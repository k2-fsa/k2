#include "hip/hip_runtime.h"
/**
 * @brief python wrappers for FSA.
 *
 * @copyright
 * Copyright (c)  2020  Mobvoi Inc.        (authors: Fangjun Kuang)
 *                      Xiaomi Corporation (authors: Haowen Qiu)
 *                      Guoguo Chen
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <limits>
#include <memory>
#include <string>
#include <utility>
#include <vector>

#include "k2/csrc/array.h"
#include "k2/csrc/fsa.h"
#include "k2/csrc/fsa_utils.h"
#include "k2/csrc/host_shim.h"
#include "k2/python/csrc/torch/fsa.h"
#include "k2/python/csrc/torch/torch_util.h"
#include "torch/extension.h"

namespace k2 {

static void PybindFsaBasicProperties(py::module &m) {
  m.def("fsa_properties_as_str", &FsaPropertiesAsString, py::arg("properties"));

  m.def("get_fsa_basic_properties", &GetFsaBasicProperties, py::arg("fsa"));

  m.def(
      "get_fsa_vec_basic_properties",
      [](FsaVec &fsa_vec) -> int32_t {
        int32_t tot_properties;
        Array1<int32_t> properties;
        GetFsaVecBasicProperties(fsa_vec, &properties, &tot_properties);
        return tot_properties;
      },
      py::arg("fsa_vec"));
  // We don't wrap the flag values from C++ to Python, we just reproduce in
  // Python directly.
}

static void PybindFsaUtil(py::module &m) {
  // TODO(fangjun): add docstring in Python describing
  // the format of the input tensor when it is a FsaVec.
  m.def(
      "fsa_from_tensor",
      [](torch::Tensor tensor) -> FsaOrVec {
        auto k2_tensor = FromTensor(tensor, TensorTag{});
        bool error = true;
        Fsa fsa;
        if (tensor.dim() == 2)
          fsa = FsaFromTensor(k2_tensor, &error);
        else if (tensor.dim() == 1)
          fsa = FsaVecFromTensor(k2_tensor, &error);
        else
          K2_LOG(FATAL)
              << "Expect dim: 2 (a single FSA) or 1 (a vector of FSAs). "
              << "Given: " << tensor.dim();

        K2_CHECK(!error);
        return fsa;
      },
      py::arg("tensor"));

  m.def(
      "fsa_to_tensor",
      [](const FsaOrVec &fsa) -> torch::Tensor {
        if (fsa.NumAxes() == 2) {
          Tensor tensor = FsaToTensor(fsa);
          return ToTensor(tensor);
        } else if (fsa.NumAxes() == 3) {
          Tensor tensor = FsaVecToTensor(fsa);
          return ToTensor(tensor);
        } else {
          K2_LOG(FATAL) << "Unsupported num_axes: " << fsa.NumAxes();
          return {};
        }
      },
      py::arg("fsa"));

  m.def(
      "fsa_to_str",
      [](Fsa &fsa, bool openfst = false,
         torch::optional<torch::Tensor> aux_labels =
             torch::nullopt) -> std::string {
        Array1<int32_t> array;
        if (aux_labels.has_value())
          array = FromTensor<int32_t>(aux_labels.value());
        return FsaToString(fsa, openfst, aux_labels ? &array : nullptr);
      },
      py::arg("fsa"), py::arg("openfst") = false,
      py::arg("aux_labels") = py::none());

  m.def(
      "fsa_from_str",
      [](const std::string &s, int num_aux_labels = 0, bool openfst = false)
          -> std::pair<Fsa, torch::optional<torch::Tensor>> {
        Array2<int32_t> aux_labels;
        Fsa fsa = FsaFromString(s, openfst, num_aux_labels,
                                &aux_labels);
        torch::optional<torch::Tensor> tensor;
        if (num_aux_labels != 0) tensor = ToTensor(aux_labels);
        return std::make_pair(fsa, tensor);
      },
      py::arg("s"), py::arg("num_aux_labels") = 0, py::arg("openfst") = false,
      "It returns a tuple with two elements. Element 0 is the FSA; element 1 "
      "is a 2-D tensor of dtype torch.int32 and shape "
      "(num_aux_labels, num_arcs) if num_aux_labels > 0; otherwise None.");

  // the following methods are for debugging only
  m.def("fsa_to_fsa_vec", &FsaToFsaVec, py::arg("fsa"));

  m.def("get_fsa_vec_element", &GetFsaVecElement, py::arg("vec"), py::arg("i"));

  m.def(
      "create_fsa_vec",
      [](std::vector<Fsa *> &fsas) -> FsaVec {
        return CreateFsaVec(fsas.size(), fsas.data());
      },
      py::arg("fsas"));

  // returns Ragged<int32_t>
  m.def("get_state_batches", &GetStateBatches, py::arg("fsas"),
        py::arg("transpose") = true);

  m.def(
      "get_dest_states",
      [](FsaVec &fsas, bool as_idx01) -> torch::Tensor {
        Array1<int32_t> ans = GetDestStates(fsas, as_idx01);
        return ToTensor(ans);
      },
      py::arg("fsas"), py::arg("as_idx01"));

  m.def(
      "get_incoming_arcs",
      [](FsaVec &fsas, torch::Tensor dest_states) -> Ragged<int32_t> {
        Array1<int32_t> dest_states_array = FromTensor<int32_t>(dest_states);
        return GetIncomingArcs(fsas, dest_states_array);
      },
      py::arg("fsas"), py::arg("dest_states"));

  // returns Ragged<int32_t>
  m.def("get_entering_arc_index_batches", &GetEnteringArcIndexBatches,
        py::arg("fsas"), py::arg("incoming_arcs"), py::arg("state_batches"));

  // returns Ragged<int32_t>
  m.def("get_leaving_arc_index_batches", &GetLeavingArcIndexBatches,
        py::arg("fsas"), py::arg("state_batches"));

  m.def(
      "is_rand_equivalent",
      [](FsaOrVec &a, FsaOrVec &b, bool log_semiring,
         float beam = k2host::kFloatInfinity,
         bool treat_epsilons_specially = true, float delta = 1e-6,
         int32_t npath = 100) -> bool {
        // if we pass npath as type `std::size_t` here, pybind11 will
        // report warning `pointless comparison of unsigned integer
        // with zero` when instantiating this binding (I guess it's
        // related to pybind11's implementation), so we here pass
        // npath as type int32_t and cast it to std::size_t. Anyway,
        // it's safe to do the cast here.
        return IsRandEquivalent(a, b, log_semiring, beam,
                                treat_epsilons_specially, delta,
                                static_cast<std::size_t>(npath));
      },
      py::arg("a"), py::arg("b"), py::arg("log_semiring"),
      py::arg("beam") = k2host::kFloatInfinity,
      py::arg("treat_epsilons_specially") = true, py::arg("delta") = 1e-6,
      py::arg("npath") = 100);
}

template <typename T>
static void PybindGetForwardScores(py::module &m, const char *name) {
  // Return a std::pair
  //   - forward_scores, a torch::Tensor of dtype torch.float32 or torch.float64
  //   (depending on T) containing the scores
  //
  //   - entering_arcs (optional)
  //     - if log_semiring is true, it is None
  //     - else it is a torch::Tensor of dtype torch.int32
  m.def(
      name,
      [](FsaVec &fsas, Ragged<int32_t> &state_batches,
         Ragged<int32_t> &entering_arc_batches, bool log_semiring)
          -> std::pair<torch::Tensor, torch::optional<torch::Tensor>> {
        Array1<int32_t> entering_arcs;
        Array1<T> scores = GetForwardScores<T>(
            fsas, state_batches, entering_arc_batches, log_semiring,
            log_semiring ? nullptr : &entering_arcs);

        torch::optional<torch::Tensor> entering_arcs_tensor;
        if (!log_semiring) entering_arcs_tensor = ToTensor(entering_arcs);

        return std::make_pair(ToTensor(scores), entering_arcs_tensor);
      },
      py::arg("fsas"), py::arg("state_batches"),
      py::arg("entering_arc_batches"), py::arg("log_semiring"));
}

template <typename T>
static void PybindBackpropGetForwardScores(py::module &m, const char *name) {
  // entering_arcs is not empty only if log_semiring is false
  m.def(
      name,
      [](FsaVec &fsas, Ragged<int32_t> &state_batches,
         Ragged<int32_t> &leaving_arc_batches, bool log_semiring,
         torch::optional<torch::Tensor> entering_arcs,
         torch::Tensor forward_scores,
         torch::Tensor forward_scores_deriv) -> torch::Tensor {
        Array1<T> forward_scores_array = FromTensor<T>(forward_scores);
        Array1<T> forward_scores_deriv_array =
            FromTensor<T>(forward_scores_deriv);
        Array1<int32_t> entering_arcs_array;
        const Array1<int32_t> *p_entering_arcs = nullptr;

        if (!log_semiring) {
          K2_CHECK(entering_arcs.has_value())
              << "You have to provide entering_arcs for tropical semiring";
          entering_arcs_array = FromTensor<int32_t>(*entering_arcs);
          p_entering_arcs = &entering_arcs_array;
        }
        Array1<T> ans = BackpropGetForwardScores<T>(
            fsas, state_batches, leaving_arc_batches, log_semiring,
            p_entering_arcs, forward_scores_array, forward_scores_deriv_array);

        return ToTensor(ans);
      },
      py::arg("fsas"), py::arg("state_batches"), py::arg("leaving_arc_batches"),
      py::arg("log_semiring"), py::arg("entering_arcs"),
      py::arg("forward_scores"), py::arg("forward_scores_deriv"));
}

template <typename T>
static void PybindGetBackwardScores(py::module &m, const char *name) {
  m.def(
      name,
      [](FsaVec &fsas, Ragged<int32_t> &state_batches,
         Ragged<int32_t> &leaving_arc_batches,
         bool log_semiring = true) -> torch::Tensor {
        Array1<T> ans = GetBackwardScores<T>(fsas, state_batches,
                                             leaving_arc_batches, log_semiring);

        return ToTensor(ans);
      },
      py::arg("fsas"), py::arg("state_batches"), py::arg("leaving_arc_batches"),
      py::arg("log_semiring") = true);
}

template <typename T>
static void PybindBackpropGetBackwardScores(py::module &m, const char *name) {
  m.def(
      name,
      [](FsaVec &fsas, Ragged<int32_t> &state_batches,
         Ragged<int32_t> &entering_arc_batches, bool log_semiring,
         torch::Tensor backward_scores,
         torch::Tensor backward_scores_deriv) -> torch::Tensor {
        Array1<T> backward_scores_array = FromTensor<T>(backward_scores);
        Array1<T> backward_scores_deriv_array =
            FromTensor<T>(backward_scores_deriv);

        Array1<T> ans = BackpropGetBackwardScores<T>(
            fsas, state_batches, entering_arc_batches, log_semiring,
            backward_scores_array, backward_scores_deriv_array);

        return ToTensor(ans);
      },
      py::arg("fsas"), py::arg("state_batches"),
      py::arg("entering_arc_batches"), py::arg("log_semiring"),
      py::arg("backward_scores"), py::arg("backward_scores_deriv"));
}

template <typename T>
static void PybindGetTotScores(py::module &m, const char *name) {
  m.def(
      name,
      [](FsaVec &fsas, torch::Tensor forward_scores) -> torch::Tensor {
        Array1<T> forward_scores_array = FromTensor<T>(forward_scores);
        Array1<T> tot_scores = GetTotScores(fsas, forward_scores_array);
        return ToTensor(tot_scores);
      },
      py::arg("fsas"), py::arg("forward_scores"));
}

static void PybindDenseFsaVec(py::module &m) {
  using PyClass = DenseFsaVec;
  py::class_<PyClass> pyclass(m, "DenseFsaVec");
  // We do not need to access its members in Python

  // TODO(fangjun): add docstring for this funciton
  pyclass.def(
      py::init([](torch::Tensor scores,
                  torch::Tensor row_splits) -> std::unique_ptr<DenseFsaVec> {
        // remove the contiguous check once the following comment
        // https://github.com/k2-fsa/k2/commit/60b8e97b1838033b45b83cc88a58ec91912ce91e#r43174753
        // is resolved.
        K2_CHECK(scores.is_contiguous());
        Array1<int32_t> row_splits_array = FromTensor<int32_t>(row_splits);

        RaggedShape shape = RaggedShape2(&row_splits_array, nullptr, -1);
        Array2<float> scores_array = FromTensor<float>(scores, Array2Tag{});

        return std::make_unique<DenseFsaVec>(shape, scores_array);
      }),
      py::arg("scores"), py::arg("row_splits"));

  pyclass.def(
      "dim0", [](PyClass &self) -> int32_t { return self.shape.Dim0(); },
      "Returns number of supervisions contained in it");

  pyclass.def("shape", [](PyClass &self) -> RaggedShape { return self.shape; });

  pyclass.def("scores_dim1",
              [](PyClass &self) -> int32_t { return self.scores.Dim1(); });

  // the `to_str` method is for debugging only
  pyclass.def("to_str", [](PyClass &self) -> std::string {
    std::ostringstream os;
    os << "num_axes: " << self.shape.NumAxes() << '\n';
    os << "device_type: " << self.shape.Context()->GetDeviceType() << '\n';
    os << "row_splits1: " << self.shape.RowSplits(1) << '\n';
    os << "row_ids1: " << self.shape.RowIds(1) << '\n';
    os << "scores:" << self.scores << '\n';
    return os.str();
  });

  pyclass.def(
      "to",
      [](const PyClass &self, py::object device) -> PyClass {
        return To(self, device);
      },
      py::arg("device"));
}

static void PybindConvertDenseToFsaVec(py::module &m) {
  m.def(
      "convert_dense_to_fsa_vec",
      [](DenseFsaVec &dense_fsa_vec) -> FsaVec {
        return ConvertDenseToFsaVec(dense_fsa_vec);
      },
      py::arg("dense_fsa_vec"));
}

template <typename T>
static void PybindGetArcPost(py::module &m, const char *name) {
  m.def(
      name,
      [](FsaVec &fsas, torch::Tensor forward_scores,
         torch::Tensor backward_scores) -> torch::Tensor {
        Array1<T> forward_scores_array = FromTensor<T>(forward_scores);
        Array1<T> backward_scores_array = FromTensor<T>(backward_scores);
        Array1<T> arc_post =
            GetArcPost<T>(fsas, forward_scores_array, backward_scores_array);
        return ToTensor(arc_post);
      },
      py::arg("fsas"), py::arg("forward_scores"), py::arg("backward_scores"));
}

template <typename T>
static void PybindBackpropGetArcPost(py::module &m, const char *name) {
  // return a pair of tensors:
  //   - forward_scores_deriv
  //   - backward_scores_deriv
  m.def(
      name,
      [](FsaVec &fsas, Ragged<int32_t> &incoming_arcs,
         torch::Tensor arc_post_deriv)
          -> std::pair<torch::Tensor, torch::Tensor> {
        Array1<T> arc_post_deriv_array = FromTensor<T>(arc_post_deriv);
        Array1<T> forward_scores_deriv;
        Array1<T> backward_scores_deriv;

        BackpropGetArcPost<T>(fsas, incoming_arcs, arc_post_deriv_array,
                              &forward_scores_deriv, &backward_scores_deriv);
        return std::make_pair(ToTensor(forward_scores_deriv),
                              ToTensor(backward_scores_deriv));
      },
      py::arg("fsas"), py::arg("incoming_arcs"), py::arg("arc_post_deriv"));
}

/* Compute the backward propagation of GetTotScores in tropical semiring.

   @param [in] fsa_vec  The input FsaVec for computing `GetTotScores`
                        and `ShortestPath`.
   @param [in] best_path_arc_indexes The arc indexes that contribute to
                                     the total scores. It is the return value
                                     of `ShortestPath`.
   @param [in] tot_scores_grad  The gradient of total scores.
   @return It returns the gradient of scores of all arcs.
 */
template <typename T>
static torch::Tensor GetTotScoresTropicalBackward(
    FsaVec &fsas, const Ragged<int32_t> &best_path_arc_indexes,
    torch::Tensor tot_scores_grad) {
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  K2_CHECK_EQ(best_path_arc_indexes.NumAxes(), 2);

  int32_t num_fsas = fsas.Dim0();
  K2_CHECK_EQ(best_path_arc_indexes.Dim0(), num_fsas);
  K2_CHECK_EQ(tot_scores_grad.sizes()[0], static_cast<int64_t>(num_fsas));
  K2_CHECK_EQ(tot_scores_grad.dim(), 1);
  K2_CHECK_EQ(tot_scores_grad.scalar_type(), ToScalarType<T>::value);

  std::vector<int64_t> dims = {fsas.NumElements()};
  auto options = torch::TensorOptions()
                     .dtype(torch::kFloat32)
                     .device(tot_scores_grad.device());
  torch::Tensor ans_grad = torch::zeros(dims, options);  // it is contiguous
  float *ans_grad_data = ans_grad.data_ptr<float>();

  const T *tot_scores_grad_data = tot_scores_grad.data_ptr<T>();
  int64_t tot_scores_grad_stride = tot_scores_grad.strides()[0];

  const int32_t *fsas_row_ids1 = fsas.RowIds(1).Data();
  const int32_t *fsas_row_ids2 = fsas.RowIds(2).Data();
  const int32_t *best_path_arc_indexes_data =
      best_path_arc_indexes.values.Data();

  K2_EVAL(
      fsas.Context(), best_path_arc_indexes.NumElements(), lambda,
      (int32_t best_path_arc_idx012)->void {
        int32_t arc_idx012 = best_path_arc_indexes_data[best_path_arc_idx012];
        int32_t state_idx01 = fsas_row_ids2[arc_idx012];
        int32_t fsas_idx0 = fsas_row_ids1[state_idx01];
        ans_grad_data[arc_idx012] =
            tot_scores_grad_data[fsas_idx0 * tot_scores_grad_stride];
      });
  return ans_grad;
}

/* Compute the backward propagation of GetTotScores in log semiring.
 *
   @param [in] fsa_vec     The input FsaVec for computing `GetTotScores`
                           and `GetArcPost`.
   @param [in] arc_post    It is the return value of `GetArcPost`.
   @param [in] tot_scores_grad  The gradient of total scores.
   @return It returns the gradient of scores of all arcs.
 */
template <typename T>
static torch::Tensor GetTotScoresLogBackward(FsaVec &fsas,
                                             torch::Tensor arc_post,
                                             torch::Tensor tot_scores_grad) {
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  K2_CHECK_EQ(fsas.NumElements(), arc_post.numel());
  K2_CHECK(arc_post.is_contiguous())
      << "arc_post is supposed to be computed by k2 "
         "so it should be contiguous!";
  K2_CHECK_EQ(arc_post.dim(), 1);
  K2_CHECK_EQ(arc_post.scalar_type(), ToScalarType<T>::value);
  K2_CHECK_EQ(tot_scores_grad.dim(), 1);
  K2_CHECK_EQ(tot_scores_grad.sizes()[0], static_cast<int64_t>(fsas.Dim0()));
  K2_CHECK_EQ(tot_scores_grad.scalar_type(), ToScalarType<T>::value);

  std::vector<int64_t> dims = {fsas.NumElements()};
  auto options = torch::TensorOptions()
                     .dtype(torch::kFloat32)
                     .device(tot_scores_grad.device());
  torch::Tensor ans_grad = torch::empty(dims, options);  // it is contiguous
  float *ans_grad_data = ans_grad.data_ptr<float>();
  const T *tot_scores_grad_data = tot_scores_grad.data_ptr<T>();
  int64_t tot_scores_grad_stride = tot_scores_grad.strides()[0];

  const int32_t *fsas_row_ids1 = fsas.RowIds(1).Data();
  const int32_t *fsas_row_ids2 = fsas.RowIds(2).Data();
  const T *arc_post_data = arc_post.data_ptr<T>();

  if (std::is_same<T, float>::value) {
    K2_EVAL(
        fsas.Context(), fsas.NumElements(), lambda, (int32_t arc_idx012)->void {
          int32_t state_idx01 = fsas_row_ids2[arc_idx012];
          int32_t fsa_idx0 = fsas_row_ids1[state_idx01];
          ans_grad_data[arc_idx012] =
              expf(arc_post_data[arc_idx012]) *
              tot_scores_grad_data[fsa_idx0 * tot_scores_grad_stride];
        });
  } else {
    K2_EVAL(
        fsas.Context(), fsas.NumElements(), lambda, (int32_t arc_idx012)->void {
          int32_t state_idx01 = fsas_row_ids2[arc_idx012];
          int32_t fsa_idx0 = fsas_row_ids1[state_idx01];
          ans_grad_data[arc_idx012] =
              exp(arc_post_data[arc_idx012]) *
              tot_scores_grad_data[fsa_idx0 * tot_scores_grad_stride];
        });
  }
  return ans_grad;
}

template <typename T>
static void PybindGetTotScoresTropicalBackward(py::module &m,
                                               const char *name) {
  m.def(name, &GetTotScoresTropicalBackward<T>, py::arg("fsas"),
        py::arg("best_path_arc_indexes"), py::arg("tot_scores_grad"));
}

template <typename T>
static void PybindGetTotScoresLogBackward(py::module &m, const char *name) {
  m.def(name, &GetTotScoresLogBackward<T>, py::arg("fsas"), py::arg("arc_post"),
        py::arg("tot_scores_grad"));
}

template <typename T>
static void PybindGetArcCdf(py::module &m, const char *name) {
  m.def(
      name,
      [](FsaOrVec &fsas, torch::Tensor arc_post) -> torch::Tensor {
        Array1<T> arc_post_array = FromTensor<T>(arc_post);
        Array1<T> ans = GetArcCdf(fsas, arc_post_array);
        return ToTensor(ans);
      },
      py::arg("fsas"), py::arg("arc_post"));
}

template <typename T>
static void PybindRandomPaths(py::module &m, const char *name) {
  m.def(
      name,
      [](FsaVec &fsas, torch::Tensor arc_cdf, int32_t num_paths,
         torch::Tensor tot_scores,
         Ragged<int32_t> &state_batches) -> Ragged<int32_t> {
        Array1<T> arc_cdf_array = FromTensor<T>(arc_cdf);
        Array1<T> tot_scores_array = FromTensor<T>(tot_scores);

        Ragged<int32_t> ans = RandomPaths(fsas, arc_cdf_array, num_paths,
                                          tot_scores_array, state_batches);
        return ans;
      },
      py::arg("fsas"), py::arg("arc_cdf"), py::arg("num_paths"),
      py::arg("tot_scores"), py::arg("state_batches"));
}

template <typename T>
static void PybindPruneOnArcPost(py::module &m, const char *name) {
  m.def(
      name,
      [](FsaVec &fsas, torch::Tensor arc_post, T threshold_prob,
         bool need_arc_map =
             true) -> std::pair<FsaVec, torch::optional<torch::Tensor>> {
        Array1<T> arc_post_array = FromTensor<T>(arc_post);
        Array1<int32_t> arc_map;
        FsaVec ans = PruneOnArcPost(fsas, arc_post_array, threshold_prob,
                                    need_arc_map ? &arc_map : nullptr);
        torch::optional<torch::Tensor> arc_map_tensor;
        if (need_arc_map) arc_map_tensor = ToTensor(arc_map);
        return std::make_pair(ans, arc_map_tensor);
      },
      py::arg("fsas"), py::arg("arc_post"), py::arg("threshold_prob"),
      py::arg("need_arc_map") = true);
}

}  // namespace k2

void PybindFsa(py::module &m) {
  k2::PybindFsaUtil(m);
  k2::PybindDenseFsaVec(m);
  k2::PybindConvertDenseToFsaVec(m);
  k2::PybindFsaBasicProperties(m);
  k2::PybindGetForwardScores<float>(m, "get_forward_scores_float");
  k2::PybindGetForwardScores<double>(m, "get_forward_scores_double");
  k2::PybindBackpropGetForwardScores<float>(
      m, "backprop_get_forward_scores_float");
  k2::PybindBackpropGetForwardScores<double>(
      m, "backprop_get_forward_scores_double");
  k2::PybindGetBackwardScores<float>(m, "get_backward_scores_float");
  k2::PybindGetBackwardScores<double>(m, "get_backward_scores_double");
  k2::PybindBackpropGetBackwardScores<float>(
      m, "backprop_get_backward_scores_float");
  k2::PybindBackpropGetBackwardScores<double>(
      m, "backprop_get_backward_scores_double");
  k2::PybindGetTotScores<float>(m, "get_tot_scores_float");
  k2::PybindGetTotScores<double>(m, "get_tot_scores_double");
  k2::PybindGetArcPost<float>(m, "get_arc_post_float");
  k2::PybindGetArcPost<double>(m, "get_arc_post_double");
  k2::PybindBackpropGetArcPost<float>(m, "backprop_get_arc_post_float");
  k2::PybindBackpropGetArcPost<double>(m, "backprop_get_arc_post_double");
  k2::PybindGetTotScoresTropicalBackward<float>(
      m, "get_tot_scores_float_tropical_backward");
  k2::PybindGetTotScoresTropicalBackward<double>(
      m, "get_tot_scores_double_tropical_backward");
  k2::PybindGetTotScoresLogBackward<float>(m,
                                           "get_tot_scores_float_log_backward");
  k2::PybindGetTotScoresLogBackward<double>(
      m, "get_tot_scores_double_log_backward");

  k2::PybindGetArcCdf<float>(m, "get_arc_cdf_float");
  k2::PybindGetArcCdf<double>(m, "get_arc_cdf_double");

  k2::PybindRandomPaths<float>(m, "random_paths_float");
  k2::PybindRandomPaths<double>(m, "random_paths_double");
  k2::PybindPruneOnArcPost<float>(m, "prune_on_arc_post_float");
  k2::PybindPruneOnArcPost<double>(m, "prune_on_arc_post_double");
}
