/**
 * @brief Index select for k2.
 *
 * Unlike torch.index_select, when an entry is -1, it sets
 * the destination entry to 0.
 *
 * @copyright
 * Copyright (c)  2020  Mobvoi Inc.        (authors: Fangjun Kuang)
 *                      Xiaomi Corp.       (author: Daniel Povey, Haowen Qiu)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */
#include <vector>

#include "k2/csrc/context.h"
#include "k2/csrc/device_guard.h"
#include "k2/csrc/macros.h"
#include "k2/csrc/nvtx.h"
#include "k2/csrc/ragged.h"
#include "k2/csrc/ragged_ops.h"
#include "k2/csrc/tensor_ops.h"
#include "k2/python/csrc/torch/index_select.h"
#include "k2/python/csrc/torch/torch_util.h"
#include "torch/extension.h"

namespace k2 {

/* Returns a 1-D tensor which indexes the src tensor using entries
   from `index`.

   @param  [in]  src    A 1-D tensor.
   @param  [in]  index  A 1-D tensor with dtype torch.int32.
                        It has to satisfy:
                            -1 <= index[i] < src.numel()
                            for i in [0, index.numel())
                        CAUTION: We require that index.is_contiguous() is true.
   @param [in] default_value  The value for ans[i] when index[i] is -1.
   @return
      Returns a 1-D contiguous tensor such that:
          ans[i] = src[index[i]] if index[i] > 0
          ans[i] = default_value if index[i] is -1
 */
template <typename T>
static torch::Tensor IndexSelect1D(torch::Tensor src, torch::Tensor index,
                                   T default_value) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(src.dim(), 1) << "Expected dim: 1. Given: " << src.dim();
  K2_CHECK_EQ(src.scalar_type(), ToScalarType<T>::value);

  K2_CHECK_EQ(index.dim(), 1);
  K2_CHECK_EQ(index.scalar_type(), ToScalarType<int32_t>::value);
  K2_CHECK(index.is_contiguous());
  K2_CHECK_EQ(src.device(), index.device());

  bool allow_minus_one = true;
  Array1<int32_t> index_array = FromTorch<int32_t>(index);
  if (src.is_contiguous()) {
    Array1<T> src_array = FromTorch<T>(src);
    Array1<T> ans_array =
        Index(src_array, index_array, allow_minus_one, default_value);
    return ToTorch(ans_array);
  }

  Tensor tensor = FromTorch(src, TensorTag{});
  Tensor ans = Index(tensor, index_array, allow_minus_one, default_value);
  return ToTorch(ans);
}

/* Returns a 2-D tensor which indexes the src tensor using entries
   from `index`.

   @param  [in]  src    A 2-D tensor. If it is non-contiguous, then it
                        has to satisfy src.strides()[1] == 1.

   @param  [in]  index  A 1-D tensor with dtype torch.int32.
                        It has to satisfy:
                            -1 <= index[i] < src.shape()[0]
                            for i in [0, index.numel())
                        CAUTION: We require that index.is_contiguous() is true.
   @return
      Returns a 2-D contiguous tensor such that:
          ans[i] = src[index[i]] if index[i] > 0
          ans[i] = zero tensor whose numel() is src.shape()[1],
                   if index[i] is -1
 */
template <typename T>
static torch::Tensor IndexSelect2D(torch::Tensor src, torch::Tensor index) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(src.dim(), 2) << "Expected dim: 2. Given: " << src.dim();
  K2_CHECK_EQ(src.scalar_type(), ToScalarType<T>::value);

  K2_CHECK_EQ(index.dim(), 1);
  K2_CHECK_EQ(index.scalar_type(), ToScalarType<int32_t>::value);
  K2_CHECK(index.is_contiguous());
  K2_CHECK_EQ(src.device(), index.device());

  Array2<T> src_array = FromTorch<T>(src, Array2Tag{});
  Array1<int32_t> index_array = FromTorch<int32_t>(index);
  bool allow_minus_one = true;
  Array2<T> ans_array = IndexRows(src_array, index_array, allow_minus_one);

  return ToTorch(ans_array);
}

static torch::Tensor IndexSelectWrapper(torch::Tensor src, torch::Tensor index,
                                        double default_value = 0) {
  NVTX_RANGE(K2_FUNC);
  DeviceGuard guard(GetContext(src));
  auto scalar_type = src.scalar_type();
  if (src.dim() == 1) {
    switch (scalar_type) {
      case ToScalarType<int32_t>::value: {
        int32_t i = static_cast<int32_t>(default_value);
        K2_CHECK_EQ(static_cast<double>(i), default_value);
        return IndexSelect1D<int32_t>(src, index, i);
      }
      case ToScalarType<int64_t>::value: {
        int64_t i = static_cast<int64_t>(default_value);
        K2_CHECK_EQ(static_cast<double>(i), default_value);
        return IndexSelect1D<int64_t>(src, index, i);
      }
      case ToScalarType<float>::value:
        return IndexSelect1D<float>(src, index, default_value);
      case ToScalarType<double>::value:
        return IndexSelect1D<double>(src, index, default_value);
      default:
        K2_LOG(FATAL) << "Unsupported scalar type: " << scalar_type;
        return {};
    }
  } else if (src.dim() == 2) {
    switch (scalar_type) {
      case ToScalarType<int32_t>::value:
        return IndexSelect2D<int32_t>(src, index);
      case ToScalarType<int64_t>::value:
        return IndexSelect2D<int64_t>(src, index);
      case ToScalarType<float>::value:
        return IndexSelect2D<float>(src, index);
      case ToScalarType<double>::value:
        return IndexSelect2D<double>(src, index);
      default:
        K2_LOG(FATAL) << "Unsupported scalar type: " << scalar_type;
        return {};
    }
  } else {
    K2_LOG(FATAL) << "Unsupported dim: " << src.dim()
                  << ".\nIt supports only 1-D and 2-D tensors.";
    return {};
  }
}

/*
  Returns a 1-D Tensor that is a result of indexing 1-D `src` with Ragged array
  `indexes` whose NumAxes() is 2. ans.numel() will equal to indexes.Dim0() as we
  suppose there is at most one non-zero element in `src` for any indexes
  sub-list in `indexes`.

     @param [in] src  Source tensor, to be indexed.
     @param [in] indexes   Indexes to use whose NumAxes() == 2, for any
                      sub-list `i` in `indexes`, we suppose there is at most
                      one non-zero values in `src` and we'll set ans[i]
                      with that non-zero value; if all values for
                      sub-list `i` is zero or the sub-list is empty, we just
                      set ans[i] == 0.
     @return   Returns a Tensor with the same dtype as `src` and shape
                     (indexes.Dim0()), i.e. a 1-D tensor with numel() equal
                     to `indexes.Dim0()`.
                     Noted the ans would be contiguous even though `src`
                     is not contiguous.
 */
template <typename T>
static torch::Tensor SimpleRaggedIndexSelect1D(torch::Tensor src,
                                               Ragged<int32_t> &indexes) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(src.dim(), 1) << "Expected dim: 1. Given: " << src.dim();
  K2_CHECK_EQ(src.scalar_type(), ToScalarType<T>::value);
  K2_CHECK_EQ(indexes.NumAxes(), 2);
  ContextPtr context = GetContext(src);
  K2_CHECK(context->IsCompatible(*indexes.Context()));

  Tensor tensor = FromTorch(src, TensorTag{});
  Tensor ans = SimpleRaggedIndexSelect1D(tensor, indexes);
  return ToTorch(ans);
}

static torch::Tensor SimpleRaggedIndexSelectWrapper(torch::Tensor src,
                                                    Ragged<int32_t> &indexes) {
  DeviceGuard guard(GetContext(src));
  auto scalar_type = src.scalar_type();
  if (src.dim() == 1) {
    switch (scalar_type) {
      case ToScalarType<int32_t>::value:
        return SimpleRaggedIndexSelect1D<int32_t>(src, indexes);
      case ToScalarType<float>::value:
        return SimpleRaggedIndexSelect1D<float>(src, indexes);
      default:
        K2_LOG(FATAL) << "Unsupported scalar type: " << scalar_type;
        return {};
    }
  } else {
    K2_LOG(FATAL) << "Unsupported dim: " << src.dim()
                  << ". It supports only 1-D tensors for now";
    return {};
  }
}

static void IndexSelect(py::module &m) {
  m.def("index_select", &IndexSelectWrapper, py::arg("src"), py::arg("index"),
        py::arg("default_value") = 0,
        R"(
      Args:
        src:
          It can be either a 1-D or a 2-D tensor. Supported dtypes are:
          `torch.int32`, `torch.int64`, `torch.float32`, and `torch.float64`.
        index:
          It has to be a 1-D **contiguous** tensor with dtype `torch.int32`.
          Must satisfy `-1 <= index[i] < src.shape[0]`.
        default_value:
          It is the default value for ans[i] if index[i] is -1.
          Used only when `src` is a 1-D tensor.
      Returns:
        Return a tensor:
          - `ans.ndim == src.ndim`
          - `ans.shape[0] == index.shape[0]`
          - If `ans.ndim == 2`, then `ans.shape[1] == src.shape[1]`
          - `ans[i] = src[index[i]]` if `index[i] != -1`.
          - `ans[i] = default_value` if `index[i] == -1`
      )");
  m.def("simple_ragged_index_select", &SimpleRaggedIndexSelectWrapper,
        py::arg("src"), py::arg("indexes"));
}

}  // namespace k2

void PybindIndexSelect(py::module &m) { k2::IndexSelect(m); }
