/**
 * @copyright
 * Copyright      2020  Mobvoi Inc.        (authors: Fangjun Kuang)
 *                2021  Xiaomi Corp.       (author: Daniel Povey,
 *                                                  Haowen Qiu,
 *                                                  Wei Kang)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "k2/csrc/context.h"
#include "k2/csrc/device_guard.h"
#include "k2/csrc/macros.h"
#include "k2/csrc/nvtx.h"
#include "k2/csrc/ragged.h"
#include "k2/csrc/ragged_ops.h"
#include "k2/csrc/tensor_ops.h"
#include "k2/python/csrc/torch/v2/ops.h"

namespace k2 {

void IndexAdd(torch::Tensor index, torch::Tensor value, torch::Tensor *in_out) {
  NVTX_RANGE(K2_FUNC);
  DeviceGuard guard(GetContext(index));

  Array1<int32_t> indexes = FromTorch<int32_t>(index);
  Tensor src = FromTorch(value, TensorTag{});
  Tensor dest = FromTorch(*in_out, TensorTag{});
  IndexAdd(src, indexes, true, &dest);
}

/* Returns a 1-D tensor which indexes the src tensor using entries
   from `index`.

   @param  [in]  src    A 1-D tensor.
   @param  [in]  index  A 1-D tensor with dtype torch.int32.
                        It has to satisfy:
                            -1 <= index[i] < src.numel()
                            for i in [0, index.numel())
                        CAUTION: We require that index.is_contiguous()
                                 is true.
   @param [in] default_value  The value for ans[i] when index[i] is -1.
   @return
      Returns a 1-D contiguous tensor such that:
          ans[i] = src[index[i]] if index[i] > 0
          ans[i] = default_value if index[i] is -1
 */
template <typename T>
torch::Tensor IndexSelect1D(torch::Tensor src, torch::Tensor index,
                            T default_value) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(src.dim(), 1) << "Expected dim: 1. Given: " << src.dim();
  K2_CHECK_EQ(src.scalar_type(), ToScalarType<T>::value)
      << "Expected equal type"
      << " Given : " << src.scalar_type() << ", " << ToScalarType<T>::value;

  K2_CHECK_EQ(index.dim(), 1)
      << "Expected index dim: 1. Given : " << index.dim();
  K2_CHECK_EQ(index.scalar_type(), ToScalarType<int32_t>::value)
      << "Expected type int32_t Given : " << index.scalar_type();
  K2_CHECK(index.is_contiguous()) << "Expected contiguous";
  K2_CHECK_EQ(src.device(), index.device())
      << "Expected in the same device"
      << " Given : " << src.device() << ", " << index.device();

  bool allow_minus_one = true;
  Array1<int32_t> index_array = FromTorch<int32_t>(index);
  // If index_array.Dim() equals to zero, the `Index` below would produce an
  // ans with `ans.Data()` be a nullptr, which will cause crash when calling
  // `torch::from_blob`. Just return an empty tensor here.
  // If src is an empty tensor, we should return an empty torch.
  if (index_array.Dim() == 0 || src.numel() == 0)
    return torch::empty({0}, src.options());
  if (src.is_contiguous()) {
    Array1<T> src_array = FromTorch<T>(src);
    Array1<T> ans_array =
        Index(src_array, index_array, allow_minus_one, default_value);
    return ToTorch(ans_array);
  }
  Tensor tensor = FromTorch(src, TensorTag{});
  Tensor ans = Index(tensor, index_array, allow_minus_one, default_value);
  return ToTorch(ans);
}

/* Returns a 2-D tensor which indexes the src tensor using entries
   from `index`.

   @param  [in]  src    A 2-D tensor. If it is non-contiguous, then it
                        has to satisfy src.strides()[1] == 1.

   @param  [in]  index  A 1-D tensor with dtype torch.int32.
                        It has to satisfy:
                            -1 <= index[i] < src.shape()[0]
                            for i in [0, index.numel())
                        CAUTION: We require that index.is_contiguous()
                                 is true.
   @return
      Returns a 2-D contiguous tensor such that:
          ans[i] = src[index[i]] if index[i] > 0
          ans[i] = zero tensor whose numel() is src.shape()[1],
                   if index[i] is -1
 */
template <typename T>
torch::Tensor IndexSelect2D(torch::Tensor src, torch::Tensor index) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(src.dim(), 2) << "Expected dim: 2. Given: " << src.dim();
  K2_CHECK_EQ(src.scalar_type(), ToScalarType<T>::value);

  K2_CHECK_EQ(index.dim(), 1);
  K2_CHECK_EQ(index.scalar_type(), ToScalarType<int32_t>::value);
  K2_CHECK(index.is_contiguous());
  K2_CHECK_EQ(src.device(), index.device());

  Array2<T> src_array = FromTorch<T>(src, Array2Tag{});
  Array1<int32_t> index_array = FromTorch<int32_t>(index);
  // If index_array.Dim() equals to zero, the `IndexRows` below would produce
  // an ans with `ans.Data()` be a nullptr, which will cause crash when
  // calling `torch::from_blob`. Just return an empty tensor here.
  // If src is an empty tensor, we should return an empty torch.
  if (index_array.Dim() == 0 || src.sizes()[0] == 0)
    return torch::empty({0, src.sizes()[1]}, src.options());
  bool allow_minus_one = true;
  Array2<T> ans_array = IndexRows(src_array, index_array, allow_minus_one);

  return ToTorch(ans_array);
}

torch::Tensor IndexSelect(torch::Tensor src, torch::Tensor index,
                          double default_value /*= 0*/) {
  NVTX_RANGE(K2_FUNC);
  DeviceGuard guard(GetContext(src));
  auto scalar_type = src.scalar_type();
  if (src.dim() == 1) {
    switch (scalar_type) {
      case ToScalarType<int32_t>::value: {
        int32_t i = static_cast<int32_t>(default_value);
        K2_CHECK_EQ(static_cast<double>(i), default_value);
        return IndexSelect1D<int32_t>(src, index, i);
      }
      case ToScalarType<int64_t>::value: {
        int64_t i = static_cast<int64_t>(default_value);
        K2_CHECK_EQ(static_cast<double>(i), default_value);
        return IndexSelect1D<int64_t>(src, index, i);
      }
      case ToScalarType<float>::value:
        return IndexSelect1D<float>(src, index, default_value);
      case ToScalarType<double>::value:
        return IndexSelect1D<double>(src, index, default_value);
      default:
        K2_LOG(FATAL) << "Unsupported scalar type: " << scalar_type;
        return {};
    }
  } else if (src.dim() == 2) {
    switch (scalar_type) {
      case ToScalarType<int32_t>::value:
        return IndexSelect2D<int32_t>(src, index);
      case ToScalarType<int64_t>::value:
        return IndexSelect2D<int64_t>(src, index);
      case ToScalarType<float>::value:
        return IndexSelect2D<float>(src, index);
      case ToScalarType<double>::value:
        return IndexSelect2D<double>(src, index);
      default:
        K2_LOG(FATAL) << "Unsupported scalar type: " << scalar_type;
        return {};
    }
  } else {
    K2_LOG(FATAL) << "Unsupported dim: " << src.dim()
                  << ".\nIt supports only 1-D and 2-D tensors.";
    return {};
  }
}

/*
  Returns a 1-D Tensor that is a result of indexing 1-D `src` with Ragged
  array `indexes` whose NumAxes() is 2. ans.numel() will equal to
  indexes.Dim0() as we suppose there is at most one non-zero element in `src`
  for any indexes sub-list in `indexes`.

     @param [in] src  Source tensor, to be indexed.
     @param [in] indexes   Indexes to use whose NumAxes() == 2, for any
                      sub-list `i` in `indexes`, we suppose there is at most
                      one non-zero values in `src` and we'll set ans[i]
                      with that non-zero value; if all values for
                      sub-list `i` is zero or the sub-list is empty, we just
                      set ans[i] == 0.
     @return   Returns a Tensor with the same dtype as `src` and shape
                     (indexes.Dim0()), i.e. a 1-D tensor with numel() equal
                     to `indexes.Dim0()`.
                     Noted the ans would be contiguous even though `src`
                     is not contiguous.
 */
template <typename T>
torch::Tensor SimpleRaggedIndexSelect1D(torch::Tensor src,
                                        Ragged<int32_t> &indexes) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(src.dim(), 1) << "Expected dim: 1. Given: " << src.dim();
  K2_CHECK_EQ(src.scalar_type(), ToScalarType<T>::value);
  K2_CHECK_EQ(indexes.NumAxes(), 2);
  ContextPtr context = GetContext(src);
  K2_CHECK(context->IsCompatible(*indexes.Context()));

  Tensor tensor = FromTorch(src, TensorTag{});
  Tensor ans = SimpleRaggedIndexSelect1D(tensor, indexes);
  return ToTorch(ans);
}

torch::Tensor SimpleRaggedIndexSelect(torch::Tensor src, RaggedAny &ragged) {
  DeviceGuard guard(GetContext(src));
  Ragged<int32_t> indexes = ragged.any.Specialize<int32_t>();
  auto scalar_type = src.scalar_type();
  if (src.dim() == 1) {
    switch (scalar_type) {
      case ToScalarType<int32_t>::value:
        return SimpleRaggedIndexSelect1D<int32_t>(src, indexes);
      case ToScalarType<float>::value:
        return SimpleRaggedIndexSelect1D<float>(src, indexes);
      default:
        K2_LOG(FATAL) << "Unsupported scalar type: " << scalar_type;
        return {};
    }
  } else {
    K2_LOG(FATAL) << "Unsupported dim: " << src.dim()
                  << ". It supports only 1-D tensors for now";
    return {};
  }
}

}  // namespace k2
