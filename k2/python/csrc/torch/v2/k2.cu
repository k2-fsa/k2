/**
 * @brief python wrapper for k2 2.0
 *
 * @copyright
 * Copyright      2021  Xiaomi Corp.  (authors: Fangjun Kuang)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "k2/python/csrc/torch/v2/any.h"
#include "k2/python/csrc/torch/v2/autograd/ragged_arc_holder.h"
#include "k2/python/csrc/torch/v2/fsa.h"
#include "k2/python/csrc/torch/v2/k2.h"
#include "k2/python/csrc/torch/v2/k2_ops.h"
#include "k2/python/csrc/torch/v2/ragged_shape.h"

namespace k2 {

void PybindV2(py::module &m) {
  py::module ragged = m.def_submodule(
      "ragged", "Sub module containing operations for ragged tensors in k2");

  PybindRaggedShape(ragged);

  m.attr("RaggedShape") = ragged.attr("RaggedShape");  // TODO: remove it

  PybindRaggedAny(ragged);
  PybindRaggedArc(ragged);

  // Add v2 submodule to avoid conflicts with original version, will remove it
  // later.
  py::module v2 = m.def_submodule(
      "v2", "Sub module containing operations for v2 ops in k2");
  PybindK2Ops(v2);

  // The following **static** variable is to resolve the following exception:
  //
  // RuntimeError: Trying to instantiate a class that isn't a registered custom
  // class: k2::RaggedArcHolder
  //
  static auto register_ragged_arc_holder =
      torch::class_<RaggedArcHolder>("MyRaggedArcHolder", "RaggedArcHolder");
}

}  // namespace k2
