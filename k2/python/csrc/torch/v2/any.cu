#include "hip/hip_runtime.h"
/**
 * @brief Wraps Ragged<Any>
 *
 * @copyright
 * Copyright      2021  Xiaomi Corp.  (authors: Fangjun Kuang)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <memory>
#include <string>
#include <utility>
#include <vector>

#include "k2/csrc/ragged.h"
#include "k2/csrc/torch_util.h"
#include "k2/python/csrc/torch/v2/any.h"
#include "k2/python/csrc/torch/v2/doc/any.h"
#include "k2/python/csrc/torch/v2/doc/doc.h"
#include "k2/python/csrc/torch/v2/ragged_any.h"

namespace k2 {

void PybindRaggedAny(py::module &m) {
  py::class_<RaggedAny> any(m, "RaggedTensor");

  //==================================================
  //      k2.ragged.Tensor methods
  //--------------------------------------------------

  any.def(py::init([](py::list data, py::object dtype = py::none(),
                      py::object device =
                          py::str("cpu")) -> std::unique_ptr<RaggedAny> {
            std::string device_str = device.is_none() ? "cpu" : py::str(device);
            return std::make_unique<RaggedAny>(data, dtype,
                                               torch::Device(device_str));
          }),
          py::arg("data"), py::arg("dtype") = py::none(),
          py::arg("device") = py::str("cpu"), kRaggedAnyInitDataDeviceDoc);

  any.def(py::init<py::list, py::object, const std::string &>(),
          py::arg("data"), py::arg("dtype") = py::none(),
          py::arg("device") = "cpu", kRaggedAnyInitDataDeviceDoc);

  any.def(py::init([](const std::string &s, py::object dtype = py::none(),
                      py::object device =
                          py::str("cpu")) -> std::unique_ptr<RaggedAny> {
            std::string device_str = device.is_none() ? "cpu" : py::str(device);
            return std::make_unique<RaggedAny>(s, dtype, device_str);
          }),
          py::arg("s"), py::arg("dtype") = py::none(),
          py::arg("device") = py::str("cpu"), kRaggedAnyInitStrDeviceDoc);

  any.def(py::init<const std::string &, py::object, const std::string &>(),
          py::arg("s"), py::arg("dtype") = py::none(),
          py::arg("device") = "cpu", kRaggedAnyInitStrDeviceDoc);

  any.def(py::init<const RaggedShape &, torch::Tensor>(), py::arg("shape"),
          py::arg("value"), kRaggedInitFromShapeAndTensorDoc);

  any.def(py::init<torch::Tensor>(), py::arg("tensor"),
          kRaggedAnyInitTensorDoc);

  any.def(
      "__str__",
      [](const RaggedAny &self) -> std::string { return self.ToString(); },
      kRaggedAnyStrDoc);

  any.def(
      "to_str_simple",
      [](const RaggedAny &self) -> std::string {
        return self.ToString(/*compact*/ true);
      },
      kRaggedAnyToStrSimpleDoc);

  any.def(
      "__repr__",
      [](const RaggedAny &self) -> std::string { return self.ToString(); },
      kRaggedAnyStrDoc);

  any.def(
      "__getitem__",
      [](RaggedAny &self, int32_t i) -> py::object {
        if (self.any.NumAxes() > 2) {
          RaggedAny ragged = self.Index(/*axis*/ 0, i);
          return py::cast(ragged);
        } else {
          DeviceGuard guard(self.any.Context());
          K2_CHECK_EQ(self.any.NumAxes(), 2);
          Array1<int32_t> row_split = self.any.RowSplits(1).To(GetCpuContext());
          const int32_t *row_split_data = row_split.Data();
          int32_t begin = row_split_data[i], end = row_split_data[i + 1];
          Dtype t = self.any.GetDtype();
          FOR_REAL_AND_INT32_TYPES(t, T, {
            Array1<T> array =
                self.any.Specialize<T>().values.Arange(begin, end);
            torch::Tensor tensor = ToTorch(array);
            return py::cast(tensor);
          });
        }
        // Unreachable code
        return py::none();
      },
      py::arg("i"), kRaggedAnyGetItemDoc);

  any.def(
      "__getitem__",
      [](RaggedAny &self, const py::slice &slice) -> RaggedAny {
        py::size_t start = 0, stop = 0, step = 0, slicelength = 0;
        if (!slice.compute(self.any.Dim0(), &start, &stop, &step, &slicelength))
          throw py::error_already_set();
        int32_t istart = static_cast<int32_t>(start);
        int32_t istop = static_cast<int32_t>(stop);
        int32_t istep = static_cast<int32_t>(step);
        K2_CHECK_EQ(istep, 1)
            << "Only support slicing with step 1, given : " << istep;

        return self.Arange(/*axis*/ 0, istart, istop);
      },
      py::arg("key"), kRaggedAnyGetItemSliceDoc);

  any.def(
      "__getitem__",
      [](RaggedAny &self, torch::Tensor key) -> RaggedAny {
        // key is a 1-d torch tensor with dtype torch.int32
        DeviceGuard guard(self.any.Context());
        Array1<int32_t> indexes = FromTorch<int32_t>(key);
        Dtype t = self.any.GetDtype();
        FOR_REAL_AND_INT32_TYPES(t, T, {
          Ragged<T> ans =
              k2::Index<T>(self.any.Specialize<T>(), /*axis*/ 0, indexes,
                           /*value_indexes*/ nullptr);

          return RaggedAny(ans.Generic());
        });
        // Unreachable code
        return {};
      },
      py::arg("key"), kRaggedAnyGetItem1DTensorDoc);

  any.def("index",
          static_cast<RaggedAny (RaggedAny::*)(RaggedAny &)>(&RaggedAny::Index),
          py::arg("indexes"), kRaggedAnyRaggedIndexDoc);

  any.def("index",
          static_cast<std::pair<RaggedAny, torch::optional<torch::Tensor>> (
              RaggedAny::*)(torch::Tensor, int32_t, bool)>(&RaggedAny::Index),
          py::arg("indexes"), py::arg("axis"),
          py::arg("need_value_indexes") = false, kRaggedAnyTensorIndexDoc);

  m.def(
      "index",
      [](torch::Tensor src, RaggedAny &indexes,
         py::object default_value = py::none()) -> RaggedAny {
        return indexes.Index(src, default_value);
      },
      py::arg("src"), py::arg("indexes"), py::arg("default_value") = py::none(),
      kRaggedAnyIndexTensorWithRaggedDoc);

  m.def(
      "index_and_sum",
      [](torch::Tensor src, RaggedAny &indexes) -> torch::Tensor {
        return indexes.IndexAndSum(src);
      },
      py::arg("src"), py::arg("indexes"), kRaggedAnyIndexAndSumDoc);

  any.def(
      "to",
      [](RaggedAny &self, py::object device) -> RaggedAny {
        std::string device_str = device.is_none() ? "cpu" : py::str(device);
        return self.To(torch::Device(device_str));
      },
      py::arg("device"), kRaggedAnyToDeviceDoc);

  any.def("to",
          static_cast<RaggedAny (RaggedAny::*)(const std::string &) const>(
              &RaggedAny::To),
          py::arg("device"), kRaggedAnyToDeviceStrDoc);

  any.def("to",
          static_cast<RaggedAny (RaggedAny::*)(torch::ScalarType) const>(
              &RaggedAny::To),
          py::arg("dtype"), kRaggedAnyToDtypeDoc);

  any.def(
      "clone",
      [](const RaggedAny &self) -> RaggedAny {
        DeviceGuard guard(self.any.Context());
        return self.Clone();
      },
      kRaggedAnyCloneDoc);

  any.def(
      "__eq__",
      [](const RaggedAny &self, const RaggedAny &other) -> bool {
        DeviceGuard guard(self.any.Context());
        Dtype t = self.any.GetDtype();
        bool ans = false;
        FOR_REAL_AND_INT32_TYPES(t, T, {
          ans = Equal<T>(self.any.Specialize<T>(), other.any.Specialize<T>());
        });
        return ans;
      },
      py::arg("other"), kRaggedAnyEqDoc);

  any.def(
      "__ne__",
      [](const RaggedAny &self, const RaggedAny &other) -> bool {
        DeviceGuard guard(self.any.Context());
        Dtype t = self.any.GetDtype();
        bool ans = false;
        FOR_REAL_AND_INT32_TYPES(t, T, {
          ans = !Equal<T>(self.any.Specialize<T>(), other.any.Specialize<T>());
        });
        return ans;
      },
      py::arg("other"), kRaggedAnyNeDoc);

  any.def("requires_grad_", &RaggedAny::SetRequiresGrad,
          py::arg("requires_grad") = true, kRaggedAnyRequiresGradMethodDoc);

  any.def("sum", &RaggedAny::Sum, py::arg("initial_value") = 0,
          kRaggedAnySumDoc);

  any.def(
      "numel",
      [](RaggedAny &self) -> int32_t {
        DeviceGuard guard(self.any.Context());
        return self.any.NumElements();
      },
      kRaggedAnyNumelDoc);

  any.def(
      "tot_size",
      [](const RaggedAny &self, int32_t axis) -> int32_t {
        DeviceGuard guard(self.any.Context());
        return self.any.TotSize(axis);
      },
      py::arg("axis"), kRaggedAnyTotSizeDoc);

  any.def(py::pickle(
      [](const RaggedAny &self) -> py::tuple {
        DeviceGuard guard(self.any.Context());
        K2_CHECK(self.any.NumAxes() == 2 || self.any.NumAxes() == 3)
            << "Only support Ragged with NumAxes() == 2 or 3 for now, "
               "given "
            << self.any.NumAxes();
        Array1<int32_t> row_splits1 = self.any.RowSplits(1);
        Dtype t = self.any.GetDtype();

        FOR_REAL_AND_INT32_TYPES(t, T, {
          auto values = self.any.Specialize<T>().values;
          // We use "row_ids" placeholder here to make it compatible for the
          // old format file.
          if (self.any.NumAxes() == 2) {
            return py::make_tuple(ToTorch(row_splits1), "row_ids1",
                                  ToTorch(values));
          } else {
            Array1<int32_t> row_splits2 = self.any.RowSplits(2);
            return py::make_tuple(ToTorch(row_splits1), "row_ids1",
                                  ToTorch(row_splits2), "row_ids2",
                                  ToTorch(values));
          }
        });
        // Unreachable code
        return py::none();
      },
      [](const py::tuple &t) -> RaggedAny {
        K2_CHECK(t.size() == 3 || t.size() == 5)
            << "Invalid state. "
            << "Expect a size of 3 or 5. Given: " << t.size();

        torch::Tensor row_splits1_tensor = t[0].cast<torch::Tensor>();
        DeviceGuard guard(GetContext(row_splits1_tensor));
        Array1<int32_t> row_splits1 = FromTorch<int32_t>(row_splits1_tensor);

        RaggedShape shape;
        if (t.size() == 3) {
          auto values_tensor = t[2].cast<torch::Tensor>();
          Dtype t = ScalarTypeToDtype(values_tensor.scalar_type());
          FOR_REAL_AND_INT32_TYPES(t, T, {
            auto values = FromTorch<T>(values_tensor);
            shape = RaggedShape2(&row_splits1, nullptr, values.Dim());
            Ragged<T> any(shape, values);
            return RaggedAny(any.Generic());
          });
        } else if (t.size() == 5) {
          torch::Tensor row_splits2_tensor = t[2].cast<torch::Tensor>();
          Array1<int32_t> row_splits2 = FromTorch<int32_t>(row_splits2_tensor);

          auto values_tensor = t[4].cast<torch::Tensor>();
          Dtype t = ScalarTypeToDtype(values_tensor.scalar_type());

          FOR_REAL_AND_INT32_TYPES(t, T, {
            auto values = FromTorch<T>(values_tensor);
            shape = RaggedShape3(&row_splits1, nullptr, -1, &row_splits2,
                                 nullptr, values.Dim());
            Ragged<T> any(shape, values);
            return RaggedAny(any.Generic());
          });
        } else {
          K2_LOG(FATAL) << "Invalid size : " << t.size();
        }

        // Unreachable code
        return {};
      }));
  SetMethodDoc(&any, "__getstate__", kRaggedAnyGetStateDoc);
  SetMethodDoc(&any, "__setstate__", kRaggedAnySetStateDoc);

  any.def("remove_axis", &RaggedAny::RemoveAxis, py::arg("axis"),
          kRaggedAnyRemoveAxisDoc);

  any.def("arange", &RaggedAny::Arange, py::arg("axis"), py::arg("begin"),
          py::arg("end"), kRaggedAnyArangeDoc);

  any.def("remove_values_leq", &RaggedAny::RemoveValuesLeq, py::arg("cutoff"),
          kRaggedAnyRemoveValuesLeqDoc);

  any.def("remove_values_eq", &RaggedAny::RemoveValuesEq, py::arg("target"),
          kRaggedAnyRemoveValuesEqDoc);

  any.def("argmax", &RaggedAny::ArgMax, py::arg("initial_value") = py::none(),
          kRaggedAnyArgMaxDoc);

  any.def("max", &RaggedAny::Max, py::arg("initial_value") = py::none(),
          kRaggedAnyMaxDoc);

  any.def("min", &RaggedAny::Min, py::arg("initial_value") = py::none(),
          kRaggedAnyMinDoc);

  any.def_static("cat", &RaggedAny::Cat, py::arg("srcs"), py::arg("axis"),
                 kRaggedCatDoc);
  m.attr("cat") = any.attr("cat");

  any.def("unique", &RaggedAny::Unique, py::arg("need_num_repeats") = false,
          py::arg("need_new2old_indexes") = false, kRaggedAnyUniqueDoc);

  any.def("normalize", &RaggedAny::Normalize, py::arg("use_log"),
          kRaggedAnyNormalizeDoc);

  any.def("pad", &RaggedAny::Pad, py::arg("mode"), py::arg("padding_value"),
          kRaggedAnyPadDoc);

  any.def("tolist", &RaggedAny::ToList, kRaggedAnyToListDoc);

  any.def("sort_", &RaggedAny::Sort, py::arg("descending") = false,
          py::arg("need_new2old_indexes") = false, kRaggedAnySortDoc);

  //==================================================
  //      k2.ragged.Tensor properties
  //--------------------------------------------------

  any.def_property_readonly(
      "dtype",
      [](const RaggedAny &self) -> py::object {
        Dtype t = self.any.GetDtype();
        auto torch = py::module::import("torch");
        switch (t) {
          case kFloatDtype:
            return torch.attr("float32");
          case kDoubleDtype:
            return torch.attr("float64");
          case kInt32Dtype:
            return torch.attr("int32");
          default:
            K2_LOG(FATAL) << "Unsupported dtype: " << TraitsOf(t).Name();
        }

        // Unreachable code
        return py::none();
      },
      kRaggedAnyDtypeDoc);

  any.def_property_readonly(
      "device",
      [](const RaggedAny &self) -> py::object {
        DeviceType d = self.any.Context()->GetDeviceType();
        torch::DeviceType device_type = ToTorchDeviceType(d);

        torch::Device device(device_type, self.any.Context()->GetDeviceId());

        auto torch_device = py::module::import("torch").attr("device");
        return torch_device(device.str());
      },
      kRaggedAnyDeviceDoc);

  // Return the underlying memory of this tensor.
  // No data is copied. Memory is shared.
  any.def_property_readonly(
      "values", [](RaggedAny &self) -> torch::Tensor { return self.Data(); },
      kRaggedAnyValuesDoc);

  any.def_property_readonly(
      "shape", [](RaggedAny &self) -> RaggedShape { return self.any.shape; },
      kRaggedAnyShapeDoc);

  any.def_property_readonly(
      "grad",
      [](RaggedAny &self) -> torch::optional<torch::Tensor> {
        if (!self.data.defined()) return {};

        return self.Data().grad();
      },
      kRaggedAnyGradPropDoc);

  any.def_property(
      "requires_grad",
      [](RaggedAny &self) -> bool {
        if (!self.data.defined()) return false;

        return self.Data().requires_grad();
      },
      [](RaggedAny &self, bool requires_grad) -> void {
        self.SetRequiresGrad(requires_grad);
      },
      kRaggedAnyRequiresGradPropDoc);

  any.def_property_readonly(
      "is_cuda",
      [](RaggedAny &self) -> bool {
        return self.any.Context()->GetDeviceType() == kCuda;
      },
      kRaggedAnyIsCudaDoc);

  // NumAxes() does not access GPU memory
  any.def_property_readonly(
      "num_axes",
      [](const RaggedAny &self) -> int32_t { return self.any.NumAxes(); },
      kRaggedAnyNumAxesDoc);

  // Dim0() does not access GPU memory
  any.def_property_readonly(
      "dim0", [](const RaggedAny &self) -> int32_t { return self.any.Dim0(); },
      kRaggedAnyDim0Doc);

  //==================================================
  //      _k2.ragged.functions
  //--------------------------------------------------

  m.def(
      "create_ragged_tensor",
      [](py::list data, py::object dtype = py::none(),
         py::object device = py::str("cpu")) -> RaggedAny {
        std::string device_str = device.is_none() ? "cpu" : py::str(device);
        return RaggedAny(data, dtype, torch::Device(device_str));
      },
      py::arg("data"), py::arg("dtype") = py::none(),
      py::arg("device") = py::str("cpu"), kCreateRaggedTensorDataDoc);

  m.def(
      "create_ragged_tensor",
      [](py::list data, py::object dtype = py::none(),
         const std::string &device = "cpu") -> RaggedAny {
        return RaggedAny(data, dtype, device);
      },
      py::arg("data"), py::arg("dtype") = py::none(), py::arg("device") = "cpu",
      kCreateRaggedTensorDataDoc);

  m.def(
      "create_ragged_tensor",
      [](const std::string &s, py::object dtype = py::none(),
         py::object device = py::str("cpu")) -> RaggedAny {
        std::string device_str = device.is_none() ? "cpu" : py::str(device);
        return RaggedAny(s, dtype, torch::Device(device_str));
      },
      py::arg("s"), py::arg("dtype") = py::none(),
      py::arg("device") = py::str("cpu"), kCreateRaggedTensorStrDoc);

  m.def(
      "create_ragged_tensor",
      [](const std::string &s, py::object dtype = py::none(),
         const std::string &device = "cpu") -> RaggedAny {
        return RaggedAny(s, dtype, device);
      },
      py::arg("s"), py::arg("dtype") = py::none(), py::arg("device") = "cpu",
      kCreateRaggedTensorStrDoc);

  m.def(
      "create_ragged_tensor",
      [](torch::Tensor tensor) -> RaggedAny { return RaggedAny(tensor); },
      py::arg("tensor"), kCreateRaggedTensorTensorDoc);
}

}  // namespace k2
