#include "hip/hip_runtime.h"
/**
 * @brief python wrappers for ragged_ops.h
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corp.       (authors: Fangjun Kuang
 *                                                   Daniel Povey)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <tuple>
#include <utility>
#include <vector>

#include "k2/csrc/device_guard.h"
#include "k2/csrc/ragged_ops.h"
#include "k2/python/csrc/torch/ragged_ops.h"
#include "k2/python/csrc/torch/torch_util.h"

namespace k2 {

template <typename T>
static void PybindRaggedRemoveAxis(py::module &m) {
  // src is a Ragged<T>
  //  there is another `remove_axis` in k2/python/csrc/torch/ragged.cu
  //  taking a RaggedShape as input.
  m.def(
      "remove_axis",
      [](Ragged<T> &src, int32_t axis) -> Ragged<T> {
        DeviceGuard guard(src.Context());
        return RemoveAxis<T>(src, axis);
      },
      py::arg("src"), py::arg("axis"));
}

template <typename T>
static void PybindRaggedArange(py::module &m, const char *name) {
  m.def(
      name,
      [](Ragged<T> &src, int32_t axis, int32_t begin,
         int32_t end) -> Ragged<T> {
        DeviceGuard guard(src.Context());
        return Arange<T>(src, axis, begin, end);
      },
      py::arg("src"), py::arg("axis"), py::arg("begin"), py::arg("end"));
}

template <typename T>
static void PybindRemoveValuesLeq(py::module &m, const char *name) {
  m.def(
      name,
      [](Ragged<T> &src, T cutoff) -> Ragged<T> {
        DeviceGuard guard(src.Context());
        return RemoveValuesLeq(src, cutoff);
      },
      py::arg("src"), py::arg("cutoff"));
}

template <typename T>
static void PybindRemoveValuesEq(py::module &m, const char *name) {
  m.def(
      name,
      [](Ragged<T> &src, T target) -> Ragged<T> {
        DeviceGuard guard(src.Context());
        return RemoveValuesEq(src, target);
      },
      py::arg("src"), py::arg("target"));
}

// Recursive implementation function used inside PybindToLists().
// Returns a list containing elements `begin` through `end-1` on
// axis `axis` of `r`, with 0 <= axis < r.NumAxes(), and
// 0 <= begin <= end <= r.TotSize(axis).
static py::list RaggedInt32ToList(Ragged<int32_t> &r, int32_t axis,
                                  int32_t begin, int32_t end) {
  K2_CHECK_LT(static_cast<uint32_t>(axis), static_cast<uint32_t>(r.NumAxes()));
  K2_CHECK_LE(end, r.TotSize(axis));
  py::list ans(end - begin);
  int32_t num_axes = r.NumAxes();
  int32_t *data;
  if (axis == num_axes - 1)
    data = r.values.Data();
  else
    data = r.RowSplits(axis + 1).Data();
  for (int32_t i = begin; i < end; i++) {
    if (axis == num_axes - 1) {
      ans[i - begin] = data[i];
    } else {
      int32_t row_begin = data[i], row_end = data[i + 1];
      ans[i - begin] = RaggedInt32ToList(r, axis + 1, row_begin, row_end);
    }
  }
  return ans;
};

static void PybindRaggedIntToList(py::module &m, const char *name) {
  m.def(
      name,
      [](Ragged<int32_t> &src) -> py::list {
        DeviceGuard guard(src.Context());
        Ragged<int32_t> r = src.To(GetCpuContext());
        return RaggedInt32ToList(r, 0, 0, r.Dim0());
      },
      py::arg("src"));
}

template <typename T>
static void PybindNormalizePerSublist(py::module &m, const char *name) {
  m.def(
      name,
      [](Ragged<T> &src, bool use_log) -> Ragged<T> {
        DeviceGuard guard(src.Context());
        return NormalizePerSublist(src, use_log);
      },
      py::arg("src"), py::arg("use_log"));
}

/* Backward propagation for NormalizePerSublist.

   @param [in] out      It is the output of `NormalizePerSublist(src)`.
   @param [in] use_log  It indicates which kind of normalization was used.
   @param [in] out_grad The gradient for `out`; must have same type as `out`
                        (float or double), and shape (out.NumElements(),).
   @return  Return the gradient for `src`.  A torch.Tensor with shape
                        (out.NumElements(),).
 */
template <typename T>
static torch::Tensor NormalizePerSublistBackward(Ragged<T> &out, bool use_log,
                                                 torch::Tensor out_grad) {
  NVTX_RANGE(K2_FUNC);
  DeviceGuard guard(out.Context());
  K2_CHECK_EQ(out_grad.dim(), 1)
      << "Expected dim: 1. Given: " << out_grad.dim();
  K2_CHECK_EQ(out_grad.scalar_type(), ToScalarType<T>::value)
      << "Expected scalar type: " << ToScalarType<T>::value
      << ". Given: " << out_grad.scalar_type();
  K2_CHECK(use_log) << "It supports only use_log==True at present";

  ContextPtr context = GetContext(out_grad);
  K2_CHECK(context->IsCompatible(*out.Context()));

  int32_t num_axes = out.NumAxes();
  Array1<T> out_grad_sum(context, out.TotSize(num_axes - 2));
  T *out_grad_sum_data = out_grad_sum.Data();
  const T *out_grad_data = out_grad.data_ptr<T>();

  int64_t stride = out_grad.strides()[0];
  if (stride != 0) {
    Array1<T> out_grad_array = FromTorch<T>(out_grad);
    K2_CHECK_EQ(out.values.Dim(), out_grad_array.Dim());

    Ragged<T> out_grad_ragged(out.shape, out_grad_array);
    SumPerSublist<T>(out_grad_ragged, 0, &out_grad_sum);
  } else {
    // stride is 0;
    // the sum is the number_of_elements_in_the_sublist * out_grad[0]
    const int32_t *row_splits_data = out.RowSplits(num_axes - 1).Data();
    K2_EVAL(
        context, out_grad_sum.Dim(), lambda_compute_out_grad_sum,
        (int32_t i)->void {
          int32_t begin = row_splits_data[i];
          int32_t end = row_splits_data[i + 1];
          out_grad_sum_data[i] = (end - begin) * out_grad_data[0];
        });
  }

  Array1<T> ans_grad_array(context, out.NumElements());
  T *ans_grad_data = ans_grad_array.Data();
  const T *out_data = out.values.Data();
  const int32_t *row_ids_data = out.RowIds(num_axes - 1).Data();
  int32_t num_elements = ans_grad_array.Dim();

  if (std::is_same<T, float>::value) {
    // use `expf` for float
    K2_EVAL(
        context, num_elements, lambda_set_ans_grad, (int32_t i)->void {
          int32_t row = row_ids_data[i];
          T scale = out_grad_sum_data[row];
          ans_grad_data[i] =
              out_grad_data[i * stride] - expf(out_data[i]) * scale;
        });
  } else {
    // use `exp` for double
    K2_EVAL(
        context, num_elements, lambda_set_ans_grad, (int32_t i)->void {
          int32_t row = row_ids_data[i];
          T scale = out_grad_sum_data[row];
          ans_grad_data[i] =
              out_grad_data[i * stride] - exp(out_data[i]) * scale;
        });
  }
  return ToTorch(ans_grad_array);
}

template <typename T>
static void PybindNormalizePerSublistBackward(py::module &m, const char *name) {
  // the device guard is used inside NormalizePerSublistBackward<T>
  m.def(name, NormalizePerSublistBackward<T>, py::arg("out"),
        py::arg("use_log"), py::arg("out_grad"));
}

template <typename T, typename Op>
static void PybindOpPerSublist(py::module &m, Op op, const char *name) {
  m.def(
      name,
      [op](Ragged<T> &src, T initial_value) -> torch::Tensor {
        DeviceGuard guard(src.Context());
        Array1<T> values(src.Context(), src.TotSize(src.NumAxes() - 2));
        op(src, initial_value, &values);
        return ToTorch(values);
      },
      py::arg("src"), py::arg("initial_value"));
}

template <typename T>
static void PybindCat(py::module &m) {
  // py::list is more efficient, but it requires more code
  m.def(
      "cat",
      [](std::vector<Ragged<T>> &srcs, int32_t axis) -> Ragged<T> {
        DeviceGuard guard(srcs[0].Context());
        return Cat(axis, srcs.size(), &srcs[0]);
      },
      py::arg("srcs"), py::arg("axis"));
}

template <typename T>
static void PybindCreateRagged2(py::module &m) {
  m.def(
      "create_ragged2",
      [](const std::vector<std::vector<T>> &vecs) -> Ragged<T> {
        return CreateRagged2(vecs);
      },
      py::arg("vecs"));
}

static void PybindGetLayer(py::module &m) {
  m.def("get_layer", &GetLayer, py::arg("src"), py::arg("layer"));
}

static void PybindUniqueSequences(py::module &m) {
  m.def(
      "unique_sequences",
      [](Ragged<int32_t> &src, bool need_num_repeats = true,
         bool need_new2old_indexes = false)
          -> std::tuple<Ragged<int32_t>, torch::optional<Ragged<int32_t>>,
                        torch::optional<torch::Tensor>> {
        DeviceGuard guard(src.Context());
        Ragged<int32_t> num_repeats;
        Array1<int32_t> new2old_indexes;
        Ragged<int32_t> ans =
            UniqueSequences(src, need_num_repeats ? &num_repeats : nullptr,
                            need_new2old_indexes ? &new2old_indexes : nullptr);

        torch::optional<Ragged<int32_t>> num_repeats_tensor;
        if (need_num_repeats) num_repeats_tensor = num_repeats;

        torch::optional<torch::Tensor> new2old_indexes_tensor;
        if (need_new2old_indexes)
          new2old_indexes_tensor = ToTorch(new2old_indexes);

        return std::make_tuple(ans, num_repeats_tensor, new2old_indexes_tensor);
      },
      py::arg("src"), py::arg("need_num_repeats") = true,
      py::arg("need_new2old_indexes") = false);
}

static void PybindIndex(py::module &m) {
  // Note there are several overloads of `index`
  // in k2/python/csrc/torch/ragged.cu

  // return a pair:
  //  - ans (RaggedShape)
  //  - value_indexes (optional)
  //
  m.def(
      "index",
      [](RaggedShape &src, int32_t axis, torch::Tensor indexes,
         bool need_value_indexes =
             true) -> std::pair<RaggedShape, torch::optional<torch::Tensor>> {
        DeviceGuard guard(src.Context());
        Array1<int32_t> indexes_array = FromTorch<int32_t>(indexes);
        Array1<int32_t> value_indexes;
        RaggedShape ans = Index(src, axis, indexes_array,
                                need_value_indexes ? &value_indexes : nullptr);

        torch::optional<torch::Tensor> value_indexes_tensor;
        if (need_value_indexes) value_indexes_tensor = ToTorch(value_indexes);

        return std::make_pair(ans, value_indexes_tensor);
      },
      py::arg("src"), py::arg("axis"), py::arg("indexes"),
      py::arg("need_value_indexes") = true);
}

static void PybindRegularRaggedShape(py::module &m) {
  // TODO(fangjun): pass a torch.device to specify the context
  //
  // As a workaround, the user can use
  // _k2.regular_ragged_shape(...).to(torch.device)
  // to move it to a given device
  m.def(
      "regular_ragged_shape",
      [](int32_t dim0, int32_t dim1) -> RaggedShape {
        ContextPtr c = GetCpuContext();
        return RegularRaggedShape(c, dim0, dim1);
      },
      py::arg("dim0"), py::arg("dim1"));
}

template <typename T>
static void PybindArgMaxPerSublist(py::module &m) {
  m.def(
      "argmax_per_sublist",
      [](Ragged<T> &src, T initial_value) -> torch::Tensor {
        DeviceGuard guard(src.Context());
        int32_t last_axis = src.NumAxes() - 1;
        const Array1<int32_t> &row_splits_array = src.RowSplits(last_axis);
        int32_t num_rows = row_splits_array.Dim() - 1;

        Array1<int32_t> indexes(src.Context(), num_rows);
        ArgMaxPerSublist(src, initial_value, &indexes);

        return ToTorch(indexes);
      },
      py::arg("src"), py::arg("initial_value"));
}

template <typename T>
static void PybindMaxPerSublist(py::module &m) {
  m.def(
      "max_per_sublist",
      [](Ragged<T> &src, T initial_value) -> torch::Tensor {
        DeviceGuard guard(src.Context());
        int32_t last_axis = src.NumAxes() - 1;
        const Array1<int32_t> &row_splits_array = src.RowSplits(last_axis);
        int32_t num_rows = row_splits_array.Dim() - 1;

        Array1<T> max_values(src.Context(), num_rows);
        MaxPerSublist(src, initial_value, &max_values);

        return ToTorch(max_values);
      },
      py::arg("src"), py::arg("initial_value"));
}

}  // namespace k2

void PybindRaggedOps(py::module &m) {
  using namespace k2;  // NOLINT
  PybindRaggedRemoveAxis<int32_t>(m);
  PybindRaggedArange<int32_t>(m, "ragged_int_arange");
  PybindRemoveValuesLeq<int32_t>(m, "ragged_int_remove_values_leq");
  PybindRemoveValuesEq<int32_t>(m, "ragged_int_remove_values_eq");
  PybindRaggedIntToList(m, "ragged_int_to_list");
  PybindNormalizePerSublist<float>(m, "normalize_per_sublist");
  PybindNormalizePerSublistBackward<float>(m, "normalize_per_sublist_backward");
  PybindOpPerSublist<float>(m, SumPerSublist<float>, "sum_per_sublist");
  PybindCat<int32_t>(m);
  PybindCat<Arc>(m);
  PybindCreateRagged2<int32_t>(m);
  PybindCreateRagged2<float>(m);
  PybindGetLayer(m);
  PybindUniqueSequences(m);
  PybindIndex(m);
  PybindRegularRaggedShape(m);
  PybindArgMaxPerSublist<float>(m);
  PybindArgMaxPerSublist<int32_t>(m);
  PybindMaxPerSublist<float>(m);
  PybindMaxPerSublist<int32_t>(m);
}
