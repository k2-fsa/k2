#include "hip/hip_runtime.h"
/**
 * @brief python wrappers for ragged_ops.h
 *
 * @copyright
 * Copyright      2020  Xiaomi Corp.       (authors: Fangjun Kuang
 *                                                   Daniel Povey)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <string>
#include <tuple>
#include <utility>
#include <vector>

#include "k2/csrc/device_guard.h"
#include "k2/csrc/ragged_ops.h"
#include "k2/csrc/torch_util.h"
#include "k2/python/csrc/torch/ragged_ops.h"

namespace k2 {

template <typename T>
static void PybindRaggedRemoveAxis(py::module &m) {
  // src is a Ragged<T>
  //  there is another `remove_axis` in k2/python/csrc/torch/ragged.cu
  //  taking a RaggedShape as input.
  m.def(
      "remove_axis",
      [](Ragged<T> &src, int32_t axis) -> Ragged<T> {
        DeviceGuard guard(src.Context());
        return RemoveAxis<T>(src, axis);
      },
      py::arg("src"), py::arg("axis"));
}

template <typename T>
static void PybindRaggedArange(py::module &m, const char *name) {
  m.def(
      name,
      [](Ragged<T> &src, int32_t axis, int32_t begin,
         int32_t end) -> Ragged<T> {
        DeviceGuard guard(src.Context());
        return Arange<T>(src, axis, begin, end);
      },
      py::arg("src"), py::arg("axis"), py::arg("begin"), py::arg("end"));
}

template <typename T>
static void PybindRemoveValuesLeq(py::module &m, const char *name) {
  m.def(
      name,
      [](Ragged<T> &src, T cutoff) -> Ragged<T> {
        DeviceGuard guard(src.Context());
        return RemoveValuesLeq(src, cutoff);
      },
      py::arg("src"), py::arg("cutoff"));
}

template <typename T>
static void PybindRemoveValuesEq(py::module &m, const char *name) {
  m.def(
      name,
      [](Ragged<T> &src, T target) -> Ragged<T> {
        DeviceGuard guard(src.Context());
        return RemoveValuesEq(src, target);
      },
      py::arg("src"), py::arg("target"));
}

// Recursive implementation function used inside PybindToLists().
// Returns a list containing elements `begin` through `end-1` on
// axis `axis` of `r`, with 0 <= axis < r.NumAxes(), and
// 0 <= begin <= end <= r.TotSize(axis).
static py::list RaggedInt32ToList(Ragged<int32_t> &r, int32_t axis,
                                  int32_t begin, int32_t end) {
  K2_CHECK_LT(static_cast<uint32_t>(axis), static_cast<uint32_t>(r.NumAxes()));
  K2_CHECK_LE(end, r.TotSize(axis));
  py::list ans(end - begin);
  int32_t num_axes = r.NumAxes();
  int32_t *data;
  if (axis == num_axes - 1)
    data = r.values.Data();
  else
    data = r.RowSplits(axis + 1).Data();
  for (int32_t i = begin; i < end; i++) {
    if (axis == num_axes - 1) {
      ans[i - begin] = data[i];
    } else {
      int32_t row_begin = data[i], row_end = data[i + 1];
      ans[i - begin] = RaggedInt32ToList(r, axis + 1, row_begin, row_end);
    }
  }
  return ans;
};

static void PybindRaggedIntToList(py::module &m, const char *name) {
  m.def(
      name,
      [](Ragged<int32_t> &src) -> py::list {
        DeviceGuard guard(src.Context());
        Ragged<int32_t> r = src.To(GetCpuContext());
        return RaggedInt32ToList(r, 0, 0, r.Dim0());
      },
      py::arg("src"));
}

template <typename T>
static void PybindPadRaggedToTensor(py::module &m) {
  m.def(
      "pad_ragged",
      [](Ragged<T> &src, const std::string &mode,
         T padding_value) -> torch::Tensor {
        DeviceGuard guard(src.Context());
        Array2<T> res = PadRagged(src, mode, padding_value);
        return ToTorch(res);
      },
      py::arg("src"), py::arg("mode"), py::arg("padding_value"));
}

template <typename T>
static void PybindNormalizePerSublist(py::module &m, const char *name) {
  m.def(
      name,
      [](Ragged<T> &src, bool use_log) -> Ragged<T> {
        DeviceGuard guard(src.Context());
        return NormalizePerSublist(src, use_log);
      },
      py::arg("src"), py::arg("use_log"));
}

/* Backward propagation for NormalizePerSublist.

   @param [in] out      It is the output of `NormalizePerSublist(src)`.
   @param [in] use_log  It indicates which kind of normalization was used.
   @param [in] out_grad The gradient for `out`; must have same type as `out`
                        (float or double), and shape (out.NumElements(),).
   @return  Return the gradient for `src`.  A torch.Tensor with shape
                        (out.NumElements(),).
 */
template <typename T>
/*static*/ torch::Tensor NormalizePerSublistBackward(Ragged<T> &out,
                                                     bool use_log,
                                                     torch::Tensor out_grad) {
  NVTX_RANGE(K2_FUNC);
  DeviceGuard guard(out.Context());
  K2_CHECK_EQ(out_grad.dim(), 1)
      << "Expected dim: 1. Given: " << out_grad.dim();
  K2_CHECK_EQ(out_grad.scalar_type(), ToScalarType<T>::value)
      << "Expected scalar type: " << ToScalarType<T>::value
      << ". Given: " << out_grad.scalar_type();
  K2_CHECK(use_log) << "It supports only use_log==True at present";

  ContextPtr context = GetContext(out_grad);
  K2_CHECK(context->IsCompatible(*out.Context()));

  int32_t num_axes = out.NumAxes();
  Array1<T> out_grad_sum(context, out.TotSize(num_axes - 2));
  T *out_grad_sum_data = out_grad_sum.Data();
  const T *out_grad_data = out_grad.data_ptr<T>();

  int64_t stride = out_grad.strides()[0];
  if (stride != 0) {
    Array1<T> out_grad_array = FromTorch<T>(out_grad);
    K2_CHECK_EQ(out.values.Dim(), out_grad_array.Dim());

    Ragged<T> out_grad_ragged(out.shape, out_grad_array);
    SumPerSublist<T>(out_grad_ragged, 0, &out_grad_sum);
  } else {
    // stride is 0;
    // the sum is the number_of_elements_in_the_sublist * out_grad[0]
    const int32_t *row_splits_data = out.RowSplits(num_axes - 1).Data();
    K2_EVAL(
        context, out_grad_sum.Dim(), lambda_compute_out_grad_sum,
        (int32_t i)->void {
          int32_t begin = row_splits_data[i];
          int32_t end = row_splits_data[i + 1];
          out_grad_sum_data[i] = (end - begin) * out_grad_data[0];
        });
  }

  Array1<T> ans_grad_array(context, out.NumElements());
  T *ans_grad_data = ans_grad_array.Data();
  const T *out_data = out.values.Data();
  const int32_t *row_ids_data = out.RowIds(num_axes - 1).Data();
  int32_t num_elements = ans_grad_array.Dim();

  if (std::is_same<T, float>::value) {
    // use `expf` for float
    K2_EVAL(
        context, num_elements, lambda_set_ans_grad, (int32_t i)->void {
          int32_t row = row_ids_data[i];
          T scale = out_grad_sum_data[row];
          ans_grad_data[i] =
              out_grad_data[i * stride] - expf(out_data[i]) * scale;
        });
  } else {
    // use `exp` for double
    K2_EVAL(
        context, num_elements, lambda_set_ans_grad, (int32_t i)->void {
          int32_t row = row_ids_data[i];
          T scale = out_grad_sum_data[row];
          ans_grad_data[i] =
              out_grad_data[i * stride] - exp(out_data[i]) * scale;
        });
  }
  return ToTorch(ans_grad_array);
}

template <typename T>
static void PybindNormalizePerSublistBackward(py::module &m, const char *name) {
  // the device guard is used inside NormalizePerSublistBackward<T>
  m.def(name, NormalizePerSublistBackward<T>, py::arg("out"),
        py::arg("use_log"), py::arg("out_grad"));
}

template <typename T, typename Op>
static void PybindOpPerSublist(py::module &m, Op op, const char *name) {
  m.def(
      name,
      [op](Ragged<T> &src, T initial_value) -> torch::Tensor {
        DeviceGuard guard(src.Context());
        Array1<T> values(src.Context(), src.TotSize(src.NumAxes() - 2));
        op(src, initial_value, &values);
        return ToTorch(values);
      },
      py::arg("src"), py::arg("initial_value"));
}

template <typename T>
static void PybindCat(py::module &m) {
  // py::list is more efficient, but it requires more code
  m.def(
      "cat",
      [](std::vector<Ragged<T>> &srcs, int32_t axis) -> Ragged<T> {
        DeviceGuard guard(srcs[0].Context());
        return Cat(axis, srcs.size(), &srcs[0]);
      },
      py::arg("srcs"), py::arg("axis"));
}

template <typename T>
static void PybindCreateRagged2(py::module &m) {
  m.def(
      "create_ragged2",
      [](const std::vector<std::vector<T>> &vecs) -> Ragged<T> {
        return CreateRagged2(vecs);
      },
      py::arg("vecs"));
}

static void PybindGetLayer(py::module &m) {
  m.def("get_layer", &GetLayer, py::arg("src"), py::arg("layer"));
}

static void PybindUniqueSequences(py::module &m) {
  m.def(
      "unique_sequences",
      [](Ragged<int32_t> &src, bool need_num_repeats = true,
         bool need_new2old_indexes = false)
          -> std::tuple<Ragged<int32_t>, torch::optional<Ragged<int32_t>>,
                        torch::optional<torch::Tensor>> {
        DeviceGuard guard(src.Context());
        Ragged<int32_t> num_repeats;
        Array1<int32_t> new2old_indexes;
        Ragged<int32_t> ans =
            UniqueSequences(src, need_num_repeats ? &num_repeats : nullptr,
                            need_new2old_indexes ? &new2old_indexes : nullptr);

        torch::optional<Ragged<int32_t>> num_repeats_tensor;
        if (need_num_repeats) num_repeats_tensor = num_repeats;

        torch::optional<torch::Tensor> new2old_indexes_tensor;
        if (need_new2old_indexes)
          new2old_indexes_tensor = ToTorch(new2old_indexes);

        return std::make_tuple(ans, num_repeats_tensor, new2old_indexes_tensor);
      },
      py::arg("src"), py::arg("need_num_repeats") = true,
      py::arg("need_new2old_indexes") = false);
}

static void PybindIndex(py::module &m) {
  // Note there are several overloads of `index`
  // in k2/python/csrc/torch/ragged.cu

  // return a pair:
  //  - ans (RaggedShape)
  //  - value_indexes (optional)
  //
  m.def(
      "index",
      [](RaggedShape &src, int32_t axis, torch::Tensor indexes,
         bool need_value_indexes =
             true) -> std::pair<RaggedShape, torch::optional<torch::Tensor>> {
        DeviceGuard guard(src.Context());
        Array1<int32_t> indexes_array = FromTorch<int32_t>(indexes);
        Array1<int32_t> value_indexes;
        RaggedShape ans = Index(src, axis, indexes_array,
                                need_value_indexes ? &value_indexes : nullptr);

        torch::optional<torch::Tensor> value_indexes_tensor;
        if (need_value_indexes) value_indexes_tensor = ToTorch(value_indexes);

        return std::make_pair(ans, value_indexes_tensor);
      },
      py::arg("src"), py::arg("axis"), py::arg("indexes"),
      py::arg("need_value_indexes") = true);
}

template <typename T>
static void PybindArgMaxPerSublist(py::module &m) {
  m.def(
      "argmax_per_sublist",
      [](Ragged<T> &src, T initial_value) -> torch::Tensor {
        DeviceGuard guard(src.Context());
        int32_t last_axis = src.NumAxes() - 1;
        const Array1<int32_t> &row_splits_array = src.RowSplits(last_axis);
        int32_t num_rows = row_splits_array.Dim() - 1;

        Array1<int32_t> indexes(src.Context(), num_rows);
        ArgMaxPerSublist(src, initial_value, &indexes);

        return ToTorch(indexes);
      },
      py::arg("src"), py::arg("initial_value"));
}

template <typename T>
static void PybindMaxPerSublist(py::module &m) {
  m.def(
      "max_per_sublist",
      [](Ragged<T> &src, T initial_value) -> torch::Tensor {
        DeviceGuard guard(src.Context());
        int32_t last_axis = src.NumAxes() - 1;
        const Array1<int32_t> &row_splits_array = src.RowSplits(last_axis);
        int32_t num_rows = row_splits_array.Dim() - 1;

        Array1<T> max_values(src.Context(), num_rows);
        MaxPerSublist(src, initial_value, &max_values);

        return ToTorch(max_values);
      },
      py::arg("src"), py::arg("initial_value"));
}

template <typename T>
static void PybindSortSublists(py::module &m) {
  // If need_new2old_indexes is true, returns an extra tensor.
  // The returned tensor contains entries mapping from the sorted tensor
  // to the unsorted tensor.
  //
  // in_unsorted[ans_tensor] is sorted
  //
  m.def(
      "sort_sublists",
      [](Ragged<T> &in_out, bool descending = false,
         bool need_new2old_indexes = false) -> torch::optional<torch::Tensor> {
        ContextPtr &c = in_out.Context();
        DeviceGuard guard(c);
        Array1<int32_t> new2old;
        if (need_new2old_indexes)
          new2old = Array1<int32_t>(c, in_out.NumElements());

        if (descending) {
          SortSublists<T, GreaterThan<T>>(
              &in_out, need_new2old_indexes ? &new2old : nullptr);
        } else {
          SortSublists<T, LessThan<T>>(
              &in_out, need_new2old_indexes ? &new2old : nullptr);
        }
        torch::optional<torch::Tensor> ans;
        if (need_new2old_indexes) ans = ToTorch(new2old);
        return ans;
      },
      py::arg("in_out"), py::arg("descending") = false,
      py::arg("need_new2old_indexes") = false);
}

}  // namespace k2

void PybindRaggedOps(py::module &m) {
  using namespace k2;  // NOLINT
  PybindArgMaxPerSublist<float>(m);
  PybindArgMaxPerSublist<int32_t>(m);
  PybindCat<int32_t>(m);
  PybindCat<float>(m);
  PybindCat<k2::Arc>(m);
  PybindCreateRagged2<int32_t>(m);
  PybindCreateRagged2<float>(m);
  PybindGetLayer(m);
  PybindIndex(m);
  PybindMaxPerSublist<float>(m);
  PybindMaxPerSublist<int32_t>(m);
  PybindNormalizePerSublist<float>(m, "normalize_per_sublist");
  PybindNormalizePerSublistBackward<float>(m, "normalize_per_sublist_backward");
  PybindOpPerSublist<float>(m, SumPerSublist<float>, "sum_per_sublist");
  PybindPadRaggedToTensor<int32_t>(m);
  PybindPadRaggedToTensor<float>(m);
  PybindRaggedArange<int32_t>(m, "ragged_int_arange");
  PybindRaggedIntToList(m, "ragged_int_to_list");
  PybindRaggedRemoveAxis<int32_t>(m);
  PybindRemoveValuesEq<int32_t>(m, "ragged_int_remove_values_eq");
  PybindRemoveValuesLeq<int32_t>(m, "ragged_int_remove_values_leq");
  PybindSortSublists<float>(m);
  PybindSortSublists<int32_t>(m);
  PybindUniqueSequences(m);
}
