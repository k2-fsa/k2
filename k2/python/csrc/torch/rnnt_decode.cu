/**
 * @brief python wrappers for rnnt_decode.h
 *
 * @copyright
 * Copyright      2022  Xiaomi Corp.       (authors: Wei Kang)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <memory>
#include <sstream>
#include <string>
#include <utility>
#include <vector>

#include "k2/csrc/device_guard.h"
#include "k2/csrc/fsa.h"
#include "k2/csrc/rnnt_decode.h"
#include "k2/python/csrc/torch/rnnt_decode.h"
#include "k2/python/csrc/torch/torch_util.h"

namespace k2 {
static void PybindRnntDecodingConfig(py::module &m) {
  using PyClass = rnnt_decoding::RnntDecodingConfig;
  py::class_<PyClass> config(m, "RnntDecodingConfig");
  config.def(py::init<int32_t, int32_t, double, int32_t, int32_t>(),
             py::arg("vocab_size"), py::arg("decoder_history_len"),
             py::arg("beam"), py::arg("max_states"), py::arg("max_contexts"),
             R"(
             Construct a RnntDecodingConfig object, it contains the parameters
             needed by rnnt decoding.

             Args:
               vocab_size:
                 It indicates how many symbols we are using, equals the
                 largest-symbol plus one.
               decoder_history_len:
                 The number of symbols of history the
                 decoder takes; will normally be one or two
                 ("stateless decoder"), our RNN-T decoding setup does not
                 support unlimited decoder context such as with LSTMs.
               beam:
                 `beam` imposes a limit on the score of a state, relative to the
                 best-scoring state on the same frame.  E.g. 10.
               max_states:
                 `max_states` is a limit on the number of distinct states that
                 we allow per frame, per stream; the number of states will not
                 be allowed to exceed this limit.
               max_contexts:
                 `max_contexts` is a limit on the number of distinct contexts
                 that we allow per frame, per stream; the number of contexts
                 will not be allowed to exceed this limit.
             )");

  config.def_readwrite("vocab_size", &PyClass::vocab_size)
      .def_readwrite("decoder_history_len", &PyClass::decoder_history_len)
      .def_readwrite("beam", &PyClass::beam)
      .def_readwrite("max_states", &PyClass::max_states)
      .def_readwrite("max_contexts", &PyClass::max_contexts);

  config.def("__str__", [](const PyClass &self) -> std::string {
    std::ostringstream os;
    os << "RnntDecodingConfig : {\n"
       << "  vocab_size : " << self.vocab_size << "\n"
       << "  decoder_history_len : " << self.decoder_history_len << "\n"
       << "  beam : " << self.beam << "\n"
       << "  max_states : " << self.max_states << "\n"
       << "  max_contexts : " << self.max_contexts << "\n"
       << "}";
    return os.str();
  });
}

static void PybindRnntDecodingStream(py::module &m) {
  using PyClass = rnnt_decoding::RnntDecodingStream;
  py::class_<PyClass, std::shared_ptr<PyClass>> stream(m, "RnntDecodingStream");

  stream.def("__str__", [](const PyClass &self) -> std::string {
    std::ostringstream os;
    os << "RnntDecodingStream : {\n"
       << "  num graph states : " << self.graph->Dim0() << "\n"
       << "  num graph arcs : " << self.graph->NumElements() << "\n"
       << "  num contexts : " << self.states.Dim0() << "\n"
       << "  num states : " << self.states.NumElements() << "\n"
       << "  num prev frames : " << self.prev_frames.size() << "\n"
       << "}";
    return os.str();
  });

  m.def("create_rnnt_decoding_stream",
        [](Fsa &graph) -> std::shared_ptr<PyClass> {
          DeviceGuard guard(graph.Context());
          return rnnt_decoding::CreateStream(std::make_shared<Fsa>(graph));
        });
}

static void PybindRnntDecodingStreams(py::module &m) {
  using PyClass = rnnt_decoding::RnntDecodingStreams;
  py::class_<PyClass> streams(m, "RnntDecodingStreams");

  streams.def(py::init(
      [](std::vector<std::shared_ptr<rnnt_decoding::RnntDecodingStream>> &srcs,
         const rnnt_decoding::RnntDecodingConfig &config)
          -> std::unique_ptr<PyClass> {
        K2_CHECK_GE(srcs.size(), 1);
        DeviceGuard guard(srcs[0]->graph->Context());
        return std::make_unique<PyClass>(srcs, config);
      }));

  streams.def("advance", [](PyClass &self, torch::Tensor logprobs) -> void {
    DeviceGuard guard(self.Context());
    logprobs = logprobs.to(torch::kFloat);
    Array2<float> logprobs_array = FromTorch<float>(logprobs, Array2Tag{});
    self.Advance(logprobs_array);
  });

  streams.def("get_contexts",
              [](PyClass &self) -> std::pair<RaggedShape, torch::Tensor> {
                DeviceGuard guard(self.Context());
                RaggedShape shape;
                Array2<int32_t> contexts;
                self.GetContexts(&shape, &contexts);
                torch::Tensor contexts_tensor = ToTorch<int32_t>(contexts);
                return std::make_pair(shape, contexts_tensor);
              });

  streams.def("terminate_and_flush_to_streams", [](PyClass &self) -> void {
    DeviceGuard guard(self.Context());
    self.TerminateAndFlushToStreams();
  });

  streams.def(
      "format_output",
      [](PyClass &self,
         std::vector<int32_t> &num_frames) -> std::pair<FsaVec, torch::Tensor> {
        DeviceGuard guard(self.Context());
        FsaVec ofsa;
        Array1<int32_t> out_map;
        self.FormatOutput(num_frames, &ofsa, &out_map);
        torch::Tensor out_map_tensor = ToTorch<int32_t>(out_map);
        return std::make_pair(ofsa, out_map_tensor);
      });
}

}  // namespace k2

void PybindRnntDecode(py::module &m) {
  k2::PybindRnntDecodingConfig(m);
  k2::PybindRnntDecodingStream(m);
  k2::PybindRnntDecodingStreams(m);
}
