/**
 * @brief python wrappers for k2.
 *
 * @copyright
 * Copyright (c)  2020  Mobvoi Inc.        (authors: Fangjun Kuang)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include "k2/python/csrc/k2.h"
#include "k2/python/csrc/torch.h"
#include "k2/python/csrc/version.h"

PYBIND11_MODULE(_k2, m) {
  m.doc() = "pybind11 binding of k2";
  PybindVersion(m);
  PybindTorch(m);
}
