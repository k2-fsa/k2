/**
 * @brief python wrappers for k2.
 *
 * @copyright
 * Copyright (c)  2020  Mobvoi AI Lab, Beijing, China (authors: Fangjun Kuang)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include "k2/python/csrc/k2.h"

#include "k2/python/csrc/torch.h"

PYBIND11_MODULE(_k2, m) {
  m.doc() = "pybind11 binding of k2";
  PybindTorch(m);
}
