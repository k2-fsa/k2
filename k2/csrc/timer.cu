#include "hip/hip_runtime.h"
/**
 * Copyright      2020  Xiaomi Corporation (authors: Haowen Qiu)
 *                      Mobvoi Inc.        (authors: Fangjun Kuang)
 *
 * See LICENSE for clarification regarding multiple authors
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <chrono>  // NOLINT
#include <memory>

#include "k2/csrc/context.h"
#include "k2/csrc/log.h"
#include "k2/csrc/timer.h"

namespace k2 {

class TimerImpl {
 public:
  TimerImpl() = default;
  virtual ~TimerImpl() = default;
  virtual void Reset() = 0;
  // Return time in seconds
  virtual double Elapsed() = 0;
};

// modified from https://github.com/kaldi-asr/kaldi/blob/master/src/base/timer.h
class CpuTimerImpl : public TimerImpl {
 public:
  CpuTimerImpl() { Reset(); }

  using high_resolution_clock = std::chrono::high_resolution_clock;

  void Reset() override { begin_ = high_resolution_clock::now(); }

  // Return time in seconds
  double Elapsed() override {
    auto end = high_resolution_clock::now();
    auto diff =
        std::chrono::duration_cast<std::chrono::microseconds>(end - begin_);
    return diff.count() / 1000000.0;
  }

 private:
  high_resolution_clock::time_point begin_;
};

class CudaTimerImpl : public TimerImpl {
 public:
  explicit CudaTimerImpl(hipStream_t stream) : stream_(stream) {
    K2_CUDA_SAFE_CALL(hipEventCreate(&time_start_));
    K2_CUDA_SAFE_CALL(hipEventCreate(&time_end_));
    Reset();
  }

  ~CudaTimerImpl() override {
    K2_CUDA_SAFE_CALL(hipEventDestroy(time_start_));
    K2_CUDA_SAFE_CALL(hipEventDestroy(time_end_));
  }

  void Reset() override {
    K2_CUDA_SAFE_CALL(hipEventRecord(time_start_, stream_));
  }

  // Return time in seconds
  double Elapsed() override {
    K2_CUDA_SAFE_CALL(hipEventRecord(time_end_, stream_));
    K2_CUDA_SAFE_CALL(hipEventSynchronize(time_end_));

    float ms_elapsed = 0;
    K2_CUDA_SAFE_CALL(
        hipEventElapsedTime(&ms_elapsed, time_start_, time_end_));
    return ms_elapsed / 1000;
  }

 private:
  hipEvent_t time_start_;
  hipEvent_t time_end_;
  hipStream_t stream_;
};

Timer::Timer(ContextPtr context) {
  switch (context->GetDeviceType()) {
    case kCpu:
      timer_impl_ = std::make_unique<CpuTimerImpl>();
      break;
    case kCuda:
      timer_impl_ = std::make_unique<CudaTimerImpl>(context->GetCudaStream());
      break;
    default:
      K2_LOG(FATAL) << "Unsupported device type: " << context->GetDeviceType();
      break;
  }
}

Timer::~Timer() = default;

void Timer::Reset() const { timer_impl_->Reset(); }

// Return time in seconds
double Timer::Elapsed() const { return timer_impl_->Elapsed(); }

}  // namespace k2
