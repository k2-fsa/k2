#include "hip/hip_runtime.h"

/**
 * @brief
 * timer
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Haowen Qiu)
 *                      Mobvoi Inc.        (authors: Fangjun Kuang)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <sys/time.h>

#include <memory>

#include "k2/csrc/context.h"
#include "k2/csrc/log.h"
#include "k2/csrc/timer.h"

namespace k2 {

class TimerImpl {
 public:
  TimerImpl() = default;
  virtual ~TimerImpl() = default;
  virtual void Reset() = 0;
  // Return time in seconds
  virtual double Elapsed() = 0;
};

// modified from https://github.com/kaldi-asr/kaldi/blob/master/src/base/timer.h
class CpuTimerImpl : public TimerImpl {
 public:
  CpuTimerImpl() { Reset(); }

  void Reset() override { gettimeofday(&time_start_, nullptr); }

  // Return time in seconds
  double Elapsed() override {
    struct timeval time_end;
    gettimeofday(&time_end, nullptr);
    double t1, t2;
    t1 = static_cast<double>(time_start_.tv_sec) +
         static_cast<double>(time_start_.tv_usec) / (1000 * 1000);
    t2 = static_cast<double>(time_end.tv_sec) +
         static_cast<double>(time_end.tv_usec) / (1000 * 1000);
    return t2 - t1;
  }

 private:
  struct timeval time_start_;
};

class CudaTimerImpl : public TimerImpl {
 public:
  explicit CudaTimerImpl(hipStream_t stream) : stream_(stream) {
    K2_CUDA_SAFE_CALL(hipEventCreate(&time_start_));
    K2_CUDA_SAFE_CALL(hipEventCreate(&time_end_));
    Reset();
  }

  ~CudaTimerImpl() override {
    K2_CUDA_SAFE_CALL(hipEventDestroy(time_start_));
    K2_CUDA_SAFE_CALL(hipEventDestroy(time_end_));
  }

  void Reset() override {
    K2_CUDA_SAFE_CALL(hipEventRecord(time_start_, stream_));
  }

  // Return time in seconds
  double Elapsed() override {
    K2_CUDA_SAFE_CALL(hipEventRecord(time_end_, stream_));
    K2_CUDA_SAFE_CALL(hipEventSynchronize(time_end_));

    float ms_elapsed;
    K2_CUDA_SAFE_CALL(
        hipEventElapsedTime(&ms_elapsed, time_start_, time_end_));
    return ms_elapsed / 1000;
  }

 private:
  hipEvent_t time_start_;
  hipEvent_t time_end_;
  hipStream_t stream_;
};

Timer::Timer(ContextPtr context) {
  switch (context->GetDeviceType()) {
    case kCpu:
      timer_impl_ = std::make_unique<CpuTimerImpl>();
      break;
    case kCuda:
      timer_impl_ = std::make_unique<CudaTimerImpl>(context->GetCudaStream());
      break;
    default:
      K2_LOG(FATAL) << "Unsupported device type: " << context->GetDeviceType();
      break;
  }
}

Timer::~Timer() = default;

void Timer::Reset() const { timer_impl_->Reset(); }

// Return time in seconds
double Timer::Elapsed() const { return timer_impl_->Elapsed(); }

}  // namespace k2
