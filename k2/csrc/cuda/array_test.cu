#include "hip/hip_runtime.h"
// k2/csrc/cuda/array_test.cu

// Copyright (c)  2020  Xiaomi Corporation (author: Haowen Qiu)

// See ../../../LICENSE for clarification regarding multiple authors

#include <algorithm>
#include <numeric>
#include <vector>

#include "gmock/gmock.h"
#include "gtest/gtest.h"
#include "k2/csrc/cuda/array.h"
#include "k2/csrc/cuda/context.h"
#include "k2/csrc/cuda/dtype.h"
#include "k2/csrc/cuda/log.h"
#include "k2/csrc/cuda/tensor.h"

namespace k2 {
template <typename T, DeviceType d>
void TestArray1() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  {
    // created with Array1(ContextPtr ctx, int32_t size), test Array1.Data()
    Array1<T> array(context, 5);
    ASSERT_EQ(array.Dim(), 5);
    std::vector<T> data(array.Dim());
    std::iota(data.begin(), data.end(), 0);
    T *array_data = array.Data();
    // copy data from CPU to CPU/GPU
    auto kind = GetMemoryCopyKind(*cpu, *array.Context());
    MemoryCopy(static_cast<void *>(array_data),
               static_cast<void *>(data.data()),
               array.Dim() * array.ElementSize(), kind);
    // copy data from CPU/GPU to CPU
    kind = GetMemoryCopyKind(*array.Context(), *cpu);
    std::vector<T> cpu_data(array.Dim());
    MemoryCopy(static_cast<void *>(cpu_data.data()),
               static_cast<const void *>(array_data),
               array.Dim() * array.ElementSize(), kind);
    for (int32_t i = 0; i < array.Dim(); ++i) {
      EXPECT_EQ(cpu_data[i], i);
    }
  }

  {
    // test operator=(T t)
    Array1<T> array(context, 5);
    ASSERT_EQ(array.Dim(), 5);
    // operator=(T t)
    array = 2;
    // copy data from CPU/GPU to CPU
    const T *array_data = array.Data();
    auto kind = GetMemoryCopyKind(*array.Context(), *cpu);
    std::vector<T> cpu_data(array.Dim());
    MemoryCopy(static_cast<void *>(cpu_data.data()),
               static_cast<const void *>(array_data),
               array.Dim() * array.ElementSize(), kind);
    for (int32_t i = 0; i < array.Dim(); ++i) {
      EXPECT_EQ(cpu_data[i], 2);
    }
  }

  {
    // created with Array1(ContextPtr, int32_t size, T elem)
    Array1<T> array(context, 5, 2);
    ASSERT_EQ(array.Dim(), 5);
    // copy data from CPU/GPU to CPU
    const T *array_data = array.Data();
    auto kind = GetMemoryCopyKind(*array.Context(), *cpu);
    std::vector<T> cpu_data(array.Dim());
    MemoryCopy(static_cast<void *>(cpu_data.data()),
               static_cast<const void *>(array_data),
               array.Dim() * array.ElementSize(), kind);
    for (int32_t i = 0; i < array.Dim(); ++i) {
      EXPECT_EQ(cpu_data[i], 2);
    }
  }

  {
    // created with Array(ContextPtr, const std:vector<T>&)
    std::vector<T> data(5);
    std::iota(data.begin(), data.end(), 0);
    Array1<T> array(context, data);
    ASSERT_EQ(array.Dim(), 5);
    // copy data from CPU/GPU to CPU
    const T *array_data = array.Data();
    auto kind = GetMemoryCopyKind(*array.Context(), *cpu);
    std::vector<T> cpu_data(array.Dim());
    MemoryCopy(static_cast<void *>(cpu_data.data()),
               static_cast<const void *>(array_data),
               array.Dim() * array.ElementSize(), kind);
    for (int32_t i = 0; i < array.Dim(); ++i) {
      EXPECT_EQ(cpu_data[i], data[i]);
    }
  }

  {
    // test Range(start, size)
    std::vector<T> data(10);
    std::iota(data.begin(), data.end(), 0);
    Array1<T> array(context, data);
    int32_t start = 2;
    int32_t size = 6;
    Array1<T> sub_array = array.Range(start, size);
    ASSERT_EQ(sub_array.Dim(), size);
    // copy data from CPU/GPU to CPU
    const T *sub_array_data = sub_array.Data();
    auto kind = GetMemoryCopyKind(*sub_array.Context(), *cpu);
    std::vector<T> cpu_data(sub_array.Dim());
    MemoryCopy(static_cast<void *>(cpu_data.data()),
               static_cast<const void *>(sub_array_data),
               sub_array.Dim() * sub_array.ElementSize(), kind);
    for (int32_t i = 0; i < sub_array.Dim(); ++i) {
      EXPECT_EQ(cpu_data[i], data[i + start]);
    }
  }

  {
    // test Range(start, size, inc)
    std::vector<T> data(20);
    std::iota(data.begin(), data.end(), 0);
    Array1<T> array(context, data);
    int32_t start = 3;
    int32_t size = 8;
    int32_t inc = 2;
    Tensor sub_tensor = array.Range(start, size, inc);
    Dtype type = DtypeOf<T>::dtype;
    EXPECT_EQ(sub_tensor.GetDtype(), type);
    Shape shape = sub_tensor.GetShape();
    EXPECT_EQ(shape.Ndim(), 1);
    EXPECT_EQ(shape.Nelement(), size);
    EXPECT_EQ(shape.StorageSize(), (size - 1) * inc + 1);
    EXPECT_EQ(shape.Dim(0), size);
    EXPECT_EQ(shape.Stride(0), inc);
    // copy data from CPU/GPU to CPU
    const T *sub_tensor_data = sub_tensor.Data<T>();
    auto kind = GetMemoryCopyKind(*(sub_tensor.GetRegion()->context), *cpu);
    std::vector<T> cpu_data(shape.StorageSize());
    MemoryCopy(static_cast<void *>(cpu_data.data()),
               static_cast<const void *>(sub_tensor_data),
               shape.StorageSize() * TraitsOf(sub_tensor.GetDtype()).NumBytes(),
               kind);
    int32_t dim0 = shape.Dim(0);
    int32_t stride0 = shape.Stride(0);
    for (int32_t i = 0, j = start; i < dim0; ++i, j += inc) {
      EXPECT_EQ(cpu_data[i * stride0], data[j]);
    }
  }

  {
    // test ToTensor
    int32_t size = 20;
    std::vector<T> data(size);
    std::iota(data.begin(), data.end(), 0);
    Array1<T> array(context, data);
    Tensor tensor = array.ToTensor();
    Dtype type = DtypeOf<T>::dtype;
    EXPECT_EQ(tensor.GetDtype(), type);
    Shape shape = tensor.GetShape();
    EXPECT_EQ(shape.Ndim(), 1);
    EXPECT_EQ(shape.Nelement(), size);
    EXPECT_EQ(shape.StorageSize(), size);
    EXPECT_EQ(shape.Dim(0), size);
    EXPECT_EQ(shape.Stride(0), 1);
    // copy data from CPU/GPU to CPU
    const T *tensor_data = tensor.Data<T>();
    auto kind = GetMemoryCopyKind(*(tensor.GetRegion()->context), *cpu);
    std::vector<T> cpu_data(shape.StorageSize());
    MemoryCopy(static_cast<void *>(cpu_data.data()),
               static_cast<const void *>(tensor_data),
               shape.StorageSize() * TraitsOf(tensor.GetDtype()).NumBytes(),
               kind);
    int32_t dim0 = shape.Dim(0);
    int32_t stride0 = shape.Stride(0);
    for (int32_t i = 0, j = 0; i < dim0; ++i, ++j) {
      EXPECT_EQ(cpu_data[i * stride0], data[j]);
    }
  }

  {
    // test operator[](const Array1<int32_t> indexes)
    std::vector<T> data(20);
    std::iota(data.begin(), data.end(), 1);
    Array1<T> array(context, data);
    std::vector<int32_t> indexes = {0, 1, 2, 5, 1, 6, 8, 9, 2, 4, 6, 3};
    Array1<int32_t> indexes_array(context, indexes);
    std::vector<T> expected_data = {1, 2, 3, 6, 2, 7, 9, 10, 3, 5, 7, 4};
    Array1<T> ans_array = array[indexes_array];
    // copy data from CPU/GPU to CPU
    ASSERT_EQ(ans_array.Dim(), expected_data.size());
    const T *ans_array_data = ans_array.Data();
    auto kind = GetMemoryCopyKind(*ans_array.Context(), *cpu);
    std::vector<T> cpu_data(ans_array.Dim());
    MemoryCopy(static_cast<void *>(cpu_data.data()),
               static_cast<const void *>(ans_array_data),
               ans_array.Dim() * ans_array.ElementSize(), kind);
    for (int32_t i = 0; i < ans_array.Dim(); ++i) {
      EXPECT_EQ(cpu_data[i], expected_data[i]);
    }
  }

  {
    // test Resize
    std::vector<T> data(5);
    std::iota(data.begin(), data.end(), 1);
    Array1<T> array(context, data);
    EXPECT_EQ(array.Dim(), data.size());

    // new_size <= array.Dim()
    int32_t new_size = 3;
    array.Resize(new_size);
    EXPECT_EQ(array.Dim(), data.size());
    new_size = 5;
    array.Resize(new_size);
    EXPECT_EQ(array.Dim(), data.size());
    // new_size > array.Dim()
    new_size = 8;
    array.Resize(new_size);
    EXPECT_EQ(array.Dim(), new_size);
    // copy data from CPU/GPU to CPU
    const T *array_data = array.Data();
    auto kind = GetMemoryCopyKind(*array.Context(), *cpu);
    std::vector<T> cpu_data(array.Dim());
    MemoryCopy(static_cast<void *>(cpu_data.data()),
               static_cast<const void *>(array_data),
               array.Dim() * array.ElementSize(), kind);
    // data.size() == 5, array.Dim() == 8, there are 3 uninitialized elements.
    for (int32_t i = 0; i < data.size(); ++i) {
      EXPECT_EQ(cpu_data[i], data[i]);
    }
  }
}

template <typename T, DeviceType d>
void TestArray2() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  {
    // created with Array2(Array1, dim0, dim1), contiguous
    std::vector<T> data(20);
    std::iota(data.begin(), data.end(), 0);
    Array1<T> array1(context, data);
    Array2<T> array(array1, 4, 5);
    EXPECT_EQ(array.Dim0(), 4);
    EXPECT_EQ(array.Dim1(), 5);
    EXPECT_EQ(array.ElemStride0(), 5);
    T *array_data = array.Data();

    {
      // test Data()
      const T *array_data = array.Data();
      // copy data from CPU/GPU to CPU
      int32_t elem_stride0 = array.ElemStride0();
      int32_t num_element_copy = array.Dim0() * array.ElemStride0();
      std::vector<T> cpu_data(num_element_copy);
      auto kind = GetMemoryCopyKind(*array.Context(), *cpu);
      MemoryCopy(static_cast<void *>(cpu_data.data()),
                 static_cast<const void *>(array_data),
                 num_element_copy * array.ElementSize(), kind);
      for (int32_t i = 0, n = 0; i < array.Dim0(); ++i) {
        for (int32_t j = 0; j < array.Dim1(); ++j) {
          EXPECT_EQ(cpu_data[i * elem_stride0 + j], data[n++]);
        }
      }
    }

    {
      // test operator[]
      for (int32_t i = 0; i < array.Dim0(); ++i) {
        Array1<T> sub_array = array[i];
        const T *sub_array_data = sub_array.Data();
        ASSERT_EQ(sub_array.Dim(), array.Dim1());
        auto kind = GetMemoryCopyKind(*sub_array.Context(), *cpu);
        std::vector<T> sub_array_cpu_data(sub_array.Dim());
        MemoryCopy(static_cast<void *>(sub_array_cpu_data.data()),
                   static_cast<const void *>(sub_array_data),
                   sub_array.Dim() * sub_array.ElementSize(), kind);
        for (int32_t j = 0; j < sub_array.Dim(); ++j) {
          EXPECT_EQ(sub_array_cpu_data[j], data[i * array.ElemStride0() + j]);
        }
      }
    }

    {
      // test Flatten()
      Array1<T> sub_array = array.Flatten();
      const T *sub_array_data = sub_array.Data();
      ASSERT_EQ(sub_array.Dim(), array.Dim0() * array.Dim1());
      auto kind = GetMemoryCopyKind(*sub_array.Context(), *cpu);
      std::vector<T> sub_array_cpu_data(sub_array.Dim());
      MemoryCopy(static_cast<void *>(sub_array_cpu_data.data()),
                 static_cast<const void *>(sub_array_data),
                 sub_array.Dim() * sub_array.ElementSize(), kind);
      for (int32_t i = 0; i < sub_array.Dim(); ++i) {
        EXPECT_EQ(sub_array_cpu_data[i], data[i]);
      }
    }

    {
      // test ToTensor()
      Tensor tensor = array.ToTensor();
      Dtype array_dtype = DtypeOf<T>::dtype;
      EXPECT_EQ(tensor.GetDtype(), array_dtype);
      Shape shape = tensor.GetShape();
      EXPECT_EQ(shape.Ndim(), 2);
      EXPECT_EQ(shape.Nelement(), array.Dim0() * array.Dim1());
      EXPECT_EQ(shape.Dim(0), array.Dim0());
      EXPECT_EQ(shape.Dim(1), array.Dim1());
      EXPECT_EQ(shape.Stride(0), array.ElemStride0());
      EXPECT_EQ(shape.Stride(1), 1);
      const T *tensor_data = tensor.Data<T>();
      auto kind = GetMemoryCopyKind(*tensor.GetRegion()->context, *cpu);
      std::vector<T> cpu_tensor_data(shape.StorageSize());
      MemoryCopy(static_cast<void *>(cpu_tensor_data.data()),
                 static_cast<const void *>(tensor_data),
                 shape.StorageSize() * TraitsOf(tensor.GetDtype()).NumBytes(),
                 kind);
      for (int32_t m = 0; m < shape.Dim(0); ++m) {
        for (int32_t n = 0; n < shape.Dim(1); ++n) {
          int32_t value =
              cpu_tensor_data[m * shape.Stride(0) + n * shape.Stride(1)];
          EXPECT_EQ(value, data[m * array.ElemStride0() + n]);
        }
      }
    }

    {
      // test constAccessor
      const auto &const_array = array;
      ConstArray2Accessor<T> accessor = const_array.Accessor();
      if (array.Context()->GetDeviceType() == kCpu) {
        EXPECT_EQ(accessor(0, 0), data[0 * array.ElemStride0() + 0]);
        EXPECT_EQ(accessor(2, 3), data[2 * array.ElemStride0() + 3]);
      }
    }

    {
      Array2Accessor<T> accessor = array.Accessor();
      if (array.Context()->GetDeviceType() == kCpu) {
        EXPECT_EQ(accessor(0, 0), 0);
        accessor(0, 0) = -10;
        EXPECT_EQ(accessor(0, 0), -10);
        EXPECT_EQ(array.Data()[0], -10);
      }
    }
  }

  {
    // created with region
    const int32_t element_size = TraitsOf(DtypeOf<T>::dtype).NumBytes();
    const int32_t num_element = 20;
    RegionPtr region = NewRegion(context, num_element * element_size);
    std::vector<T> src_data(num_element);
    std::iota(src_data.begin(), src_data.end(), 0);
    T *data = region->GetData<T, d>();
    auto kind = GetMemoryCopyKind(*cpu, *region->context);
    MemoryCopy(static_cast<void *>(data),
               static_cast<const void *>(src_data.data()),
               num_element * element_size, kind);

    {
      // created with region, contiguous on 0 aixs
      Array2<T> array(4, 5, 5, 0, region);
      EXPECT_EQ(array.Dim0(), 4);
      EXPECT_EQ(array.Dim1(), 5);
      EXPECT_EQ(array.ElemStride0(), 5);
      EXPECT_EQ(array.ElementSize(), element_size);
      // test Data()
      const T *array_data = array.Data();
      // copy data from CPU/GPU to CPU
      int32_t elem_stride0 = array.ElemStride0();
      int32_t num_element_copy = array.Dim0() * array.ElemStride0();
      std::vector<T> cpu_data(num_element_copy);
      kind = GetMemoryCopyKind(*array.Context(), *cpu);
      MemoryCopy(static_cast<void *>(cpu_data.data()),
                 static_cast<const void *>(array_data),
                 num_element_copy * array.ElementSize(), kind);
      for (int32_t i = 0, n = 0; i < array.Dim0(); ++i) {
        for (int32_t j = 0; j < array.Dim1(); ++j) {
          EXPECT_EQ(cpu_data[i * elem_stride0 + j], src_data[n++]);
        }
      }

      {
        // test Flatten()
        Array1<T> sub_array = array.Flatten();
        const T *sub_array_data = sub_array.Data();
        ASSERT_EQ(sub_array.Dim(), array.Dim0() * array.Dim1());
        kind = GetMemoryCopyKind(*sub_array.Context(), *cpu);
        std::vector<T> sub_array_cpu_data(sub_array.Dim());
        MemoryCopy(static_cast<void *>(sub_array_cpu_data.data()),
                   static_cast<const void *>(sub_array_data),
                   sub_array.Dim() * sub_array.ElementSize(), kind);
        for (int32_t i = 0, n = 0; i < array.Dim0(); ++i) {
          for (int32_t j = 0; j < array.Dim1(); ++j) {
            EXPECT_EQ(sub_array_cpu_data[n++], src_data[i * elem_stride0 + j]);
          }
        }
      }
    }

    {
      // created with region, non-contiguous on 0 aixs
      int32_t element_offset = 2;
      Array2<T> array(3, 5, 6, element_offset * element_size, region);
      EXPECT_EQ(array.Dim0(), 3);
      EXPECT_EQ(array.Dim1(), 5);
      EXPECT_EQ(array.ElemStride0(), 6);
      EXPECT_EQ(array.ElementSize(), element_size);
      // test Data()
      const T *array_data = array.Data();
      // copy data from CPU/GPU to CPU
      int32_t elem_stride0 = array.ElemStride0();
      int32_t num_element_copy = array.Dim0() * array.ElemStride0();
      std::vector<T> cpu_data(num_element_copy);
      kind = GetMemoryCopyKind(*array.Context(), *cpu);
      MemoryCopy(static_cast<void *>(cpu_data.data()),
                 static_cast<const void *>(array_data),
                 num_element_copy * array.ElementSize(), kind);
      for (int32_t i = 0; i < array.Dim0(); ++i) {
        for (int32_t j = 0; j < array.Dim1(); ++j) {
          EXPECT_EQ(cpu_data[i * elem_stride0 + j],
                    src_data[element_offset + i * elem_stride0 + j]);
        }
      }

      {
        // test Flatten()
        Array1<T> sub_array = array.Flatten();
        const T *sub_array_data = sub_array.Data();
        ASSERT_EQ(sub_array.Dim(), array.Dim0() * array.Dim1());
        kind = GetMemoryCopyKind(*sub_array.Context(), *cpu);
        std::vector<T> sub_array_cpu_data(sub_array.Dim());
        MemoryCopy(static_cast<void *>(sub_array_cpu_data.data()),
                   static_cast<const void *>(sub_array_data),
                   sub_array.Dim() * sub_array.ElementSize(), kind);
        for (int32_t i = 0, n = 0; i < array.Dim0(); ++i) {
          for (int32_t j = 0; j < array.Dim1(); ++j) {
            EXPECT_EQ(sub_array_cpu_data[n++],
                      src_data[element_offset + i * elem_stride0 + j]);
          }
        }
      }
    }
  }

  {
    // created with tensor, stride on 1st axis is 1
    const int32_t element_size = TraitsOf(DtypeOf<T>::dtype).NumBytes();
    const int32_t num_element = 24;
    RegionPtr region = NewRegion(context, num_element * element_size);
    std::vector<T> src_data(num_element);
    std::iota(src_data.begin(), src_data.end(), 0);
    T *data = region->GetData<T, d>();
    auto kind = GetMemoryCopyKind(*cpu, *region->context);
    MemoryCopy(static_cast<void *>(data),
               static_cast<const void *>(src_data.data()),
               num_element * element_size, kind);
    std::vector<int32_t> dims = {2, 4};
    std::vector<int32_t> strides = {10, 1};
    Shape shape(dims, strides);
    const int32_t element_offset = 4;
    const int32_t bytes_offset = element_offset * element_size;
    Tensor tensor(DtypeOf<T>::dtype, shape, region, bytes_offset);
    Array2<T> array(tensor, false);
    int32_t elem_stride0 = array.ElemStride0();
    int32_t elem_stride1 = tensor.GetShape().Stride(1);
    EXPECT_EQ(elem_stride1, 1);
    {
      // check_data
      const T *array_data = array.Data();
      int32_t num_element_copy = array.Dim0() * array.ElemStride0();
      std::vector<T> cpu_data(num_element_copy);
      kind = GetMemoryCopyKind(*array.Context(), *cpu);
      MemoryCopy(static_cast<void *>(cpu_data.data()),
                 static_cast<const void *>(array_data),
                 num_element_copy * array.ElementSize(), kind);
      for (int32_t i = 0; i < array.Dim0(); ++i) {
        for (int32_t j = 0; j < array.Dim1(); ++j) {
          EXPECT_EQ(
              cpu_data[i * elem_stride0 + j],
              src_data[element_offset + i * elem_stride0 + j * elem_stride1]);
        }
      }
    }

    {
      // test Flatten()
      Array1<T> sub_array = array.Flatten();
      const T *sub_array_data = sub_array.Data();
      ASSERT_EQ(sub_array.Dim(), array.Dim0() * array.Dim1());
      kind = GetMemoryCopyKind(*sub_array.Context(), *cpu);
      std::vector<T> sub_array_cpu_data(sub_array.Dim());
      MemoryCopy(static_cast<void *>(sub_array_cpu_data.data()),
                 static_cast<const void *>(sub_array_data),
                 sub_array.Dim() * sub_array.ElementSize(), kind);
      for (int32_t i = 0, n = 0; i < array.Dim0(); ++i) {
        for (int32_t j = 0; j < array.Dim1(); ++j) {
          EXPECT_EQ(
              sub_array_cpu_data[n++],
              src_data[element_offset + i * elem_stride0 + j * elem_stride1]);
        }
      }
    }
  }

  {
    // created with tensor, stride on 1st axis is not 1
    const int32_t element_size = TraitsOf(DtypeOf<T>::dtype).NumBytes();
    const int32_t num_element = 24;
    RegionPtr region = NewRegion(context, num_element * element_size);
    std::vector<T> src_data(num_element);
    std::iota(src_data.begin(), src_data.end(), 0);
    T *data = region->GetData<T, d>();
    auto kind = GetMemoryCopyKind(*cpu, *region->context);
    MemoryCopy(static_cast<void *>(data),
               static_cast<const void *>(src_data.data()),
               num_element * element_size, kind);
    std::vector<int32_t> dims = {2, 4};
    std::vector<int32_t> strides = {10, 2};
    Shape shape(dims, strides);
    const int32_t element_offset = 4;
    const int32_t bytes_offset = element_offset * element_size;
    Tensor tensor(DtypeOf<T>::dtype, shape, region, bytes_offset);
    Array2<T> array(tensor, true);
    int32_t elem_stride0 = array.ElemStride0();
    int32_t elem_stride1 = tensor.GetShape().Stride(1);
    {
      // check_data
      const T *array_data = array.Data();
      int32_t num_element_copy = array.Dim0() * array.ElemStride0();
      std::vector<T> cpu_data(num_element_copy);
      kind = GetMemoryCopyKind(*array.Context(), *cpu);
      MemoryCopy(static_cast<void *>(cpu_data.data()),
                 static_cast<const void *>(array_data),
                 num_element_copy * array.ElementSize(), kind);
      for (int32_t i = 0; i < array.Dim0(); ++i) {
        for (int32_t j = 0; j < array.Dim1(); ++j) {
          EXPECT_EQ(
              cpu_data[i * elem_stride0 + j],
              src_data[element_offset + i * elem_stride0 + j * elem_stride1]);
        }
      }
    }

    {
      // test Flatten()
      Array1<T> sub_array = array.Flatten();
      const T *sub_array_data = sub_array.Data();
      ASSERT_EQ(sub_array.Dim(), array.Dim0() * array.Dim1());
      kind = GetMemoryCopyKind(*sub_array.Context(), *cpu);
      std::vector<T> sub_array_cpu_data(sub_array.Dim());
      MemoryCopy(static_cast<void *>(sub_array_cpu_data.data()),
                 static_cast<const void *>(sub_array_data),
                 sub_array.Dim() * sub_array.ElementSize(), kind);
      for (int32_t i = 0, n = 0; i < array.Dim0(); ++i) {
        for (int32_t j = 0; j < array.Dim1(); ++j) {
          EXPECT_EQ(
              sub_array_cpu_data[n++],
              src_data[element_offset + i * elem_stride0 + j * elem_stride1]);
        }
      }
    }
  }
}

TEST(ArrayTest, Array1Test) {
  TestArray1<int32_t, kCpu>();
  TestArray1<int32_t, kCuda>();
  TestArray1<double, kCpu>();
  TestArray1<double, kCuda>();
}

TEST(ArrayTest, Array2Test) {
  TestArray2<int32_t, kCpu>();
  TestArray2<int32_t, kCuda>();
  TestArray2<double, kCpu>();
  TestArray2<double, kCuda>();
}

}  // namespace k2
