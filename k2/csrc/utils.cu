/**
 * @brief
 * context
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey
 *                                                   Meixu Song)
 *                      Fangjun Kuang (csukuangfj@gmail.com)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include "k2/csrc/utils.h"

#include <hipcub/hipcub.hpp>

namespace k2 {

template <typename SrcPtr, typename DestPtr>
void ExclusivePrefixSum(ContextPtr &c, int32_t n, SrcPtr src, DestPtr dest) {
  DeviceType d = c->GetDeviceType();
  using SumType = typename std::decay<decltype(dest[0])>::type;
  if (d == kCpu) {
    SumType sum = 0;
    for (int32_t i = 0; i != n; ++i) {
      dest[i] = sum;
      sum += src[i];
    }
  } else {
    assert(d == kCuda);
    // Determine temporary device storage requirements
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    // since d_temp_storage is nullptr, the following function will compute
    // the number of required bytes for d_temp_storage
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, src, dest,
                                  n, c->GetCudaStream());
    void *deleter_context;
    d_temp_storage = c->Allocate(temp_storage_bytes, &deleter_context);
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, src, dest,
                                  n, c->GetCudaStream());
    c->Deallocate(d_temp_storage, deleter_context);
  }
}

}  // namespace k2
