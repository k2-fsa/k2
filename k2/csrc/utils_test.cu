/**
 * @brief
 * utils_test
 *
 * @copyright
 * Copyright (c)  2020  Fangjun Kuang (csukuangfj@gmail.com)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include <numeric>
#include <vector>

#include "k2/csrc/array.h"
#include "k2/csrc/utils.h"

namespace k2 {

TEST(UtilsTest, CpuExclusiveSum) {
  void *deleter_context;
  ContextPtr c = GetCpuContext();
  int32_t n = 5;
  // [0, 1, 2, 3, 4]
  // the exclusive prefix sum is [0, 0, 1, 3, 6]
  auto *src = reinterpret_cast<int32_t *>(
      c->Allocate(n * sizeof(int32_t), &deleter_context));
  std::iota(src, src + n, 0);

  auto *dst = reinterpret_cast<int32_t *>(
      c->Allocate(n * sizeof(int32_t), &deleter_context));
  ExclusiveSum(c, n, src, dst);

  EXPECT_THAT(std::vector<int32_t>(dst, dst + n),
              ::testing::ElementsAre(0, 0, 1, 3, 6));

  c->Deallocate(dst, deleter_context);
  c->Deallocate(src, deleter_context);
}

TEST(UtilsTest, CudaExclusiveSum) {
  void *deleter_context;
  ContextPtr c = GetCudaContext();
  int32_t n = 5;
  auto *src = reinterpret_cast<int32_t *>(
      c->Allocate(n * sizeof(int32_t), &deleter_context));

  std::vector<int32_t> h(n);
  std::iota(h.begin(), h.end(), 0);
  hipMemcpy(src, h.data(), sizeof(int32_t) * n, hipMemcpyHostToDevice);

  auto *dst = reinterpret_cast<int32_t *>(
      c->Allocate(n * sizeof(int32_t), &deleter_context));
  ExclusiveSum(c, n, src, dst);

  hipMemcpy(h.data(), dst, sizeof(int32_t) * n, hipMemcpyDeviceToHost);

  EXPECT_THAT(h, ::testing::ElementsAre(0, 0, 1, 3, 6));

  c->Deallocate(dst, deleter_context);
  c->Deallocate(src, deleter_context);
}

template <DeviceType d>
void TestRowSplitsToRowIds() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  {
    // test empty case
    const std::vector<int32_t> row_splits_vec = {0};
    const std::vector<int32_t> row_ids_vec;
    Array1<int32_t> row_splits(context, row_splits_vec);
    int32_t num_rows = row_splits.Dim() - 1;
    int32_t num_elements = row_splits[num_rows];
    Array1<int32_t> row_ids(context, num_elements);
    int32_t *row_ids_data = row_ids.Data();
    EXPECT_EQ(row_ids.Dim(), num_elements);
    // just run to check if there is any error
    RowSplitsToRowIds(context, num_rows, row_splits.Data(), num_elements,
                      row_ids_data);
  }

  {
    const std::vector<int32_t> row_splits_vec = {0,  2,  3,  5,  8, 9,
                                                 12, 13, 15, 15, 16};
    const std::vector<int32_t> row_ids_vec = {0, 0, 1, 2, 2, 3, 3, 3,
                                              4, 5, 5, 5, 6, 7, 7, 9};
    Array1<int32_t> row_splits(context, row_splits_vec);
    int32_t num_rows = row_splits.Dim() - 1;
    int32_t num_elements = row_splits[num_rows];
    Array1<int32_t> row_ids(context, num_elements);
    int32_t *row_ids_data = row_ids.Data();
    EXPECT_EQ(row_ids.Dim(), num_elements);
    RowSplitsToRowIds(context, num_rows, row_splits.Data(), num_elements,
                      row_ids_data);
    // copy data from CPU/GPU to CPU
    Array1<int32_t> cpu_array = row_ids.To(cpu);
    std::vector<int32_t> cpu_data(cpu_array.Data(),
                                  cpu_array.Data() + cpu_array.Dim());
    EXPECT_EQ(cpu_data, row_ids_vec);
  }
}

TEST(UtilsTest, RowSplitsToRowIds) {
  TestRowSplitsToRowIds<kCpu>();
  TestRowSplitsToRowIds<kCuda>();
}

template <DeviceType d>
void TestRowIdsToRowSplits() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  {
    // test empty case
    const std::vector<int32_t> row_ids_vec;
    const std::vector<int32_t> row_splits_vec;
    Array1<int32_t> row_ids(context, row_ids_vec);
    int32_t num_rows = 0;
    int32_t num_elements = row_ids.Dim();
    Array1<int32_t> row_splits(context, num_rows + 1);
    int32_t *row_splits_data = row_splits.Data();
    RowIdsToRowSplits(context, num_elements, row_ids.Data(), true, num_rows,
                      row_splits_data);
    EXPECT_EQ(row_splits[0], 0);
  }

  {
    // no empty rows
    const std::vector<int32_t> row_ids_vec = {0, 0, 1, 1, 1, 2};
    const std::vector<int32_t> row_splits_vec = {0, 2, 5, 6};
    Array1<int32_t> row_ids(context, row_ids_vec);
    int32_t num_elements = row_ids.Dim();
    int32_t num_rows = row_ids[num_elements - 1] + 1;
    Array1<int32_t> row_splits(context, num_rows + 1);
    EXPECT_EQ(row_splits.Dim(), num_rows + 1);
    int32_t *row_splits_data = row_splits.Data();
    RowIdsToRowSplits(context, num_elements, row_ids.Data(), true, num_rows,
                      row_splits_data);
    // copy data from CPU/GPU to CPU
    Array1<int32_t> cpu_array = row_splits.To(cpu);
    std::vector<int32_t> cpu_data(cpu_array.Data(),
                                  cpu_array.Data() + cpu_array.Dim());
    EXPECT_EQ(cpu_data, row_splits_vec);
  }

  {
    // has empty rows
    const std::vector<int32_t> row_splits_vec = {0,  2,  3,  5,  8, 9,
                                                 12, 13, 15, 15, 16};
    const std::vector<int32_t> row_ids_vec = {0, 0, 1, 2, 2, 3, 3, 3,
                                              4, 5, 5, 5, 6, 7, 7, 9};
    Array1<int32_t> row_ids(context, row_ids_vec);
    int32_t num_elements = row_ids.Dim();
    int32_t num_rows = row_ids[num_elements - 1] + 1;
    Array1<int32_t> row_splits(context, num_rows + 1);
    EXPECT_EQ(row_splits.Dim(), num_rows + 1);
    int32_t *row_splits_data = row_splits.Data();
    RowIdsToRowSplits(context, num_elements, row_ids.Data(), false, num_rows,
                      row_splits_data);
    // copy data from CPU/GPU to CPU
    Array1<int32_t> cpu_array = row_splits.To(cpu);
    std::vector<int32_t> cpu_data(cpu_array.Data(),
                                  cpu_array.Data() + cpu_array.Dim());
    EXPECT_EQ(cpu_data, row_splits_vec);
  }
}

TEST(UtilsTest, RowIdsToRowSplits) {
  TestRowIdsToRowSplits<kCpu>();
  TestRowIdsToRowSplits<kCuda>();
}
}  // namespace k2
