/**
 * Copyright      2022  Xiaomi Corporation (authors: Fangjun Kuang)
 *
 * See LICENSE for clarification regarding multiple authors
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "k2/csrc/ragged_ops.h"
#include "k2/csrc/torch_api.h"
#include "k2/csrc/torch_util.h"

namespace k2 {

void ExclusiveSum(torch::Tensor src, torch::Tensor *dst) {
  Array1<int32_t> src_arr = FromTorch<int32_t>(src);
  Array1<int32_t> dst_arr = FromTorch<int32_t>(*dst);

  ExclusiveSum(src_arr, &dst_arr);
}

RaggedShapePtr RaggedShape2(torch::Tensor row_splits, torch::Tensor row_ids,
                            int32_t cached_tot_size /*=-1*/) {
  if (!row_splits.defined()) {
    K2_CHECK(row_ids.defined())
        << "You have to provide row_ids if row_splits is empty";
  }

  Array1<int32_t> row_splits_arr, row_ids_arr;

  if (row_splits.defined()) {
    row_splits_arr = FromTorch<int32_t>(row_splits);
  }

  if (row_ids.defined()) {
    row_ids_arr = FromTorch<int32_t>(row_ids);
  }

  return std::make_shared<RaggedShape>(RaggedShape2(
      row_splits.defined() ? &row_splits_arr : nullptr,
      row_ids.defined() ? &row_ids_arr : nullptr, cached_tot_size));
}

int32_t TotSize(RaggedShapePtr shape, int32_t axis) {
  return shape->TotSize(axis);
}

torch::Tensor RowIds(RaggedShapePtr shape, int32_t axis) {
  return ToTorch(shape->RowIds(axis));
}

torch::Tensor RowSplits(RaggedShapePtr shape, int32_t axis) {
  return ToTorch(shape->RowSplits(axis));
}

}  // namespace k2
