/**
 * Copyright      2022  Xiaomi Corporation (authors: Wei Kang)
 *
 * See LICENSE for clarification regarding multiple authors
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gtest/gtest.h"
#include "k2/csrc/test_utils.h"
//
#include "k2/csrc/array.h"
#include "k2/csrc/device_guard.h"
#include "k2/csrc/context.h"

namespace k2 {

// Use a separate function because there is a lambda function inside K2_EVAL().
static void TestImpl() {
  int num_devices;
  auto ret = hipGetDeviceCount(&num_devices);
  K2_LOG(INFO) << "Number of devices: " << num_devices;

  // Set the default device to 1
  ret = hipSetDevice(1);
  K2_CHECK_CUDA_ERROR(ret);

  int current_device;
  ret = hipGetDevice(&current_device);
  K2_CHECK_CUDA_ERROR(ret);
  EXPECT_EQ(current_device, 1);

  ContextPtr c = GetCudaContext(0);
  EXPECT_EQ(c->GetDeviceId(), 0);

  // Test zero byte allocation.
  {
    std::vector<int32_t> data;
    Array1<int32_t> src(c, data);
    EXPECT_EQ(src.Dim(), 0);
  }

  // the default device should still be 1
  ret = hipGetDevice(&current_device);
  K2_CHECK_CUDA_ERROR(ret);
  EXPECT_EQ(current_device, 1);

  Array1<int32_t> a(c, "[1 2]");
  EXPECT_EQ(a.Context()->GetDeviceId(), 0);

  // b uses the default device, which is 1
  Array1<int32_t> b(GetCudaContext(), "[10 20]");
  EXPECT_EQ(b.Context()->GetDeviceId(), 1);

  int32_t *a_data = a.Data();
  int32_t *b_data = b.Data();

  {
    DeviceGuard guard(0);
    // a is on device 0
    K2_EVAL(
        a.Context(), 2, set_a, (int32_t i)->void { a_data[i] += 1; });
    CheckArrayData(a, {2, 3});
  }

  {
    DeviceGuard guard(1);
    // b is on device 1
    K2_EVAL(
        b.Context(), 2, set_b, (int32_t i)->void { b_data[i] += 2; });

    CheckArrayData(b, {12, 22});
  }
}


TEST(DefaultContext, GetCudaContext) {
  // skip this test is CUDA is not available
  int n;
  auto ret = hipGetDeviceCount(&n);
  if (ret == hipSuccess && n > 1) {
    TestImpl();
  }
}

}  // namespace k2
