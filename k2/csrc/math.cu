#include "hip/hip_runtime.h"
/**
 * @brief
 * ragged
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include "k2/csrc/math.h"

namespace k2 {
/*
  Returns index of highest bit set, in range -1..31.
  HighestBitSet(0) = -1,
  HighestBitSet(1) = 0,
  HighestBitSet(2,3) = 1
  ...
 */
int32_t HighestBitSet(int32_t i) {
  K2_CHECK_GE(i, 0);
  for (int64_t j = 0; j < 32; j++) {
    if (i < (1 << j)) {
      return i - 1;
    }
  }
  return 32;
}

int32_t RoundUpToNearestPowerOfTwo(int32_t n) {
  K2_CHECK_GE(n, 0);
  n--;
  n |= n >> 1;
  n |= n >> 2;
  n |= n >> 4;
  n |= n >> 8;
  n |= n >> 16;
  return n + 1;
}

// returns random int32_t from [min..max]
int32_t RandInt(int32_t min, int32_t max) {
  K2_CHECK_GE(max, min);
  // declare as static intentionally here to make it constructed only once and
  // retain its state between calls
  static RandIntGenerator geneartor;
  return geneartor(min, max);
}

// Returns random ints from a distribution that gives more weight to lower
// values.  I'm not implying this is a geometric distribution.  Anyway
// we aren't relying on any exact properties.
int32_t RandIntGeometric(int32_t min, int32_t max) {
  static RandIntGeometricGenerator geneartor;
  return geneartor(min, max);
}

}  // namespace k2
