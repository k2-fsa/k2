#include "hip/hip_runtime.h"
/**
 * @brief
 * ragged
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include "k2/csrc/macros.h"
#include "k2/csrc/math.h"
#include "k2/csrc/nvtx.h"

namespace k2 {
/*
  Returns index of highest bit set, in range -1..31.
  HighestBitSet(0) = -1,
  HighestBitSet(1) = 0,
  HighestBitSet(2,3) = 1
  ...
 */
int32_t HighestBitSet(int32_t i) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_GE(i, 0);
  for (int64_t j = 0; j < 32; j++) {
    if (i < (1 << j)) {
      return i - 1;
    }
  }
  return 32;
}

int32_t RoundUpToNearestPowerOfTwo(int32_t n) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_GE(n, 0);
  n--;
  n |= n >> 1;
  n |= n >> 2;
  n |= n >> 4;
  n |= n >> 8;
  n |= n >> 16;
  return n + 1;
}

// returns random int32_t from [min..max]
int32_t RandInt(int32_t min, int32_t max) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_GE(max, min);
  // declare as static intentionally here to make it constructed only once and
  // retain its state between calls
  static RandIntGenerator geneartor;
  return geneartor(min, max);
}

// Returns random ints from a distribution that gives more weight to lower
// values.  I'm not implying this is a geometric distribution.  Anyway
// we aren't relying on any exact properties.
int32_t RandIntGeometric(int32_t min, int32_t max) {
  NVTX_RANGE(K2_FUNC);
  static RandIntGeometricGenerator geneartor;
  return geneartor(min, max);
}

namespace internal {
template <typename Real>
Real FixedRead(std::istream &is) {
  NVTX_RANGE(K2_FUNC);
  is >> std::ws;
  char c = is.peek();
  if (c == '-') {
    is.get();
    return -FixedRead<Real>(is);
  } else if (c == 'i' || c == 'I') {
    char c[10];
    int pos = 0;
    while (pos < 9 && isalpha(is.peek())) c[pos++] = tolower(is.get());
    c[pos] = '\0';
    if (strcmp(c, "inf") && strcmp(c, "infinity"))
      is.setstate(std::ios::failbit);
    return std::numeric_limits<Real>::infinity();
    // can handle NaN's later, with:
    //} else if (c == 'n' || c == 'N') {
    // (NaN's are printed in a more complicated way though.
  } else {
    Real r;
    is >> r;
    return r;
  }
}
// Instantiate the template above.
template float FixedRead(std::istream &is);
template double FixedRead(std::istream &is);
}  // namespace internal

}  // namespace k2
