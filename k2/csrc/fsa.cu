// k2/csrc/cuda/fsa.cu

// Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey
//                                                   Haowen Qiu)

// See ../../LICENSE for // clarification regarding multiple authors

#include "k2/csrc/array_ops.h"
#include "k2/csrc/fsa.h"

namespace {
/* Will be used in FsaVecFromTensor to call ExclusiveSum (which calls
   hipcub::DeviceScan::ExclusiveSum internally).

   This is a kind of pseudo-vector (that we don't have to allocate
   memory for) It behaves like a pointer to a vector of size
   `num_arcs`, of 'tails' (see `tails concept` in utils.h) which
   tells us if this is the last arc within this FSA.
 */
struct IsLastArcOfFsa {
  int32_t num_arcs;
  const k2::Arc *arcs;

  explicit IsLastArcOfFsa(int32_t num_arcs, const k2::Arc *arcs)
      : num_arcs(num_arcs), arcs(arcs) {}
  __host__ __device__ IsLastArcOfFsa(const IsLastArcOfFsa &other)
      : num_arcs(num_arcs), arcs(other.arcs)  {}

  // operator[] and operator+ are required by hipcub::DeviceScan::ExclusiveSum
  __host__ __device__ bool operator[](int32_t i) const {
    return (i + 1 >= num_arcs || arcs[i + 1].src_state < arcs[i].src_state);
  }
  __host__ __device__ IsLastArcOfFsa operator+(int32_t n) const {
    IsLastArcOfFsa tmp(*this);
    tmp.arcs += n;
    return tmp;
  }
};




}  // namespace

namespace std {
// value_type is required by hipcub::DeviceScan::ExclusiveSum
template <>
struct iterator_traits<::IsLastArcOfFsa> {
  typedef bool value_type;
};
}  // namespace std

namespace k2 {

int32_t GetFsaVecBasicProperties(FsaVec &fsa_vec) {
  if (fsa_vec.NumAxes() != 3)
    return 0;
  ContextPtr c = fsa_vec.Context();
  const int32_t *row_ids1_data = fsa_vec.shape.RowIds(1).Data(),
                *row_splits1_data = fsa_vec.shape.RowSplits(1).Data(),
                *row_ids2_data = fsa_vec.shape.RowIds(2).Data(),
                *row_splits2_data = fsa_vec.shape.RowSplits(2).Data();
  Arc *arcs_data = fsa_vec.values.Data();

  int32_t num_arcs = fsa_vec.values.Dim();

  // `neg_` means negated. It's more convenient to do it this way.
  Array1<int32_t> neg_properties(c, num_arcs);
  int32_t num_states = fsa_vec.shape.RowIds(1).Dim(),
          num_fsas = fsa_vec.shape.Dim0();

  // `reachable[idx01]` will be true if the state with index idx01 has an arc
  // entering it or is state 0 of its FSA, not counting self-loops; it's a
  // looser condition than being 'accessible' in FSA terminlogy, simply meaning
  // it's reachable from some state (which might not itself be reachable).
  //
  // reachable[num_states + idx01] will be true if the state with index idx01 is
  // the final-state of its FSA (i.e. last-numbered) or has at least one arc
  // leaving it, not counting self-loops. Again, it's a looser condition than
  // being 'co-accessible' in FSA terminology.
  Array1<char> reachable(c, num_states * 2 + 1, static_cast<char>(0));
  Array1<char> flag = reachable.Range(num_states * 1, 1);
  Array1<char> co_reachable = reachable.Range(num_states, num_states);
  reachable = reachable.Range(0, num_states);
  int32_t *neg_properties_data = neg_properties.Data();
  char *reachable_data = reachable.Data();  // access co_reachable via this.

  auto lambda_get_properties = [=] __host__ __device__(int32_t idx012) -> void {
    Arc arc = arcs_data[idx012];
    Arc prev_arc;
    if (idx012 > 0) prev_arc = arcs_data[idx012 - 1];
    int32_t idx01 = row_ids2_data[idx012], idx01x = row_splits2_data[idx012],
            idx2 = idx012 - idx01x, idx0 = row_ids1_data[idx01],
            idx0x = row_splits1_data[idx0],
            idx0x_next = row_splits1_data[idx0 + 1], idx1 = idx01 - idx0x,
            idx0xx = row_splits2_data[idx0x];
    int32_t this_fsa_num_states = idx0x_next - idx0x;

    int32_t neg_property = 0;
    if (arc.src_state != idx1) neg_property |= kFsaPropertiesValid;
    if (arc.dest_state <= arc.src_state) {
      neg_property |= kFsaPropertiesTopSortedAndAcyclic;
      if (arc.dest_state < arc.src_state)
        neg_property |= kFsaPropertiesTopSorted;
    }
    if (arc.symbol == 0) neg_property |= kFsaPropertiesEpsilonFree;
    if (arc.symbol < 0) {
      if (arc.symbol != -1) {  // neg. symbols != -1 are not allowed.
        neg_property |= kFsaPropertiesValid;
      } else {
        if (arc.dest_state != this_fsa_num_states - 1)
          neg_property |= kFsaPropertiesValid;
      }
    }
    if (arc.symbol != -1 && arc.dest_state == this_fsa_num_states - 1)
      neg_property |= kFsaPropertiesValid;
    if (arc.dest_state < 0 || arc.dest_state >= this_fsa_num_states)
      neg_property |= kFsaPropertiesValid;
    else if (arc.dest_state != arc.src_state)
      reachable_data[idx0x + arc.dest_state] = static_cast<char>(1);

    if (idx0xx == idx012) {
      // first arc in this FSA (whether or not it's from state 0..)
      reachable_data[idx0x] = static_cast<char>(1);  // state 0 is reachable.
      // final state is always co-reachable.
      reachable_data[num_states + idx0x_next - 1] = static_cast<char>(1);
      // there was a problem with the state-indexes which makes this
      // impossible to deserialize from a list of arcs.
      if (idx012 > 0 && prev_arc.src_state <= arc.src_state)
        neg_property |= kFsaPropertiesSerializable;
    }

    if (idx2 == 0) {
      // First arc leaving this state record that this state has arcs leaving
      // it.
      if (arc.dest_state != arc.src_state)
        reachable_data[num_states + idx01] = 1;
    }
    neg_properties_data[idx012] = ~neg_property;
  };
  Eval(c, num_arcs, lambda_get_properties);
  // Note: eventually, for more diagnostics, we could use AndPerSublist to get
  // the properties one by  one.

  Array1<int32_t> and_properties = neg_properties.Range(0, 1);  // ok to overlap
  Array1<char> and_reachable = reachable.Range(0, 1);
  Array1<char> and_co_reachable = reachable.Range(0, 1);
  ParallelRunner pr(c);
  {
    With(pr.NewStream());
    And(neg_properties, static_cast<int32_t>(kFsaAllProperties),
        &and_properties);
  }
  {
    With(pr.NewStream());
    And(reachable, static_cast<char>(1), &and_reachable);
  }
  {
    With(pr.NewStream());
    And(co_reachable, static_cast<char>(1), &and_co_reachable);
  }
  {
    char *flag_data = flag.Data();
    auto lambda_find_empty_fsas = [=] __host__ __device__(int32_t i) -> void {
      if (row_ids1_data[i + 1] == row_ids1_data[i])
        *flag_data = 1;  // There is an empty FSA.
    };
    Eval(pr.NewStream(), num_fsas, lambda_find_empty_fsas);
  }

  int32_t properties =
      and_properties[0] |
      (and_reachable[0] ? kFsaPropertiesMaybeAccessible : 0) |
      (and_co_reachable[0] ? kFsaPropertiesMaybeCoaccessible : 0);
  if (flag[0])  // not serializable because has empty FSA.
    properties &= ~kFsaPropertiesSerializable;

  // probably tons of bugs in this :-(
  return properties;
}

FsaVec FsaVecFromFsa(const Fsa &fsa) {
  ContextPtr c = fsa.values.Context();
  K2_CHECK(fsa.NumAxes() == 2);
  RaggedShape first_axis = TrivialShape(c, fsa.shape.Dim0());
  RaggedShape fsa_vec_shape = ComposeRaggedShapes(first_axis, fsa.shape);
  return Ragged<Arc>(fsa_vec_shape, fsa.values);
}

int32_t GetFsaBasicProperties(const Fsa &fsa) {
  if (fsa.NumAxes() != 2)
    return 0;
  FsaVec vec = FsaVecFromFsa(fsa);
  return GetFsaVecBasicProperties(vec);
}


Fsa FsaFromArray1(Array1<Arc> &array, bool *error) {
  const Arc *arcs_data = reinterpret_cast<const Arc *>(array.Data());
  ContextPtr c = array.Context();
  const int32_t num_arcs = array.Dim();


  // If the FSA has arcs entering the final state, that will
  // tell us what the final-state id is.
  // If there are no arcs entering the final-state, we let the final state be
  // (highest numbered state that has arcs leaving it) + 1, so num_states
  // (highest numbered state that has arcs leaving it) + 2.

  // element 0 is num-states, element is error flag that's set to
  // 0 on error.

  Array1<int32_t> num_states_array(c, 2, -1);
  int32_t *num_states_data = num_states_array.Data();


  Array1<int32_t> row_ids1(c, num_arcs);  // maps arc->state.
  int32_t *row_ids1_data = row_ids1.Data();

  auto lambda_misc = [=] __host__ __device__(int32_t i) -> void {
    row_ids1_data[i] = arcs_data[i].src_state;
    if (arcs_data[i].symbol == -1) {
      int32_t final_state = arcs_data[i].dest_state;
      int32_t old_value = num_states_data[0];
      if (old_value >= 0 && old_value != final_state + 1)
        num_states_data[1] = 0;  // set error flag.
      num_states_data[0] = final_state + 1;
    }
  };
  Eval(c, num_arcs, lambda_misc);
  num_states_array = num_states_array.To(GetCpuContext());
  int32_t num_states = num_states_array[0],
      error_flag = num_states_array[1];
  if (error_flag == 0) {
    K2_LOG(WARNING)
        << "Could not convert tensor to FSA, there was a problem "
           "working out the num-states in the FSA, num_states="
        << num_states;
    *error = true;
    return Fsa();
  }

  if (!ValidateRowIds(row_ids1)) {
    K2_LOG(WARNING)
        << "Could not convert tensor to FSA, src_states of arcs were out of "
        "order";
    *error = true;
    return Fsa();
  }
  Array1<int32_t> row_splits1(c, num_states + 1);
  RowIdsToRowSplits(c, num_arcs, row_ids1_data, false, num_states,
                    row_splits1.Data());
#ifndef NDEBUG
  if (!ValidateRowSplitsAndIds(
          row_splits1, row_ids1, NULL)) {
    K2_LOG(FATAL) << "Failure validating row-splits/row-ids, likely code error";
  }
#endif

  RaggedShape fsas_shape = RaggedShape2(&row_splits1, &row_ids1,
                                        row_ids1.Dim());
  FsaVec ans = Ragged<Arc>(fsas_shape, array);
  int32_t properties = GetFsaVecBasicProperties(ans);
  // TODO: check properties, at least
  int32_t required_props = (kFsaPropertiesValid | kFsaPropertiesNonempty |
                            kFsaPropertiesSerializable);
  if (properties & required_props) {
    K2_LOG(WARNING) << "Did not have expected properties "
                    << (properties & required_props) << " vs. "
                    << required_props;
    // TODO: better way of displaying properties.
    *error = true;
  }
  return ans;
}

Fsa FsaFromTensor(Tensor &t, bool *error) {
  if (!t.IsContiguous())
    t = ToContiguous(t);

  *error = false;
  if (t.GetDtype() != kInt32Dtype) {
    K2_LOG(WARNING) << "Could not convert tensor to FSA, wrong dtype, got "
                    << TraitsOf(t.GetDtype()).Name() << " but expected "
                    << TraitsOf(kInt32Dtype).Name();
    *error = true;
    return Fsa();  // Invalid, empty FSA
  }
  if (t.NumAxes() != 2 || t.Dim(1) != 4) {
    K2_LOG(WARNING) << "Could not convert tensor to FSA, shape was "
                    << t.Dims();
  }
  K2_CHECK_EQ(sizeof(Arc), sizeof(int32_t) * 4);
  int32_t *tensor_data = t.Data<int32_t>();

  Array1<Arc> arc_array(t.Dim(0), t.GetRegion(), t.ByteOffset());
  return FsaFromArray1(arc_array, error);
}


Fsa FsaVecFromArray1(Array1<Arc> &array, bool *error) {
  const Arc *arcs_data = reinterpret_cast<const Arc *>(array.Data());
  ContextPtr c = array.Context();
  const int32_t num_arcs = array.Dim();
  Array1<int32_t> row_ids12(c, num_arcs + 1);  // maps arc->fsa_id, like
                                               // row_ids1[row_ids2]
  IsLastArcOfFsa fsa_tails(num_arcs, arcs_data);
  ExclusiveSum(c, num_arcs + 1, fsa_tails, row_ids12.Data());
  int32_t num_fsas = row_ids12[num_arcs];
  row_ids12 = row_ids12.Range(0, num_arcs);

  int32_t *fsa_ids_data = row_ids12.Data();

  // Get the num-states per FSA, including the final-state which must be
  // numbered last.  If the FSA has arcs entering the final state, that will
  // tell us what the final-state id is.  (that goes in num_states_per_fsa).
  // If there are no arcs entering the final-state, we let the final state be
  // (highest numbered state that has arcs leaving it) + 1, so num_states
  // (highest numbered state that has arcs leaving it) + 2.
  //
  // num_states_per_fsa[num_fsas] is an error flag that gets set to 0 on error.
  Array1<int32_t> num_states_per_fsa(c, num_fsas + 1, -1);
  int32_t *num_states_per_fsa_data = num_states_per_fsa.Data();
  auto lambda_get_num_states_a = [=] __host__ __device__(int32_t i) -> void {
    if (arcs_data[i].symbol == -1) {
      int32_t final_state = arcs_data[i].dest_state, fsa_id = fsa_ids_data[i];
      num_states_per_fsa_data[fsa_id] = final_state + 1;
    }
  };
  Eval(c, num_arcs, lambda_get_num_states_a);

  Array1<int32_t> row_splits12(c, num_fsas + 1);
  int32_t * row_splits12_data = row_splits12.Data();
  RowIdsToRowSplits(c, num_arcs, row_ids12.Data(), true,
                    num_fsas, row_splits12.Data());
  auto lambda_get_num_states_b = [=] __host__ __device__(int32_t i) -> void {
     int32_t num_states_1 = num_states_per_fsa_data[i],
         num_states_2 = arcs_data[row_splits12_data[i+1] - 1].src_state + 2;
    if (num_states_2 <= 0 ||
        (num_states_1 >= 0 && num_states_2 > num_states_1)) {
      // Note: num_states_2 is a lower bound on the final-state, something is
      // wrong if num_states_1 != -1 and num_states_2  is greater than
      // num_states_1.
      num_states_per_fsa_data[2 * num_fsas] = 0;  // Error
    } else {
      int32_t num_states = (num_states_1 < 0 ? num_states_2 : num_states_1);
      num_states_per_fsa_data[i] = num_states;
    }
  };
  Eval(c, num_arcs, lambda_get_num_states_b);
  if (num_states_per_fsa[2 * num_fsas] == 0) {
    K2_LOG(WARNING)
        << "Could not convert tensor to FSAs, there was a problem "
           "working out the num-states in the FSAs, num_states_per_fsa="
        << num_states_per_fsa;
    *error = true;
    return Fsa();
  }
  num_states_per_fsa = num_states_per_fsa.Range(0, num_fsas + 1);
  // row_splits1 is of size num_fsas + 1.
  // TODO(dan): make this in-place?
  Array1<int32_t> row_splits1 = ExclusiveSum(num_states_per_fsa);
  int32_t tot_num_states = row_splits1[num_fsas];

  const int32_t *row_splits1_data = row_splits1.Data();

  // by `row_ids2` we mean row_ids for axis=2. This is the second
  // of two row_ids vectors. It maps from idx012 to idx01.
  Array1<int32_t> row_ids2(c, num_arcs);
  int32_t *row_ids2_data = row_ids2.Data();
  auto lambda_set_row_ids2 = [=] __host__ __device__(int32_t i) -> void {
    int32_t src_state = arcs_data[i].src_state, fsa_id = fsa_ids_data[i];
    row_ids2_data[i] = row_splits1_data[fsa_id] + src_state;
  };
  Eval(c, num_arcs, lambda_set_row_ids2);

  if (!ValidateRowIds(row_ids2)) {
    K2_LOG(WARNING)
        << "Could not convert tensor to FSA, src_states of arcs were out of "
        "order";
    *error = true;
    return Fsa();
  }

  Array1<int32_t> row_splits2(c, tot_num_states + 1);
  RowIdsToRowSplits(c, num_arcs, row_ids2_data, false, tot_num_states,
                    row_splits2.Data());
#ifndef NDEBUG
  if (!ValidateRowSplitsAndIds(
          row_splits2, row_ids2,
          &num_states_per_fsa)) {  // last arg is temp space
    K2_LOG(FATAL) << "Failure validating row-splits/row-ids, likely code error";
  }
#endif

  // row_ids1 maps from idx01 to idx0.
  // row_ids12 maps from idx012 to idx0.  row_splits2 maps from idx01 to idx012.
  Array1<int32_t> row_ids1 = row_ids12[row_splits2];

#ifndef NDEBUG
  if (!ValidateRowSplitsAndIds(
          row_splits1, row_ids1,
          &num_states_per_fsa)) {  // last arg is temp space
    K2_LOG(FATAL) << "Failure validating row-splits/row-ids, likely code error";
  }
#endif


  RaggedShape fsas_shape =
      RaggedShape3(&row_splits1, &row_ids1, row_ids1.Dim(),
                   &row_splits2, &row_ids2, row_ids2.Dim());
  FsaVec ans = Ragged<Arc>(fsas_shape, array);
  int32_t properties = GetFsaVecBasicProperties(ans);
  // TODO: check properties, at least
  int32_t required_props = (kFsaPropertiesValid | kFsaPropertiesNonempty |
                            kFsaPropertiesSerializable);
  if (properties & required_props) {
    K2_LOG(WARNING) << "Did not have expected properties "
                    << (properties & required_props) << " vs. "
                    << required_props;
    // TODO: better way of displaying properties.
    *error = true;
  }
  return ans;
}



FsaVec FsaVecFromTensor(Tensor &t, bool *error) {
  if (!t.IsContiguous())
    t = ToContiguous(t);

  *error = false;
  if (t.GetDtype() != kInt32Dtype) {
    K2_LOG(WARNING) << "Could not convert tensor to FSA, wrong dtype, got "
                    << TraitsOf(t.GetDtype()).Name() << " but expected "
                    << TraitsOf(kInt32Dtype).Name();
    *error = true;
    return Fsa();  // Invalid, empty FSA
  }
  if (t.NumAxes() != 2 || t.Dim(1) != 4) {
    K2_LOG(WARNING) << "Could not convert tensor to FSA, shape was "
                    << t.Dims();
  }
  K2_CHECK_EQ(sizeof(Arc), sizeof(int32_t) * 4);
  int32_t *tensor_data = t.Data<int32_t>();

  Array1<Arc> arc_array(t.Dim(0), t.GetRegion(), t.ByteOffset());
  return FsaVecFromArray1(arc_array, error);
}

}  // namespace k2
