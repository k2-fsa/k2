#include "hip/hip_runtime.h"
/**
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey)
 *
 * See LICENSE for clarification regarding multiple authors
 */

#include <algorithm>
#include <limits>
#include <vector>

#include "k2/csrc/array_ops.h"
#include "k2/csrc/device_guard.h"
#include "k2/csrc/fsa_algo.h"
#include "k2/csrc/fsa_utils.h"
#include "k2/csrc/hash.h"
#include "k2/csrc/ragged_ops.h"
#include "k2/csrc/thread_pool.h"

namespace k2 {

namespace intersect_pruned_internal {

/* Information associated with a state active on a particular frame..  */
struct StateInfo {
  /* abs_state_id is the state-index in a_fsas_.  Note: the ind0 in here
     won't necessarily match the ind0 within FrameInfo::state if
     a_fsas_stride_ == 0. */
  int32_t a_fsas_state_idx01;

  /* Caution: this is ACTUALLY A FLOAT that has been bit-twiddled using
     FloatToOrderedInt/OrderedIntToFloat so we can use atomic max.  It
     represents a Viterbi-style 'forward probability'.  (Viterbi, meaning: we
     use max not log-sum).  You can take the pruned lattice and rescore it if
     you want log-sum.  */
  int32_t forward_loglike;

  /* Note: this `backward_loglike` is the best score of any path from here to
     the end, minus the best path in the overall FSA, i.e. it's the backward
     score you get if, at the final-state, you set backward_loglike ==
     -forward_loglike. So backward_loglike + OrderedIntToFloat(forward_loglike)
     <= 0, and you can treat it somewhat like a posterior (except they don't sum
     to one as we're using max, not log-add).
  */
  float backward_loglike;
};

struct ArcInfo {              // for an arc that wasn't pruned away...
  int32_t a_fsas_arc_idx012;  // the arc-index in a_fsas_.
  float arc_loglike;          // loglike on this arc: equals loglike from data
  // (nnet output, == b_fsas), plus loglike from
  // the arc in a_fsas.

  union {
    // these 2 different ways of storing the index of the destination state
    // are used at different stages of the algorithm; we give them different
    // names for clarity.
    int32_t dest_a_fsas_state_idx01;  // The destination-state as an index
    // into a_fsas_.
    int32_t dest_info_state_idx1;  // The destination-state as an idx1 into the
                                   // next FrameInfo's `arcs` or `states`,
                                   // omitting the FSA-index which can be worked
                                   // out from the structure of this frame's
                                   // ArcInfo.
  } u;
  float end_loglike;  // loglike at the end of the arc just before
  // (conceptually) it joins the destination state.
};

/*
static std::ostream &operator<<(std::ostream &os, const StateInfo &s) {
  os << "StateInfo{" << s.a_fsas_state_idx01 << ","
     << OrderedIntToFloat(s.forward_loglike) << "," << s.backward_loglike
     << "}";
  return os;
}

static std::ostream &operator<<(std::ostream &os, const ArcInfo &a) {
  os << "ArcInfo{" << a.a_fsas_arc_idx012 << "," << a.arc_loglike << ","
     << a.u.dest_a_fsas_state_idx01 << "," << a.end_loglike
     << "[i=" << FloatToOrderedInt(a.end_loglike) << "]"
     << "}";
  return os;
}
*/

}  // namespace intersect_pruned_internal

using namespace intersect_pruned_internal;  // NOLINT

/*
   Pruned intersection (a.k.a. composition) that corresponds to decoding for
   speech recognition-type tasks.  Can use either different decoding graphs (one
   per acoustic sequence) or a shared graph
*/
class MultiGraphDenseIntersectPruned {
 public:
  /**
     Pruned intersection (a.k.a. composition) that corresponds to decoding for
     speech recognition-type tasks

       @param [in] a_fsas  The decoding graphs, one per sequence.  E.g. might
                           just be a linear sequence of phones, or might be
                           something more complicated.  Must have either the
                           same Dim0() as b_fsas, or Dim0()==1 in which
                           case the graph is shared.
       @param [in] b_fsas  The neural-net output, with each frame containing the
                           log-likes of each phone.  A series of sequences of
                           (in general) different length.
       @param [in] search_beam    "Default" search/decoding beam.  The actual
                           beam is dynamic and also depends on max_active and
                           min_active.
       @param [in] output_beam    Beam for pruning the output FSA, will
                                  typically be smaller than search_beam.
       @param [in] min_active  Minimum number of FSA states that are allowed to
                           be active on any given frame for any given
                           intersection/composition task. This is advisory,
                           in that it will try not to have fewer than this
                           number active.
       @param [in] max_active  Maximum number of FSA states that are allowed to
                           be active on any given frame for any given
                           intersection/composition task. This is advisory,
                           in that it will try not to exceed that but may not
                           always succeed.  This determines the hash size.
   */
  MultiGraphDenseIntersectPruned(FsaVec &a_fsas, DenseFsaVec &b_fsas,
                                 float search_beam, float output_beam,
                                 int32_t min_active, int32_t max_active)
      : a_fsas_(a_fsas),
        b_fsas_(b_fsas),
        search_beam_(search_beam),
        output_beam_(output_beam),
        min_active_(min_active),
        max_active_(max_active),
        dynamic_beams_(a_fsas.Context(), b_fsas.shape.Dim0(), search_beam),
        forward_semaphore_(1) {
    NVTX_RANGE(K2_FUNC);
    c_ = GetContext(a_fsas.shape, b_fsas.shape);
    T_ = b_fsas_.shape.MaxSize(1);
    K2_CHECK_GT(search_beam, 0);
    K2_CHECK_GT(output_beam, 0);
    K2_CHECK_GE(min_active, 0);
    K2_CHECK_GT(max_active, min_active);
    K2_CHECK(a_fsas.shape.Dim0() == b_fsas.shape.Dim0() ||
             a_fsas.shape.Dim0() == 1);
    K2_CHECK_GE(b_fsas.shape.Dim0(), 1);
    int32_t num_seqs = b_fsas.shape.Dim0();

    int32_t num_buckets = RoundUpToNearestPowerOfTwo(num_seqs * 4 *
                                                     max_active);
    if (num_buckets < 128)
      num_buckets = 128;
    int32_t num_a_copies;
    if (a_fsas.shape.Dim0() == 1) {
      a_fsas_stride_ = 0;
      state_map_fsa_stride_ = a_fsas.TotSize(1);
      num_a_copies = b_fsas.shape.Dim0();
    } else {
      K2_CHECK_EQ(a_fsas.shape.Dim0(), b_fsas.shape.Dim0());
      a_fsas_stride_ = 1;
      state_map_fsa_stride_ = 0;
      num_a_copies = 1;
    }
    // +1, because all-ones is not a valid key.
    int64_t num_keys = num_a_copies * (int64_t)a_fsas.TotSize(1) + 1;

    // To reduce the number of template instantiations, we limit the
    // code to use either 32 or 36 or 40 bits.
    // 32 can be optimized in future so if the num_keys is less than
    // 1<<32, we favor that value.
    int32_t num_key_bits;
    if ((num_keys >> 32) == 0)
      num_key_bits = 32;
    else if ((num_keys >> 36) == 0)
      num_key_bits = 36;
    else {
      num_key_bits = 40;
      if ((num_keys >> 40) != 0) {
        K2_LOG(FATAL) << "Too many keys for hash, please extend this code "
            "with more options: num_keys=" << num_keys;
      }
    }
    state_map_ = Hash(c_, num_buckets, num_key_bits);


    { // set up do_pruning_after_ and prune_t_begin_end_.

      do_pruning_after_.resize(T_ + 1, (char)0);

      // each time we prune, prune 30 frames; but shift by 20 frames each
      // time so there are 10 frames of overlap.
      int32_t prune_num_frames = 30,
                   prune_shift = 20,
                             T = T_;
      K2_CHECK_GT(prune_num_frames, prune_shift);
      // The first begin_t is negative but will be rounded up to zero to get the
      // start of the range.  The motivation is: we don't want to wait until we
      // have processed `prune_num_frames` frames to prune for the first time,
      // because that first interval of not-pruning, being larger than normal,
      // would dominate the maximum memory used by intersection.
      for (int32_t begin_t = prune_shift - prune_num_frames; ;
           begin_t += prune_shift) {
        int32_t prune_begin = std::max<int32_t>(0, begin_t),
                  prune_end = begin_t + prune_num_frames;
        bool last = false;
        if (prune_end >= T) {
          prune_end = T;
          last = true;
        }
        K2_CHECK_LT(prune_begin, prune_end);
        do_pruning_after_[prune_end - 1] = (char)1;
        prune_t_begin_end_.push_back({prune_begin, prune_end});
        if (last)
          break;
      }
    }
  }

  // The information we have for each frame of the pruned-intersection (really:
  // decoding) algorithm.  We keep an array of these, one for each frame, up to
  // the length of the longest sequence we're decoding plus one.
  struct FrameInfo {
    // States that are active at the beginning of this frame.  Indexed
    // [fsa_idx][state_idx], where fsa_idx indexes b_fsas_ (and a_fsas_, if
    // a_fsas_stride_ != 0); and state_idx just enumerates the active states
    // on this frame (as state_idx01's in a_fsas_).
    Ragged<StateInfo> states;  // 2 axes: fsa, state

    // Indexed [fsa_idx][state_idx][arc_idx].. the first 2 indexes are
    // the same as those into 'states' (the first 2 levels of the structure
    // are shared), and the last one enumerates the arcs leaving each of those
    // states.
    //
    // Note: there may be indexes [fsa_idx] that have no states (because that
    // FSA had fewer frames than the max), and indexes [fsa_idx][state_idx] that
    // have no arcs due to pruning.
    Ragged<ArcInfo> arcs;  // 3 axes: fsa, state, arc
  };

  /* Does the main work of intersection/composition, but doesn't produce any
     output; the output is provided when you call FormatOutput(). */
  void Intersect() {
    /*
      T is the largest number of (frames+1) of neural net output, or the largest
      number of frames of log-likelihoods we count the final frame with (0,
      -inf, -inf..) that is used for the final-arc.  The largest number of
      states in the fsas represented by b_fsas equals T+1 (e.g. 1 frame would
      require 2 states, because that 1 frame is the arc from state 0 to state
      1).  So the #states is 2 greater than the actual number of frames in the
      neural-net output.
    */
    int32_t num_fsas = b_fsas_.shape.Dim0(), T = T_;

    std::ostringstream os;
    os << "Intersect:T=" << T << ",num_fsas=" << num_fsas
       << ",TotSize(1)=" << b_fsas_.shape.TotSize(1);
    NVTX_RANGE(os.str().c_str());

    ThreadPool* pool = GetThreadPool();
    pool->SubmitTask([this]() { BackwardPassStatic(this); });

    // we'll initially populate frames_[0.. T+1], but discard the one at T+1,
    // which has no arcs or states, the ones we use are from 0 to T.
    frames_.reserve(T + 2);

    frames_.push_back(InitialFrameInfo());

    for (int32_t t = 0; t <= T; t++) {
      if (state_map_.NumKeyBits() == 32) {
        frames_.push_back(PropagateForward<32>(t, frames_.back().get()));
      } else if (state_map_.NumKeyBits() == 36) {
        frames_.push_back(PropagateForward<36>(t, frames_.back().get()));
      } else {
        K2_CHECK_EQ(state_map_.NumKeyBits(), 40);
        frames_.push_back(PropagateForward<40>(t, frames_.back().get()));
      }
      if (do_pruning_after_[t]) {
        // let a phase of backward-pass pruning commence.
        backward_semaphore_.Signal(c_);
        // note: normally we should acquire forward_semaphore_ without having to
        // wait.  It avoids the backward pass getting too far behind the forward
        // pass, which could mean too much memory is used.
        forward_semaphore_.acquire();
      }
    }
    // The FrameInfo for time T+1 will have no states.  We did that
    // last PropagateForward so that the 'arcs' member of frames_[T]
    // is set up (it has no arcs but we need the shape).
    frames_.pop_back();

    pool->WaitAllTasksFinished();
  }

  void BackwardPass() {
    int32_t num_fsas = b_fsas_.shape.Dim0(),
      num_work_items = max_active_ * num_fsas * T_;
    ParallelRunner pr(c_);
    // if num_work_items is big enough, it will actually create a new stream.
    hipStream_t stream = pr.NewStream(num_work_items);
    With w(stream);  // This overrides whatever stream c_ contains with `stream`, if it's not


    NVTX_RANGE(K2_FUNC);
    for (size_t i = 0; i < prune_t_begin_end_.size(); i++) {
      backward_semaphore_.Wait(c_);
      int32_t prune_t_begin = prune_t_begin_end_[i].first,
                prune_t_end = prune_t_begin_end_[i].second;
      PruneTimeRange(prune_t_begin, prune_t_end);
      forward_semaphore_.release();
    }
  }

  static void BackwardPassStatic(MultiGraphDenseIntersectPruned *c) {
    // WARNING(fangjun): this is run in a separate thread, so we have
    // to reset its default device. Otherwise, it will throw later
    // if the main thread is using a different device.
    DeviceGuard guard(c->c_);
    c->BackwardPass();
  }

  // Return FrameInfo for 1st frame, with `states` set but `arcs` not set.
  std::unique_ptr<FrameInfo> InitialFrameInfo() {
    NVTX_RANGE("InitialFrameInfo");
    int32_t num_fsas = b_fsas_.shape.Dim0();
    std::unique_ptr<FrameInfo> ans = std::make_unique<FrameInfo>();

    if (a_fsas_.Dim0() == 1) {
      int32_t start_states_per_seq = (a_fsas_.shape.TotSize(1) > 0),  // 0 or 1
          num_start_states = num_fsas * start_states_per_seq;
      ans->states = Ragged<StateInfo>(
          RegularRaggedShape(c_, num_fsas, start_states_per_seq),
          Array1<StateInfo>(c_, num_start_states));
      StateInfo *states_data = ans->states.values.Data();
      K2_EVAL(
          c_, num_start_states, lambda_set_states, (int32_t i)->void {
            StateInfo info;
            info.a_fsas_state_idx01 = 0;  // start state of a_fsas_
            info.forward_loglike = FloatToOrderedInt(0.0);
            states_data[i] = info;
          });
    } else {
      Ragged<int32_t> start_states = GetStartStates(a_fsas_);
      ans->states =
          Ragged<StateInfo>(start_states.shape,
                            Array1<StateInfo>(c_, start_states.NumElements()));
      StateInfo *ans_states_values_data = ans->states.values.Data();
      const int32_t *start_states_values_data = start_states.values.Data();

      K2_EVAL(
          c_, start_states.NumElements(), lambda_set_state_info,
          (int32_t states_idx01)->void {
            StateInfo info;
            info.a_fsas_state_idx01 = start_states_values_data[states_idx01];
            info.forward_loglike = FloatToOrderedInt(0.0);
            ans_states_values_data[states_idx01] = info;
          });
    }
    return ans;
  }

  void FormatOutput(FsaVec *ofsa, Array1<int32_t> *arc_map_a,
                    Array1<int32_t> *arc_map_b) {
    NVTX_RANGE("FormatOutput");

    int32_t T = T_;


    ContextPtr c_cpu = GetCpuContext();
    Array1<ArcInfo *> arcs_data_ptrs(c_cpu, T + 1);
    Array1<int32_t *> arcs_row_splits1_ptrs(c_cpu, T + 1);
    for (int32_t t = 0; t <= T; t++) {
      arcs_data_ptrs.Data()[t] = frames_[t]->arcs.values.Data();
      arcs_row_splits1_ptrs.Data()[t] = frames_[t]->arcs.RowSplits(1).Data();
    }
    // transfer to GPU if we're using a GPU
    arcs_data_ptrs = arcs_data_ptrs.To(c_);
    ArcInfo **arcs_data_ptrs_data = arcs_data_ptrs.Data();
    arcs_row_splits1_ptrs = arcs_row_splits1_ptrs.To(c_);
    int32_t **arcs_row_splits1_ptrs_data = arcs_row_splits1_ptrs.Data();
    const int32_t *b_fsas_row_splits1 = b_fsas_.shape.RowSplits(1).Data();
    const int32_t *a_fsas_row_splits1 = a_fsas_.RowSplits(1).Data();
    int32_t a_fsas_stride = a_fsas_stride_;  // 0 or 1 depending if the decoding
                                             // graph is shared.
    int32_t num_fsas = b_fsas_.shape.Dim0();

    RaggedShape final_arcs_shape;
    { /*  This block populates `final_arcs_shape`.  It is the shape of a ragged
          tensor of arcs that conceptually would live at frames_[T+1]->arcs.  It
          contains no actual arcs, but may contain some states, that represent
          "missing" final-states.  The problem we are trying to solve is that
          there was a start-state for an FSA but no final-state because it did
          not survive pruning, and this could lead to an output FSA that is
          invalid or is misinterpreted (because we are interpreting a non-final
          state as a final state).
       */
      Array1<int32_t> num_extra_states(c_, num_fsas + 1);
      int32_t *num_extra_states_data = num_extra_states.Data();
      K2_EVAL(c_, num_fsas, lambda_set_num_extra_states, (int32_t i) -> void {
          int32_t final_t = b_fsas_row_splits1[i+1] - b_fsas_row_splits1[i];
          int32_t *arcs_row_splits1_data = arcs_row_splits1_ptrs_data[final_t];
          int32_t num_states_final_t = arcs_row_splits1_data[i + 1] -
                                       arcs_row_splits1_data[i];
          K2_CHECK_LE(num_states_final_t, 1);

          // has_start_state is 1 if there is a start-state; note, we don't prune
          // the start-states, so they'll be present if they were present in a_fsas_.
          int32_t has_start_state = (a_fsas_row_splits1[i * a_fsas_stride] <
                                     a_fsas_row_splits1[i * a_fsas_stride + 1]);

          // num_extra_states_data[i] will be 1 if there was a start state but no final-state;
          // else, 0.
          num_extra_states_data[i] = has_start_state * (1 - num_states_final_t);
        });
      ExclusiveSum(num_extra_states, &num_extra_states);

      RaggedShape top_shape = RaggedShape2(&num_extra_states, nullptr, -1),
               bottom_shape = RegularRaggedShape(c_, top_shape.NumElements(), 0);
      final_arcs_shape = ComposeRaggedShapes(top_shape, bottom_shape);
    }


    RaggedShape oshape;
    // see documentation of Stack() in ragged_ops.h for explanation.
    Array1<uint32_t> oshape_merge_map;

    {
      NVTX_RANGE("InitOshape");
      // each of these have 3 axes.
      std::vector<RaggedShape *> arcs_shapes(T + 2);
      for (int32_t t = 0; t <= T; t++)
        arcs_shapes[t] = &(frames_[t]->arcs.shape);
      arcs_shapes[T + 1] = &final_arcs_shape;

      // oshape is a 4-axis ragged tensor which is indexed:
      //   oshape[fsa_index][t][state_idx][arc_idx]
      int32_t axis = 1;
      oshape = Stack(axis, T + 2, arcs_shapes.data(), &oshape_merge_map);
    }


    int32_t *oshape_row_ids3 = oshape.RowIds(3).Data(),
            *oshape_row_ids2 = oshape.RowIds(2).Data(),
            *oshape_row_ids1 = oshape.RowIds(1).Data(),
            *oshape_row_splits3 = oshape.RowSplits(3).Data(),
            *oshape_row_splits2 = oshape.RowSplits(2).Data(),
            *oshape_row_splits1 = oshape.RowSplits(1).Data();


    int32_t num_arcs = oshape.NumElements();
    *arc_map_a = Array1<int32_t>(c_, num_arcs);
    *arc_map_b = Array1<int32_t>(c_, num_arcs);
    int32_t *arc_map_a_data = arc_map_a->Data(),
            *arc_map_b_data = arc_map_b->Data();
    Array1<Arc> arcs_out(c_, num_arcs);
    Arc *arcs_out_data = arcs_out.Data();
    const Arc *a_fsas_arcs = a_fsas_.values.Data();
    int32_t b_fsas_num_cols = b_fsas_.scores.Dim1();
    const int32_t *b_fsas_row_ids1 = b_fsas_.shape.RowIds(1).Data();

    const uint32_t *oshape_merge_map_data = oshape_merge_map.Data();

    K2_EVAL(
        c_, num_arcs, lambda_format_arc_data,
        (int32_t oarc_idx0123)->void {  // by 'oarc' we mean arc with shape `oshape`.
          int32_t oarc_idx012 = oshape_row_ids3[oarc_idx0123],
                   oarc_idx01 = oshape_row_ids2[oarc_idx012],
                    oarc_idx0 = oshape_row_ids1[oarc_idx01],
                   oarc_idx0x = oshape_row_splits1[oarc_idx0],
                  oarc_idx0xx = oshape_row_splits2[oarc_idx0x],
                    oarc_idx1 = oarc_idx01 - oarc_idx0x,
             oarc_idx01x_next = oshape_row_splits2[oarc_idx01 + 1];

          int32_t m = oshape_merge_map_data[oarc_idx0123],
                  t = m % (T + 2),  // actually we won't get t == T or t == T + 1
                                    // here since those frames have no arcs.
        arcs_idx012 = m / (T + 2);  // arc_idx012 into FrameInfo::arcs on time t,
                                    // index of the arc on that frame.

          K2_CHECK_EQ(t, oarc_idx1);

          const ArcInfo *arcs_data = arcs_data_ptrs_data[t];

          ArcInfo arc_info = arcs_data[arcs_idx012];
          Arc arc;
          arc.src_state = oarc_idx012 - oarc_idx0xx;
          // Note: the idx1 w.r.t. the frame's `arcs` is an idx2 w.r.t. `oshape`.
          int32_t dest_state_idx012 = oarc_idx01x_next +
                                      arc_info.u.dest_info_state_idx1;
          arc.dest_state = dest_state_idx012 - oarc_idx0xx;
          arc.label = a_fsas_arcs[arc_info.a_fsas_arc_idx012].label;

          int32_t fsa_id = oarc_idx0,
            b_fsas_idx0x = b_fsas_row_splits1[fsa_id],
            b_fsas_idx01 = b_fsas_idx0x + t,
             b_fsas_idx2 = (arc.label + 1),
       b_fsas_arc_idx012 = b_fsas_idx01 * b_fsas_num_cols + b_fsas_idx2;

          arc.score = arc_info.arc_loglike;
          arc_map_a_data[oarc_idx0123] = arc_info.a_fsas_arc_idx012;
          arc_map_b_data[oarc_idx0123] = b_fsas_arc_idx012;
          arcs_out_data[oarc_idx0123] = arc;
        });

    // Remove axis 1, which corresponds to time.
    *ofsa = FsaVec(RemoveAxis(oshape, 1), arcs_out);
  }

  /*
    Computes pruning cutoffs for this frame: these are the cutoffs for the arc
    "forward score", one per FSA.  This is a dynamic process involving
    dynamic_beams_ which are updated on each frame (they start off at
    search_beam_).

       @param [in] arc_end_scores  The "forward log-probs" (scores) at the
                    end of each arc, i.e. its contribution to the following
                    state.  Is a tensor indexed [fsa_id][state][arc]; we
                    will get rid of the [state] dim, combining it with the
                    [arc] dim, so it's just [fsa_id][arc]
                    It is conceptually unchanged by this operation but non-const
                    because row-ids of its shape may need to be generated.
       @return      Returns a vector of log-likelihood cutoffs, one per FSA (the
                    cutoff will be -infinity for FSAs that don't have any active
                    states).  The cutoffs will be of the form: the best score
                    for any arc, minus the dynamic beam.  See the code for how
                    the dynamic beam is adjusted; it will approach
                    'search_beam_' as long as the number of active states in
                    each FSA is between min_active and max_active.
  */
  Array1<float> GetPruningCutoffs(Ragged<float> &arc_end_scores) {
    NVTX_RANGE(K2_FUNC);
    int32_t num_fsas = arc_end_scores.shape.Dim0();

    // get the maximum score from each sub-list (i.e. each FSA, on this frame).
    // Note: can probably do this with a cub Reduce operation using an operator
    // that has side effects (that notices when it's operating across a
    // boundary).
    // the max will be -infinity for any FSA-id that doesn't have any active
    // states (e.g. because that stream has finished).
    // Casting to ragged2 just considers the top 2 indexes, ignoring the 3rd.
    // i.e. it's indexed by [fsa_id][arc].
    Ragged<float> end_scores_per_fsa = arc_end_scores.RemoveAxis(1);
    Array1<float> max_per_fsa(c_, end_scores_per_fsa.Dim0());
    MaxPerSublist(end_scores_per_fsa, -std::numeric_limits<float>::infinity(),
                  &max_per_fsa);
    const int32_t *arc_end_scores_row_splits1_data =
        arc_end_scores.RowSplits(1).Data();
    const float *max_per_fsa_data = max_per_fsa.Data();
    float *dynamic_beams_data = dynamic_beams_.Data();

    float default_beam = search_beam_, max_active = max_active_,
          min_active = min_active_;
    K2_CHECK_LT(min_active, max_active);

    Array1<float> cutoffs(c_, num_fsas);
    float *cutoffs_data = cutoffs.Data();

    K2_EVAL(
        c_, num_fsas, lambda_set_beam_and_cutoffs, (int32_t i)->void {
          float best_loglike = max_per_fsa_data[i],
                dynamic_beam = dynamic_beams_data[i];
          int32_t active_states = arc_end_scores_row_splits1_data[i + 1] -
                                  arc_end_scores_row_splits1_data[i];
          if (active_states <= max_active) {
            // Not constrained by max_active...
            if (active_states >= min_active || active_states == 0) {
              // Neither the max_active nor min_active constraints
              // apply.  Gradually approach 'beam'
              // (Also approach 'beam' if active_states == 0; we might as
              // well, since there is nothing to prune here).
              dynamic_beam = 0.8 * dynamic_beam + 0.2 * default_beam;
            } else {
              // We violated the min_active constraint -> increase beam
              if (dynamic_beam < default_beam) dynamic_beam = default_beam;
              // gradually make the beam larger as long
              // as we are below min_active
              dynamic_beam *= 1.25;
            }
          } else {
            // We violated the max_active constraint -> decrease beam
            if (dynamic_beam > default_beam) dynamic_beam = default_beam;
            // Decrease the beam as long as we have more than
            // max_active active states.
            dynamic_beam *= 0.8;
          }
          dynamic_beams_data[i] = dynamic_beam;
          cutoffs_data[i] = best_loglike - dynamic_beam;
        });

    return cutoffs;
  }

  /*
    Returns list of arcs on this frame, consisting of all arcs leaving
    the states active on 'cur_frame'.

       @param [in] t       The time-index (on which to look up log-likes),
                           t >= 0
       @param [in] cur_frame   The FrameInfo for the current frame; only its
                       'states' member is expected to be set up on entry.
   */
  Ragged<ArcInfo> GetArcs(int32_t t, FrameInfo *cur_frame) {
    NVTX_RANGE(K2_FUNC);
    Ragged<StateInfo> &states = cur_frame->states;
    const StateInfo *state_values = states.values.Data();

    // in a_fsas_ (the decoding graphs), maps from state_idx01 to arc_idx01x.
    const int32_t *fsa_arc_splits = a_fsas_.shape.RowSplits(2).Data();

    int32_t num_states = states.values.Dim();
    Array1<int32_t> num_arcs(c_, num_states + 1);
    int32_t *num_arcs_data = num_arcs.Data();
    // `num_arcs` gives the num-arcs for each state in `states`.
    K2_EVAL(
        c_, num_states, num_arcs_lambda, (int32_t state_idx01)->void {
          int32_t a_fsas_state_idx01 =
                      state_values[state_idx01].a_fsas_state_idx01,
                  a_fsas_arc_idx01x = fsa_arc_splits[a_fsas_state_idx01],
                  a_fsas_arc_idx01x_next =
                      fsa_arc_splits[a_fsas_state_idx01 + 1],
                  a_fsas_num_arcs = a_fsas_arc_idx01x_next - a_fsas_arc_idx01x;
          num_arcs_data[state_idx01] = a_fsas_num_arcs;
        });
    ExclusiveSum(num_arcs, &num_arcs);

    // initialize shape of array that will hold arcs leaving the active states.
    // Its shape is [fsa_index][state][arc]; the top two levels are shared with
    // `states`.  'ai' means ArcInfo.
    RaggedShape ai_shape =
        ComposeRaggedShapes(states.shape, RaggedShape2(&num_arcs, nullptr, -1));

    // from state_idx01 (into `states` or `ai_shape`) -> fsa_idx0
    const int32_t *ai_row_ids1 = ai_shape.RowIds(1).Data();
    // from arc_idx012 (into `ai_shape`) to state_idx01
    const int32_t *ai_row_ids2 = ai_shape.RowIds(2).Data();
    // from state_idx01 to arc_idx01x
    const int32_t *ai_row_splits2 = ai_shape.RowSplits(2).Data();
    // from state_idx01 (into a_fsas_) to arc_idx01x (into a_fsas_)
    const int32_t *a_fsas_row_splits2 = a_fsas_.shape.RowSplits(2).Data();

    const Arc *arcs = a_fsas_.values.Data();
    // fsa_idx0 to ind0x (into b_fsas_), which gives the 1st row for this
    // sequence.
    const int32_t *b_fsas_row_ids1 = b_fsas_.shape.RowIds(1).Data();
    const int32_t *b_fsas_row_splits1 = b_fsas_.shape.RowSplits(1).Data();
    const float *score_data = b_fsas_.scores.Data();
    int32_t scores_num_cols = b_fsas_.scores.Dim1();
    auto scores_acc = b_fsas_.scores.Accessor();

    Ragged<ArcInfo> ai(ai_shape);
    ArcInfo *ai_data = ai.values.Data();  // uninitialized

    K2_EVAL(
        c_, ai.values.Dim(), ai_lambda, (int32_t ai_arc_idx012)->void {
          int32_t ai_state_idx01 = ai_row_ids2[ai_arc_idx012],
                  ai_fsa_idx0 = ai_row_ids1[ai_state_idx01],
                  ai_arc_idx01x = ai_row_splits2[ai_state_idx01],
                  ai_arc_idx2 = ai_arc_idx012 - ai_arc_idx01x;
          StateInfo sinfo = state_values[ai_state_idx01];
          int32_t a_fsas_arc_idx01x =
                      a_fsas_row_splits2[sinfo.a_fsas_state_idx01],
                  a_fsas_arc_idx012 = a_fsas_arc_idx01x + ai_arc_idx2;
          Arc arc = arcs[a_fsas_arc_idx012];

          int32_t scores_idx0x = b_fsas_row_splits1[ai_fsa_idx0],
                  scores_idx01 = scores_idx0x + t,  // t == ind1 into 'scores'
              scores_idx2 =
                  arc.label + 1;  // the +1 is so that -1 can be handled
          K2_DCHECK_LT(static_cast<uint32_t>(scores_idx2),
                       static_cast<uint32_t>(scores_num_cols));
          float acoustic_score = scores_acc(scores_idx01, scores_idx2);
          ArcInfo ai;
          ai.a_fsas_arc_idx012 = a_fsas_arc_idx012;
          ai.arc_loglike = acoustic_score + arc.score;
          ai.end_loglike =
              OrderedIntToFloat(sinfo.forward_loglike) + ai.arc_loglike;
          // at least currently, the ArcInfo object's src_state and dest_state
          // are idx1's not idx01's, i.e. they don't contain the FSA-index,
          // where as the ai element is an idx01, so we need to do this to
          // convert to an idx01; this relies on the fact that
          // sinfo.abs_state_id == arc.src_state + a_fsas_fsa_idx0x.
          ai.u.dest_a_fsas_state_idx01 =
              sinfo.a_fsas_state_idx01 + arc.dest_state - arc.src_state;
          ai_data[ai_arc_idx012] = ai;
        });
    return ai;
  }

  // Later we may choose to support b_fsas_.Dim0() == 1 and a_fsas_.Dim0() > 1,
  // and we'll have to change various bits of code for that to work.
  inline int32_t NumFsas() const { return b_fsas_.shape.Dim0(); }

  /*
    Does the forward-propagation (basically: the decoding step) and
    returns a newly allocated FrameInfo* object for the next frame.

      num_key_bits (template argument): either 32 (normal case) or 40: it is
            the number number of bits in `state_map_idx`.

      @param [in] t   Time-step that we are processing arcs leaving from;
                   will be called with t=0, t=1, ...
      @param [in] cur_frame  FrameInfo object for the states corresponding to
                   time t; will have its 'states' member set up but not its
                   'arcs' member (this function will create that).
     @return  Returns FrameInfo object corresponding to time t+1; will have its
             'states' member set up but not its 'arcs' member.
   */
  template <int32_t NUM_KEY_BITS>
  std::unique_ptr<FrameInfo> PropagateForward(int32_t t, FrameInfo *cur_frame) {
    NVTX_RANGE("PropagateForward");
    int32_t num_fsas = NumFsas();
    // Ragged<StateInfo> &states = cur_frame->states;
    // arc_info has 3 axes: fsa_id, state, arc.
    cur_frame->arcs = GetArcs(t, cur_frame);

    if (NUM_KEY_BITS > 32) { // a check.
      constexpr int32_t NUM_VALUE_BITS = 64 - NUM_KEY_BITS,
                             shift = std::min<int32_t>(31, NUM_VALUE_BITS);
      // the 'min' part is to avoid a compiler warning about 'shift count too
      // large' for code that is anyway unreachable.
      K2_CHECK_EQ(cur_frame->arcs.NumElements() >> shift, 0) <<
          "Too many arcs to store in hash; try smaller NUM_KEY_BITS (would "
          "require code change) or reduce max_states or minibatch size.";
    }

    Ragged<ArcInfo> &arc_info = cur_frame->arcs;

    ArcInfo *ai_data = arc_info.values.Data();
    Array1<float> ai_data_array1(c_, cur_frame->arcs.values.Dim());
    float *ai_data_array1_data = ai_data_array1.Data();
    K2_EVAL(
        c_, ai_data_array1.Dim(), lambda_set_ai_data,
        (int32_t i)->void { ai_data_array1_data[i] = ai_data[i].end_loglike; });
    Ragged<float> ai_loglikes(arc_info.shape, ai_data_array1);

    // `cutoffs` is of dimension num_fsas.
    Array1<float> cutoffs = GetPruningCutoffs(ai_loglikes);
    float *cutoffs_data = cutoffs.Data();

    // write certain indexes (into ai.values) to state_map_.Data().  Keeps
    // track of the active states and will allow us to assign a numbering to
    // them.
    const int32_t *ai_row_ids1 = arc_info.shape.RowIds(1).Data(),
                  *ai_row_ids2 = arc_info.shape.RowIds(2).Data();
    int64_t state_map_fsa_stride = state_map_fsa_stride_;

    // renumber_states will be a renumbering that dictates which of the arcs in
    // 'ai' correspond to unique states.  Only one arc for each dest-state is
    // kept (it doesn't matter which one).
    Renumbering renumber_states(c_, arc_info.NumElements());
    char *keep_this_state_data = renumber_states.Keep().Data();

    int32_t new_hash_size = RoundUpToNearestPowerOfTwo(
        int32_t(arc_info.NumElements() * 1.0));
    if (new_hash_size > state_map_.NumBuckets()) {
      bool copy_data = false;  // The hash is empty right now, so there is
                               // nothing to copy.
      state_map_.Resize(new_hash_size, NUM_KEY_BITS, -1, copy_data);
    }

    auto state_map_acc = state_map_.GetAccessor<Hash::Accessor<NUM_KEY_BITS>>();

    {
      NVTX_RANGE("LambdaSetStateMap");
      K2_EVAL(
          c_, arc_info.NumElements(), lambda_set_state_map,
          (int32_t arc_idx012)->void {
            int32_t fsa_id = ai_row_ids1[ai_row_ids2[arc_idx012]];
            int32_t dest_state_idx01 =
                ai_data[arc_idx012].u.dest_a_fsas_state_idx01;
            float end_loglike = ai_data[arc_idx012].end_loglike,
                  cutoff = cutoffs_data[fsa_id];
            char keep_this_state = 0;  // only one arc entering any state will
                                       // have its 'keep_this_state_data' entry
                                       // set to 1.
            if (end_loglike > cutoff) {
              uint64_t state_map_idx = dest_state_idx01 +
                          fsa_id * state_map_fsa_stride;
              if (state_map_acc.Insert(state_map_idx, (uint64_t)arc_idx012))
                keep_this_state = 1;
            }
            keep_this_state_data[arc_idx012] = keep_this_state;
          });
    }


    int32_t num_states = renumber_states.NumNewElems();
    // state_reorder_data maps from (state_idx01 on next frame) to (the
    // arc_idx012 on this frame which is the source arc which we arbitrarily
    // choose as being "responsible" for the creation of that state).
    const int32_t *state_reorder_data = renumber_states.Old2New().Data();

    // state_to_fsa_id maps from an index into the next frame's
    // FrameInfo::states.values() vector to the sequence-id (fsa_id) associated
    // with it.  It should be non-decreasing.
    Array1<int32_t> state_to_fsa_id(c_, num_states);
    {  // This block sets 'state_to_fsa_id'.
      NVTX_RANGE("LambdaSetStateToFsaId");
      int32_t *state_to_fsa_id_data = state_to_fsa_id.Data();
      K2_EVAL(
          c_, arc_info.NumElements(), lambda_state_to_fsa_id,
          (int32_t arc_idx012)->void {
            int32_t fsa_id = ai_row_ids1[ai_row_ids2[arc_idx012]],
                    this_state_j = state_reorder_data[arc_idx012],
                    next_state_j = state_reorder_data[arc_idx012 + 1];
            if (next_state_j > this_state_j) {
              state_to_fsa_id_data[this_state_j] = fsa_id;
            }
          });

      K2_DCHECK(IsMonotonic(state_to_fsa_id));
    }

    std::unique_ptr<FrameInfo> ans = std::make_unique<FrameInfo>();
    Array1<int32_t> states_row_splits1(c_, num_fsas + 1);
    RowIdsToRowSplits(state_to_fsa_id, &states_row_splits1);
    ans->states = Ragged<StateInfo>(
        RaggedShape2(&states_row_splits1, &state_to_fsa_id, num_states),
        Array1<StateInfo>(c_, num_states));
    StateInfo *ans_states_data = ans->states.values.Data();
    const int32_t minus_inf_int =
        FloatToOrderedInt(-std::numeric_limits<float>::infinity());
    K2_EVAL(
        c_, num_states, lambda_init_loglike, (int32_t i)->void {
          ans_states_data[i].forward_loglike = minus_inf_int;
        });

    {
      NVTX_RANGE("LambdaModifyStateMap");
      // Modify the elements of `state_map` to refer to the indexes into
      // `ans->states` / `kept_states_data`, rather than the indexes into
      // ai_data. This will decrease some of the values in `state_map`, in
      // general.
      K2_EVAL(
          c_, arc_info.NumElements(), lambda_modify_state_map,
          (int32_t arc_idx012)->void {
            int32_t fsa_id = ai_row_ids1[ai_row_ids2[arc_idx012]];
            int32_t dest_state_idx01 =
                ai_data[arc_idx012].u.dest_a_fsas_state_idx01;
            int32_t this_j = state_reorder_data[arc_idx012],
                    next_j = state_reorder_data[arc_idx012 + 1];
            if (next_j > this_j) {
              uint64_t state_map_idx = dest_state_idx01 +
                                      fsa_id * state_map_fsa_stride;
              uint64_t value, *key_value_addr = nullptr;
              bool ans = state_map_acc.Find(state_map_idx,
                                            &value, &key_value_addr);
              K2_DCHECK(ans);
              K2_DCHECK_EQ(static_cast<int32_t>(value), arc_idx012);
              // Note: this_j is an idx01 into ans->states.  previously it
              // contained an arc_idx012 (of the entering arc that won the
              // race).
              state_map_acc.SetValue(key_value_addr, state_map_idx,
                                     (uint64_t)this_j);
            }
          });
    }

    // We'll set up the data of the kept states below...
    StateInfo *kept_states_data = ans->states.values.Data();

    {
      int32_t *ans_states_row_splits1_data = ans->states.RowSplits(1).Data();

      NVTX_RANGE("LambdaSetStates");
      K2_EVAL(
          c_, arc_info.NumElements(), lambda_set_arcs_and_states,
          (int32_t arc_idx012)->void {
            int32_t fsa_id = ai_row_ids1[ai_row_ids2[arc_idx012]];

            ArcInfo &info = ai_data[arc_idx012];

            int32_t dest_a_fsas_state_idx01 = info.u.dest_a_fsas_state_idx01;


            uint64_t state_map_idx = dest_a_fsas_state_idx01 +
                                     fsa_id * state_map_fsa_stride;
            uint64_t state_idx01;
            const uint64_t minus_one = ~(uint64_t)0;
            if (!state_map_acc.Find(state_map_idx, &state_idx01))
              state_idx01 = minus_one;   // The destination state did not survive
                                         // pruning.

            int32_t state_idx1;
            if (state_idx01 != minus_one) {
              int32_t state_idx0x = ans_states_row_splits1_data[fsa_id];
              state_idx1 = static_cast<int32_t>(state_idx01) - state_idx0x;
            } else {
              state_idx1 = -1;  // Meaning: invalid.
            }
            // state_idx1 is the idx1 into ans->states, of the destination
            // state.
            info.u.dest_info_state_idx1 = state_idx1;
            if (state_idx1 < 0)
              return;

            // multiple threads may write the same value to the address written
            // to in the next line.
            kept_states_data[state_idx01].a_fsas_state_idx01 =
                dest_a_fsas_state_idx01;
            int32_t end_loglike_int = FloatToOrderedInt(info.end_loglike);
            // Set the forward log-like of the dest state to the largest of any
            // of those of the incoming arcs.  Note: we initialized this in
            // lambda_init_loglike above.
            AtomicMax(&(kept_states_data[state_idx01].forward_loglike),
                      end_loglike_int);
          });
    }
    {
      NVTX_RANGE("LambdaResetStateMap");
      const int32_t *next_states_row_ids1 = ans->states.shape.RowIds(1).Data();
      K2_EVAL(
          c_, ans->states.NumElements(), lambda_reset_state_map,
          (int32_t state_idx01)->void {
            int32_t a_fsas_state_idx01 =
                        kept_states_data[state_idx01].a_fsas_state_idx01,
                fsa_idx0 = next_states_row_ids1[state_idx01];
            int64_t state_map_idx = a_fsas_state_idx01 +
                                    fsa_idx0 * state_map_fsa_stride;
            state_map_acc.Delete(state_map_idx);
          });
    }
    return ans;
  }


  /*
    Sets backward_loglike fields of StateInfo to the negative of the forward
    prob if (this is the final-state or !only_final_probs), else -infinity.

    This is used in computing the backward loglikes/scores for purposes of
    pruning.  This may be done after we're finished decoding/intersecting,
    or while we are still decoding.

    Note: something similar to this (setting backward-prob == forward-prob) is
    also done in PropagateBackward() when we detect final-states.  That's needed
    because not all sequences have the same length, so some may have reached
    their final state earlier.  (Note: we only get to the final-state of a_fsas_
    if we've reached the final frame of the input, because for non-final frames
    we always have -infinity as the log-prob corresponding to the symbol -1.)

    While we are still decoding, a background process will do pruning
    concurrently with the forward computation, for purposes of reducing memory
    usage (and so that most of the pruning can be made concurrent with the
    forward computation).  In this case we want to avoid pruning away anything
    that wouldn't have been pruned away if we were to have waited to the end;
    and it turns out that setting the backward probs to the negative of the
    forward probs (i.e.  for all states, not just final states) accomplishes
    this.  The issue was mentioned in the "Exact Lattice Generation.." paper and
    also in the code for Kaldi's lattice-faster-decoder; search for "As in [3],
    to save memory..."

      @param [in] cur_frame    Frame on which to set the backward probs
  */
  void SetBackwardProbsFinal(FrameInfo *cur_frame) {
    NVTX_RANGE("SetBackwardProbsFinal");
    Ragged<StateInfo> &cur_states = cur_frame->states;  // 2 axes: fsa,state
    int32_t num_states = cur_states.values.Dim();
    if (num_states == 0)
      return;
    StateInfo *cur_states_data = cur_states.values.Data();
    const int32_t *a_fsas_row_ids1_data = a_fsas_.shape.RowIds(1).Data(),
               *a_fsas_row_splits1_data = a_fsas_.shape.RowSplits(1).Data(),
              *cur_states_row_ids1_data = cur_states.RowIds(1).Data();
    double minus_inf = -std::numeric_limits<double>::infinity();

    K2_EVAL(c_, num_states, lambda_set_backward_prob, (int32_t state_idx01) -> void {
        StateInfo *info = cur_states_data + state_idx01;
        double backward_loglike,
            forward_loglike = OrderedIntToFloat(info->forward_loglike);
        if (forward_loglike - forward_loglike == 0) { // not -infinity...
          // canonically we'd set this to zero, but setting it to the forward
          // loglike when this is the final-state (in a_fsas_) has the effect of
          // making the (forward+backward) probs equivalent to the logprob minus
          // the best-path log-prob, which is convenient for pruning.  If this
          // is not actually the last frame of this sequence, which can happen
          // if this was called before the forward decoding process was
          // finished, what we are doing is a form of pruning that is guaranteed
          // not to prune anything out that would not have been pruned out if we
          // had waited until the real end of the file to do the pruning.
          backward_loglike = -forward_loglike;
        } else {
          backward_loglike = minus_inf;
        }
        info->backward_loglike = backward_loglike;
      });
  }

  /*
    Does backward propagation of log-likes, which means setting the
    backward_loglike field of the StateInfo variable (for cur_frame);
    and works out which arcs and which states are to be pruned
    on cur_frame; this information is output to Array1<char>'s which
    are supplied by the caller.

    These backward log-likes are normalized in such a way that you can add them
    with the forward log-likes to produce the log-likelihood ratio vs the best
    path (this will be non-positive).  (To do this, for the final state we have
    to set the backward log-like to the negative of the forward log-like; see
    SetBackwardProbsFinal()).

    This function also prunes arc-indexes on `cur_frame` and state-indexes
    on `next_frame`.

       @param [in] t      The time-index (on which to look up log-likes);
                          equals time index of `cur_frame`; t >= 0
       @param [in]  cur_frame The FrameInfo for the frame on which we want to
                          set the forward log-like, and output pruning info
                          for arcs and states
       @param [in]  next_frame The next frame's FrameInfo, on which to look
                           up log-likes for the next frame; the
                           `backward_loglike` values of states on `next_frame`
                           are assumed to already be set, either by
                           SetBackwardProbsFinal() or a previous call to
                           PropagateBackward().
       @param [out] cur_frame_states_keep   An array, created by the caller,
                        to which we'll write 1s for elements of cur_frame->states
                        which we need to keep, and 0s for others.
       @param [out] cur_frame_arcs_keep   An array, created by the caller,
                        to which we'll write 1s for elements of cur_frame->arcs
                        which we need to keep (because they survived pruning),
                        and 0s for others.
  */
  void PropagateBackward(int32_t t,
                         FrameInfo *cur_frame,
                         FrameInfo *next_frame,
                         Array1<char> *cur_frame_states_keep,
                         Array1<char> *cur_frame_arcs_keep) {
    NVTX_RANGE("PropagateBackward");
    int32_t num_states = cur_frame->states.NumElements(),
            num_arcs = cur_frame->arcs.NumElements();
    K2_CHECK_EQ(num_states, cur_frame_states_keep->Dim());
    K2_CHECK_EQ(num_arcs, cur_frame_arcs_keep->Dim());

    int32_t *a_fsas_row_ids1_data = a_fsas_.shape.RowIds(1).Data(),
            *a_fsas_row_splits1_data = a_fsas_.shape.RowSplits(1).Data();

    float minus_inf = -std::numeric_limits<float>::infinity();

    Ragged<float> arc_backward_prob(cur_frame->arcs.shape,
                                    Array1<float>(c_, cur_frame->arcs.NumElements()));
    float *arc_backward_prob_data = arc_backward_prob.values.Data();

    ArcInfo *ai_data = cur_frame->arcs.values.Data();
    int32_t *arcs_rowids1 = cur_frame->arcs.shape.RowIds(1).Data(),
            *arcs_rowids2 = cur_frame->arcs.shape.RowIds(2).Data(),
            *arcs_row_splits1 = cur_frame->arcs.shape.RowSplits(1).Data(),
            *arcs_row_splits2 = cur_frame->arcs.shape.RowSplits(2).Data();
    float output_beam = output_beam_;

    // compute arc backward probs, and set elements of 'keep_cur_arcs_data'
    int32_t next_num_states = next_frame->states.TotSize(1);

    char *keep_cur_arcs_data = cur_frame_arcs_keep->Data(),
        *keep_cur_states_data = cur_frame_states_keep->Data();

    const int32_t *next_states_row_splits1_data =
        next_frame->states.RowSplits(1).Data();

    StateInfo *next_states_data = next_frame->states.values.Data();
    StateInfo *cur_states_data = cur_frame->states.values.Data();

    K2_EVAL(c_, num_arcs, lambda_set_arc_backward_prob_and_keep,
            (int32_t arcs_idx012) -> void {
      ArcInfo *arc = ai_data + arcs_idx012;
      int32_t state_idx01 = arcs_rowids2[arcs_idx012],
                 seq_idx0 = arcs_rowids1[state_idx01],  // 'seq' == fsa-idx in b
        next_states_idx0x = next_states_row_splits1_data[seq_idx0];

      // Note: if dest_state_idx1 == -1, dest_state_idx01 has a meaningless
      // value below, but it's never referenced.
      int32_t dest_state_idx1 = arc->u.dest_info_state_idx1,
             dest_state_idx01 = next_states_idx0x + dest_state_idx1;
      float backward_loglike = minus_inf;
      char keep_this_arc = 0;
      if (dest_state_idx1 == -1) {
          // dest_state_idx1 == -1 means this arc was already pruned in
          // the forward pass.. do nothing.
      } else {
        float arc_loglike = arc->arc_loglike;
        float dest_state_backward_loglike =
            next_states_data[dest_state_idx01].backward_loglike;
        // 'backward_loglike' is the loglike at the beginning of the arc
        backward_loglike = arc_loglike + dest_state_backward_loglike;
        float src_state_forward_loglike = OrderedIntToFloat(
            cur_states_data[arcs_rowids2[arcs_idx012]].forward_loglike);

        // should be <= 0.0, mathematically.
        K2_CHECK_LT(backward_loglike, -src_state_forward_loglike + 2.0);
        if (backward_loglike + src_state_forward_loglike >= -output_beam) {
          keep_this_arc = 1;
        } else {
          backward_loglike = minus_inf;  // Don't let arcs outside beam
                                         // contribute to their start-states's
                                         // backward prob (we'll use that to
                                         // prune the start-states away.)
        }
      }
      keep_cur_arcs_data[arcs_idx012] = keep_this_arc;
      arc_backward_prob_data[arcs_idx012] = backward_loglike;
      });

    /* note, the elements of state_backward_prob that don't have arcs leaving
       them will be set to the supplied default.  */
    Array1<float> state_backward_prob(c_, num_states);
    MaxPerSublist(arc_backward_prob, minus_inf, &state_backward_prob);

    const float *state_backward_prob_data = state_backward_prob.Data();
    const int32_t *cur_states_row_ids1 =
        cur_frame->states.shape.RowIds(1).Data();

    int32_t num_fsas = NumFsas();
    K2_DCHECK_EQ(cur_frame->states.shape.Dim0(), num_fsas);
    K2_EVAL(
        c_, cur_frame->states.NumElements(), lambda_set_state_backward_prob,
        (int32_t state_idx01)->void {
          StateInfo *info = cur_states_data + state_idx01;
          int32_t fsas_state_idx01 = info->a_fsas_state_idx01,
                  a_fsas_idx0 = a_fsas_row_ids1_data[fsas_state_idx01],
                  fsas_state_idx0x_next = a_fsas_row_splits1_data[a_fsas_idx0 + 1];
          float forward_loglike = OrderedIntToFloat(info->forward_loglike),
                backward_loglike;
          // `is_final_state` means this is the final-state in a_fsas.  this
          // implies it's final in b_fsas too, since they both would have seen
          // symbols -1.
          int32_t is_final_state =
              (fsas_state_idx01 + 1 >= fsas_state_idx0x_next);
          if (is_final_state) {
            // Note: there is only one final-state.
            backward_loglike = -forward_loglike;
          } else {
            backward_loglike = state_backward_prob_data[state_idx01];
          }
          info->backward_loglike = backward_loglike;
          keep_cur_states_data[state_idx01] = (backward_loglike != minus_inf);
        });
  }

  /*
    This function does backward propagation and pruning of arcs and states for a
    specific time range.
        @param [in] begin_t   Lowest `t` value to call PropagateBackward() for
                            and to prune its arcs and states.  Require t >= 0.
        @param [in] end_t    One-past-the-highest `t` value to call PropagateBackward()
                            and to prune its arcs and states.  Require that
                            `frames_[t+1]` already be set up; this requires at least
                            end_t <= T.
    Arcs on frames t >= end_t and states on frame t > end_t are ignored; the backward
    probs on time end_t are set by SetBackwardProbsFinal(), see its documentation
    to understand what this does if we haven't yet reached the end of one of the
    sequences.

    After this function is done, the arcs for `frames_[t]` with begin_t <= t < end_t and
    the states for `frames_[t]` with begin_t < t < end_t will have their numbering changed.
    (We don't renumber the states on begin_t because that would require the dest-states
     of the arcs on time `begin_t - 1` to be modified).  TODO: check this...
   */
  void PruneTimeRange(int32_t begin_t,
                      int32_t end_t) {
    NVTX_RANGE(K2_FUNC);
    SetBackwardProbsFinal(frames_[end_t].get());
    ContextPtr cpu = GetCpuContext();
    int32_t num_fsas = b_fsas_.shape.Dim0(),
               num_t = end_t - begin_t;
    Array1<int32_t> old_states_offsets(cpu, num_t + 1),
        old_arcs_offsets(cpu, num_t + 1);
    int32_t tot_states = 0, tot_arcs = 0;
    {
      int32_t *old_states_offsets_data = old_states_offsets.Data(),
                *old_arcs_offsets_data = old_arcs_offsets.Data();
      for (int32_t i = 0; i <= num_t; i++) {
        int32_t t = begin_t + i;
        old_states_offsets_data[i] = tot_states;
        old_arcs_offsets_data[i] = tot_arcs;
        if (i < num_t) {
          tot_states += frames_[t]->arcs.TotSize(1);
          tot_arcs += frames_[t]->arcs.TotSize(2);
        }
      }
    }


    // contains respectively: row_splits1_ptrs, row_ids1_ptrs,
    // row_splits1_ptrs, row_splits2_ptrs,
    // old_arcs_ptrs (really type ArcInfo*),
    // old_states_ptrs (really type StateInfo*).
    Array1<void*> old_all_ptrs(cpu, num_t * 6);

    Renumbering renumber_states(c_, tot_states),
        renumber_arcs(c_, tot_arcs);
    {
      void                    **all_p = old_all_ptrs.Data();
      int32_t **old_row_splits1_ptrs_data = (int32_t**)all_p,
                 **old_row_ids1_ptrs_data = (int32_t**)all_p + num_t,
          **old_row_splits2_ptrs_data = (int32_t**)all_p + 2 * num_t,
             **old_row_ids2_ptrs_data = (int32_t**)all_p + 3 * num_t;
      StateInfo **old_states_ptrs_data = (StateInfo**)all_p + 4 * num_t;
      ArcInfo **old_arcs_ptrs_data = (ArcInfo**)all_p + 5 * num_t;
      int32_t *old_states_offsets_data = old_states_offsets.Data(),
                *old_arcs_offsets_data = old_arcs_offsets.Data();

      for (int32_t t = end_t - 1; t >= begin_t; --t) {
        int32_t i = t - begin_t;
        Array1<char> this_states_keep =
            renumber_states.Keep().Arange(old_states_offsets_data[i],
                                          old_states_offsets_data[i + 1]),
            this_arcs_keep =
            renumber_arcs.Keep().Arange(old_arcs_offsets_data[i],
                                        old_arcs_offsets_data[i + 1]);
        FrameInfo *cur_frame = frames_[t].get();
        PropagateBackward(t, cur_frame, frames_[t+1].get(),
                          &this_states_keep, &this_arcs_keep);

        old_row_splits1_ptrs_data[i] = cur_frame->arcs.RowSplits(1).Data();
        old_row_ids1_ptrs_data[i] = cur_frame->arcs.RowIds(1).Data();
        old_row_splits2_ptrs_data[i] = cur_frame->arcs.RowSplits(2).Data();
        old_row_ids2_ptrs_data[i] = cur_frame->arcs.RowIds(2).Data();
        old_arcs_ptrs_data[i] = cur_frame->arcs.values.Data();
        old_states_ptrs_data[i] = cur_frame->states.values.Data();

        // We can't discard any states on t == begin_t because: if it is not t ==
        // 0, it would be inconvenient to map the dest-states of arcs on t - 1;
        // and if it is t == 0, this may remove the start-state, which would make
        // it more complex to avoid invalid FSAs (e.g. with an end-state but no
        // start-state, or in which we incorrectly interpret a non-start state as
        // the start state).
        if (i == 0)  // t == begin_t
          this_states_keep = (char)1;  // set all elements of the array
        // `states_keep` to 1.
      }
    }

    old_states_offsets = old_states_offsets.To(c_);
    old_arcs_offsets = old_arcs_offsets.To(c_);
    Array1<int32_t> new_states_offsets = renumber_states.Old2New(true)[old_states_offsets],
                      new_arcs_offsets = renumber_arcs.Old2New(true)[old_arcs_offsets];
    int32_t new_num_states = renumber_states.NumNewElems(),
              new_num_arcs =  renumber_arcs.NumNewElems();
    // These arrays map to the (t - begin_t) corresponding to this state or arc
    // in the new numbering, i.e. the frame index minus begin_t.
    Array1<int32_t> new_state_to_frame(c_, new_num_states),
        new_arc_to_frame(c_, new_num_arcs);
    RowSplitsToRowIds(new_states_offsets, &new_state_to_frame);
    RowSplitsToRowIds(new_arcs_offsets, &new_arc_to_frame);
    const int32_t *old_states_offsets_data = old_states_offsets.Data(),
                  *new_states_offsets_data = new_states_offsets.Data(),
                    *old_arcs_offsets_data = old_arcs_offsets.Data(),
                    *new_arcs_offsets_data = new_arcs_offsets.Data(),
                  *new_state_to_frame_data = new_state_to_frame.Data(),
                    *new_arc_to_frame_data = new_arc_to_frame.Data(),
                      *states_old2new_data = renumber_states.Old2New().Data(),
                      *states_new2old_data = renumber_states.New2Old().Data(),
                        *arcs_old2new_data = renumber_arcs.Old2New().Data(),
                        *arcs_new2old_data = renumber_arcs.New2Old().Data();

    // Allocate the new row_splits and row_ids vectors for the shapes on the
    // individual frames, and the new arc-info and state-info.
    Array2<int32_t> all_row_splits1(c_, num_t, num_fsas + 1);
    auto all_row_splits1_acc = all_row_splits1.Accessor();
    Array1<int32_t> all_row_ids1(c_, new_num_states);
    // the "+ num_t" below is for the extra element of each row_splits array.
    Array1<int32_t> all_row_splits2(c_, new_num_states + num_t);
    Array1<int32_t> all_row_ids2(c_, new_num_arcs);
    Array1<StateInfo> all_states(c_, new_num_states);
    Array1<ArcInfo> all_arcs(c_, new_num_arcs);

    int32_t *all_row_ids1_data = all_row_ids1.Data(),
            *all_row_ids2_data = all_row_ids2.Data(),
         *all_row_splits2_data = all_row_splits2.Data();
    StateInfo *all_states_data = all_states.Data();
    ArcInfo *all_arcs_data = all_arcs.Data();

    old_all_ptrs = old_all_ptrs.To(c_);
    void **all_p = old_all_ptrs.Data();

    K2_EVAL2(c_, num_t, num_fsas + 1,
             lambda_set_new_row_splits1, (int32_t t_offset,
                                          int32_t seq_idx) -> void {
      // note, t_offset is t - t_start.
      int32_t *old_row_splits1 = (int32_t*) all_p[t_offset];
      int32_t old_idx0x = old_row_splits1[seq_idx];
      // "pos" means position in appended states vector
      // old_start_pos means start for this `t`.
      int32_t old_start_pos = old_states_offsets_data[t_offset],
                    old_pos = old_start_pos + old_idx0x,
              new_start_pos = states_old2new_data[old_start_pos],
                    new_pos = states_old2new_data[old_pos],
                  new_idx0x = new_pos - new_start_pos;
      all_row_splits1_acc(t_offset, seq_idx) = new_idx0x;
      // TODO: set elem zero of row-splits?

      if (seq_idx == 0) {
        // We assign the `seq_idx == 0` version of the kernel to set the initial
        // zero in each row_splits vector.
        all_row_splits2_data[new_pos + t_offset] = 0;
      }
             });

    K2_EVAL(c_, new_num_states, lambda_per_state, (int32_t new_i) -> void {
      // new_i is position in appended vector of all states.
      int32_t    t_offset = new_state_to_frame_data[new_i],
      old_state_start_pos = old_states_offsets_data[t_offset],
        new_arc_start_pos = new_arcs_offsets_data[t_offset],
        old_arc_start_pos = old_arcs_offsets_data[t_offset],
                    old_i = states_new2old_data[new_i],
          old_state_idx01 = old_i - old_state_start_pos;


      // this old_states_data is from its FrameInfo::states.
      const StateInfo *old_states_data = (StateInfo*)all_p[4 * num_t + t_offset];
      const int32_t *old_row_ids1_data = (int32_t*)all_p[1 * num_t + t_offset],
                 *old_row_splits2_data = (int32_t*)all_p[2 * num_t + t_offset];

      // set the row-ids1 (these contain FSA-ids).
      all_row_ids1_data[new_i] = old_row_ids1_data[old_state_idx01];


      {  // set the row-splits2.
        // We make each kernel responsible for the *next* row_splits entry,
        // i.e. for its new_state_idx01 plus one.  This solves the problem of no
        // kernel being responsible for the last row-splits entry.  We
        // separately wrote the zeros for the 1st row-splits entry, in a
        // previous kernel.
        //
        // It's safe to use old_state_idx01+1 instead of doing the same mapping
        // from new_i+1 that we do from new_i to old_state_idx01, because
        // we know this state was kept (because it has a new_i index.)
        int32_t old_arc_idx01x_next = old_row_splits2_data[old_state_idx01+1],
                   old_arc_pos_next = old_arc_idx01x_next + old_arc_start_pos,
                   new_arc_pos_next = arcs_old2new_data[old_arc_pos_next],
                new_arc_idx01x_next = new_arc_pos_next - new_arc_start_pos;

        // "+ t_offset" is to compensate for the extra element of each row_splits
        // vector.  The "+ 1" is about the "next", i.e. each kernel is responsible
        // for the next row_splits element, and none is responsible for the initial zero;
        // that is set in a previous kernel.
        all_row_splits2_data[new_i + t_offset + 1] = new_arc_idx01x_next;
      }
      all_states_data[new_i] = old_states_data[old_state_idx01];
      });

    K2_EVAL(c_, new_num_arcs, lambda_set_arcs, (int32_t new_i) -> void {
      // new_i is position in appended vector of all arcs
      int32_t    t_offset = new_arc_to_frame_data[new_i],
      new_state_start_pos = new_states_offsets_data[t_offset],
      old_state_start_pos = old_states_offsets_data[t_offset],
 next_old_state_start_pos = old_states_offsets_data[t_offset + 1],
        old_arc_start_pos = old_arcs_offsets_data[t_offset],
                    old_i = arcs_new2old_data[new_i],
           old_arc_idx012 = old_i - old_arc_start_pos;

      ArcInfo *old_info_data =  (ArcInfo*)all_p[5 * num_t + t_offset];
      int32_t *old_row_ids2_data = (int32_t*)all_p[3 * num_t + t_offset],
             *old_row_ids1_data  = (int32_t*)all_p[1 * num_t + t_offset],
      *next_old_row_splits1_data = (int32_t*)all_p[t_offset + 1];

      int32_t old_src_state_idx01 = old_row_ids2_data[old_arc_idx012],
                         fsa_idx0 = old_row_ids1_data[old_src_state_idx01],
                old_src_state_pos = old_src_state_idx01 + old_state_start_pos,
                new_src_state_pos = states_old2new_data[old_src_state_pos],
              new_src_state_idx01 = new_src_state_pos - new_state_start_pos;

      all_row_ids2_data[new_i] = new_src_state_idx01;

      ArcInfo info = old_info_data[old_arc_idx012];

      if (t_offset + 1 == num_t) {
        // Do nothing; this is the last frame of the batch of frames that we are
        // pruning, so we don't need to renumber the destination-states of the
        // arcs leaving it because the next frame's states have not been pruned
        // (so the numbering stays the same).
      } else {
        // idx1 of the state in the next frame's `states` object.
        int32_t dest_info_state_idx1 = info.u.dest_info_state_idx1;

        // the naming below is unusual; by "pos" we mean position in the old or
        // new "all_states" or "all_arcs" vectors, which have all frames appended.
        // (the new ones physically exist; the old ones don't, but they are the
        // numberings used in renumber_states.Keep() and renumber_arcs.Keep().)
        int32_t old_dest_state_idx0x = next_old_row_splits1_data[fsa_idx0],
        old_dest_state_idx01 = old_dest_state_idx0x + dest_info_state_idx1,
        old_dest_state_idx0x_pos = next_old_state_start_pos + old_dest_state_idx0x,
        old_dest_state_idx01_pos = next_old_state_start_pos + old_dest_state_idx01,
        new_dest_state_idx0x_pos = states_old2new_data[old_dest_state_idx0x_pos],
        new_dest_state_idx01_pos = states_old2new_data[old_dest_state_idx01_pos],
        new_dest_state_idx1 = new_dest_state_idx01_pos - new_dest_state_idx0x_pos;
        info.u.dest_info_state_idx1 = new_dest_state_idx1;
      }
      all_arcs_data[new_i] = info;
      });

    // Now reconstruct the states and arcs for all the frames we pruned, from
    // sub-parts of the arrays we just created.
    new_states_offsets = new_states_offsets.To(cpu);
    new_arcs_offsets = new_arcs_offsets.To(cpu);
    new_states_offsets_data = new_states_offsets.Data();
    new_arcs_offsets_data = new_arcs_offsets.Data();
    for (int32_t i = 0; i < num_t; i++) {  // i corresponds to "t_offset".
      int32_t state_offset = new_states_offsets_data[i],
         next_state_offset = new_states_offsets_data[i + 1],
                arc_offset = new_arcs_offsets_data[i],
           next_arc_offset = new_arcs_offsets_data[i + 1];

      // next line: operator[] into Array2 gives Array1, one row.
      Array1<int32_t> row_splits1 = all_row_splits1.Row(i),
                         row_ids1 = all_row_ids1.Arange(state_offset, next_state_offset),
                      row_splits2 = all_row_splits2.Arange(state_offset + i, next_state_offset + (i+1)),
                         row_ids2 = all_row_ids2.Arange(arc_offset, next_arc_offset);
      Array1<ArcInfo> arcs = all_arcs.Arange(arc_offset, next_arc_offset);

      RaggedShape arcs_shape = RaggedShape3(&row_splits1, &row_ids1, -1,
                                            &row_splits2, &row_ids2, -1);
      int32_t t = begin_t + i;
      frames_[t]->arcs = Ragged<ArcInfo>(arcs_shape, arcs);
      Array1<StateInfo> states = all_states.Arange(state_offset, next_state_offset);
      RaggedShape states_shape = GetLayer(arcs_shape, 0);
      frames_[t]->states = Ragged<StateInfo>(states_shape, states);
    }
  }


  ContextPtr c_;
  FsaVec &a_fsas_;         // Note: a_fsas_ has 3 axes.
  int32_t a_fsas_stride_;  // 1 if we use a different FSA per sequence
                           // (a_fsas_.Dim0() > 1), 0 if the decoding graph is
                           // shared (a_fsas_.Dim0() == 1).
  DenseFsaVec &b_fsas_;
  int32_t T_;  // == b_fsas_.shape.MaxSize(1).
  float search_beam_;
  float output_beam_;
  int32_t min_active_;
  int32_t max_active_;
  Array1<float> dynamic_beams_;  // dynamic beams (initially just search_beam_
                                 // but change due to max_active/min_active
                                 // constraints).

  int32_t state_map_fsa_stride_;  // state_map_fsa_stride_ is a_fsas_.TotSize(1)
                                  // if a_fsas_.Dim0() == 1, else 0.


  Hash state_map_;    // state_map_ maps from:
                      // key == (state_map_fsa_stride_*n) + a_fsas_state_idx01,
                      //    where n is the fsa_idx, i.e. the index into b_fsas_
                      // to
                      // value, where at different stages of PropagateForward(),
                      // value is an arc_idx012 (into cur_frame->arcs), and
                      // then later a state_idx01 into the next frame's `state`
                      // member.

  // The 1st dim is needed because If all the
  // streams share the same FSA in a_fsas_, we need
  // separate maps for each).  This map is used on
  // each frame to compute and store the mapping
  // from active states to the position in the
  // `states` array.  Between frames, all values
  // have -1 in them.
  std::vector<std::unique_ptr<FrameInfo>> frames_;

  // logically an array of bool, of size T_ + 1; for each 0 <= t <= T, after the
  // forward pass finishes propagation with cur_frame_ == t, if
  // do_pruning_after_[t] is false it will continue as normal; otherwise (if
  // true), it will signal `semaphore_`.
  std::vector<char> do_pruning_after_;

  // For each t for which do_pruning_after_[t] is true, there will be a
  // pair (begin_t, end_t) in prune_t_begin_end giving the
  // arguments for which we will invoke PruneTimeRange() after the forward-pass
  // for time t has completed.  The size of this array equals the sum
  // of nonzero elements of do_pruning_after_.
  std::vector<std::pair<int32_t, int32_t> > prune_t_begin_end_;

  // Each time the forward-pass finishes forward processing for a t value for
  // which do_pruning_after_[t] is true, it will signal this semaphore; the
  // backward-pass thread (which does pruning) will wait on it as many times as
  // do_pruning_after_[t] is set to true.
  Semaphore backward_semaphore_;

  // The function of forward_semaphore_ is to ensure that the backward (pruning)
  // pass doesn't "get too far behind" relative to the forward pass, which might
  // cause us to use more memory than expected.  (Note: the backward pass is
  // normally a bit faster than the forward pass, so typically this won't be a
  // problem).  Each time the backward pass has finished one round of pruning it
  // signals this semaphore.  each time after the forward pass signals the
  // backward pass that it's ready to prune, it waits on this semaphore
  // immediately afterward.  But because forward_semaphore_ is initialized to 1
  // rather than zero, the effect is that the forward pass is waiting for the
  // *previous* phase of backward pruning to complete, rather than the current
  // one.
  k2std::counting_semaphore forward_semaphore_;
};

void IntersectDensePruned(FsaVec &a_fsas, DenseFsaVec &b_fsas,
                          float search_beam, float output_beam,
                          int32_t min_active_states, int32_t max_active_states,
                          FsaVec *out, Array1<int32_t> *arc_map_a,
                          Array1<int32_t> *arc_map_b) {
  NVTX_RANGE("IntersectDensePruned");
  FsaVec a_vec = FsaToFsaVec(a_fsas);
  MultiGraphDenseIntersectPruned intersector(a_vec, b_fsas, search_beam,
                                             output_beam, min_active_states,
                                             max_active_states);

  intersector.Intersect();
  intersector.FormatOutput(out, arc_map_a, arc_map_b);
}
}  // namespace k2
