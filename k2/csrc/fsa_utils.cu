#include "hip/hip_runtime.h"
/**
 * @brief Utilities for creating FSAs.
 *
 * Note that serializations are done in Python.
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey, Haowen Qiu)
 *                      Mobvoi Inc.        (authors: Fangjun Kuang)
 *                      Guoguo Chen
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <algorithm>
#include <limits>
#include <sstream>
#include <utility>
#include <vector>

#include "k2/csrc/array.h"
#include "k2/csrc/context.h"
#include "k2/csrc/fsa.h"
#include "k2/csrc/fsa_utils.h"
#include "k2/csrc/math.h"
#include "k2/csrc/ragged.h"

namespace k2 {

// field separator within a line for a text form FSA
static constexpr const char *kDelim = " \t";

// Convert a string to an integer. Abort the program on failure.
static int32_t StringToInt(const std::string &s) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK(!s.empty());

  bool ok = false;
  char *p = nullptr;
  // std::strtol requires a `long` type
  long n = std::strtol(s.c_str(), &p, 10);  // NOLINT
  if (*p == '\0') ok = true;

  auto res = static_cast<int32_t>(n);
  if (n != res) ok = false;  // out of range

  K2_CHECK(ok) << "Failed to convert " << s << " to an integer";

  return res;
}

// Convert a string to a float. Abort the program on failure.
// TODO(guoguo): We may run into locale problems, with comma vs. period for
//               decimals. We have to test if the C code will behave the same
//               w.r.t. locale as Python does.
static float StringToFloat(const std::string &s) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK(!s.empty());
  char *p = nullptr;
  float f = std::strtof(s.c_str(), &p);
  if (*p != '\0') K2_LOG(FATAL) << "Failed to convert " << s << " to a float";
  return f;
}

// Trim leading and trailing spaces of a string.
static void TrimString(std::string *s) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_NE(s, nullptr);
  auto not_space = [](int32_t c) -> bool { return std::isspace(c) == 0; };

  s->erase(s->begin(), std::find_if(s->begin(), s->end(), not_space));
  s->erase(std::find_if(s->rbegin(), s->rend(), not_space).base(), s->end());
}

/* Split a string to a vector of strings using a set of delimiters.

   Example usage:

   @code
    std::string in = "1 2 3";
    const char *delim = " \t";
    std::vector<std::string> out;
    SplitStringToVector(in, delim, &out);
   @endcode

   @param [in]  in    The input string to be split.
   @param [in]  delim A string of delimiters.
   @param [out] out   It saves the split result.
*/
static void SplitStringToVector(const std::string &in, const char *delim,
                                std::vector<std::string> *out) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_NE(delim, nullptr);
  K2_CHECK_NE(out, nullptr);
  out->clear();
  std::size_t start = 0;
  while (true) {
    auto pos = in.find_first_of(delim, start);
    if (pos == std::string::npos) break;

    auto sub = in.substr(start, pos - start);
    start = pos + 1;

    TrimString(&sub);
    if (!sub.empty()) out->emplace_back(std::move(sub));
  }

  if (start < in.size()) {
    auto sub = in.substr(start);
    TrimString(&sub);
    if (!sub.empty()) out->emplace_back(std::move(sub));
  }
}

/* Create an acceptor from a stream, assuming the acceptor is in the k2 format:

   src_state1 dest_state1 label1 score1
   src_state2 dest_state2 label2 score2
   ... ...
   final_state

   The source states will be in non-descending order, and the final state does
   not bear a cost/score -- we put the cost/score on the arc that connects to
   the final state and set its label to -1.

   @param [in]  is    The input stream that contains the acceptor.

   @return It returns an Fsa on CPU.
*/
static Fsa K2AcceptorFromStream(std::istringstream &is) {
  NVTX_RANGE(K2_FUNC);
  std::vector<Arc> arcs;
  std::vector<std::string> splits;
  std::string line;

  bool finished = false;  // when the final state is read, set it to true.
  while (std::getline(is, line)) {
    SplitStringToVector(line, kDelim,
                        &splits);  // splits is cleared in the function
    if (splits.empty()) continue;  // this is an empty line

    K2_CHECK_EQ(finished, false);

    auto num_fields = splits.size();
    if (num_fields == 4u) {
      //   0            1          2      3
      // src_state  dest_state   label  score
      int32_t src_state = StringToInt(splits[0]);
      int32_t dest_state = StringToInt(splits[1]);
      int32_t symbol = StringToInt(splits[2]);
      float score = StringToFloat(splits[3]);
      arcs.emplace_back(src_state, dest_state, symbol, score);
    } else if (num_fields == 1u) {
      //   0
      // final_state
      (void)StringToInt(splits[0]);  // this is a final state
      finished = true;               // set finish
    } else {
      K2_LOG(FATAL) << "Invalid line: " << line
                    << "\nk2 acceptor expects a line with 1 (final_state) or "
                       "4 (src_state dest_state label score) fields";
    }
  }

  K2_CHECK_EQ(finished, true) << "The last line should be the final state";

  bool error = true;
  Array1<Arc> array(GetCpuContext(), arcs);
  auto fsa = FsaFromArray1(array, &error);
  K2_CHECK_EQ(error, false);

  return fsa;
}

/* Create a transducer from a stream, assuming the transducer is in the K2
   format:

   src_state1 dest_state1 label1 aux_label1 score1
   src_state2 dest_state2 label2 aux_label2 score2
   ... ...
   final_state

   The source states will be in non-descending order, and the final state does
   not bear a cost/score -- we put the cost/score on the arc that connects to
   the final state and set its label to -1.

   @param [in]  is    The input stream that contains the transducer.

   @return It returns an Fsa on CPU.
*/
static Fsa K2TransducerFromStream(std::istringstream &is,
                                  Array1<int32_t> *aux_labels) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK(aux_labels != nullptr);

  std::vector<int32_t> aux_labels_internal;
  std::vector<Arc> arcs;
  std::vector<std::string> splits;
  std::string line;

  bool finished = false;  // when the final state is read, set it to true.
  while (std::getline(is, line)) {
    SplitStringToVector(line, kDelim,
                        &splits);  // splits is cleared in the function
    if (splits.empty()) continue;  // this is an empty line

    K2_CHECK_EQ(finished, false);

    auto num_fields = splits.size();
    if (num_fields == 5u) {
      //   0           1         2         3        4
      // src_state  dest_state label   aux_label  score
      int32_t src_state = StringToInt(splits[0]);
      int32_t dest_state = StringToInt(splits[1]);
      int32_t symbol = StringToInt(splits[2]);
      int32_t aux_label = StringToInt(splits[3]);
      float score = StringToFloat(splits[4]);
      arcs.emplace_back(src_state, dest_state, symbol, score);
      aux_labels_internal.push_back(aux_label);
    } else if (num_fields == 1u) {
      //   0
      // final_state
      (void)StringToInt(splits[0]);
      finished = true;  // set finish
    } else {
      K2_LOG(FATAL) << "Invalid line: " << line
                    << "\nk2 transducer expects a line with 1 (final_state) or "
                       "5 (src_state dest_state label aux_label score) fields";
    }
  }

  K2_CHECK_EQ(finished, true) << "The last line should be the final state";

  auto cpu_context = GetCpuContext();
  *aux_labels = Array1<int32_t>(cpu_context, aux_labels_internal);
  Array1<Arc> array(cpu_context, arcs);

  bool error = true;
  auto fsa = FsaFromArray1(array, &error);
  K2_CHECK_EQ(error, false);

  return fsa;
}

/* Create an acceptor from a stream, assuming the acceptor is in the OpenFST
   format:

   src_state1 dest_state1 label1 score1
   src_state2 dest_state2 label2 score2
   ... ...
   final_state final_score

   We will negate the cost/score when we read them in. Also note, OpenFST may
   omit the cost/score if it is 0.0.

   We always create the super final state. If there are final state(s) in the
   original FSA, then we add arc(s) from the original final state(s) to the
   super final state, with the (negated) old final state cost/score as its
   cost/score, and -1 as its label.

   @param [in]  is    The input stream that contains the acceptor.

   @return It returns an Fsa on CPU.
*/
static Fsa OpenFstAcceptorFromStream(std::istringstream &is) {
  NVTX_RANGE(K2_FUNC);
  std::vector<Arc> arcs;
  std::vector<std::vector<Arc>> state_to_arcs;  // indexed by states
  std::vector<std::string> splits;
  std::string line;

  int32_t max_state = -1;
  int32_t num_arcs = 0;
  std::vector<int32_t> original_final_states;
  std::vector<float> original_final_weights;
  while (std::getline(is, line)) {
    SplitStringToVector(line, kDelim,
                        &splits);  // splits is cleared in the function
    if (splits.empty()) continue;  // this is an empty line

    auto num_fields = splits.size();
    if (num_fields == 3u || num_fields == 4u) {
      //   0            1          2
      // src_state  dest_state   label
      //
      // or
      //
      //   0            1          2      3
      // src_state  dest_state   label  score
      int32_t src_state = StringToInt(splits[0]);
      int32_t dest_state = StringToInt(splits[1]);
      int32_t symbol = StringToInt(splits[2]);
      float score = 0.0f;
      if (num_fields == 4u) score = -1.0f * StringToFloat(splits[3]);

      // Add the arc to "state_to_arcs".
      ++num_arcs;
      max_state = std::max(max_state, std::max(src_state, dest_state));
      if (static_cast<int32_t>(state_to_arcs.size()) <= src_state)
        state_to_arcs.resize(src_state + 1);
      state_to_arcs[src_state].emplace_back(src_state, dest_state, symbol,
                                            score);
    } else if (num_fields == 1u || num_fields == 2u) {
      //   0            1
      // final_state  score
      float score = 0.0f;
      if (num_fields == 2u) score = -1.0f * StringToFloat(splits[1]);
      original_final_states.push_back(StringToInt(splits[0]));
      original_final_weights.push_back(score);
      max_state = std::max(max_state, original_final_states.back());
    } else {
      K2_LOG(FATAL) << "Invalid line: " << line
                    << "\nOpenFST acceptor expects a line with 1 (final_state),"
                       " 2 (final_state score), 3 (src_state dest_state label) "
                       "or 4 (src_state dest_state label score) fields.";
    }
  }

  K2_CHECK(is.eof());

  // Post processing on final states. If there are final state(s) in the
  // original FSA, we add the super final state as well as arc(s) from original
  // final state(s) to the super final state. Otherwise, the super final state
  // will be added by FsaFromArray1 (since there's no arc with label
  // kFinalSymbol).
  if (original_final_states.size() > 0) {
    K2_CHECK_EQ(original_final_states.size(), original_final_weights.size());
    int32_t super_final_state = max_state + 1;
    state_to_arcs.resize(super_final_state);
    for (std::size_t i = 0; i != original_final_states.size(); ++i) {
      state_to_arcs[original_final_states[i]].emplace_back(
          original_final_states[i], super_final_state,
          -1,  // kFinalSymbol
          original_final_weights[i]);
      ++num_arcs;
    }
  }

  // Move arcs from "state_to_arcs" to "arcs".
  int32_t arc_index = 0;
  arcs.resize(num_arcs);
  for (std::size_t s = 0; s < state_to_arcs.size(); ++s) {
    for (std::size_t a = 0; a < state_to_arcs[s].size(); ++a) {
      K2_CHECK_GT(num_arcs, arc_index);
      arcs[arc_index] = state_to_arcs[s][a];
      ++arc_index;
    }
  }
  K2_CHECK_EQ(num_arcs, arc_index);

  bool error = true;
  Array1<Arc> array(GetCpuContext(), arcs);
  // FsaFromArray1 will add a super final state if the original FSA doesn't have
  // a final state.
  auto fsa = FsaFromArray1(array, &error);
  K2_CHECK_EQ(error, false);

  return fsa;
}

/* Create a transducer from a stream, assuming the transducer is in the OpenFST
   format:

   src_state1 dest_state1 label1 aux_label1 score1
   src_state2 dest_state2 label2 aux_label2 score2
   ... ...
   final_state final_score

   We will negate the cost/score when we read them in. Also note, OpenFST may
   omit the cost/score if it is 0.0.

   We always create the super final state. If there are final state(s) in the
   original FST, then we add arc(s) from the original final state(s) to the
   super final state, with the (negated) old final state cost/score as its
   cost/score, -1 as its label and -1 as its aux_label.

   @param [in]  is    The input stream that contains the transducer.

   @return It returns an Fsa on CPU.
*/
static Fsa OpenFstTransducerFromStream(std::istringstream &is,
                                       Array1<int32_t> *aux_labels) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK(aux_labels != nullptr);

  std::vector<std::vector<int32_t>> state_to_aux_labels;  // indexed by states
  std::vector<std::vector<Arc>> state_to_arcs;            // indexed by states
  std::vector<int32_t> aux_labels_internal;
  std::vector<Arc> arcs;
  std::vector<std::string> splits;
  std::string line;

  int32_t max_state = -1;
  int32_t num_arcs = 0;
  std::vector<int32_t> original_final_states;
  std::vector<float> original_final_weights;
  while (std::getline(is, line)) {
    SplitStringToVector(line, kDelim,
                        &splits);  // splits is cleared in the function
    if (splits.empty()) continue;  // this is an empty line

    auto num_fields = splits.size();
    if (num_fields == 4u || num_fields == 5u) {
      //   0           1         2         3
      // src_state  dest_state label   aux_label
      //
      // or
      //
      //   0           1         2         3        4
      // src_state  dest_state label   aux_label  score
      int32_t src_state = StringToInt(splits[0]);
      int32_t dest_state = StringToInt(splits[1]);
      int32_t symbol = StringToInt(splits[2]);
      int32_t aux_label = StringToInt(splits[3]);
      float score = 0.0f;
      if (num_fields == 5u) score = -1.0f * StringToFloat(splits[4]);

      // Add the arc to "state_to_arcs", and aux_label to "state_to_aux_labels"
      ++num_arcs;
      max_state = std::max(max_state, std::max(src_state, dest_state));
      if (static_cast<int32_t>(state_to_arcs.size()) <= src_state) {
        state_to_arcs.resize(src_state + 1);
        state_to_aux_labels.resize(src_state + 1);
      }
      state_to_arcs[src_state].emplace_back(src_state, dest_state, symbol,
                                            score);
      state_to_aux_labels[src_state].push_back(aux_label);
    } else if (num_fields == 1u || num_fields == 2u) {
      //   0
      // final_state
      //
      // or
      //
      //   0            1
      // final_state  score
      // There could be multiple final states, so we first have to collect all
      // the final states, and then work out the super final state.
      float score = 0.0f;
      if (num_fields == 2u) score = -1.0f * StringToFloat(splits[1]);
      original_final_states.push_back(StringToInt(splits[0]));
      original_final_weights.push_back(score);
      max_state = std::max(max_state, original_final_states.back());
    } else {
      K2_LOG(FATAL) << "Invalid line: " << line
                    << "\nOpenFST transducer expects a line with "
                       "1 (final_state), 2 (final_state score), "
                       "4 (src_state dest_state label aux_label) or "
                       "5 (src_state dest_state label aux_label score) fields.";
    }
  }

  K2_CHECK(is.eof());

  // Post processing on final states. If there are final state(s) in the
  // original FST, we add the super final state as well as arc(s) from original
  // final state(s) to the super final state. Otherwise, the super final state
  // will be added by FsaFromArray1 (since there's no arc with label
  // kFinalSymbol).
  if (original_final_states.size() > 0) {
    K2_CHECK_EQ(original_final_states.size(), original_final_weights.size());
    int32_t super_final_state = max_state + 1;
    state_to_arcs.resize(super_final_state);
    state_to_aux_labels.resize(super_final_state);
    for (std::size_t i = 0; i != original_final_states.size(); ++i) {
      state_to_arcs[original_final_states[i]].emplace_back(
          original_final_states[i], super_final_state,
          -1,  // kFinalSymbol
          original_final_weights[i]);
      state_to_aux_labels[original_final_states[i]].push_back(
          -1);  // kFinalSymbol
      ++num_arcs;
    }
  }

  // Move arcs from "state_to_arcs" to "arcs", and aux_labels from
  // "state_to_aux_labels" to "aux_labels_internal"
  int32_t arc_index = 0;
  arcs.resize(num_arcs);
  aux_labels_internal.resize(num_arcs);
  K2_CHECK_EQ(state_to_arcs.size(), state_to_aux_labels.size());
  for (std::size_t s = 0; s < state_to_arcs.size(); ++s) {
    K2_CHECK_EQ(state_to_arcs[s].size(), state_to_aux_labels[s].size());
    for (std::size_t a = 0; a < state_to_arcs[s].size(); ++a) {
      K2_CHECK_GT(num_arcs, arc_index);
      arcs[arc_index] = state_to_arcs[s][a];
      aux_labels_internal[arc_index] = state_to_aux_labels[s][a];
      ++arc_index;
    }
  }
  K2_CHECK_EQ(num_arcs, arc_index);

  auto cpu_context = GetCpuContext();
  *aux_labels = Array1<int32_t>(cpu_context, aux_labels_internal);
  Array1<Arc> array(cpu_context, arcs);

  bool error = true;
  // FsaFromArray1 will add a super final state if the original FSA doesn't have
  // a final state.
  auto fsa = FsaFromArray1(array, &error);
  K2_CHECK_EQ(error, false);

  return fsa;
}

Fsa FsaFromString(const std::string &s, bool openfst /*= false*/,
                  Array1<int32_t> *aux_labels /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  std::istringstream is(s);
  K2_CHECK(is);

  if (openfst == false && aux_labels == nullptr)
    return K2AcceptorFromStream(is);
  else if (openfst == false && aux_labels != nullptr)
    return K2TransducerFromStream(is, aux_labels);
  else if (openfst == true && aux_labels == nullptr)
    return OpenFstAcceptorFromStream(is);
  else if (openfst == true && aux_labels != nullptr)
    return OpenFstTransducerFromStream(is, aux_labels);

  return Fsa();  // unreachable code
}

std::string FsaToString(const Fsa &fsa, bool openfst /*= false*/,
                        const Array1<int32_t> *aux_labels /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(fsa.NumAxes(), 2);

  if (fsa.Context()->GetDeviceType() != kCpu) {
    Fsa _fsa = fsa.To(GetCpuContext());
    Array1<int32_t> _aux_labels;
    if (aux_labels) _aux_labels = aux_labels->To(_fsa.Context());
    return FsaToString(_fsa, openfst, aux_labels ? &_aux_labels : nullptr);
  }

  K2_CHECK_EQ(fsa.Context()->GetDeviceType(), kCpu);
  const Array1<int32_t> &row_splits = fsa.shape.RowSplits(1);
  const Array1<Arc> &arcs = fsa.values;

  const int32_t *p = nullptr;
  if (aux_labels != nullptr) {
    K2_CHECK(IsCompatible(fsa, *aux_labels));
    K2_CHECK_EQ(aux_labels->Dim(), arcs.Dim());
    p = aux_labels->Data();
  }
  float scale = 1;
  if (openfst) scale = -1;

  std::ostringstream os;

  int32_t n = arcs.Dim();
  char sep = ' ';
  char line_sep = '\n';
  for (int32_t i = 0; i != n; ++i) {
    const auto &arc = arcs[i];
    os << arc.src_state << sep << arc.dest_state << sep << arc.label << sep;
    if (p != nullptr) os << p[i] << sep;
    os << (scale * arc.score) << line_sep;
  }
  os << (fsa.shape.Dim0() - 1) << line_sep;
  return os.str();
}

Array1<int32_t> GetDestStates(FsaVec &fsas, bool as_idx01) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  ContextPtr &c = fsas.Context();
  int32_t num_arcs = fsas.NumElements();
  Array1<int32_t> ans(c, num_arcs);
  const Arc *arcs_data = fsas.values.Data();
  int32_t *ans_data = ans.Data();
  if (!as_idx01) {
    K2_EVAL(
        c, num_arcs, lambda_set_dest_states1, (int32_t arc_idx012) {
          ans_data[arc_idx012] = arcs_data[arc_idx012].dest_state;
        });
  } else {
    const int32_t *row_ids2 = fsas.RowIds(2).Data();
    K2_EVAL(
        c, num_arcs, lambda_set_dest_states01, (int32_t arc_idx012) {
          int32_t src_state = arcs_data[arc_idx012].src_state,
                  dest_state = arcs_data[arc_idx012].dest_state;
          // (row_ids2[arc_idx012] - src_state) is the same as
          // row_splits1[row_ids1[row_ids2[arc_idx012]]]; it's the idx01 of the
          // 1st state in this FSA.
          ans_data[arc_idx012] =
              dest_state + (row_ids2[arc_idx012] - src_state);
        });
  }
  return ans;
}

Ragged<int32_t> GetStateBatches(FsaVec &fsas, bool transpose) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  ContextPtr &c = fsas.Context();
  Array1<int32_t> arc_dest_states = GetDestStates(fsas, true);

  MonotonicLowerBound(arc_dest_states, &arc_dest_states);

  int32_t num_fsas = fsas.Dim0(), num_states = fsas.TotSize(1),
          num_arcs = fsas.TotSize(2);

  // We can tune `log_power` as a tradeoff between work done and clock time on
  // GPU.
  int32_t log_power = (c->GetDeviceType() == kCpu ? 0 : 4);

  int32_t max_num_states = fsas.shape.MaxSize(1);
  // the following avoids doing too much extra work accumulating powers
  // of 'dest_states' for very small problem sizes.
  while (log_power > 0 && (1 << (1 + log_power)) > max_num_states) log_power--;

  // Ignoring edge effects: `dest_states_powers[0]` is just an array indexed by
  // state_idx01, that gives us the dest_state_idx01 that would be the beginning
  // of the next batch if state_idx01 were the beginning of the current batch.
  // So if we follow this chain forward from the start of one of the FSAs until
  // it passes the end of this FSA, we get the beginnings of the batches
  // we want.  The natural algorithm to find the beginnings of the batches
  // is sequential.
  Array2<int32_t> dest_states_powers(c, log_power + 1, num_states);
  const int32_t *arc_dest_states_data = arc_dest_states.Data(),
                *fsas_row_splits2_data = fsas.RowSplits(2).Data();
  int32_t *dest_states_power_data =
      dest_states_powers.Data();  // only process Row[0] below
  const int32_t int_max = std::numeric_limits<int32_t>::max();
  K2_EVAL(
      c, num_states, lambda_set_dest_states, (int32_t state_idx01)->void {
        int32_t arc_idx01x = fsas_row_splits2_data[state_idx01];
        // If this state has arcs, let its `dest_state` be the smallest
        // `dest_state` of any of its arcs (which is the first element of those
        // arcs' dest states in `arc_dest_states_data`); otherwise, take the
        // `dest_state` from the 1st arc of the next state, which is the largest
        // value we can take (if the definition is: the highest-numbered state s
        // for which neither this state nor any later-numbered state has an arc
        // to a state lower than s).

        // if this state has arcs,
        //    arc_idx01x is the first arc index of this state, we get the
        //    smallest dest state of this state's arcs using
        //    arc_dest_states_data[arc_idx01x]
        // else
        //    arc_idx01x is the first arc index of the next state, then
        //    arc_dest_states_data[arc_idx01x] is the largest value we can take,
        //    which is also the smallest dest state in the next state.
        int32_t dest_state =
            (arc_idx01x < num_arcs ? arc_dest_states_data[arc_idx01x]
                                   : int_max);
        dest_states_power_data[state_idx01] = dest_state;
        // if the following fails, it's either a code error or the input FSA had
        // cycles.
        K2_CHECK_GT(dest_state, state_idx01);
      });

  // `num_batches_per_fsa` will be set to the number of batches of states that
  // we'll use for each FSA... it corresponds to the number of times we have
  // to follow links forward in the dest_states array till we pass the
  // end of the array for this fSA.
  Array1<int32_t> num_batches_per_fsa(c, num_fsas + 1, 0);

  // `batch_starts` will contain the locations of the first state_idx01 for each
  // batch, but in an 'un-consolidated' format.  Specifically, for FSA with
  // index i, the batch_starts for that FSA begin at element fsa.RowSplits(1)[i]
  // of `batch_starts`.  This is just a convenient layout because we know there
  // can't be more batches than there are states.  We'll later consolidate the
  // information into a single array.
  Array1<int32_t> batch_starts(c, num_states + 1);

  int32_t *num_batches_per_fsa_data = num_batches_per_fsa.Data(),
          *batch_starts_data = batch_starts.Data();
  const int32_t *fsas_row_splits1_data = fsas.RowSplits(1).Data();

#if 0
  // This is a simple version of the kernel that demonstrates what we're trying
  // to do with the more complex code.
  K2_EVAL(
      c, num_fsas, lambda_set_batch_info_simple, (int32_t fsa_idx) {
        int32_t begin_state_idx01 = fsas_row_splits1_data[fsa_idx],
                end_state_idx01 = fsas_row_splits1_data[fsa_idx + 1];
        int32_t i = 0, cur_state_idx01 = begin_state_idx01;
        while (cur_state_idx01 < end_state_idx01) {
          batch_starts_data[begin_state_idx01 + i] = cur_state_idx01;
          cur_state_idx01 = dest_states_power_data[cur_state_idx01];
          ++i;
        }
        num_batches_per_fsa_data[fsa_idx] = i;
      });
#else
  int32_t stride = dest_states_powers.ElemStride0();
  for (int32_t power = 1; power <= log_power; power++) {
    const int32_t *src_data = dest_states_powers.Data() + (power - 1) * stride;
    int32_t *dest_data = dest_states_powers.Data() + power * stride;
    K2_EVAL(
        c, num_states, lambda_square_array, (int32_t state_idx01)->void {
          int32_t dest_state = src_data[state_idx01],
                  dest_state_sq =
                      (dest_state < num_states ? src_data[dest_state]
                                               : int_max);
          dest_data[state_idx01] = dest_state_sq;
        });
  }
  // jobs_per_fsa tells us how many separate chains of states we'll follow for
  // each FSA.
  // jobs_multiple is a kind of trick to ensure any given warp doesn't
  // issue more memory requests than it can handle at a time (we drop
  // some threads).
  int32_t jobs_per_fsa = (1 << log_power),
          jobs_multiple = (c->GetDeviceType() == kCuda ? 8 : 1);
  while (jobs_multiple > 1 && jobs_per_fsa * jobs_multiple * num_fsas > 10000)
    jobs_multiple /= 2;  // Likely won't get here.  Just reduce multiple if
                         // num-jobs is ridiculous.

  auto dest_states_powers_acc = dest_states_powers.Accessor();
  K2_EVAL2(
      c, num_fsas, jobs_per_fsa * jobs_multiple, lambda_set_batch_info,
      (int32_t fsa_idx, int32_t j) {
        if (j % jobs_multiple != 0)
          return;  // a trick to avoid too much random
                   // memory access for any given warp
        int32_t task_idx =
            j / jobs_multiple;  // Now 0 <= task_idx < jobs_per_fsa.

        // The task indexed `task_idx` is responsible for batches numbered
        // task_idx, task_idx + jobs_per_fsa, task_index + 2 * job_per_fsa and
        // so on, for the FSA numbered `fsa_idx`. Comparing this code to
        // `lambda_set_batch_info_simple`, this task is responsible for the
        // assignment to batch_starts_data for all i such that i % jobs_per_fsas
        // == task_idx, together with the assignment to
        // num_batchess_per_fsa_data if
        //  i % jobs_per_fsas == task_idx (here referring to the i value finally
        // assigned to that location).

        int32_t begin_state_idx01 = fsas_row_splits1_data[fsa_idx],
                end_state_idx01 = fsas_row_splits1_data[fsa_idx + 1];
        int32_t num_states_this_fsa = end_state_idx01 - begin_state_idx01;
        int32_t i = 0, cur_state_idx01 = begin_state_idx01;

        if (task_idx >= num_states_this_fsa) return;

        // The next loop advances `cur_state_idx01` by
        // a number of steps equal to `task_idx`.
        for (int32_t m = 0; m < log_power; ++m) {
          int32_t n = 1 << m;
          if ((task_idx & n) != 0) {
            i += n;
            int32_t next = dest_states_powers_acc(m, cur_state_idx01);
            if (next >= end_state_idx01) return;
            cur_state_idx01 = next;
          }
        }
        K2_CHECK_EQ(i, task_idx);

        while (1) {
          if (i >= num_states_this_fsa) return;
          batch_starts_data[begin_state_idx01 + i] = cur_state_idx01;
          int32_t next_state_idx01 = dest_states_powers_acc(
              log_power,
              cur_state_idx01);  // advance jobs_per_fsa = (1 << log_power)
                                 // steps
          if (next_state_idx01 >= end_state_idx01) {
            // if exactly one step would also be enough to take us past the
            // boundary...
            if (dest_states_powers_acc(0, cur_state_idx01) >= end_state_idx01) {
              num_batches_per_fsa_data[fsa_idx] = i + 1;
            }
            return;
          } else {
            i += jobs_per_fsa;
            cur_state_idx01 = next_state_idx01;
          }
        }
      });
#endif
  ExclusiveSum(num_batches_per_fsa, &num_batches_per_fsa);
  Array1<int32_t> &ans_row_splits1 = num_batches_per_fsa;
  int32_t num_batches = num_batches_per_fsa[num_fsas];
  Array1<int32_t> ans_row_ids1(c, num_batches);
  RowSplitsToRowIds(ans_row_splits1, &ans_row_ids1);
  Array1<int32_t> ans_row_splits2(c, num_batches + 1);
  const int32_t *ans_row_splits1_data = ans_row_splits1.Data(),
                *ans_row_ids1_data = ans_row_ids1.Data();
  int32_t *ans_row_splits2_data = ans_row_splits2.Data();
  ans_row_splits2.Range(num_batches, 1) = num_states;  // The kernel below won't
                                                       // set this last element
  K2_EVAL(
      c, num_batches, lambda_set_ans_row_splits2, (int32_t idx01)->void {
        int32_t idx0 = ans_row_ids1_data[idx01],  // Fsa index
            idx0x = ans_row_splits1_data[idx0], idx1 = idx01 - idx0x,
                fsas_idx0x =
                    fsas_row_splits1_data[idx0];  // 1st state-idx (idx01)
                                                  // in fsas_, for this FSA

        int32_t fsas_idx01 =
            fsas_idx0x + idx1;  // the idx1 is actually the
                                // batch-index, this statement
                                // reflects the 'un-consolidated'
                                // format of `batch_starts`.

        int32_t this_batch_start = batch_starts_data[fsas_idx01];
        ans_row_splits2_data[idx01] = this_batch_start;
      });

  RaggedShape ans_shape =
      RaggedShape3(&ans_row_splits1, &ans_row_ids1, num_batches,
                   &ans_row_splits2, nullptr, num_states);
  Array1<int32_t> ans_value = Range(c, num_states, 0);
  if (transpose) {
    ans_shape = MakeTransposable(ans_shape);
    Ragged<int32_t> ans(ans_shape, ans_value);
    return Transpose(ans);
  } else {
    return Ragged<int32_t>(ans_shape, ans_value);
  }
}

Ragged<int32_t> GetIncomingArcs(FsaVec &fsas,
                                const Array1<int32_t> &dest_states) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  K2_CHECK(IsCompatible(fsas, dest_states));
  ContextPtr &c = fsas.Context();
  Ragged<int32_t> dest_states_tensor(fsas.shape, dest_states);
  int32_t num_fsas = fsas.Dim0(), num_states = fsas.TotSize(1),
          num_arcs = fsas.TotSize(2);

  Array1<int32_t> incoming_arcs_order =
                      GetTransposeReordering(dest_states_tensor, num_states),

                  ans_row_ids2 = dest_states[incoming_arcs_order];
  // Note: incoming_arcs_row_ids2 will be monotonically increasing

  Array1<int32_t> ans_row_splits2(c, num_states + 1);
  RowIdsToRowSplits(ans_row_ids2, &ans_row_splits2);

  // Axis 1 corresponds to FSA states, so the row-ids and row-splits for axis
  // 1 are the same as for `fsas`.
  Array1<int32_t> ans_row_ids1 = fsas.RowIds(1),
                  ans_row_splits1 = fsas.RowSplits(1);
  return Ragged<int32_t>(
      RaggedShape3(&ans_row_splits1, &ans_row_ids1, num_states,
                   &ans_row_splits2, &ans_row_ids2, num_arcs),
      incoming_arcs_order);
}

Ragged<int32_t> GetLeavingArcIndexBatches(FsaVec &fsas,
                                          Ragged<int32_t> &state_batches) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK(IsCompatible(fsas, state_batches));
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  K2_CHECK_EQ(state_batches.NumAxes(), 3);
  ContextPtr &c = fsas.Context();
  int32_t num_fsas = fsas.Dim0(), num_states = fsas.TotSize(1),
          num_arcs = fsas.TotSize(2);
  int32_t num_batches = state_batches.Dim0();
  K2_DCHECK(state_batches.TotSize(1) == num_fsas * num_batches);
  K2_DCHECK_EQ(state_batches.NumElements(), num_states);

  // get ans_shape
  Array1<int32_t> ans_row_splits3(c, num_states + 1);
  int32_t *ans_row_splits3_data = ans_row_splits3.Data();
  const int32_t *fsa_states_row_splits_data = fsas.RowSplits(2).Data();
  const int32_t *batch_states_data = state_batches.values.Data();
  K2_EVAL(
      c, num_states, lambda_set_ans_row_splits3, (int32_t idx) {
        int32_t state_idx = batch_states_data[idx];
        ans_row_splits3_data[idx] = fsa_states_row_splits_data[state_idx + 1] -
                                    fsa_states_row_splits_data[state_idx];
      });
  ExclusiveSum(ans_row_splits3, &ans_row_splits3);
  Array1<int32_t> ans_row_ids3(c, num_arcs);
  RowSplitsToRowIds(ans_row_splits3, &ans_row_ids3);
  RaggedShape ans_shape = ComposeRaggedShapes(
      state_batches.shape,
      RaggedShape2(&ans_row_splits3, &ans_row_ids3, num_arcs));

  // get ans_values
  Array1<int32_t> ans_values(c, num_arcs);
  int32_t *ans_values_data = ans_values.Data();
  const int32_t *ans_row_ids3_data = ans_row_ids3.Data();
  K2_EVAL(
      c, num_arcs, lambda_set_ans_values, (int32_t idx0123) {
        int32_t ans_idx012 = ans_row_ids3_data[idx0123];
        int32_t state_idx =
            batch_states_data[ans_idx012];  // state_idx is idx01 in fsas
        int32_t fsa_idx01x = fsa_states_row_splits_data[state_idx];
        // ans_idx3 is fsas_idx2, i.e. the arc idx in a state
        int32_t ans_idx3 = idx0123 - ans_row_splits3_data[ans_idx012];
        ans_values_data[idx0123] = fsa_idx01x + ans_idx3;
      });

  return Ragged<int32_t>(ans_shape, ans_values);
}

Ragged<int32_t> GetEnteringArcIndexBatches(FsaVec &fsas,
                                           Ragged<int32_t> &incoming_arcs,
                                           Ragged<int32_t> &state_batches) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK(IsCompatible(fsas, state_batches));
  K2_CHECK(IsCompatible(fsas, incoming_arcs));
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  K2_CHECK_EQ(incoming_arcs.NumAxes(), 3);
  K2_CHECK_EQ(state_batches.NumAxes(), 3);
  ContextPtr &c = fsas.Context();
  int32_t num_fsas = fsas.Dim0(), num_states = fsas.TotSize(1),
          num_arcs = fsas.TotSize(2);
  int32_t num_batches = state_batches.Dim0();
  // just using DCHECK below to save time in production code
  K2_DCHECK(state_batches.TotSize(1) == num_fsas * num_batches);
  K2_DCHECK_EQ(state_batches.NumElements(), num_states);
  K2_DCHECK_EQ(incoming_arcs.Dim0(), num_fsas);
  K2_DCHECK_EQ(incoming_arcs.TotSize(1), num_states);
  K2_DCHECK_EQ(incoming_arcs.NumElements(), num_arcs);

  // get ans_shape
  Array1<int32_t> ans_row_splits3(c, num_states + 1);
  int32_t *ans_row_splits3_data = ans_row_splits3.Data();
  const int32_t *incoming_arcs_row_splits_data =
      incoming_arcs.RowSplits(2).Data();
  const int32_t *batch_states_data = state_batches.values.Data();
  K2_EVAL(
      c, num_states, lambda_set_ans_row_splits3, (int32_t idx) {
        int32_t state_idx = batch_states_data[idx];
        ans_row_splits3_data[idx] =
            incoming_arcs_row_splits_data[state_idx + 1] -
            incoming_arcs_row_splits_data[state_idx];
      });
  ExclusiveSum(ans_row_splits3, &ans_row_splits3);
  Array1<int32_t> ans_row_ids3(c, num_arcs);
  RowSplitsToRowIds(ans_row_splits3, &ans_row_ids3);
  RaggedShape ans_shape = ComposeRaggedShapes(
      state_batches.shape,
      RaggedShape2(&ans_row_splits3, &ans_row_ids3, num_arcs));

  // get ans_values
  Array1<int32_t> ans_values(c, num_arcs);
  int32_t *ans_values_data = ans_values.Data();
  const int32_t *ans_row_ids3_data = ans_row_ids3.Data();
  const int32_t *incoming_arcs_data = incoming_arcs.values.Data();
  K2_EVAL(
      c, num_arcs, lambda_set_ans_values, (int32_t idx0123) {
        int32_t ans_idx012 = ans_row_ids3_data[idx0123];
        int32_t state_idx =
            batch_states_data[ans_idx012];  // state_idx is idx01 in
                                            // incoming_arcs
        int32_t incoming_arcs_idx01x = incoming_arcs_row_splits_data[state_idx];
        // ans_idx3 is incoming_arcs_idx2, i.e. the entering arc idx for a state
        int32_t ans_idx3 = idx0123 - ans_row_splits3_data[ans_idx012];
        int32_t incoming_arcs_idx012 = incoming_arcs_idx01x + ans_idx3;
        ans_values_data[idx0123] = incoming_arcs_data[incoming_arcs_idx012];
      });

  return Ragged<int32_t>(ans_shape, ans_values);
}

FsaVec ConvertDenseToFsaVec(DenseFsaVec &src) {
  NVTX_RANGE(K2_FUNC);
  ContextPtr &c = src.shape.Context();
  // caution: 'num_symbols' is the number of symbols excluding the final-symbol
  // -1.
  int32_t num_fsas = src.shape.Dim0(), num_symbols = src.scores.Dim1() - 1;
  // the "1" is the extra state per FSA we need in the FsaVec format,
  // for the final-state.
  RaggedShape fsa2state = ChangeSublistSize(src.shape, 1);
  // again, the "+num_fsas" below is the extra state per FSA we need in the
  // FsaVec format, for the final-state.
  int32_t num_states = src.shape.NumElements() + num_fsas;
  // The explanation num-arcs below is as follows:
  // Firstly, all rows of src.scores (==all elements of src.shape) correspond
  // to states with arcs leaving them.  Most of them have `num_symbols` arcs,
  // but the final one for each FSA has 1 arc (with symbol -1)
  int32_t num_arcs =
      src.shape.NumElements() * num_symbols - (num_symbols - 1) * num_fsas;
  Array1<int32_t> row_splits2(c, num_states + 1), row_ids2(c, num_arcs);
  const int32_t *row_ids1_data = fsa2state.RowIds(1).Data(),
                *src_row_ids1_data = src.shape.RowIds(1).Data(),
                *src_row_splits1_data = src.shape.RowSplits(1).Data();
  Array1<Arc> arcs(c, num_arcs);
  Arc *arcs_data = arcs.Data();

  auto scores_acc = src.scores.Accessor();

  int32_t *row_splits2_data = row_splits2.Data(),
          *row_ids2_data = row_ids2.Data();

  // 0 <= s < num_symbols; note, `num_symbols` excludes the final-symbol (-1).
  // note: `src` means: w.r.t. the numbering in the original DenseFsaVec.
  K2_EVAL2(
      c, src.shape.NumElements(), num_symbols, lambda_set_arcs_etc,
      (int32_t src_state_idx01, int32_t s)->void {
        int32_t fsa_idx0 = src_row_ids1_data[src_state_idx01],
                src_state_idx0x = src_row_splits1_data[fsa_idx0],
                state_idx1 = src_state_idx01 - src_state_idx0x,
                src_next_state_idx0x = src_row_splits1_data[fsa_idx0 + 1],
                src_num_states1 = src_next_state_idx0x - src_state_idx0x,
                ans_state_idx01 = src_state_idx01 +
                                  fsa_idx0;  // we add one final-state per FSA..
                                             // "+ fsa_idx0" gives the
                                             // difference from old->new
                                             // numbering.

        // arc_idx0xx is the 1st arc-index of the FSA we are creating.. each
        // source state has `num_symbols` arcs leaving it except the last one of
        // each FSA, which has 1 arc leaving it (to the final-state).
        int32_t arc_idx0xx = (src_state_idx0x * num_symbols) -
                             fsa_idx0 * (num_symbols - 1),
                arc_idx01x = arc_idx0xx + (state_idx1 * num_symbols),
                arc_idx012 = arc_idx01x + s;
        int32_t symbol_offset;
        if (state_idx1 + 1 == src_num_states1) {
          symbol_offset = -1;
          if (s > 0) return;  // we just need the arc with -1.

          // if this is the state before the final state of this FSA. it has the
          // responsibility to write the row_splits2 value for the final state.
          // It's arc_idx012 + 1; the "+1" corresponds to the single arc with
          // the final-symbol on it.
          row_splits2_data[ans_state_idx01 + 1] = arc_idx012 + 1;
        } else {
          symbol_offset = 0;
        }
        // the "+ 1" is because index 0 in `scores` is for the final-symbol -1,
        // then 0, 1, etc.
        int32_t symbol_index_in_scores = s + symbol_offset + 1;
        arcs_data[arc_idx012] =
            Arc(state_idx1, state_idx1 + 1, s + symbol_offset,
                scores_acc(src_state_idx01, symbol_index_in_scores));
        row_ids2_data[arc_idx012] = ans_state_idx01;
        if (s == 0) {  // 1st arc for this state.
          row_splits2_data[ans_state_idx01] = arc_idx012;
          K2_CHECK(row_ids1_data[ans_state_idx01] == fsa_idx0);
          if (src_state_idx01 == 0) row_splits2_data[num_states] = num_arcs;
        }
      });

  RaggedShape state2arc = RaggedShape2(&row_splits2, &row_ids2, num_arcs);
  return Ragged<Arc>(ComposeRaggedShapes(fsa2state, state2arc), arcs);
}

template <typename FloatType>
Array1<FloatType> GetForwardScores(FsaVec &fsas, Ragged<int32_t> &state_batches,
                                   Ragged<int32_t> &entering_arc_batches,
                                   bool log_semiring,
                                   Array1<int32_t> *entering_arcs) {
  NVTX_RANGE(K2_FUNC);
  K2_STATIC_ASSERT((std::is_same<float, FloatType>::value ||
                    std::is_same<double, FloatType>::value));
  K2_CHECK(IsCompatible(fsas, state_batches));
  K2_CHECK(IsCompatible(fsas, entering_arc_batches));
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  K2_CHECK_EQ(state_batches.NumAxes(), 3);
  K2_CHECK_EQ(entering_arc_batches.NumAxes(), 4);
  ContextPtr &c = fsas.Context();
  int32_t num_fsas = fsas.Dim0(), num_states = fsas.TotSize(1),
          num_arcs = fsas.TotSize(2);
  int32_t num_batches = state_batches.Dim0();
  // just using DCHECK below to save time in production code
  K2_DCHECK(state_batches.TotSize(1) == num_fsas * num_batches);
  K2_DCHECK_EQ(state_batches.NumElements(), num_states);
  K2_DCHECK_EQ(entering_arc_batches.Dim0(), num_batches);
  K2_DCHECK_EQ(entering_arc_batches.TotSize(1), state_batches.TotSize(1));
  K2_DCHECK_EQ(entering_arc_batches.TotSize(2), num_states);
  K2_DCHECK_EQ(entering_arc_batches.NumElements(), num_arcs);

  FloatType negative_infinity = -std::numeric_limits<FloatType>::infinity();
  Array1<FloatType> state_scores(c, num_states, negative_infinity);
  FloatType *state_scores_data = state_scores.Data();
  // set the score of start state in each fsa to be 0
  const int32_t *fsa_row_splits1 = fsas.RowSplits(1).Data();
  K2_EVAL(
      c, num_fsas, lambda_set_start_state_score, (int32_t fsa_idx) {
        int32_t start_state = fsa_row_splits1[fsa_idx],
                start_state_next_fsa = fsa_row_splits1[fsa_idx + 1];
        if (start_state_next_fsa - start_state > 0)
          state_scores_data[start_state] = 0;
      });

  // get the 1st entering arc index in each batch, +1 so we can get the number
  // of entering arcs in each batch by taking the difference of adjacent
  // elements
  Array1<int32_t> entering_arc_start_index(c, num_batches + 1);
  int32_t *entering_arc_start_index_data = entering_arc_start_index.Data();
  const int32_t *arc_batches_row_splits1 =
      entering_arc_batches.RowSplits(1).Data();
  const int32_t *arc_batches_row_splits2 =
      entering_arc_batches.RowSplits(2).Data();
  const int32_t *arc_batches_row_splits3 =
      entering_arc_batches.RowSplits(3).Data();
  K2_EVAL(
      c, num_batches, lambda_set_entering_arc_start_index, (int32_t batch_idx) {
        int32_t this_state_idx0xx =
            arc_batches_row_splits2[batch_idx * num_fsas];
        int32_t this_arc_idx0xxx = arc_batches_row_splits3[this_state_idx0xx];
        entering_arc_start_index_data[batch_idx] = this_arc_idx0xxx;
        if (batch_idx == num_batches - 1) {
          // process the last element
          int32_t next_state_idx0xx =
              arc_batches_row_splits2[num_batches * num_fsas];
          int32_t next_arc_idx0xxx = arc_batches_row_splits3[next_state_idx0xx];
          entering_arc_start_index_data[num_batches] = next_arc_idx0xxx;
        }
      });

  const int32_t *arc_batches_row_ids1 = entering_arc_batches.RowIds(1).Data();
  const int32_t *arc_batches_row_ids2 = entering_arc_batches.RowIds(2).Data();
  const int32_t *arc_batches_row_ids3 = entering_arc_batches.RowIds(3).Data();
  const int32_t *entering_arc_ids = entering_arc_batches.values.Data();
  const int32_t *states_data = state_batches.values.Data();
  const Arc *arcs = fsas.values.Data();
  Array1<FloatType> entering_arc_score_values(
      c, num_arcs);  // entering arc_scores in batches
  FloatType *arc_scores_data = entering_arc_score_values.Data();
  // copy entering_arc_start_index to cpu as we will access its elements in
  // below Eval function for `lambda_set_entering_arc_scores`
  Array1<int32_t> cpu_entering_arc_start_index =
      entering_arc_start_index.To(GetCpuContext());
  const int32_t *cpu_entering_arc_start = cpu_entering_arc_start_index.Data();
  // copy the index of start state in each fsa to CPU
  Array1<int32_t> &arc_batches_row_splits1_array =
      entering_arc_batches.RowSplits(1);
  Array1<int32_t> arc_batches_row_splits12_cpu =
      entering_arc_batches.RowSplits(2)[arc_batches_row_splits1_array].To(
          GetCpuContext());
  K2_CHECK_EQ(arc_batches_row_splits12_cpu.Dim(), num_batches + 1);
  const int32_t *arc_batches_row_splits12_cpu_data =
      arc_batches_row_splits12_cpu.Data();
  Array1<int32_t> arc_row_splits_mem(c, num_states + 1);
  Array1<FloatType> score_cache(c, num_states + 1);

  int32_t *entering_arcs_data = nullptr;
  if (entering_arcs) {
    K2_CHECK_EQ(log_semiring, false) << " entering_arcs supplied";
    *entering_arcs = Array1<int32_t>(c, num_states, -1);
    entering_arcs_data = entering_arcs->Data();
  }

  // process batch sequentially.
  for (int32_t i = 0; i < num_batches; ++i) {
    // get the range we would call Max/LogSum per sub list
    int32_t this_state_idx0xx = arc_batches_row_splits12_cpu_data[i],
            next_state_idx0xx = arc_batches_row_splits12_cpu_data[i + 1];
    K2_CHECK_LT(this_state_idx0xx, num_states);
    K2_CHECK_LE(next_state_idx0xx, num_states);
    int32_t num_states_this_batch = next_state_idx0xx - this_state_idx0xx;
    K2_CHECK_LT(num_states_this_batch, arc_row_splits_mem.Dim());
    // we always use the first `num_states_this_batch` elements in
    // arc_row_splits_mem.
    Array1<int32_t> arc_row_splits_part = arc_row_splits_mem.Range(
        0, num_states_this_batch + 1);  // +1 for the last element
    int32_t num_arcs_this_batch =
        cpu_entering_arc_start[i + 1] - cpu_entering_arc_start[i];
    {
      ParallelRunner pr(c);
      // get entering arc scores
      {
        With w(pr.NewStream());
        K2_EVAL(
            c, num_arcs_this_batch, lambda_set_entering_arc_score,
            (int32_t idx123) {
              // all idx** in below code are the indexes to entering_arc_batches
              int32_t idx0123 = entering_arc_start_index_data[i] + idx123;
              int32_t idx012 = arc_batches_row_ids3[idx0123];
              int32_t idx01 = arc_batches_row_ids2[idx012];
              K2_CHECK_EQ(idx01 / num_fsas, i);  // idx01/num_fsas is batch_id
              int32_t fsa_id = idx01 % num_fsas;

              int32_t entering_arc_id = entering_arc_ids[idx0123];
              float curr_arc_score = arcs[entering_arc_id].score;
              int32_t src_state_idx1 = arcs[entering_arc_id].src_state;
              int32_t src_state_idx01 =
                  fsa_row_splits1[fsa_id] + src_state_idx1;
              arc_scores_data[idx0123] =
                  state_scores_data[src_state_idx01] + curr_arc_score;
            });
      }
      {
        With w(pr.NewStream());
        // make entering arc row splits info in each batch starting from zero,
        // we will use it to call MaxPerSublist or LogSumPerSubList
        int32_t *sum_splits_data = arc_row_splits_part.Data();
        K2_EVAL(
            c, num_states_this_batch + 1, lambda_set_row_splits_for_sum,
            (int32_t idx) {
              sum_splits_data[idx] =
                  arc_batches_row_splits3[idx + this_state_idx0xx] -
                  arc_batches_row_splits3[this_state_idx0xx];
            });
      }
    }
    int32_t this_arc_idx0xxx = cpu_entering_arc_start[i];
    Array1<FloatType> sub_scores_values =
        entering_arc_score_values.Range(this_arc_idx0xxx, num_arcs_this_batch);
    RaggedShape sub_scores_shape =
        RaggedShape2(&arc_row_splits_part, nullptr, sub_scores_values.Dim());
    Ragged<FloatType> sub_scores(sub_scores_shape, sub_scores_values);
    // we always use the first num_rows elements in score_cache.
    Array1<FloatType> sub_state_scores =
        score_cache.Range(0, num_states_this_batch);
    // get scores per state in this batch
    if (log_semiring) {
      LogSumPerSublist(sub_scores, negative_infinity, &sub_state_scores);
    } else {
      MaxPerSublist(sub_scores, negative_infinity, &sub_state_scores);
      if (entering_arcs_data != nullptr) {
        FloatType *sub_state_scores_data = sub_state_scores.Data(),
                  *sub_scores_data = sub_scores.values.Data();
        int32_t *sub_scores_row_ids_data = sub_scores.RowIds(1).Data();
        const int32_t *sub_state_ids_data = states_data + this_state_idx0xx,
                      *sub_entering_arc_ids_data =
                          entering_arc_ids + this_arc_idx0xxx;
        // arc_idx01 below is an index into sub_scores, it is also an arc_idx123
        // into entering_arc_batches.
        K2_EVAL(
            c, sub_scores.NumElements(), lambda_set_entering_arcs,
            (int32_t arc_idx01) {
              // state_idx0 below is idx0 into `sub_scores`, also an index into
              // `sub_scores`.
              int32_t state_idx0 = sub_scores_row_ids_data[arc_idx01];
              if (sub_scores_data[arc_idx01] ==
                  sub_state_scores_data[state_idx0]) {
                int32_t fsas_state_idx01 = sub_state_ids_data[state_idx0],
                        fsas_entering_arc_idx012 =
                            sub_entering_arc_ids_data[arc_idx01];
                // The following statement has a race condition if there is a
                // tie on scores, but this is OK and by design.  It makes the
                // choice of traceback non-deterministic in these cases.
                entering_arcs_data[fsas_state_idx01] = fsas_entering_arc_idx012;
              }
            });
      }
    }
    const FloatType *sub_state_scores_data = sub_state_scores.Data();
    // Copy those scores to corresponding state in state_scores.
    // `state_idx12` is an idx12 w.r.t. state_batches and entering_arc_batches,
    // but an idx1 w.r.t. sub_scores and an index into the array
    // sub_state_scores.
    K2_EVAL(
        c, num_states_this_batch, lambda_copy_state_scores,
        (int32_t state_idx12) {
          int32_t batches_idx012 = this_state_idx0xx + state_idx12;
          int32_t fsas_state_idx01 = states_data[batches_idx012];
          int32_t batches_idx01 = arc_batches_row_ids2[batches_idx012];
          int32_t fsa_idx0 = batches_idx01 % num_fsas;
          int32_t start_state_idx01 = fsa_row_splits1[fsa_idx0];
          // don't override score 0 in the start state in each fsa.
          if (fsas_state_idx01 != start_state_idx01)
            state_scores_data[fsas_state_idx01] =
                sub_state_scores_data[state_idx12];
        });
  }

  return state_scores;
}

template <typename FloatType>
Array1<FloatType> GetBackwardScores(
    FsaVec &fsas, Ragged<int32_t> &state_batches,
    Ragged<int32_t> &leaving_arc_batches,
    const Array1<FloatType> *tot_scores /*= nullptr*/,
    bool log_semiring /*= true*/) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK(IsCompatible(fsas, state_batches));
  K2_CHECK(IsCompatible(fsas, leaving_arc_batches));
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  K2_CHECK_EQ(state_batches.NumAxes(), 3);
  K2_CHECK_EQ(leaving_arc_batches.NumAxes(), 4);
  ContextPtr &c = fsas.Context();
  int32_t num_fsas = fsas.Dim0(), num_states = fsas.TotSize(1),
          num_arcs = fsas.TotSize(2);
  int32_t num_batches = state_batches.Dim0();
  K2_DCHECK(state_batches.TotSize(1) == num_fsas * num_batches);
  // just using DCHECK below to save time in production code
  K2_DCHECK_EQ(state_batches.NumElements(), num_states);
  K2_DCHECK_EQ(leaving_arc_batches.Dim0(), num_batches);
  K2_DCHECK_EQ(leaving_arc_batches.TotSize(1), state_batches.TotSize(1));
  K2_DCHECK_EQ(leaving_arc_batches.TotSize(2), num_states);
  K2_DCHECK_EQ(leaving_arc_batches.NumElements(), num_arcs);

  FloatType negative_infinity = -std::numeric_limits<FloatType>::infinity();
  Array1<FloatType> state_scores(c, num_states, negative_infinity);
  FloatType *state_scores_data = state_scores.Data();
  const int32_t *fsa_row_splits1 = fsas.RowSplits(1).Data();
  if (tot_scores != nullptr) {
    K2_CHECK(IsCompatible(fsas, *tot_scores));
    K2_CHECK_EQ(tot_scores->Dim(), num_fsas);
    const FloatType *tot_scores_data = tot_scores->Data();
    // set the score of final state in fsa i to be negative of tot_scores[i]
    K2_EVAL(
        c, num_fsas, lambda_set_final_state_score, (int32_t fsa_idx) {
          int32_t start_state = fsa_row_splits1[fsa_idx],
                  start_state_next_fsa = fsa_row_splits1[fsa_idx + 1];
          if (start_state_next_fsa - start_state > 0) {
            // We never set the score of a state to positive_infinity, otherwise
            // we may get NaN when add it with negative_infinity. But this
            // usually would not happen for a connected FSA.
            if (tot_scores_data[fsa_idx] != negative_infinity) {
              state_scores_data[start_state_next_fsa - 1] =
                  -tot_scores_data[fsa_idx];
            } else {
              state_scores_data[start_state_next_fsa - 1] = negative_infinity;
            }
          }
        });
  } else {
    // set the score of final state in each fsa to be 0
    K2_EVAL(
        c, num_fsas, lambda_set_final_state_score, (int32_t fsa_idx) {
          int32_t start_state = fsa_row_splits1[fsa_idx],
                  start_state_next_fsa = fsa_row_splits1[fsa_idx + 1];
          if (start_state_next_fsa - start_state > 0)
            state_scores_data[start_state_next_fsa - 1] = 0;
        });
  }

  // get the 1st leaving arc index in each batch, +1 so we can get the number of
  // leaving arcs in each batch by taking the difference of adjacent elements
  Array1<int32_t> leaving_arc_start_index(c, num_batches + 1);
  int32_t *leaving_arc_start_index_data = leaving_arc_start_index.Data();
  const int32_t *arc_batches_row_splits1 =
      leaving_arc_batches.RowSplits(1).Data();
  const int32_t *arc_batches_row_splits2 =
      leaving_arc_batches.RowSplits(2).Data();
  const int32_t *arc_batches_row_splits3 =
      leaving_arc_batches.RowSplits(3).Data();
  K2_EVAL(
      c, num_batches, lambda_set_leaving_arc_start_index, (int32_t batch_idx) {
        int32_t this_state_idx0xx =
            arc_batches_row_splits2[batch_idx * num_fsas];
        int32_t this_arc_idx0xxx = arc_batches_row_splits3[this_state_idx0xx];
        leaving_arc_start_index_data[batch_idx] = this_arc_idx0xxx;
        if (batch_idx == num_batches - 1) {
          // process the last element
          int32_t next_state_idx0xx =
              arc_batches_row_splits2[num_batches * num_fsas];
          int32_t next_arc_idx0xxx = arc_batches_row_splits3[next_state_idx0xx];
          leaving_arc_start_index_data[num_batches] = next_arc_idx0xxx;
        }
      });

  const int32_t *arc_batches_row_ids1 = leaving_arc_batches.RowIds(1).Data();
  const int32_t *arc_batches_row_ids2 = leaving_arc_batches.RowIds(2).Data();
  const int32_t *arc_batches_row_ids3 = leaving_arc_batches.RowIds(3).Data();
  const int32_t *leaving_arc_ids = leaving_arc_batches.values.Data();
  const int32_t *states_data = state_batches.values.Data();
  const Arc *arcs = fsas.values.Data();
  Array1<FloatType> leaving_arc_score_values(
      c, num_arcs);  // leaving arc_scores in batches
  FloatType *arc_scores_data = leaving_arc_score_values.Data();
  // copy leaving_arc_start_index to cpu as we will access its elements in below
  // Eval function for `lambda_set_leaving_arc_scores`
  Array1<int32_t> cpu_leaving_arc_start_index =
      leaving_arc_start_index.To(GetCpuContext());
  const int32_t *cpu_leaving_arc_start = cpu_leaving_arc_start_index.Data();
  // copy the index of start state in each fsa to CPU
  Array1<int32_t> arc_batches_row_splits1_array =
      leaving_arc_batches.RowSplits(1);
  Array1<int32_t> arc_batches_row_splits12_cpu =
      leaving_arc_batches.RowSplits(2)[arc_batches_row_splits1_array].To(
          GetCpuContext());
  K2_CHECK_EQ(arc_batches_row_splits12_cpu.Dim(), num_batches + 1);
  const int32_t *arc_batches_row_splits12_cpu_data =
      arc_batches_row_splits12_cpu.Data();
  Array1<int32_t> arc_row_splits_mem(c, num_states + 1);
  Array1<FloatType> score_cache(c, num_states + 1);
  // process batch sequentially.
  for (int32_t i = num_batches - 1; i >= 0; --i) {
    // get the range we would call Max/LogSum per sub list
    int32_t this_state_idx0xx = arc_batches_row_splits12_cpu_data[i];
    int32_t next_state_idx0xx =
        arc_batches_row_splits12_cpu_data[i + 1];  // the 1st state idx in the
                                                   // next batch
    K2_CHECK_LT(this_state_idx0xx, num_states);
    K2_CHECK_LE(next_state_idx0xx, num_states);
    int32_t num_states_this_batch = next_state_idx0xx - this_state_idx0xx;
    K2_CHECK_LT(num_states_this_batch, arc_row_splits_mem.Dim());
    // we always use the first `num_states_this_batch` elements in
    // arc_row_splits_mem.
    Array1<int32_t> arc_row_splits_part = arc_row_splits_mem.Range(
        0, num_states_this_batch + 1);  // +1 for the last element
    int32_t num_arcs_this_batch =
        cpu_leaving_arc_start[i + 1] - cpu_leaving_arc_start[i];
    {
      ParallelRunner pr(c);
      // get leaving arc scores
      {
        With w(pr.NewStream());
        K2_EVAL(
            c, num_arcs_this_batch, lambda_set_leaving_arc_score,
            (int32_t idx123) {
              // all idx** in below code are the indexes to leaving_arc_batches
              int32_t idx0123 = leaving_arc_start_index_data[i] + idx123;
              int32_t idx012 = arc_batches_row_ids3[idx0123];
              int32_t idx01 = arc_batches_row_ids2[idx012];
              K2_CHECK_EQ(idx01 / num_fsas, i);  // idx01/num_fsas is batch_id
              int32_t fsa_id = idx01 % num_fsas;

              int32_t leaving_arc_id = leaving_arc_ids[idx0123];
              float curr_arc_score = arcs[leaving_arc_id].score;
              int32_t dest_state_idx1 = arcs[leaving_arc_id].dest_state;
              int32_t dest_state_idx01 =
                  fsa_row_splits1[fsa_id] + dest_state_idx1;
              arc_scores_data[idx0123] =
                  state_scores_data[dest_state_idx01] + curr_arc_score;
            });
      }
      {
        With w(pr.NewStream());
        // make leaving arc row splits info in each batch starting from zero,
        // we will use it to call MaxPerSublist or LogSumPerSubList
        int32_t *sum_splits_data = arc_row_splits_part.Data();
        K2_EVAL(
            c, num_states_this_batch + 1, lambda_set_row_splits_for_sum,
            (int32_t idx) {
              sum_splits_data[idx] =
                  arc_batches_row_splits3[idx + this_state_idx0xx] -
                  arc_batches_row_splits3[this_state_idx0xx];
            });
      }
    }
    int32_t this_arc_idx0xxx = cpu_leaving_arc_start[i];
    Array1<FloatType> sub_scores_values =
        leaving_arc_score_values.Range(this_arc_idx0xxx, num_arcs_this_batch);
    RaggedShape sub_scores_shape =
        RaggedShape2(&arc_row_splits_part, nullptr, sub_scores_values.Dim());
    Ragged<FloatType> sub_scores(sub_scores_shape, sub_scores_values);
    // we always use the first num_rows elements in score_cache.
    Array1<FloatType> sub_state_scores =
        score_cache.Range(0, num_states_this_batch);
    // get scores per state in this batch
    if (log_semiring)
      LogSumPerSublist(sub_scores, negative_infinity, &sub_state_scores);
    else
      MaxPerSublist(sub_scores, negative_infinity, &sub_state_scores);
    const FloatType *sub_state_scores_data = sub_state_scores.Data();
    // copy those scores to corresponding state in state_scores
    K2_EVAL(
        c, num_states_this_batch, lambda_copy_state_scores, (int32_t idx2) {
          int32_t idx012 = this_state_idx0xx + idx2;
          int32_t state_idx012 = states_data[idx012];
          int32_t idx01 = arc_batches_row_ids2[idx012];
          int32_t fsa_id = idx01 % num_fsas;
          int32_t start_state = fsa_row_splits1[fsa_id],
                  start_state_next_fsa = fsa_row_splits1[fsa_id + 1];
          if (start_state_next_fsa - start_state > 0) {  // non-empty fsa
            int32_t final_state_idx = start_state_next_fsa - 1;
            // don't override score in the final state in each fsa.
            if (state_idx012 != final_state_idx)
              state_scores_data[state_idx012] = sub_state_scores_data[idx2];
          }
        });
  }

  return state_scores;
}

template <typename FloatType>
Array1<FloatType> GetTotScores(FsaVec &fsas,
                               const Array1<FloatType> &forward_scores) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK(IsCompatible(fsas, forward_scores));
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  ContextPtr &c = fsas.Context();
  int32_t num_fsas = fsas.Dim0(), num_states = fsas.TotSize(1);
  K2_CHECK_EQ(num_states, forward_scores.Dim());

  FloatType negative_infinity = -std::numeric_limits<FloatType>::infinity();
  Array1<FloatType> tot_scores(c, num_fsas, negative_infinity);
  FloatType *tot_scores_data = tot_scores.Data();

  const int32_t *fsa_row_splits1 = fsas.RowSplits(1).Data();
  const FloatType *forward_scores_data = forward_scores.Data();
  K2_EVAL(
      c, num_fsas, lambda_copy_tot_scores, (int32_t fsa_idx) {
        int32_t start_state = fsa_row_splits1[fsa_idx],
                start_state_next_fsa = fsa_row_splits1[fsa_idx + 1];
        if (start_state_next_fsa > start_state) {  // non-empty fsa
          int32_t final_state_idx = start_state_next_fsa - 1;
          tot_scores_data[fsa_idx] = forward_scores_data[final_state_idx];
        }
      });

  return tot_scores;
}

template <typename FloatType>
Array1<FloatType> GetArcScores(FsaVec &fsas,
                               const Array1<FloatType> &forward_scores,
                               const Array1<FloatType> &backward_scores) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK(IsCompatible(fsas, forward_scores));
  K2_CHECK(IsCompatible(fsas, backward_scores));
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  ContextPtr &c = fsas.Context();
  int32_t num_fsas = fsas.Dim0(), num_states = fsas.TotSize(1),
          num_arcs = fsas.TotSize(2);
  K2_CHECK_EQ(num_states, forward_scores.Dim());
  K2_CHECK_EQ(num_states, backward_scores.Dim());

  Array1<FloatType> arc_scores(c, num_arcs);
  FloatType *arc_scores_data = arc_scores.Data();

  const int32_t *fsa_row_splits1 = fsas.RowSplits(1).Data();
  const int32_t *fsa_row_ids1 = fsas.RowIds(1).Data();
  const int32_t *fsa_row_ids2 = fsas.RowIds(2).Data();
  const Arc *arcs = fsas.values.Data();
  const FloatType *forward_scores_data = forward_scores.Data();
  const FloatType *backward_scores_data = backward_scores.Data();
  K2_EVAL(
      c, num_arcs, lambda_get_arc_scores, (int32_t arc_idx012) {
        int32_t src_state_idx1 = arcs[arc_idx012].src_state;
        int32_t dest_state_idx1 = arcs[arc_idx012].dest_state;
        float arc_score = arcs[arc_idx012].score;

        int32_t idx01 = fsa_row_ids2[arc_idx012];
        int32_t idx0 = fsa_row_ids1[idx01];
        int32_t idx0x = fsa_row_splits1[idx0];
        int32_t src_state_idx01 = idx0x + src_state_idx1;
        int32_t dest_state_idx01 = idx0x + dest_state_idx1;
        arc_scores_data[arc_idx012] = arc_score +
                                      forward_scores_data[src_state_idx01] +
                                      backward_scores_data[dest_state_idx01];
      });

  return arc_scores;
}

// explicit instantiation for those score computation functions above
template Array1<float> GetForwardScores(FsaVec &fsas,
                                        Ragged<int32_t> &state_batches,
                                        Ragged<int32_t> &entering_arc_batches,
                                        bool log_semiring,
                                        Array1<int32_t> *entering_arcs);
template Array1<double> GetForwardScores(FsaVec &fsas,
                                         Ragged<int32_t> &state_batches,
                                         Ragged<int32_t> &entering_arc_batches,
                                         bool log_semiring,
                                         Array1<int32_t> *entering_arcs);

template Array1<float> GetBackwardScores(FsaVec &fsas,
                                         Ragged<int32_t> &state_batches,
                                         Ragged<int32_t> &leaving_arc_batches,
                                         const Array1<float> *tot_scores,
                                         bool log_semiring);
template Array1<double> GetBackwardScores(FsaVec &fsas,
                                          Ragged<int32_t> &state_batches,
                                          Ragged<int32_t> &leaving_arc_batches,
                                          const Array1<double> *tot_scores,
                                          bool log_semiring);

template Array1<float> GetArcScores(FsaVec &fsas,
                                    const Array1<float> &forward_scores,
                                    const Array1<float> &backward_scores);
template Array1<double> GetArcScores(FsaVec &fsas,
                                     const Array1<double> &forward_scores,
                                     const Array1<double> &backward_scores);

template Array1<float> GetTotScores(FsaVec &fsas,
                                    const Array1<float> &forward_scores);
template Array1<double> GetTotScores(FsaVec &fsas,
                                     const Array1<double> &forward_scores);

Fsa RandomFsa(bool acyclic /*=true*/, int32_t max_symbol /*=50*/,
              int32_t min_num_arcs /*=0*/, int32_t max_num_arcs /*=1000*/) {
  NVTX_RANGE(K2_FUNC);
  ContextPtr c = GetCpuContext();
  K2_CHECK_GE(min_num_arcs, 0);
  K2_CHECK_GE(max_num_arcs, min_num_arcs);
  K2_CHECK_GE(max_symbol, 0);
  RaggedShape shape =
      RandomRaggedShape(false, 2, 2, min_num_arcs, max_num_arcs);
  int32_t dim0 = shape.Dim0();
  // empty Fsa
  if (dim0 == 0) return Fsa(shape, Array1<Arc>(c, std::vector<Arc>{}));
  // as there should be no arcs leaving the final_state, we always push back an
  // empty row here.
  Array1<int32_t> ans_row_splits1(c, dim0 + 2);
  Array1<int32_t> sub_range = ans_row_splits1.Range(0, dim0 + 1);
  sub_range.CopyFrom(shape.RowSplits(1));
  int32_t *ans_row_splits1_data = ans_row_splits1.Data();
  ans_row_splits1_data[dim0 + 1] = ans_row_splits1_data[dim0];
  // create returned shape
  RaggedShapeLayer ans_shape_dim;
  ans_shape_dim.row_splits = ans_row_splits1;
  ans_shape_dim.cached_tot_size = shape.TotSize(1);
  RaggedShape ans_shape(std::vector<RaggedShapeLayer>{ans_shape_dim}, true);
  ans_shape.Populate();

  // will be used to generate scores on arcs.
  std::random_device rd;
  std::mt19937 gen(rd());
  // TODO(haowen): let the users set the range of scores? it's fine to use it
  // for now as we just use it to test.
  std::uniform_real_distribution<float> dis_score(0, 10);

  // create arcs
  int32_t *row_ids1 = ans_shape.RowIds(1).Data();
  int32_t num_states = ans_shape.Dim0(), num_arcs = ans_shape.TotSize(1);
  int32_t start_state = 0, final_state = num_states - 1;
  std::vector<Arc> arcs(num_arcs);
  for (int32_t i = 0; i != num_arcs; ++i) {
    int32_t curr_state = row_ids1[i];
    int32_t dest_state = acyclic ? RandInt(curr_state + 1, final_state)
                                 : RandInt(start_state, final_state);
    int32_t symbol = dest_state == final_state ? -1 : RandInt(0, max_symbol);
    float score = dis_score(gen);
    arcs[i] = Arc(curr_state, dest_state, symbol, score);
  }
  return Fsa(ans_shape, Array1<Arc>(c, arcs));
}

FsaVec RandomFsaVec(int32_t min_num_fsas /*=1*/, int32_t max_num_fsas /*=1000*/,
                    bool acyclic /*=true*/, int32_t max_symbol /*=50*/,
                    int32_t min_num_arcs /*=0*/,
                    int32_t max_num_arcs /*=1000*/) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_GE(min_num_fsas, 0);
  K2_CHECK_GE(max_num_fsas, min_num_fsas);
  int32_t num_fsas = RandInt(min_num_fsas, max_num_fsas);
  std::vector<Fsa> fsas(num_fsas);
  for (int32_t i = 0; i != num_fsas; ++i) {
    fsas[i] = RandomFsa(acyclic, max_symbol, min_num_arcs, max_num_arcs);
  }
  return Stack(0, num_fsas, fsas.data());
}

DenseFsaVec RandomDenseFsaVec(int32_t min_num_fsas, int32_t max_num_fsas,
                              int32_t min_frames, int32_t max_frames,
                              int32_t min_symbols, int32_t max_symbols,
                              float scores_scale) {
  NVTX_RANGE(K2_FUNC);
  ContextPtr c = GetCpuContext();
  int32_t num_fsas = RandInt(min_num_fsas, max_num_fsas);

  // num_symbols includes epsilon but not final-symbol -1.
  int32_t num_symbols = RandInt(min_symbols, max_symbols);

  // `num_frames` includes the extra 1 frame for the final-symbol.
  std::vector<int32_t> num_frames(num_fsas + 1);
  int32_t tot_frames = 0;
  for (int32_t i = 0; i < num_fsas; i++) {
    num_frames[i] = RandInt(min_frames, max_frames) + 1;
    tot_frames += num_frames[i];
  }

  Array2<float> scores(c, tot_frames, num_symbols + 1);
  auto scores_acc = scores.Accessor();

  std::vector<int32_t> row_splits_vec(num_fsas + 1);
  row_splits_vec[0] = 0;
  int32_t cur_start_frame = 0;
  RandIntGenerator gen;
  for (int32_t i = 0; i < num_fsas; i++) {
    int32_t this_num_frames = num_frames[i],
            end_frame = cur_start_frame + this_num_frames;
    for (int32_t f = cur_start_frame; f + 1 < end_frame; f++) {
      scores_acc(f, 0) = -std::numeric_limits<float>::infinity();
      for (int32_t j = 0; j < num_symbols; j++)
        scores_acc(f, j + 1) = scores_scale * gen(-50, 50) * 0.01;
    }
    // on the last frame the placement of infinity vs. finite is reversed:
    // -1 gets finite value, others get infinity.
    int32_t f = end_frame - 1;
    scores_acc(f, 0) = scores_scale * gen(-50, 50) * 0.01;
    for (int32_t j = 0; j < num_symbols; j++)
      scores_acc(f, j + 1) = -std::numeric_limits<float>::infinity();
    row_splits_vec[i + 1] = cur_start_frame = end_frame;
  }
  Array1<int32_t> row_splits(c, row_splits_vec);
  return DenseFsaVec(RaggedShape2(&row_splits, nullptr, tot_frames), scores);
}

Ragged<int32_t> GetStartStates(FsaVec &src) {
  NVTX_RANGE(K2_FUNC);
  ContextPtr c = src.Context();
  K2_CHECK(src.NumAxes() == 3);
  int32_t num_fsas = src.Dim0();
  const int32_t *src_row_splits1_data = src.RowSplits(1).Data();

  Array1<int32_t> ans_row_splits(c, num_fsas + 1);
  // will first set the elements of ans_row_splits to the number of states kept
  // from this FSA (either 0 or 1).
  int32_t *num_states_data = ans_row_splits.Data();
  K2_EVAL(
      c, num_fsas, lambda_set_num_states, (int32_t fsa_idx0)->void {
        // 1 if the FSA is not empty, 0 if empty.
        num_states_data[fsa_idx0] = (src_row_splits1_data[fsa_idx0 + 1] >
                                     src_row_splits1_data[fsa_idx0]);
      });
  ExclusiveSum(ans_row_splits, &ans_row_splits);
  int32_t ans_dim = ans_row_splits.Back();
  Ragged<int32_t> ans(RaggedShape2(&ans_row_splits, nullptr, ans_dim),
                      Array1<int32_t>(c, ans_dim));
  const int32_t *ans_row_ids1_data = ans.shape.RowIds(1).Data();
  int32_t *ans_values_data = ans.values.Data();
  K2_EVAL(
      c, ans_dim, lambda_set_ans_values, (int32_t ans_idx01)->void {
        int32_t idx0 = ans_row_ids1_data[ans_idx01];
        int32_t src_start_state_idx01 = src_row_splits1_data[idx0];
        K2_CHECK_GT(src_row_splits1_data[idx0 + 1], src_row_splits1_data[idx0]);
        ans_values_data[ans_idx01] = src_start_state_idx01;
      });
  return ans;
}

FsaVec FsaVecFromArcIndexes(FsaVec &fsas, Ragged<int32_t> &best_arc_indexes) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  K2_CHECK_EQ(best_arc_indexes.NumAxes(), 2);
  K2_CHECK(IsCompatible(fsas, best_arc_indexes));
  K2_CHECK_EQ(fsas.Dim0(), best_arc_indexes.Dim0());

  // if there are n arcs (for n > 0), there are n + 1 states; if there are 0
  // arcs, there are 0 states (that FSA will have no arcs or states).
  RaggedShape states_shape = ChangeSublistSizePinned(best_arc_indexes.shape, 1);
  const int32_t *states_shape_row_splits1_data =
      states_shape.RowSplits(1).Data();

  int32_t num_fsas = fsas.Dim0();
  int32_t num_states = states_shape.NumElements();
  int32_t num_arcs = best_arc_indexes.shape.NumElements();
  ContextPtr &context = fsas.Context();

  if (num_arcs == 0) {
    RaggedShape shape_a = RegularRaggedShape(context, num_fsas, 0),
                shape_b = RegularRaggedShape(context, 0, 0);
    return FsaVec(ComposeRaggedShapes(shape_a, shape_b),
                  Array1<Arc>(context, 0));
  }

  Array1<int32_t> row_splits2(context, num_states + 1);
  Array1<int32_t> row_ids2(context, num_arcs);
  int32_t *row_splits2_data = row_splits2.Data();
  int32_t *row_ids2_data = row_ids2.Data();

  Array1<Arc> arcs(context, num_arcs);
  Arc *arcs_data = arcs.Data();

  const int32_t *best_arc_indexes_row_splits1_data =
      best_arc_indexes.RowSplits(1).Data();

  const int32_t *best_arc_indexes_row_ids1_data =
      best_arc_indexes.RowIds(1).Data();

  const int32_t *best_arc_indexes_data = best_arc_indexes.values.Data();
  const Arc *fsas_values_data = fsas.values.Data();

  K2_EVAL(
      context, num_arcs, lambda_set_arcs, (int32_t best_arc_idx01) {
        int32_t fsas_idx0 = best_arc_indexes_row_ids1_data[best_arc_idx01];
        int32_t best_arc_idx0x = best_arc_indexes_row_splits1_data[fsas_idx0];
        int32_t best_arc_idx0x_next =
            best_arc_indexes_row_splits1_data[fsas_idx0 + 1];
        int32_t num_best_arcs = best_arc_idx0x_next - best_arc_idx0x;
        int32_t best_arc_idx1 = best_arc_idx01 - best_arc_idx0x;

        int32_t state_offset = states_shape_row_splits1_data[fsas_idx0];

        const Arc &arc =
            fsas_values_data[best_arc_indexes_data[best_arc_idx01]];
        int32_t src_state = best_arc_idx1;
        int32_t dest_state = src_state + 1;
        int32_t label = arc.label;
        float score = arc.score;
        arcs_data[best_arc_idx01] = Arc(src_state, dest_state, label, score);

        int32_t state_idx01 = state_offset + src_state;
        row_ids2_data[best_arc_idx01] = state_idx01;
        row_splits2_data[state_idx01 + 1] = best_arc_idx01 + 1;
        if (best_arc_idx01 == 0) row_splits2_data[0] = 0;

        if (best_arc_idx1 + 1 == num_best_arcs)
          row_splits2_data[state_idx01 + 2] = best_arc_idx01 + 1;
      });
  RaggedShape shape =
      RaggedShape3(&states_shape.RowSplits(1), &states_shape.RowIds(1),
                   num_states, &row_splits2, &row_ids2, num_arcs);
  Ragged<Arc> ans(shape, arcs);
  return ans;
}

FsaVec GetIncomingFsaVec(FsaVec &fsas) {
  Array1<int32_t> dest_states = GetDestStates(fsas, true);
  Ragged<int32_t> arc_indexes = GetIncomingArcs(fsas, dest_states);
  return FsaVec(arc_indexes.shape, fsas.values[arc_indexes.values]);
}

Ragged<int32_t> ComposeArcMaps(Ragged<int32_t> &step1_arc_map,
                               Ragged<int32_t> &step2_arc_map) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(step1_arc_map.NumAxes(), 2);
  K2_CHECK_EQ(step2_arc_map.NumAxes(), 2);
  ContextPtr c = GetContext(step1_arc_map, step2_arc_map);
  int32_t arc_map1_dim0 = step1_arc_map.Dim0(),
          arc_map2_dim0 = step2_arc_map.Dim0();

  Ragged<int32_t> step1_elements = Index(step1_arc_map, step2_arc_map.values);
  RaggedShape composed_shape =
      ComposeRaggedShapes(step2_arc_map.shape, step1_elements.shape);
  return Ragged<int32_t>(RemoveAxis(composed_shape, 1), step1_elements.values);
}

}  // namespace k2
