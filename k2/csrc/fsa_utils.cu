#include "hip/hip_runtime.h"
/**
 * @brief Utilities for creating FSAs.
 *
 * Note that serializations are done in Python.
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey, Haowen Qiu)
 *                      Mobvoi Inc.        (authors: Fangjun Kuang)
 *                      Guoguo Chen
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <algorithm>
#include <limits>
#include <sstream>
#include <utility>
#include <vector>

#include "k2/csrc/context.h"
#include "k2/csrc/fsa_utils.h"

namespace k2 {

// field separator within a line for a text form FSA
static constexpr const char *kDelim = " \t";

// Convert a string to an integer. Abort the program on failure.
static int32_t StringToInt(const std::string &s) {
  K2_CHECK(!s.empty());

  bool ok = false;
  char *p = nullptr;
  // std::strtol requires a `long` type
  long n = std::strtol(s.c_str(), &p, 10);  // NOLINT
  if (*p == '\0') ok = true;

  auto res = static_cast<int32_t>(n);
  if (n != res) ok = false;  // out of range

  K2_CHECK(ok) << "Failed to convert " << s << " to an integer";

  return res;
}

// Convert a string to a float. Abort the program on failure.
// TODO(guoguo): We may run into locale problems, with comma vs. period for
//               decimals. We have to test if the C code will behave the same
//               w.r.t. locale as Python does.
static float StringToFloat(const std::string &s) {
  K2_CHECK(!s.empty());
  char *p = nullptr;
  float f = std::strtof(s.c_str(), &p);
  if (*p != '\0') K2_LOG(FATAL) << "Failed to convert " << s << " to a float";
  return f;
}

// Trim leading and trailing spaces of a string.
static void TrimString(std::string *s) {
  K2_CHECK_NE(s, nullptr);
  auto not_space = [](int32_t c) -> bool { return std::isspace(c) == 0; };

  s->erase(s->begin(), std::find_if(s->begin(), s->end(), not_space));
  s->erase(std::find_if(s->rbegin(), s->rend(), not_space).base(), s->end());
}

/* Split a string to a vector of strings using a set of delimiters.

   Example usage:

   @code
    std::string in = "1 2 3";
    const char *delim = " \t";
    std::vector<std::string> out;
    SplitStringToVector(in, delim, &out);
   @endcode

   @param [in]  in    The input string to be split.
   @param [in]  delim A string of delimiters.
   @param [out] out   It saves the split result.
*/
static void SplitStringToVector(const std::string &in, const char *delim,
                                std::vector<std::string> *out) {
  K2_CHECK_NE(delim, nullptr);
  K2_CHECK_NE(out, nullptr);
  out->clear();
  std::size_t start = 0;
  while (true) {
    auto pos = in.find_first_of(delim, start);
    if (pos == std::string::npos) break;

    auto sub = in.substr(start, pos - start);
    start = pos + 1;

    TrimString(&sub);
    if (!sub.empty()) out->emplace_back(std::move(sub));
  }

  if (start < in.size()) {
    auto sub = in.substr(start);
    TrimString(&sub);
    if (!sub.empty()) out->emplace_back(std::move(sub));
  }
}

/* Create an acceptor from a stream, assuming the acceptor is in the k2 format:

   src_state1 dest_state1 label1 score1
   src_state2 dest_state2 label2 score2
   ... ...
   final_state

   The source states will be in non-descending order, and the final state does
   not bear a cost/score -- we put the cost/score on the arc that connects to
   the final state and set its label to -1.

   @param [in]  is    The input stream that contains the acceptor.

   @return It returns an Fsa on CPU.
*/
static Fsa K2AcceptorFromStream(std::istringstream &is) {
  std::vector<Arc> arcs;
  std::vector<std::string> splits;
  std::string line;

  bool finished = false;  // when the final state is read, set it to true.
  while (std::getline(is, line)) {
    SplitStringToVector(line, kDelim,
                        &splits);  // splits is cleared in the function
    if (splits.empty()) continue;  // this is an empty line

    K2_CHECK_EQ(finished, false);

    auto num_fields = splits.size();
    if (num_fields == 4u) {
      //   0            1          2      3
      // src_state  dest_state   label  score
      int32_t src_state = StringToInt(splits[0]);
      int32_t dest_state = StringToInt(splits[1]);
      int32_t symbol = StringToInt(splits[2]);
      float score = StringToFloat(splits[3]);
      arcs.emplace_back(src_state, dest_state, symbol, score);
    } else if (num_fields == 1u) {
      //   0
      // final_state
      (void)StringToInt(splits[0]);  // this is a final state
      finished = true;               // set finish
    } else {
      K2_LOG(FATAL) << "Invalid line: " << line
                    << "\nk2 acceptor expects a line with 1 (final_state) or "
                       "4 (src_state dest_state label score) fields";
    }
  }

  K2_CHECK_EQ(finished, true) << "The last line should be the final state";

  bool error = true;
  Array1<Arc> array(GetCpuContext(), arcs);
  auto fsa = FsaFromArray1(array, &error);
  K2_CHECK_EQ(error, false);

  return fsa;
}

/* Create a transducer from a stream, assuming the transducer is in the K2
   format:

   src_state1 dest_state1 label1 aux_label1 score1
   src_state2 dest_state2 label2 aux_label2 score2
   ... ...
   final_state

   The source states will be in non-descending order, and the final state does
   not bear a cost/score -- we put the cost/score on the arc that connects to
   the final state and set its label to -1.

   @param [in]  is    The input stream that contains the transducer.

   @return It returns an Fsa on CPU.
*/
static Fsa K2TransducerFromStream(std::istringstream &is,
                                  Array1<int32_t> *aux_labels) {
  K2_CHECK(aux_labels != nullptr);

  std::vector<int32_t> aux_labels_internal;
  std::vector<Arc> arcs;
  std::vector<std::string> splits;
  std::string line;

  bool finished = false;  // when the final state is read, set it to true.
  while (std::getline(is, line)) {
    SplitStringToVector(line, kDelim,
                        &splits);  // splits is cleared in the function
    if (splits.empty()) continue;  // this is an empty line

    K2_CHECK_EQ(finished, false);

    auto num_fields = splits.size();
    if (num_fields == 5u) {
      //   0           1         2         3        4
      // src_state  dest_state label   aux_label  score
      int32_t src_state = StringToInt(splits[0]);
      int32_t dest_state = StringToInt(splits[1]);
      int32_t symbol = StringToInt(splits[2]);
      int32_t aux_label = StringToInt(splits[3]);
      float score = StringToFloat(splits[4]);
      arcs.emplace_back(src_state, dest_state, symbol, score);
      aux_labels_internal.push_back(aux_label);
    } else if (num_fields == 1u) {
      //   0
      // final_state
      (void)StringToInt(splits[0]);
      finished = true;  // set finish
    } else {
      K2_LOG(FATAL) << "Invalid line: " << line
                    << "\nk2 transducer expects a line with 1 (final_state) or "
                       "5 (src_state dest_state label aux_label score) fields";
    }
  }

  K2_CHECK_EQ(finished, true) << "The last line should be the final state";

  auto cpu_context = GetCpuContext();
  *aux_labels = Array1<int32_t>(cpu_context, aux_labels_internal);
  Array1<Arc> array(cpu_context, arcs);

  bool error = true;
  auto fsa = FsaFromArray1(array, &error);
  K2_CHECK_EQ(error, false);

  return fsa;
}

/* Create an acceptor from a stream, assuming the acceptor is in the OpenFST
   format:

   src_state1 dest_state1 label1 score1
   src_state2 dest_state2 label2 score2
   ... ...
   final_state final_score

   We will negate the cost/score when we read them in. Also note, OpenFST may
   omit the cost/score if it is 0.0.

   We always create the super final state. If there are final state(s) in the
   original FSA, then we add arc(s) from the original final state(s) to the
   super final state, with the (negated) old final state cost/score as its
   cost/score, and -1 as its label.

   @param [in]  is    The input stream that contains the acceptor.

   @return It returns an Fsa on CPU.
*/
static Fsa OpenFstAcceptorFromStream(std::istringstream &is) {
  std::vector<Arc> arcs;
  std::vector<std::vector<Arc>> state_to_arcs;  // indexed by states
  std::vector<std::string> splits;
  std::string line;

  int32_t max_state = -1;
  int32_t num_arcs = 0;
  std::vector<int32_t> original_final_states;
  std::vector<float> original_final_weights;
  while (std::getline(is, line)) {
    SplitStringToVector(line, kDelim,
                        &splits);  // splits is cleared in the function
    if (splits.empty()) continue;  // this is an empty line

    auto num_fields = splits.size();
    if (num_fields == 3u || num_fields == 4u) {
      //   0            1          2
      // src_state  dest_state   label
      //
      // or
      //
      //   0            1          2      3
      // src_state  dest_state   label  score
      int32_t src_state = StringToInt(splits[0]);
      int32_t dest_state = StringToInt(splits[1]);
      int32_t symbol = StringToInt(splits[2]);
      float score = 0.0f;
      if (num_fields == 4u) score = -1.0f * StringToFloat(splits[3]);

      // Add the arc to "state_to_arcs".
      ++num_arcs;
      max_state = std::max(max_state, std::max(src_state, dest_state));
      if (static_cast<int32_t>(state_to_arcs.size()) <= src_state)
        state_to_arcs.resize(src_state + 1);
      state_to_arcs[src_state].emplace_back(src_state, dest_state, symbol,
                                            score);
    } else if (num_fields == 1u || num_fields == 2u) {
      //   0            1
      // final_state  score
      float score = 0.0f;
      if (num_fields == 2u) score = -1.0f * StringToFloat(splits[1]);
      original_final_states.push_back(StringToInt(splits[0]));
      original_final_weights.push_back(score);
      max_state = std::max(max_state, original_final_states.back());
    } else {
      K2_LOG(FATAL) << "Invalid line: " << line
                    << "\nOpenFST acceptor expects a line with 1 (final_state),"
                       " 2 (final_state score), 3 (src_state dest_state label) "
                       "or 4 (src_state dest_state label score) fields.";
    }
  }

  K2_CHECK(is.eof());

  // Post processing on final states. If there are final state(s) in the
  // original FSA, we add the super final state as well as arc(s) from original
  // final state(s) to the super final state. Otherwise, the super final state
  // will be added by FsaFromArray1 (since there's no arc with label
  // kFinalSymbol).
  if (original_final_states.size() > 0) {
    K2_CHECK_EQ(original_final_states.size(), original_final_weights.size());
    int32_t super_final_state = max_state + 1;
    state_to_arcs.resize(super_final_state);
    for (std::size_t i = 0; i != original_final_states.size(); ++i) {
      state_to_arcs[original_final_states[i]].emplace_back(
          original_final_states[i], super_final_state,
          -1,  // kFinalSymbol
          original_final_weights[i]);
      ++num_arcs;
    }
  }

  // Move arcs from "state_to_arcs" to "arcs".
  int32_t arc_index = 0;
  arcs.resize(num_arcs);
  for (std::size_t s = 0; s < state_to_arcs.size(); ++s) {
    for (std::size_t a = 0; a < state_to_arcs[s].size(); ++a) {
      K2_CHECK_GT(num_arcs, arc_index);
      arcs[arc_index] = state_to_arcs[s][a];
      ++arc_index;
    }
  }
  K2_CHECK_EQ(num_arcs, arc_index);

  bool error = true;
  Array1<Arc> array(GetCpuContext(), arcs);
  // FsaFromArray1 will add a super final state if the original FSA doesn't have
  // a final state.
  auto fsa = FsaFromArray1(array, &error);
  K2_CHECK_EQ(error, false);

  return fsa;
}

/* Create a transducer from a stream, assuming the transducer is in the OpenFST
   format:

   src_state1 dest_state1 label1 aux_label1 score1
   src_state2 dest_state2 label2 aux_label2 score2
   ... ...
   final_state final_score

   We will negate the cost/score when we read them in. Also note, OpenFST may
   omit the cost/score if it is 0.0.

   We always create the super final state. If there are final state(s) in the
   original FST, then we add arc(s) from the original final state(s) to the
   super final state, with the (negated) old final state cost/score as its
   cost/score, -1 as its label and 0 as its aux_label.

   @param [in]  is    The input stream that contains the transducer.

   @return It returns an Fsa on CPU.
*/
static Fsa OpenFstTransducerFromStream(std::istringstream &is,
                                       Array1<int32_t> *aux_labels) {
  K2_CHECK(aux_labels != nullptr);

  std::vector<std::vector<int32_t>> state_to_aux_labels;  // indexed by states
  std::vector<std::vector<Arc>> state_to_arcs;            // indexed by states
  std::vector<int32_t> aux_labels_internal;
  std::vector<Arc> arcs;
  std::vector<std::string> splits;
  std::string line;

  int32_t max_state = -1;
  int32_t num_arcs = 0;
  std::vector<int32_t> original_final_states;
  std::vector<float> original_final_weights;
  while (std::getline(is, line)) {
    SplitStringToVector(line, kDelim,
                        &splits);  // splits is cleared in the function
    if (splits.empty()) continue;  // this is an empty line

    auto num_fields = splits.size();
    if (num_fields == 4u || num_fields == 5u) {
      //   0           1         2         3
      // src_state  dest_state label   aux_label
      //
      // or
      //
      //   0           1         2         3        4
      // src_state  dest_state label   aux_label  score
      int32_t src_state = StringToInt(splits[0]);
      int32_t dest_state = StringToInt(splits[1]);
      int32_t symbol = StringToInt(splits[2]);
      int32_t aux_label = StringToInt(splits[3]);
      float score = 0.0f;
      if (num_fields == 5u) score = -1.0f * StringToFloat(splits[4]);

      // Add the arc to "state_to_arcs", and aux_label to "state_to_aux_labels"
      ++num_arcs;
      max_state = std::max(max_state, std::max(src_state, dest_state));
      if (static_cast<int32_t>(state_to_arcs.size()) <= src_state) {
        state_to_arcs.resize(src_state + 1);
        state_to_aux_labels.resize(src_state + 1);
      }
      state_to_arcs[src_state].emplace_back(src_state, dest_state, symbol,
                                            score);
      state_to_aux_labels[src_state].push_back(aux_label);
    } else if (num_fields == 1u || num_fields == 2u) {
      //   0
      // final_state
      //
      // or
      //
      //   0            1
      // final_state  score
      // There could be multiple final states, so we first have to collect all
      // the final states, and then work out the super final state.
      float score = 0.0f;
      if (num_fields == 2u) score = -1.0f * StringToFloat(splits[1]);
      original_final_states.push_back(StringToInt(splits[0]));
      original_final_weights.push_back(score);
      max_state = std::max(max_state, original_final_states.back());
    } else {
      K2_LOG(FATAL) << "Invalid line: " << line
                    << "\nOpenFST transducer expects a line with "
                       "1 (final_state), 2 (final_state score), "
                       "4 (src_state dest_state label aux_label) or "
                       "5 (src_state dest_state label aux_label score) fields.";
    }
  }

  K2_CHECK(is.eof());

  // Post processing on final states. If there are final state(s) in the
  // original FST, we add the super final state as well as arc(s) from original
  // final state(s) to the super final state. Otherwise, the super final state
  // will be added by FsaFromArray1 (since there's no arc with label
  // kFinalSymbol).
  if (original_final_states.size() > 0) {
    K2_CHECK_EQ(original_final_states.size(), original_final_weights.size());
    int32_t super_final_state = max_state + 1;
    state_to_arcs.resize(super_final_state);
    state_to_aux_labels.resize(super_final_state);
    for (std::size_t i = 0; i != original_final_states.size(); ++i) {
      state_to_arcs[original_final_states[i]].emplace_back(
          original_final_states[i], super_final_state,
          -1,  // kFinalSymbol
          original_final_weights[i]);
      // TODO(guoguo) We are not sure yet what to put as the auxiliary label for
      //              arcs entering the super final state. The only real choices
      //              are kEpsilon or kFinalSymbol. We are using kEpsilon for
      //              now.
      state_to_aux_labels[original_final_states[i]].push_back(0);  // kEpsilon
      ++num_arcs;
    }
  }

  // Move arcs from "state_to_arcs" to "arcs", and aux_labels from
  // "state_to_aux_labels" to "aux_labels_internal"
  int32_t arc_index = 0;
  arcs.resize(num_arcs);
  aux_labels_internal.resize(num_arcs);
  K2_CHECK_EQ(state_to_arcs.size(), state_to_aux_labels.size());
  for (std::size_t s = 0; s < state_to_arcs.size(); ++s) {
    K2_CHECK_EQ(state_to_arcs[s].size(), state_to_aux_labels[s].size());
    for (std::size_t a = 0; a < state_to_arcs[s].size(); ++a) {
      K2_CHECK_GT(num_arcs, arc_index);
      arcs[arc_index] = state_to_arcs[s][a];
      aux_labels_internal[arc_index] = state_to_aux_labels[s][a];
      ++arc_index;
    }
  }
  K2_CHECK_EQ(num_arcs, arc_index);

  auto cpu_context = GetCpuContext();
  *aux_labels = Array1<int32_t>(cpu_context, aux_labels_internal);
  Array1<Arc> array(cpu_context, arcs);

  bool error = true;
  // FsaFromArray1 will add a super final state if the original FSA doesn't have
  // a final state.
  auto fsa = FsaFromArray1(array, &error);
  K2_CHECK_EQ(error, false);

  return fsa;
}

Fsa FsaFromString(const std::string &s, bool openfst /*= false*/,
                  Array1<int32_t> *aux_labels /*= nullptr*/) {
  std::istringstream is(s);
  K2_CHECK(is);

  if (openfst == false && aux_labels == nullptr)
    return K2AcceptorFromStream(is);
  else if (openfst == false && aux_labels != nullptr)
    return K2TransducerFromStream(is, aux_labels);
  else if (openfst == true && aux_labels == nullptr)
    return OpenFstAcceptorFromStream(is);
  else if (openfst == true && aux_labels != nullptr)
    return OpenFstTransducerFromStream(is, aux_labels);

  return Fsa();  // unreachable code
}

std::string FsaToString(const Fsa &fsa, bool openfst /*= false*/,
                        const Array1<int32_t> *aux_labels /*= nullptr*/) {
  K2_CHECK_EQ(fsa.NumAxes(), 2);

  if (fsa.Context()->GetDeviceType() != kCpu) {
    Fsa _fsa = fsa.To(GetCpuContext());
    Array1<int32_t> _aux_labels;
    if (aux_labels) _aux_labels = aux_labels->To(_fsa.Context());
    return FsaToString(_fsa, openfst, aux_labels ? &_aux_labels : nullptr);
  }

  K2_CHECK_EQ(fsa.Context()->GetDeviceType(), kCpu);
  const Array1<int32_t> &row_splits = fsa.shape.RowSplits(1);
  const Array1<Arc> &arcs = fsa.values;

  const int32_t *p = nullptr;
  if (aux_labels != nullptr) {
    K2_CHECK(IsCompatible(fsa, *aux_labels));
    K2_CHECK_EQ(aux_labels->Dim(), arcs.Dim());
    p = aux_labels->Data();
  }
  float scale = 1;
  if (openfst) scale = -1;

  std::ostringstream os;

  int32_t n = arcs.Dim();
  char sep = ' ';
  char line_sep = '\n';
  for (int32_t i = 0; i != n; ++i) {
    const auto &arc = arcs[i];
    os << arc.src_state << sep << arc.dest_state << sep << arc.symbol << sep;
    if (p != nullptr) os << p[i] << sep;
    os << (scale * arc.score) << line_sep;
  }
  os << (fsa.shape.Dim0() - 1) << line_sep;
  return os.str();
}

Array1<int32_t> GetDestStates(FsaVec &fsas, bool as_idx01) {
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  ContextPtr &c = fsas.Context();
  int32_t num_arcs = fsas.NumElements();
  Array1<int32_t> ans(c, num_arcs);
  const Arc *arcs_data = fsas.values.Data();
  int32_t *ans_data = ans.Data();
  if (!as_idx01) {
    auto lambda_set_dest_states1 = [=] __host__ __device__(int32_t arc_idx012) {
      ans_data[arc_idx012] = arcs_data[arc_idx012].dest_state;
    };
    Eval(c, num_arcs, lambda_set_dest_states1);
  } else {
    const int32_t *row_ids2 = fsas.RowIds(2).Data();
    auto lambda_set_dest_states01 = [=] __host__ __device__(
                                        int32_t arc_idx012) {
      int32_t src_state = arcs_data[arc_idx012].src_state,
              dest_state = arcs_data[arc_idx012].dest_state;
      // (row_ids2[arc_idx012] - src_state) is the same as
      // row_splits1[row_ids1[row_ids2[arc_idx012]]]; it's the idx01 of the 1st
      // state in this FSA.
      ans_data[arc_idx012] = dest_state + (row_ids2[arc_idx012] - src_state);
    };
    Eval(c, num_arcs, lambda_set_dest_states01);
  }
  return ans;
}

Ragged<int32_t> GetStateBatches(FsaVec &fsas, bool transpose) {
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  ContextPtr &c = fsas.Context();
  Array1<int32_t> arc_dest_states = GetDestStates(fsas, true);

  MonotonicLowerBound(arc_dest_states, &arc_dest_states);

  int32_t num_fsas = fsas.Dim0(), num_states = fsas.TotSize(1),
          num_arcs = fsas.TotSize(2);

  // We can tune `log_power` as a tradeoff between work done and clock time on
  // GPU.
  int32_t log_power = (c->GetDeviceType() == kCpu ? 0 : 4);

  int32_t max_num_states = fsas.shape.MaxSize(1);
  // the following avoids doing too much extra work accumulating powers
  // of 'dest_states' for very small problem sizes.
  while (log_power > 0 && (1 << (1 + log_power)) > max_num_states) log_power--;

  // Ignoring edge effects: `dest_states_powers[0]` is just an array indexed by
  // state_idx01, that gives us the dest_state_idx01 that would be the beginning
  // of the next batch if state_idx01 were the beginning of the current batch.
  // So if we follow this chain forward from the start of one of the FSAs until
  // it passes the end of this FSA, we get the beginnings of the batches
  // we want.  The natural algorithm to find the beginnings of the batches
  // is sequential.
  Array2<int32_t> dest_states_powers(c, log_power + 1, num_states);
  const int32_t *arc_dest_states_data = arc_dest_states.Data(),
                *fsas_row_splits2_data = fsas.RowSplits(2).Data();
  int32_t *dest_states_power_data =
      dest_states_powers.Data();  // only process Row[0] below
  const int32_t int_max = std::numeric_limits<int32_t>::max();
  auto lambda_set_dest_states =
      [=] __host__ __device__(int32_t state_idx01) -> void {
    int32_t arc_idx01x = fsas_row_splits2_data[state_idx01];
    // If this state has arcs, let its `dest_state` be the smallest `dest_state`
    // of any of its arcs (which is the first element of those arcs' dest states
    // in `arc_dest_states_data`); otherwise, take the `dest_state` from the 1st
    // arc of the next state, which is the largest value we can take (if the
    // definition is: the highest-numbered state s for which neither this state
    // nor any later-numbered state has an arc to a state lower than s).

    // if this state has arcs,
    //    arc_idx01x is the first arc index of this state, we get the
    //    smallest dest state of this state's arcs using
    //    arc_dest_states_data[arc_idx01x]
    // else
    //    arc_idx01x is the first arc index of the next state, then
    //    arc_dest_states_data[arc_idx01x] is the largest value we can take,
    //    which is also the smallest dest state in the next state.
    int32_t dest_state =
        (arc_idx01x < num_arcs ? arc_dest_states_data[arc_idx01x] : int_max);
    dest_states_power_data[state_idx01] = dest_state;
    // if the following fails, it's either a code error or the input FSA had
    // cycles.
    K2_CHECK_GT(dest_state, state_idx01);
  };
  Eval(c, num_states, lambda_set_dest_states);

  // `num_batches_per_fsa` will be set to the number of batches of states that
  // we'll use for each FSA... it corresponds to the number of times we have
  // to follow links forward in the dest_states array till we pass the
  // end of the array for this fSA.
  Array1<int32_t> num_batches_per_fsa(c, num_fsas + 1);

  // `batch_starts` will contain the locations of the first state_idx01 for each
  // batch, but in an 'un-consolidated' format.  Specifically, for FSA with
  // index i, the batch_starts for that FSA begin at element fsa.RowSplits(1)[i]
  // of `batch_starts`.  This is just a convenient layout because we know there
  // can't be more batches than there are states.  We'll later consolidate the
  // information into a single array.
  Array1<int32_t> batch_starts(c, num_states + 1);

  int32_t *num_batches_per_fsa_data = num_batches_per_fsa.Data(),
          *batch_starts_data = batch_starts.Data();
  const int32_t *fsas_row_splits1_data = fsas.RowSplits(1).Data();

#if 0
  // This is a simple version of the kernel that demonstrates what we're trying
  // to do with the more complex code.
  auto lambda_set_batch_info_simple = [=] __host__ __device__(int32_t fsa_idx) {
    int32_t begin_state_idx01 = fsas_row_splits1_data[fsa_idx],
            end_state_idx01 = fsas_row_splits1_data[fsa_idx + 1];
    int32_t i = 0, cur_state_idx01 = begin_state_idx01;
    while (cur_state_idx01 < end_state_idx01) {
      batch_starts_data[begin_state_idx01 + i] = cur_state_idx01;
      cur_state_idx01 = dest_states_power_data[cur_state_idx01];
      ++i;
    }
    num_batches_per_fsa_data[fsa_idx] = i;
  };
  Eval(c, num_fsas, lambda_set_batch_info_simple);
#else
  int32_t stride = dest_states_powers.ElemStride0();
  for (int32_t power = 1; power <= log_power; power++) {
    const int32_t *src_data = dest_states_powers.Data() + (power - 1) * stride;
    int32_t *dest_data = dest_states_powers.Data() + power * stride;
    auto lambda_square_array =
        [=] __host__ __device__(int32_t state_idx01) -> void {
      int32_t dest_state = src_data[state_idx01],
              dest_state_sq =
                  (dest_state < num_states ? src_data[dest_state] : int_max);
      dest_data[state_idx01] = dest_state_sq;
    };
    Eval(c, num_states, lambda_square_array);
  }
  // jobs_per_fsa tells us how many separate chains of states we'll follow for
  // each FSA.
  // jobs_multiple is a kind of trick to ensure any given warp doesn't
  // issue more memory requests than it can handle at a time (we drop
  // some threads).
  int32_t jobs_per_fsa = (1 << log_power),
          jobs_multiple = (c->GetDeviceType() == kCuda ? 8 : 1);
  while (jobs_multiple > 1 && jobs_per_fsa * jobs_multiple * num_fsas > 10000)
    jobs_multiple /= 2;  // Likely won't get here.  Just reduce multiple if
                         // num-jobs is ridiculous.

  auto dest_states_powers_acc = dest_states_powers.Accessor();
  auto lambda_set_batch_info = [=] __host__ __device__(int32_t fsa_idx,
                                                       int32_t j) {
    if (j % jobs_multiple != 0)
      return;                              // a trick to avoid too much random
                                           // memory access for any given warp
    int32_t task_idx = j / jobs_multiple;  // Now 0 <= task_idx < jobs_per_fsa.

    // The task indexed `task_idx` is responsible for batches numbered
    // task_idx, task_idx + jobs_per_fsa, task_index + 2 * job_per_fsa and so
    // on, for the FSA numbered `fsa_idx`. Comparing this code to
    // `lambda_set_batch_info_simple`, this task is responsible for the
    // assignment to batch_starts_data for all i such that i % jobs_per_fsas ==
    // task_idx, together with the assignment to num_batchess_per_fsa_data if
    //  i % jobs_per_fsas == task_idx (here referring to the i value finally
    // assigned to that location).

    int32_t begin_state_idx01 = fsas_row_splits1_data[fsa_idx],
            end_state_idx01 = fsas_row_splits1_data[fsa_idx + 1];
    int32_t num_states_this_fsa = end_state_idx01 - begin_state_idx01;
    int32_t i = 0, cur_state_idx01 = begin_state_idx01;

    if (task_idx >= num_states_this_fsa) return;

    // The next loop advances `cur_state_idx01` by
    // a number of steps equal to `task_idx`.
    for (int32_t m = 0; m < log_power; ++m) {
      int32_t n = 1 << m;
      if ((task_idx & n) != 0) {
        i += n;
        int32_t next = dest_states_powers_acc(m, cur_state_idx01);
        if (next >= end_state_idx01) return;
        cur_state_idx01 = next;
      }
    }
    K2_CHECK_EQ(i, task_idx);

    while (1) {
      if (i >= num_states_this_fsa) return;
      batch_starts_data[begin_state_idx01 + i] = cur_state_idx01;
      int32_t next_state_idx01 = dest_states_powers_acc(
          log_power,
          cur_state_idx01);  // advance jobs_per_fsa = (1 << log_power) steps
      if (next_state_idx01 >= end_state_idx01) {
        // if exactly one step would also be enough to take us past the
        // boundary...
        if (dest_states_powers_acc(0, cur_state_idx01) >= end_state_idx01) {
          num_batches_per_fsa_data[fsa_idx] = i + 1;
        }
        return;
      } else {
        i += jobs_per_fsa;
        cur_state_idx01 = next_state_idx01;
      }
    }
  };
  Eval2(c, num_fsas, jobs_per_fsa * jobs_multiple, lambda_set_batch_info);
#endif
  ExclusiveSum(num_batches_per_fsa, &num_batches_per_fsa);
  Array1<int32_t> &ans_row_splits1 = num_batches_per_fsa;
  int32_t num_batches = num_batches_per_fsa[num_fsas];
  Array1<int32_t> ans_row_ids1(c, num_batches);
  RowSplitsToRowIds(ans_row_splits1, &ans_row_ids1);
  Array1<int32_t> ans_row_splits2(c, num_batches + 1);
  const int32_t *ans_row_splits1_data = ans_row_splits1.Data(),
                *ans_row_ids1_data = ans_row_ids1.Data();
  int32_t *ans_row_splits2_data = ans_row_splits2.Data();
  ans_row_splits2.Range(num_batches, 1) = num_states;  // The kernel below won't
                                                       // set this last element
  auto lambda_set_ans_row_splits2 =
      [=] __host__ __device__(int32_t idx01) -> void {
    int32_t idx0 = ans_row_ids1_data[idx01],  // Fsa index
        idx0x = ans_row_splits1_data[idx0], idx1 = idx01 - idx0x,
            fsas_idx0x = fsas_row_splits1_data[idx0],  // 1st state-idx (idx01)
                                                       // in fsas_, for this FSA
        fsas_idx01 = fsas_idx0x + idx1,  // the idx1 is actually the
                                         // batch-index, this statement reflects
                                         // the 'un-consolidated' format of
                                         // `batch_starts`.
        this_batch_start = batch_starts_data[fsas_idx01];
    ans_row_splits2_data[idx01] = this_batch_start;
  };
  Eval(c, num_batches, lambda_set_ans_row_splits2);

  RaggedShape ans_shape =
      RaggedShape3(&ans_row_splits1, &ans_row_ids1, num_batches,
                   &ans_row_splits2, nullptr, num_states);
  Array1<int32_t> ans_value = Range(c, num_states, 0);
  if (transpose) {
    ans_shape = MakeTransposable(ans_shape);
    Ragged<int32_t> ans(ans_shape, ans_value);
    return Transpose(ans);
  } else {
    return Ragged<int32_t>(ans_shape, ans_value);
  }
}

Ragged<int32_t> GetIncomingArcs(FsaVec &fsas,
                                const Array1<int32_t> &dest_states) {
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  K2_CHECK(IsCompatible(fsas, dest_states));
  ContextPtr &c = fsas.Context();
  Ragged<int32_t> dest_states_tensor(fsas.shape, dest_states);
  int32_t num_fsas = fsas.Dim0(), num_states = fsas.TotSize(1),
          num_arcs = fsas.TotSize(2);

  Array1<int32_t> incoming_arcs_order =
                      GetTransposeReordering(dest_states_tensor, num_states),

                  ans_row_ids2 = dest_states[incoming_arcs_order];
  // Note: incoming_arcs_row_ids2 will be monotonically increasing

  Array1<int32_t> ans_row_splits2(c, num_states + 1);
  RowIdsToRowSplits(ans_row_ids2, &ans_row_splits2);

  // Axis 1 corresponds to FSA states, so the row-ids and row-splits for axis
  // 1 are the same as for `fsas`.
  Array1<int32_t> ans_row_ids1 = fsas.RowIds(1),
                  ans_row_splits1 = fsas.RowSplits(1);
  return Ragged<int32_t>(
      RaggedShape3(&ans_row_splits1, &ans_row_ids1, num_states,
                   &ans_row_splits2, &ans_row_ids2, num_arcs),
      incoming_arcs_order);
}

Ragged<int32_t> GetLeavingArcIndexBatches(FsaVec &fsas,
                                          Ragged<int32_t> &state_batches) {
  K2_CHECK(IsCompatible(fsas, state_batches));
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  K2_CHECK_EQ(state_batches.NumAxes(), 3);
  ContextPtr &c = fsas.Context();
  int32_t num_fsas = fsas.Dim0(), num_states = fsas.TotSize(1),
          num_arcs = fsas.TotSize(2);
  int32_t num_batches = state_batches.Dim0();
  K2_DCHECK_EQ((state_batches.TotSize(1) / num_batches), num_fsas);
  K2_DCHECK_EQ(state_batches.NumElements(), num_states);

  // get ans_shape
  Array1<int32_t> ans_row_splits3(c, num_states + 1);
  int32_t *ans_row_splits3_data = ans_row_splits3.Data();
  const int32_t *fsa_states_row_splits_data = fsas.RowSplits(2).Data();
  const int32_t *batch_states_data = state_batches.values.Data();
  auto lambda_set_ans_row_splits3 = [=] __host__ __device__(int32_t idx) {
    int32_t state_idx = batch_states_data[idx];
    ans_row_splits3_data[idx] = fsa_states_row_splits_data[state_idx + 1] -
                                fsa_states_row_splits_data[state_idx];
  };
  Eval(c, num_states, lambda_set_ans_row_splits3);
  ExclusiveSum(ans_row_splits3, &ans_row_splits3);
  Array1<int32_t> ans_row_ids3(c, num_arcs);
  RowSplitsToRowIds(ans_row_splits3, &ans_row_ids3);
  RaggedShape ans_shape = ComposeRaggedShapes(
      state_batches.shape,
      RaggedShape2(&ans_row_splits3, &ans_row_ids3, num_arcs));

  // get ans_values
  Array1<int32_t> ans_values(c, num_arcs);
  int32_t *ans_values_data = ans_values.Data();
  const int32_t *ans_row_ids3_data = ans_row_ids3.Data();
  auto lambda_set_ans_values = [=] __host__ __device__(int32_t idx0123) {
    int32_t ans_idx012 = ans_row_ids3_data[idx0123];
    int32_t state_idx =
        batch_states_data[ans_idx012];  // state_idx is idx01 in fsas
    int32_t fsa_idx01x = fsa_states_row_splits_data[state_idx];
    // ans_idx3 is fsas_idx2, i.e. the arc idx in a state
    int32_t ans_idx3 = idx0123 - ans_row_splits3_data[ans_idx012];
    ans_values_data[idx0123] = fsa_idx01x + ans_idx3;
  };
  Eval(c, num_arcs, lambda_set_ans_values);

  return Ragged<int32_t>(ans_shape, ans_values);
}

Ragged<int32_t> GetEnteringArcIndexBatches(FsaVec &fsas,
                                           Ragged<int32_t> &incoming_arcs,
                                           Ragged<int32_t> &state_batches) {
  K2_CHECK(IsCompatible(fsas, state_batches));
  K2_CHECK(IsCompatible(fsas, incoming_arcs));
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  K2_CHECK_EQ(incoming_arcs.NumAxes(), 3);
  K2_CHECK_EQ(state_batches.NumAxes(), 3);
  ContextPtr &c = fsas.Context();
  int32_t num_fsas = fsas.Dim0(), num_states = fsas.TotSize(1),
          num_arcs = fsas.TotSize(2);
  int32_t num_batches = state_batches.Dim0();
  // just using DCHECK below to save time in production code
  K2_DCHECK_EQ((state_batches.TotSize(1) / num_batches), num_fsas);
  K2_DCHECK_EQ(state_batches.NumElements(), num_states);
  K2_DCHECK_EQ(incoming_arcs.Dim0(), num_fsas);
  K2_DCHECK_EQ(incoming_arcs.TotSize(1), num_states);
  K2_DCHECK_EQ(incoming_arcs.NumElements(), num_arcs);

  // get ans_shape
  Array1<int32_t> ans_row_splits3(c, num_states + 1);
  int32_t *ans_row_splits3_data = ans_row_splits3.Data();
  const int32_t *incoming_arcs_row_splits_data =
      incoming_arcs.RowSplits(2).Data();
  const int32_t *batch_states_data = state_batches.values.Data();
  auto lambda_set_ans_row_splits3 = [=] __host__ __device__(int32_t idx) {
    int32_t state_idx = batch_states_data[idx];
    ans_row_splits3_data[idx] = incoming_arcs_row_splits_data[state_idx + 1] -
                                incoming_arcs_row_splits_data[state_idx];
  };
  Eval(c, num_states, lambda_set_ans_row_splits3);
  ExclusiveSum(ans_row_splits3, &ans_row_splits3);
  Array1<int32_t> ans_row_ids3(c, num_arcs);
  RowSplitsToRowIds(ans_row_splits3, &ans_row_ids3);
  RaggedShape ans_shape = ComposeRaggedShapes(
      state_batches.shape,
      RaggedShape2(&ans_row_splits3, &ans_row_ids3, num_arcs));

  // get ans_values
  Array1<int32_t> ans_values(c, num_arcs);
  int32_t *ans_values_data = ans_values.Data();
  const int32_t *ans_row_ids3_data = ans_row_ids3.Data();
  const int32_t *incoming_arcs_data = incoming_arcs.values.Data();
  auto lambda_set_ans_values = [=] __host__ __device__(int32_t idx0123) {
    int32_t ans_idx012 = ans_row_ids3_data[idx0123];
    int32_t state_idx =
        batch_states_data[ans_idx012];  // state_idx is idx01 in incoming_arcs
    int32_t incoming_arcs_idx01x = incoming_arcs_row_splits_data[state_idx];
    // ans_idx3 is incoming_arcs_idx2, i.e. the entering arc idx for a state
    int32_t ans_idx3 = idx0123 - ans_row_splits3_data[ans_idx012];
    int32_t incoming_arcs_idx012 = incoming_arcs_idx01x + ans_idx3;
    ans_values_data[idx0123] = incoming_arcs_data[incoming_arcs_idx012];
  };
  Eval(c, num_arcs, lambda_set_ans_values);

  return Ragged<int32_t>(ans_shape, ans_values);
}

FsaVec ConvertDenseToFsaVec(DenseFsaVec &src) {
  ContextPtr &c = src.shape.Context();
  // caution: 'num_symbols' is the number of symbols excluding the final-symbol
  // -1.
  int32_t num_fsas = src.shape.Dim0(), num_symbols = src.scores.Dim1() - 1;
  // the "1" is the extra state per FSA we need in the FsaVec format,
  // for the final-state.
  RaggedShape fsa2state = ChangeSublistSize(src.shape, 1);

  int32_t num_states = src.shape.NumElements() + num_fsas,
          num_arcs = src.shape.NumElements() * num_symbols -
                     (num_symbols - 1) * num_fsas;
  Array1<int32_t> row_splits2(c, num_states), row_ids2(c, num_arcs);
  const int32_t *row_ids1_data = fsa2state.RowIds(1).Data(),
                *src_row_ids1_data = src.shape.RowIds(1).Data(),
                *src_row_splits1_data = src.shape.RowSplits(1).Data();
  Array1<Arc> arcs(c, num_arcs);
  Arc *arcs_data = arcs.Data();

  auto scores_acc = src.scores.Accessor();

  // each FSA we return has one extra state (we add the final state).
  int32_t *row_splits2_data = row_splits2.Data(),
          *row_ids2_data = row_ids2.Data();

  // 0 <= s < num_symbols; note, `num_symbols` excludes the final-symbol (-1).
  auto lambda_set_arcs_etc = [=] __host__ __device__(int32_t src_state_idx01,
                                                     int32_t s) -> void {
    int32_t fsa_idx0 = src_row_ids1_data[src_state_idx01],
            src_state_idx0x = src_row_splits1_data[fsa_idx0],
            state_idx1 = src_state_idx01 - src_state_idx0x,
            src_next_state_idx0x = src_row_splits1_data[fsa_idx0 + 1],
            src_num_states1 = src_next_state_idx0x - src_state_idx0x,
            ans_state_idx01 =
                src_state_idx01 + fsa_idx0;  // add final-state per FSA..

    // arc_idx0x is the 1st arc-index of the FSA we are creating.. each source
    // state has `num_symbols` arcs leaving it except the last one of each FSA,
    // which has 1 arc leaving it (to the final-state).
    int32_t arc_idx0xx =
                (src_state_idx0x * num_symbols) - fsa_idx0 * (num_symbols - 1),
            arc_idx01x = arc_idx0xx + (state_idx1 * num_symbols),
            arc_idx012 = arc_idx01x + s;
    int32_t symbol_offset;
    if (state_idx1 + 1 < src_num_states1) {
      symbol_offset = -1;
      if (s > 0) return;  // we just need the arc with -1.
    } else {
      symbol_offset = 0;
    }
    // the "+ 1" is because index 0 in `scores` is for the final-symbol -1,
    // then 0, 1, etc.
    int32_t symbol_index_in_scores = s + symbol_offset + 1;
    arcs_data[arc_idx012] =
        Arc(state_idx1, state_idx1 + 1, s + symbol_offset,
            FloatAsInt(scores_acc(src_state_idx01, symbol_index_in_scores)));
    row_ids2_data[arc_idx012] = ans_state_idx01;
    if (s == 0) {  // 1st arc for this state.
      row_splits2_data[ans_state_idx01] = arc_idx012;
      K2_CHECK(row_ids1_data[ans_state_idx01] == fsa_idx0);
      if (src_state_idx01 == 0) row_splits2_data[num_states] = num_arcs;
    }
  };
  Eval2(c, src.shape.NumElements(), num_symbols, lambda_set_arcs_etc);

  RaggedShape state2arc = RaggedShape2(&row_splits2, &row_ids2, num_arcs);
  return Ragged<Arc>(ComposeRaggedShapes(fsa2state, state2arc), arcs);
}

}  // namespace k2
