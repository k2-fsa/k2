#include "hip/hip_runtime.h"
/**
 * @brief Utilities for creating FSAs.
 *
 * Note that serializations are done in Python.
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey)
 *                      Mobvoi Inc.        (authors: Fangjun Kuang)
 *                      Guoguo Chen
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <algorithm>
#include <limits>
#include <sstream>
#include <utility>
#include <vector>

#include "k2/csrc/context.h"
#include "k2/csrc/fsa_utils.h"

namespace k2 {

// field separator within a line for a text form FSA
static constexpr const char *kDelim = " \t";

// Convert a string to an integer. Abort the program on failure.
static int32_t StringToInt(const std::string &s) {
  K2_CHECK(!s.empty());

  bool ok = false;
  char *p = nullptr;
  // std::strtol requires a `long` type
  long n = std::strtol(s.c_str(), &p, 10);  // NOLINT
  if (*p == '\0') ok = true;

  auto res = static_cast<int32_t>(n);
  if (n != res) ok = false;  // out of range

  K2_CHECK(ok) << "Failed to convert " << s << " to an integer";

  return res;
}

// Convert a string to a float. Abort the program on failure.
// TODO(guoguo): We may run into locale problems, with comma vs. period for
//               decimals. We have to test if the C code will behave the same
//               w.r.t. locale as Python does.
static float StringToFloat(const std::string &s) {
  K2_CHECK(!s.empty());
  char *p = nullptr;
  float f = std::strtof(s.c_str(), &p);
  if (*p != '\0') K2_LOG(FATAL) << "Failed to convert " << s << " to a float";
  return f;
}

// Trim leading and trailing spaces of a string.
static void TrimString(std::string *s) {
  K2_CHECK_NE(s, nullptr);
  auto not_space = [](int32_t c) -> bool { return std::isspace(c) == 0; };

  s->erase(s->begin(), std::find_if(s->begin(), s->end(), not_space));
  s->erase(std::find_if(s->rbegin(), s->rend(), not_space).base(), s->end());
}

/* Split a string to a vector of strings using a set of delimiters.

   Example usage:

   @code
    std::string in = "1 2 3";
    const char *delim = " \t";
    std::vector<std::string> out;
    SplitStringToVector(in, delim, &out);
   @endcode

   @param [in]  in    The input string to be split.
   @param [in]  delim A string of delimiters.
   @param [out] out   It saves the split result.
*/
static void SplitStringToVector(const std::string &in, const char *delim,
                                std::vector<std::string> *out) {
  K2_CHECK_NE(delim, nullptr);
  K2_CHECK_NE(out, nullptr);
  out->clear();
  std::size_t start = 0;
  while (true) {
    auto pos = in.find_first_of(delim, start);
    if (pos == std::string::npos) break;

    auto sub = in.substr(start, pos - start);
    start = pos + 1;

    TrimString(&sub);
    if (!sub.empty()) out->emplace_back(std::move(sub));
  }

  if (start < in.size()) {
    auto sub = in.substr(start);
    TrimString(&sub);
    if (!sub.empty()) out->emplace_back(std::move(sub));
  }
}

/* Create an acceptor from a stream, assuming the acceptor is in the k2 format:

   src_state1 dest_state1 label1 score1
   src_state2 dest_state2 label2 score2
   ... ...
   final_state

   The source states will be in non-descending order, and the final state does
   not bear a cost/score -- we put the cost/score on the arc that connects to
   the final state and set its label to -1.

   @param [in]  is    The input stream that contains the acceptor.

   @return It returns an Fsa on CPU.
*/
static Fsa K2AcceptorFromStream(std::istringstream &is) {
  std::vector<Arc> arcs;
  std::vector<std::string> splits;
  std::string line;

  bool finished = false;  // when the final state is read, set it to true.
  while (std::getline(is, line)) {
    SplitStringToVector(line, kDelim,
                        &splits);  // splits is cleared in the function
    if (splits.empty()) continue;  // this is an empty line

    K2_CHECK_EQ(finished, false);

    auto num_fields = splits.size();
    if (num_fields == 4u) {
      //   0            1          2      3
      // src_state  dest_state   label  score
      int32_t src_state = StringToInt(splits[0]);
      int32_t dest_state = StringToInt(splits[1]);
      int32_t symbol = StringToInt(splits[2]);
      float score = StringToFloat(splits[3]);
      arcs.emplace_back(src_state, dest_state, symbol, score);
    } else if (num_fields == 1u) {
      //   0
      // final_state
      (void)StringToInt(splits[0]);  // this is a final state
      finished = true;               // set finish
    } else {
      K2_LOG(FATAL) << "Invalid line: " << line
                    << "\nk2 acceptor expects a line with 1 (final_state) or "
                       "4 (src_state dest_state label score) fields";
    }
  }

  K2_CHECK_EQ(finished, true) << "The last line should be the final state";

  bool error = true;
  Array1<Arc> array(GetCpuContext(), arcs);
  auto fsa = FsaFromArray1(array, &error);
  K2_CHECK_EQ(error, false);

  return fsa;
}

/* Create a transducer from a stream, assuming the transducer is in the K2
   format:

   src_state1 dest_state1 label1 aux_label1 score1
   src_state2 dest_state2 label2 aux_label2 score2
   ... ...
   final_state

   The source states will be in non-descending order, and the final state does
   not bear a cost/score -- we put the cost/score on the arc that connects to
   the final state and set its label to -1.

   @param [in]  is    The input stream that contains the transducer.

   @return It returns an Fsa on CPU.
*/
static Fsa K2TransducerFromStream(std::istringstream &is,
                                  Array1<int32_t> *aux_labels) {
  K2_CHECK(aux_labels != nullptr);

  std::vector<int32_t> aux_labels_internal;
  std::vector<Arc> arcs;
  std::vector<std::string> splits;
  std::string line;

  bool finished = false;  // when the final state is read, set it to true.
  while (std::getline(is, line)) {
    SplitStringToVector(line, kDelim,
                        &splits);  // splits is cleared in the function
    if (splits.empty()) continue;  // this is an empty line

    K2_CHECK_EQ(finished, false);

    auto num_fields = splits.size();
    if (num_fields == 5u) {
      //   0           1         2         3        4
      // src_state  dest_state label   aux_label  score
      int32_t src_state = StringToInt(splits[0]);
      int32_t dest_state = StringToInt(splits[1]);
      int32_t symbol = StringToInt(splits[2]);
      int32_t aux_label = StringToInt(splits[3]);
      float score = StringToFloat(splits[4]);
      arcs.emplace_back(src_state, dest_state, symbol, score);
      aux_labels_internal.push_back(aux_label);
    } else if (num_fields == 1u) {
      //   0
      // final_state
      (void)StringToInt(splits[0]);
      finished = true;  // set finish
    } else {
      K2_LOG(FATAL) << "Invalid line: " << line
                    << "\nk2 transducer expects a line with 1 (final_state) or "
                       "5 (src_state dest_state label aux_label score) fields";
    }
  }

  K2_CHECK_EQ(finished, true) << "The last line should be the final state";

  auto cpu_context = GetCpuContext();
  *aux_labels = Array1<int32_t>(cpu_context, aux_labels_internal);
  Array1<Arc> array(cpu_context, arcs);

  bool error = true;
  auto fsa = FsaFromArray1(array, &error);
  K2_CHECK_EQ(error, false);

  return fsa;
}

/* Create an acceptor from a stream, assuming the acceptor is in the OpenFST
   format:

   src_state1 dest_state1 label1 score1
   src_state2 dest_state2 label2 score2
   ... ...
   final_state final_score

   We will negate the cost/score when we read them in. Also note, OpenFST may
   omit the cost/score if it is 0.0.

   We always create the super final state. If there are final state(s) in the
   original FSA, then we add arc(s) from the original final state(s) to the
   super final state, with the (negated) old final state cost/score as its
   cost/score, and -1 as its label.

   @param [in]  is    The input stream that contains the acceptor.

   @return It returns an Fsa on CPU.
*/
static Fsa OpenFstAcceptorFromStream(std::istringstream &is) {
  std::vector<Arc> arcs;
  std::vector<std::vector<Arc>> state_to_arcs;  // indexed by states
  std::vector<std::string> splits;
  std::string line;

  int32_t max_state = -1;
  int32_t num_arcs = 0;
  std::vector<int32_t> original_final_states;
  std::vector<float> original_final_weights;
  while (std::getline(is, line)) {
    SplitStringToVector(line, kDelim,
                        &splits);  // splits is cleared in the function
    if (splits.empty()) continue;  // this is an empty line

    auto num_fields = splits.size();
    if (num_fields == 3u || num_fields == 4u) {
      //   0            1          2
      // src_state  dest_state   label
      //
      // or
      //
      //   0            1          2      3
      // src_state  dest_state   label  score
      int32_t src_state = StringToInt(splits[0]);
      int32_t dest_state = StringToInt(splits[1]);
      int32_t symbol = StringToInt(splits[2]);
      float score = 0.0f;
      if (num_fields == 4u) score = -1.0f * StringToFloat(splits[3]);

      // Add the arc to "state_to_arcs".
      ++num_arcs;
      max_state = std::max(max_state, std::max(src_state, dest_state));
      if (static_cast<int32_t>(state_to_arcs.size()) <= src_state)
        state_to_arcs.resize(src_state + 1);
      state_to_arcs[src_state].emplace_back(src_state, dest_state, symbol,
                                            score);
    } else if (num_fields == 1u || num_fields == 2u) {
      //   0            1
      // final_state  score
      float score = 0.0f;
      if (num_fields == 2u) score = -1.0f * StringToFloat(splits[1]);
      original_final_states.push_back(StringToInt(splits[0]));
      original_final_weights.push_back(score);
      max_state = std::max(max_state, original_final_states.back());
    } else {
      K2_LOG(FATAL) << "Invalid line: " << line
                    << "\nOpenFST acceptor expects a line with 1 (final_state),"
                       " 2 (final_state score), 3 (src_state dest_state label) "
                       "or 4 (src_state dest_state label score) fields.";
    }
  }

  K2_CHECK(is.eof());

  // Post processing on final states. If there are final state(s) in the
  // original FSA, we add the super final state as well as arc(s) from original
  // final state(s) to the super final state. Otherwise, the super final state
  // will be added by FsaFromArray1 (since there's no arc with label
  // kFinalSymbol).
  if (original_final_states.size() > 0) {
    K2_CHECK_EQ(original_final_states.size(), original_final_weights.size());
    int32_t super_final_state = max_state + 1;
    state_to_arcs.resize(super_final_state);
    for (std::size_t i = 0; i != original_final_states.size(); ++i) {
      state_to_arcs[original_final_states[i]].emplace_back(
          original_final_states[i], super_final_state,
          -1,  // kFinalSymbol
          original_final_weights[i]);
      ++num_arcs;
    }
  }

  // Move arcs from "state_to_arcs" to "arcs".
  int32_t arc_index = 0;
  arcs.resize(num_arcs);
  for (std::size_t s = 0; s < state_to_arcs.size(); ++s) {
    for (std::size_t a = 0; a < state_to_arcs[s].size(); ++a) {
      K2_CHECK_GT(num_arcs, arc_index);
      arcs[arc_index] = state_to_arcs[s][a];
      ++arc_index;
    }
  }
  K2_CHECK_EQ(num_arcs, arc_index);

  bool error = true;
  Array1<Arc> array(GetCpuContext(), arcs);
  // FsaFromArray1 will add a super final state if the original FSA doesn't have
  // a final state.
  auto fsa = FsaFromArray1(array, &error);
  K2_CHECK_EQ(error, false);

  return fsa;
}

/* Create a transducer from a stream, assuming the transducer is in the OpenFST
   format:

   src_state1 dest_state1 label1 aux_label1 score1
   src_state2 dest_state2 label2 aux_label2 score2
   ... ...
   final_state final_score

   We will negate the cost/score when we read them in. Also note, OpenFST may
   omit the cost/score if it is 0.0.

   We always create the super final state. If there are final state(s) in the
   original FST, then we add arc(s) from the original final state(s) to the
   super final state, with the (negated) old final state cost/score as its
   cost/score, -1 as its label and 0 as its aux_label.

   @param [in]  is    The input stream that contains the transducer.

   @return It returns an Fsa on CPU.
*/
static Fsa OpenFstTransducerFromStream(std::istringstream &is,
                                       Array1<int32_t> *aux_labels) {
  K2_CHECK(aux_labels != nullptr);

  std::vector<std::vector<int32_t>> state_to_aux_labels;  // indexed by states
  std::vector<std::vector<Arc>> state_to_arcs;            // indexed by states
  std::vector<int32_t> aux_labels_internal;
  std::vector<Arc> arcs;
  std::vector<std::string> splits;
  std::string line;

  int32_t max_state = -1;
  int32_t num_arcs = 0;
  std::vector<int32_t> original_final_states;
  std::vector<float> original_final_weights;
  while (std::getline(is, line)) {
    SplitStringToVector(line, kDelim,
                        &splits);  // splits is cleared in the function
    if (splits.empty()) continue;  // this is an empty line

    auto num_fields = splits.size();
    if (num_fields == 4u || num_fields == 5u) {
      //   0           1         2         3
      // src_state  dest_state label   aux_label
      //
      // or
      //
      //   0           1         2         3        4
      // src_state  dest_state label   aux_label  score
      int32_t src_state = StringToInt(splits[0]);
      int32_t dest_state = StringToInt(splits[1]);
      int32_t symbol = StringToInt(splits[2]);
      int32_t aux_label = StringToInt(splits[3]);
      float score = 0.0f;
      if (num_fields == 5u) score = -1.0f * StringToFloat(splits[4]);

      // Add the arc to "state_to_arcs", and aux_label to "state_to_aux_labels"
      ++num_arcs;
      max_state = std::max(max_state, std::max(src_state, dest_state));
      if (static_cast<int32_t>(state_to_arcs.size()) <= src_state) {
        state_to_arcs.resize(src_state + 1);
        state_to_aux_labels.resize(src_state + 1);
      }
      state_to_arcs[src_state].emplace_back(src_state, dest_state, symbol,
                                            score);
      state_to_aux_labels[src_state].push_back(aux_label);
    } else if (num_fields == 1u || num_fields == 2u) {
      //   0
      // final_state
      //
      // or
      //
      //   0            1
      // final_state  score
      // There could be multiple final states, so we first have to collect all
      // the final states, and then work out the super final state.
      float score = 0.0f;
      if (num_fields == 2u) score = -1.0f * StringToFloat(splits[1]);
      original_final_states.push_back(StringToInt(splits[0]));
      original_final_weights.push_back(score);
      max_state = std::max(max_state, original_final_states.back());
    } else {
      K2_LOG(FATAL) << "Invalid line: " << line
                    << "\nOpenFST transducer expects a line with "
                       "1 (final_state), 2 (final_state score), "
                       "4 (src_state dest_state label aux_label) or "
                       "5 (src_state dest_state label aux_label score) fields.";
    }
  }

  K2_CHECK(is.eof());

  // Post processing on final states. If there are final state(s) in the
  // original FST, we add the super final state as well as arc(s) from original
  // final state(s) to the super final state. Otherwise, the super final state
  // will be added by FsaFromArray1 (since there's no arc with label
  // kFinalSymbol).
  if (original_final_states.size() > 0) {
    K2_CHECK_EQ(original_final_states.size(), original_final_weights.size());
    int32_t super_final_state = max_state + 1;
    state_to_arcs.resize(super_final_state);
    state_to_aux_labels.resize(super_final_state);
    for (std::size_t i = 0; i != original_final_states.size(); ++i) {
      state_to_arcs[original_final_states[i]].emplace_back(
          original_final_states[i], super_final_state,
          -1,  // kFinalSymbol
          original_final_weights[i]);
      // TODO(guoguo) We are not sure yet what to put as the auxiliary label for
      //              arcs entering the super final state. The only real choices
      //              are kEpsilon or kFinalSymbol. We are using kEpsilon for
      //              now.
      state_to_aux_labels[original_final_states[i]].push_back(0);  // kEpsilon
      ++num_arcs;
    }
  }

  // Move arcs from "state_to_arcs" to "arcs", and aux_labels from
  // "state_to_aux_labels" to "aux_labels_internal"
  int32_t arc_index = 0;
  arcs.resize(num_arcs);
  aux_labels_internal.resize(num_arcs);
  K2_CHECK_EQ(state_to_arcs.size(), state_to_aux_labels.size());
  for (std::size_t s = 0; s < state_to_arcs.size(); ++s) {
    K2_CHECK_EQ(state_to_arcs[s].size(), state_to_aux_labels[s].size());
    for (std::size_t a = 0; a < state_to_arcs[s].size(); ++a) {
      K2_CHECK_GT(num_arcs, arc_index);
      arcs[arc_index] = state_to_arcs[s][a];
      aux_labels_internal[arc_index] = state_to_aux_labels[s][a];
      ++arc_index;
    }
  }
  K2_CHECK_EQ(num_arcs, arc_index);

  auto cpu_context = GetCpuContext();
  *aux_labels = Array1<int32_t>(cpu_context, aux_labels_internal);
  Array1<Arc> array(cpu_context, arcs);

  bool error = true;
  // FsaFromArray1 will add a super final state if the original FSA doesn't have
  // a final state.
  auto fsa = FsaFromArray1(array, &error);
  K2_CHECK_EQ(error, false);

  return fsa;
}

Fsa FsaFromString(const std::string &s, bool openfst /*= false*/,
                  Array1<int32_t> *aux_labels /*= nullptr*/) {
  std::istringstream is(s);
  K2_CHECK(is);

  if (openfst == false && aux_labels == nullptr)
    return K2AcceptorFromStream(is);
  else if (openfst == false && aux_labels != nullptr)
    return K2TransducerFromStream(is, aux_labels);
  else if (openfst == true && aux_labels == nullptr)
    return OpenFstAcceptorFromStream(is);
  else if (openfst == true && aux_labels != nullptr)
    return OpenFstTransducerFromStream(is, aux_labels);

  return Fsa();  // unreachable code
}

std::string FsaToString(const Fsa &fsa, bool openfst /*= false*/,
                        const Array1<int32_t> *aux_labels /*= nullptr*/) {
  K2_CHECK_EQ(fsa.NumAxes(), 2);

  if (fsa.Context()->GetDeviceType() != kCpu) {
    Fsa _fsa = fsa.To(GetCpuContext());
    Array1<int32_t> _aux_labels;
    if (aux_labels) _aux_labels = aux_labels->To(_fsa.Context());
    return FsaToString(_fsa, openfst, aux_labels ? &_aux_labels : nullptr);
  }

  K2_CHECK_EQ(fsa.Context()->GetDeviceType(), kCpu);
  const Array1<int32_t> &row_splits = fsa.shape.RowSplits(1);
  const Array1<Arc> &arcs = fsa.values;

  const int32_t *p = nullptr;
  if (aux_labels != nullptr) {
    K2_CHECK(IsCompatible(fsa, *aux_labels));
    K2_CHECK_EQ(aux_labels->Dim(), arcs.Dim());
    p = aux_labels->Data();
  }
  float scale = 1;
  if (openfst) scale = -1;

  std::ostringstream os;

  int32_t n = arcs.Dim();
  char sep = ' ';
  char line_sep = '\n';
  for (int32_t i = 0; i != n; ++i) {
    const auto &arc = arcs[i];
    os << arc.src_state << sep << arc.dest_state << sep << arc.symbol << sep;
    if (p != nullptr) os << p[i] << sep;
    os << (scale * arc.score) << line_sep;
  }
  os << (fsa.shape.Dim0() - 1) << line_sep;
  return os.str();
}

Array1<int32_t> GetDestStates(FsaVec &fsas, bool as_idx01) {
  ContextPtr c = fsas.Context();
  int32_t num_arcs = fsas.NumElements();
  Array1<int32_t> ans(c, num_arcs);
  Arc *arcs_data = fsas.values.Data();
  int32_t *ans_data = ans.Data();
  if (!as_idx01) {
    const Arc *arcs = fsas.values.Data();
    auto lambda_set_dest_states1 = [=] __host__ __device__(int32_t arc_idx012) {
      ans_data[arc_idx012] = arcs[arc_idx012].dest_state;
    };
    Eval(c, num_arcs, lambda_set_dest_states1);
  } else {
    const int32_t *row_ids2 = fsas.RowIds(2).Data();
    auto lambda_set_dest_states01 = [=] __host__ __device__(
                                        int32_t arc_idx012) {
      int32_t src_state = arcs_data[arc_idx012].src_state,
              dest_state = arcs_data[arc_idx012].dest_state;
      // (row_ids2[arc_idx012] - src_state) is the same as
      // row_splits1[row_ids1[row_ids2[arc_idx012]]]; it's the idx01 of the 1st
      // state in this FSA.
      ans_data[arc_idx012] = dest_state + (row_ids2[arc_idx012] - src_state);
    };
    Eval(c, num_arcs, lambda_set_dest_states01);
  }
  return ans;
}

Ragged<int32_t> GetBatches(FsaVec &fsas, bool transpose) {
  K2_CHECK(fsas.NumAxes() == 3);
  ContextPtr c = fsas.Context();
  Array1<int32_t> arc_dest_states = GetDestStates(fsas, true);

  MonotonicLowerBound(arc_dest_states, &arc_dest_states);

  int32_t num_fsas = fsas.Dim0(), num_states = fsas.TotSize(1),
          num_arcs = fsas.TotSize(2);

  // We can tune `log_power` as a tradeoff between work done and clock time on
  // GPU.
  int32_t log_power = (c->GetDeviceType() == kCpu ? 0 : 4);

  int32_t max_num_states = fsas.shape.MaxSize(1);
  // the following avoids doing too much extra work accumulating powers
  // of 'dest_states' for very small problem sizes.
  while (log_power > 0 && (1 << (1 + log_power)) > max_num_states) log_power--;

  // Ignoring edge effects: `dest_states_powers[0]` is just an array indexed by
  // state_idx01, that gives us the dest_state_idx01 that would be the beginning
  // of the next batch if state_idx01 were the beginning of the current batch.
  // So if we follow this chain forward from the start of one of the FSAs until
  // it passes the end of this FSA, we get the beginnings of the batches
  // we want.  The natural algorithm to find the beginnings of the batches
  // is sequential.
  Array2<int32_t> dest_states_powers(c, num_states, log_power + 1);
  const int32_t *arc_dest_states_data = arc_dest_states.Data(),
                *fsas_row_splits2_data = fsas.RowSplits(2).Data();
  int32_t *dest_states_data = dest_states_powers.Data();
  const int32_t int_max = std::numeric_limits<int32_t>::max();
  auto lambda_set_dest_states =
      [=] __host__ __device__(int32_t state_idx01) -> void {
    int32_t arc_idx01x = fsas_row_splits2_data[state_idx01],
            next_arc_idx01x = fsas_row_splits2_data[arc_idx01x];
    // If this state has arcs, let its `dest_state` be the largest `dest_state`
    // of any of its arcs (which is the last one); otherwise, take the
    // `dest_state` from the 1st arc of the next state, which is the largest
    // value we can take (if the definition is: the highest-numbered state s for
    // which neither this state nor any later-numbered state has an arc to a
    // state lower than s).
    int32_t arc_idx012 = max(arc_idx01x, next_arc_idx01x - 1);
    int32_t dest_state =
        (arc_idx012 < num_arcs ? arc_dest_states_data[arc_idx012] : int_max);
    dest_states_data[state_idx01] = dest_state;
    // if the following fails, it's either a code error or the input FSA had
    // cycles.
    K2_CHECK_GT(dest_state, state_idx01);
  };
  Eval(c, num_states, lambda_set_dest_states);

  for (int32_t power = 1; power <= log_power; power++) {
    int32_t stride = dest_states_powers.ElemStride0();
    const int32_t *src_data = dest_states_powers.Data() + (power - 1) * stride;
    int32_t *dest_data = dest_states_powers.Data() + power * stride;
    auto lambda_square_array =
        [=] __host__ __device__(int32_t state_idx01) -> void {
      int32_t dest_state = src_data[state_idx01],
              dest_state_sq =
                  (dest_state < num_states ? src_data[dest_state] : int_max);
      dest_data[state_idx01] = dest_state_sq;
    };
    Eval(c, num_states, lambda_square_array);
  }

  // `num_batches_per_fsa` will be set to the number of batches of states that
  // we'll use for each FSA... it corresponds to the number of times we have
  // to follow links forward in the dest_states array till we pass the
  // end of the array for this fSA.
  Array1<int32_t> num_batches_per_fsa(c, num_fsas + 1);

  // `batch_starts` will contain the locations of the first state_idx01 for each
  // batch, but in an 'un-consolidated' format.  Specifically, for FSA with
  // index i, the batch_starts for that FSA begin at element fsa.RowSplits(1)[i]
  // of `batch_starts`.  This is just a convenient layout because we know there
  // can't be more batches than there are states.  We'll later consolidate the
  // information into a single array.
  Array1<int32_t> batch_starts(c, num_states + 1);

  int32_t *num_batches_per_fsa_data = num_batches_per_fsa.Data(),
          *batch_starts_data = batch_starts.Data();
  const int32_t *fsas_row_splits1_data = fsas.RowSplits(1).Data();

  // jobs_per_fsa tells us how many separate chains of states we'll follow for
  // each FSA.
  // jobs_multiple is a kind of trick to ensure any given warp doesn't
  // issue more memory requests than it can handle at a time (we drop
  // some threads).
  int32_t jobs_per_fsa = (1 << log_power),
          jobs_multiple = (c->GetDeviceType() == kCuda ? 8 : 1);
  while (jobs_multiple > 1 && jobs_per_fsa * jobs_multiple * num_fsas > 10000)
    jobs_multiple /= 2;  // Likely won't get here.  Just reduce multiple if
                         // num-jobs is ridiculous.

    // TODO(Dan): after debugging this version, change it to 0 and debug the
    // more complex version.
#if 1
  // This is a simple version of the kernel that demonstrates what we're trying
  // to do with the more complex code.
  auto lambda_set_batch_info_simple = [=] __host__ __device__(int32_t fsa_idx) {
    int32_t begin_state_idx01 = fsas_row_splits1_data[fsa_idx],
            end_state_idx01 = fsas_row_splits1_data[fsa_idx + 1];
    int32_t i = 0, cur_state_idx01 = begin_state_idx01;
    while (cur_state_idx01 < end_state_idx01) {
      batch_starts_data[begin_state_idx01 + (i++)] = cur_state_idx01;
      cur_state_idx01 = dest_states_data[cur_state_idx01];
    }
    num_batches_per_fsa_data[fsa_idx] = i;
  };
  Eval(c, num_fsas, lambda_set_batch_info_simple);
#else
  auto dest_states_powers_acc = dest_states_powers.Accessor();
  auto lambda_set_batch_info = [=] __host__ __device__(int32_t fsa_idx,
                                                       int32_t j) {
    if (j % jobs_multiple != 0)
      return;                              // a trick to avoid too much random
                                           // memory access for any given warp
    int32_t task_idx = j / jobs_multiple;  // Now 0 <= j < jobs_per_fsa.

    // The task indexed `task_idx` is responsible for batches numbered
    // task_idx, task_idx + jobs_per_fsa, task_index + 2 * job_per_fsa and so
    // on, for the FSA numbered `fsa_idx`. Comparing this code to
    // `lambda_set_batch_info_simple`, this task is responsible for the
    // assignment to batch_starts_data for all i such that i % jobs_per_fsas ==
    // task_idx, together with the assignment to num_batchess_per_fsa_data if
    //  i % jobs_per_fsas == task_idx (here referring to the i value finally
    // assigned to that location).

    int32 begin_state_idx01 = fsas_row_splits1_data[fsa_idx],
          end_state_idx01 = fsas_row_splits1_data[fsa_idx + 1];
    int32_t i = 0, cur_state_idx01 = begin_state_idx01;

    // The next loop advances `cur_state_idx01` by
    // a number of steps equal to `task_idx`.
    for (int32_t j = 0; j < log_power; j++) {
      int32_t n = 1 << j;
      if (task_idx % n != 0) {
        i += n;
        int32_t next = dest_state_powers_acc(j, cur_state_idx01);
        if (next >= end_state_idx01) return;
        cur_state_idx01 = next;
      }
    }
    K2_CHECK_EQ(i, task_idx);

    while (1) {
      batch_starts_data[begin_state_idx01 + i] = cur_state_idx01;
      int32_t next_state_idx01 =
          dest_states_powers_acc(log_power, cur_state_idx01);
      if (next_state_idx01 >= end_state_idx01) {
        // if exactly one step would also be enough to take us past the
        // boundary...
        if (dest_states_powers_acc(0, cur_state_idx01) >= next_state_idx01) {
          num_batches_per_fsa_data[fsa_idx] = i + 1;
          return;
        } else {
          i += cur_state_idx01;
        }
      }
    }
  };
  Eval(c, num_fsas, jobs_per_fsa * jobs_multiple, lambda_set_batch_info);
#endif
  ExclusiveSum(num_batches_per_fsa, &num_batches_per_fsa);
  Array1<int32_t> &ans_row_splits1 = num_batches_per_fsa;
  int32_t num_batches = num_batches_per_fsa[num_fsas];
  Array1<int32_t> ans_row_ids1(c, num_batches);
  RowSplitsToRowIds(ans_row_splits1, &ans_row_ids1);
  Array1<int32_t> ans_row_splits2(c, num_batches + 1);
  const int32_t *ans_row_splits1_data = ans_row_splits1.Data(),
                *ans_row_ids1_data = ans_row_ids1.Data();
  int32_t *ans_row_splits2_data = ans_row_splits2.Data();
  ans_row_splits2.Range(num_batches, 1) = num_states;  // The kernel below won't
                                                       // set this last element
  auto lambda_set_ans_row_ids2 =
      [=] __host__ __device__(int32_t idx01) -> void {
    int32_t idx0 = ans_row_ids1_data[idx01],  // Fsa index
        idx0x = ans_row_splits1_data[idx0], idx1 = idx01 - idx0x,
            fsas_idx0x = fsas_row_splits1_data[idx0],  // 1st state-idx (idx01)
                                                       // in fsas_, for this FSA
        fsas_idx01 = fsas_idx0x + idx1,  // the idx1 is actually the
                                         // batch-index, this statement reflects
                                         // the 'un-consolidated' format of
                                         // `batch_starts`.
        this_batch_start = batch_starts_data[fsas_idx01];
    ans_row_splits2_data[idx01] = this_batch_start;
  };
  Eval(c, num_batches, lambda_set_ans_row_ids2);

  Ragged<int32_t> ans(RaggedShape3(&ans_row_splits1, &ans_row_ids1, num_batches,
                                   &ans_row_splits2, nullptr, num_states),
                      Range(c, 0, num_states));
  if (transpose)
    return Transpose(ans);
  else
    return ans;
}

}  // namespace k2
