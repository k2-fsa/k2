#include "hip/hip_runtime.h"
/**
 * @brief fsa_algo  Implementation of FSA algorithm wrappers from fsa_algo.h

 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey)
 *                      Mobvoi Inc.        (authors: Fangjun Kuang)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <vector>

#include "k2/csrc/array_ops.h"
#include "k2/csrc/fsa_algo.h"
#include "k2/csrc/host/connect.h"
#include "k2/csrc/host/intersect.h"
#include "k2/csrc/host_shim.h"

// this contains a subset of the algorithms in fsa_algo.h; currently it just
// contains one that are wrappings of the corresponding algorithms in
// host/.
namespace k2 {

bool RecursionWrapper(bool (*f)(Fsa &, Fsa *, Array1<int32_t> *), Fsa &src,
                      Fsa *dest, Array1<int32_t> *arc_map) {
  // src is actually an FsaVec.  Just recurse for now.
  int32_t num_fsas = src.shape.Dim0();
  std::vector<Fsa> srcs(num_fsas), dests(num_fsas);
  std::vector<Array1<int32_t>> arc_maps(num_fsas);
  for (int32_t i = 0; i < num_fsas; i++) {
    srcs[i] = src.Index(0, i);
    // Recurse.
    if (!f(srcs[i], &(dests[i]), (arc_map ? &(arc_maps[i]) : nullptr)))
      return false;
  }
  *dest = Stack(0, num_fsas, &(dests[0]));
  if (arc_map) *arc_map = Append(num_fsas, &(arc_maps[0]));
  return true;
}

bool ConnectFsa(Fsa &src, Fsa *dest, Array1<int32_t> *arc_map) {
  int32_t num_axes = src.NumAxes();
  if (num_axes < 2 || num_axes > 3) {
    K2_LOG(FATAL) << "Input has bad num-axes " << num_axes;
  } else if (num_axes == 3) {
    return RecursionWrapper(ConnectFsa, src, dest, arc_map);
  }

  k2host::Fsa host_fsa = FsaToHostFsa(src);
  k2host::Connection c(host_fsa);
  k2host::Array2Size<int32_t> size;
  c.GetSizes(&size);
  FsaCreator creator(size);
  k2host::Fsa host_dest_fsa = creator.GetHostFsa();
  int32_t *arc_map_data = nullptr;
  if (arc_map != nullptr) {
    *arc_map = Array1<int32_t>(src.Context(), size.size2);
    arc_map_data = arc_map->Data();
  }
  bool ans = c.GetOutput(&host_dest_fsa, arc_map_data);
  *dest = creator.GetFsa();
  return ans;
}

void Intersect(FsaOrVec &a_fsas, FsaOrVec &b_fsas,
               FsaVec *out,
               Array1<int32_t> *arc_map_a,
               Array1<int32_t> *arc_map_b) {
  K2_CHECK(a_fsas.NumAxes() >= 2 && a_fsas.NumAxes() <= 3);
  K2_CHECK(b_fsas.NumAxes() >= 2 && b_fsas.NumAxes() <= 3);
  ContextPtr c = a_fsas.Context();
  K2_CHECK_EQ(c->GetDeviceType(), kCpu);
  if (a_fsas.NumAxes() == 2) {
    FsaVec a_fsas_vec = FsaToFsaVec(a_fsas);
    Intersect(a_fsas_vec, b_fsas, out, arc_map_a, arc_map_b);
    return;
  }
  if (b_fsas.NumAxes() == 2) {
    FsaVec b_fsas_vec = FsaToFsaVec(b_fsas);
    Intersect(a_fsas, b_fsas_vec, out, arc_map_a, arc_map_b);
    return;
  }

  int32_t num_fsas_a = a_fsas.Dim0(),
    num_fsas_b = b_fsas.Dim0();
  K2_CHECK_GT(num_fsas_a, 0);
  K2_CHECK_GT(num_fsas_b, 0);
  int32_t stride_a = 1, stride_b = 1;
  if (num_fsas_a != num_fsas_b) {
    if (num_fsas_a == 1) { stride_a = 0; }
    else if (num_fsas_b == 1) { stride_b = 0; }
    else { K2_CHECK_EQ(num_fsas_a, num_fsas_b); }
    // the check on the previous line will fail.
  }
  int32_t num_fsas = std::max(num_fsas_a, num_fsas_b);

  std::vector<std::unique_ptr<k2host::Intersection>> intersections(num_fsas);
  std::vector<k2host::Array2Size<int32_t>> sizes(num_fsas);
  for (int32_t i = 0; i < num_fsas; i++) {
    k2host::Fsa host_fsa_a = FsaVecToHostFsa(a_fsas, i * stride_a),
      host_fsa_b = FsaVecToHostFsa(b_fsas, i * stride_b);
    intersections[i] = std::make_unique<k2host::Intersection>(host_fsa_a,
                                                              host_fsa_b);
    intersections[i]->GetSizes(&(sizes[i]));
  }
  FsaVecCreator creator(sizes);
  int32_t num_arcs = creator.NumArcs();

  if (arc_map_a) *arc_map_a = Array1<int32_t>(c, num_arcs);
  if (arc_map_b) *arc_map_b = Array1<int32_t>(c, num_arcs);


  // the following few lines will allow us to add suitable offsets to the
  // `arc_map`.
  Array1<int32_t> a_fsas_row_splits12 = a_fsas.RowSplits(2)[a_fsas.RowSplits(1)],
    b_fsas_row_splits12 = b_fsas.RowSplits(2)[b_fsas.RowSplits(1)];
  const int32_t *a_fsas_row_splits12_data = a_fsas_row_splits12.Data(),
    *b_fsas_row_splits12_data = b_fsas_row_splits12.Data();

  for (int32_t i = 0; i < num_fsas; i++) {
    k2host::Fsa host_fsa_out = creator.GetHostFsa(i);
    int32_t arc_offset = creator.GetArcOffsetFor(i);
    int32_t *this_arc_map_a = (arc_map_a ? arc_map_a->Data() + arc_offset :
                               nullptr),
      *this_arc_map_b = (arc_map_b ? arc_map_b->Data() + arc_offset :
                         nullptr);
    bool ans = intersections[i]->GetOutput(&host_fsa_out,
                                           this_arc_map_a, this_arc_map_b);
    int32_t this_num_arcs = creator.GetArcOffsetFor(i+1) - arc_offset;
    if (arc_map_a) {
      int32_t arc_offset_a = a_fsas_row_splits12_data[i * stride_a];
      for (int32_t i = 0; i < this_num_arcs; i++)
        this_arc_map_a[i] += arc_offset_a;
    }
    if (arc_map_b) {
      int32_t arc_offset_b = b_fsas_row_splits12_data[i * stride_b];
      for (int32_t i = 0; i < this_num_arcs; i++)
        this_arc_map_b[i] += arc_offset_b;
    }
  }
  *out = creator.GetFsaVec();
}

namespace {
struct ArcComparer {
  __host__ __device__ __forceinline__ bool operator()(const Arc &lhs,
                                                      const Arc &rhs) const {
    return lhs.symbol < rhs.symbol;
  }
};
}  // namespace

void ArcSort(Fsa *fsa) {
  if (fsa->NumAxes() < 2) return;  // it is empty
  SortSublists<Arc, ArcComparer>(fsa);
}

}  // namespace k2
