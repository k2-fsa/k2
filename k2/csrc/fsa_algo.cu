#include "hip/hip_runtime.h"
/**
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey, Haowen Qiu,
 *                                                   Wei Kang)
 *                      Mobvoi Inc.        (authors: Fangjun Kuang)
 *
 * See LICENSE for clarification regarding multiple authors
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <limits>
#include <memory>
#include <type_traits>
#include <utility>
#include <vector>

#include "k2/csrc/array_ops.h"
#include "k2/csrc/fsa_algo.h"
#include "k2/csrc/fsa_utils.h"
#include "k2/csrc/host/aux_labels.h"
#include "k2/csrc/host/connect.h"
#include "k2/csrc/host/determinize.h"
#include "k2/csrc/host/intersect.h"
#include "k2/csrc/host/rmepsilon.h"
#include "k2/csrc/host/topsort.h"
#include "k2/csrc/host_shim.h"
#include "k2/csrc/macros.h"
#include "k2/csrc/rm_epsilon.h"


// this contains a subset of the algorithms in fsa_algo.h; currently it just
// contains one that are wrappings of the corresponding algorithms in
// host/.
namespace k2 {

bool RecursionWrapper(bool (*f)(Fsa &, Fsa *, Array1<int32_t> *), Fsa &src,
                      Fsa *dest, Array1<int32_t> *arc_map) {
  NVTX_RANGE(K2_FUNC);
  // src is actually an FsaVec.  Just recurse for now.
  int32_t num_fsas = src.shape.Dim0();
  std::vector<Fsa> srcs(num_fsas), dests(num_fsas);
  std::vector<Array1<int32_t>> arc_maps(num_fsas);
  int32_t tot_num_arcs = 0;
  for (int32_t i = 0; i < num_fsas; ++i) {
    srcs[i] = src.Index(0, i);
    // Recurse.
    if (!f(srcs[i], &(dests[i]),
           (arc_map != nullptr ? &(arc_maps[i]) : nullptr)))
      return false;
    if (arc_map != nullptr) {
      // convert arc indexes in arc_maps from idx2 to idx012
      arc_maps[i] = Plus(arc_maps[i], tot_num_arcs);
      tot_num_arcs += srcs[i].NumElements();
    }
  }
  *dest = Stack(0, num_fsas, dests.data());
  if (arc_map != nullptr)
    *arc_map = Cat(src.Context(), num_fsas, arc_maps.data());
  return true;
}

bool ConnectHost(Fsa &src, Fsa *dest, Array1<int32_t> *arc_map /*=nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  int32_t num_axes = src.NumAxes();
  if (num_axes < 2 || num_axes > 3) {
    K2_LOG(FATAL) << "Input has bad num-axes " << num_axes;
  } else if (num_axes == 3) {
    return RecursionWrapper(ConnectHost, src, dest, arc_map);
  }

  k2host::Fsa host_fsa = FsaToHostFsa(src);
  k2host::Connection c(host_fsa);
  k2host::Array2Size<int32_t> size;
  c.GetSizes(&size);
  FsaCreator creator(size);
  k2host::Fsa host_dest_fsa = creator.GetHostFsa();
  int32_t *arc_map_data = nullptr;
  if (arc_map != nullptr) {
    *arc_map = Array1<int32_t>(src.Context(), size.size2);
    arc_map_data = arc_map->Data();
  }
  bool ans = c.GetOutput(&host_dest_fsa, arc_map_data);
  *dest = creator.GetFsa();
  return ans;
}

bool TopSortHost(Fsa &src, Fsa *dest, Array1<int32_t> *arc_map /*=nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  int32_t num_axes = src.NumAxes();
  if (num_axes < 2 || num_axes > 3) {
    K2_LOG(FATAL) << "Input has bad num-axes " << num_axes;
  } else if (num_axes == 3) {
    return RecursionWrapper(TopSortHost, src, dest, arc_map);
  }

  k2host::Fsa host_fsa = FsaToHostFsa(src);
  k2host::TopSorter sorter(host_fsa);
  k2host::Array2Size<int32_t> size;
  sorter.GetSizes(&size);
  FsaCreator creator(size);
  k2host::Fsa host_dest_fsa = creator.GetHostFsa();
  int32_t *arc_map_data = nullptr;
  if (arc_map != nullptr) {
    *arc_map = Array1<int32_t>(src.Context(), size.size2);
    arc_map_data = arc_map->Data();
  }
  bool ans = sorter.GetOutput(&host_dest_fsa, arc_map_data);
  *dest = creator.GetFsa();
  return ans;
}

bool Intersect(FsaOrVec &a_fsas, int32_t properties_a, FsaOrVec &b_fsas,
               int32_t properties_b, bool treat_epsilons_specially, FsaVec *out,
               Array1<int32_t> *arc_map_a, Array1<int32_t> *arc_map_b) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK(a_fsas.NumAxes() >= 2 && a_fsas.NumAxes() <= 3);
  K2_CHECK(b_fsas.NumAxes() >= 2 && b_fsas.NumAxes() <= 3);
  ContextPtr c = a_fsas.Context();
  K2_CHECK_EQ(c->GetDeviceType(), kCpu);
  if (a_fsas.NumAxes() == 2) {
    FsaVec a_fsas_vec = FsaToFsaVec(a_fsas);
    return Intersect(a_fsas_vec, properties_a, b_fsas, properties_b,
                     treat_epsilons_specially, out, arc_map_a, arc_map_b);
  }
  if (b_fsas.NumAxes() == 2) {
    FsaVec b_fsas_vec = FsaToFsaVec(b_fsas);
    return Intersect(a_fsas, properties_a, b_fsas_vec, properties_b,
                     treat_epsilons_specially, out, arc_map_a, arc_map_b);
  }

  int32_t num_fsas_a = a_fsas.Dim0(), num_fsas_b = b_fsas.Dim0();
  K2_CHECK_GT(num_fsas_a, 0);
  K2_CHECK_GT(num_fsas_b, 0);
  int32_t stride_a = 1, stride_b = 1;
  if (num_fsas_a != num_fsas_b) {
    if (num_fsas_a == 1) {
      stride_a = 0;
    } else if (num_fsas_b == 1) {
      stride_b = 0;
    } else {
      K2_CHECK_EQ(num_fsas_a, num_fsas_b);
    }
    // the check on the previous line will fail.
  }
  if (properties_a < 0) {
    Array1<int32_t> properties_a_out(c, num_fsas_a);
    GetFsaVecBasicProperties(a_fsas, &properties_a_out, &properties_a);
  }
  if (properties_b < 0) {
    Array1<int32_t> properties_b_out(c, num_fsas_b);
    GetFsaVecBasicProperties(b_fsas, &properties_b_out, &properties_b);
  }
  bool arc_sorted = (properties_a & kFsaPropertiesArcSorted) &&
                    (properties_b & kFsaPropertiesArcSorted);
  K2_CHECK(arc_sorted) << "Both a_fsas and b_fsas should be arc-sorted";
  int32_t num_fsas = std::max(num_fsas_a, num_fsas_b);

  std::vector<std::unique_ptr<k2host::Intersection>> intersections(num_fsas);
  std::vector<k2host::Array2Size<int32_t>> sizes(num_fsas);
  for (int32_t i = 0; i < num_fsas; ++i) {
    k2host::Fsa host_fsa_a = FsaVecToHostFsa(a_fsas, i * stride_a),
                host_fsa_b = FsaVecToHostFsa(b_fsas, i * stride_b);
    intersections[i] = std::make_unique<k2host::Intersection>(
        host_fsa_a, host_fsa_b, treat_epsilons_specially, false);
    intersections[i]->GetSizes(&(sizes[i]));
  }
  FsaVecCreator creator(sizes);
  int32_t num_arcs = creator.NumArcs();

  if (arc_map_a) *arc_map_a = Array1<int32_t>(c, num_arcs);
  if (arc_map_b) *arc_map_b = Array1<int32_t>(c, num_arcs);

  // the following few lines will allow us to add suitable offsets to the
  // `arc_map`.
  Array1<int32_t> a_fsas_row_splits12 =
                      a_fsas.RowSplits(2)[a_fsas.RowSplits(1)],
                  b_fsas_row_splits12 =
                      b_fsas.RowSplits(2)[b_fsas.RowSplits(1)];
  const int32_t *a_fsas_row_splits12_data = a_fsas_row_splits12.Data(),
                *b_fsas_row_splits12_data = b_fsas_row_splits12.Data();

  bool ok = true;
  for (int32_t i = 0; i < num_fsas; ++i) {
    k2host::Fsa host_fsa_out = creator.GetHostFsa(i);
    int32_t arc_offset = creator.GetArcOffsetFor(i);
    int32_t *this_arc_map_a =
                (arc_map_a ? arc_map_a->Data() + arc_offset : nullptr),
            *this_arc_map_b =
                (arc_map_b ? arc_map_b->Data() + arc_offset : nullptr);
    bool ans = intersections[i]->GetOutput(&host_fsa_out, this_arc_map_a,
                                           this_arc_map_b);
    ok = ok && ans;
    int32_t this_num_arcs = creator.GetArcOffsetFor(i + 1) - arc_offset;
    if (arc_map_a) {
      int32_t arc_offset_a = a_fsas_row_splits12_data[i * stride_a];
      for (int32_t i = 0; i < this_num_arcs; i++)
        if (this_arc_map_a[i] != -1) this_arc_map_a[i] += arc_offset_a;
    }
    if (arc_map_b) {
      int32_t arc_offset_b = b_fsas_row_splits12_data[i * stride_b];
      for (int32_t i = 0; i < this_num_arcs; i++)
        if (this_arc_map_b[i] != -1) this_arc_map_b[i] += arc_offset_b;
    }
  }
  *out = creator.GetFsaVec();
  return ok;
}

// Will be used in RemoveEpsilonHost and Determinize below to process FsaVec
// input recursively.
void RecursionWrapper(void (*f)(FsaOrVec &, FsaOrVec *, Ragged<int32_t> *),
                      FsaOrVec &src, FsaOrVec *dest,
                      Ragged<int32_t> *arc_deriv) {
  NVTX_RANGE(K2_FUNC);
  // src is actually an FsaVec.  Just recurse for now.
  K2_CHECK_EQ(src.NumAxes(), 3);
  int32_t num_fsas = src.shape.Dim0();
  std::vector<Fsa> srcs(num_fsas), dests(num_fsas);
  std::vector<Ragged<int32_t>> arc_derivs(num_fsas);
  int32_t tot_num_arcs = 0;
  for (int32_t i = 0; i < num_fsas; ++i) {
    srcs[i] = src.Index(0, i);
    f(srcs[i], &(dests[i]), arc_deriv != nullptr ? &(arc_derivs[i]) : nullptr);
    if (arc_deriv != nullptr) {
      // convert arc indexes in arc_derivs from idx2 to idx012
      Array1<int32_t> &values = arc_derivs[i].values;
      values = Plus(values, tot_num_arcs);
      tot_num_arcs += srcs[i].NumElements();
    }
  }
  *dest = Stack(0, num_fsas, dests.data());
  if (arc_deriv != nullptr) *arc_deriv = Cat(0, num_fsas, arc_derivs.data());
}

void RemoveEpsilonHost(FsaOrVec &src, FsaOrVec *dest,
                       Ragged<int32_t> *arc_derivs /*=nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  int32_t num_axes = src.NumAxes();
  if (num_axes < 2 || num_axes > 3) {
    K2_LOG(FATAL) << "Input has bad num-axes " << num_axes;
  } else if (num_axes == 3) {
    return RecursionWrapper(RemoveEpsilonHost, src, dest, arc_derivs);
  }
  k2host::Fsa host_fsa = FsaToHostFsa(src);
  int32_t num_states = host_fsa.NumStates();
  K2_CHECK_EQ(num_states, src.Dim0());
  std::vector<double> max_forward_weights(num_states);
  std::vector<double> max_backward_weights(num_states);
  k2host::WfsaWithFbWeights max_wfsa(host_fsa, k2host::kMaxWeight,
                                     max_forward_weights.data(),
                                     max_backward_weights.data());
  // pass infinity as beam since we don't do pruning here.
  float beam = std::numeric_limits<float>::infinity();
  k2host::EpsilonsRemoverPrunedMax eps_remover(max_wfsa, beam);
  k2host::Array2Size<int32_t> fsa_size, arc_derivs_size;
  eps_remover.GetSizes(&fsa_size, &arc_derivs_size);
  FsaCreator fsa_creator(fsa_size);
  k2host::Fsa host_dest_fsa = fsa_creator.GetHostFsa();
  K2_STATIC_ASSERT(
      (std::is_same<k2host::MaxTracebackState::DerivType, int32_t>::value));
  Ragged2Creator<int32_t> ragged_creator(arc_derivs_size);
  k2host::Array2<int32_t *, int32_t> host_arc_derivs =
      ragged_creator.GetHostArray2();
  eps_remover.GetOutput(&host_dest_fsa, &host_arc_derivs);
  *dest = fsa_creator.GetFsa();
  if (arc_derivs != nullptr) *arc_derivs = ragged_creator.GetRagged2();
}


void RemoveEpsilon(FsaOrVec &src, int32_t properties,
                   FsaOrVec *dest,
                   Ragged<int32_t> *arc_derivs) {
  if ((properties & kFsaPropertiesTopSortedAndAcyclic) != 0 &&
      src.Context()->GetDeviceType() == kCpu) {
    // Host version of the algorithm
    RemoveEpsilonHost(src, dest, arc_derivs);
  } else {
    RemoveEpsilonDevice(src, dest, arc_derivs);
  }
}


void RemoveEpsilonAndAddSelfLoops(FsaOrVec &src, int32_t properties,
                                  FsaOrVec *dest,
                                  Ragged<int32_t> *arc_derivs) {
  NVTX_RANGE(K2_FUNC);
  Ragged<int32_t> arc_derivs1;

  FsaOrVec temp;
  RemoveEpsilon(src, properties, &temp,
                (arc_derivs != nullptr ? &arc_derivs1 : nullptr));

  Array1<int32_t> arc_derivs2;
  AddEpsilonSelfLoops(temp, dest,
                      (arc_derivs != nullptr ? &arc_derivs2 : nullptr));

  if (arc_derivs != nullptr) {
    *arc_derivs = Index(arc_derivs1, 0, arc_derivs2, nullptr);
  }
}




void Determinize(FsaOrVec &src, FsaOrVec *dest,
                 Ragged<int32_t> *arc_derivs /*=nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  int32_t num_axes = src.NumAxes();
  if (num_axes < 2 || num_axes > 3) {
    K2_LOG(FATAL) << "Input has bad num-axes " << num_axes;
  } else if (num_axes == 3) {
    return RecursionWrapper(Determinize, src, dest, arc_derivs);
  }
  k2host::Fsa host_fsa = FsaToHostFsa(src);
  int32_t num_states = host_fsa.NumStates();
  K2_CHECK_EQ(num_states, src.Dim0());
  int32_t max_step = -1;  // no limit
  k2host::DeterminizerMax determinizer(host_fsa, max_step);
  k2host::Array2Size<int32_t> fsa_size, arc_derivs_size;
  determinizer.GetSizes(&fsa_size, &arc_derivs_size);
  FsaCreator fsa_creator(fsa_size);
  k2host::Fsa host_dest_fsa = fsa_creator.GetHostFsa();
  K2_STATIC_ASSERT(
      (std::is_same<k2host::MaxTracebackState::DerivType, int32_t>::value));
  Ragged2Creator<int32_t> ragged_creator(arc_derivs_size);
  k2host::Array2<int32_t *, int32_t> host_arc_derivs =
      ragged_creator.GetHostArray2();
  determinizer.GetOutput(&host_dest_fsa, &host_arc_derivs);
  *dest = fsa_creator.GetFsa();
  if (arc_derivs != nullptr) *arc_derivs = ragged_creator.GetRagged2();
}

Fsa LinearFsa(const Array1<int32_t> &symbols) {
  NVTX_RANGE(K2_FUNC);
  ContextPtr &c = symbols.Context();
  int32_t n = symbols.Dim(), num_states = n + 2, num_arcs = n + 1;
  Array1<int32_t> row_splits1 = Range(c, num_states + 1, 0),
                  row_ids1 = Range(c, num_arcs, 0);
  int32_t *row_splits1_data = row_splits1.Data();
  Array1<Arc> arcs(c, num_arcs);
  Arc *arcs_data = arcs.Data();
  const int32_t *symbols_data = symbols.Data();
  K2_EVAL(
      c, num_arcs, lambda_set_arcs, (int32_t arc_idx01)->void {
        int32_t src_state = arc_idx01, dest_state = arc_idx01 + 1,
                // -1 == kFinalSymbol
            symbol = (arc_idx01 < n ? symbols_data[arc_idx01] : -1);
        if (arc_idx01 < n) K2_CHECK_NE(symbol, -1);
        float score = 0.0;
        arcs_data[arc_idx01] = Arc(src_state, dest_state, symbol, score);
        // the final state has no leaving arcs.
        if (arc_idx01 == 0) row_splits1_data[num_states] = num_arcs;
      });
  return Ragged<Arc>(RaggedShape2(&row_splits1, &row_ids1, num_arcs), arcs);
}

FsaVec LinearFsas(const Ragged<int32_t> &symbols) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(symbols.NumAxes(), 2);
  ContextPtr &c = symbols.Context();

  // if there are n symbols, there are n+2 states and n+1 arcs.
  RaggedShape states_shape = ChangeSublistSize(symbols.shape, 2);

  int32_t num_states = states_shape.NumElements(),
          num_arcs = symbols.NumElements() + symbols.Dim0();

  // row_splits2 maps from state_idx01 to arc_idx012; row_ids2 does the reverse.
  // We'll set them in the lambda below.
  Array1<int32_t> row_splits2(c, num_states + 1), row_ids2(c, num_arcs);

  int32_t *row_ids2_data = row_ids2.Data(),
          *row_splits2_data = row_splits2.Data();
  const int32_t *row_ids1_data = states_shape.RowIds(1).Data(),
                *row_splits1_data = states_shape.RowSplits(1).Data(),
                *symbols_data = symbols.values.Data();
  Array1<Arc> arcs(c, num_arcs);
  Arc *arcs_data = arcs.Data();
  K2_EVAL(
      c, num_states, lambda, (int32_t state_idx01)->void {
        int32_t fsa_idx0 = row_ids1_data[state_idx01],
                state_idx0x = row_splits1_data[fsa_idx0],
                next_state_idx0x = row_splits1_data[fsa_idx0 + 1],
                idx1 = state_idx01 - state_idx0x;

        // the following works because each FSA has one fewer arcs than states.
        int32_t arc_idx0xx = state_idx0x - fsa_idx0,
                next_arc_idx0xx = next_state_idx0x - (fsa_idx0 + 1),
                // the following may look a bit wrong.. here, the idx1 is the
                // same as the idx12 if the arc exists, because each state has
                // one arc leaving it (except the last state).
            arc_idx012 = arc_idx0xx + idx1;
        // the following works because each FSA has one fewer symbols than arcs
        // (however it doesn't work for the last arc of each FSA; we check
        // below.)
        int32_t symbol_idx01 = arc_idx012 - fsa_idx0;
        if (arc_idx012 < next_arc_idx0xx) {
          int32_t src_state = idx1, dest_state = idx1 + 1,
                  symbol = (arc_idx012 + 1 < next_arc_idx0xx
                                ? symbols_data[symbol_idx01]
                                : -1);  // kFinalSymbol
          float score = 0.0;
          arcs_data[arc_idx012] = Arc(src_state, dest_state, symbol, score);
          row_ids2_data[arc_idx012] = state_idx01;
        } else {
          // The following ensures that the last element of row_splits1_data
          // (i.e. row_splits1[num_states]) is set to num_arcs.  It also writes
          // something unnecessary for the last state of each FSA but the last
          // one, which will cause 2 threads to write the same item to the same
          // location. Note that there is no arc with index `arc_idx01`, if you
          // reach here.
          row_splits2_data[state_idx01 + 1] = arc_idx012;
        }
        row_splits2_data[state_idx01] = arc_idx012;
      });

  return Ragged<Arc>(
      RaggedShape3(&states_shape.RowSplits(1), &states_shape.RowIds(1),
                   num_states, &row_splits2, &row_ids2, num_arcs),
      arcs);
}


FsaVec CtcGraphs(const Ragged<int32_t> &symbols, bool standard /*= true*/,
                 Array1<int32_t> *arc_map /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(symbols.NumAxes(), 2);
  ContextPtr &c = symbols.Context();

  int32_t num_fsas = symbols.Dim0();
  Array1<int32_t> num_states_for(c, num_fsas + 1);
  int32_t *num_states_for_data = num_states_for.Data();
  const int32_t *symbol_row_split1_data = symbols.RowSplits(1).Data();
  // symbols indexed with [fsa][symbol]
  // for each fsa we need `symbol_num * 2 + 1 + 1` states, `symbol_num * 2 + 1`
  // means that we need a blank state on each side of a symbol state, `+ 1` is
  // for final state in k2
  K2_EVAL(
      c, num_fsas, lambda_set_num_states, (int32_t fsa_idx0)->void {
        int32_t symbol_idx0x = symbol_row_split1_data[fsa_idx0],
                symbol_idx0x_next = symbol_row_split1_data[fsa_idx0 + 1],
                symbol_num = symbol_idx0x_next - symbol_idx0x;
        num_states_for_data[fsa_idx0] = symbol_num * 2 + 2;
      });

  ExclusiveSum(num_states_for, &num_states_for);
  Array1<int32_t> &fsa_to_states_row_splits = num_states_for;
  RaggedShape fsa_to_states =
      RaggedShape2(&fsa_to_states_row_splits, nullptr, -1);

  int32_t num_states = fsa_to_states.NumElements();
  Array1<int32_t> num_arcs_for(c, num_states + 1);
  int32_t *num_arcs_for_data = num_arcs_for.Data();
  const int32_t *fts_row_splits1_data = fsa_to_states.RowSplits(1).Data(),
                *fts_row_ids1_data = fsa_to_states.RowIds(1).Data(),
                *symbol_data = symbols.values.Data();
  // set the arcs number for each state
  K2_EVAL(
      c, num_states, lambda_set_num_arcs, (int32_t state_idx01)->void {
        int32_t fsa_idx0 = fts_row_ids1_data[state_idx01],
                // we minus fsa_idx0 here, because we are adding one more state,
                // the final state for each fsa
                sym_state_idx01 = state_idx01 / 2 - fsa_idx0,
                remainder = state_idx01 % 2,
                current_num_arcs = 2;  // normally there are two arcs, self-loop
                                       // and arc pointing to the next state
                                       // blank state always has two arcs
        if (remainder) {  // symbol state
          int32_t sym_final_state =
                    symbol_row_split1_data[fsa_idx0 + 1];
          // There are no arcs for final states
          if (sym_state_idx01 == sym_final_state) {
            current_num_arcs = 0;
          } else if (!standard) {
            current_num_arcs = 3;
          } else {
            int32_t current_symbol = symbol_data[sym_state_idx01],
                    // we set the next symbol of the last symbol to -1, so
                    // the following if clause will always be true, which means
                    // we will have 3 arcs for last symbol state
                    next_symbol = (sym_state_idx01 + 1) == sym_final_state ?
                                  -1 : symbol_data[sym_state_idx01 + 1];
            // symbols must be not equal to -1, which is specially used in k2
            K2_CHECK_NE(current_symbol, -1);
            // if current_symbol equals next_symbol, we need a blank state
            // between them, so there are two arcs for this state
            // otherwise, this state will point to blank state and next symbol
            // state, so we need three arcs here.
            // Note: for the simpilfied topology (standard equals false), there
            // are always 3 arcs leaving symbol states.
            if (current_symbol != next_symbol)
              current_num_arcs = 3;
          }
        }
        num_arcs_for_data[state_idx01] = current_num_arcs;
      });

  ExclusiveSum(num_arcs_for, &num_arcs_for);
  Array1<int32_t> &states_to_arcs_row_splits = num_arcs_for;
  RaggedShape states_to_arcs =
      RaggedShape2(&states_to_arcs_row_splits, nullptr, -1);

  // ctc_shape with a index of [fsa][state][arc]
  RaggedShape ctc_shape = ComposeRaggedShapes(fsa_to_states, states_to_arcs);
  int32_t num_arcs = ctc_shape.NumElements();
  Array1<Arc> arcs(c, num_arcs);
  Arc *arcs_data = arcs.Data();
  const int32_t *ctc_row_splits1_data = ctc_shape.RowSplits(1).Data(),
                *ctc_row_ids1_data = ctc_shape.RowIds(1).Data(),
                *ctc_row_splits2_data = ctc_shape.RowSplits(2).Data(),
                *ctc_row_ids2_data = ctc_shape.RowIds(2).Data();
  int32_t *arc_map_data = nullptr;
  if (arc_map != nullptr) {
    *arc_map = Array1<int32_t>(c, num_arcs);
    arc_map_data = arc_map->Data();
  }

  K2_EVAL(
      c, num_arcs, lambda_set_arcs, (int32_t arc_idx012)->void {
        int32_t state_idx01 = ctc_row_ids2_data[arc_idx012],
                fsa_idx0 = ctc_row_ids1_data[state_idx01],
                state_idx0x = ctc_row_splits1_data[fsa_idx0],
                state_idx1 = state_idx01 - state_idx0x,
                arc_idx01x = ctc_row_splits2_data[state_idx01],
                arc_idx2 = arc_idx012 - arc_idx01x,
                sym_state_idx01 = state_idx01 / 2 - fsa_idx0,
                remainder = state_idx01 % 2,
                sym_final_state = symbol_row_split1_data[fsa_idx0 + 1];
        bool final_state = sym_final_state == sym_state_idx01;
        int32_t current_symbol = final_state ?
            -1 : symbol_data[sym_state_idx01];
        Arc arc;
        arc.score = 0;
        arc.src_state = state_idx1;
        int32_t arc_map_value = -1;
        if (remainder) {
          if (final_state) return;
          int32_t next_symbol = (sym_state_idx01 + 1) == sym_final_state ?
              -1 : symbol_data[sym_state_idx01 + 1];
          // for standard topology, the symbol state can not point to next
          // symbol state if the next symbol is identical to current symbol.
          if (current_symbol == next_symbol && standard) {
            K2_CHECK_LT(arc_idx2, 2);
            arc.label = arc_idx2 == 0 ? 0 : current_symbol;
            arc.dest_state = arc_idx2 == 0 ? state_idx1 + 1 : state_idx1;
          } else {
            switch (arc_idx2) {
              case 0:   // the arc pointing to blank state
                arc.label = 0;
                arc.dest_state = state_idx1 + 1;
                break;
              case 1:   // the self loop arc
                arc.label = current_symbol;
                arc.dest_state = state_idx1;
                break;
              case 2:  // the arc pointing to the next symbol state
                arc.label = next_symbol;
                arc_map_value = sym_state_idx01 + 1 == sym_final_state ?
                    -1 : sym_state_idx01 + 1;
                arc.dest_state = state_idx1 + 2;
                break;
              default:
                K2_LOG(FATAL) << "Arc index must be less than 3";
            }
          }
        } else {
          K2_CHECK_LT(arc_idx2, 2);
          arc.label = arc_idx2 == 0 ? 0 : current_symbol;
          arc.dest_state = arc_idx2 == 0 ? state_idx1 : state_idx1 + 1;
          arc_map_value = (arc_idx2 == 0 || final_state) ? -1 : sym_state_idx01;
        }
        arcs_data[arc_idx012] = arc;
        if (arc_map) arc_map_data[arc_idx012] = arc_map_value;
      });
  return Ragged<Arc>(ctc_shape, arcs);
}

void ArcSort(Fsa *fsa) {
  if (fsa->NumAxes() < 2) return;  // it is empty
  SortSublists<Arc>(fsa);
}

void ArcSort(Fsa &src, Fsa *dest, Array1<int32_t> *arc_map /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  if (!src.values.IsValid()) return;

  if (arc_map != nullptr)
    *arc_map = Array1<int32_t>(src.Context(), src.NumElements());

  Fsa tmp(src.shape, src.values.Clone());
  SortSublists<Arc>(&tmp, arc_map);
  *dest = tmp;
}

// TODO(fangjun): use the following method suggested by Dan
//
// ... incidentally, it's possible to further optimize this so the run
// time is less than linear, by using methods similar to what I use
// in GetStateBatches(); imagine computing a table that instead of
// the best traceback, is the best 2-step traceback; and then the 4-step
// traceback, and so on. There's no need for this right now, since the
// forward-pass algorithm is already at least linear-time in the length
// of this path. But we can consider it for the future.
Ragged<int32_t> ShortestPath(FsaVec &fsas,
                             const Array1<int32_t> &entering_arcs) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  const int32_t *entering_arcs_data = entering_arcs.Data();
  const Arc *arcs_data = fsas.values.Data();
  int32_t num_fsas = fsas.Dim0();
  int32_t num_states = fsas.TotSize(1);
  ContextPtr &context = fsas.Context();

  // allocate an extra element for ExclusiveSum
  Array1<int32_t> num_best_arcs_per_fsa(context, num_fsas + 1);
  int32_t *num_best_arcs_per_fsa_data = num_best_arcs_per_fsa.Data();
  const int32_t *row_splits1_data = fsas.RowSplits(1).Data();

  // -1 represents an invalid arc_index.
  // This extra array avoids an extra iteration over `entering_arcs`.
  Array1<int32_t> state_best_arc_index_array(context, num_states, -1);
  int32_t *state_best_arc_index_array_data = state_best_arc_index_array.Data();

  K2_EVAL(
      context, num_fsas, lambda_set_num_best_arcs, (int32_t fsas_idx0) {
        int32_t state_idx01 = row_splits1_data[fsas_idx0];
        int32_t state_idx01_next = row_splits1_data[fsas_idx0 + 1];

        if (state_idx01_next == state_idx01) {
          // this fsa is empty, so there is no best path available
          num_best_arcs_per_fsa_data[fsas_idx0] = 0;
          return;
        }

        int32_t final_state_idx01 = state_idx01_next - 1;
        int32_t cur_state = final_state_idx01;
        int32_t cur_index = entering_arcs_data[cur_state];
        int32_t num_arcs = 0;
        int32_t *p = state_best_arc_index_array_data + final_state_idx01;
        while (cur_index != -1) {
          *p = cur_index;
          --p;

          cur_state = arcs_data[cur_index].src_state + state_idx01;
          cur_index = entering_arcs_data[cur_state];
          ++num_arcs;
        }
        num_best_arcs_per_fsa_data[fsas_idx0] = num_arcs;
      });
  ExclusiveSum(num_best_arcs_per_fsa, &num_best_arcs_per_fsa);

  RaggedShape shape = RaggedShape2(&num_best_arcs_per_fsa, nullptr, -1);
  const int32_t *shape_row_splits1_data = shape.RowSplits(1).Data();
  const int32_t *shape_row_ids1_data = shape.RowIds(1).Data();

  const int32_t *ans_row_splits_data = shape.RowSplits(1).Data();
  Array1<int32_t> best_path_arc_indexes(context, shape.NumElements());
  int32_t *best_path_arc_indexes_data = best_path_arc_indexes.Data();

  K2_EVAL(
      context, shape.NumElements(), lambda_set_best_arcs, (int32_t ans_idx01) {
        int32_t fsa_idx0 = shape_row_ids1_data[ans_idx01];
        int32_t ans_idx0x = shape_row_splits1_data[fsa_idx0];
        int32_t ans_idx1 = ans_idx01 - ans_idx0x;

        int32_t num_arcs_this_fsa = num_best_arcs_per_fsa_data[fsa_idx0 + 1] -
                                    num_best_arcs_per_fsa_data[fsa_idx0];
        if (num_arcs_this_fsa == 0) return;

        int32_t final_state_idx01_this_fsa = row_splits1_data[fsa_idx0 + 1] - 1;

        const int32_t *p_start = state_best_arc_index_array_data +
                                 final_state_idx01_this_fsa -
                                 num_arcs_this_fsa + 1;

        best_path_arc_indexes_data[ans_idx01] = p_start[ans_idx1];
      });

  Ragged<int32_t> ans(shape, best_path_arc_indexes);
  return ans;
}

void AddEpsilonSelfLoops(FsaOrVec &src, FsaOrVec *dest,
                         Array1<int32_t> *arc_map /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  ContextPtr &c = src.Context();
  const int32_t *old_row_splits1_data = src.RowSplits(1).Data(),
                *old_row_ids1_data = src.RowIds(1).Data();
  const Arc *old_arcs_data = src.values.Data();
  if (src.NumAxes() == 2) {
    int32_t num_states = src.Dim0();
    if (num_states < 2) {
      K2_CHECK_EQ(num_states, 0);
      *dest = src;
      if (arc_map != nullptr) *arc_map = Array1<int32_t>(c, 0);
      return;
    }

    int32_t old_num_arcs = src.TotSize(1),
            new_num_arcs = old_num_arcs + (num_states - 1);
    Array1<int32_t> new_row_splits(c, num_states + 1),
        new_row_ids(c, new_num_arcs);
    Array1<Arc> new_arcs(c, new_num_arcs);
    int32_t *new_row_splits1_data = new_row_splits.Data(),
            *new_row_ids1_data = new_row_ids.Data();
    Arc *new_arcs_data = new_arcs.Data();
    int32_t *arc_map_data = nullptr;
    if (arc_map) {
      *arc_map = Array1<int32_t>(c, new_num_arcs);
      arc_map_data = arc_map->Data();
    }
    ParallelRunner pr(c);
    {
      With w(pr.NewStream());
      K2_EVAL(
          c, old_num_arcs, lambda_copy_data, (int32_t arc_idx01)->void {
            int32_t state_idx0 = old_row_ids1_data[arc_idx01],
                    new_arc_idx01 = arc_idx01 + 1 + state_idx0;
            // the "+1" above is because we put the self-loop first.
            new_row_ids1_data[new_arc_idx01] = state_idx0;
            new_arcs_data[new_arc_idx01] = old_arcs_data[arc_idx01];
            if (arc_map_data) arc_map_data[new_arc_idx01] = arc_idx01;
          });
    }
    {
      With w(pr.NewStream());
      K2_EVAL(
          c, num_states, lambda_set_new_data, (int32_t state_idx0)->void {
            int32_t old_arc_idx0x = old_row_splits1_data[state_idx0],
                    new_arc_idx0x = old_arc_idx0x + state_idx0;
            new_row_splits1_data[state_idx0] = new_arc_idx0x;
            if (state_idx0 + 1 < num_states) {        // not final-state
              int32_t new_arc_idx01 = new_arc_idx0x;  // the 1st arc is the loop
              new_row_ids1_data[new_arc_idx01] = state_idx0;
              new_arcs_data[new_arc_idx01] =
                  Arc(state_idx0, state_idx0, 0, 0.0);
              if (arc_map_data) arc_map_data[new_arc_idx01] = -1;
            } else {
              // Note: if num_states was zero we would have returned above, so
              // we don't have to worry about empty FSAs.
              new_row_splits1_data[num_states] = new_arc_idx0x;
            }
          });
    }
    pr.Finish();
    *dest = Ragged<Arc>(
        RaggedShape2(&new_row_splits, &new_row_ids, new_num_arcs), new_arcs);
  } else {
    K2_CHECK_EQ(src.NumAxes(), 3);
    // Get a vector saying, for each FSA, whether it's nonempty.
    int32_t num_fsas = src.Dim0(), num_states = src.TotSize(1),
            old_num_arcs = src.TotSize(2);
    if (num_states == 0) {
      *dest = src;
      if (arc_map) *arc_map = Array1<int32_t>(c, 0);
      return;
    }
    Array1<int32_t> fsa_nonempty(c, num_fsas + 1);
    int32_t *fsa_nonempty_data = fsa_nonempty.Data();
    K2_EVAL(
        c, num_fsas, lambda_set_fsa_nonempty, (int32_t fsa_idx0)->void {
          fsa_nonempty_data[fsa_idx0] = (old_row_splits1_data[fsa_idx0 + 1] >
                                         old_row_splits1_data[fsa_idx0]);
        });
    ExclusiveSum(fsa_nonempty, &fsa_nonempty);
    const int32_t *old_row_splits2_data = src.RowSplits(2).Data(),
                  *old_row_ids2_data = src.RowIds(2).Data();
    int32_t num_nonempty_fsas = fsa_nonempty.Back(),
            new_num_arcs = old_num_arcs + num_states - num_nonempty_fsas;
    // we subtract `num_nonempty_fsas` because final-states don't get a
    // self-loop.

    Array1<int32_t> new_row_splits2(c, num_states + 1),
        new_row_ids2(c, new_num_arcs);
    Array1<Arc> new_arcs(c, new_num_arcs);
    // fsa_idx0_mod_data maps from fsa_idx0 to a modified fsa_idx0 that
    // "doesn't count" FSAs with zero states.
    const int32_t *fsa_idx0_mod_data = fsa_nonempty_data;
    int32_t *new_row_splits2_data = new_row_splits2.Data(),
            *new_row_ids2_data = new_row_ids2.Data();
    Arc *new_arcs_data = new_arcs.Data();
    int32_t *arc_map_data = nullptr;
    if (arc_map) {
      *arc_map = Array1<int32_t>(c, new_num_arcs);
      arc_map_data = arc_map->Data();
    }
    ParallelRunner pr(c);
    {
      With w(pr.NewStream());
      K2_EVAL(
          c, old_num_arcs, lambda_copy_data, (int32_t arc_idx012)->void {
            int32_t state_idx01 = old_row_ids2_data[arc_idx012],
                    fsa_idx0 = old_row_ids1_data[state_idx01],
                    fsa_idx0_mod = fsa_idx0_mod_data[fsa_idx0],
                    new_arc_idx012 =
                        arc_idx012 + 1 + state_idx01 - fsa_idx0_mod;
            // The "+1" above is because we put the self-loop first.  The
            // "-fsa_idx0_mod" is because final-states don't get a self-loop.
            new_row_ids2_data[new_arc_idx012] = state_idx01;
            new_arcs_data[new_arc_idx012] = old_arcs_data[arc_idx012];
            if (arc_map_data) arc_map_data[new_arc_idx012] = arc_idx012;
          });
    }
    {
      With w(pr.NewStream());
      K2_EVAL(
          c, num_states, lambda_set_new_data, (int32_t state_idx01)->void {
            int32_t fsa_idx0 = old_row_ids1_data[state_idx01],
                    fsa_idx0_mod = fsa_idx0_mod_data[fsa_idx0],
                    state_idx0x = old_row_splits1_data[fsa_idx0],
                    next_state_idx0x = old_row_splits1_data[fsa_idx0 + 1],
                    old_arc_idx01x = old_row_splits2_data[state_idx01];
            // Below the "+ state_idx01" is because each state gets a self-loop,
            // and the "- fsa_idx0_mod" is because final-states don't get a
            // self-loop.
            int32_t new_arc_idx01x =
                old_arc_idx01x + state_idx01 - fsa_idx0_mod;
            // The self-loop arc is the first arc:
            int32_t new_arc_idx012 = new_arc_idx01x;
            new_row_splits2_data[state_idx01] = new_arc_idx01x;
            if (state_idx01 + 1 < next_state_idx0x) {  // not final-state
              new_row_ids2_data[new_arc_idx012] = state_idx01;
              int32_t state_idx1 = state_idx01 - state_idx0x;
              new_arcs_data[new_arc_idx012] =
                  Arc(state_idx1, state_idx1, 0, 0.0);
              if (arc_map_data) arc_map_data[new_arc_idx012] = -1;
            } else if (state_idx01 + 1 == num_states) {
              // Note: if num_states was zero  we would have returned above, so
              // we dont have to worry about an empty FsaVec.
              new_row_splits2_data[num_states] = new_arc_idx01x;
            }
          });
    }
    pr.Finish();
    *dest =
        Ragged<Arc>(RaggedShape3(&src.RowSplits(1), &src.RowIds(1), num_states,
                                 &new_row_splits2, &new_row_ids2, new_num_arcs),
                    new_arcs);
  }
}

Fsa Union(FsaVec &fsas, Array1<int32_t> *arc_map /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(fsas.NumAxes(), 3);

  ContextPtr &context = fsas.Context();
  const int32_t *fsas_row_splits1_data = fsas.RowSplits(1).Data();
  const int32_t *fsas_row_splits2_data = fsas.RowSplits(2).Data();
  const int32_t *fsas_row_ids1_data = fsas.RowIds(1).Data();
  const int32_t *fsas_row_ids2_data = fsas.RowIds(2).Data();
  const Arc *arcs_data = fsas.values.Data();

  int32_t num_fsas = fsas.Dim0();
  int32_t num_states = fsas.TotSize(1);
  int32_t num_arcs = fsas.TotSize(2);

  // A new start state and a new final state are added (+2).
  // The final state of each fsa is removed (-num_fsas)
  int32_t num_out_states = num_states + 2 - num_fsas;
  int32_t out_final_state = num_out_states - 1;

  // For every fsa, a new arc is added from the new start state
  // to its original start state (+num_fsas)
  int32_t num_out_arcs = num_arcs + num_fsas;

  Array1<int32_t> out_row_ids(context, num_out_arcs);
  Array1<Arc> out_arcs(context, num_out_arcs);
  Array1<int32_t> tmp_arc_map(context, num_out_arcs, -1);
  int32_t *tmp_arc_map_data = tmp_arc_map.Data();

  int32_t *out_row_ids_data = out_row_ids.Data();
  Arc *out_arcs_data = out_arcs.Data();

  K2_EVAL(
      context, num_arcs, lambda_set_out, (int32_t fsas_arc_idx012) {
        int32_t fsas_state_idx01 = fsas_row_ids2_data[fsas_arc_idx012];
        int32_t fsas_idx0 = fsas_row_ids1_data[fsas_state_idx01];
        int32_t this_fsa_final_state_idx01 =
            fsas_row_splits1_data[fsas_idx0 + 1] - 1;

        K2_DCHECK_GT(this_fsa_final_state_idx01, fsas_state_idx01)
            << "We support only FSAs with at least two states at present";

        int32_t fsas_state_idx0x = fsas_row_splits1_data[fsas_idx0];
        int32_t fsas_state_idx1 = fsas_state_idx01 - fsas_state_idx0x;
        int32_t this_fsa_final_state_idx1 =
            this_fsa_final_state_idx01 - fsas_state_idx0x;

        int32_t fsas_arc_idx0xx = fsas_row_splits2_data[fsas_state_idx0x];

        // fsa0: +1 (a new start state)
        // fsa1: +0 (the final state of fsa0 is removed)
        // fsa2: -1 (the final state of fsa1 is removed)
        // fsa3: -2 (the final state of fsa2 is removed)
        int32_t state_offset = 1 - fsas_idx0;
        int32_t out_state_idx0 = fsas_state_idx01 + state_offset;

        int32_t out_arc_idx01 = fsas_arc_idx012 + num_fsas;
        out_row_ids_data[out_arc_idx01] = out_state_idx0;
        Arc arc = arcs_data[fsas_arc_idx012];

        K2_DCHECK_EQ(arc.src_state, fsas_state_idx1);

        if (arc.dest_state == this_fsa_final_state_idx1)
          arc.dest_state = out_final_state;
        else
          arc.dest_state = arc.dest_state - arc.src_state + out_state_idx0;

        arc.src_state = out_state_idx0;
        out_arcs_data[out_arc_idx01] = arc;
        tmp_arc_map_data[out_arc_idx01] = fsas_arc_idx012;

        if (fsas_arc_idx0xx == fsas_arc_idx012) {
          // add a new arc from the new start state to the start state
          // of this fsa
          //
          // WARNING: we cannot use fsas_state_idx01 here
          // since the start state may have no leaving arcs!
          Arc arc(0, fsas_state_idx0x + state_offset, 0, 0);
          out_arcs_data[fsas_idx0] = arc;
          out_row_ids_data[fsas_idx0] = 0;
        }
      });

  if (arc_map != nullptr) *arc_map = std::move(tmp_arc_map);
  Array1<int32_t> out_row_splits(context, num_out_states + 1);
  RowIdsToRowSplits(out_row_ids, &out_row_splits);
  RaggedShape shape = RaggedShape2(&out_row_splits, &out_row_ids, num_out_arcs);
  Fsa ans = Ragged<Arc>(shape, out_arcs);
  return ans;
}

Fsa Closure(Fsa &fsa, Array1<int32_t> *arc_map /* = nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(fsa.NumAxes(), 2) << "We support only a single FSA.";
  ContextPtr &c = fsa.Context();

  int32_t num_states = fsa.Dim0();
  if (num_states < 2) {
    K2_CHECK_EQ(num_states, 0)
        << "An empty fsa should contain no states at all";
    if (arc_map != nullptr) *arc_map = Array1<int32_t>(c, 0);
    return fsa;  // return itself if the input fsa is empty
  }

  const int32_t *fsa_row_splits_data = fsa.RowSplits(1).Data();
  const int32_t *fsa_row_ids_data = fsa.RowIds(1).Data();
  const Arc *fsa_arcs_data = fsa.values.Data();
  int32_t fsa_final_state = num_states - 1;

  int32_t num_out_states = num_states;

  // An arc from the start state to the final state with label == -1 is added.
  int32_t num_out_arcs = fsa.values.Dim() + 1;

  Array1<int32_t> out_row_ids(c, num_out_arcs);
  int32_t *out_row_ids_data = out_row_ids.Data();

  Array1<Arc> out_arcs(c, num_out_arcs);
  Arc *out_arcs_data = out_arcs.Data();

  Array1<int32_t> tmp_arc_map(c, num_out_arcs);
  int32_t *tmp_arc_map_data = tmp_arc_map.Data();

  K2_EVAL(
      c, fsa.values.Dim(), lambda_set_arcs, (int32_t fsa_arc_idx01) {
        int32_t fsa_state_idx0 = fsa_row_ids_data[fsa_arc_idx01];
        int32_t fsa_arc_idx0x = fsa_row_splits_data[fsa_state_idx0];
        int32_t fsa_arc_idx1 = fsa_arc_idx01 - fsa_arc_idx0x;
        int32_t this_state_num_arcs =
            fsa_row_splits_data[fsa_state_idx0 + 1] - fsa_arc_idx0x;

        Arc arc = fsa_arcs_data[fsa_arc_idx01];
        if (arc.dest_state == fsa_final_state) {
          // modify arcs entering the final state such that:
          //   - dest_state == 0
          //   - label == 0
          arc.dest_state = 0;
          K2_DCHECK_EQ(arc.label, -1);
          arc.label = 0;
        }

        int out_arc_idx01;
        if (arc.src_state > 0) {
          // this arc is not originated from the start state, so its index is
          // incremented
          out_arc_idx01 = fsa_arc_idx01 + 1;
        } else {
          out_arc_idx01 = fsa_arc_idx01;
          if (fsa_arc_idx1 == this_state_num_arcs - 1) {
            // This is the last arc of the original start state,
            // so we add a new arc just after it.
            Arc new_arc(0, fsa_final_state, -1, 0.0f);
            out_arcs_data[out_arc_idx01 + 1] = new_arc;
            out_row_ids_data[out_arc_idx01 + 1] = 0;
            tmp_arc_map_data[out_arc_idx01 + 1] = -1;
          }
        }

        // it may happen that the start state has no leaving arcs
        if (fsa_row_splits_data[1] == 0) {
          Arc new_arc(0, fsa_final_state, -1, 0.0f);
          out_arcs_data[0] = new_arc;
          out_row_ids_data[0] = 0;
          tmp_arc_map_data[0] = -1;
        }

        tmp_arc_map_data[out_arc_idx01] = fsa_arc_idx01;

        out_arcs_data[out_arc_idx01] = arc;
        out_row_ids_data[out_arc_idx01] = arc.src_state;
      });

  if (arc_map != nullptr) *arc_map = std::move(tmp_arc_map);

  Array1<int32_t> out_row_splits(c, num_out_states + 1);
  int32_t *out_row_splits_data = out_row_splits.Data();

  K2_EVAL(
      c, out_row_splits.Dim(), lambda_set_row_splits, (int32_t i) {
        if (i == 0)
          out_row_splits_data[i] = 0;
        else
          out_row_splits_data[i] = fsa_row_splits_data[i] + 1;
      });

  RaggedShape shape = RaggedShape2(&out_row_splits, &out_row_ids, num_out_arcs);
  Fsa ans = Ragged<Arc>(shape, out_arcs);
  return ans;
}

FsaOrVec ExpandArcs(FsaOrVec &fsas, RaggedShape &labels_shape,
                    Array1<int32_t> *fsas_arc_map /*=nullptr*/,
                    Array1<int32_t> *labels_arc_map /*=nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  if (fsas.NumAxes() == 2) {
    FsaVec fsas_temp = FsaToFsaVec(fsas);
    return ExpandArcs(fsas_temp, labels_shape, fsas_arc_map, labels_arc_map)
        .RemoveAxis(0);
  }
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  K2_CHECK_EQ(labels_shape.NumAxes(), 2);
  K2_CHECK_EQ(fsas.NumElements(), labels_shape.Dim0());
  ContextPtr &c = fsas.Context();
  K2_CHECK(c->IsCompatible(*labels_shape.Context()));

  RaggedShape state_to_arcs = GetLayer(fsas.shape, 1);

  // `state_to_foo` is a RaggedShape that, for each state in `fsas`, has a list
  // of length `num_arcs + 1`, where `num_arcs` is the number of arcs leaving
  // this state in `fsas`.  Interpret this as: one element for the state
  // itself, then one for each arc leaving it.  This `foo` is an index that
  // corresponds to num-arcs plus one, but because it is really a placeholder
  // and we want to keep it distinct from other things, we call it `foo`.
  RaggedShape state_to_foo = ChangeSublistSize(state_to_arcs, 1);

  int32_t foo_size = state_to_foo.NumElements();

  // For each element of `state_to_foo`, `num_ostates_for` says how many states
  // there will be for this (state,foo) in the returned (output) FSA.  Here, the
  // idx0 is the state, the idx1 is foo.  If idx1 == 0 (interpret this as "the
  // state itself"), then `num_ostates_for[idx01] = 1`, meaning "keep the
  // original state".  Otherwise, idx1 - 1 represents an arc_idx2 [into `fsas`],
  // and we set `num_ostates_for[idx01] = max(0, seq_len-1)`, where seq_len is
  // the length of the sequence in `labels_shape` corresponding to this
  // arc-index.
  Array1<int32_t> num_ostates_for(c, foo_size + 1);
  int32_t *num_ostates_for_data = num_ostates_for.Data();

  const int32_t *labels_row_splits1_data = labels_shape.RowSplits(1).Data(),
                *fsas_row_splits2_data = fsas.RowSplits(2).Data(),
                *state_to_foo_row_splits1_data =
                    state_to_foo.RowSplits(1).Data(),
                *state_to_foo_row_ids1_data = state_to_foo.RowIds(1).Data();

  K2_EVAL(
      c, foo_size, lambda_set_num_ostates, (int32_t idx01)->void {
        // note: the idx01, idx0, idx0x are into `state_to_foo`.
        // This idx0 is a state-index into `fsas` (an idx01 w.r.t. `fsas`).
        int32_t idx0 = state_to_foo_row_ids1_data[idx01],
                idx0x = state_to_foo_row_splits1_data[idx0],
                idx1 = idx01 - idx0x;  // idx1 is `foo`.
        int32_t num_ostates;
        if (idx1 == 0) {
          num_ostates = 1;  // this is a copy of the original state.
        } else {
          int32_t fsas_arc_idx2 = idx1 - 1, fsas_state_idx01 = idx0,
                  fsas_arc_idx01x = fsas_row_splits2_data[fsas_state_idx01],
                  fsas_arc_idx012 = fsas_arc_idx01x + fsas_arc_idx2,
                  labels_shape_idx0 = fsas_arc_idx012,
                  labels_shape_idx0x =
                      labels_row_splits1_data[labels_shape_idx0],
                  labels_shape_idx0x_next =
                      labels_row_splits1_data[labels_shape_idx0 + 1],
                  labels_shape_len1 =
                      labels_shape_idx0x_next - labels_shape_idx0x;
          // A sequence of n symbols will require n-1 extra states to represent
          // it.
          num_ostates = max(labels_shape_len1 - 1, (int32_t)0);
        }
        num_ostates_for_data[idx01] = num_ostates;
      });
  ExclusiveSum(num_ostates_for, &num_ostates_for);
  Array1<int32_t> &foo_to_ostates_row_splits = num_ostates_for;
  RaggedShape foo_to_ostates =
      RaggedShape2(&foo_to_ostates_row_splits, nullptr, -1);

  // to_ostates_shape has 4 axes: [fsa_id][orig_state][foo][ostate]
  // where foo is a general-purpose index that ranges over the (num_arcs + 1) of
  // the original state.
  RaggedShape to_ostates_shape = ComposeRaggedShapes3(
      GetLayer(fsas.shape, 0), state_to_foo, foo_to_ostates);

  // Below, `tos` means `to_ostates_shape`.
  const int32_t *tos_row_splits1_data = to_ostates_shape.RowSplits(1).Data(),
                *tos_row_ids1_data = to_ostates_shape.RowIds(1).Data(),
                *tos_row_splits2_data = to_ostates_shape.RowSplits(2).Data(),
                *tos_row_ids2_data = to_ostates_shape.RowIds(2).Data(),
                *tos_row_splits3_data = to_ostates_shape.RowSplits(3).Data(),
                *tos_row_ids3_data = to_ostates_shape.RowIds(3).Data();

  // `num_oarcs` gives the number of arcs in the returned (output) FSA for each
  // `ostate` (i.e. leaving each state in the returned FSA).
  int32_t tot_ostates = to_ostates_shape.NumElements();
  Array1<int32_t> num_oarcs(c, tot_ostates + 1);
  int32_t *num_oarcs_data = num_oarcs.Data();
  K2_EVAL(
      c, tot_ostates, lambda_set_num_oarcs, (int32_t idx0123)->void {
        // All these indexes are into `to_ostates_shape`, indexed
        // `[fsa][state][foo][ostate].`
        int32_t idx012 = tos_row_ids3_data[idx0123],
                idx012x = tos_row_splits3_data[idx012],
                idx01 = tos_row_ids2_data[idx012],
                idx01x = tos_row_splits2_data[idx01],
                idx01x_next = tos_row_splits2_data[idx01 + 1],
                len2 = idx01x_next - idx01x, idx2 = idx012 - idx01x,
                idx3 = idx0123 - idx012x;
        int32_t num_arcs;
        if (idx2 == 0) {
          K2_CHECK_EQ(idx3, 0);
          // This ostate corresponds to the original state; it is not one of the
          // extra states added to support chains of arcs.
          // The original state had `orig_num_arcs` leaving it, which is the
          // number of `foo` indexes minus one.
          int32_t orig_num_arcs = len2 - 1;
          num_arcs = orig_num_arcs;
        } else {
          // All newly-created states have exactly one arc leaving them.
          num_arcs = 1;
        }
        num_oarcs_data[idx0123] = num_arcs;
      });
  ExclusiveSum(num_oarcs, &num_oarcs);
  Array1<int32_t> &ostate_to_oarcs_row_splits = num_oarcs;
  RaggedShape ostate_to_oarcs =
      RaggedShape2(&ostate_to_oarcs_row_splits, nullptr, -1);

  // `full_shape` has 5 axes: [fsa][orig_state][foo][ostate][oarc]
  RaggedShape full_shape =
      ComposeRaggedShapes(to_ostates_shape, ostate_to_oarcs);
  // for the lower-order row-splits and row-ids, use tot_row_{splits,idx}n_data
  const int32_t *full_row_splits4_data = full_shape.RowSplits(4).Data(),
                *full_row_ids4_data = full_shape.RowIds(4).Data();
  int32_t tot_oarcs = full_shape.NumElements();
  K2_CHECK_GE(tot_oarcs, fsas.NumElements());

  int32_t *fsas_arc_map_data = nullptr, *labels_arc_map_data = nullptr;
  if (fsas_arc_map) {
    *fsas_arc_map = Array1<int32_t>(c, tot_oarcs);
    fsas_arc_map_data = fsas_arc_map->Data();
  }
  if (labels_arc_map) {
    *labels_arc_map = Array1<int32_t>(c, tot_oarcs);
    labels_arc_map_data = labels_arc_map->Data();
  }
  Array1<Arc> oarcs(c, tot_oarcs);
  Arc *oarcs_data = oarcs.Data();
  const Arc *arcs_data = fsas.values.Data();

  K2_EVAL(
      c, tot_oarcs, lambda_set_arcs, (int32_t idx01234)->void {
        // All these indexes are into `full_shape`, indexed
        // `[fsa][state][foo][ostate][oarc].`
        int32_t idx0123 = full_row_ids4_data[idx01234],
                idx0123x = full_row_splits4_data[idx0123],
                idx4 = idx01234 - idx0123x, idx012 = tos_row_ids3_data[idx0123],
                idx012x = tos_row_splits3_data[idx012],
                idx3 = idx0123 - idx012x, idx01 = tos_row_ids2_data[idx012],
                idx01x = tos_row_splits2_data[idx01], idx2 = idx012 - idx01x,
                idx0 = tos_row_ids1_data[idx01],
                idx0x = tos_row_splits1_data[idx0],
                idx0xxx = tos_row_splits3_data[tos_row_splits2_data[idx0x]];

        int32_t fsa_idx01x = fsas_row_splits2_data[idx01];

        int32_t fsa_idx2;  // the idx2 (arc-index) into `fsas` of the input arc
                           // that's most relevant to us..
        int32_t seq_pos;  // seq_pos is our index into the sequence of arcs that
                          // we produce for each original arc
        if (idx2 == 0) {
          K2_CHECK_EQ(idx3, 0);
          fsa_idx2 = idx4;  // corresponds to foo=0, so idx3 will be 0; the idx4
                            // enumerates the arcs leaving it..
          seq_pos = 0;
        } else {
          // this is one of the extra `foo` indexes, one per arc in the input
          // FSA that leaves this state; each of those `foo` indexes has
          // (seq_len - 1) states in it (idx3=0,1..seq_len-1); and each state
          // has one arc leaving it (idx4==0).
          K2_CHECK_EQ(idx4, 0);
          fsa_idx2 = idx2 - 1;
          seq_pos = idx3 + 1;
        }
        int32_t fsa_idx012 = fsa_idx01x + fsa_idx2;  // index of the arc in
                                                     // source FSA FSA that
                                                     // we're expanding..
        Arc iarc = arcs_data[fsa_idx012];

        int32_t labels_idx0x = labels_row_splits1_data[fsa_idx012],
                labels_next_idx0x = labels_row_splits1_data[fsa_idx012 + 1],
                labels_len1 = labels_next_idx0x - labels_idx0x;
        // labels_len1 is length of label sequence for this arc
        K2_CHECK_LT(seq_pos, max(int32_t(1), labels_len1));

        int32_t dest_idx01 = idx0x + iarc.dest_state,  // original destination
                                                       // state-index
            orig_dest_idx0123 =
                tos_row_splits3_data[tos_row_splits2_data[dest_idx01]];

        Arc oarc;
        oarc.src_state = idx0123 - idx0xxx;
        // If this is the last arc in the sequence, the dest-state is the
        // original dest-state of the arc.  Otherwise the dest-state is one of
        // the new states that we created. The idx123 will be an idx1 after
        // removing axes.
        int32_t dest_idx123;
        if (seq_pos + 1 >= labels_len1) {  // last arc in sequence..
          dest_idx123 = orig_dest_idx0123 - idx0xxx;
        } else {
          int32_t dest_state_idx2 = fsa_idx2 + 1,  // index `foo` equals
                                                   // orig_arc_idx+1
              dest_state_idx3 = seq_pos,           // ostate index..
              dest_idx012 = idx01x + dest_state_idx2,
                  dest_idx012x = tos_row_splits3_data[dest_idx012],
                  dest_idx0123 = dest_idx012x + dest_state_idx3;
          dest_idx123 = dest_idx0123 - idx0xxx;
        }
        oarc.dest_state = dest_idx123;  // indexes 1,2,3 will be combined; in
                                        // the output FSA it will be an idx1.

        if (fsas_arc_map_data)
          fsas_arc_map_data[idx01234] = (seq_pos == 0 ? fsa_idx012 : -1);
        if (labels_arc_map_data)
          labels_arc_map_data[idx01234] =
              (seq_pos < labels_len1 ? labels_idx0x + seq_pos : -1);
        if (iarc.label != -1) {
          // normal case.. label goes on 1st arc in sequence
          oarc.label = (seq_pos == 0 ? iarc.label : 0);
        } else {
          // If the arc was to the final-state, we need to keep the label on the
          // last arc of the sequence to keep the output valid.  The following
          // would be "seq_pos + 1 == labels_len1 ? -1 : 0", but we make it ">="
          // not "=" to account for the case seq_pos=0, labels_len1 = 0.
          oarc.label = (seq_pos + 1 >= labels_len1 ? -1 : 0);
        }
        oarc.score = (seq_pos == 0 ? iarc.score : 0.0);
        oarcs_data[idx01234] = oarc;
      });

  // remove current axes 1 and 2... [after removing axis 1, old axis 2 becomes
  // axis 1, so remove axis 1 twice].
  RaggedShape temp = RemoveAxis(full_shape, 1);
  return FsaVec(RemoveAxis(temp, 1), oarcs);
}


void Invert(FsaOrVec &src, Ragged<int32_t> &src_aux_labels, FsaOrVec *dest,
            Ragged<int32_t> *dest_aux_labels,
            Array1<int32_t> *arc_map /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(src_aux_labels.NumAxes(), 2);
  K2_CHECK_EQ(src_aux_labels.Dim0(), src.NumElements());
  K2_CHECK(dest != nullptr && dest_aux_labels != nullptr);
  ContextPtr c = GetContext(src, src_aux_labels);
  if (src.NumAxes() == 2) {
    Fsa *srcs = &src;
    FsaVec src_vec = CreateFsaVec(1, &srcs), dest_vec;
    Invert(src_vec, src_aux_labels, &dest_vec, dest_aux_labels, arc_map);
    *dest = GetFsaVecElement(dest_vec, 0);
    return;
  }
  Array1<int32_t> src_arc_map, labels_arc_map;
  *dest = ExpandArcs(src, src_aux_labels.shape, &src_arc_map, &labels_arc_map);
  // swap labels and aux_labels
  int32_t dest_num_arcs = dest->NumElements();
  Arc *dest_arcs_data = dest->values.Data();
  const int32_t *labels_arc_map_data = labels_arc_map.Data(),
                *src_aux_labels_data = src_aux_labels.values.Data();
  Array1<int32_t> dest_aux_labels_row_splits(c, dest_num_arcs + 1);
  int32_t *dest_aux_labels_row_splits_data = dest_aux_labels_row_splits.Data();
  K2_EVAL(
      c, dest_num_arcs, lambda_set_dest_aux_labels_num,
      (int32_t dest_idx012)->void {
        Arc &dest_arc = dest_arcs_data[dest_idx012];
        // we'll remove epsilons in dest_aux_labels
        dest_aux_labels_row_splits_data[dest_idx012] =
            dest_arc.label == 0 ? 0 : 1;
      });
  ExclusiveSum(dest_aux_labels_row_splits.Arange(0, dest_num_arcs),
               &dest_aux_labels_row_splits);
  RaggedShape dest_aux_labels_shape =
      RaggedShape2(&dest_aux_labels_row_splits, nullptr, -1);
  Array1<int32_t> dest_aux_labels_values(c,
                                         dest_aux_labels_shape.NumElements());
  int32_t *dest_aux_labels_values_data = dest_aux_labels_values.Data();
  K2_EVAL(
      c, dest_num_arcs, lambda_set_dest_labels_and_aux_labels,
      (int32_t dest_idx012)->void {
        Arc &dest_arc = dest_arcs_data[dest_idx012];
        // swap label and aux_label
        if (dest_arc.label != 0) {
          int32_t dest_aux_labels_idx0x =
              dest_aux_labels_row_splits_data[dest_idx012];
          // every arc in dest has at most one aux_label (as the aux_label is
          // the label of src on this arc)
          dest_aux_labels_values_data[dest_aux_labels_idx0x] = dest_arc.label;
        }
        int32_t src_aux_labels_idx01 = labels_arc_map_data[dest_idx012];
        dest_arc.label = src_aux_labels_idx01 == -1
                             ? 0
                             : src_aux_labels_data[src_aux_labels_idx01];
      });
  *dest_aux_labels =
      Ragged<int32_t>(dest_aux_labels_shape, dest_aux_labels_values);
  if (arc_map != nullptr) *arc_map = src_arc_map;
}

// Will be used in InvertHost to process FsaVec input recursively.
void RecursionWrapperAuxLabels(void (*f)(FsaOrVec &, Ragged<int32_t> &,
                                         FsaOrVec *, Ragged<int32_t> *),
                               FsaOrVec &src, Ragged<int32_t> &src_aux_labels,
                               FsaOrVec *dest,
                               Ragged<int32_t> *dest_aux_labels) {
  NVTX_RANGE(K2_FUNC);
  // src is actually an FsaVec.  Just recurse for now.
  K2_CHECK_EQ(src.NumAxes(), 3);
  int32_t num_fsas = src.shape.Dim0();
  std::vector<Fsa> srcs(num_fsas), dests(num_fsas);
  std::vector<Ragged<int32_t>> src_aux_labels_vec(num_fsas),
      dest_aux_labels_vec(num_fsas);
  int32_t tot_num_arcs = 0;
  Array1<int32_t> src_aux_labels_row_splits = src_aux_labels.RowSplits(1),
                  src_aux_labels_values = src_aux_labels.values;
  for (int32_t i = 0; i < num_fsas; ++i) {
    srcs[i] = src.Index(0, i);
    int32_t cur_num_arcs = srcs[i].NumElements();
    // below block get aux_labels for srcs[i]
    // TODO(haowen): replace with Range op for ragged
    {
      Array1<int32_t> row_splits = src_aux_labels_row_splits.Arange(
          tot_num_arcs, tot_num_arcs + cur_num_arcs + 1);
      Array1<int32_t> values =
          src_aux_labels_values.Arange(row_splits[0], row_splits.Back());
      row_splits = Minus(row_splits, row_splits[0]);
      RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
      src_aux_labels_vec[i] = Ragged<int32_t>(shape, values);
    }
    f(srcs[i], src_aux_labels_vec[i], &(dests[i]), &(dest_aux_labels_vec[i]));
    tot_num_arcs += cur_num_arcs;
  }
  *dest = Stack(0, num_fsas, dests.data());
  *dest_aux_labels = Cat(0, num_fsas, dest_aux_labels_vec.data());
}

void InvertHost(FsaOrVec &src, Ragged<int32_t> &src_aux_labels, FsaOrVec *dest,
                Ragged<int32_t> *dest_aux_labels) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(src_aux_labels.NumAxes(), 2);
  K2_CHECK_EQ(src_aux_labels.Dim0(), src.NumElements());
  K2_CHECK(dest != nullptr && dest_aux_labels != nullptr);
  int32_t num_axes = src.NumAxes();
  if (num_axes < 2 || num_axes > 3) {
    K2_LOG(FATAL) << "Input has bad num-axes " << num_axes;
  } else if (num_axes == 3) {
    return RecursionWrapperAuxLabels(InvertHost, src, src_aux_labels, dest,
                                     dest_aux_labels);
  }

  k2host::Fsa host_fsa = FsaToHostFsa(src);
  // k2host::AuxLabels is a k2host::Array2
  k2host::AuxLabels host_aux_labels(
      src_aux_labels.Dim0(), src_aux_labels.NumElements(),
      src_aux_labels.RowSplits(1).Data(), src_aux_labels.values.Data());
  k2host::FstInverter inverter(host_fsa, host_aux_labels);
  k2host::Array2Size<int32_t> fsa_size, aux_size;
  inverter.GetSizes(&fsa_size, &aux_size);
  FsaCreator fsa_creator(fsa_size);
  k2host::Fsa host_dest_fsa = fsa_creator.GetHostFsa();
  Ragged2Creator<int32_t> ragged_creator(aux_size);
  k2host::AuxLabels host_dest_aux_labels = ragged_creator.GetHostArray2();
  inverter.GetOutput(&host_dest_fsa, &host_dest_aux_labels);
  *dest = fsa_creator.GetFsa();
  *dest_aux_labels = ragged_creator.GetRagged2();
}

FsaOrVec ReplaceFsa(FsaVec &src, FsaOrVec &index, int32_t symbol_range_begin,
                    Array1<int32_t> *arc_map_src /* = nullptr */,
                    Array1<int32_t> *arc_map_index /* = nullptr */) {
  NVTX_RANGE(K2_FUNC);
  if (index.NumAxes() == 2) {
    FsaVec index_temp = FsaToFsaVec(index);
    return ReplaceFsa(src, index_temp, symbol_range_begin, arc_map_src,
                      arc_map_index).RemoveAxis(0);
  }
  K2_CHECK_EQ(index.NumAxes(), 3);
  ContextPtr &c = index.Context();
  K2_CHECK(c->IsCompatible(*src.Context()));

  RaggedShape state_to_arcs = GetLayer(index.shape, 1);

  // `state_to_foo` is a RaggedShape that, for each state in `index`, has a list
  // of length `tot_arcs + 1`.  Interpret this as: one element for the state
  // itself, then one for each arc leaving it.  This `foo` is an index that
  // corresponds to num-arcs plus one, but because it is really a placeholder
  // and we want to keep it distinct from other things, we call it `foo`.
  RaggedShape state_to_foo = ChangeSublistSize(state_to_arcs, 1);

  int32_t foo_size = state_to_foo.NumElements(),
          num_src_fsas = src.Dim0();
  // For each element of `state_to_foo`, `num_ostates_for` says how many states
  // there will be for this (state,foo) in the returned (output) FSA.  Here, the
  // idx0 is the state, the idx1 is foo.  If idx1 == 0 (interpret this as "the
  // state itself"), then `num_ostates_for[idx01] = 1`, meaning "keep the
  // original state".  Otherwise, idx1 - 1 represents an arc_idx2 [into `index`]
  // and we set `num_ostates_for[idx01] = max(0, state_num-1)`, where state_num
  // is the states number of the fsa in `src` that would repalce into this arc,
  // the final state of this fsa will identify with the dest-state of this arc,
  // so we minus 1.
  Array1<int32_t> num_ostates_for(c, foo_size + 1);
  int32_t *num_ostates_for_data = num_ostates_for.Data();
  const Arc *index_arcs_data = index.values.Data();

  const int32_t *src_row_splits1_data = src.RowSplits(1).Data(),
                *index_row_splits2_data = index.RowSplits(2).Data(),
                *state_to_foo_row_splits1_data =
                    state_to_foo.RowSplits(1).Data(),
                *state_to_foo_row_ids1_data = state_to_foo.RowIds(1).Data();

  K2_EVAL(
      c, foo_size, lambda_set_num_ostates, (int32_t idx01)->void {
        // note: the idx01, idx0, idx0x are into `state_to_foo`.
        // This idx0 is a state-index into `index` (an idx01 w.r.t. `index`).
        int32_t idx0 = state_to_foo_row_ids1_data[idx01],
                idx0x = state_to_foo_row_splits1_data[idx0],
                idx1 = idx01 - idx0x;  // idx1 is `foo`.
        int32_t num_ostates;
        if (idx1 == 0) {
          num_ostates = 1;  // this is a copy of the original state.
        } else {
          int32_t index_arc_idx2 = idx1 - 1, index_state_idx01 = idx0,
                  index_arc_idx01x = index_row_splits2_data[index_state_idx01],
                  index_arc_idx012 = index_arc_idx01x + index_arc_idx2,
                  index_label = index_arcs_data[index_arc_idx012].label,
                  src_idx0 = index_label - symbol_range_begin;
          // will not replace for this arc
          if (src_idx0 < 0 || src_idx0 >= num_src_fsas) {
            num_ostates = 0;
          } else {
            int32_t src_idx0x = src_row_splits1_data[src_idx0],
                    src_idx0x_next = src_row_splits1_data[src_idx0 + 1],
                    src_len1 = src_idx0x_next - src_idx0x;
            num_ostates = max(src_len1 - 1, (int32_t)0);
          }
        }
        num_ostates_for_data[idx01] = num_ostates;
      });
  ExclusiveSum(num_ostates_for, &num_ostates_for);
  Array1<int32_t> &foo_to_ostates_row_splits = num_ostates_for;
  RaggedShape foo_to_ostates =
      RaggedShape2(&foo_to_ostates_row_splits, nullptr, -1);

  // to_ostates_shape has 4 axes: [fsa_id][orig_state][foo][ostate]
  // where foo is a general-purpose index that ranges over the (num_arcs + 1) of
  // the original state.
  RaggedShape to_ostates_shape = ComposeRaggedShapes3(
      GetLayer(index.shape, 0), state_to_foo, foo_to_ostates);

  // Below, `tos` means `to_ostates_shape`.
  const int32_t *tos_row_splits1_data = to_ostates_shape.RowSplits(1).Data(),
                *tos_row_ids1_data = to_ostates_shape.RowIds(1).Data(),
                *tos_row_splits2_data = to_ostates_shape.RowSplits(2).Data(),
                *tos_row_ids2_data = to_ostates_shape.RowIds(2).Data(),
                *tos_row_splits3_data = to_ostates_shape.RowSplits(3).Data(),
                *tos_row_ids3_data = to_ostates_shape.RowIds(3).Data(),
                *src_row_splits2_data = src.RowSplits(2).Data();

  // `num_oarcs` gives the number of arcs in the returned (output) FSA for each
  // `ostate` (i.e. leaving each state in the returned FSA).
  int32_t tot_ostates = to_ostates_shape.NumElements();
  Array1<int32_t> num_oarcs(c, tot_ostates + 1);
  int32_t *num_oarcs_data = num_oarcs.Data();
  K2_EVAL(
      c, tot_ostates, lambda_set_num_oarcs, (int32_t idx0123)->void {
        // All these indexes are into `to_ostates_shape`, indexed
        // `[fsa][state][foo][ostate].`
        int32_t idx012 = tos_row_ids3_data[idx0123],
                idx012x = tos_row_splits3_data[idx012],
                idx01 = tos_row_ids2_data[idx012],
                idx01x = tos_row_splits2_data[idx01],
                idx01x_next = tos_row_splits2_data[idx01 + 1],
                len2 = idx01x_next - idx01x, idx2 = idx012 - idx01x,
                idx3 = idx0123 - idx012x;
        int32_t num_arcs;
        if (idx2 == 0) {
          K2_CHECK_EQ(idx3, 0);
          // This ostate corresponds to the original state;
          // The original state had `orig_num_arcs` leaving it, which is the
          // number of `foo` indexes minus one.
          int32_t orig_num_arcs = len2 - 1;
          num_arcs = orig_num_arcs;
        } else {
          // All inserted states have the same num of arcs as in the src.
          // note: the prefix `index_` means it is an idxXXX w.r.t. `index`.
          // the prefix `src_` means the variable is an idxXXX w.r.t. `src`.
          int32_t index_arc_idx2 = idx2 - 1,
                  index_arc_idx01x = index_row_splits2_data[idx01],
                  index_arc_idx012 = index_arc_idx01x + index_arc_idx2,
                  index_label = index_arcs_data[index_arc_idx012].label,
                  src_fsa_idx0 = index_label - symbol_range_begin;
          K2_CHECK_GE(src_fsa_idx0, 0);
          K2_CHECK_LT(src_fsa_idx0, num_src_fsas);
          int32_t src_state_idx1 = idx3,
                  src_state_idx0x = src_row_splits1_data[src_fsa_idx0],
                  src_state_idx01 = src_state_idx0x + src_state_idx1,
                  src_arc_idx01x = src_row_splits2_data[src_state_idx01],
                  src_arc_idx01x_next =
                    src_row_splits2_data[src_state_idx01 + 1],
                  src_num_arcs = src_arc_idx01x_next - src_arc_idx01x;
          num_arcs = src_num_arcs;
        }
        num_oarcs_data[idx0123] = num_arcs;
      });
  ExclusiveSum(num_oarcs, &num_oarcs);
  Array1<int32_t> &ostate_to_oarcs_row_splits = num_oarcs;
  RaggedShape ostate_to_oarcs =
      RaggedShape2(&ostate_to_oarcs_row_splits, nullptr, -1);

  // `full_shape` has 5 axes: [fsa][orig_state][foo][ostate][oarc]
  RaggedShape full_shape =
      ComposeRaggedShapes(to_ostates_shape, ostate_to_oarcs);

  // for the lower-order row-splits and row-ids, use tot_row_{splits,ids}n_data
  const int32_t *full_row_splits4_data = full_shape.RowSplits(4).Data(),
                *full_row_ids4_data = full_shape.RowIds(4).Data();
  int32_t tot_oarcs = full_shape.NumElements();
  K2_CHECK_GE(tot_oarcs, index.NumElements());

  int32_t *arc_map_src_data = nullptr, *arc_map_index_data = nullptr;
  if (arc_map_src) {
    *arc_map_src = Array1<int32_t>(c, tot_oarcs);
    arc_map_src_data = arc_map_src->Data();
  }
  if (arc_map_index) {
    *arc_map_index = Array1<int32_t>(c, tot_oarcs);
    arc_map_index_data = arc_map_index->Data();
  }
  Array1<Arc> oarcs(c, tot_oarcs);
  Arc *oarcs_data = oarcs.Data();
  const Arc *src_arcs_data = src.values.Data();

  K2_EVAL(
      c, tot_oarcs, lambda_set_arcs, (int32_t idx01234)->void {
        // All these indexes are into `full_shape`, indexed
        // `[fsa][state][foo][ostate][oarc].`
        // The prefix `index_` means it is an idxXXX w.r.t. `index`.
        // the prefix `src_` means the variable is an idxXXX w.r.t. `src`.
        int32_t idx0123 = full_row_ids4_data[idx01234],
                idx0123x = full_row_splits4_data[idx0123],
                idx4 = idx01234 - idx0123x,
                idx012 = tos_row_ids3_data[idx0123],
                idx012x = tos_row_splits3_data[idx012],
                idx3 = idx0123 - idx012x,
                idx01 = tos_row_ids2_data[idx012],
                idx01x = tos_row_splits2_data[idx01],
                idx2 = idx012 - idx01x,
                idx0 = tos_row_ids1_data[idx01],
                idx0x = tos_row_splits1_data[idx0],
                idx0xxx = tos_row_splits3_data[tos_row_splits2_data[idx0x]];

        int32_t index_arc_idx2;  // the idx2 (arc-index) into `index`
        if (idx2 == 0) {
          K2_CHECK_EQ(idx3, 0);
          index_arc_idx2 = idx4;  // corresponds to foo=0, so idx3 will be 0;
                                  // the idx4 enumerates the arcs leaving it..
        } else {
          // this is one of the extra `foo` indexes, it's conrespoding index
          // into `index` is `foo` index minus 1
          index_arc_idx2 = idx2 - 1;
        }

        int32_t index_arc_idx01x = index_row_splits2_data[idx01];
        // index of the arc in source FSA, FSA that we're replaceing..
        int32_t index_arc_idx012 = index_arc_idx01x + index_arc_idx2;

        Arc index_arc = index_arcs_data[index_arc_idx012];
        // original destination state-index
        int32_t dest_state_idx01 = idx0x + index_arc.dest_state,
                orig_dest_state_idx0123 =
                  tos_row_splits3_data[tos_row_splits2_data[dest_state_idx01]];

        Arc src_arc;
        Arc oarc;
        oarc.src_state = idx0123 - idx0xxx;
        // initialize mapping index
        int32_t arc_src_map_idx = -1,
                arc_index_map_idx = -1;
        int32_t src_fsa_idx0 = index_arc.label - symbol_range_begin;
        // will not replace for this arc
        // dest state is the dest state of index arc
        if (src_fsa_idx0 < 0 || src_fsa_idx0 >= num_src_fsas) {
          K2_CHECK_EQ(idx2, 0);
          oarc.dest_state = orig_dest_state_idx0123 - idx0xxx;
          oarc.label = index_arc.label;
          oarc.score = index_arc.score;
          arc_index_map_idx = index_arc_idx012;
        } else {
          int32_t src_state_idx0x = src_row_splits1_data[src_fsa_idx0],
                  src_state_idx0x_next = src_row_splits1_data[src_fsa_idx0 + 1],
                  num_states = src_state_idx0x_next - src_state_idx0x,
                  src_state_idx1 = idx3,
                  src_state_idx01 = src_state_idx0x + src_state_idx1,
                  src_arc_idx01x = src_row_splits2_data[src_state_idx01],
                  src_arc_idx2 = idx4,
                  src_arc_idx012 = src_arc_idx01x + src_arc_idx2;
          src_arc = src_arcs_data[src_arc_idx012];
          // handle the arcs belongs to index
          if (idx2 == 0) {
            // if the fsa to be replaced in is empty, this arc would point to
            // its original dest-state
            if (0 == num_states) {
              oarc.dest_state = orig_dest_state_idx0123 - idx0xxx;
            } else {
              // this arc would point to the initial state of the fsa in src,
              // the state id bias to current state(the src-state) is the count
              // of all the ostates coresponding to the original state util now,
              // the idx4 enumerates foo index
              int32_t idx012_t = idx01x + 0,
                      idx2_t = idx4,
                      idx012x_t = tos_row_splits3_data[idx012_t],
                      idx012x_next_t =
                        tos_row_splits3_data[idx012_t + idx2_t + 1],
                      bias = idx012x_next_t - idx012x_t;
              oarc.dest_state = idx0123 + bias - idx0xxx;
            }
            // set the label of the arc we are replacing to be 0(epsilon)
            oarc.label = 0;
            oarc.score = index_arc.score;
            arc_index_map_idx = index_arc_idx012;
          } else {   // handle the arcs belongs to src
            // the arc point to the final state of the fsa in src would point to
            // the dest state of the arc we're replaceing
            if (src_arc.label == -1) {
              oarc.dest_state = orig_dest_state_idx0123 - idx0xxx;
            } else {
              // this is the inner arc of the fsa in src
              int32_t dest_state_idx012x = idx0123 - idx3,
                  dest_state_idx0123 = dest_state_idx012x + src_arc.dest_state;
              oarc.dest_state = dest_state_idx0123 - idx0xxx;
            }
            // arcs in src fsas that point to final state would set to epsilon
            // arc (label from -1 to 0)
            oarc.label = src_arc.label == -1 ? 0 : src_arc.label;
            oarc.score = src_arc.score;
            arc_src_map_idx = src_arc_idx012;
          }
        }
        if (arc_map_src_data)
          arc_map_src_data[idx01234] = arc_src_map_idx;
        if (arc_map_index_data)
          arc_map_index_data[idx01234] = arc_index_map_idx;
        oarcs_data[idx01234] = oarc;
      });
  // remove current axes 1 and 2... [after removing axis 1, old axis 2 becomes
  // axis 1, so remove axis 1 twice].
  RaggedShape temp = RemoveAxis(full_shape, 1);
  return FsaVec(RemoveAxis(temp, 1), oarcs);
}

FsaOrVec RemoveEpsilonSelfLoops(FsaOrVec &src,
                                Array1<int32_t> *arc_map /* = nullptr */) {
  NVTX_RANGE(K2_FUNC);
  if (src.NumAxes() == 2) {
    FsaVec temp = FsaToFsaVec(src);
    return RemoveEpsilonSelfLoops(temp, arc_map).RemoveAxis(0);
  }
  K2_CHECK_EQ(src.NumAxes(), 3);

  ContextPtr &c = src.Context();
  int32_t num_arcs = src.NumElements();
  Renumbering renumber_lists(c, num_arcs);
  char *keep_list_data = renumber_lists.Keep().Data();

  const Arc *arcs_data = src.values.Data();
  K2_EVAL(
      c, num_arcs, lambda_set_keep, (int32_t i)->void {
        Arc arc = arcs_data[i];
        char keep;
        if (arc.label == 0 && arc.src_state == arc.dest_state) {
          // This arc is an epsilon self-loop, so it should be removed
          keep = 0;
        } else {
          keep = 1;
        }
        keep_list_data[i] = keep;
      });
  FsaVec ans = Index(src, 2, renumber_lists.New2Old(), arc_map);
  return ans;
}

}  // namespace k2
