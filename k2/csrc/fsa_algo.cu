#include "hip/hip_runtime.h"
/**
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey, Haowen Qiu)
 *                      Mobvoi Inc.        (authors: Fangjun Kuang)
 *
 * See LICENSE for clarification regarding multiple authors
 */

#include <algorithm>
#include <limits>
#include <memory>
#include <type_traits>
#include <utility>
#include <vector>

#include "k2/csrc/array_ops.h"
#include "k2/csrc/fsa_algo.h"
#include "k2/csrc/fsa_utils.h"
#include "k2/csrc/host/aux_labels.h"
#include "k2/csrc/host/connect.h"
#include "k2/csrc/host/determinize.h"
#include "k2/csrc/host/intersect.h"
#include "k2/csrc/host/rmepsilon.h"
#include "k2/csrc/host/topsort.h"
#include "k2/csrc/host_shim.h"
#include "k2/csrc/macros.h"
#include "k2/csrc/rm_epsilon.h"


// this contains a subset of the algorithms in fsa_algo.h; currently it just
// contains one that are wrappings of the corresponding algorithms in
// host/.
namespace k2 {

bool RecursionWrapper(bool (*f)(Fsa &, Fsa *, Array1<int32_t> *), Fsa &src,
                      Fsa *dest, Array1<int32_t> *arc_map) {
  NVTX_RANGE(K2_FUNC);
  // src is actually an FsaVec.  Just recurse for now.
  int32_t num_fsas = src.shape.Dim0();
  std::vector<Fsa> srcs(num_fsas), dests(num_fsas);
  std::vector<Array1<int32_t>> arc_maps(num_fsas);
  int32_t tot_num_arcs = 0;
  for (int32_t i = 0; i < num_fsas; ++i) {
    srcs[i] = src.Index(0, i);
    // Recurse.
    if (!f(srcs[i], &(dests[i]),
           (arc_map != nullptr ? &(arc_maps[i]) : nullptr)))
      return false;
    if (arc_map != nullptr) {
      // convert arc indexes in arc_maps from idx2 to idx012
      arc_maps[i] = Plus(arc_maps[i], tot_num_arcs);
      tot_num_arcs += srcs[i].NumElements();
    }
  }
  *dest = Stack(0, num_fsas, dests.data());
  if (arc_map != nullptr)
    *arc_map = Append(src.Context(), num_fsas, arc_maps.data());
  return true;
}

bool Connect(Fsa &src, Fsa *dest, Array1<int32_t> *arc_map /*=nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  int32_t num_axes = src.NumAxes();
  if (num_axes < 2 || num_axes > 3) {
    K2_LOG(FATAL) << "Input has bad num-axes " << num_axes;
  } else if (num_axes == 3) {
    return RecursionWrapper(Connect, src, dest, arc_map);
  }

  k2host::Fsa host_fsa = FsaToHostFsa(src);
  k2host::Connection c(host_fsa);
  k2host::Array2Size<int32_t> size;
  c.GetSizes(&size);
  FsaCreator creator(size);
  k2host::Fsa host_dest_fsa = creator.GetHostFsa();
  int32_t *arc_map_data = nullptr;
  if (arc_map != nullptr) {
    *arc_map = Array1<int32_t>(src.Context(), size.size2);
    arc_map_data = arc_map->Data();
  }
  bool ans = c.GetOutput(&host_dest_fsa, arc_map_data);
  *dest = creator.GetFsa();
  return ans;
}

bool TopSortHost(Fsa &src, Fsa *dest, Array1<int32_t> *arc_map /*=nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  int32_t num_axes = src.NumAxes();
  if (num_axes < 2 || num_axes > 3) {
    K2_LOG(FATAL) << "Input has bad num-axes " << num_axes;
  } else if (num_axes == 3) {
    return RecursionWrapper(TopSortHost, src, dest, arc_map);
  }

  k2host::Fsa host_fsa = FsaToHostFsa(src);
  k2host::TopSorter sorter(host_fsa);
  k2host::Array2Size<int32_t> size;
  sorter.GetSizes(&size);
  FsaCreator creator(size);
  k2host::Fsa host_dest_fsa = creator.GetHostFsa();
  int32_t *arc_map_data = nullptr;
  if (arc_map != nullptr) {
    *arc_map = Array1<int32_t>(src.Context(), size.size2);
    arc_map_data = arc_map->Data();
  }
  bool ans = sorter.GetOutput(&host_dest_fsa, arc_map_data);
  *dest = creator.GetFsa();
  return ans;
}

bool Intersect(FsaOrVec &a_fsas, int32_t properties_a, FsaOrVec &b_fsas,
               int32_t properties_b, bool treat_epsilons_specially, FsaVec *out,
               Array1<int32_t> *arc_map_a, Array1<int32_t> *arc_map_b) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK(a_fsas.NumAxes() >= 2 && a_fsas.NumAxes() <= 3);
  K2_CHECK(b_fsas.NumAxes() >= 2 && b_fsas.NumAxes() <= 3);
  ContextPtr c = a_fsas.Context();
  K2_CHECK_EQ(c->GetDeviceType(), kCpu);
  if (a_fsas.NumAxes() == 2) {
    FsaVec a_fsas_vec = FsaToFsaVec(a_fsas);
    return Intersect(a_fsas_vec, properties_a, b_fsas, properties_b,
                     treat_epsilons_specially, out, arc_map_a, arc_map_b);
  }
  if (b_fsas.NumAxes() == 2) {
    FsaVec b_fsas_vec = FsaToFsaVec(b_fsas);
    return Intersect(a_fsas, properties_a, b_fsas_vec, properties_b,
                     treat_epsilons_specially, out, arc_map_a, arc_map_b);
  }

  int32_t num_fsas_a = a_fsas.Dim0(), num_fsas_b = b_fsas.Dim0();
  K2_CHECK_GT(num_fsas_a, 0);
  K2_CHECK_GT(num_fsas_b, 0);
  int32_t stride_a = 1, stride_b = 1;
  if (num_fsas_a != num_fsas_b) {
    if (num_fsas_a == 1) {
      stride_a = 0;
    } else if (num_fsas_b == 1) {
      stride_b = 0;
    } else {
      K2_CHECK_EQ(num_fsas_a, num_fsas_b);
    }
    // the check on the previous line will fail.
  }
  if (properties_a < 0) {
    Array1<int32_t> properties_a_out(c, num_fsas_a);
    GetFsaVecBasicProperties(a_fsas, &properties_a_out, &properties_a);
  }
  if (properties_b < 0) {
    Array1<int32_t> properties_b_out(c, num_fsas_b);
    GetFsaVecBasicProperties(b_fsas, &properties_b_out, &properties_b);
  }
  bool arc_sorted = (properties_a & kFsaPropertiesArcSorted) &&
                    (properties_b & kFsaPropertiesArcSorted);
  K2_CHECK(arc_sorted) << "Both a_fsas and b_fsas should be arc-sorted";
  int32_t num_fsas = std::max(num_fsas_a, num_fsas_b);

  std::vector<std::unique_ptr<k2host::Intersection>> intersections(num_fsas);
  std::vector<k2host::Array2Size<int32_t>> sizes(num_fsas);
  for (int32_t i = 0; i < num_fsas; ++i) {
    k2host::Fsa host_fsa_a = FsaVecToHostFsa(a_fsas, i * stride_a),
                host_fsa_b = FsaVecToHostFsa(b_fsas, i * stride_b);
    intersections[i] = std::make_unique<k2host::Intersection>(
        host_fsa_a, host_fsa_b, treat_epsilons_specially, false);
    intersections[i]->GetSizes(&(sizes[i]));
  }
  FsaVecCreator creator(sizes);
  int32_t num_arcs = creator.NumArcs();

  if (arc_map_a) *arc_map_a = Array1<int32_t>(c, num_arcs);
  if (arc_map_b) *arc_map_b = Array1<int32_t>(c, num_arcs);

  // the following few lines will allow us to add suitable offsets to the
  // `arc_map`.
  Array1<int32_t> a_fsas_row_splits12 =
                      a_fsas.RowSplits(2)[a_fsas.RowSplits(1)],
                  b_fsas_row_splits12 =
                      b_fsas.RowSplits(2)[b_fsas.RowSplits(1)];
  const int32_t *a_fsas_row_splits12_data = a_fsas_row_splits12.Data(),
                *b_fsas_row_splits12_data = b_fsas_row_splits12.Data();

  bool ok = true;
  for (int32_t i = 0; i < num_fsas; ++i) {
    k2host::Fsa host_fsa_out = creator.GetHostFsa(i);
    int32_t arc_offset = creator.GetArcOffsetFor(i);
    int32_t *this_arc_map_a =
                (arc_map_a ? arc_map_a->Data() + arc_offset : nullptr),
            *this_arc_map_b =
                (arc_map_b ? arc_map_b->Data() + arc_offset : nullptr);
    bool ans = intersections[i]->GetOutput(&host_fsa_out, this_arc_map_a,
                                           this_arc_map_b);
    ok = ok && ans;
    int32_t this_num_arcs = creator.GetArcOffsetFor(i + 1) - arc_offset;
    if (arc_map_a) {
      int32_t arc_offset_a = a_fsas_row_splits12_data[i * stride_a];
      for (int32_t i = 0; i < this_num_arcs; i++)
        this_arc_map_a[i] += arc_offset_a;
    }
    if (arc_map_b) {
      int32_t arc_offset_b = b_fsas_row_splits12_data[i * stride_b];
      for (int32_t i = 0; i < this_num_arcs; i++)
        this_arc_map_b[i] += arc_offset_b;
    }
  }
  *out = creator.GetFsaVec();
  return ok;
}

// Will be used in RemoveEpsilonHost and Determinize below to process FsaVec
// input recursively.
void RecursionWrapper(void (*f)(FsaOrVec &, FsaOrVec *, Ragged<int32_t> *),
                      FsaOrVec &src, FsaOrVec *dest,
                      Ragged<int32_t> *arc_deriv) {
  NVTX_RANGE(K2_FUNC);
  // src is actually an FsaVec.  Just recurse for now.
  K2_CHECK_EQ(src.NumAxes(), 3);
  int32_t num_fsas = src.shape.Dim0();
  std::vector<Fsa> srcs(num_fsas), dests(num_fsas);
  std::vector<Ragged<int32_t>> arc_derivs(num_fsas);
  int32_t tot_num_arcs = 0;
  for (int32_t i = 0; i < num_fsas; ++i) {
    srcs[i] = src.Index(0, i);
    f(srcs[i], &(dests[i]), arc_deriv != nullptr ? &(arc_derivs[i]) : nullptr);
    if (arc_deriv != nullptr) {
      // convert arc indexes in arc_derivs from idx2 to idx012
      Array1<int32_t> &values = arc_derivs[i].values;
      values = Plus(values, tot_num_arcs);
      tot_num_arcs += srcs[i].NumElements();
    }
  }
  *dest = Stack(0, num_fsas, dests.data());
  if (arc_deriv != nullptr) *arc_deriv = Append(0, num_fsas, arc_derivs.data());
}

void RemoveEpsilonHost(FsaOrVec &src, FsaOrVec *dest,
                       Ragged<int32_t> *arc_derivs /*=nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  int32_t num_axes = src.NumAxes();
  if (num_axes < 2 || num_axes > 3) {
    K2_LOG(FATAL) << "Input has bad num-axes " << num_axes;
  } else if (num_axes == 3) {
    return RecursionWrapper(RemoveEpsilonHost, src, dest, arc_derivs);
  }
  k2host::Fsa host_fsa = FsaToHostFsa(src);
  int32_t num_states = host_fsa.NumStates();
  K2_CHECK_EQ(num_states, src.Dim0());
  std::vector<double> max_forward_weights(num_states);
  std::vector<double> max_backward_weights(num_states);
  k2host::WfsaWithFbWeights max_wfsa(host_fsa, k2host::kMaxWeight,
                                     max_forward_weights.data(),
                                     max_backward_weights.data());
  // pass infinity as beam since we don't do pruning here.
  float beam = std::numeric_limits<float>::infinity();
  k2host::EpsilonsRemoverPrunedMax eps_remover(max_wfsa, beam);
  k2host::Array2Size<int32_t> fsa_size, arc_derivs_size;
  eps_remover.GetSizes(&fsa_size, &arc_derivs_size);
  FsaCreator fsa_creator(fsa_size);
  k2host::Fsa host_dest_fsa = fsa_creator.GetHostFsa();
  K2_STATIC_ASSERT(
      (std::is_same<k2host::MaxTracebackState::DerivType, int32_t>::value));
  Ragged2Creator<int32_t> ragged_creator(arc_derivs_size);
  k2host::Array2<int32_t *, int32_t> host_arc_derivs =
      ragged_creator.GetHostArray2();
  eps_remover.GetOutput(&host_dest_fsa, &host_arc_derivs);
  *dest = fsa_creator.GetFsa();
  if (arc_derivs != nullptr) *arc_derivs = ragged_creator.GetRagged2();
}


void RemoveEpsilon(FsaOrVec &src, int32_t properties,
                   FsaOrVec *dest,
                   Ragged<int32_t> *arc_derivs) {
  if ((properties & kFsaPropertiesTopSortedAndAcyclic) != 0 &&
      src.Context()->GetDeviceType() == kCpu) {
    // Host version of the algorithm
    RemoveEpsilonHost(src, dest, arc_derivs);
  } else {
    RemoveEpsilonDevice(src, dest, arc_derivs);
  }
}


void RemoveEpsilonAndAddSelfLoops(FsaOrVec &src, int32_t properties,
                                  FsaOrVec *dest,
                                  Ragged<int32_t> *arc_derivs) {
  Ragged<int32_t> arc_derivs1;

  FsaOrVec temp;
  RemoveEpsilon(src, properties, &temp,
                (arc_derivs != nullptr ? &arc_derivs1 : nullptr));

  Array1<int32_t> arc_derivs2;
  AddEpsilonSelfLoops(temp, dest,
                      (arc_derivs != nullptr ? &arc_derivs2 : nullptr));

  if (arc_derivs != nullptr) {
    *arc_derivs = Index(arc_derivs1, 0, arc_derivs2, nullptr);
  }
}




void Determinize(FsaOrVec &src, FsaOrVec *dest,
                 Ragged<int32_t> *arc_derivs /*=nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  int32_t num_axes = src.NumAxes();
  if (num_axes < 2 || num_axes > 3) {
    K2_LOG(FATAL) << "Input has bad num-axes " << num_axes;
  } else if (num_axes == 3) {
    return RecursionWrapper(Determinize, src, dest, arc_derivs);
  }
  k2host::Fsa host_fsa = FsaToHostFsa(src);
  int32_t num_states = host_fsa.NumStates();
  K2_CHECK_EQ(num_states, src.Dim0());
  int32_t max_step = -1;  // no limit
  k2host::DeterminizerMax determinizer(host_fsa, max_step);
  k2host::Array2Size<int32_t> fsa_size, arc_derivs_size;
  determinizer.GetSizes(&fsa_size, &arc_derivs_size);
  FsaCreator fsa_creator(fsa_size);
  k2host::Fsa host_dest_fsa = fsa_creator.GetHostFsa();
  K2_STATIC_ASSERT(
      (std::is_same<k2host::MaxTracebackState::DerivType, int32_t>::value));
  Ragged2Creator<int32_t> ragged_creator(arc_derivs_size);
  k2host::Array2<int32_t *, int32_t> host_arc_derivs =
      ragged_creator.GetHostArray2();
  determinizer.GetOutput(&host_dest_fsa, &host_arc_derivs);
  *dest = fsa_creator.GetFsa();
  if (arc_derivs != nullptr) *arc_derivs = ragged_creator.GetRagged2();
}

Fsa LinearFsa(const Array1<int32_t> &symbols) {
  NVTX_RANGE(K2_FUNC);
  ContextPtr &c = symbols.Context();
  int32_t n = symbols.Dim(), num_states = n + 2, num_arcs = n + 1;
  Array1<int32_t> row_splits1 = Range(c, num_states + 1, 0),
                  row_ids1 = Range(c, num_arcs, 0);
  int32_t *row_splits1_data = row_splits1.Data();
  Array1<Arc> arcs(c, num_arcs);
  Arc *arcs_data = arcs.Data();
  const int32_t *symbols_data = symbols.Data();
  K2_EVAL(
      c, num_arcs, lambda_set_arcs, (int32_t arc_idx01)->void {
        int32_t src_state = arc_idx01, dest_state = arc_idx01 + 1,
                // -1 == kFinalSymbol
            symbol = (arc_idx01 < n ? symbols_data[arc_idx01] : -1);
        if (arc_idx01 < n) K2_CHECK_NE(symbol, -1);
        float score = 0.0;
        arcs_data[arc_idx01] = Arc(src_state, dest_state, symbol, score);
        // the final state has no leaving arcs.
        if (arc_idx01 == 0) row_splits1_data[num_states] = num_arcs;
      });
  return Ragged<Arc>(RaggedShape2(&row_splits1, &row_ids1, num_arcs), arcs);
}

FsaVec LinearFsas(const Ragged<int32_t> &symbols) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(symbols.NumAxes(), 2);
  ContextPtr &c = symbols.Context();

  // if there are n symbols, there are n+2 states and n+1 arcs.
  RaggedShape states_shape = ChangeSublistSize(symbols.shape, 2);

  int32_t num_states = states_shape.NumElements(),
          num_arcs = symbols.NumElements() + symbols.Dim0();

  // row_splits2 maps from state_idx01 to arc_idx012; row_ids2 does the reverse.
  // We'll set them in the lambda below.
  Array1<int32_t> row_splits2(c, num_states + 1), row_ids2(c, num_arcs);

  int32_t *row_ids2_data = row_ids2.Data(),
          *row_splits2_data = row_splits2.Data();
  const int32_t *row_ids1_data = states_shape.RowIds(1).Data(),
                *row_splits1_data = states_shape.RowSplits(1).Data(),
                *symbols_data = symbols.values.Data();
  Array1<Arc> arcs(c, num_arcs);
  Arc *arcs_data = arcs.Data();
  K2_EVAL(
      c, num_states, lambda, (int32_t state_idx01)->void {
        int32_t fsa_idx0 = row_ids1_data[state_idx01],
                state_idx0x = row_splits1_data[fsa_idx0],
                next_state_idx0x = row_splits1_data[fsa_idx0 + 1],
                idx1 = state_idx01 - state_idx0x;

        // the following works because each FSA has one fewer arcs than states.
        int32_t arc_idx0xx = state_idx0x - fsa_idx0,
                next_arc_idx0xx = next_state_idx0x - (fsa_idx0 + 1),
                // the following may look a bit wrong.. here, the idx1 is the
                // same as the idx12 if the arc exists, because each state has
                // one arc leaving it (except the last state).
            arc_idx012 = arc_idx0xx + idx1;
        // the following works because each FSA has one fewer symbols than arcs
        // (however it doesn't work for the last arc of each FSA; we check
        // below.)
        int32_t symbol_idx01 = arc_idx012 - fsa_idx0;
        if (arc_idx012 < next_arc_idx0xx) {
          int32_t src_state = idx1, dest_state = idx1 + 1,
                  symbol = (arc_idx012 + 1 < next_arc_idx0xx
                                ? symbols_data[symbol_idx01]
                                : -1);  // kFinalSymbol
          float score = 0.0;
          arcs_data[arc_idx012] = Arc(src_state, dest_state, symbol, score);
          row_ids2_data[arc_idx012] = state_idx01;
        } else {
          // The following ensures that the last element of row_splits1_data
          // (i.e. row_splits1[num_states]) is set to num_arcs.  It also writes
          // something unnecessary for the last state of each FSA but the last
          // one, which will cause 2 threads to write the same item to the same
          // location. Note that there is no arc with index `arc_idx01`, if you
          // reach here.
          row_splits2_data[state_idx01 + 1] = arc_idx012;
        }
        row_splits2_data[state_idx01] = arc_idx012;
      });

  return Ragged<Arc>(
      RaggedShape3(&states_shape.RowSplits(1), &states_shape.RowIds(1),
                   num_states, &row_splits2, &row_ids2, num_arcs),
      arcs);
}

void ArcSort(Fsa *fsa) {
  NVTX_RANGE(K2_FUNC);
  if (fsa->NumAxes() < 2) return;  // it is empty
  SortSublists<Arc>(fsa);
}

void ArcSort(Fsa &src, Fsa *dest, Array1<int32_t> *arc_map /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  if (!src.values.IsValid()) return;

  if (arc_map != nullptr)
    *arc_map = Array1<int32_t>(src.Context(), src.NumElements());

  Fsa tmp(src.shape, src.values.Clone());
  SortSublists<Arc>(&tmp, arc_map);
  *dest = tmp;
}

// TODO(fangjun): use the following method suggested by Dan
//
// ... incidentally, it's possible to further optimize this so the run
// time is less than linear, by using methods similar to what I use
// in GetStateBatches(); imagine computing a table that instead of
// the best traceback, is the best 2-step traceback; and then the 4-step
// traceback, and so on. There's no need for this right now, since the
// forward-pass algorithm is already at least linear-time in the length
// of this path. But we can consider it for the future.
Ragged<int32_t> ShortestPath(FsaVec &fsas,
                             const Array1<int32_t> &entering_arcs) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  const int32_t *entering_arcs_data = entering_arcs.Data();
  const Arc *arcs_data = fsas.values.Data();
  int32_t num_fsas = fsas.Dim0();
  int32_t num_states = fsas.TotSize(1);
  ContextPtr &context = fsas.Context();

  // allocate an extra element for ExclusiveSum
  Array1<int32_t> num_best_arcs_per_fsa(context, num_fsas + 1);
  int32_t *num_best_arcs_per_fsa_data = num_best_arcs_per_fsa.Data();
  const int32_t *row_splits1_data = fsas.RowSplits(1).Data();

  // -1 represents an invalid arc_index.
  // This extra array avoids an extra iteration over `entering_arcs`.
  Array1<int32_t> state_best_arc_index_array(context, num_states, -1);
  int32_t *state_best_arc_index_array_data = state_best_arc_index_array.Data();

  K2_EVAL(
      context, num_fsas, lambda_set_num_best_arcs, (int32_t fsas_idx0) {
        int32_t state_idx01 = row_splits1_data[fsas_idx0];
        int32_t state_idx01_next = row_splits1_data[fsas_idx0 + 1];

        if (state_idx01_next == state_idx01) {
          // this fsa is empty, so there is no best path available
          num_best_arcs_per_fsa_data[fsas_idx0] = 0;
          return;
        }

        int32_t final_state_idx01 = state_idx01_next - 1;
        int32_t cur_state = final_state_idx01;
        int32_t cur_index = entering_arcs_data[cur_state];
        int32_t num_arcs = 0;
        int32_t *p = state_best_arc_index_array_data + final_state_idx01;
        while (cur_index != -1) {
          *p = cur_index;
          --p;

          cur_state = arcs_data[cur_index].src_state + state_idx01;
          cur_index = entering_arcs_data[cur_state];
          ++num_arcs;
        }
        num_best_arcs_per_fsa_data[fsas_idx0] = num_arcs;
      });
  ExclusiveSum(num_best_arcs_per_fsa, &num_best_arcs_per_fsa);

  RaggedShape shape = RaggedShape2(&num_best_arcs_per_fsa, nullptr, -1);
  const int32_t *shape_row_splits1_data = shape.RowSplits(1).Data();
  const int32_t *shape_row_ids1_data = shape.RowIds(1).Data();

  const int32_t *ans_row_splits_data = shape.RowSplits(1).Data();
  Array1<int32_t> best_path_arc_indexes(context, shape.NumElements());
  int32_t *best_path_arc_indexes_data = best_path_arc_indexes.Data();

  K2_EVAL(
      context, shape.NumElements(), lambda_set_best_arcs, (int32_t ans_idx01) {
        int32_t fsa_idx0 = shape_row_ids1_data[ans_idx01];
        int32_t ans_idx0x = shape_row_splits1_data[fsa_idx0];
        int32_t ans_idx1 = ans_idx01 - ans_idx0x;

        int32_t num_arcs_this_fsa = num_best_arcs_per_fsa_data[fsa_idx0 + 1] -
                                    num_best_arcs_per_fsa_data[fsa_idx0];
        if (num_arcs_this_fsa == 0) return;

        int32_t final_state_idx01_this_fsa = row_splits1_data[fsa_idx0 + 1] - 1;

        const int32_t *p_start = state_best_arc_index_array_data +
                                 final_state_idx01_this_fsa -
                                 num_arcs_this_fsa + 1;

        best_path_arc_indexes_data[ans_idx01] = p_start[ans_idx1];
      });

  Ragged<int32_t> ans(shape, best_path_arc_indexes);
  return ans;
}

void AddEpsilonSelfLoops(FsaOrVec &src, FsaOrVec *dest,
                         Array1<int32_t> *arc_map /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  ContextPtr &c = src.Context();
  const int32_t *old_row_splits1_data = src.RowSplits(1).Data(),
                *old_row_ids1_data = src.RowIds(1).Data();
  const Arc *old_arcs_data = src.values.Data();
  if (src.NumAxes() == 2) {
    int32_t num_states = src.Dim0();
    if (num_states < 2) {
      K2_CHECK_EQ(num_states, 0);
      *dest = src;
      if (arc_map != nullptr) *arc_map = Array1<int32_t>(c, 0);
      return;
    }

    int32_t old_num_arcs = src.TotSize(1),
            new_num_arcs = old_num_arcs + (num_states - 1);
    Array1<int32_t> new_row_splits(c, num_states + 1),
        new_row_ids(c, new_num_arcs);
    Array1<Arc> new_arcs(c, new_num_arcs);
    int32_t *new_row_splits1_data = new_row_splits.Data(),
            *new_row_ids1_data = new_row_ids.Data();
    Arc *new_arcs_data = new_arcs.Data();
    int32_t *arc_map_data = nullptr;
    if (arc_map) {
      *arc_map = Array1<int32_t>(c, new_num_arcs);
      arc_map_data = arc_map->Data();
    }
    ParallelRunner pr(c);
    {
      With w(pr.NewStream());
      K2_EVAL(
          c, old_num_arcs, lambda_copy_data, (int32_t arc_idx01)->void {
            int32_t state_idx0 = old_row_ids1_data[arc_idx01],
                    new_arc_idx01 = arc_idx01 + 1 + state_idx0;
            // the "+1" above is because we put the self-loop first.
            new_row_ids1_data[new_arc_idx01] = state_idx0;
            new_arcs_data[new_arc_idx01] = old_arcs_data[arc_idx01];
            if (arc_map_data) arc_map_data[new_arc_idx01] = arc_idx01;
          });
    }
    {
      With w(pr.NewStream());
      K2_EVAL(
          c, num_states, lambda_set_new_data, (int32_t state_idx0)->void {
            int32_t old_arc_idx0x = old_row_splits1_data[state_idx0],
                    new_arc_idx0x = old_arc_idx0x + state_idx0;
            new_row_splits1_data[state_idx0] = new_arc_idx0x;
            if (state_idx0 + 1 < num_states) {        // not final-state
              int32_t new_arc_idx01 = new_arc_idx0x;  // the 1st arc is the loop
              new_row_ids1_data[new_arc_idx01] = state_idx0;
              new_arcs_data[new_arc_idx01] =
                  Arc(state_idx0, state_idx0, 0, 0.0);
              if (arc_map_data) arc_map_data[new_arc_idx01] = -1;
            } else {
              // Note: if num_states was zero we would have returned above, so
              // we don't have to worry about empty FSAs.
              new_row_splits1_data[num_states] = new_arc_idx0x;
            }
          });
    }
    pr.Finish();
    *dest = Ragged<Arc>(
        RaggedShape2(&new_row_splits, &new_row_ids, new_num_arcs), new_arcs);
  } else {
    K2_CHECK_EQ(src.NumAxes(), 3);
    // Get a vector saying, for each FSA, whether it's nonempty.
    int32_t num_fsas = src.Dim0(), num_states = src.TotSize(1),
            old_num_arcs = src.TotSize(2);
    if (num_states == 0) {
      *dest = src;
      if (arc_map) *arc_map = Array1<int32_t>(c, 0);
      return;
    }
    Array1<int32_t> fsa_nonempty(c, num_fsas + 1);
    int32_t *fsa_nonempty_data = fsa_nonempty.Data();
    K2_EVAL(
        c, num_fsas, lambda_set_fsa_nonempty, (int32_t fsa_idx0)->void {
          fsa_nonempty_data[fsa_idx0] = (old_row_splits1_data[fsa_idx0 + 1] >
                                         old_row_splits1_data[fsa_idx0]);
        });
    ExclusiveSum(fsa_nonempty, &fsa_nonempty);
    const int32_t *old_row_splits2_data = src.RowSplits(2).Data(),
                  *old_row_ids2_data = src.RowIds(2).Data();
    int32_t num_nonempty_fsas = fsa_nonempty.Back(),
            new_num_arcs = old_num_arcs + num_states - num_nonempty_fsas;
    // we subtract `num_nonempty_fsas` because final-states don't get a
    // self-loop.

    Array1<int32_t> new_row_splits2(c, num_states + 1),
        new_row_ids2(c, new_num_arcs);
    Array1<Arc> new_arcs(c, new_num_arcs);
    // fsa_idx0_mod_data maps from fsa_idx0 to a modified fsa_idx0 that
    // "doesn't count" FSAs with zero states.
    const int32_t *fsa_idx0_mod_data = fsa_nonempty_data;
    int32_t *new_row_splits2_data = new_row_splits2.Data(),
            *new_row_ids2_data = new_row_ids2.Data();
    Arc *new_arcs_data = new_arcs.Data();
    int32_t *arc_map_data = nullptr;
    if (arc_map) {
      *arc_map = Array1<int32_t>(c, new_num_arcs);
      arc_map_data = arc_map->Data();
    }
    ParallelRunner pr(c);
    {
      With w(pr.NewStream());
      K2_EVAL(
          c, old_num_arcs, lambda_copy_data, (int32_t arc_idx012)->void {
            int32_t state_idx01 = old_row_ids2_data[arc_idx012],
                    fsa_idx0 = old_row_ids1_data[state_idx01],
                    fsa_idx0_mod = fsa_idx0_mod_data[fsa_idx0],
                    new_arc_idx012 =
                        arc_idx012 + 1 + state_idx01 - fsa_idx0_mod;
            // The "+1" above is because we put the self-loop first.  The
            // "-fsa_idx0_mod" is because final-states don't get a self-loop.
            new_row_ids2_data[new_arc_idx012] = state_idx01;
            new_arcs_data[new_arc_idx012] = old_arcs_data[arc_idx012];
            if (arc_map_data) arc_map_data[new_arc_idx012] = arc_idx012;
          });
    }
    {
      With w(pr.NewStream());
      K2_EVAL(
          c, num_states, lambda_set_new_data, (int32_t state_idx01)->void {
            int32_t fsa_idx0 = old_row_ids1_data[state_idx01],
                    fsa_idx0_mod = fsa_idx0_mod_data[fsa_idx0],
                    state_idx0x = old_row_splits1_data[fsa_idx0],
                    next_state_idx0x = old_row_splits1_data[fsa_idx0 + 1],
                    old_arc_idx01x = old_row_splits2_data[state_idx01];
            // Below the "+ state_idx01" is because each state gets a self-loop,
            // and the "- fsa_idx0_mod" is because final-states don't get a
            // self-loop.
            int32_t new_arc_idx01x =
                old_arc_idx01x + state_idx01 - fsa_idx0_mod;
            // The self-loop arc is the first arc:
            int32_t new_arc_idx012 = new_arc_idx01x;
            new_row_splits2_data[state_idx01] = new_arc_idx01x;
            if (state_idx01 + 1 < next_state_idx0x) {  // not final-state
              new_row_ids2_data[new_arc_idx012] = state_idx01;
              int32_t state_idx1 = state_idx01 - state_idx0x;
              new_arcs_data[new_arc_idx012] =
                  Arc(state_idx1, state_idx1, 0, 0.0);
              if (arc_map_data) arc_map_data[new_arc_idx012] = -1;
            } else if (state_idx01 + 1 == num_states) {
              // Note: if num_states was zero  we would have returned above, so
              // we dont have to worry about an empty FsaVec.
              new_row_splits2_data[num_states] = new_arc_idx01x;
            }
          });
    }
    pr.Finish();
    *dest =
        Ragged<Arc>(RaggedShape3(&src.RowSplits(1), &src.RowIds(1), num_states,
                                 &new_row_splits2, &new_row_ids2, new_num_arcs),
                    new_arcs);
  }
}

Fsa Union(FsaVec &fsas, Array1<int32_t> *arc_map /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(fsas.NumAxes(), 3);

  ContextPtr &context = fsas.Context();
  const int32_t *fsas_row_splits1_data = fsas.RowSplits(1).Data();
  const int32_t *fsas_row_splits2_data = fsas.RowSplits(2).Data();
  const int32_t *fsas_row_ids1_data = fsas.RowIds(1).Data();
  const int32_t *fsas_row_ids2_data = fsas.RowIds(2).Data();
  const Arc *arcs_data = fsas.values.Data();

  int32_t num_fsas = fsas.Dim0();
  int32_t num_states = fsas.TotSize(1);
  int32_t num_arcs = fsas.TotSize(2);

  // A new start state and a new final state are added (+2).
  // The final state of each fsa is removed (-num_fsas)
  int32_t num_out_states = num_states + 2 - num_fsas;
  int32_t out_final_state = num_out_states - 1;

  // For every fsa, a new arc is added from the new start state
  // to its original start state (+num_fsas)
  int32_t num_out_arcs = num_arcs + num_fsas;

  Array1<int32_t> out_row_ids(context, num_out_arcs);
  Array1<Arc> out_arcs(context, num_out_arcs);
  Array1<int32_t> tmp_arc_map(context, num_out_arcs, -1);
  int32_t *tmp_arc_map_data = tmp_arc_map.Data();

  int32_t *out_row_ids_data = out_row_ids.Data();
  Arc *out_arcs_data = out_arcs.Data();

  K2_EVAL(
      context, num_arcs, lambda_set_out, (int32_t fsas_arc_idx012) {
        int32_t fsas_state_idx01 = fsas_row_ids2_data[fsas_arc_idx012];
        int32_t fsas_idx0 = fsas_row_ids1_data[fsas_state_idx01];
        int32_t this_fsa_final_state_idx01 =
            fsas_row_splits1_data[fsas_idx0 + 1] - 1;

        K2_DCHECK_GT(this_fsa_final_state_idx01, fsas_state_idx01)
            << "We support only FSAs with at least two states at present";

        int32_t fsas_state_idx0x = fsas_row_splits1_data[fsas_idx0];
        int32_t fsas_state_idx1 = fsas_state_idx01 - fsas_state_idx0x;
        int32_t this_fsa_final_state_idx1 =
            this_fsa_final_state_idx01 - fsas_state_idx0x;

        int32_t fsas_arc_idx0xx = fsas_row_splits2_data[fsas_state_idx0x];

        // fsa0: +1 (a new start state)
        // fsa1: +0 (the final state of fsa0 is removed)
        // fsa2: -1 (the final state of fsa1 is removed)
        // fsa3: -2 (the final state of fsa2 is removed)
        int32_t state_offset = 1 - fsas_idx0;
        int32_t out_state_idx0 = fsas_state_idx01 + state_offset;

        int32_t out_arc_idx01 = fsas_arc_idx012 + num_fsas;
        out_row_ids_data[out_arc_idx01] = out_state_idx0;
        Arc arc = arcs_data[fsas_arc_idx012];

        K2_DCHECK_EQ(arc.src_state, fsas_state_idx1);

        if (arc.dest_state == this_fsa_final_state_idx1)
          arc.dest_state = out_final_state;
        else
          arc.dest_state = arc.dest_state - arc.src_state + out_state_idx0;

        arc.src_state = out_state_idx0;
        out_arcs_data[out_arc_idx01] = arc;
        tmp_arc_map_data[out_arc_idx01] = fsas_arc_idx012;

        if (fsas_arc_idx0xx == fsas_arc_idx012) {
          // add a new arc from the new start state to the start state
          // of this fsa
          //
          // WARNING: we cannot use fsas_state_idx01 here
          // since the start state may have no leaving arcs!
          Arc arc(0, fsas_state_idx0x + state_offset, 0, 0);
          out_arcs_data[fsas_idx0] = arc;
          out_row_ids_data[fsas_idx0] = 0;
        }
      });

  if (arc_map != nullptr) *arc_map = std::move(tmp_arc_map);
  Array1<int32_t> out_row_splits(context, num_out_states + 1);
  RowIdsToRowSplits(out_row_ids, &out_row_splits);
  RaggedShape shape = RaggedShape2(&out_row_splits, &out_row_ids, num_out_arcs);
  Fsa ans = Ragged<Arc>(shape, out_arcs);
  return ans;
}

Fsa Closure(Fsa &fsa, Array1<int32_t> *arc_map /* = nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(fsa.NumAxes(), 2) << "We support only a single FSA.";
  ContextPtr &c = fsa.Context();

  int32_t num_states = fsa.Dim0();
  if (num_states < 2) {
    K2_CHECK_EQ(num_states, 0)
        << "An empty fsa should contain no states at all";
    if (arc_map != nullptr) *arc_map = Array1<int32_t>(c, 0);
    return fsa;  // return itself if the input fsa is empty
  }

  const int32_t *fsa_row_splits_data = fsa.RowSplits(1).Data();
  const int32_t *fsa_row_ids_data = fsa.RowIds(1).Data();
  const Arc *fsa_arcs_data = fsa.values.Data();
  int32_t fsa_final_state = num_states - 1;

  int32_t num_out_states = num_states;

  // An arc from the start state to the final state with label == -1 is added.
  int32_t num_out_arcs = fsa.values.Dim() + 1;

  Array1<int32_t> out_row_ids(c, num_out_arcs);
  int32_t *out_row_ids_data = out_row_ids.Data();

  Array1<Arc> out_arcs(c, num_out_arcs);
  Arc *out_arcs_data = out_arcs.Data();

  Array1<int32_t> tmp_arc_map(c, num_out_arcs);
  int32_t *tmp_arc_map_data = tmp_arc_map.Data();

  K2_EVAL(
      c, fsa.values.Dim(), lambda_set_arcs, (int32_t fsa_arc_idx01) {
        int32_t fsa_state_idx0 = fsa_row_ids_data[fsa_arc_idx01];
        int32_t fsa_arc_idx0x = fsa_row_splits_data[fsa_state_idx0];
        int32_t fsa_arc_idx1 = fsa_arc_idx01 - fsa_arc_idx0x;
        int32_t this_state_num_arcs =
            fsa_row_splits_data[fsa_state_idx0 + 1] - fsa_arc_idx0x;

        Arc arc = fsa_arcs_data[fsa_arc_idx01];
        if (arc.dest_state == fsa_final_state) {
          // modify arcs entering the final state such that:
          //   - dest_state == 0
          //   - label == 0
          arc.dest_state = 0;
          K2_DCHECK_EQ(arc.label, -1);
          arc.label = 0;
        }

        int out_arc_idx01;
        if (arc.src_state > 0) {
          // this arc is not originated from the start state, so its index is
          // incremented
          out_arc_idx01 = fsa_arc_idx01 + 1;
        } else {
          out_arc_idx01 = fsa_arc_idx01;
          if (fsa_arc_idx1 == this_state_num_arcs - 1) {
            // This is the last arc of the original start state,
            // so we add a new arc just after it.
            Arc new_arc(0, fsa_final_state, -1, 0.0f);
            out_arcs_data[out_arc_idx01 + 1] = new_arc;
            out_row_ids_data[out_arc_idx01 + 1] = 0;
            tmp_arc_map_data[out_arc_idx01 + 1] = -1;
          }
        }

        // it may happen that the start state has no leaving arcs
        if (fsa_row_splits_data[1] == 0) {
          Arc new_arc(0, fsa_final_state, -1, 0.0f);
          out_arcs_data[0] = new_arc;
          out_row_ids_data[0] = 0;
          tmp_arc_map_data[0] = -1;
        }

        tmp_arc_map_data[out_arc_idx01] = fsa_arc_idx01;

        out_arcs_data[out_arc_idx01] = arc;
        out_row_ids_data[out_arc_idx01] = arc.src_state;
      });

  if (arc_map != nullptr) *arc_map = std::move(tmp_arc_map);

  Array1<int32_t> out_row_splits(c, num_out_states + 1);
  int32_t *out_row_splits_data = out_row_splits.Data();

  K2_EVAL(
      c, out_row_splits.Dim(), lambda_set_row_splits, (int32_t i) {
        if (i == 0)
          out_row_splits_data[i] = 0;
        else
          out_row_splits_data[i] = fsa_row_splits_data[i] + 1;
      });

  RaggedShape shape = RaggedShape2(&out_row_splits, &out_row_ids, num_out_arcs);
  Fsa ans = Ragged<Arc>(shape, out_arcs);
  return ans;
}

FsaOrVec ExpandArcs(FsaOrVec &fsas, RaggedShape &labels_shape,
                    Array1<int32_t> *fsas_arc_map /*=nullptr*/,
                    Array1<int32_t> *labels_arc_map /*=nullptr*/) {
  if (fsas.NumAxes() == 2) {
    FsaVec fsas_temp = FsaToFsaVec(fsas);
    return ExpandArcs(fsas_temp, labels_shape, fsas_arc_map, labels_arc_map)
        .RemoveAxis(0);
  }
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  K2_CHECK_EQ(labels_shape.NumAxes(), 2);
  K2_CHECK_EQ(fsas.NumElements(), labels_shape.Dim0());
  ContextPtr c = fsas.Context();
  K2_CHECK(c->IsCompatible(*labels_shape.Context()));

  RaggedShape state_to_arcs = GetLayer(fsas.shape, 1);

  // `state_to_foo` is a RaggedShape that, for each state in `fsas`, has a list
  // of length `tot_arcs + 1`.  Interpret this as: one element for the state
  // itself, then one for each arc leaving it.  This `foo` is an index that
  // corresponds to num-arcs plus one, but because it is really a placeholder
  // and we want to keep it distinct from other things, we call it `foo`.
  RaggedShape state_to_foo = ChangeSublistSize(state_to_arcs, 1);

  int32_t foo_size = state_to_foo.NumElements();

  // For each element of `state_to_foo`, `num_ostates_for` says how many states
  // there will be for this (state,foo) in the returned (output) FSA.  Here, the
  // idx0 is the state, the idx1 is foo.  If idx1 == 0 (interpret this as "the
  // state itself"), then `num_ostates_for[idx01] = 1`, meaning "keep the
  // original state".  Otherwise, idx1 - 1 represents an arc_idx2 [into `fsas`],
  // and we set `num_ostates_for[idx01] = max(0, seq_len-1)`, where seq_len is
  // the length of the sequence in `labels_shape` corresponding to this
  // arc-index.
  Array1<int32_t> num_ostates_for(c, foo_size + 1);
  int32_t *num_ostates_for_data = num_ostates_for.Data();

  const int32_t *labels_row_splits1_data = labels_shape.RowSplits(1).Data(),
                *fsas_row_splits2_data = fsas.RowSplits(2).Data(),
                *state_to_foo_row_splits1_data =
                    state_to_foo.RowSplits(1).Data(),
                *state_to_foo_row_ids1_data = state_to_foo.RowIds(1).Data();

  K2_EVAL(
      c, foo_size, lambda_set_num_ostates, (int32_t idx01)->void {
        // note: the idx01, idx0, idx0x are into `state_to_foo`.
        // This idx0 is a state-index into `fsas` (an idx01 w.r.t. `fsas`).
        int32_t idx0 = state_to_foo_row_ids1_data[idx01],
                idx0x = state_to_foo_row_splits1_data[idx0],
                idx1 = idx01 - idx0x;  // idx1 is `foo`.
        int32_t num_ostates;
        if (idx1 == 0) {
          num_ostates = 1;  // this is a copy of the original state.
        } else {
          int32_t fsas_arc_idx2 = idx1 - 1, fsas_state_idx01 = idx0,
                  fsas_arc_idx01x = fsas_row_splits2_data[fsas_state_idx01],
                  fsas_arc_idx012 = fsas_arc_idx01x + fsas_arc_idx2,
                  labels_shape_idx0 = fsas_arc_idx012,
                  labels_shape_idx0x =
                      labels_row_splits1_data[labels_shape_idx0],
                  labels_shape_idx0x_next =
                      labels_row_splits1_data[labels_shape_idx0 + 1],
                  labels_shape_len1 =
                      labels_shape_idx0x_next - labels_shape_idx0x;
          // A sequence of n symbols will require n-1 extra states to represent
          // it.
          num_ostates = max(labels_shape_len1 - 1, (int32_t)0);
        }
        num_ostates_for_data[idx01] = num_ostates;
      });
  ExclusiveSum(num_ostates_for, &num_ostates_for);
  Array1<int32_t> &foo_to_ostates_row_splits = num_ostates_for;
  RaggedShape foo_to_ostates =
      RaggedShape2(&foo_to_ostates_row_splits, nullptr, -1);

  // to_ostates_shape has 4 axes: [fsa_id][orig_state][foo][ostate]
  // where foo is a general-purpose index that ranges over the (num_arcs + 1) of
  // the original state.
  RaggedShape to_ostates_shape = ComposeRaggedShapes3(
      GetLayer(fsas.shape, 0), state_to_foo, foo_to_ostates);

  // Below, `tos` means `to_ostates_shape`.
  const int32_t *tos_row_splits1_data = to_ostates_shape.RowSplits(1).Data(),
                *tos_row_ids1_data = to_ostates_shape.RowIds(1).Data(),
                *tos_row_splits2_data = to_ostates_shape.RowSplits(2).Data(),
                *tos_row_ids2_data = to_ostates_shape.RowIds(2).Data(),
                *tos_row_splits3_data = to_ostates_shape.RowSplits(3).Data(),
                *tos_row_ids3_data = to_ostates_shape.RowIds(3).Data();

  // `num_oarcs` gives the number of arcs in the returned (output) FSA for each
  // `ostate` (i.e. leaving each state in the returned FSA).
  int32_t tot_ostates = to_ostates_shape.NumElements();
  Array1<int32_t> num_oarcs(c, tot_ostates + 1);
  int32_t *num_oarcs_data = num_oarcs.Data();
  K2_EVAL(
      c, tot_ostates, lambda_set_num_oarcs, (int32_t idx0123)->void {
        // All these indexes are into `to_ostates_shape`, indexed
        // `[fsa][state][foo][ostate].`
        int32_t idx012 = tos_row_ids3_data[idx0123],
                idx012x = tos_row_splits3_data[idx012],
                idx01 = tos_row_ids2_data[idx012],
                idx01x = tos_row_splits2_data[idx01],
                idx01x_next = tos_row_splits2_data[idx01 + 1],
                len2 = idx01x_next - idx01x, idx2 = idx012 - idx01x,
                idx3 = idx0123 - idx012x;
        int32_t num_arcs;
        if (idx2 == 0) {
          K2_CHECK_EQ(idx3, 0);
          // This ostate corresponds to the original state; it is not one of the
          // extra states added to support chains of arcs.
          // The original state had `orig_num_arcs` leaving it, which is the
          // number of `foo` indexes minus one.
          int32_t orig_num_arcs = len2 - 1;
          num_arcs = orig_num_arcs;
        } else {
          // All newly-created states have exactly one arc leaving them.
          num_arcs = 1;
        }
        num_oarcs_data[idx0123] = num_arcs;
      });
  ExclusiveSum(num_oarcs, &num_oarcs);
  Array1<int32_t> &ostate_to_oarcs_row_splits = num_oarcs;
  RaggedShape ostate_to_oarcs =
      RaggedShape2(&ostate_to_oarcs_row_splits, nullptr, -1);

  // `full_shape` has 5 axes: [fsa][orig_state][foo][ostate][oarc]
  RaggedShape full_shape =
      ComposeRaggedShapes(to_ostates_shape, ostate_to_oarcs);
  // for the lower-order row-splits and row-ids, use tot_row_{splits,idx}n_data
  const int32_t *full_row_splits4_data = full_shape.RowSplits(4).Data(),
                *full_row_ids4_data = full_shape.RowIds(4).Data();
  int32_t tot_oarcs = full_shape.NumElements();
  K2_CHECK_GE(tot_oarcs, fsas.NumElements());

  int32_t *fsas_arc_map_data = nullptr, *labels_arc_map_data = nullptr;
  if (fsas_arc_map) {
    *fsas_arc_map = Array1<int32_t>(c, tot_oarcs);
    fsas_arc_map_data = fsas_arc_map->Data();
  }
  if (labels_arc_map) {
    *labels_arc_map = Array1<int32_t>(c, tot_oarcs);
    labels_arc_map_data = labels_arc_map->Data();
  }
  Array1<Arc> oarcs(c, tot_oarcs);
  Arc *oarcs_data = oarcs.Data();
  const Arc *arcs_data = fsas.values.Data();

  K2_EVAL(
      c, tot_oarcs, lambda_set_arcs, (int32_t idx01234)->void {
        // All these indexes are into `full_shape`, indexed
        // `[fsa][state][foo][ostate][oarc].`
        int32_t idx0123 = full_row_ids4_data[idx01234],
                idx0123x = full_row_splits4_data[idx0123],
                idx4 = idx01234 - idx0123x, idx012 = tos_row_ids3_data[idx0123],
                idx012x = tos_row_splits3_data[idx012],
                idx3 = idx0123 - idx012x, idx01 = tos_row_ids2_data[idx012],
                idx01x = tos_row_splits2_data[idx01], idx2 = idx012 - idx01x,
                idx0 = tos_row_ids1_data[idx01],
                idx0x = tos_row_splits1_data[idx0],
                idx0xxx = tos_row_splits3_data[tos_row_splits2_data[idx0x]];

        int32_t fsa_idx01x = fsas_row_splits2_data[idx01];

        int32_t fsa_idx2;  // the idx2 (arc-index) into `fsas` of the input arc
                           // that's most relevant to us..
        int32_t seq_pos;  // seq_pos is our index into the sequence of arcs that
                          // we produce for each original arc
        if (idx2 == 0) {
          K2_CHECK_EQ(idx3, 0);
          fsa_idx2 = idx4;  // corresponds to foo=0, so idx3 will be 0; the idx4
                            // enumerates the arcs leaving it..
          seq_pos = 0;
        } else {
          // this is one of the extra `foo` indexes, one per arc in the input
          // FSA that leaves this state; each of those `foo` indexes has
          // (seq_len - 1) states in it (idx3=0,1..seq_len-1); and each state
          // has one arc leaving it (idx4==0).
          K2_CHECK_EQ(idx4, 0);
          fsa_idx2 = idx2 - 1;
          seq_pos = idx3 + 1;
        }
        int32_t fsa_idx012 = fsa_idx01x + fsa_idx2;  // index of the arc in
                                                     // source FSA FSA that
                                                     // we're expanding..
        Arc iarc = arcs_data[fsa_idx012];

        int32_t labels_idx0x = labels_row_splits1_data[fsa_idx012],
                labels_next_idx0x = labels_row_splits1_data[fsa_idx012 + 1],
                labels_len1 = labels_next_idx0x - labels_idx0x;
        // labels_len1 is length of label sequence for this arc
        K2_CHECK_LT(seq_pos, max(int32_t(1), labels_len1));

        int32_t dest_idx01 = idx0x + iarc.dest_state,  // original destination
                                                       // state-index
            orig_dest_idx0123 =
                tos_row_splits3_data[tos_row_splits2_data[dest_idx01]];

        Arc oarc;
        oarc.src_state = idx0123 - idx0xxx;
        // If this is the last arc in the sequence, the dest-state is the
        // original dest-state of the arc.  Otherwise the dest-state is one of
        // the new states that we created. The idx123 will be an idx1 after
        // removing axes.
        int32_t dest_idx123;
        if (seq_pos + 1 >= labels_len1) {  // last arc in sequence..
          dest_idx123 = orig_dest_idx0123 - idx0xxx;
        } else {
          int32_t dest_state_idx2 = fsa_idx2 + 1,  // index `foo` equals
                                                   // orig_arc_idx+1
              dest_state_idx3 = seq_pos,           // ostate index..
              dest_idx012 = idx01x + dest_state_idx2,
                  dest_idx012x = tos_row_splits3_data[dest_idx012],
                  dest_idx0123 = dest_idx012x + dest_state_idx3;
          dest_idx123 = dest_idx0123 - idx0xxx;
        }
        oarc.dest_state = dest_idx123;  // indexes 1,2,3 will be combined; in
                                        // the output FSA it will be an idx1.

        if (fsas_arc_map_data)
          fsas_arc_map_data[idx01234] = (seq_pos == 0 ? fsa_idx012 : -1);
        if (labels_arc_map_data)
          labels_arc_map_data[idx01234] =
              (seq_pos < labels_len1 ? labels_idx0x + seq_pos : -1);
        if (iarc.label != -1) {
          // normal case.. label goes on 1st arc in sequence
          oarc.label = (seq_pos == 0 ? iarc.label : 0);
        } else {
          // If the arc was to the final-state, we need to keep the label on the
          // last arc of the sequence to keep the output valid.  The following
          // would be "seq_pos + 1 == labels_len1 ? -1 : 0", but we make it ">="
          // not "=" to account for the case seq_pos=0, labels_len1 = 0.
          oarc.label = (seq_pos + 1 >= labels_len1 ? -1 : 0);
        }
        oarc.score = (seq_pos == 0 ? iarc.score : 0.0);
        oarcs_data[idx01234] = oarc;
      });

  // remove current axes 1 and 2... [after removing axis 1, old axis 2 becomes
  // axis 1, so remove axis 1 twice].
  RaggedShape temp = RemoveAxis(full_shape, 1);
  return FsaVec(RemoveAxis(temp, 1), oarcs);
}

void Invert(FsaOrVec &src, Ragged<int32_t> &src_aux_labels, FsaOrVec *dest,
            Ragged<int32_t> *dest_aux_labels,
            Array1<int32_t> *arc_map /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(src_aux_labels.NumAxes(), 2);
  K2_CHECK_EQ(src_aux_labels.Dim0(), src.NumElements());
  K2_CHECK(dest != nullptr && dest_aux_labels != nullptr);
  ContextPtr c = GetContext(src, src_aux_labels);
  if (src.NumAxes() == 2) {
    Fsa *srcs = &src;
    FsaVec src_vec = CreateFsaVec(1, &srcs), dest_vec;
    Invert(src_vec, src_aux_labels, &dest_vec, dest_aux_labels, arc_map);
    *dest = GetFsaVecElement(dest_vec, 0);
    return;
  }
  Array1<int32_t> src_arc_map, labels_arc_map;
  *dest = ExpandArcs(src, src_aux_labels.shape, &src_arc_map, &labels_arc_map);
  // swap labels and aux_labels
  int32_t dest_num_arcs = dest->NumElements();
  Arc *dest_arcs_data = dest->values.Data();
  const int32_t *labels_arc_map_data = labels_arc_map.Data(),
                *src_aux_labels_data = src_aux_labels.values.Data();
  Array1<int32_t> dest_aux_labels_row_splits(c, dest_num_arcs + 1);
  int32_t *dest_aux_labels_row_splits_data = dest_aux_labels_row_splits.Data();
  K2_EVAL(
      c, dest_num_arcs, lambda_set_dest_aux_labels_num,
      (int32_t dest_idx012)->void {
        Arc &dest_arc = dest_arcs_data[dest_idx012];
        // we'll remove epsilons in dest_aux_labels
        dest_aux_labels_row_splits_data[dest_idx012] =
            dest_arc.label == 0 ? 0 : 1;
      });
  ExclusiveSum(dest_aux_labels_row_splits.Arange(0, dest_num_arcs),
               &dest_aux_labels_row_splits);
  RaggedShape dest_aux_labels_shape =
      RaggedShape2(&dest_aux_labels_row_splits, nullptr, -1);
  Array1<int32_t> dest_aux_labels_values(c,
                                         dest_aux_labels_shape.NumElements());
  int32_t *dest_aux_labels_values_data = dest_aux_labels_values.Data();
  K2_EVAL(
      c, dest_num_arcs, lambda_set_dest_labels_and_aux_labels,
      (int32_t dest_idx012)->void {
        Arc &dest_arc = dest_arcs_data[dest_idx012];
        // swap label and aux_label
        if (dest_arc.label != 0) {
          int32_t dest_aux_labels_idx0x =
              dest_aux_labels_row_splits_data[dest_idx012];
          // every arc in dest has at most one aux_label (as the aux_label is
          // the label of src on this arc)
          dest_aux_labels_values_data[dest_aux_labels_idx0x] = dest_arc.label;
        }
        int32_t src_aux_labels_idx01 = labels_arc_map_data[dest_idx012];
        dest_arc.label = src_aux_labels_idx01 == -1
                             ? 0
                             : src_aux_labels_data[src_aux_labels_idx01];
      });
  *dest_aux_labels =
      Ragged<int32_t>(dest_aux_labels_shape, dest_aux_labels_values);
  if (arc_map != nullptr) *arc_map = src_arc_map;
}

// Will be used in InvertHost to process FsaVec input recursively.
void RecursionWrapperAuxLabels(void (*f)(FsaOrVec &, Ragged<int32_t> &,
                                         FsaOrVec *, Ragged<int32_t> *),
                               FsaOrVec &src, Ragged<int32_t> &src_aux_labels,
                               FsaOrVec *dest,
                               Ragged<int32_t> *dest_aux_labels) {
  NVTX_RANGE(K2_FUNC);
  // src is actually an FsaVec.  Just recurse for now.
  K2_CHECK_EQ(src.NumAxes(), 3);
  int32_t num_fsas = src.shape.Dim0();
  std::vector<Fsa> srcs(num_fsas), dests(num_fsas);
  std::vector<Ragged<int32_t>> src_aux_labels_vec(num_fsas),
      dest_aux_labels_vec(num_fsas);
  int32_t tot_num_arcs = 0;
  Array1<int32_t> src_aux_labels_row_splits = src_aux_labels.RowSplits(1),
                  src_aux_labels_values = src_aux_labels.values;
  for (int32_t i = 0; i < num_fsas; ++i) {
    srcs[i] = src.Index(0, i);
    int32_t cur_num_arcs = srcs[i].NumElements();
    // below block get aux_labels for srcs[i]
    // TODO(haowen): replace with Range op for ragged
    {
      Array1<int32_t> row_splits = src_aux_labels_row_splits.Arange(
          tot_num_arcs, tot_num_arcs + cur_num_arcs + 1);
      Array1<int32_t> values =
          src_aux_labels_values.Arange(row_splits[0], row_splits.Back());
      row_splits = Minus(row_splits, row_splits[0]);
      RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
      src_aux_labels_vec[i] = Ragged<int32_t>(shape, values);
    }
    f(srcs[i], src_aux_labels_vec[i], &(dests[i]), &(dest_aux_labels_vec[i]));
    tot_num_arcs += cur_num_arcs;
  }
  *dest = Stack(0, num_fsas, dests.data());
  *dest_aux_labels = Append(0, num_fsas, dest_aux_labels_vec.data());
}

void InvertHost(FsaOrVec &src, Ragged<int32_t> &src_aux_labels, FsaOrVec *dest,
                Ragged<int32_t> *dest_aux_labels) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(src_aux_labels.NumAxes(), 2);
  K2_CHECK_EQ(src_aux_labels.Dim0(), src.NumElements());
  K2_CHECK(dest != nullptr && dest_aux_labels != nullptr);
  int32_t num_axes = src.NumAxes();
  if (num_axes < 2 || num_axes > 3) {
    K2_LOG(FATAL) << "Input has bad num-axes " << num_axes;
  } else if (num_axes == 3) {
    return RecursionWrapperAuxLabels(InvertHost, src, src_aux_labels, dest,
                                     dest_aux_labels);
  }

  k2host::Fsa host_fsa = FsaToHostFsa(src);
  // k2host::AuxLabels is a k2host::Array2
  k2host::AuxLabels host_aux_labels(
      src_aux_labels.Dim0(), src_aux_labels.NumElements(),
      src_aux_labels.RowSplits(1).Data(), src_aux_labels.values.Data());
  k2host::FstInverter inverter(host_fsa, host_aux_labels);
  k2host::Array2Size<int32_t> fsa_size, aux_size;
  inverter.GetSizes(&fsa_size, &aux_size);
  FsaCreator fsa_creator(fsa_size);
  k2host::Fsa host_dest_fsa = fsa_creator.GetHostFsa();
  Ragged2Creator<int32_t> ragged_creator(aux_size);
  k2host::AuxLabels host_dest_aux_labels = ragged_creator.GetHostArray2();
  inverter.GetOutput(&host_dest_fsa, &host_dest_aux_labels);
  *dest = fsa_creator.GetFsa();
  *dest_aux_labels = ragged_creator.GetRagged2();
}

FsaOrVec RemoveEpsilonSelfLoops(FsaOrVec &src,
                                Array1<int32_t> *arc_map /* = nullptr */) {
  if (src.NumAxes() == 2) {
    FsaVec temp = FsaToFsaVec(src);
    return RemoveEpsilonSelfLoops(temp, arc_map).RemoveAxis(0);
  }
  K2_CHECK_EQ(src.NumAxes(), 3);

  ContextPtr &c = src.Context();
  int32_t num_arcs = src.NumElements();
  Renumbering renumber_lists(c, num_arcs);
  char *keep_list_data = renumber_lists.Keep().Data();

  const Arc *arcs_data = src.values.Data();
  K2_EVAL(
      c, num_arcs, lambda_set_keep, (int32_t i)->void {
        Arc arc = arcs_data[i];
        char keep;
        if (arc.label == 0 && arc.src_state == arc.dest_state) {
          // This arc is an epsilon self-loop, so it should be removed
          keep = 0;
        } else {
          keep = 1;
        }
        keep_list_data[i] = keep;
      });
  FsaVec ans = Index(src, 2, renumber_lists.New2Old(), arc_map);
  return ans;
}

}  // namespace k2
