#include "hip/hip_runtime.h"
/**
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey, Haowen Qiu,
 *                                                   Wei Kang)
 *                      Mobvoi Inc.        (authors: Fangjun Kuang)
 *
 * See LICENSE for clarification regarding multiple authors
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <limits>
#include <memory>
#include <type_traits>
#include <utility>
#include <vector>

#include "k2/csrc/array_ops.h"
#include "k2/csrc/fsa_algo.h"
#include "k2/csrc/fsa_utils.h"
#include "k2/csrc/host/aux_labels.h"
#include "k2/csrc/host/connect.h"
#include "k2/csrc/host/determinize.h"
#include "k2/csrc/host/intersect.h"
#include "k2/csrc/host/rmepsilon.h"
#include "k2/csrc/host/topsort.h"
#include "k2/csrc/host_shim.h"
#include "k2/csrc/macros.h"
#include "k2/csrc/rm_epsilon.h"


// this contains a subset of the algorithms in fsa_algo.h; currently it just
// contains one that are wrappings of the corresponding algorithms in
// host/.
namespace k2 {

bool RecursionWrapper(bool (*f)(Fsa &, Fsa *, Array1<int32_t> *), Fsa &src,
                      Fsa *dest, Array1<int32_t> *arc_map) {
  NVTX_RANGE(K2_FUNC);
  // src is actually an FsaVec.  Just recurse for now.
  int32_t num_fsas = src.shape.Dim0();
  std::vector<Fsa> srcs(num_fsas), dests(num_fsas);
  std::vector<Array1<int32_t>> arc_maps(num_fsas);
  int32_t tot_num_arcs = 0;
  for (int32_t i = 0; i < num_fsas; ++i) {
    srcs[i] = src.Index(0, i);
    // Recurse.
    if (!f(srcs[i], &(dests[i]),
           (arc_map != nullptr ? &(arc_maps[i]) : nullptr)))
      return false;
    if (arc_map != nullptr) {
      // convert arc indexes in arc_maps from idx2 to idx012
      arc_maps[i] = Plus(arc_maps[i], tot_num_arcs);
      tot_num_arcs += srcs[i].NumElements();
    }
  }
  *dest = Stack(0, num_fsas, dests.data());
  if (arc_map != nullptr)
    *arc_map = Cat(src.Context(), num_fsas, arc_maps.data());
  return true;
}

bool ConnectHost(Fsa &src, Fsa *dest, Array1<int32_t> *arc_map /*=nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  int32_t num_axes = src.NumAxes();
  if (num_axes < 2 || num_axes > 3) {
    K2_LOG(FATAL) << "Input has bad num-axes " << num_axes;
  } else if (num_axes == 3) {
    return RecursionWrapper(ConnectHost, src, dest, arc_map);
  }

  k2host::Fsa host_fsa = FsaToHostFsa(src);
  k2host::Connection c(host_fsa);
  k2host::Array2Size<int32_t> size;
  c.GetSizes(&size);
  FsaCreator creator(size);
  k2host::Fsa host_dest_fsa = creator.GetHostFsa();
  int32_t *arc_map_data = nullptr;
  if (arc_map != nullptr) {
    *arc_map = Array1<int32_t>(src.Context(), size.size2);
    arc_map_data = arc_map->Data();
  }
  bool ans = c.GetOutput(&host_dest_fsa, arc_map_data);
  *dest = creator.GetFsa();
  return ans;
}

bool TopSortHost(Fsa &src, Fsa *dest, Array1<int32_t> *arc_map /*=nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  int32_t num_axes = src.NumAxes();
  if (num_axes < 2 || num_axes > 3) {
    K2_LOG(FATAL) << "Input has bad num-axes " << num_axes;
  } else if (num_axes == 3) {
    return RecursionWrapper(TopSortHost, src, dest, arc_map);
  }

  k2host::Fsa host_fsa = FsaToHostFsa(src);
  k2host::TopSorter sorter(host_fsa);
  k2host::Array2Size<int32_t> size;
  sorter.GetSizes(&size);
  FsaCreator creator(size);
  k2host::Fsa host_dest_fsa = creator.GetHostFsa();
  int32_t *arc_map_data = nullptr;
  if (arc_map != nullptr) {
    *arc_map = Array1<int32_t>(src.Context(), size.size2);
    arc_map_data = arc_map->Data();
  }
  bool ans = sorter.GetOutput(&host_dest_fsa, arc_map_data);
  *dest = creator.GetFsa();
  return ans;
}

bool Intersect(FsaOrVec &a_fsas, int32_t properties_a, FsaOrVec &b_fsas,
               int32_t properties_b, bool treat_epsilons_specially, FsaVec *out,
               Array1<int32_t> *arc_map_a, Array1<int32_t> *arc_map_b) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK(a_fsas.NumAxes() >= 2 && a_fsas.NumAxes() <= 3);
  K2_CHECK(b_fsas.NumAxes() >= 2 && b_fsas.NumAxes() <= 3);
  ContextPtr c = a_fsas.Context();
  K2_CHECK_EQ(c->GetDeviceType(), kCpu);
  if (a_fsas.NumAxes() == 2) {
    FsaVec a_fsas_vec = FsaToFsaVec(a_fsas);
    return Intersect(a_fsas_vec, properties_a, b_fsas, properties_b,
                     treat_epsilons_specially, out, arc_map_a, arc_map_b);
  }
  if (b_fsas.NumAxes() == 2) {
    FsaVec b_fsas_vec = FsaToFsaVec(b_fsas);
    return Intersect(a_fsas, properties_a, b_fsas_vec, properties_b,
                     treat_epsilons_specially, out, arc_map_a, arc_map_b);
  }

  int32_t num_fsas_a = a_fsas.Dim0(), num_fsas_b = b_fsas.Dim0();
  K2_CHECK_GT(num_fsas_a, 0);
  K2_CHECK_GT(num_fsas_b, 0);
  int32_t stride_a = 1, stride_b = 1;
  if (num_fsas_a != num_fsas_b) {
    if (num_fsas_a == 1) {
      stride_a = 0;
    } else if (num_fsas_b == 1) {
      stride_b = 0;
    } else {
      K2_CHECK_EQ(num_fsas_a, num_fsas_b);
    }
    // the check on the previous line will fail.
  }
  if (properties_a < 0) {
    Array1<int32_t> properties_a_out(c, num_fsas_a);
    GetFsaVecBasicProperties(a_fsas, &properties_a_out, &properties_a);
  }
  if (properties_b < 0) {
    Array1<int32_t> properties_b_out(c, num_fsas_b);
    GetFsaVecBasicProperties(b_fsas, &properties_b_out, &properties_b);
  }
  bool arc_sorted = (properties_a & kFsaPropertiesArcSorted) &&
                    (properties_b & kFsaPropertiesArcSorted);
  K2_CHECK(arc_sorted) << "Both a_fsas and b_fsas should be arc-sorted";
  int32_t num_fsas = std::max(num_fsas_a, num_fsas_b);

  std::vector<std::unique_ptr<k2host::Intersection>> intersections(num_fsas);
  std::vector<k2host::Array2Size<int32_t>> sizes(num_fsas);
  for (int32_t i = 0; i < num_fsas; ++i) {
    k2host::Fsa host_fsa_a = FsaVecToHostFsa(a_fsas, i * stride_a),
                host_fsa_b = FsaVecToHostFsa(b_fsas, i * stride_b);
    intersections[i] = std::make_unique<k2host::Intersection>(
        host_fsa_a, host_fsa_b, treat_epsilons_specially, false);
    intersections[i]->GetSizes(&(sizes[i]));
  }
  FsaVecCreator creator(sizes);
  int32_t num_arcs = creator.NumArcs();

  if (arc_map_a) *arc_map_a = Array1<int32_t>(c, num_arcs);
  if (arc_map_b) *arc_map_b = Array1<int32_t>(c, num_arcs);

  // the following few lines will allow us to add suitable offsets to the
  // `arc_map`.
  Array1<int32_t> a_fsas_row_splits12 =
                      a_fsas.RowSplits(2)[a_fsas.RowSplits(1)],
                  b_fsas_row_splits12 =
                      b_fsas.RowSplits(2)[b_fsas.RowSplits(1)];
  const int32_t *a_fsas_row_splits12_data = a_fsas_row_splits12.Data(),
                *b_fsas_row_splits12_data = b_fsas_row_splits12.Data();

  bool ok = true;
  for (int32_t i = 0; i < num_fsas; ++i) {
    k2host::Fsa host_fsa_out = creator.GetHostFsa(i);
    int32_t arc_offset = creator.GetArcOffsetFor(i);
    int32_t *this_arc_map_a =
                (arc_map_a ? arc_map_a->Data() + arc_offset : nullptr),
            *this_arc_map_b =
                (arc_map_b ? arc_map_b->Data() + arc_offset : nullptr);
    bool ans = intersections[i]->GetOutput(&host_fsa_out, this_arc_map_a,
                                           this_arc_map_b);
    ok = ok && ans;
    int32_t this_num_arcs = creator.GetArcOffsetFor(i + 1) - arc_offset;
    if (arc_map_a) {
      int32_t arc_offset_a = a_fsas_row_splits12_data[i * stride_a];
      for (int32_t i = 0; i < this_num_arcs; i++)
        if (this_arc_map_a[i] != -1) this_arc_map_a[i] += arc_offset_a;
    }
    if (arc_map_b) {
      int32_t arc_offset_b = b_fsas_row_splits12_data[i * stride_b];
      for (int32_t i = 0; i < this_num_arcs; i++)
        if (this_arc_map_b[i] != -1) this_arc_map_b[i] += arc_offset_b;
    }
  }
  *out = creator.GetFsaVec();
  return ok;
}

// Will be used in RemoveEpsilonHost and Determinize below to process FsaVec
// input recursively.
void RecursionWrapper(void (*f)(FsaOrVec &, FsaOrVec *, Ragged<int32_t> *),
                      FsaOrVec &src, FsaOrVec *dest,
                      Ragged<int32_t> *arc_deriv) {
  NVTX_RANGE(K2_FUNC);
  // src is actually an FsaVec.  Just recurse for now.
  K2_CHECK_EQ(src.NumAxes(), 3);
  int32_t num_fsas = src.shape.Dim0();
  std::vector<Fsa> srcs(num_fsas), dests(num_fsas);
  std::vector<Ragged<int32_t>> arc_derivs(num_fsas);
  int32_t tot_num_arcs = 0;
  for (int32_t i = 0; i < num_fsas; ++i) {
    srcs[i] = src.Index(0, i);
    f(srcs[i], &(dests[i]), arc_deriv != nullptr ? &(arc_derivs[i]) : nullptr);
    if (arc_deriv != nullptr) {
      // convert arc indexes in arc_derivs from idx2 to idx012
      Array1<int32_t> &values = arc_derivs[i].values;
      values = Plus(values, tot_num_arcs);
      tot_num_arcs += srcs[i].NumElements();
    }
  }
  *dest = Stack(0, num_fsas, dests.data());
  if (arc_deriv != nullptr) *arc_deriv = Cat(0, num_fsas, arc_derivs.data());
}

void RemoveEpsilonHost(FsaOrVec &src, FsaOrVec *dest,
                       Ragged<int32_t> *arc_derivs /*=nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  int32_t num_axes = src.NumAxes();
  if (num_axes < 2 || num_axes > 3) {
    K2_LOG(FATAL) << "Input has bad num-axes " << num_axes;
  } else if (num_axes == 3) {
    return RecursionWrapper(RemoveEpsilonHost, src, dest, arc_derivs);
  }
  k2host::Fsa host_fsa = FsaToHostFsa(src);
  int32_t num_states = host_fsa.NumStates();
  K2_CHECK_EQ(num_states, src.Dim0());
  std::vector<double> max_forward_weights(num_states);
  std::vector<double> max_backward_weights(num_states);
  k2host::WfsaWithFbWeights max_wfsa(host_fsa, k2host::kMaxWeight,
                                     max_forward_weights.data(),
                                     max_backward_weights.data());
  // pass infinity as beam since we don't do pruning here.
  float beam = std::numeric_limits<float>::infinity();
  k2host::EpsilonsRemoverPrunedMax eps_remover(max_wfsa, beam);
  k2host::Array2Size<int32_t> fsa_size, arc_derivs_size;
  eps_remover.GetSizes(&fsa_size, &arc_derivs_size);
  FsaCreator fsa_creator(fsa_size);
  k2host::Fsa host_dest_fsa = fsa_creator.GetHostFsa();
  K2_STATIC_ASSERT(
      (std::is_same<k2host::MaxTracebackState::DerivType, int32_t>::value));
  Ragged2Creator<int32_t> ragged_creator(arc_derivs_size);
  k2host::Array2<int32_t *, int32_t> host_arc_derivs =
      ragged_creator.GetHostArray2();
  eps_remover.GetOutput(&host_dest_fsa, &host_arc_derivs);
  *dest = fsa_creator.GetFsa();
  if (arc_derivs != nullptr) *arc_derivs = ragged_creator.GetRagged2();
}


void RemoveEpsilon(FsaOrVec &src, int32_t properties,
                   FsaOrVec *dest,
                   Ragged<int32_t> *arc_derivs) {
  if ((properties & kFsaPropertiesTopSortedAndAcyclic) != 0 &&
      src.Context()->GetDeviceType() == kCpu) {
    // Host version of the algorithm
    RemoveEpsilonHost(src, dest, arc_derivs);
  } else {
    RemoveEpsilonDevice(src, dest, arc_derivs);
  }
}


void RemoveEpsilonAndAddSelfLoops(FsaOrVec &src, int32_t properties,
                                  FsaOrVec *dest,
                                  Ragged<int32_t> *arc_derivs) {
  NVTX_RANGE(K2_FUNC);
  Ragged<int32_t> arc_derivs1;

  FsaOrVec temp;
  RemoveEpsilon(src, properties, &temp,
                (arc_derivs != nullptr ? &arc_derivs1 : nullptr));

  Array1<int32_t> arc_derivs2;
  AddEpsilonSelfLoops(temp, dest,
                      (arc_derivs != nullptr ? &arc_derivs2 : nullptr));

  if (arc_derivs != nullptr) {
    *arc_derivs = Index(arc_derivs1, 0, arc_derivs2, nullptr);
  }
}


void Determinize(FsaOrVec &src,
                 DeterminizeWeightPushingType weight_pushing_type,
                 FsaOrVec *dest, Ragged<int32_t> *arc_derivs /*=nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  int32_t num_axes = src.NumAxes();
  if (num_axes < 2 || num_axes > 3) {
    K2_LOG(FATAL) << "Input has bad num-axes " << num_axes;
  } else if (num_axes == 3) {
    int32_t num_fsas = src.shape.Dim0();
    std::vector<Fsa> srcs(num_fsas), dests(num_fsas);
    std::vector<Ragged<int32_t>> derivs_vector(num_fsas);
    int32_t tot_num_arcs = 0;
    for (int32_t i = 0; i < num_fsas; ++i) {
      srcs[i] = src.Index(0, i);
      Determinize(srcs[i], weight_pushing_type, &(dests[i]),
                 arc_derivs != nullptr ? &(derivs_vector[i]) : nullptr);
      if (arc_derivs != nullptr) {
        // convert arc indexes in arc_derivs from idx2 to idx012
        Array1<int32_t> &values = arc_derivs[i].values;
        values = Plus(values, tot_num_arcs);
        tot_num_arcs += srcs[i].NumElements();
      }
    }
    *dest = Stack(0, num_fsas, dests.data());
    if (arc_derivs != nullptr) *arc_derivs = Cat(0, num_fsas,
                                                 derivs_vector.data());
    return;
  }
  k2host::Fsa host_fsa = FsaToHostFsa(src);
  int32_t num_states = host_fsa.NumStates();
  K2_CHECK_EQ(num_states, src.Dim0());
  int32_t max_step = -1;  // no limit
  k2host::FbWeightType host_weight_pushing_type =
      static_cast<k2host::FbWeightType>(static_cast<int>(weight_pushing_type));
  k2host::DeterminizerMax determinizer(host_fsa, max_step,
                                       host_weight_pushing_type);
  k2host::Array2Size<int32_t> fsa_size, arc_derivs_size;
  determinizer.GetSizes(&fsa_size, &arc_derivs_size);
  FsaCreator fsa_creator(fsa_size);
  k2host::Fsa host_dest_fsa = fsa_creator.GetHostFsa();
  K2_STATIC_ASSERT(
      (std::is_same<k2host::MaxTracebackState::DerivType, int32_t>::value));
  Ragged2Creator<int32_t> ragged_creator(arc_derivs_size);
  k2host::Array2<int32_t *, int32_t> host_arc_derivs =
      ragged_creator.GetHostArray2();
  determinizer.GetOutput(&host_dest_fsa, &host_arc_derivs);
  *dest = fsa_creator.GetFsa();
  if (arc_derivs != nullptr) *arc_derivs = ragged_creator.GetRagged2();
}

Fsa LinearFsa(const Array1<int32_t> &symbols) {
  NVTX_RANGE(K2_FUNC);
  ContextPtr &c = symbols.Context();
  int32_t n = symbols.Dim(), num_states = n + 2, num_arcs = n + 1;
  Array1<int32_t> row_splits1 = Range(c, num_states + 1, 0),
                  row_ids1 = Range(c, num_arcs, 0);
  int32_t *row_splits1_data = row_splits1.Data();
  Array1<Arc> arcs(c, num_arcs);
  Arc *arcs_data = arcs.Data();
  const int32_t *symbols_data = symbols.Data();
  K2_EVAL(
      c, num_arcs, lambda_set_arcs, (int32_t arc_idx01)->void {
        int32_t src_state = arc_idx01, dest_state = arc_idx01 + 1,
                // -1 == kFinalSymbol
            symbol = (arc_idx01 < n ? symbols_data[arc_idx01] : -1);
        if (arc_idx01 < n) K2_CHECK_NE(symbol, -1);
        float score = 0.0;
        arcs_data[arc_idx01] = Arc(src_state, dest_state, symbol, score);
        // the final state has no leaving arcs.
        if (arc_idx01 == 0) row_splits1_data[num_states] = num_arcs;
      });
  return Ragged<Arc>(RaggedShape2(&row_splits1, &row_ids1, num_arcs), arcs);
}

FsaVec LinearFsas(const Ragged<int32_t> &symbols) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(symbols.NumAxes(), 2);
  ContextPtr &c = symbols.Context();

  // if there are n symbols, there are n+2 states and n+1 arcs.
  RaggedShape states_shape = ChangeSublistSize(symbols.shape, 2);

  int32_t num_states = states_shape.NumElements(),
          num_arcs = symbols.NumElements() + symbols.Dim0();

  // row_splits2 maps from state_idx01 to arc_idx012; row_ids2 does the reverse.
  // We'll set them in the lambda below.
  Array1<int32_t> row_splits2(c, num_states + 1), row_ids2(c, num_arcs);
  // If num_states equals to zero, the code below won't set the last value of
  // row_splits2, we should initialize here, or it will be a random value.
  if (num_states == 0) row_splits2 = 0;

  int32_t *row_ids2_data = row_ids2.Data(),
          *row_splits2_data = row_splits2.Data();
  const int32_t *row_ids1_data = states_shape.RowIds(1).Data(),
                *row_splits1_data = states_shape.RowSplits(1).Data(),
                *symbols_data = symbols.values.Data();
  Array1<Arc> arcs(c, num_arcs);
  Arc *arcs_data = arcs.Data();
  K2_EVAL(
      c, num_states, lambda, (int32_t state_idx01)->void {
        int32_t fsa_idx0 = row_ids1_data[state_idx01],
                state_idx0x = row_splits1_data[fsa_idx0],
                next_state_idx0x = row_splits1_data[fsa_idx0 + 1],
                idx1 = state_idx01 - state_idx0x;

        // the following works because each FSA has one fewer arcs than states.
        int32_t arc_idx0xx = state_idx0x - fsa_idx0,
                next_arc_idx0xx = next_state_idx0x - (fsa_idx0 + 1),
                // the following may look a bit wrong.. here, the idx1 is the
                // same as the idx12 if the arc exists, because each state has
                // one arc leaving it (except the last state).
            arc_idx012 = arc_idx0xx + idx1;
        // the following works because each FSA has one fewer symbols than arcs
        // (however it doesn't work for the last arc of each FSA; we check
        // below.)
        int32_t symbol_idx01 = arc_idx012 - fsa_idx0;
        if (arc_idx012 < next_arc_idx0xx) {
          int32_t src_state = idx1, dest_state = idx1 + 1,
                  symbol = (arc_idx012 + 1 < next_arc_idx0xx
                                ? symbols_data[symbol_idx01]
                                : -1);  // kFinalSymbol
          float score = 0.0;
          arcs_data[arc_idx012] = Arc(src_state, dest_state, symbol, score);
          row_ids2_data[arc_idx012] = state_idx01;
        } else {
          // The following ensures that the last element of row_splits1_data
          // (i.e. row_splits1[num_states]) is set to num_arcs.  It also writes
          // something unnecessary for the last state of each FSA but the last
          // one, which will cause 2 threads to write the same item to the same
          // location. Note that there is no arc with index `arc_idx01`, if you
          // reach here.
          row_splits2_data[state_idx01 + 1] = arc_idx012;
        }
        row_splits2_data[state_idx01] = arc_idx012;
      });
  return Ragged<Arc>(
      RaggedShape3(&states_shape.RowSplits(1), &states_shape.RowIds(1),
                   num_states, &row_splits2, &row_ids2, num_arcs),
      arcs);
}

FsaVec LevenshteinGraphs(const Ragged<int32_t> &symbols,
                         float ins_del_score /* = -0.501 */,
                         Array1<int32_t> *aux_labels /*= nullptr*/,
                         Array1<float> *score_offsets /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(symbols.NumAxes(), 2);
  ContextPtr &c = symbols.Context();

  // For each fsa, the number of states will be number of symbols plus 2, we
  // plus 2 because we need an extra super final arc for each fsa.
  RaggedShape fsa_to_states = ChangeSublistSize(symbols.shape, 2);

  int32_t num_states = fsa_to_states.NumElements();
  Array1<int32_t> num_arcs_for(c, num_states + 1);
  int32_t *num_arcs_for_data = num_arcs_for.Data();
  // "fts" is short for fsa to states
  const int32_t *fts_row_splits1_data = fsa_to_states.RowSplits(1).Data(),
                *fts_row_ids1_data = fsa_to_states.RowIds(1).Data();
  // set the arcs number for each state
  K2_EVAL(
      c, num_states, lambda_set_num_arcs, (int32_t state_idx01)->void {
        int32_t fsa_idx0 = fts_row_ids1_data[state_idx01],
                final_state = fts_row_splits1_data[fsa_idx0 + 1] - 1,
                current_num_arcs = 3;  // normally there are three arcs,
                                       // self-loop and two arcs pointing to
                                       // the next state.
        if (state_idx01 == final_state - 1)
          current_num_arcs = 2;
        else if (state_idx01 == final_state)
          current_num_arcs = 0;
        num_arcs_for_data[state_idx01] = current_num_arcs;
      });
  ExclusiveSum(num_arcs_for, &num_arcs_for);
  Array1<int32_t> &states_to_arcs_row_splits = num_arcs_for;
  int32_t num_arcs = symbols.NumElements() * 3 + symbols.Dim0() * 2;
  RaggedShape states_to_arcs =
      RaggedShape2(&states_to_arcs_row_splits, nullptr, num_arcs);

  // shape with a index of [fsa][state][arc]
  RaggedShape shape = ComposeRaggedShapes(fsa_to_states, states_to_arcs);
  Array1<Arc> arcs(c, num_arcs);
  Arc *arcs_data = arcs.Data();
  const int32_t *row_splits1_data = shape.RowSplits(1).Data(),
                *row_ids1_data = shape.RowIds(1).Data(),
                *row_splits2_data = shape.RowSplits(2).Data(),
                *row_ids2_data = shape.RowIds(2).Data(),
                *symbols_data = symbols.values.Data();

  int32_t *aux_labels_data = nullptr;
  if (aux_labels != nullptr) {
    *aux_labels = Array1<int32_t>(c, num_arcs);
    aux_labels_data = aux_labels->Data();
  }
  float *score_offsets_data = nullptr;
  if (score_offsets != nullptr) {
    *score_offsets = Array1<float>(c, num_arcs);
    score_offsets_data = score_offsets->Data();
  }

  K2_EVAL(
      c, num_arcs, lambda_set_arcs, (int32_t arc_idx012)->void {
        int32_t state_idx01 = row_ids2_data[arc_idx012],
                fsa_idx0 = row_ids1_data[state_idx01],
                state_idx0x = row_splits1_data[fsa_idx0],
                final_state_idx01 = row_splits1_data[fsa_idx0 + 1] - 1,
                state_idx1 = state_idx01 - state_idx0x,
                arc_idx01x = row_splits2_data[state_idx01],
                arc_idx2 = arc_idx012 - arc_idx01x,
                sym_state_idx01 = state_idx01 - 2 * fsa_idx0,
                current_symbol = 0,
                aux_labels_value = 0;

        if (state_idx01 != final_state_idx01 - 1 &&
            state_idx01 != final_state_idx01) {
          current_symbol = symbols_data[sym_state_idx01];
          K2_CHECK((current_symbol != 0) && (current_symbol != -1))
            << "0 and -1 are not expected to be a symbol.";
        }

        float score_offset_value = 0;
        Arc arc;
        arc.src_state = state_idx1;

        switch (arc_idx2) {
          case 0:  // the self loop arc
            arc.label = 0;
            arc.dest_state = state_idx1;
            arc.score = ins_del_score;
            aux_labels_value = 0;
            score_offset_value = ins_del_score - (-0.5);
            break;
          case 1:   // the arc pointing to next state with blank
            if (state_idx01 == final_state_idx01 - 1) {  // the arc pointing to
                                                         // final state
              arc.label = -1;
              arc.score = 0;
              aux_labels_value = -1;
            } else {
              arc.label = 0;
              arc.score = -0.5;
              aux_labels_value = current_symbol;
            }
            arc.dest_state = state_idx1 + 1;
            break;
          case 2:  // the arc pointing to the next state with symbol
            arc.label = current_symbol;
            arc.dest_state = state_idx1 + 1;
            arc.score = 0;
            aux_labels_value = current_symbol;
            break;
          default:
            K2_LOG(FATAL) << "Arc index must be less than 3";
        }

        arcs_data[arc_idx012] = arc;
        if (aux_labels) aux_labels_data[arc_idx012] = aux_labels_value;
        if (score_offsets) score_offsets_data[arc_idx012] = score_offset_value;
      });
  return Ragged<Arc>(shape, arcs);
}

FsaVec CtcGraphs(const Ragged<int32_t> &symbols, bool modified /*= false*/,
                 Array1<int32_t> *aux_labels /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(symbols.NumAxes(), 2);
  ContextPtr &c = symbols.Context();

  int32_t num_fsas = symbols.Dim0();
  Array1<int32_t> num_states_for(c, num_fsas + 1);
  int32_t *num_states_for_data = num_states_for.Data();
  const int32_t *symbol_row_split1_data = symbols.RowSplits(1).Data();
  // symbols indexed with [fsa][symbol]
  // for each fsa we need `symbol_num * 2 + 1 + 1` states, `symbol_num * 2 + 1`
  // means that we need a blank state on each side of a symbol state, `+ 1` is
  // for final state in k2
  K2_EVAL(
      c, num_fsas, lambda_set_num_states, (int32_t fsa_idx0)->void {
        int32_t symbol_idx0x = symbol_row_split1_data[fsa_idx0],
                symbol_idx0x_next = symbol_row_split1_data[fsa_idx0 + 1],
                symbol_num = symbol_idx0x_next - symbol_idx0x;
        num_states_for_data[fsa_idx0] = symbol_num * 2 + 2;
      });

  ExclusiveSum(num_states_for, &num_states_for);
  Array1<int32_t> &fsa_to_states_row_splits = num_states_for;
  RaggedShape fsa_to_states =
      RaggedShape2(&fsa_to_states_row_splits, nullptr, -1);

  int32_t num_states = fsa_to_states.NumElements();
  Array1<int32_t> num_arcs_for(c, num_states + 1);
  int32_t *num_arcs_for_data = num_arcs_for.Data();
  const int32_t *fts_row_splits1_data = fsa_to_states.RowSplits(1).Data(),
                *fts_row_ids1_data = fsa_to_states.RowIds(1).Data(),
                *symbol_data = symbols.values.Data();
  // set the arcs number for each state
  K2_EVAL(
      c, num_states, lambda_set_num_arcs, (int32_t state_idx01)->void {
        int32_t fsa_idx0 = fts_row_ids1_data[state_idx01],
                // we minus fsa_idx0 here, because we are adding one more state,
                // the final state for each fsa
                sym_state_idx01 = state_idx01 / 2 - fsa_idx0,
                remainder = state_idx01 % 2,
                current_num_arcs = 2;  // normally there are two arcs, self-loop
                                       // and arc pointing to the next state
                                       // blank state always has two arcs
        if (remainder) {  // symbol state
          int32_t sym_final_state =
                    symbol_row_split1_data[fsa_idx0 + 1];
          // There are no arcs for final states
          if (sym_state_idx01 == sym_final_state) {
            current_num_arcs = 0;
          } else if (modified) {
            current_num_arcs = 3;
          } else {
            int32_t current_symbol = symbol_data[sym_state_idx01],
                    // we set the next symbol of the last symbol to -1, so
                    // the following if clause will always be true, which means
                    // we will have 3 arcs for last symbol state
                    next_symbol = (sym_state_idx01 + 1) == sym_final_state ?
                                  -1 : symbol_data[sym_state_idx01 + 1];
            // symbols must be not equal to -1, which is specially used in k2
            K2_CHECK_NE(current_symbol, -1);
            // if current_symbol equals next_symbol, we need a blank state
            // between them, so there are two arcs for this state
            // otherwise, this state will point to blank state and next symbol
            // state, so we need three arcs here.
            // Note: for the simplified topology (standard equals false), there
            // are always 3 arcs leaving symbol states.
            if (current_symbol != next_symbol)
              current_num_arcs = 3;
          }
        }
        num_arcs_for_data[state_idx01] = current_num_arcs;
      });

  ExclusiveSum(num_arcs_for, &num_arcs_for);
  Array1<int32_t> &states_to_arcs_row_splits = num_arcs_for;
  RaggedShape states_to_arcs =
      RaggedShape2(&states_to_arcs_row_splits, nullptr, -1);

  // ctc_shape with a index of [fsa][state][arc]
  RaggedShape ctc_shape = ComposeRaggedShapes(fsa_to_states, states_to_arcs);
  int32_t num_arcs = ctc_shape.NumElements();
  Array1<Arc> arcs(c, num_arcs);
  Arc *arcs_data = arcs.Data();
  const int32_t *ctc_row_splits1_data = ctc_shape.RowSplits(1).Data(),
                *ctc_row_ids1_data = ctc_shape.RowIds(1).Data(),
                *ctc_row_splits2_data = ctc_shape.RowSplits(2).Data(),
                *ctc_row_ids2_data = ctc_shape.RowIds(2).Data();
  int32_t *aux_labels_data = nullptr;
  if (aux_labels != nullptr) {
    *aux_labels = Array1<int32_t>(c, num_arcs);
    aux_labels_data = aux_labels->Data();
  }

  K2_EVAL(
      c, num_arcs, lambda_set_arcs, (int32_t arc_idx012)->void {
        int32_t state_idx01 = ctc_row_ids2_data[arc_idx012],
                fsa_idx0 = ctc_row_ids1_data[state_idx01],
                state_idx0x = ctc_row_splits1_data[fsa_idx0],
                state_idx1 = state_idx01 - state_idx0x,
                arc_idx01x = ctc_row_splits2_data[state_idx01],
                arc_idx2 = arc_idx012 - arc_idx01x,
                sym_state_idx01 = state_idx01 / 2 - fsa_idx0,
                remainder = state_idx01 % 2,
                sym_final_state = symbol_row_split1_data[fsa_idx0 + 1];
        bool final_state = sym_final_state == sym_state_idx01;
        int32_t current_symbol = final_state ?
            -1 : symbol_data[sym_state_idx01];
        Arc arc;
        arc.score = 0;
        arc.src_state = state_idx1;
        int32_t aux_labels_value = 0;
        if (remainder) {
          if (final_state) return;
          int32_t next_symbol = (sym_state_idx01 + 1) == sym_final_state ?
              -1 : symbol_data[sym_state_idx01 + 1];
          // for standard topology, the symbol state can not point to next
          // symbol state if the next symbol is identical to current symbol.
          if (current_symbol == next_symbol && !modified) {
            K2_CHECK_LT(arc_idx2, 2);
            arc.label = arc_idx2 == 0 ? 0 : current_symbol;
            arc.dest_state = arc_idx2 == 0 ? state_idx1 + 1 : state_idx1;
          } else {
            switch (arc_idx2) {
              case 0:   // the arc pointing to blank state
                arc.label = 0;
                arc.dest_state = state_idx1 + 1;
                break;
              case 1:   // the self loop arc
                arc.label = current_symbol;
                arc.dest_state = state_idx1;
                break;
              case 2:  // the arc pointing to the next symbol state
                arc.label = next_symbol;
                aux_labels_value = sym_state_idx01 + 1 == sym_final_state ?
                    -1 : next_symbol;
                arc.dest_state = state_idx1 + 2;
                break;
              default:
                K2_LOG(FATAL) << "Arc index must be less than 3";
            }
          }
        } else {
          K2_CHECK_LT(arc_idx2, 2);
          arc.label = arc_idx2 == 0 ? 0 : current_symbol;
          arc.dest_state = arc_idx2 == 0 ? state_idx1 : state_idx1 + 1;
          aux_labels_value = arc_idx2 == 0 ? 0 : current_symbol;
          if (final_state && arc_idx2 != 0) aux_labels_value = -1;
        }
        arcs_data[arc_idx012] = arc;
        if (aux_labels) aux_labels_data[arc_idx012] = aux_labels_value;
      });
  return Ragged<Arc>(ctc_shape, arcs);
}

Fsa CtcTopo(const ContextPtr &c, int32_t max_token, bool modified,
            Array1<int32_t> *aux_labels) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK(aux_labels);
  if (modified) {
    // plusing 2 here to include 0(epsilon) and final state
    int32_t states = max_token + 2;
    // for modified topology, the number of self loops and leaving arcs for
    // state 0 are all the number of states minus one.
    // and there two arcs(one for self loop, the other points to state 0) for
    // each of other states. see links below for details :
    // https://github.com/k2-fsa/k2/issues/746#issuecomment-856421616
    // https://github.com/k2-fsa/snowfall/pull/209
    int32_t num_arcs = (states - 1) * 2 + (states - 2) * 2;
    *aux_labels = Array1<int32_t>(c, num_arcs);
    Array1<int32_t> row_ids(c, num_arcs);
    Array1<Arc> arcs(c, num_arcs);
    int32_t *row_ids_data = row_ids.Data(),
            *aux_labels_data = aux_labels->Data();
    Arc *arcs_data = arcs.Data();
    K2_EVAL(
      c, num_arcs, lambad_set_row_ids_and_arcs, (int32_t idx01) -> void {
        Arc arc;
        arc.score = 0;
        if (idx01 < states - 1) {  // state 0 self loop
          arc.src_state = 0;
          arc.dest_state = 0;
          arc.label = idx01;
          row_ids_data[idx01] = 0;
          aux_labels_data[idx01] = idx01;
        } else if (idx01 < (states - 1) * 2) {  // arcs leaving state 0
          int32_t dest_state = idx01 - (states - 1) + 1;
          arc.src_state = 0;
          arc.dest_state = dest_state;
          arc.label = dest_state == states - 1 ? -1 : dest_state;
          row_ids_data[idx01] = 0;
          aux_labels_data[idx01] = dest_state == states -1 ? -1 : dest_state;
        } else {  // arcs for other states
          int32_t bias = idx01 - (states - 1) * 2;
          int32_t state = bias / 2 + 1;
          arc.src_state = state;
          arc.label = state;
          if (bias % 2)
            arc.dest_state = 0;
          else
            arc.dest_state = state;
          row_ids_data[idx01] = state;
          aux_labels_data[idx01] = 0;
        }
        arcs_data[idx01] = arc;
      });
    Array1<int32_t> row_splits(c, states + 1);
    RowIdsToRowSplits(row_ids, &row_splits);
    return Ragged<Arc>(RaggedShape2(&row_splits, &row_ids, num_arcs), arcs);
  } else {
    // plusing 2 here to include 0(epsilon) and final state
    int32_t states = max_token + 2,
            dim0 = states - 1,  // minusing 1 here because there is not
                                // any leaving arcs for final state
            dim1 = max_token + 2,  // there are number of states arcs leaving
                                   // each state for standard topolopy
            num_arcs = dim0 * dim1;
    *aux_labels = Array1<int32_t>(c, num_arcs);
    Array1<int32_t> row_ids(c, num_arcs);
    Array1<Arc> arcs(c, num_arcs);
    int32_t *row_ids_data = row_ids.Data(),
            *aux_labels_data = aux_labels->Data();
    Arc *arcs_data = arcs.Data();
    K2_EVAL2(
      c, dim0, dim1, lambda_set_row_ids_and_arcs,
        (int32_t i, int32_t j)->void {
          row_ids_data[i * dim1 + j] = i;
          Arc arc;
          arc.src_state = i;
          arc.dest_state = j;
          arc.label = j == (dim1 - 1) ? -1 : j;
          arc.score = 0;
          arcs_data[i * dim1 + j] = arc;
          int32_t olabel = i == j ? 0 : (j == (dim1 - 1) ? -1 : j);
          aux_labels_data[i * dim1 + j] = olabel;
      });
    Array1<int32_t> row_splits(c, states + 1);
    RowIdsToRowSplits(row_ids, &row_splits);
    return Ragged<Arc>(RaggedShape2(&row_splits, &row_ids, dim0 * dim1), arcs);
  }
}

Fsa TrivialGraph(const ContextPtr &c, int32_t max_token,
    Array1<int32_t> *aux_labels) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK(aux_labels);
  int32_t num_arcs = max_token + 1;
  Array1<int32_t> row_splits(c, std::vector<int32_t>{0, num_arcs, num_arcs});
  Array1<int32_t> row_ids(c, num_arcs);
  Array1<Arc> values(c, num_arcs);
  *aux_labels = Array1<int32_t>(c, num_arcs);
  int32_t *row_ids_data = row_ids.Data(),
          *aux_labels_data = aux_labels->Data();
  Arc *values_data = values.Data();

  K2_EVAL(
      c, num_arcs, lambda, (int32_t idx)->void {
        Arc arc;
        arc.score = 0;
        arc.src_state = 0;
        arc.dest_state = 0;
        arc.label = idx + 1;
        int32_t aux_label = idx + 1, row_id = 0;
        if (idx == num_arcs - 1) {
          arc.dest_state = 1;
          arc.label = -1;
          aux_label = -1;
        }
        row_ids_data[idx] = row_id;
        values_data[idx] = arc;
        aux_labels_data[idx] = aux_label;
      });
  return Ragged<Arc>(RaggedShape2(&row_splits, &row_ids, num_arcs), values);
}

void ArcSort(Fsa *fsa) {
  if (fsa->NumAxes() < 2) return;  // it is empty
  SortSublists<Arc>(fsa);
}

void ArcSort(Fsa &src, Fsa *dest, Array1<int32_t> *arc_map /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  if (!src.values.IsValid()) return;

  if (arc_map != nullptr)
    *arc_map = Array1<int32_t>(src.Context(), src.NumElements());

  Fsa tmp(src.shape, src.values.Clone());
  SortSublists<Arc>(&tmp, arc_map);
  *dest = tmp;
}

Ragged<int32_t> ShortestPath(FsaVec &fsas,
                             const Array1<int32_t> &entering_arcs) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  const int32_t *entering_arcs_data = entering_arcs.Data();
  const Arc *arcs_data = fsas.values.Data();
  int32_t num_fsas = fsas.Dim0();
  int32_t num_states = fsas.TotSize(1);
  ContextPtr &context = fsas.Context();

  // allocate an extra element for ExclusiveSum
  Array1<int32_t> num_best_arcs_per_fsa(context, num_fsas + 1, 0);
  int32_t *num_best_arcs_per_fsa_data = num_best_arcs_per_fsa.Data();
  const int32_t *row_splits1_data = fsas.RowSplits(1).Data();

  // -1 represents an invalid arc_index.
  // This extra array avoids an extra iteration over `entering_arcs`.
  Array1<int32_t> state_best_arc_index_array(context, num_states, -1);
  int32_t *state_best_arc_index_array_data = state_best_arc_index_array.Data();

#if 0
  // This is a simple version of the kernel that demonstrates what we're trying
  // to do with the more complex code.
  K2_EVAL(
      context, num_fsas, lambda_set_num_best_arcs, (int32_t fsas_idx0) {
        int32_t state_idx01 = row_splits1_data[fsas_idx0];
        int32_t state_idx01_next = row_splits1_data[fsas_idx0 + 1];

        if (state_idx01_next == state_idx01) {
          // this fsa is empty, so there is no best path available
          num_best_arcs_per_fsa_data[fsas_idx0] = 0;
          return;
        }

        int32_t final_state_idx01 = state_idx01_next - 1;
        int32_t cur_state = final_state_idx01;
        int32_t cur_index = entering_arcs_data[cur_state];
        int32_t num_arcs = 0;
        int32_t *p = state_best_arc_index_array_data + final_state_idx01;
        while (cur_index != -1) {
          *p = cur_index;
          --p;

          cur_state = arcs_data[cur_index].src_state + state_idx01;
          cur_index = entering_arcs_data[cur_state];
          ++num_arcs;
        }
        if (cur_state != state_idx01) {
          // Previous condition equals to
          // arcs_data[cur_index].src_state != 0.
          // Current fsa is non-connected.
          num_arcs = 0;
        }
        num_best_arcs_per_fsa_data[fsas_idx0] = num_arcs;
      });
#else
  // Comparing with previous simple version,
  // the run time of following code is less than linear,
  // by using methods similar to that in GetStateBatches();
  // imagine computing a table that including the entering arc,
  // then the entering arc of 2-step traceback;
  // and then the entering arc of 4-step traceback, and so on.

  // We can tune `log_power` as a tradeoff between work done and clock time on
  // GPU.
  int32_t log_power = (context->GetDeviceType() == kCpu ? 0 : 4);

  int32_t max_num_states = fsas.shape.MaxSize(1);
  // The following avoids doing too much extra work accumulating powers
  // of 'entering_arcs' for very small problem sizes.
  while (log_power > 0 && (1 << (1 + log_power)) > max_num_states) log_power--;

  Array2<int32_t> entering_arcs_powers(context, log_power + 1, num_states);
  const int32_t stride = entering_arcs_powers.ElemStride0();
  int32_t *entering_arcs_powers_data = entering_arcs_powers.Data();

  const int32_t *row_ids1_data = fsas.RowIds(1).Data(),
                *row_splits2_data = fsas.RowSplits(2).Data();

  // Row 0 tracks entering arc of 1-step traceback for each state.
  context->CopyDataTo(
      entering_arcs.Dim() * entering_arcs.ElementSize(),
      entering_arcs_data,
      context,
      entering_arcs_powers_data);

  // Row 1 tracks entering arc of 2-step traceback for each state;
  // Row 2 tracks entering arc of 4-step traceback for each state, and so on.
  for (int32_t power = 1; power <= log_power; power++) {
    const int32_t *src_data =
      entering_arcs_powers.Data() + (power - 1) * stride;
    int32_t *dest_data =
      entering_arcs_powers.Data() + power * stride;

    K2_EVAL(
        context, num_states, lambda_set_entering_arcs_powers,
        (int32_t state_idx01)->void {
          int32_t fsas_idx0 = row_ids1_data[state_idx01];
          // The first state of current fsas_idx0.
          int32_t begin_state_idx01 = row_splits1_data[fsas_idx0];
          int32_t cur_index = src_data[state_idx01];

          if (cur_index != -1) {
            int32_t cur_state =
              arcs_data[cur_index].src_state + begin_state_idx01;
            cur_index = src_data[cur_state];
          }
          dest_data[state_idx01] = cur_index;
        });
  }

  // jobs_per_fsa tells us how many separate chains of states we'll follow for
  // each FSA.
  // jobs_multiple is a kind of trick to ensure any given warp doesn't
  // issue more memory requests than it can handle at a time (we drop
  // some threads).
  int32_t jobs_per_fsa = (1 << log_power),
          jobs_multiple = (context->GetDeviceType() == kCuda ? 8 : 1);
  while (jobs_multiple > 1 && jobs_per_fsa * jobs_multiple * num_fsas > 10000)
    jobs_multiple /= 2;  // Likely won't get here.  Just reduce multiple if
                         // num-jobs is ridiculous.

  auto entering_arcs_powers_acc = entering_arcs_powers.Accessor();
  K2_EVAL2(
      context, num_fsas, jobs_per_fsa * jobs_multiple,
      lambda_set_numbert_best_arcs2, (int32_t fsas_idx0, int32_t j) {
        if (j % jobs_multiple != 0)
          return;  // a trick to avoid too much random
                   // memory access for any given warp
        int32_t task_idx =
            j / jobs_multiple;  // Now 0 <= task_idx < jobs_per_fsa.

        int32_t begin_state_idx01 = row_splits1_data[fsas_idx0];

        int32_t end_state_idx01 = row_splits1_data[fsas_idx0 + 1];

        int32_t begin_arc_idx012 = row_splits2_data[begin_state_idx01];
        int32_t end_arc_idx012 = row_splits2_data[end_state_idx01];

        int32_t num_states_this_fsa = end_state_idx01 - begin_state_idx01;
        int32_t num_arcs_this_fsa = end_arc_idx012 - begin_arc_idx012;
        if (num_arcs_this_fsa == 0 || num_states_this_fsa == 0) {
          // This fsa is empty, so there is no shortest path available.
          num_best_arcs_per_fsa_data[fsas_idx0] = 0;
          return;
        }

        int32_t least_num_best_arcs_this_fsa = task_idx + 1;

        if (least_num_best_arcs_this_fsa > num_arcs_this_fsa ||
            least_num_best_arcs_this_fsa >= num_states_this_fsa) return;

        // Eventually,
        // num_best_arcs_this_fsa[fsa_idx0] = cur_num_best_states_this_fsa + 1.
        // cur_num_best_states_this_fsa is 0-based to make it easier
        // to compute offset(i.e. "p" in following code) of arc index.
        int32_t cur_num_best_states_this_fsa = 0;

        // Initialized for task_idx == 0.
        int32_t cur_dest_state_idx01 = end_state_idx01 - 1;
        int32_t cur_index = entering_arcs_powers_acc(0, cur_dest_state_idx01);

        // Initialized for task_idx > 0.
        for (int32_t m = 0; m < log_power; ++m) {
          int32_t n = 1 << m;
          if ((task_idx & n) != 0) {
            cur_num_best_states_this_fsa += n;
            cur_index = entering_arcs_powers_acc(m, cur_dest_state_idx01);
            if (cur_index == -1) return;

            // The new dest_state is the src_state of cur_index.
            // It's not a typo cur_dest_state_idx01 is assigned with src_state.
            cur_dest_state_idx01 =
              arcs_data[cur_index].src_state + begin_state_idx01;
          }
        }

        // In previous for loop, cur_dest_state_idx01 is assigned to the
        // "first" state for each task_idx.
        // To get shortest path, the original fsa is visited in a reversed way,
        // so the "first" states here are "tailing" states in original fsa.
        // e.g.:
        // For task_idx = 0,
        // the "first" state is the final state in original fsa,
        // i.e. end_state_idx01 - 1.
        //
        // For task_idx = 1, the "first" state is a penultimate state,
        // i.e. the one that owns the "entering arc" to the final state.
        //
        // cur_idx is the "entering arc" of the "first" state for each task_idx.
        cur_index = entering_arcs_powers_acc(0, cur_dest_state_idx01);
        if (cur_index == -1) return;
        int32_t cur_src_state_idx01 =
          arcs_data[cur_index].src_state + begin_state_idx01;

        K2_CHECK_EQ(cur_num_best_states_this_fsa, task_idx);

        // cur_num_best_states_this_fsa is 0-based.
        // It's slightly easier to compute the storage offset(i.e. p)
        // for arc_index than 1-based.
        int32_t *p = state_best_arc_index_array_data + end_state_idx01 - 1
          - cur_num_best_states_this_fsa;

        // Used to detect states whose entering_arc_idx == -1
        // and calculate num_best_arcs_per_fsa[fsa_idx0].
        int32_t next_num_best_states_this_fsa = cur_num_best_states_this_fsa;
        int32_t prev_src_state_idx01 = cur_src_state_idx01;

        while (1) {
          if (cur_index == -1) {
            // If exactly one step would also be enough to take us past the
            // boundary.
            if (entering_arcs_powers_acc(0, prev_src_state_idx01) == -1) {
              if (prev_src_state_idx01 == begin_state_idx01) {
                // cur_num_best_states is 0-based.
                // "+ 1" makes it 1-based.
                num_best_arcs_per_fsa_data[fsas_idx0] =
                  cur_num_best_states_this_fsa + 1;
              } else {
                // Current fsa is non-connected.
                num_best_arcs_per_fsa_data[fsas_idx0] = 0;
              }
            }
            return;
          } else {
            // Storage cur_index and calculate the new offset for
            // a step with "jobs_per_fsa" arcs.
            *p = cur_index;
            p -= jobs_per_fsa;

            // Cache current environment before trying
            // a step with "jobs_per_fsa" arcs.
            cur_num_best_states_this_fsa = next_num_best_states_this_fsa;
            next_num_best_states_this_fsa += jobs_per_fsa;
            prev_src_state_idx01 =
              arcs_data[cur_index].src_state + begin_state_idx01;

            // Try a step with "jobs_per_fsa" arcs.
            cur_index =
              entering_arcs_powers_acc(log_power, prev_src_state_idx01);
          }
        }
      });
#endif
  ExclusiveSum(num_best_arcs_per_fsa, &num_best_arcs_per_fsa);

  RaggedShape shape = RaggedShape2(&num_best_arcs_per_fsa, nullptr, -1);
  const int32_t *shape_row_splits1_data = shape.RowSplits(1).Data();
  const int32_t *shape_row_ids1_data = shape.RowIds(1).Data();

  const int32_t *ans_row_splits_data = shape.RowSplits(1).Data();
  Array1<int32_t> best_path_arc_indexes(context, shape.NumElements());
  int32_t *best_path_arc_indexes_data = best_path_arc_indexes.Data();

  K2_EVAL(
      context, shape.NumElements(), lambda_set_best_arcs, (int32_t ans_idx01) {
        int32_t fsa_idx0 = shape_row_ids1_data[ans_idx01];
        int32_t ans_idx0x = shape_row_splits1_data[fsa_idx0];
        int32_t ans_idx1 = ans_idx01 - ans_idx0x;

        int32_t num_arcs_this_fsa = num_best_arcs_per_fsa_data[fsa_idx0 + 1] -
                                    num_best_arcs_per_fsa_data[fsa_idx0];
        if (num_arcs_this_fsa == 0) return;

        int32_t final_state_idx01_this_fsa = row_splits1_data[fsa_idx0 + 1] - 1;

        const int32_t *p_start = state_best_arc_index_array_data +
                                 final_state_idx01_this_fsa -
                                 num_arcs_this_fsa + 1;

        best_path_arc_indexes_data[ans_idx01] = p_start[ans_idx1];
      });

  Ragged<int32_t> ans(shape, best_path_arc_indexes);
  return ans;
}

void AddEpsilonSelfLoops(FsaOrVec &src, FsaOrVec *dest,
                         Array1<int32_t> *arc_map /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  ContextPtr &c = src.Context();
  const int32_t *old_row_splits1_data = src.RowSplits(1).Data(),
                *old_row_ids1_data = src.RowIds(1).Data();
  const Arc *old_arcs_data = src.values.Data();
  if (src.NumAxes() == 2) {
    int32_t num_states = src.Dim0();
    if (num_states < 2) {
      K2_CHECK_EQ(num_states, 0);
      *dest = src;
      if (arc_map != nullptr) *arc_map = Array1<int32_t>(c, 0);
      return;
    }

    int32_t old_num_arcs = src.TotSize(1),
            new_num_arcs = old_num_arcs + (num_states - 1);
    Array1<int32_t> new_row_splits(c, num_states + 1),
        new_row_ids(c, new_num_arcs);
    Array1<Arc> new_arcs(c, new_num_arcs);
    int32_t *new_row_splits1_data = new_row_splits.Data(),
            *new_row_ids1_data = new_row_ids.Data();
    Arc *new_arcs_data = new_arcs.Data();
    int32_t *arc_map_data = nullptr;
    if (arc_map) {
      *arc_map = Array1<int32_t>(c, new_num_arcs);
      arc_map_data = arc_map->Data();
    }
    ParallelRunner pr(c);
    {
      With w(pr.NewStream());
      K2_EVAL(
          c, old_num_arcs, lambda_copy_data, (int32_t arc_idx01)->void {
            int32_t state_idx0 = old_row_ids1_data[arc_idx01],
                    new_arc_idx01 = arc_idx01 + 1 + state_idx0;
            // the "+1" above is because we put the self-loop first.
            new_row_ids1_data[new_arc_idx01] = state_idx0;
            new_arcs_data[new_arc_idx01] = old_arcs_data[arc_idx01];
            if (arc_map_data) arc_map_data[new_arc_idx01] = arc_idx01;
          });
    }
    {
      With w(pr.NewStream());
      K2_EVAL(
          c, num_states, lambda_set_new_data, (int32_t state_idx0)->void {
            int32_t old_arc_idx0x = old_row_splits1_data[state_idx0],
                    new_arc_idx0x = old_arc_idx0x + state_idx0;
            new_row_splits1_data[state_idx0] = new_arc_idx0x;
            if (state_idx0 + 1 < num_states) {        // not final-state
              int32_t new_arc_idx01 = new_arc_idx0x;  // the 1st arc is the loop
              new_row_ids1_data[new_arc_idx01] = state_idx0;
              new_arcs_data[new_arc_idx01] =
                  Arc(state_idx0, state_idx0, 0, 0.0);
              if (arc_map_data) arc_map_data[new_arc_idx01] = -1;
            } else {
              // Note: if num_states was zero we would have returned above, so
              // we don't have to worry about empty FSAs.
              new_row_splits1_data[num_states] = new_arc_idx0x;
            }
          });
    }
    pr.Finish();
    *dest = Ragged<Arc>(
        RaggedShape2(&new_row_splits, &new_row_ids, new_num_arcs), new_arcs);
  } else {
    K2_CHECK_EQ(src.NumAxes(), 3);
    // Get a vector saying, for each FSA, whether it's nonempty.
    int32_t num_fsas = src.Dim0(), num_states = src.TotSize(1),
            old_num_arcs = src.TotSize(2);
    if (num_states == 0) {
      *dest = src;
      if (arc_map) *arc_map = Array1<int32_t>(c, 0);
      return;
    }
    Array1<int32_t> fsa_nonempty(c, num_fsas + 1);
    int32_t *fsa_nonempty_data = fsa_nonempty.Data();
    K2_EVAL(
        c, num_fsas, lambda_set_fsa_nonempty, (int32_t fsa_idx0)->void {
          fsa_nonempty_data[fsa_idx0] = (old_row_splits1_data[fsa_idx0 + 1] >
                                         old_row_splits1_data[fsa_idx0]);
        });
    ExclusiveSum(fsa_nonempty, &fsa_nonempty);
    const int32_t *old_row_splits2_data = src.RowSplits(2).Data(),
                  *old_row_ids2_data = src.RowIds(2).Data();
    int32_t num_nonempty_fsas = fsa_nonempty.Back(),
            new_num_arcs = old_num_arcs + num_states - num_nonempty_fsas;
    // we subtract `num_nonempty_fsas` because final-states don't get a
    // self-loop.

    Array1<int32_t> new_row_splits2(c, num_states + 1),
        new_row_ids2(c, new_num_arcs);
    Array1<Arc> new_arcs(c, new_num_arcs);
    // fsa_idx0_mod_data maps from fsa_idx0 to a modified fsa_idx0 that
    // "doesn't count" FSAs with zero states.
    const int32_t *fsa_idx0_mod_data = fsa_nonempty_data;
    int32_t *new_row_splits2_data = new_row_splits2.Data(),
            *new_row_ids2_data = new_row_ids2.Data();
    Arc *new_arcs_data = new_arcs.Data();
    int32_t *arc_map_data = nullptr;
    if (arc_map) {
      *arc_map = Array1<int32_t>(c, new_num_arcs);
      arc_map_data = arc_map->Data();
    }
    ParallelRunner pr(c);
    {
      With w(pr.NewStream());
      K2_EVAL(
          c, old_num_arcs, lambda_copy_data, (int32_t arc_idx012)->void {
            int32_t state_idx01 = old_row_ids2_data[arc_idx012],
                    fsa_idx0 = old_row_ids1_data[state_idx01],
                    fsa_idx0_mod = fsa_idx0_mod_data[fsa_idx0],
                    new_arc_idx012 =
                        arc_idx012 + 1 + state_idx01 - fsa_idx0_mod;
            // The "+1" above is because we put the self-loop first.  The
            // "-fsa_idx0_mod" is because final-states don't get a self-loop.
            new_row_ids2_data[new_arc_idx012] = state_idx01;
            new_arcs_data[new_arc_idx012] = old_arcs_data[arc_idx012];
            if (arc_map_data) arc_map_data[new_arc_idx012] = arc_idx012;
          });
    }
    {
      With w(pr.NewStream());
      K2_EVAL(
          c, num_states, lambda_set_new_data, (int32_t state_idx01)->void {
            int32_t fsa_idx0 = old_row_ids1_data[state_idx01],
                    fsa_idx0_mod = fsa_idx0_mod_data[fsa_idx0],
                    state_idx0x = old_row_splits1_data[fsa_idx0],
                    next_state_idx0x = old_row_splits1_data[fsa_idx0 + 1],
                    old_arc_idx01x = old_row_splits2_data[state_idx01];
            // Below the "+ state_idx01" is because each state gets a self-loop,
            // and the "- fsa_idx0_mod" is because final-states don't get a
            // self-loop.
            int32_t new_arc_idx01x =
                old_arc_idx01x + state_idx01 - fsa_idx0_mod;
            // The self-loop arc is the first arc:
            int32_t new_arc_idx012 = new_arc_idx01x;
            new_row_splits2_data[state_idx01] = new_arc_idx01x;
            if (state_idx01 + 1 < next_state_idx0x) {  // not final-state
              new_row_ids2_data[new_arc_idx012] = state_idx01;
              int32_t state_idx1 = state_idx01 - state_idx0x;
              new_arcs_data[new_arc_idx012] =
                  Arc(state_idx1, state_idx1, 0, 0.0);
              if (arc_map_data) arc_map_data[new_arc_idx012] = -1;
            } else if (state_idx01 + 1 == num_states) {
              // Note: if num_states was zero  we would have returned above, so
              // we dont have to worry about an empty FsaVec.
              new_row_splits2_data[num_states] = new_arc_idx01x;
            }
          });
    }
    pr.Finish();
    *dest =
        Ragged<Arc>(RaggedShape3(&src.RowSplits(1), &src.RowIds(1), num_states,
                                 &new_row_splits2, &new_row_ids2, new_num_arcs),
                    new_arcs);
  }
}

Fsa Union(FsaVec &fsas, Array1<int32_t> *arc_map /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(fsas.NumAxes(), 3);

  ContextPtr &context = fsas.Context();
  const int32_t *fsas_row_splits1_data = fsas.RowSplits(1).Data();
  const int32_t *fsas_row_splits2_data = fsas.RowSplits(2).Data();
  const int32_t *fsas_row_ids1_data = fsas.RowIds(1).Data();
  const int32_t *fsas_row_ids2_data = fsas.RowIds(2).Data();
  const Arc *arcs_data = fsas.values.Data();

  int32_t num_fsas = fsas.Dim0();
  int32_t num_states = fsas.TotSize(1);
  int32_t num_arcs = fsas.TotSize(2);

  // A new start state and a new final state are added (+2).
  // The final state of each fsa is removed (-num_fsas)
  int32_t num_out_states = num_states + 2 - num_fsas;
  int32_t out_final_state = num_out_states - 1;

  // For every fsa, a new arc is added from the new start state
  // to its original start state (+num_fsas)
  int32_t num_out_arcs = num_arcs + num_fsas;

  Array1<int32_t> out_row_ids(context, num_out_arcs);
  Array1<Arc> out_arcs(context, num_out_arcs);
  Array1<int32_t> tmp_arc_map(context, num_out_arcs, -1);
  int32_t *tmp_arc_map_data = tmp_arc_map.Data();

  int32_t *out_row_ids_data = out_row_ids.Data();
  Arc *out_arcs_data = out_arcs.Data();

  K2_EVAL(
      context, num_arcs, lambda_set_out, (int32_t fsas_arc_idx012) {
        int32_t fsas_state_idx01 = fsas_row_ids2_data[fsas_arc_idx012];
        int32_t fsas_idx0 = fsas_row_ids1_data[fsas_state_idx01];
        int32_t this_fsa_final_state_idx01 =
            fsas_row_splits1_data[fsas_idx0 + 1] - 1;

        K2_DCHECK_GT(this_fsa_final_state_idx01, fsas_state_idx01)
            << "We support only FSAs with at least two states at present";

        int32_t fsas_state_idx0x = fsas_row_splits1_data[fsas_idx0];
        int32_t fsas_state_idx1 = fsas_state_idx01 - fsas_state_idx0x;
        int32_t this_fsa_final_state_idx1 =
            this_fsa_final_state_idx01 - fsas_state_idx0x;

        int32_t fsas_arc_idx0xx = fsas_row_splits2_data[fsas_state_idx0x];

        // fsa0: +1 (a new start state)
        // fsa1: +0 (the final state of fsa0 is removed)
        // fsa2: -1 (the final state of fsa1 is removed)
        // fsa3: -2 (the final state of fsa2 is removed)
        int32_t state_offset = 1 - fsas_idx0;
        int32_t out_state_idx0 = fsas_state_idx01 + state_offset;

        int32_t out_arc_idx01 = fsas_arc_idx012 + num_fsas;
        out_row_ids_data[out_arc_idx01] = out_state_idx0;
        Arc arc = arcs_data[fsas_arc_idx012];

        K2_DCHECK_EQ(arc.src_state, fsas_state_idx1);

        if (arc.dest_state == this_fsa_final_state_idx1)
          arc.dest_state = out_final_state;
        else
          arc.dest_state = arc.dest_state - arc.src_state + out_state_idx0;

        arc.src_state = out_state_idx0;
        out_arcs_data[out_arc_idx01] = arc;
        tmp_arc_map_data[out_arc_idx01] = fsas_arc_idx012;

        if (fsas_arc_idx0xx == fsas_arc_idx012) {
          // add a new arc from the new start state to the start state
          // of this fsa
          //
          // WARNING: we cannot use fsas_state_idx01 here
          // since the start state may have no leaving arcs!
          Arc arc(0, fsas_state_idx0x + state_offset, 0, 0);
          out_arcs_data[fsas_idx0] = arc;
          out_row_ids_data[fsas_idx0] = 0;
        }
      });

  if (arc_map != nullptr) *arc_map = std::move(tmp_arc_map);
  Array1<int32_t> out_row_splits(context, num_out_states + 1);
  RowIdsToRowSplits(out_row_ids, &out_row_splits);
  RaggedShape shape = RaggedShape2(&out_row_splits, &out_row_ids, num_out_arcs);
  Fsa ans = Ragged<Arc>(shape, out_arcs);
  return ans;
}

Fsa Closure(Fsa &fsa, Array1<int32_t> *arc_map /* = nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(fsa.NumAxes(), 2) << "We support only a single FSA.";
  ContextPtr &c = fsa.Context();

  int32_t num_states = fsa.Dim0();
  if (num_states < 2) {
    K2_CHECK_EQ(num_states, 0)
        << "An empty fsa should contain no states at all";
    if (arc_map != nullptr) *arc_map = Array1<int32_t>(c, 0);
    return fsa;  // return itself if the input fsa is empty
  }

  const int32_t *fsa_row_splits_data = fsa.RowSplits(1).Data();
  const int32_t *fsa_row_ids_data = fsa.RowIds(1).Data();
  const Arc *fsa_arcs_data = fsa.values.Data();
  int32_t fsa_final_state = num_states - 1;

  int32_t num_out_states = num_states;

  // An arc from the start state to the final state with label == -1 is added.
  int32_t num_out_arcs = fsa.values.Dim() + 1;

  Array1<int32_t> out_row_ids(c, num_out_arcs);
  int32_t *out_row_ids_data = out_row_ids.Data();

  Array1<Arc> out_arcs(c, num_out_arcs);
  Arc *out_arcs_data = out_arcs.Data();

  Array1<int32_t> tmp_arc_map(c, num_out_arcs);
  int32_t *tmp_arc_map_data = tmp_arc_map.Data();

  K2_EVAL(
      c, fsa.values.Dim(), lambda_set_arcs, (int32_t fsa_arc_idx01) {
        int32_t fsa_state_idx0 = fsa_row_ids_data[fsa_arc_idx01];
        int32_t fsa_arc_idx0x = fsa_row_splits_data[fsa_state_idx0];
        int32_t fsa_arc_idx1 = fsa_arc_idx01 - fsa_arc_idx0x;
        int32_t this_state_num_arcs =
            fsa_row_splits_data[fsa_state_idx0 + 1] - fsa_arc_idx0x;

        Arc arc = fsa_arcs_data[fsa_arc_idx01];
        if (arc.dest_state == fsa_final_state) {
          // modify arcs entering the final state such that:
          //   - dest_state == 0
          //   - label == 0
          arc.dest_state = 0;
          K2_DCHECK_EQ(arc.label, -1);
          arc.label = 0;
        }

        int out_arc_idx01;
        if (arc.src_state > 0) {
          // this arc is not originated from the start state, so its index is
          // incremented
          out_arc_idx01 = fsa_arc_idx01 + 1;
        } else {
          out_arc_idx01 = fsa_arc_idx01;
          if (fsa_arc_idx1 == this_state_num_arcs - 1) {
            // This is the last arc of the original start state,
            // so we add a new arc just after it.
            Arc new_arc(0, fsa_final_state, -1, 0.0f);
            out_arcs_data[out_arc_idx01 + 1] = new_arc;
            out_row_ids_data[out_arc_idx01 + 1] = 0;
            tmp_arc_map_data[out_arc_idx01 + 1] = -1;
          }
        }

        // it may happen that the start state has no leaving arcs
        if (fsa_row_splits_data[1] == 0) {
          Arc new_arc(0, fsa_final_state, -1, 0.0f);
          out_arcs_data[0] = new_arc;
          out_row_ids_data[0] = 0;
          tmp_arc_map_data[0] = -1;
        }

        tmp_arc_map_data[out_arc_idx01] = fsa_arc_idx01;

        out_arcs_data[out_arc_idx01] = arc;
        out_row_ids_data[out_arc_idx01] = arc.src_state;
      });

  if (arc_map != nullptr) *arc_map = std::move(tmp_arc_map);

  Array1<int32_t> out_row_splits(c, num_out_states + 1);
  int32_t *out_row_splits_data = out_row_splits.Data();

  K2_EVAL(
      c, out_row_splits.Dim(), lambda_set_row_splits, (int32_t i) {
        if (i == 0)
          out_row_splits_data[i] = 0;
        else
          out_row_splits_data[i] = fsa_row_splits_data[i] + 1;
      });

  RaggedShape shape = RaggedShape2(&out_row_splits, &out_row_ids, num_out_arcs);
  Fsa ans = Ragged<Arc>(shape, out_arcs);
  return ans;
}

FsaOrVec ExpandArcs(FsaOrVec &fsas, RaggedShape &labels_shape,
                    Array1<int32_t> *fsas_arc_map /*=nullptr*/,
                    Array1<int32_t> *labels_arc_map /*=nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  if (fsas.NumAxes() == 2) {
    FsaVec fsas_temp = FsaToFsaVec(fsas);
    return ExpandArcs(fsas_temp, labels_shape, fsas_arc_map, labels_arc_map)
        .RemoveAxis(0);
  }
  K2_CHECK_EQ(fsas.NumAxes(), 3);
  K2_CHECK_EQ(labels_shape.NumAxes(), 2);
  K2_CHECK_EQ(fsas.NumElements(), labels_shape.Dim0());
  ContextPtr &c = fsas.Context();
  K2_CHECK(c->IsCompatible(*labels_shape.Context()));

  RaggedShape state_to_arcs = GetLayer(fsas.shape, 1);

  // `state_to_foo` is a RaggedShape that, for each state in `fsas`, has a list
  // of length `num_arcs + 1`, where `num_arcs` is the number of arcs leaving
  // this state in `fsas`.  Interpret this as: one element for the state
  // itself, then one for each arc leaving it.  This `foo` is an index that
  // corresponds to num-arcs plus one, but because it is really a placeholder
  // and we want to keep it distinct from other things, we call it `foo`.
  RaggedShape state_to_foo = ChangeSublistSize(state_to_arcs, 1);

  int32_t foo_size = state_to_foo.NumElements();

  // For each element of `state_to_foo`, `num_ostates_for` says how many states
  // there will be for this (state,foo) in the returned (output) FSA.  Here, the
  // idx0 is the state, the idx1 is foo.  If idx1 == 0 (interpret this as "the
  // state itself"), then `num_ostates_for[idx01] = 1`, meaning "keep the
  // original state".  Otherwise, idx1 - 1 represents an arc_idx2 [into `fsas`],
  // and we set `num_ostates_for[idx01] = max(0, seq_len-1)`, where seq_len is
  // the length of the sequence in `labels_shape` corresponding to this
  // arc-index.
  Array1<int32_t> num_ostates_for(c, foo_size + 1);
  int32_t *num_ostates_for_data = num_ostates_for.Data();

  const int32_t *labels_row_splits1_data = labels_shape.RowSplits(1).Data(),
                *fsas_row_splits2_data = fsas.RowSplits(2).Data(),
                *state_to_foo_row_splits1_data =
                    state_to_foo.RowSplits(1).Data(),
                *state_to_foo_row_ids1_data = state_to_foo.RowIds(1).Data();

  K2_EVAL(
      c, foo_size, lambda_set_num_ostates, (int32_t idx01)->void {
        // note: the idx01, idx0, idx0x are into `state_to_foo`.
        // This idx0 is a state-index into `fsas` (an idx01 w.r.t. `fsas`).
        int32_t idx0 = state_to_foo_row_ids1_data[idx01],
                idx0x = state_to_foo_row_splits1_data[idx0],
                idx1 = idx01 - idx0x;  // idx1 is `foo`.
        int32_t num_ostates;
        if (idx1 == 0) {
          num_ostates = 1;  // this is a copy of the original state.
        } else {
          int32_t fsas_arc_idx2 = idx1 - 1, fsas_state_idx01 = idx0,
                  fsas_arc_idx01x = fsas_row_splits2_data[fsas_state_idx01],
                  fsas_arc_idx012 = fsas_arc_idx01x + fsas_arc_idx2,
                  labels_shape_idx0 = fsas_arc_idx012,
                  labels_shape_idx0x =
                      labels_row_splits1_data[labels_shape_idx0],
                  labels_shape_idx0x_next =
                      labels_row_splits1_data[labels_shape_idx0 + 1],
                  labels_shape_len1 =
                      labels_shape_idx0x_next - labels_shape_idx0x;
          // A sequence of n symbols will require n-1 extra states to represent
          // it.
          num_ostates = max(labels_shape_len1 - 1, (int32_t)0);
        }
        num_ostates_for_data[idx01] = num_ostates;
      });
  ExclusiveSum(num_ostates_for, &num_ostates_for);
  Array1<int32_t> &foo_to_ostates_row_splits = num_ostates_for;
  RaggedShape foo_to_ostates =
      RaggedShape2(&foo_to_ostates_row_splits, nullptr, -1);

  // to_ostates_shape has 4 axes: [fsa_id][orig_state][foo][ostate]
  // where foo is a general-purpose index that ranges over the (num_arcs + 1) of
  // the original state.
  RaggedShape to_ostates_shape = ComposeRaggedShapes3(
      GetLayer(fsas.shape, 0), state_to_foo, foo_to_ostates);

  // Below, `tos` means `to_ostates_shape`.
  const int32_t *tos_row_splits1_data = to_ostates_shape.RowSplits(1).Data(),
                *tos_row_ids1_data = to_ostates_shape.RowIds(1).Data(),
                *tos_row_splits2_data = to_ostates_shape.RowSplits(2).Data(),
                *tos_row_ids2_data = to_ostates_shape.RowIds(2).Data(),
                *tos_row_splits3_data = to_ostates_shape.RowSplits(3).Data(),
                *tos_row_ids3_data = to_ostates_shape.RowIds(3).Data();

  // `num_oarcs` gives the number of arcs in the returned (output) FSA for each
  // `ostate` (i.e. leaving each state in the returned FSA).
  int32_t tot_ostates = to_ostates_shape.NumElements();
  Array1<int32_t> num_oarcs(c, tot_ostates + 1);
  int32_t *num_oarcs_data = num_oarcs.Data();
  K2_EVAL(
      c, tot_ostates, lambda_set_num_oarcs, (int32_t idx0123)->void {
        // All these indexes are into `to_ostates_shape`, indexed
        // `[fsa][state][foo][ostate].`
        int32_t idx012 = tos_row_ids3_data[idx0123],
                idx012x = tos_row_splits3_data[idx012],
                idx01 = tos_row_ids2_data[idx012],
                idx01x = tos_row_splits2_data[idx01],
                idx01x_next = tos_row_splits2_data[idx01 + 1],
                len2 = idx01x_next - idx01x, idx2 = idx012 - idx01x,
                idx3 = idx0123 - idx012x;
        int32_t num_arcs;
        if (idx2 == 0) {
          K2_CHECK_EQ(idx3, 0);
          // This ostate corresponds to the original state; it is not one of the
          // extra states added to support chains of arcs.
          // The original state had `orig_num_arcs` leaving it, which is the
          // number of `foo` indexes minus one.
          int32_t orig_num_arcs = len2 - 1;
          num_arcs = orig_num_arcs;
        } else {
          // All newly-created states have exactly one arc leaving them.
          num_arcs = 1;
        }
        num_oarcs_data[idx0123] = num_arcs;
      });
  ExclusiveSum(num_oarcs, &num_oarcs);
  Array1<int32_t> &ostate_to_oarcs_row_splits = num_oarcs;
  RaggedShape ostate_to_oarcs =
      RaggedShape2(&ostate_to_oarcs_row_splits, nullptr, -1);

  // `full_shape` has 5 axes: [fsa][orig_state][foo][ostate][oarc]
  RaggedShape full_shape =
      ComposeRaggedShapes(to_ostates_shape, ostate_to_oarcs);
  // for the lower-order row-splits and row-ids, use tot_row_{splits,idx}n_data
  const int32_t *full_row_splits4_data = full_shape.RowSplits(4).Data(),
                *full_row_ids4_data = full_shape.RowIds(4).Data();
  int32_t tot_oarcs = full_shape.NumElements();
  K2_CHECK_GE(tot_oarcs, fsas.NumElements());

  int32_t *fsas_arc_map_data = nullptr, *labels_arc_map_data = nullptr;
  if (fsas_arc_map) {
    *fsas_arc_map = Array1<int32_t>(c, tot_oarcs);
    fsas_arc_map_data = fsas_arc_map->Data();
  }
  if (labels_arc_map) {
    *labels_arc_map = Array1<int32_t>(c, tot_oarcs);
    labels_arc_map_data = labels_arc_map->Data();
  }
  Array1<Arc> oarcs(c, tot_oarcs);
  Arc *oarcs_data = oarcs.Data();
  const Arc *arcs_data = fsas.values.Data();

  K2_EVAL(
      c, tot_oarcs, lambda_set_arcs, (int32_t idx01234)->void {
        // All these indexes are into `full_shape`, indexed
        // `[fsa][state][foo][ostate][oarc].`
        int32_t idx0123 = full_row_ids4_data[idx01234],
                idx0123x = full_row_splits4_data[idx0123],
                idx4 = idx01234 - idx0123x, idx012 = tos_row_ids3_data[idx0123],
                idx012x = tos_row_splits3_data[idx012],
                idx3 = idx0123 - idx012x, idx01 = tos_row_ids2_data[idx012],
                idx01x = tos_row_splits2_data[idx01], idx2 = idx012 - idx01x,
                idx0 = tos_row_ids1_data[idx01],
                idx0x = tos_row_splits1_data[idx0],
                idx0xxx = tos_row_splits3_data[tos_row_splits2_data[idx0x]];

        int32_t fsa_idx01x = fsas_row_splits2_data[idx01];

        int32_t fsa_idx2;  // the idx2 (arc-index) into `fsas` of the input arc
                           // that's most relevant to us..
        int32_t seq_pos;  // seq_pos is our index into the sequence of arcs that
                          // we produce for each original arc
        if (idx2 == 0) {
          K2_CHECK_EQ(idx3, 0);
          fsa_idx2 = idx4;  // corresponds to foo=0, so idx3 will be 0; the idx4
                            // enumerates the arcs leaving it..
          seq_pos = 0;
        } else {
          // this is one of the extra `foo` indexes, one per arc in the input
          // FSA that leaves this state; each of those `foo` indexes has
          // (seq_len - 1) states in it (idx3=0,1..seq_len-1); and each state
          // has one arc leaving it (idx4==0).
          K2_CHECK_EQ(idx4, 0);
          fsa_idx2 = idx2 - 1;
          seq_pos = idx3 + 1;
        }
        int32_t fsa_idx012 = fsa_idx01x + fsa_idx2;  // index of the arc in
                                                     // source FSA FSA that
                                                     // we're expanding..
        Arc iarc = arcs_data[fsa_idx012];

        int32_t labels_idx0x = labels_row_splits1_data[fsa_idx012],
                labels_next_idx0x = labels_row_splits1_data[fsa_idx012 + 1],
                labels_len1 = labels_next_idx0x - labels_idx0x;
        // labels_len1 is length of label sequence for this arc
        K2_CHECK_LT(seq_pos, max(int32_t(1), labels_len1));

        int32_t dest_idx01 = idx0x + iarc.dest_state,  // original destination
                                                       // state-index
            orig_dest_idx0123 =
                tos_row_splits3_data[tos_row_splits2_data[dest_idx01]];

        Arc oarc;
        oarc.src_state = idx0123 - idx0xxx;
        // If this is the last arc in the sequence, the dest-state is the
        // original dest-state of the arc.  Otherwise the dest-state is one of
        // the new states that we created. The idx123 will be an idx1 after
        // removing axes.
        int32_t dest_idx123;
        if (seq_pos + 1 >= labels_len1) {  // last arc in sequence..
          dest_idx123 = orig_dest_idx0123 - idx0xxx;
        } else {
          int32_t dest_state_idx2 = fsa_idx2 + 1,  // index `foo` equals
                                                   // orig_arc_idx+1
              dest_state_idx3 = seq_pos,           // ostate index..
              dest_idx012 = idx01x + dest_state_idx2,
                  dest_idx012x = tos_row_splits3_data[dest_idx012],
                  dest_idx0123 = dest_idx012x + dest_state_idx3;
          dest_idx123 = dest_idx0123 - idx0xxx;
        }
        oarc.dest_state = dest_idx123;  // indexes 1,2,3 will be combined; in
                                        // the output FSA it will be an idx1.

        if (fsas_arc_map_data)
          fsas_arc_map_data[idx01234] = (seq_pos == 0 ? fsa_idx012 : -1);
        if (labels_arc_map_data)
          labels_arc_map_data[idx01234] =
              (seq_pos < labels_len1 ? labels_idx0x + seq_pos : -1);
        if (iarc.label != -1) {
          // normal case.. label goes on 1st arc in sequence
          oarc.label = (seq_pos == 0 ? iarc.label : 0);
        } else {
          // If the arc was to the final-state, we need to keep the label on the
          // last arc of the sequence to keep the output valid.  The following
          // would be "seq_pos + 1 == labels_len1 ? -1 : 0", but we make it ">="
          // not "=" to account for the case seq_pos=0, labels_len1 = 0.
          oarc.label = (seq_pos + 1 >= labels_len1 ? -1 : 0);
        }
        oarc.score = (seq_pos == 0 ? iarc.score : 0.0);
        oarcs_data[idx01234] = oarc;
      });

  // remove current axes 1 and 2... [after removing axis 1, old axis 2 becomes
  // axis 1, so remove axis 1 twice].
  RaggedShape temp = RemoveAxis(full_shape, 1);
  return FsaVec(RemoveAxis(temp, 1), oarcs);
}


void Invert(FsaOrVec &src, Ragged<int32_t> &src_aux_labels, FsaOrVec *dest,
            Ragged<int32_t> *dest_aux_labels,
            Array1<int32_t> *arc_map /*= nullptr*/) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(src_aux_labels.NumAxes(), 2);
  K2_CHECK_EQ(src_aux_labels.Dim0(), src.NumElements());
  K2_CHECK(dest != nullptr && dest_aux_labels != nullptr);
  ContextPtr c = GetContext(src, src_aux_labels);
  if (src.NumAxes() == 2) {
    Fsa *srcs = &src;
    FsaVec src_vec = CreateFsaVec(1, &srcs), dest_vec;
    Invert(src_vec, src_aux_labels, &dest_vec, dest_aux_labels, arc_map);
    *dest = GetFsaVecElement(dest_vec, 0);
    return;
  }
  Array1<int32_t> src_arc_map, labels_arc_map;
  *dest = ExpandArcs(src, src_aux_labels.shape, &src_arc_map, &labels_arc_map);
  // swap labels and aux_labels
  int32_t dest_num_arcs = dest->NumElements();
  Arc *dest_arcs_data = dest->values.Data();
  const int32_t *labels_arc_map_data = labels_arc_map.Data(),
                *src_aux_labels_data = src_aux_labels.values.Data();
  Array1<int32_t> dest_aux_labels_row_splits(c, dest_num_arcs + 1);
  int32_t *dest_aux_labels_row_splits_data = dest_aux_labels_row_splits.Data();
  K2_EVAL(
      c, dest_num_arcs, lambda_set_dest_aux_labels_num,
      (int32_t dest_idx012)->void {
        Arc &dest_arc = dest_arcs_data[dest_idx012];
        // we'll remove epsilons in dest_aux_labels
        dest_aux_labels_row_splits_data[dest_idx012] =
            dest_arc.label == 0 ? 0 : 1;
      });
  ExclusiveSum(dest_aux_labels_row_splits.Arange(0, dest_num_arcs),
               &dest_aux_labels_row_splits);
  RaggedShape dest_aux_labels_shape =
      RaggedShape2(&dest_aux_labels_row_splits, nullptr, -1);
  Array1<int32_t> dest_aux_labels_values(c,
                                         dest_aux_labels_shape.NumElements());
  int32_t *dest_aux_labels_values_data = dest_aux_labels_values.Data();
  K2_EVAL(
      c, dest_num_arcs, lambda_set_dest_labels_and_aux_labels,
      (int32_t dest_idx012)->void {
        Arc &dest_arc = dest_arcs_data[dest_idx012];
        // swap label and aux_label
        if (dest_arc.label != 0) {
          int32_t dest_aux_labels_idx0x =
              dest_aux_labels_row_splits_data[dest_idx012];
          // every arc in dest has at most one aux_label (as the aux_label is
          // the label of src on this arc)
          dest_aux_labels_values_data[dest_aux_labels_idx0x] = dest_arc.label;
        }
        int32_t src_aux_labels_idx01 = labels_arc_map_data[dest_idx012];
        dest_arc.label = src_aux_labels_idx01 == -1
                             ? 0
                             : src_aux_labels_data[src_aux_labels_idx01];
      });
  *dest_aux_labels =
      Ragged<int32_t>(dest_aux_labels_shape, dest_aux_labels_values);
  if (arc_map != nullptr) *arc_map = src_arc_map;
}

// Will be used in InvertHost to process FsaVec input recursively.
void RecursionWrapperAuxLabels(void (*f)(FsaOrVec &, Ragged<int32_t> &,
                                         FsaOrVec *, Ragged<int32_t> *),
                               FsaOrVec &src, Ragged<int32_t> &src_aux_labels,
                               FsaOrVec *dest,
                               Ragged<int32_t> *dest_aux_labels) {
  NVTX_RANGE(K2_FUNC);
  // src is actually an FsaVec.  Just recurse for now.
  K2_CHECK_EQ(src.NumAxes(), 3);
  int32_t num_fsas = src.shape.Dim0();
  std::vector<Fsa> srcs(num_fsas), dests(num_fsas);
  std::vector<Ragged<int32_t>> src_aux_labels_vec(num_fsas),
      dest_aux_labels_vec(num_fsas);
  int32_t tot_num_arcs = 0;
  Array1<int32_t> src_aux_labels_row_splits = src_aux_labels.RowSplits(1),
                  src_aux_labels_values = src_aux_labels.values;
  for (int32_t i = 0; i < num_fsas; ++i) {
    srcs[i] = src.Index(0, i);
    int32_t cur_num_arcs = srcs[i].NumElements();
    // below block get aux_labels for srcs[i]
    // TODO(haowen): replace with Range op for ragged
    {
      Array1<int32_t> row_splits = src_aux_labels_row_splits.Arange(
          tot_num_arcs, tot_num_arcs + cur_num_arcs + 1);
      Array1<int32_t> values =
          src_aux_labels_values.Arange(row_splits[0], row_splits.Back());
      row_splits = Minus(row_splits, row_splits[0]);
      RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
      src_aux_labels_vec[i] = Ragged<int32_t>(shape, values);
    }
    f(srcs[i], src_aux_labels_vec[i], &(dests[i]), &(dest_aux_labels_vec[i]));
    tot_num_arcs += cur_num_arcs;
  }
  *dest = Stack(0, num_fsas, dests.data());
  *dest_aux_labels = Cat(0, num_fsas, dest_aux_labels_vec.data());
}

void InvertHost(FsaOrVec &src, Ragged<int32_t> &src_aux_labels, FsaOrVec *dest,
                Ragged<int32_t> *dest_aux_labels) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK_EQ(src_aux_labels.NumAxes(), 2);
  K2_CHECK_EQ(src_aux_labels.Dim0(), src.NumElements());
  K2_CHECK(dest != nullptr && dest_aux_labels != nullptr);
  int32_t num_axes = src.NumAxes();
  if (num_axes < 2 || num_axes > 3) {
    K2_LOG(FATAL) << "Input has bad num-axes " << num_axes;
  } else if (num_axes == 3) {
    return RecursionWrapperAuxLabels(InvertHost, src, src_aux_labels, dest,
                                     dest_aux_labels);
  }

  k2host::Fsa host_fsa = FsaToHostFsa(src);
  // k2host::AuxLabels is a k2host::Array2
  k2host::AuxLabels host_aux_labels(
      src_aux_labels.Dim0(), src_aux_labels.NumElements(),
      src_aux_labels.RowSplits(1).Data(), src_aux_labels.values.Data());
  k2host::FstInverter inverter(host_fsa, host_aux_labels);
  k2host::Array2Size<int32_t> fsa_size, aux_size;
  inverter.GetSizes(&fsa_size, &aux_size);
  FsaCreator fsa_creator(fsa_size);
  k2host::Fsa host_dest_fsa = fsa_creator.GetHostFsa();
  Ragged2Creator<int32_t> ragged_creator(aux_size);
  k2host::AuxLabels host_dest_aux_labels = ragged_creator.GetHostArray2();
  inverter.GetOutput(&host_dest_fsa, &host_dest_aux_labels);
  *dest = fsa_creator.GetFsa();
  *dest_aux_labels = ragged_creator.GetRagged2();
}

FsaOrVec ReplaceFsa(FsaVec &src, FsaOrVec &index, int32_t symbol_range_begin,
                    Array1<int32_t> *arc_map_src /* = nullptr */,
                    Array1<int32_t> *arc_map_index /* = nullptr */) {
  NVTX_RANGE(K2_FUNC);
  if (index.NumAxes() == 2) {
    FsaVec index_temp = FsaToFsaVec(index);
    return ReplaceFsa(src, index_temp, symbol_range_begin, arc_map_src,
                      arc_map_index).RemoveAxis(0);
  }
  K2_CHECK_EQ(index.NumAxes(), 3);
  ContextPtr &c = index.Context();
  K2_CHECK(c->IsCompatible(*src.Context()));

  RaggedShape state_to_arcs = GetLayer(index.shape, 1);

  // `state_to_foo` is a RaggedShape that, for each state in `index`, has a list
  // of length `tot_arcs + 1`.  Interpret this as: one element for the state
  // itself, then one for each arc leaving it.  This `foo` is an index that
  // corresponds to num-arcs plus one, but because it is really a placeholder
  // and we want to keep it distinct from other things, we call it `foo`.
  RaggedShape state_to_foo = ChangeSublistSize(state_to_arcs, 1);

  int32_t foo_size = state_to_foo.NumElements(),
          num_src_fsas = src.Dim0();
  // For each element of `state_to_foo`, `num_ostates_for` says how many states
  // there will be for this (state,foo) in the returned (output) FSA.  Here, the
  // idx0 is the state, the idx1 is foo.  If idx1 == 0 (interpret this as "the
  // state itself"), then `num_ostates_for[idx01] = 1`, meaning "keep the
  // original state".  Otherwise, idx1 - 1 represents an arc_idx2 [into `index`]
  // and we set `num_ostates_for[idx01] = max(0, state_num-1)`, where state_num
  // is the states number of the fsa in `src` that would repalce into this arc,
  // the final state of this fsa will identify with the dest-state of this arc,
  // so we minus 1.
  Array1<int32_t> num_ostates_for(c, foo_size + 1);
  int32_t *num_ostates_for_data = num_ostates_for.Data();
  const Arc *index_arcs_data = index.values.Data();

  const int32_t *src_row_splits1_data = src.RowSplits(1).Data(),
                *index_row_splits2_data = index.RowSplits(2).Data(),
                *state_to_foo_row_splits1_data =
                    state_to_foo.RowSplits(1).Data(),
                *state_to_foo_row_ids1_data = state_to_foo.RowIds(1).Data();

  K2_EVAL(
      c, foo_size, lambda_set_num_ostates, (int32_t idx01)->void {
        // note: the idx01, idx0, idx0x are into `state_to_foo`.
        // This idx0 is a state-index into `index` (an idx01 w.r.t. `index`).
        int32_t idx0 = state_to_foo_row_ids1_data[idx01],
                idx0x = state_to_foo_row_splits1_data[idx0],
                idx1 = idx01 - idx0x;  // idx1 is `foo`.
        int32_t num_ostates;
        if (idx1 == 0) {
          num_ostates = 1;  // this is a copy of the original state.
        } else {
          int32_t index_arc_idx2 = idx1 - 1, index_state_idx01 = idx0,
                  index_arc_idx01x = index_row_splits2_data[index_state_idx01],
                  index_arc_idx012 = index_arc_idx01x + index_arc_idx2,
                  index_label = index_arcs_data[index_arc_idx012].label,
                  src_idx0 = index_label - symbol_range_begin;
          // will not replace for this arc
          if (src_idx0 < 0 || src_idx0 >= num_src_fsas) {
            num_ostates = 0;
          } else {
            int32_t src_idx0x = src_row_splits1_data[src_idx0],
                    src_idx0x_next = src_row_splits1_data[src_idx0 + 1],
                    src_len1 = src_idx0x_next - src_idx0x;
            num_ostates = max(src_len1 - 1, (int32_t)0);
          }
        }
        num_ostates_for_data[idx01] = num_ostates;
      });
  ExclusiveSum(num_ostates_for, &num_ostates_for);
  Array1<int32_t> &foo_to_ostates_row_splits = num_ostates_for;
  RaggedShape foo_to_ostates =
      RaggedShape2(&foo_to_ostates_row_splits, nullptr, -1);

  // to_ostates_shape has 4 axes: [fsa_id][orig_state][foo][ostate]
  // where foo is a general-purpose index that ranges over the (num_arcs + 1) of
  // the original state.
  RaggedShape to_ostates_shape = ComposeRaggedShapes3(
      GetLayer(index.shape, 0), state_to_foo, foo_to_ostates);

  // Below, `tos` means `to_ostates_shape`.
  const int32_t *tos_row_splits1_data = to_ostates_shape.RowSplits(1).Data(),
                *tos_row_ids1_data = to_ostates_shape.RowIds(1).Data(),
                *tos_row_splits2_data = to_ostates_shape.RowSplits(2).Data(),
                *tos_row_ids2_data = to_ostates_shape.RowIds(2).Data(),
                *tos_row_splits3_data = to_ostates_shape.RowSplits(3).Data(),
                *tos_row_ids3_data = to_ostates_shape.RowIds(3).Data(),
                *src_row_splits2_data = src.RowSplits(2).Data();

  // `num_oarcs` gives the number of arcs in the returned (output) FSA for each
  // `ostate` (i.e. leaving each state in the returned FSA).
  int32_t tot_ostates = to_ostates_shape.NumElements();
  Array1<int32_t> num_oarcs(c, tot_ostates + 1);
  int32_t *num_oarcs_data = num_oarcs.Data();
  K2_EVAL(
      c, tot_ostates, lambda_set_num_oarcs, (int32_t idx0123)->void {
        // All these indexes are into `to_ostates_shape`, indexed
        // `[fsa][state][foo][ostate].`
        int32_t idx012 = tos_row_ids3_data[idx0123],
                idx012x = tos_row_splits3_data[idx012],
                idx01 = tos_row_ids2_data[idx012],
                idx01x = tos_row_splits2_data[idx01],
                idx01x_next = tos_row_splits2_data[idx01 + 1],
                len2 = idx01x_next - idx01x, idx2 = idx012 - idx01x,
                idx3 = idx0123 - idx012x;
        int32_t num_arcs;
        if (idx2 == 0) {
          K2_CHECK_EQ(idx3, 0);
          // This ostate corresponds to the original state;
          // The original state had `orig_num_arcs` leaving it, which is the
          // number of `foo` indexes minus one.
          int32_t orig_num_arcs = len2 - 1;
          num_arcs = orig_num_arcs;
        } else {
          // All inserted states have the same num of arcs as in the src.
          // note: the prefix `index_` means it is an idxXXX w.r.t. `index`.
          // the prefix `src_` means the variable is an idxXXX w.r.t. `src`.
          int32_t index_arc_idx2 = idx2 - 1,
                  index_arc_idx01x = index_row_splits2_data[idx01],
                  index_arc_idx012 = index_arc_idx01x + index_arc_idx2,
                  index_label = index_arcs_data[index_arc_idx012].label,
                  src_fsa_idx0 = index_label - symbol_range_begin;
          K2_CHECK_GE(src_fsa_idx0, 0);
          K2_CHECK_LT(src_fsa_idx0, num_src_fsas);
          int32_t src_state_idx1 = idx3,
                  src_state_idx0x = src_row_splits1_data[src_fsa_idx0],
                  src_state_idx01 = src_state_idx0x + src_state_idx1,
                  src_arc_idx01x = src_row_splits2_data[src_state_idx01],
                  src_arc_idx01x_next =
                    src_row_splits2_data[src_state_idx01 + 1],
                  src_num_arcs = src_arc_idx01x_next - src_arc_idx01x;
          num_arcs = src_num_arcs;
        }
        num_oarcs_data[idx0123] = num_arcs;
      });
  ExclusiveSum(num_oarcs, &num_oarcs);
  Array1<int32_t> &ostate_to_oarcs_row_splits = num_oarcs;
  RaggedShape ostate_to_oarcs =
      RaggedShape2(&ostate_to_oarcs_row_splits, nullptr, -1);

  // `full_shape` has 5 axes: [fsa][orig_state][foo][ostate][oarc]
  RaggedShape full_shape =
      ComposeRaggedShapes(to_ostates_shape, ostate_to_oarcs);

  // for the lower-order row-splits and row-ids, use tot_row_{splits,ids}n_data
  const int32_t *full_row_splits4_data = full_shape.RowSplits(4).Data(),
                *full_row_ids4_data = full_shape.RowIds(4).Data();
  int32_t tot_oarcs = full_shape.NumElements();
  K2_CHECK_GE(tot_oarcs, index.NumElements());

  int32_t *arc_map_src_data = nullptr, *arc_map_index_data = nullptr;
  if (arc_map_src) {
    *arc_map_src = Array1<int32_t>(c, tot_oarcs);
    arc_map_src_data = arc_map_src->Data();
  }
  if (arc_map_index) {
    *arc_map_index = Array1<int32_t>(c, tot_oarcs);
    arc_map_index_data = arc_map_index->Data();
  }
  Array1<Arc> oarcs(c, tot_oarcs);
  Arc *oarcs_data = oarcs.Data();
  const Arc *src_arcs_data = src.values.Data();

  K2_EVAL(
      c, tot_oarcs, lambda_set_arcs, (int32_t idx01234)->void {
        // All these indexes are into `full_shape`, indexed
        // `[fsa][state][foo][ostate][oarc].`
        // The prefix `index_` means it is an idxXXX w.r.t. `index`.
        // the prefix `src_` means the variable is an idxXXX w.r.t. `src`.
        int32_t idx0123 = full_row_ids4_data[idx01234],
                idx0123x = full_row_splits4_data[idx0123],
                idx4 = idx01234 - idx0123x,
                idx012 = tos_row_ids3_data[idx0123],
                idx012x = tos_row_splits3_data[idx012],
                idx3 = idx0123 - idx012x,
                idx01 = tos_row_ids2_data[idx012],
                idx01x = tos_row_splits2_data[idx01],
                idx2 = idx012 - idx01x,
                idx0 = tos_row_ids1_data[idx01],
                idx0x = tos_row_splits1_data[idx0],
                idx0xxx = tos_row_splits3_data[tos_row_splits2_data[idx0x]];

        int32_t index_arc_idx2;  // the idx2 (arc-index) into `index`
        if (idx2 == 0) {
          K2_CHECK_EQ(idx3, 0);
          index_arc_idx2 = idx4;  // corresponds to foo=0, so idx3 will be 0;
                                  // the idx4 enumerates the arcs leaving it..
        } else {
          // this is one of the extra `foo` indexes, it's corresponding index
          // into `index` is `foo` index minus 1
          index_arc_idx2 = idx2 - 1;
        }

        int32_t index_arc_idx01x = index_row_splits2_data[idx01];
        // index of the arc in source FSA, FSA that we're replacing..
        int32_t index_arc_idx012 = index_arc_idx01x + index_arc_idx2;

        Arc index_arc = index_arcs_data[index_arc_idx012];
        // original destination state-index
        int32_t dest_state_idx01 = idx0x + index_arc.dest_state,
                orig_dest_state_idx0123 =
                  tos_row_splits3_data[tos_row_splits2_data[dest_state_idx01]];

        Arc src_arc;
        Arc oarc;
        oarc.src_state = idx0123 - idx0xxx;
        // initialize mapping index
        int32_t arc_src_map_idx = -1,
                arc_index_map_idx = -1;
        int32_t src_fsa_idx0 = index_arc.label - symbol_range_begin;
        // will not replace for this arc
        // dest state is the dest state of index arc
        if (src_fsa_idx0 < 0 || src_fsa_idx0 >= num_src_fsas) {
          K2_CHECK_EQ(idx2, 0);
          oarc.dest_state = orig_dest_state_idx0123 - idx0xxx;
          oarc.label = index_arc.label;
          oarc.score = index_arc.score;
          arc_index_map_idx = index_arc_idx012;
        } else {
          int32_t src_state_idx0x = src_row_splits1_data[src_fsa_idx0],
                  src_state_idx0x_next = src_row_splits1_data[src_fsa_idx0 + 1],
                  num_states = src_state_idx0x_next - src_state_idx0x,
                  src_state_idx1 = idx3,
                  src_state_idx01 = src_state_idx0x + src_state_idx1,
                  src_arc_idx01x = src_row_splits2_data[src_state_idx01],
                  src_arc_idx2 = idx4,
                  src_arc_idx012 = src_arc_idx01x + src_arc_idx2;
          src_arc = src_arcs_data[src_arc_idx012];
          // handle the arcs belongs to index
          if (idx2 == 0) {
            // if the fsa to be replaced in is empty, this arc would point to
            // its original dest-state
            if (0 == num_states) {
              oarc.dest_state = orig_dest_state_idx0123 - idx0xxx;
            } else {
              // this arc would point to the initial state of the fsa in src,
              // the state id bias to current state(the src-state) is the count
              // of all the ostates corresponding to the original state until
              // now, the idx4 enumerates foo index
              int32_t idx012_t = idx01x + 0,
                      idx2_t = idx4,
                      idx012x_t = tos_row_splits3_data[idx012_t],
                      idx012x_next_t =
                        tos_row_splits3_data[idx012_t + idx2_t + 1],
                      bias = idx012x_next_t - idx012x_t;
              oarc.dest_state = idx0123 + bias - idx0xxx;
            }
            // set the label of the arc we are replacing to be 0(epsilon)
            oarc.label = 0;
            oarc.score = index_arc.score;
            arc_index_map_idx = index_arc_idx012;
          } else {   // handle the arcs belongs to src
            // the arc point to the final state of the fsa in src would point to
            // the dest state of the arc we're replacing
            if (src_arc.label == -1) {
              oarc.dest_state = orig_dest_state_idx0123 - idx0xxx;
            } else {
              // this is the inner arc of the fsa in src
              int32_t dest_state_idx012x = idx0123 - idx3,
                  dest_state_idx0123 = dest_state_idx012x + src_arc.dest_state;
              oarc.dest_state = dest_state_idx0123 - idx0xxx;
            }
            // arcs in src fsas that point to final state would set to epsilon
            // arc (label from -1 to 0)
            oarc.label = src_arc.label == -1 ? 0 : src_arc.label;
            oarc.score = src_arc.score;
            arc_src_map_idx = src_arc_idx012;
          }
        }
        if (arc_map_src_data)
          arc_map_src_data[idx01234] = arc_src_map_idx;
        if (arc_map_index_data)
          arc_map_index_data[idx01234] = arc_index_map_idx;
        oarcs_data[idx01234] = oarc;
      });
  // remove current axes 1 and 2... [after removing axis 1, old axis 2 becomes
  // axis 1, so remove axis 1 twice].
  RaggedShape temp = RemoveAxis(full_shape, 1);
  return FsaVec(RemoveAxis(temp, 1), oarcs);
}

FsaOrVec RemoveEpsilonSelfLoops(FsaOrVec &src,
                                Array1<int32_t> *arc_map /* = nullptr */) {
  NVTX_RANGE(K2_FUNC);
  if (src.NumAxes() == 2) {
    FsaVec temp = FsaToFsaVec(src);
    return RemoveEpsilonSelfLoops(temp, arc_map).RemoveAxis(0);
  }
  K2_CHECK_EQ(src.NumAxes(), 3);

  ContextPtr &c = src.Context();
  int32_t num_arcs = src.NumElements();
  Renumbering renumber_lists(c, num_arcs);
  char *keep_list_data = renumber_lists.Keep().Data();

  const Arc *arcs_data = src.values.Data();
  K2_EVAL(
      c, num_arcs, lambda_set_keep, (int32_t i)->void {
        Arc arc = arcs_data[i];
        char keep;
        if (arc.label == 0 && arc.src_state == arc.dest_state) {
          // This arc is an epsilon self-loop, so it should be removed
          keep = 0;
        } else {
          keep = 1;
        }
        keep_list_data[i] = keep;
      });
  FsaVec ans = Index(src, 2, renumber_lists.New2Old(), arc_map);
  return ans;
}

FsaVec GenerateDenominatorLattice(Ragged<int32_t> &sampled_paths,
                                  Ragged<int32_t> &frame_ids,
                                  Ragged<int32_t> &left_symbols,
                                  Ragged<float> &sampling_probs,
                                  Array1<int32_t> &boundary,
                                  int32_t vocab_size,
                                  int32_t context_size,
                                  Array1<int32_t> *arc_map) {
  NVTX_RANGE(K2_FUNC);
  K2_CHECK(arc_map);
  K2_CHECK_EQ(sampled_paths.NumAxes(), 3);
  K2_CHECK_EQ(frame_ids.NumAxes(), 3);
  K2_CHECK_EQ(left_symbols.NumAxes(), 4);
  K2_CHECK_EQ(sampling_probs.NumAxes(), 3);

  K2_DCHECK_EQ(sampled_paths.NumElements(), frame_ids.NumElements());
  K2_DCHECK_EQ(sampled_paths.NumElements(),
      left_symbols.NumElements() * context_size);
  K2_DCHECK_EQ(sampled_paths.NumElements(), sampling_probs.NumElements());
  K2_DCHECK_EQ(sampled_paths.TotSize(0), boundary.Dim());
  for (int32_t i = 0; i < 3; ++i) {
    K2_DCHECK_EQ(sampled_paths.TotSize(i), frame_ids.TotSize(i));
    K2_DCHECK_EQ(sampled_paths.TotSize(i), left_symbols.TotSize(i));
    K2_DCHECK_EQ(sampled_paths.TotSize(i), sampling_probs.TotSize(i));
  }

  ContextPtr c = GetContext(
      sampled_paths, frame_ids, left_symbols, sampling_probs);

  // The states indicating we are in on each position of each path, which has
  // the same shape as `sampled_paths`, because each symbol in the paths is
  // sampled from a specific frame with corresponding left contexts.
  // Each state represents a tuple like (t, left_symbols1, left_symbols2...),
  // the number of left_symbols equals to the `context_size`. A state is
  // calculated from t * V ^ c + \sum_{i=1}^{c} s_i * V ^ (c - i),
  // V is vocab_size, c is context_size, s_i is the ith left_symbols.
  // For example, if context_size = 2, vocab_size = 10, so, one possible tuple
  // would be (2, 4, 5), then the corresponding state is
  // 2 * 10 ^ 2 + 4 * 10 + 5 = 245.
  Ragged<int64_t> states(sampled_paths.shape);
  int32_t num_states = states.NumElements();

  const int32_t *frame_ids_data = frame_ids.values.Data(),
                *left_symbols_row_splits3_data
                    = left_symbols.RowSplits(3).Data(),
                *left_symbols_data = left_symbols.values.Data();
  int64_t *states_data = states.values.Data();

  // This kernel calculates t * V ^ c for each state.
  K2_EVAL(
      c, num_states, lambda_init_states_with_t, (int32_t idx012) -> void {
        states_data[idx012]
            = frame_ids_data[idx012] * Pow(vocab_size, context_size);
  });

  // The following kernels calculate \sum_{i=1}^{c} s_i * V ^ (c - i)
  for (int32_t i = 0; i < context_size; ++i) {
    K2_EVAL(
        c, num_states, lambda_generate_states, (int32_t idx012) -> void {
          int32_t left_symbols_idx012x = left_symbols_row_splits3_data[idx012],
                  left_symbols_idx0123 = left_symbols_idx012x + i,
                  exp = context_size - i - 1;
          states_data[idx012]
              += left_symbols_data[left_symbols_idx0123] * Pow(vocab_size, exp);
    });
  }

  // Sort those states for each sequence, so as to merge the same states.
  // sorted_states has two axes: [seq][state]
  auto sorted_states = Ragged<int64_t>(
      RemoveAxis(states.shape, 1 /*axis*/), states.values.Clone());
  Array1<int32_t> sorted_states_new2old(c, num_states);
  SortSublists<int64_t>(&sorted_states, &sorted_states_new2old);

  // We need old2new map to find the original consecutive state.
  Array1<int32_t> sorted_states_old2new(c, num_states);
  const int32_t *sorted_states_new2old_data = sorted_states_new2old.Data();
  int32_t *sorted_states_old2new_data = sorted_states_old2new.Data();
  K2_EVAL(
      c, num_states, lambda_get_old2new, (int32_t i) -> void {
      sorted_states_old2new_data[sorted_states_new2old_data[i]] = i;
  });

  // Search "tails concept" in k2/csrc/utils.h for the details of tail array.
  // By applying ExclusiveSum on the tail_array, we can get a row_id mapping the
  // sorted states to unique_states (i.e. the merged states).
  Array1<int32_t> tail_array(c, num_states);
  const int32_t *sorted_states_row_ids1_data = sorted_states.RowIds(1).Data();
  const int64_t *sorted_states_data = sorted_states.values.Data();
  int32_t *tail_array_data = tail_array.Data();

  K2_EVAL(
      c, num_states, lambda_get_tail_array, (int32_t idx01) -> void {
      if (idx01 == num_states - 1) tail_array_data[idx01] = 1;
      int32_t idx0 = sorted_states_row_ids1_data[idx01],
              next_idx0 = sorted_states_row_ids1_data[idx01 + 1];
      if (idx0 == next_idx0 &&
          sorted_states_data[idx01] == sorted_states_data[idx01 + 1])
          tail_array_data[idx01] = 0;
      else
          tail_array_data[idx01] = 1;
  });

  Array1<int32_t> unique_states_row_ids(c, num_states);
  ExclusiveSum(tail_array, &unique_states_row_ids);

  // unique_states_shape's shape [merged state][sorted state]
  // unique_states_shape.row_splits.Dim() - 1 equals to the number of merged
  // states.
  RaggedShape unique_states_shape = RaggedShape2(
      nullptr, &unique_states_row_ids, unique_states_row_ids.Dim());

  // We are figuring out the ragged shape of the lattice.
  // First, figure out the number of states (i.e. the merged states) for each
  // sequence.
  // Second, figure out the number of arcs for each merged state.
  int32_t num_seqs = states.TotSize(0);

  // Plus 1 here because we will apply ExclusiveSum on this array.
  Array1<int32_t> num_states_for_seqs(c, states.TotSize(0) + 1);

  // "ss" is short for "sorted states"
  // "us" is short for "unique states".
  const int32_t *ss_row_splits1_data = sorted_states.RowSplits(1).Data(),
                *us_row_ids1_data = unique_states_shape.RowIds(1).Data();
  int32_t *num_states_for_seqs_data = num_states_for_seqs.Data();

  K2_EVAL(
      c, num_seqs, lambda_get_num_states, (int32_t idx0) -> void {
      int32_t ss_idx0x = ss_row_splits1_data[idx0],
              ss_idx0x_next = ss_row_splits1_data[idx0 + 1],
              us_idx0 = us_row_ids1_data[ss_idx0x],
              us_idx0_next_minus_1 = us_row_ids1_data[ss_idx0x_next - 1],
              num_unique_states = us_idx0_next_minus_1 - us_idx0 + 1;
      // Plus 3 here, because we need a super dest_state for the states sampled
      // on the last frame (this dest_state will point to the final state),
      // a fake super dest_state for the last states of linear paths that
      // are not sampled on the last frames (this fake dest_state will be
      // removed by connect operation), and a final state needed by k2.
      num_states_for_seqs_data[idx0] = num_unique_states + 3;
  });

  ExclusiveSum(num_states_for_seqs, &num_states_for_seqs);
  RaggedShape seqs_to_states_shape = RaggedShape2(
      &num_states_for_seqs, nullptr, -1);
  int32_t num_merged_states = seqs_to_states_shape.NumElements();

  K2_CHECK_EQ(unique_states_shape.RowSplits(1).Dim() - 1 + num_seqs * 3,
              num_merged_states);

  // Plus 1 here because we will apply ExclusiveSum on this array.
  Array1<int32_t> num_arcs_for_states(
      c, seqs_to_states_shape.NumElements() + 1);

  // "sts" is short for "seqs to states"
  // "us" is short for "unique states".
  const int32_t *us_row_splits1_data = unique_states_shape.RowSplits(1).Data(),
                *sts_row_ids1_data = seqs_to_states_shape.RowIds(1).Data(),
                *sts_row_splits1_data
                  = seqs_to_states_shape.RowSplits(1).Data();
  int32_t *num_arcs_for_states_data = num_arcs_for_states.Data();

  K2_EVAL(
      c, num_merged_states, lambda_get_num_arcs, (int32_t idx01) -> void {
      int32_t idx0 = sts_row_ids1_data[idx01],
              idx0x_next = sts_row_splits1_data[idx0 + 1],
              num_arcs = 0;
      // The final arc for each sequence.
      if (idx01 == idx0x_next - 2) num_arcs = 1;
      if (idx01 < idx0x_next - 3) {
          // Minus idx0 * 3, because we add extra three states for each sequence.
          int32_t us_idx0 = idx01 - idx0 * 3,
                  us_idx0x = us_row_splits1_data[us_idx0],
                  us_idx0x_next = us_row_splits1_data[us_idx0 + 1];
          num_arcs = us_idx0x_next - us_idx0x;
     }
     // idx01 == idx0x_next - 3 (i.e. the fake super dest_state) and
     // idx01 == idx0x_next -1 (i.e. the final state) don't have arcs.
     num_arcs_for_states_data[idx01] = num_arcs;
  });

  ExclusiveSum(num_arcs_for_states, &num_arcs_for_states);
  RaggedShape states_to_arcs_shape = RaggedShape2(
      &num_arcs_for_states, nullptr, -1);

  RaggedShape arcs_shape = ComposeRaggedShapes(
      seqs_to_states_shape, states_to_arcs_shape);
  int32_t num_arcs = arcs_shape.NumElements();

  // Each state (before merging) has a leaving arc, we add a final arc
  // to each sequence, so, the total number of arcs equals to
  // num_states + num_seqs
  K2_CHECK_EQ(num_arcs, num_seqs + num_states);

  // Populate arcs.
  // "ss" is short for "sorted states"
  const int32_t *sampled_paths_data = sampled_paths.values.Data(),
                *arcs_shape_row_ids1_data = arcs_shape.RowIds(1).Data(),
                *arcs_shape_row_splits1_data = arcs_shape.RowSplits(1).Data(),
                *arcs_shape_row_ids2_data = arcs_shape.RowIds(2).Data(),
                *states_row_ids2_data = states.RowIds(2).Data(),
                *boundary_data = boundary.Data(),
                *ss_row_ids1_data = sorted_states.RowIds(1).Data();
  const float *sampling_probs_data = sampling_probs.values.Data();
  Array1<Arc> arcs(c, num_arcs);
  Arc *arcs_data = arcs.Data();

  // The arc_map mapping from lattice arcs to original state indexes.
  Array1<int32_t> raw_arc_map(c, num_arcs);
  int32_t *raw_arc_map_data = raw_arc_map.Data();

  K2_EVAL(
      c, num_arcs, lambda_set_arcs, (int32_t idx012) -> void {
      Arc arc;
      int32_t arc_map_value = -1;
      int32_t idx01 = arcs_shape_row_ids2_data[idx012],
              idx0 = arcs_shape_row_ids1_data[idx01],
              idx0x = arcs_shape_row_splits1_data[idx0],
              idx1 = idx01 - idx0x;
      arc.src_state = idx1;

      // Final arc of the last sequence.
      if (idx012 == num_arcs - 1) {
        arc.dest_state = idx1 + 1;
        arc.label = -1;
        arc.score = 0.0;
      } else {
        int32_t idx01_next = arcs_shape_row_ids2_data[idx012 + 1],
                idx0_next = arcs_shape_row_ids1_data[idx01_next];
        // Final arc for each sequence, except the last sequence.
        if (idx0 != idx0_next) {
          arc.dest_state = idx1 + 1;
          arc.label = -1;
          arc.score = 0.0;
        } else {
          // ss_idx01 is the global index of sorted states, minus idx0 here
          // because we added an extra final arc for each sequence.
          int32_t ss_idx01 = idx012 - idx0,
                  states_idx012 = sorted_states_new2old_data[ss_idx01];

          arc_map_value = states_idx012;
          arc.label = sampled_paths_data[states_idx012];
          float sampling_prob = sampling_probs_data[states_idx012];

          int32_t us_idx0 = us_row_ids1_data[ss_idx01],
                  repeat_num = us_row_splits1_data[us_idx0 + 1] -
                    us_row_splits1_data[us_idx0];

          float score = -logf(1 - powf(1 - sampling_prob, repeat_num));
          if (score - score != 0) {
            arc.score = 0.0;
          } else {
            arc.score = score;
          }

          K2_DCHECK_LT(frame_ids_data[states_idx012], boundary_data[idx0]);

          int32_t idx0x_next = arcs_shape_row_splits1_data[idx0 + 1];

          // Handle the final state of last sequence.
          if (states_idx012 == num_states - 1) {
            // If current state is on final frame, it will point to the added
            // super dest_state.
            if (frame_ids_data[states_idx012] == boundary_data[idx0] - 1) {
              arc.dest_state = idx0x_next - idx0x - 2;
            } else {
              // point to the fake added dest_state.
              arc.dest_state = idx0x_next - idx0x - 3;
            }
          } else {
            // states_idx01 is path index
            int32_t states_idx01 = states_row_ids2_data[states_idx012],
                    states_idx01_next =
                      states_row_ids2_data[states_idx012 + 1];
            if (states_idx01 != states_idx01_next) {
              // If current state is on final frame, it will point to the added
              // super dest_state.
              if (frame_ids_data[states_idx012] == boundary_data[idx0] - 1) {
                arc.dest_state = idx0x_next - idx0x - 2;
              } else {
                // point to the fake added dest_state.
                arc.dest_state = idx0x_next - idx0x - 3;
              }
            } else {
              // If current state is on final frame, it will point to the added
              // super dest_state.
              if (frame_ids_data[states_idx012] == boundary_data[idx0] - 1 &&
                  frame_ids_data[states_idx012 + 1] != boundary_data[idx0] - 1) {
                arc.dest_state = idx0x_next - idx0x - 2;
              } else {
                // states_idx012 + 1 is the index of original consecutive state.
                // "ss" is short for "sorted states"
                // "us" is short for "unique states".
                int32_t ss_idx01_next =
                  sorted_states_old2new_data[states_idx012 + 1],
                        us_idx0_next = us_row_ids1_data[ss_idx01_next];
                // Plus 3 * idx0, because we add 3 state for each sequence
                arc.dest_state = us_idx0_next +  3 * idx0 - idx0x;
              }
            }
          }
        }
      }
     arcs_data[idx012] = arc;
     raw_arc_map_data[idx012] = arc_map_value;
  });

  FsaVec fsas = Ragged<Arc>(arcs_shape, arcs);
  // arcsort so as to remove duplicate arcs.
  Array1<int32_t> arc_sort_new2old(c, num_arcs);
  SortSublists<Arc>(&fsas, &arc_sort_new2old);

  // remove duplicate arcs, use renumbering
  Renumbering renumber_arcs(c, num_arcs);
  char *keep_arcs_data = renumber_arcs.Keep().Data();
  K2_EVAL(
      c, num_arcs, lambda_set_keep_arcs, (int32_t idx012) -> void {
      char keep = 1;
      if (idx012 < num_arcs - 1) {
        int32_t idx01 = arcs_shape_row_ids2_data[idx012],
                idx01_next = arcs_shape_row_ids2_data[idx012 + 1];
        // duplicate arcs, which are arcs with the same symbol going from the
        // same src_state to the same dest_state. The symbol will automatically
        // be the same if the src_state and dest_state are the same if
        // context_size > 0.
        if (idx01 == idx01_next &&
            arcs_data[idx012].src_state == arcs_data[idx012 + 1].src_state &&
            arcs_data[idx012].dest_state == arcs_data[idx012 + 1].dest_state) {
          K2_DCHECK_EQ(arcs_data[idx012].label, arcs_data[idx012 + 1].label);
          keep = 0;
        }
      }
      keep_arcs_data[idx012] = keep;
  });

  Array1<int32_t> renumber_arc_map;
  FsaVec final_fsas = Index(
      fsas, 2, renumber_arcs.New2Old(), &renumber_arc_map);

  if (arc_map != nullptr) {
    *arc_map = raw_arc_map[arc_sort_new2old][renumber_arc_map];
  }
  return final_fsas;
}

}  // namespace k2
