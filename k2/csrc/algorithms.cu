/**
 * Copyright (c)  2020  Xiaomi Corporation (authors: Haowen Qiu)
 *
 * See LICENSE for clarification regarding multiple authors
 */

#include <algorithm>
#include <vector>

#include "k2/csrc/algorithms.h"
#include "k2/csrc/array.h"
#include "k2/csrc/array_ops.h"
#include "k2/csrc/macros.h"
#include "k2/csrc/nvtx.h"

namespace k2 {
void Renumbering::ComputeOld2New() {
  NVTX_RANGE(K2_FUNC);
  old2new_ = Array1<int32_t>(keep_.Context(), keep_.Dim() + 1);
  ExclusiveSum(keep_, &old2new_);
  num_new_elems_ = old2new_.Back();
  K2_CHECK_GE(num_new_elems_, 0);
  K2_CHECK_LE(num_new_elems_, keep_.Dim());
}

namespace {
// This small piece of code had to be put in a separate function due to
// CUDA limitations about lambdas in classes with private members.
inline void ComputeNew2OldHelper(ContextPtr &c, const int32_t *old2new_data,
                                 int32_t *new2old_data, int32_t old_dim) {
  NVTX_RANGE(K2_FUNC);
  // Note: the following accesses data one past the end of (current)
  // old2new_, but it does actually exist.

  K2_EVAL(
      c, old_dim + 1, lambda_set_new2old, (int32_t old_idx) {
        if (old_idx == old_dim ||
            old2new_data[old_idx + 1] > old2new_data[old_idx])
          new2old_data[old2new_data[old_idx]] = old_idx;
      });
}

}  // namespace

void Renumbering::ComputeNew2Old() {
  NVTX_RANGE(K2_FUNC);
  if (!old2new_.IsValid()) ComputeOld2New();
  new2old_ = Array1<int32_t>(keep_.Context(), num_new_elems_ + 1);
  const int32_t *old2new_data = old2new_.Data();
  int32_t *new2old_data = new2old_.Data();
  ComputeNew2OldHelper(keep_.Context(), old2new_data, new2old_data,
                       keep_.Dim());
  new2old_ = new2old_.Range(0, num_new_elems_);
}

Renumbering::Renumbering(const Array1<char> &keep,
                         const Array1<int32_t> &old2new,
                         const Array1<int32_t> &new2old):
    keep_(keep), old2new_(old2new),
    num_new_elems_(new2old.Dim()),
    new2old_(new2old) { }


Renumbering IdentityRenumbering(ContextPtr c, int32_t size) {
  Array1<char> keep(c, size + 1);  // uninitialized.
  keep = keep.Arange(0, size);
  Array1<int32_t> range = Arange(c, 0, size + 1);
  return Renumbering(keep, range, range.Arange(0, size));
}



}  // namespace k2
