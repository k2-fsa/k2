/**
 * @brief
 * algorithms
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Haowen Qiu)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <algorithm>
#include <vector>

#include "k2/csrc/algorithms.h"
#include "k2/csrc/array.h"
#include "k2/csrc/array_ops.h"

namespace k2 {
void Renumbering::ComputeOld2New() {
  old2new_ = Array1<int32_t>(keep_.Context(), keep_.Dim() + 1);
  ExclusiveSum(keep_, &old2new_);
  num_new_elems_ = old2new_.Back();
  K2_CHECK_GE(num_new_elems_, 0);
  K2_CHECK_LE(num_new_elems_, keep_.Dim());
  old2new_ = old2new_.Range(0, keep_.Dim());
}


namespace {
// This small piece of code had to be put in a separate function due to
// CUDA limitations about lambdas in classes with private members.
inline void ComputeNew2OldHelper(ContextPtr &c,
                                 const int32_t *old2new_data,
                                 int32_t *new2old_data,
                                 int32_t old_dim) {

  // caution: the following accesses data one past the end of (current)
  // old2new_, but it does actually exist.
  auto lambda_set_old2new = [=] __host__ __device__ (int32_t old_idx) {
    if (old2new_data[old_idx + 1] > old2new_data[old_idx])
      new2old_data[old2new_data[old_idx]] = old_idx;
  };
  Eval(c, old_dim, lambda_set_old2new);
}
}  // namespace

void Renumbering::ComputeNew2Old() {
  if  (!old2new_.IsValid())
    ComputeOld2New();
  new2old_ = Array1<int32_t>(keep_.Context(), num_new_elems_);

  const int32_t *old2new_data = old2new_.Data();
  int32_t *new2old_data = new2old_.Data();
  ComputeNew2OldHelper(keep_.Context(), old2new_data, new2old_data,
                       keep_.Dim());
}


}  // namespace k2
