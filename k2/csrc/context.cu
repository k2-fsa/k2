/**
 * @brief
 * context
 *
 * @copyright
 * Copyright (c)  2020  Mobvoi Inc.        (authors: Fangjun Kuang)
 *                      Xiaomi Corporation (author: Haowen Qiu)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include "k2/csrc/context.h"

namespace k2 {

RegionPtr NewRegion(ContextPtr &context, std::size_t num_bytes) {
  // .. fairly straightforward.  Sets bytes_used to num_bytes, caller can
  // overwrite if needed.
  auto ans = std::make_shared<Region>();
  ans->context = context;
  // TODO(haowen): deleter_context is always null with above constructor,
  // we need add another constructor of Region to allow the caller
  // to provide deleter_context.
  ans->data = context->Allocate(num_bytes, &ans->deleter_context);
  ans->num_bytes = num_bytes;
  ans->bytes_used = num_bytes;
  return ans;
}

ParallelRunner::ParallelRunner(ContextPtr c) : c_(c) {
  if (c_->GetDeviceType() == kCuda) {
    auto ret = hipEventCreate(&event_);
    K2_CHECK_CUDA_ERROR(ret);
    // record event on `c_->GetCudaStream` and will be waited on `NewStream`
    ret = hipEventRecord(event_, c_->GetCudaStream());
    K2_CHECK_CUDA_ERROR(ret);
  }
}
hipStream_t ParallelRunner::NewStream() {
  DeviceType d = c_->GetDeviceType();
  if (d == kCpu) {
    return kCudaStreamInvalid;
  } else {
    K2_CHECK_EQ(d, kCuda);
    hipStream_t stream;
    auto ret = hipStreamCreate(&stream);
    K2_CHECK_CUDA_ERROR(ret);
    streams_.push_back(stream);

    ret = hipStreamWaitEvent(stream, event_, 0);
    K2_CHECK_CUDA_ERROR(ret);
    return stream;
  }
}

ParallelRunner::~ParallelRunner() {
  if (c_->GetDeviceType() == kCuda) {
    for (std::size_t i = 0; i != streams_.size(); ++i) {
      // create and record event on `stream_[i]`, and wait on c_->GetCudaStream
      hipEvent_t event;
      auto ret = hipEventCreate(&event);
      K2_CHECK_CUDA_ERROR(ret);
      ret = hipEventRecord(event, streams_[i]);
      K2_CHECK_CUDA_ERROR(ret);
      ret = hipStreamWaitEvent(c_->GetCudaStream(), event, 0);
      K2_CHECK_CUDA_ERROR(ret);
      ret = hipEventDestroy(event);
      K2_CHECK_CUDA_ERROR(ret);
      ret = hipStreamDestroy(streams_[i]);
      K2_CHECK_CUDA_ERROR(ret);
    }
    // destroy event_
    auto ret = hipEventDestroy(event_);
    K2_CHECK_CUDA_ERROR(ret);
  }
}

}  // namespace k2
