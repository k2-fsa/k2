/**
 * @brief
 * ragged_test
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey, Haowen Qiu)
 *                      Mobvoi Inc.        (authors: Fangjun Kuang)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include <algorithm>
#include <numeric>
#include <utility>
#include <vector>

#include "k2/csrc/array.h"
#include "k2/csrc/array_ops.h"
#include "k2/csrc/context.h"
#include "k2/csrc/math.h"
#include "k2/csrc/ragged.h"
#include "k2/csrc/ragged_ops.h"
#include "k2/csrc/tensor.h"

namespace {
// TODO(haowen): may move below functions to some file like `test_utils.h`,
// in case other Tests may use it?
template <typename T>
static void CheckArrayData(const k2::Array1<T> &array,
                           const std::vector<T> &target) {
  ASSERT_EQ(array.Dim(), target.size());
  const T *array_data = array.Data();
  // copy data from CPU/GPU to CPU
  auto kind = k2::GetMemoryCopyKind(*array.Context(), *k2::GetCpuContext());
  std::vector<T> cpu_data(array.Dim());
  k2::MemoryCopy(static_cast<void *>(cpu_data.data()),
                 static_cast<const void *>(array_data),
                 array.Dim() * array.ElementSize(), kind, nullptr);
  EXPECT_EQ(cpu_data, target);
}

static void CheckRowSplits(k2::RaggedShape &shape,
                           const std::vector<std::vector<int32_t>> &target) {
  for (int32_t i = 1; i < shape.NumAxes(); ++i) {
    k2::Array1<int32_t> curr_row_splits = shape.RowSplits(i);
    CheckArrayData<int32_t>(curr_row_splits, target[i - 1]);
  }
}

// check if `array` and `target` have the same values
template <typename T>
static void CheckArrayData(const k2::Array1<T> &array,
                           const k2::Array1<T> &target) {
  ASSERT_EQ(array.Dim(), target.Dim());
  int32_t dim = array.Dim();
  k2::ContextPtr cpu = k2::GetCpuContext();
  k2::Array1<T> cpu_array = array.To(cpu);
  k2::Array1<T> cpu_target = target.To(cpu);
  std::vector<T> array_data(cpu_array.Data(), cpu_array.Data() + dim);
  std::vector<T> target_data(cpu_target.Data(), cpu_target.Data() + dim);
  EXPECT_EQ(array_data, target_data);
}
}  // namespace

namespace k2 {
class RaggedShapeOpsSuiteTest : public ::testing::Test {
 protected:
  RaggedShapeOpsSuiteTest() {
    ContextPtr context = GetCpuContext();
    const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
    const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
    const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
    const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};
    const std::vector<int32_t> row_splits3 = {0,  2,  3,  5,  8, 9,
                                              12, 13, 15, 15, 16};
    const std::vector<int32_t> row_ids3 = {0, 0, 1, 2, 2, 3, 3, 3,
                                           4, 5, 5, 5, 6, 7, 7, 9};
    std::vector<RaggedShapeDim> axes;
    axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits1),
                                     Array1<int32_t>(context, row_ids1),
                                     static_cast<int32_t>(row_ids1.size())});
    axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits2),
                                     Array1<int32_t>(context, row_ids2),
                                     static_cast<int32_t>(row_ids2.size())});
    axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits3),
                                     Array1<int32_t>(context, row_ids3),
                                     static_cast<int32_t>(row_ids3.size())});

    simple_shape_ = RaggedShape(axes, true);

    // random_shape_ is on CPU
    random_shape_ = RandomRaggedShape(true,   // set_row_ids
                                      3,      // min_num_axes
                                      4,      // max_num_axes
                                      0,      // min_num_elements
                                      1000);  // max_num_elements
  }

  RaggedShape simple_shape_;
  RaggedShape random_shape_;
};

template <typename T, DeviceType d>
void TestMaxPerSubListTest() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  {
    // empty case
    const std::vector<int32_t> row_splits = {0};
    RaggedShapeDim shape_dim;
    shape_dim.row_splits = Array1<int32_t>(context, row_splits);
    shape_dim.cached_tot_size = 0;
    std::vector<RaggedShapeDim> axes = {shape_dim};
    RaggedShape shape(axes, true);
    Array1<T> values(context, 0);
    Ragged<T> ragged(shape, values);

    int32_t num_rows = ragged.shape.Dim0();
    ASSERT_EQ(num_rows, 0);
    Array1<T> max_values(context, num_rows);
    // just run to check if there's any error
    MaxPerSublist(ragged, 1, &max_values);
    EXPECT_EQ(max_values.Dim(), 0);
  }

  {
    const std::vector<int32_t> row_splits = {0, 2, 2, 5, 6};
    RaggedShapeDim shape_dim;
    shape_dim.row_splits = Array1<int32_t>(context, row_splits);
    shape_dim.cached_tot_size = row_splits.back();
    std::vector<RaggedShapeDim> axes = {shape_dim};
    RaggedShape shape(axes, true);
    const std::vector<T> values_vec = {1, 3, 2, 8, 0, -1};
    Array1<T> values(context, values_vec);
    Ragged<T> ragged(shape, values);

    int32_t num_rows = ragged.shape.Dim0();
    Array1<T> max_values(context, num_rows);
    T default_value = 2;
    MaxPerSublist(ragged, default_value, &max_values);
    // copy memory from GPU/CPU to CPU
    std::vector<T> cpu_data(max_values.Dim());
    auto kind = GetMemoryCopyKind(*max_values.Context(), *cpu);
    MemoryCopy(static_cast<void *>(cpu_data.data()),
               static_cast<const void *>(max_values.Data()),
               max_values.Dim() * max_values.ElementSize(), kind, nullptr);
    std::vector<T> expected_data = {3, default_value, 8, default_value};
    EXPECT_EQ(cpu_data, expected_data);
  }

  {
    // test with random large size
    const int32_t min_num_elements = 2000;
    // not random shape is on CPU
    RaggedShape shape = RandomRaggedShape(false, 2, 2, min_num_elements, 5000);
    ASSERT_EQ(shape.NumAxes(), 2);
    RaggedShape gpu_shape;
    if (d == kCuda) {
      // copy shape to GPU
      const Array1<T> &row_splits = shape.RowSplits(1);
      RaggedShapeDim shape_dim;
      shape_dim.row_splits = row_splits.To(GetCudaContext());
      shape_dim.cached_tot_size = shape.NumElements();
      std::vector<RaggedShapeDim> axes = {shape_dim};
      gpu_shape = RaggedShape(axes, true);
    }

    int32_t num_elems = shape.NumElements();
    std::vector<T> data(num_elems);
    for (int32_t i = 0; i != 10; ++i) {
      std::iota(data.begin(), data.end(), 0);
      // randomly set data[pos] = num_elems which is
      // greater than any element in data
      int32_t pos = RandInt(0, num_elems - 1);
      data[pos] = num_elems;
      // find the corresponding row
      int32_t num_rows = shape.Dim0();
      const int32_t *row_splits_data = shape.RowSplits(1).Data();
      int32_t row = 0;
      for (int32_t i = 0; i < num_rows; ++i) {
        if (pos >= row_splits_data[i] && pos < row_splits_data[i + 1]) {
          row = i;
          break;
        }
      }

      Array1<T> values(context, data);
      Ragged<T> ragged(d == kCuda ? gpu_shape : shape, values);
      Array1<T> max_values(context, num_rows);
      T default_value = 0;
      MaxPerSublist(ragged, default_value, &max_values);
      EXPECT_EQ(max_values[row], num_elems);
    }
  }
}

TEST(RaggedShapeOpsTest, MaxPerSubListTest) {
  TestMaxPerSubListTest<int32_t, kCpu>();
  TestMaxPerSubListTest<int32_t, kCuda>();
}

template <typename T, DeviceType d>
void TestMinPerSubListTest() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  {
    // empty case
    std::vector<int32_t> row_splits_vec = {0};
    Array1<T> row_splits(context, row_splits_vec);
    RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
    Array1<T> values(context, 0);
    Ragged<T> ragged(shape, values);

    int32_t num_rows = ragged.shape.Dim0();
    ASSERT_EQ(num_rows, 0);
    Array1<T> min_values(context, num_rows);
    // just run to check if there's any error
    MinPerSublist(ragged, 1, &min_values);
    EXPECT_EQ(min_values.Dim(), 0);
  }

  {
    std::vector<int32_t> row_splits_vec = {0, 2, 2, 5, 6};
    Array1<T> row_splits(context, row_splits_vec);
    RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
    const std::vector<T> values_vec = {1, 3, 3, 8, 4, -1};
    Array1<T> values(context, values_vec);
    Ragged<T> ragged(shape, values);

    int32_t num_rows = ragged.shape.Dim0();
    Array1<T> min_values(context, num_rows);
    T default_value = 2;
    MinPerSublist(ragged, default_value, &min_values);
    // copy memory from GPU/CPU to CPU
    min_values = min_values.To(cpu);
    std::vector<T> cpu_data(min_values.Data(),
                            min_values.Data() + min_values.Dim());
    std::vector<T> expected_data = {1, default_value, default_value, -1};
    EXPECT_EQ(cpu_data, expected_data);
  }

  // May add tests for random large size? (but maybe it's fine to not add as we
  // have tested large cases in MaxPerSubList)
}

TEST(RaggedShapeOpsTest, MinPerSubListTest) {
  TestMinPerSubListTest<int32_t, kCpu>();
  TestMinPerSubListTest<int32_t, kCuda>();
}

template <typename T, DeviceType d>
void TestAndOrPerSubListTest() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  {
    // And
    const std::vector<int32_t> row_splits = {0, 2, 2, 5, 6};
    RaggedShapeDim shape_dim;
    shape_dim.row_splits = Array1<int32_t>(context, row_splits);
    shape_dim.cached_tot_size = row_splits.back();
    std::vector<RaggedShapeDim> axes = {shape_dim};
    RaggedShape shape(axes, true);
    const std::vector<T> values_vec = {1, 3, 3, 6, 11, 0};
    Array1<T> values(context, values_vec);
    Ragged<T> ragged(shape, values);

    int32_t num_rows = ragged.shape.Dim0();
    Array1<T> dst(context, num_rows);
    T default_value = -1;
    AndPerSublist(ragged, default_value, &dst);
    // copy memory from GPU/CPU to CPU
    dst = dst.To(cpu);
    std::vector<T> cpu_data(dst.Data(), dst.Data() + dst.Dim());
    std::vector<T> expected_data = {1, -1, 2, 0};
    EXPECT_EQ(cpu_data, expected_data);
  }

  {
    // Or
    const std::vector<int32_t> row_splits = {0, 2, 2, 5, 6};
    RaggedShapeDim shape_dim;
    shape_dim.row_splits = Array1<int32_t>(context, row_splits);
    shape_dim.cached_tot_size = row_splits.back();
    std::vector<RaggedShapeDim> axes = {shape_dim};
    RaggedShape shape(axes, true);
    const std::vector<T> values_vec = {1, 3, 3, 4, 6, 0};
    Array1<T> values(context, values_vec);
    Ragged<T> ragged(shape, values);

    int32_t num_rows = ragged.shape.Dim0();
    Array1<T> dst(context, num_rows);
    T default_value = 0;
    OrPerSublist(ragged, default_value, &dst);
    // copy memory from GPU/CPU to CPU
    dst = dst.To(cpu);
    std::vector<T> cpu_data(dst.Data(), dst.Data() + dst.Dim());
    std::vector<T> expected_data = {3, 0, 7, 0};
    EXPECT_EQ(cpu_data, expected_data);
  }
}

TEST(RagedShapeOpsTest, AndOrPerSubListTest) {
  TestAndOrPerSubListTest<int32_t, kCpu>();
  TestAndOrPerSubListTest<int32_t, kCuda>();
}

void TestUnsqueeze(ContextPtr context, const RaggedShape &input_shape) {
  RaggedShape src_shape = input_shape.To(context);
  src_shape.Populate();  // set row_ids
  {
    // axis = 0.
    RaggedShape shape = Unsqueeze(src_shape, 0);
    int32_t dim0 = src_shape.Dim0();
    const std::vector<RaggedShapeDim> &src_axes = src_shape.Axes();
    const std::vector<RaggedShapeDim> &dest_axes = shape.Axes();

    {
      const Array1<int32_t> &row_splits0 = dest_axes[0].row_splits;
      std::vector<int32_t> data = {0, dim0};
      CheckArrayData(row_splits0, data);
    }

    {
      const Array1<int32_t> &row_ids0 = dest_axes[0].row_ids;
      std::vector<int32_t> data(dim0, 0);
      CheckArrayData(row_ids0, data);
    }

    {
      for (auto i = 0; i != src_axes.size(); ++i) {
        CheckArrayData(src_axes[i].row_splits, dest_axes[i + 1].row_splits);
        CheckArrayData(src_axes[i].row_ids, dest_axes[i + 1].row_ids);
      }
    }
  }

  {
    // axis = 1
    int32_t axis = 1;
    RaggedShape shape = Unsqueeze(src_shape, axis);
    int32_t tot_size = shape.TotSize(axis);
    const std::vector<RaggedShapeDim> &src_axes = src_shape.Axes();
    const std::vector<RaggedShapeDim> &dest_axes = shape.Axes();

    {
      for (auto i = 0; i < axis; ++i) {
        CheckArrayData(src_axes[i].row_splits, dest_axes[i].row_splits);
        CheckArrayData(src_axes[i].row_ids, dest_axes[i].row_ids);
      }
    }

    {
      const Array1<int32_t> &row_splits = dest_axes[axis].row_splits;
      std::vector<int32_t> data(tot_size + 1);
      std::iota(data.begin(), data.end(), 0);
      CheckArrayData(row_splits, data);
    }

    {
      const Array1<int32_t> &row_ids = dest_axes[axis].row_ids;
      std::vector<int32_t> data(tot_size);
      std::iota(data.begin(), data.end(), 0);
      CheckArrayData(row_ids, data);
    }

    {
      for (auto i = axis; i < src_axes.size(); ++i) {
        CheckArrayData(src_axes[i].row_splits, dest_axes[i + 1].row_splits);
        CheckArrayData(src_axes[i].row_ids, dest_axes[i + 1].row_ids);
      }
    }
  }
}

TEST_F(RaggedShapeOpsSuiteTest, TestUnsqueezeCpu) {
  TestUnsqueeze(GetCpuContext(), simple_shape_);
  TestUnsqueeze(GetCpuContext(), random_shape_);
}
TEST_F(RaggedShapeOpsSuiteTest, TestUnsqueezeGpu) {
  TestUnsqueeze(GetCudaContext(), simple_shape_);
  TestUnsqueeze(GetCudaContext(), random_shape_);
}

void TestRemoveAxis(ContextPtr context, const RaggedShape &input_shape) {
  RaggedShape src_shape = input_shape.To(context);
  ASSERT_EQ(src_shape.NumAxes(), 4);
  {
    // axis = 0.
    int32_t axis = 0;
    RaggedShape shape = RemoveAxis(src_shape, axis);
    const std::vector<RaggedShapeDim> &src_axes = src_shape.Axes();
    const std::vector<RaggedShapeDim> &dest_axes = shape.Axes();
    ASSERT_EQ(src_axes.size(), 3);
    ASSERT_EQ(dest_axes.size(), 2);

    {
      for (auto i = 0; i != dest_axes.size(); ++i) {
        CheckArrayData(dest_axes[i].row_splits, src_axes[i + 1].row_splits);
        CheckArrayData(dest_axes[i].row_ids, src_axes[i + 1].row_ids);
      }
    }
  }

  {
    // axis = 1
    int32_t axis = 1;
    RaggedShape shape = RemoveAxis(src_shape, axis);
    const std::vector<RaggedShapeDim> &src_axes = src_shape.Axes();
    const std::vector<RaggedShapeDim> &dest_axes = shape.Axes();
    ASSERT_EQ(src_axes.size(), 3);
    ASSERT_EQ(dest_axes.size(), 2);

    {
      const Array1<int32_t> &row_splits0 = dest_axes[0].row_splits;
      std::vector<int32_t> data = {0, 3, 7, 10};
      CheckArrayData(row_splits0, data);
    }

    {
      const Array1<int32_t> &row_ids0 = dest_axes[0].row_ids;
      std::vector<int32_t> data = {0, 0, 0, 1, 1, 1, 1, 2, 2, 2};
      CheckArrayData(row_ids0, data);
    }

    {
      for (auto i = 1; i != dest_axes.size(); ++i) {
        CheckArrayData(dest_axes[i].row_splits, src_axes[i + 1].row_splits);
        CheckArrayData(dest_axes[i].row_ids, src_axes[i + 1].row_ids);
      }
    }
  }

  {
    // axis = 3
    int32_t axis = 3;  // the last axis
    RaggedShape shape = RemoveAxis(src_shape, axis);
    const std::vector<RaggedShapeDim> &src_axes = src_shape.Axes();
    const std::vector<RaggedShapeDim> &dest_axes = shape.Axes();
    ASSERT_EQ(src_axes.size(), 3);
    ASSERT_EQ(dest_axes.size(), 2);

    {
      for (auto i = 0; i != dest_axes.size(); ++i) {
        CheckArrayData(dest_axes[i].row_splits, src_axes[i].row_splits);
        CheckArrayData(dest_axes[i].row_ids, src_axes[i].row_ids);
      }
    }
  }
}

TEST_F(RaggedShapeOpsSuiteTest, TestRemoveAxisCpu) {
  TestRemoveAxis(GetCpuContext(), simple_shape_);
}
TEST_F(RaggedShapeOpsSuiteTest, TestRemoveAxisGpu) {
  TestRemoveAxis(GetCudaContext(), simple_shape_);
}

void TestGetOffsets(ContextPtr context) {
  for (int32_t i = 0; i != 2; ++i) {
    int32_t num_shape = RandInt(10, 100);
    int32_t num_axes = RandInt(2, 4);
    std::vector<RaggedShape> shape_vec(num_shape);
    std::vector<RaggedShape *> shapes(num_shape);
    for (int32_t j = 0; j != num_shape; ++j) {
      shape_vec[j] =
          RandomRaggedShape(false, num_axes, num_axes, 0, 1000).To(context);
      shapes[j] = &shape_vec[j];
    }
    RaggedShape **shapes_ptr = shapes.data();
    Array2<int32_t> offsets = GetOffsets(num_shape, shapes_ptr);
    ASSERT_EQ(offsets.Dim0(), num_axes + 1);
    ASSERT_EQ(offsets.Dim1(), num_shape + 1);
    auto acc = offsets.Accessor();
    for (int32_t axis = 0; axis <= num_axes; ++axis) {
      int32_t sum = 0;
      for (int32_t j = 0; j <= num_shape; ++j) {
        EXPECT_EQ(acc(axis, j), sum);
        if (j < num_shape) {
          sum += (axis == 0 ? 1 : shape_vec[j].TotSize(axis - 1));
        }
      }
    }
  }
}

TEST(RaggedShapeOpsTest, TestGetOffsets) {
  TestGetOffsets(GetCpuContext());
  TestGetOffsets(GetCudaContext());
}

// returns a random ragged shape where the dims on axis 1 are all the same
// (so: can be transposed).
RaggedShape RandomRaggedShapeToTranspose(ContextPtr c) {
  ContextPtr c_cpu = GetCpuContext();

  RaggedShape random = RandomRaggedShape(false, 2, 4, 0, 5000).To(c);

  int32_t input_dim0 = random.Dim0(), divisor = 1;
  for (int32_t i = 1; i * i <= input_dim0; i++) {
    if (input_dim0 % i == 0 && i > divisor) divisor = i;
  }

  int32_t output_dim0 = divisor, output_dim1 = input_dim0 / divisor;

  Array1<int32_t> row_splits =
      Range<int32_t>(c, output_dim0 + 1, 0, output_dim1);
  int32_t cached_tot_size = input_dim0;

  RaggedShape top_level_shape =
      RaggedShape2(&row_splits, nullptr, cached_tot_size);
  return ComposeRaggedShapes(top_level_shape, random);
}

template <DeviceType d>
void TestTranspose() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  {
    const std::vector<int32_t> row_splits1_vec = {0, 2, 4, 6};
    const std::vector<int32_t> row_splits2_vec = {0, 3, 4, 7, 8, 10, 12};
    Array1<int32_t> row_splits1(context, row_splits1_vec);
    Array1<int32_t> row_splits2(context, row_splits2_vec);
    RaggedShape src_shape =
        RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);
    ASSERT_EQ(src_shape.Dim0(), 3);
    ASSERT_EQ(src_shape.TotSize(1), 6);
    RaggedShape shape = Transpose(src_shape);
    EXPECT_EQ(shape.Dim0(), 2);
    ASSERT_EQ(shape.TotSize(1), 6);
    const std::vector<int32_t> expected_row_splits = {0, 3, 6};
    const std::vector<int32_t> expected_row_ids = {0, 0, 0, 1, 1, 1};
    CheckArrayData(shape.RowSplits(1), expected_row_splits);
    CheckArrayData(shape.RowIds(1), expected_row_ids);
    CheckArrayData(shape.RowSplits(2), {0, 3, 6, 8, 9, 10, 12});
    CheckArrayData(shape.RowIds(2), {0, 0, 0, 1, 1, 1, 2, 2, 3, 4, 5, 5});
  }

  {
    // random case
    for (int32_t j = 0; j != 2; ++j) {
      RaggedShape to_transpose = RandomRaggedShapeToTranspose(context);
      RaggedShape transposed = Transpose(to_transpose);

      if (d != kCpu) {
        to_transpose = to_transpose.To(cpu);
        transposed = transposed.To(cpu);
      }

      for (auto iter = transposed.Iterator(); !iter.Done(); iter.Next()) {
        std::vector<int32_t> index = iter.Value();
        int32_t i = transposed[index];  // Just make sure this doesn't crash,
                                        // dont need the value.
        std::swap(index[0], index[1]);
        i = to_transpose[index];  // don't need the value, just need to make
                                  // sure it's an allowable index.
      }
      for (auto iter = to_transpose.Iterator(); !iter.Done(); iter.Next()) {
        std::vector<int32_t> index = iter.Value();
        std::swap(index[0], index[1]);
        int32_t i = transposed[index];  // don't need the value, just need to
                                        // make sure it's an allowable index.
      }
    }
  }
}
TEST(RaggedShapeOpsTest, TestTranspose) {
  TestTranspose<kCpu>();
  TestTranspose<kCuda>();
}

template <DeviceType d, typename T>
void TestTransposeRagged() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  {
    const std::vector<int32_t> row_splits1_vec = {0, 2, 4, 6};
    const std::vector<int32_t> row_splits2_vec = {0, 3, 4, 7, 8, 10, 12};
    Array1<int32_t> row_splits1(context, row_splits1_vec);
    Array1<int32_t> row_splits2(context, row_splits2_vec);
    RaggedShape src_shape =
        RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);
    ASSERT_EQ(src_shape.Dim0(), 3);
    ASSERT_EQ(src_shape.TotSize(1), 6);
    std::vector<T> values = {0, 1, 2, 3, 4, 5, 8, 7, 6, 9, 10, 15};
    ASSERT_EQ(values.size(), src_shape.NumElements());
    Array1<T> values_array(context, values);
    Ragged<T> ragged(src_shape, values_array);
    Ragged<T> ans = Transpose(ragged);
    RaggedShape shape = ans.shape;
    // Check shape
    ASSERT_EQ(shape.Dim0(), 2);
    ASSERT_EQ(shape.TotSize(1), 6);
    const std::vector<int32_t> expected_row_splits = {0, 3, 6};
    const std::vector<int32_t> expected_row_ids = {0, 0, 0, 1, 1, 1};
    CheckArrayData(shape.RowSplits(1), expected_row_splits);
    CheckArrayData(shape.RowIds(1), expected_row_ids);
    CheckArrayData(shape.RowSplits(2), {0, 3, 6, 8, 9, 10, 12});
    CheckArrayData(shape.RowIds(2), {0, 0, 0, 1, 1, 1, 2, 2, 3, 4, 5, 5});
    // Check values
    CheckArrayData(ans.values, {0, 1, 2, 4, 5, 8, 6, 9, 3, 7, 10, 15});
  }

  {
    // random case
    for (int32_t j = 0; j != 2; ++j) {
      RaggedShape to_transpose = RandomRaggedShapeToTranspose(context);
      int32_t num_elems = to_transpose.NumElements();
      Array1<T> src_values = RandUniformArray1<T>(context, num_elems, 0, 10000);
      Ragged<T> src(to_transpose, src_values);
      Ragged<T> ans = Transpose(src);
      if (d != kCpu) {
        src = src.To(cpu);
        ans = ans.To(cpu);
        to_transpose = to_transpose.To(cpu);
      }
      RaggedShape transposed = ans.shape;

      for (auto iter = transposed.Iterator(); !iter.Done(); iter.Next()) {
        std::vector<int32_t> index = iter.Value();
        T value = ans[index];
        std::swap(index[0], index[1]);
        EXPECT_EQ(value, src[index]);
      }
      for (auto iter = to_transpose.Iterator(); !iter.Done(); iter.Next()) {
        std::vector<int32_t> index = iter.Value();
        T value = src[index];
        std::swap(index[0], index[1]);
        EXPECT_EQ(value, ans[index]);
      }
    }
  }
}
TEST(RaggedTest, TestTransposeRagged) {
  TestTransposeRagged<kCpu, int32_t>();
  TestTransposeRagged<kCuda, int32_t>();
  TestTransposeRagged<kCpu, double>();
  TestTransposeRagged<kCuda, double>();
}

template <DeviceType d>
void TestRowSplitsPtr() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }
  RaggedShape shape = RandomRaggedShape().To(context);
  ASSERT_GE(shape.NumAxes(), 2);
  Array1<int32_t *> ptrs = GetRowSplitsPtr(shape);
  ASSERT_EQ(ptrs.Dim(), shape.NumAxes() - 1);
  // as num_axes is not so big, access (may copy memory) it in a loop is fine.
  for (int32_t i = 0; i != ptrs.Dim(); ++i) {
    EXPECT_EQ(ptrs[i], shape.RowSplits(i + 1).Data());
  }
}
TEST(RaggedShapeOpsTest, TestRowSplitsPtr) {
  TestRowSplitsPtr<kCpu>();
  TestRowSplitsPtr<kCuda>();
}

void TestRaggedShape2(ContextPtr context, const RaggedShape &shape) {
  RaggedShape src_shape = shape.To(context);
  src_shape.Populate();
  ASSERT_GE(src_shape.NumAxes(), 2);
  Array1<int32_t> row_splits = src_shape.RowSplits(1);
  Array1<int32_t> row_ids = src_shape.RowIds(1);
  int32_t cached_tot_size = src_shape.TotSize(1);

  {
    // both row_splits and row_ids are non-null
    RaggedShape result = RaggedShape2(&row_splits, &row_ids, cached_tot_size);
    CheckArrayData(result.RowSplits(1), row_splits);
    CheckArrayData(result.RowIds(1), row_ids);
    EXPECT_EQ(result.TotSize(1), cached_tot_size);
  }
  {
    // both row_splits and row_ids are non-null, cached_tot_size = -1
    RaggedShape result = RaggedShape2(&row_splits, &row_ids, -1);
    CheckArrayData(result.RowSplits(1), row_splits);
    CheckArrayData(result.RowIds(1), row_ids);
    EXPECT_EQ(result.TotSize(1), cached_tot_size);
  }
  {
    // row_ids is null
    RaggedShape result = RaggedShape2(&row_splits, nullptr, cached_tot_size);
    CheckArrayData(result.RowSplits(1), row_splits);
    CheckArrayData(result.RowIds(1), row_ids);
    EXPECT_EQ(result.TotSize(1), cached_tot_size);
  }
  {
    // row_ids is null, cached_tot_size = -1
    RaggedShape result = RaggedShape2(&row_splits, nullptr, -1);
    CheckArrayData(result.RowSplits(1), row_splits);
    CheckArrayData(result.RowIds(1), row_ids);
    EXPECT_EQ(result.TotSize(1), cached_tot_size);
  }

  // note if row_splits == null, then we suppose there's no empty rows after the
  // last row-id in row_ids
  if (row_splits.Dim() == (row_ids.Dim() == 0 ? 1 : row_ids.Back() + 2)) {
    {
      // row_splits is null
      RaggedShape result = RaggedShape2(nullptr, &row_ids, cached_tot_size);
      CheckArrayData(result.RowSplits(1), row_splits);
      CheckArrayData(result.RowIds(1), row_ids);
      EXPECT_EQ(result.TotSize(1), cached_tot_size);
    }
    {
      // row_splits is null, cached_tot_size = -1
      RaggedShape result = RaggedShape2(nullptr, &row_ids, -1);
      CheckArrayData(result.RowSplits(1), row_splits);
      CheckArrayData(result.RowIds(1), row_ids);
      EXPECT_EQ(result.TotSize(1), cached_tot_size);
    }
  }
}
TEST_F(RaggedShapeOpsSuiteTest, TestRaggedShape2Cpu) {
  TestRaggedShape2(GetCpuContext(), simple_shape_);
  TestRaggedShape2(GetCpuContext(), random_shape_);
}
TEST_F(RaggedShapeOpsSuiteTest, TestRaggedShape2Gpu) {
  TestRaggedShape2(GetCudaContext(), simple_shape_);
  TestRaggedShape2(GetCudaContext(), random_shape_);
}

void TestRaggedShape3(ContextPtr context, const RaggedShape &shape) {
  RaggedShape src_shape = shape.To(context);
  src_shape.Populate();
  ASSERT_GE(src_shape.NumAxes(), 3);
  Array1<int32_t> row_splits1 = src_shape.RowSplits(1);
  Array1<int32_t> row_ids1 = src_shape.RowIds(1);
  int32_t cached_tot_size1 = src_shape.TotSize(1);
  Array1<int32_t> row_splits2 = src_shape.RowSplits(2);
  Array1<int32_t> row_ids2 = src_shape.RowIds(2);
  int32_t cached_tot_size2 = src_shape.TotSize(2);

  {
    // both row_splits and row_ids are non-null
    RaggedShape result =
        RaggedShape3(&row_splits1, &row_ids1, cached_tot_size1, &row_splits2,
                     &row_ids2, cached_tot_size2);
    CheckArrayData(result.RowSplits(1), row_splits1);
    CheckArrayData(result.RowIds(1), row_ids1);
    EXPECT_EQ(result.TotSize(1), cached_tot_size1);
    CheckArrayData(result.RowSplits(2), row_splits2);
    CheckArrayData(result.RowIds(2), row_ids2);
    EXPECT_EQ(result.TotSize(2), cached_tot_size2);
  }
  {
    // row_ids is non-null, cached_tot_size = -1
    RaggedShape result =
        RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);
    CheckArrayData(result.RowSplits(1), row_splits1);
    CheckArrayData(result.RowIds(1), row_ids1);
    EXPECT_EQ(result.TotSize(1), cached_tot_size1);
    CheckArrayData(result.RowSplits(2), row_splits2);
    CheckArrayData(result.RowIds(2), row_ids2);
    EXPECT_EQ(result.TotSize(2), cached_tot_size2);
  }

  // note if row_splits == null, then we suppose there's no empty rows after the
  // last row-id in row_ids
  bool valid1 =
      (row_splits1.Dim() == (row_ids1.Dim() == 0 ? 1 : row_ids1.Back() + 2));
  bool valid2 =
      (row_splits2.Dim() == (row_ids2.Dim() == 0 ? 1 : row_ids2.Back() + 2));
  if (valid1 && valid2) {
    RaggedShape result =
        RaggedShape3(nullptr, &row_ids1, -1, nullptr, &row_ids2, -1);
    CheckArrayData(result.RowSplits(1), row_splits1);
    CheckArrayData(result.RowIds(1), row_ids1);
    EXPECT_EQ(result.TotSize(1), cached_tot_size1);
    CheckArrayData(result.RowSplits(2), row_splits2);
    CheckArrayData(result.RowIds(2), row_ids2);
    EXPECT_EQ(result.TotSize(2), cached_tot_size2);
  }
  // TODO(haowen): add more cases for other branches
}
TEST_F(RaggedShapeOpsSuiteTest, TestRaggedShape3Cpu) {
  TestRaggedShape3(GetCpuContext(), simple_shape_);
  TestRaggedShape3(GetCpuContext(), random_shape_);
}
TEST_F(RaggedShapeOpsSuiteTest, TestRaggedShape3Gpu) {
  TestRaggedShape3(GetCudaContext(), simple_shape_);
  TestRaggedShape3(GetCudaContext(), random_shape_);
}

void TestComposeShape(ContextPtr context, const RaggedShape &shape) {
  RaggedShape src_shape = shape.To(context);
  ASSERT_GE(src_shape.NumAxes(), 3);
  Array1<int32_t> row_splits1 = src_shape.RowSplits(1);
  Array1<int32_t> row_ids1 = src_shape.RowIds(1);
  Array1<int32_t> row_splits2 = src_shape.RowSplits(2);
  Array1<int32_t> row_ids2 = src_shape.RowIds(2);

  RaggedShape shape1 = RaggedShape2(&row_splits1, nullptr, -1);
  RaggedShape shape2 = RaggedShape2(&row_splits2, nullptr, -1);

  RaggedShape result = ComposeRaggedShapes(shape1, shape2);

  ASSERT_EQ(result.NumAxes(), 3);

  CheckArrayData(result.RowSplits(1), row_splits1);
  CheckArrayData(result.RowIds(1), row_ids1);
  CheckArrayData(result.RowSplits(2), row_splits2);
  CheckArrayData(result.RowIds(2), row_ids2);
}
TEST_F(RaggedShapeOpsSuiteTest, TestComposeShapeCpu) {
  TestComposeShape(GetCpuContext(), simple_shape_);
  TestComposeShape(GetCpuContext(), random_shape_);
}
TEST_F(RaggedShapeOpsSuiteTest, TestComposeShapeGpu) {
  TestComposeShape(GetCudaContext(), simple_shape_);
  TestComposeShape(GetCudaContext(), random_shape_);
}

void TestShapeFromTotSize(ContextPtr context, const RaggedShape &shape) {
  RaggedShape src_shape = shape.To(context);
  ASSERT_GE(src_shape.NumAxes(), 2);

  int32_t num_axes = src_shape.NumAxes();
  std::vector<int32_t> tot_sizes(num_axes);
  for (int32_t i = 0; i != num_axes; ++i) {
    tot_sizes[i] = src_shape.TotSize(i);
  }

  RaggedShape result =
      RaggedShapeFromTotSizes(context, num_axes, tot_sizes.data());

  ASSERT_EQ(result.NumAxes(), num_axes);
  for (int32_t i = 0; i < num_axes; ++i) {
    EXPECT_EQ(result.TotSize(i), src_shape.TotSize(i));
    if (i > 0) {
      EXPECT_EQ(result.RowSplits(i).Dim(), src_shape.RowSplits(i).Dim());
      EXPECT_EQ(result.RowIds(i).Dim(), src_shape.RowIds(i).Dim());
    }
  }
}
TEST_F(RaggedShapeOpsSuiteTest, TestShapeFromTotSizeCpu) {
  TestShapeFromTotSize(GetCpuContext(), simple_shape_);
  TestShapeFromTotSize(GetCpuContext(), random_shape_);
}
TEST_F(RaggedShapeOpsSuiteTest, TestShapeFromTotSizeGpu) {
  TestShapeFromTotSize(GetCudaContext(), simple_shape_);
  TestShapeFromTotSize(GetCudaContext(), random_shape_);
}

template <typename T, DeviceType d>
void TestRagged() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  {
    // constructed with row_splits and row_ids
    // RaggedTensor4 t = [
    //  [ [[ 1, 2], [4]],  [[3, 0]] ],
    //  [ [[7, 8, 9]], [[6], [3, 5, 7]], [[2]] ],
    //  [ [[3, 4], [], [8]] ]
    // ]
    const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
    const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
    const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
    const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};
    const std::vector<int32_t> row_splits3 = {0,  2,  3,  5,  8, 9,
                                              12, 13, 15, 15, 16};
    const std::vector<int32_t> row_ids3 = {0, 0, 1, 2, 2, 3, 3, 3,
                                           4, 5, 5, 5, 6, 7, 7, 9};
    const std::vector<T> values_vec = {1, 2, 4, 3, 0, 7, 8, 9,
                                       6, 3, 5, 7, 2, 3, 4, 8};
    std::vector<RaggedShapeDim> axes;
    axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits1),
                                     Array1<int32_t>(context, row_ids1),
                                     static_cast<int32_t>(row_ids1.size())});
    axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits2),
                                     Array1<int32_t>(context, row_ids2),
                                     static_cast<int32_t>(row_ids2.size())});
    axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits3),
                                     Array1<int32_t>(context, row_ids3),
                                     static_cast<int32_t>(row_ids3.size())});

    RaggedShape shape(axes, true);
    Array1<T> values(context, values_vec);
    Ragged<T> ragged(shape, values);

    // test Index(axis, i)
    {
      // values: [[[ 1, 2], [4]], [[3, 0]]]
      Ragged<T> sub_raggged = ragged.Index(0, 0);
      RaggedShape &sub_shape = sub_raggged.shape;
      EXPECT_EQ(sub_shape.NumAxes(), 3);
      const std::vector<std::vector<int32_t>> sub_row_splits_vec = {
          {0, 2, 3}, {0, 2, 3, 5}};
      CheckRowSplits(sub_shape, sub_row_splits_vec);
      const Array1<T> &sub_values = sub_raggged.values;
      const std::vector<T> sub_values_vec = {1, 2, 4, 3, 0};
      CheckArrayData<T>(sub_values, sub_values_vec);
    }
    {
      // values: [[[7, 8, 9]], [[6], [3, 5, 7]], [[2]]]
      Ragged<T> sub_raggged = ragged.Index(0, 1);
      RaggedShape &sub_shape = sub_raggged.shape;
      EXPECT_EQ(sub_shape.NumAxes(), 3);
      const std::vector<std::vector<int32_t>> sub_row_splits_vec = {
          {0, 1, 3, 4}, {0, 3, 4, 7, 8}};
      CheckRowSplits(sub_shape, sub_row_splits_vec);
      const Array1<T> &sub_values = sub_raggged.values;
      const std::vector<T> sub_values_vec = {7, 8, 9, 6, 3, 5, 7, 2};
      CheckArrayData<T>(sub_values, sub_values_vec);
    }
    {
      // values: [[[3, 4], [], [8]]]
      Ragged<T> sub_raggged = ragged.Index(0, 2);
      RaggedShape &sub_shape = sub_raggged.shape;
      EXPECT_EQ(sub_shape.NumAxes(), 3);
      const std::vector<std::vector<int32_t>> sub_row_splits_vec = {
          {0, 3}, {0, 2, 2, 3}};
      CheckRowSplits(sub_shape, sub_row_splits_vec);
      const Array1<T> &sub_values = sub_raggged.values;
      const std::vector<T> sub_values_vec = {3, 4, 8};
      CheckArrayData<T>(sub_values, sub_values_vec);
    }

    // test operator[](const std::vector<int32_t> &indexes)
    if (d == kCpu) {
      {
        std::vector<int32_t> indexes = {0, 0, 0, 0};
        EXPECT_EQ(ragged.shape[indexes], 0);
        EXPECT_EQ(ragged[indexes], 1);
      }
      {
        std::vector<int32_t> indexes = {0, 1, 0, 0};
        EXPECT_EQ(ragged.shape[indexes], 3);
        EXPECT_EQ(ragged[indexes], 3);
      }
      {
        std::vector<int32_t> indexes = {1, 0, 0, 1};
        EXPECT_EQ(ragged.shape[indexes], 6);
        EXPECT_EQ(ragged[indexes], 8);
      }
      {
        std::vector<int32_t> indexes = {1, 1, 1, 0};
        EXPECT_EQ(ragged.shape[indexes], 9);
        EXPECT_EQ(ragged[indexes], 3);
      }
      {
        std::vector<int32_t> indexes = {2, 0, 0, 1};
        EXPECT_EQ(ragged.shape[indexes], 14);
        EXPECT_EQ(ragged[indexes], 4);
      }
      {
        std::vector<int32_t> indexes = {2, 0, 2, 0};
        EXPECT_EQ(ragged.shape[indexes], 15);
        EXPECT_EQ(ragged[indexes], 8);
      }
    }

    const std::vector<std::vector<int32_t>> row_splits_vec = {
        row_splits1, row_splits2, row_splits3};
    // test To(ctx)
    {
      // to GPU
      Ragged<T> other = ragged.To(GetCudaContext());
      CheckRowSplits(other.shape, row_splits_vec);
      CheckArrayData<T>(other.values, values_vec);
    }
    {
      // to CPU
      Ragged<T> other = ragged.To(GetCpuContext());
      CheckRowSplits(other.shape, row_splits_vec);
      CheckArrayData<T>(other.values, values_vec);
    }
  }
}

template <typename T, typename OP = LessThan<T>>
static void CpuSortSublists(const Array1<int32_t> &row_splits, Array1<T> *src) {
  K2_CHECK(src->Context()->GetDeviceType() == kCpu);
  T *p = src->Data();
  OP comp = OP();
  for (int32_t i = 0; i < row_splits.Dim() - 1; ++i) {
    int32_t cur = row_splits[i];
    int32_t next = row_splits[i + 1];
    std::sort(p + cur, p + next, comp);
  }
}

template <typename T, typename OP = LessThan<T>>
static void TestSortSublists() {
  auto cpu_context = GetCpuContext();
  auto cuda_context = GetCudaContext();

  RaggedShape shape = RandomRaggedShape(false,  // set_row_ids
                                        2,      // min_num_axes
                                        4,      // max_num_axes
                                        1,      // min_num_elements
                                        2000);  // max_num_elements

  Array1<T> values =
      RandUniformArray1<T>(shape.Context(), shape.NumElements(), -2000, 2000);
  Ragged<T> ragged(shape, values);
  ragged = ragged.To(cuda_context);
  values = values.To(cpu_context);  // to be sorted by cpu

  // TODO(fangjun): add a `Clone` method to Array1<T>
  Array1<T> unsorted = values.To(cuda_context).To(cpu_context);

  Array1<int32_t> order(ragged.Context(), ragged.values.Dim());
  SortSublists<T, OP>(&ragged, &order);

  Array1<int32_t> &segment = ragged.shape.RowSplits(ragged.NumAxes() - 1);
  CpuSortSublists<T, OP>(segment, &values);

  int32_t n = order.Dim();
  for (int i = 0; i != n; ++i) {
    EXPECT_EQ(values[i], ragged.values[i]);
    EXPECT_EQ(ragged.values[i], unsorted[order[i]]);
  }
}

TEST(RaggedTest, Ragged) {
  TestRagged<int32_t, kCuda>();
  TestRagged<int32_t, kCpu>();
  TestRagged<double, kCuda>();
  TestRagged<double, kCpu>();

  TestSortSublists<int32_t>();
  TestSortSublists<double>();
}

template <DeviceType d>
void TestAppend() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  {
    // simple case
    std::vector<RaggedShape> shapes(2);
    std::vector<RaggedShape *> shapes_ptr(2);
    std::vector<std::vector<Array1<int32_t>>> row_splits_vec(2);
    {
      const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
      const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
      const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
      const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};
      Array1<int32_t> splits1(context, row_splits1);
      Array1<int32_t> ids1(context, row_ids1);
      Array1<int32_t> splits2(context, row_splits2);
      Array1<int32_t> ids2(context, row_ids2);
      row_splits_vec[0].push_back(splits1);
      row_splits_vec[1].push_back(splits2);
      shapes[0] = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2, &ids2,
                               ids2.Dim());
      shapes_ptr[0] = &shapes[0];
    }
    {
      const std::vector<int32_t> row_splits1 = {0, 1, 3};
      const std::vector<int32_t> row_ids1 = {0, 1, 1};
      const std::vector<int32_t> row_splits2 = {0, 3, 4, 7};
      const std::vector<int32_t> row_ids2 = {0, 0, 0, 1, 2, 2, 2};
      Array1<int32_t> splits1(context, row_splits1);
      Array1<int32_t> ids1(context, row_ids1);
      Array1<int32_t> splits2(context, row_splits2);
      Array1<int32_t> ids2(context, row_ids2);
      row_splits_vec[0].push_back(splits1);
      row_splits_vec[1].push_back(splits2);
      RaggedShape shape = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2,
                                       &ids2, ids2.Dim());
      shapes[1] = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2, &ids2,
                               ids2.Dim());
      shapes_ptr[1] = &shapes[1];
    }

    RaggedShape result = Append(0, 2, shapes_ptr.data());

    // get result splits with `SpliceRowSplits` and get result row-ids with
    // `RowSplitsToRowIds``
    std::vector<Array1<int32_t>> result_splits;
    std::vector<Array1<int32_t>> result_ids;
    for (auto i = 0; i < 2; ++i) {
      std::vector<const Array1<int32_t> *> splits_ptr = {&row_splits_vec[i][0],
                                                         &row_splits_vec[i][1]};
      Array1<int32_t> curr_row_splits = SpliceRowSplits(2, splits_ptr.data());
      result_splits.push_back(curr_row_splits);
      Array1<int32_t> curr_row_ids(context, curr_row_splits.Back());
      RowSplitsToRowIds(curr_row_splits, curr_row_ids);
      result_ids.push_back(curr_row_ids);
    }
    for (int32_t i = 0; i < 2; ++i) {
      CheckArrayData(result.RowSplits(i + 1), result_splits[i]);
      CheckArrayData(result.RowIds(i + 1), result_ids[i]);
    }
  }

  {
    // test with random large size
    for (int32_t i = 0; i < 2; ++i) {
      int32_t num_shape = RandInt(2, 100);
      int32_t num_axes = RandInt(2, 4);
      std::vector<RaggedShape> shape_vec(num_shape);
      std::vector<RaggedShape *> shapes(num_shape);
      for (int32_t j = 0; j != num_shape; ++j) {
        shape_vec[j] =
            RandomRaggedShape(true, num_axes, num_axes, 0, 1000).To(context);
        shapes[j] = &shape_vec[j];
      }
      RaggedShape result = Append(0, num_shape, shapes.data());
      ASSERT_EQ(result.NumAxes(), num_axes);

      // get result splits with `SpliceRowSplits` and get result row-ids with
      // `RowSplitsToRowIds``
      std::vector<Array1<int32_t>> result_splits;
      std::vector<Array1<int32_t>> result_ids;
      for (int32_t axis = 1; axis < num_axes; ++axis) {
        std::vector<Array1<int32_t>> splits_vec(num_shape);
        std::vector<const Array1<int32_t> *> splits_vec_ptr(num_shape);
        for (int32_t n = 0; n != num_shape; ++n) {
          splits_vec[n] = shape_vec[n].RowSplits(axis);
          splits_vec_ptr[n] = &splits_vec[n];
        }
        Array1<int32_t> curr_row_splits =
            SpliceRowSplits(num_shape, splits_vec_ptr.data());
        result_splits.push_back(curr_row_splits);
        Array1<int32_t> curr_row_ids(context, curr_row_splits.Back());
        RowSplitsToRowIds(curr_row_splits, curr_row_ids);
        result_ids.push_back(curr_row_ids);
      }

      // check data
      for (int32_t axis = 1; axis < num_axes; ++axis) {
        CheckArrayData(result.RowSplits(axis), result_splits[axis - 1]);
        CheckArrayData(result.RowIds(axis), result_ids[axis - 1]);
      }
    }
  }
}
TEST(RaggedShapeOpsTest, TestAppend) {
  TestAppend<kCpu>();
  TestAppend<kCuda>();
}
void CheckResultOfRenumber(const ContextPtr &context, RaggedShape &shape,
                           Array1<int32_t> &new2old, RaggedShape &result) {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  int32_t num_axes = shape.NumAxes();
  int32_t dim0 = shape.Dim0();
  if (dim0 == 0) {
    std::vector<int32_t> empty_row_splits = {0};
    for (int32_t i = 0; i < num_axes - 1; ++i) {
      CheckArrayData(result.RowSplits(i + 1), empty_row_splits);
      EXPECT_EQ(result.RowIds(i + 1).Dim(), 0);
    }
    return;
  }
  Array2<int32_t> old_offsets(context, num_axes, dim0 + 1);
  auto old_offsets_acc = old_offsets.Accessor();
  Array1<int32_t *> row_splits_ptrs = GetRowSplitsPtr(shape);
  int32_t **row_splits_ptrs_data = row_splits_ptrs.Data();
  // Set old_offsets
  auto lambda_get_old_offsets = [=] __host__ __device__(int32_t i) {
    // 0 <= i <= dim0
    int32_t cur_offset = i;
    for (int32_t axis = 0; axis < num_axes; axis++) {
      old_offsets_acc(axis, i) = cur_offset;
      if (axis + 1 == num_axes) return;
      cur_offset = row_splits_ptrs_data[axis][cur_offset];
    }
  };
  Eval(context, dim0 + 1, lambda_get_old_offsets);
  old_offsets = old_offsets.To(cpu);
  auto cpu_offsets_acc = old_offsets.Accessor();
  shape = shape.To(cpu);
  new2old = new2old.To(cpu);
  // get result splits with `SpliceRowSplits` and get result row-ids with
  // `RowSplitsToRowIds``
  std::vector<Array1<int32_t>> result_splits;
  std::vector<Array1<int32_t>> result_ids;
  for (auto axis = 0; axis < num_axes - 1; ++axis) {
    Array1<int32_t> curr_row_splits = shape.RowSplits(axis + 1);
    std::vector<Array1<int32_t>> splits_vec(dim0);
    std::vector<const Array1<int32_t> *> splits_vec_ptr(dim0);
    for (int32_t m = 0; m != dim0; ++m) {
      int32_t old_idx = new2old[m];
      int32_t start = cpu_offsets_acc(axis, old_idx);
      int32_t end = cpu_offsets_acc(axis, old_idx + 1);
      Array1<int32_t> sub_list = curr_row_splits.Range(start, end - start + 1);
      Array1<int32_t> copy_sub_list(cpu, sub_list.Dim());
      copy_sub_list.CopyFrom(sub_list);
      int32_t *data = copy_sub_list.Data();
      int32_t init = data[0];
      for (int32_t n = 0; n != copy_sub_list.Dim(); ++n) {
        data[n] -= init;
      }
      splits_vec[m] = copy_sub_list;
      splits_vec_ptr[m] = &splits_vec[m];
    }
    Array1<int32_t> result_row_splits =
        SpliceRowSplits(dim0, splits_vec_ptr.data());
    result_splits.push_back(result_row_splits);
    Array1<int32_t> result_row_ids(cpu, result_row_splits.Back());
    RowSplitsToRowIds(result_row_splits, result_row_ids);
    result_ids.push_back(result_row_ids);
  }
  for (int32_t i = 0; i < num_axes - 1; ++i) {
    CheckArrayData(result.RowSplits(i + 1), result_splits[i]);
    CheckArrayData(result.RowIds(i + 1), result_ids[i]);
  }
}

template <DeviceType d>
void TestRenumber() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  {
    // simple case
    const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
    const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
    const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
    const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};
    // std::vector<std::vector<int32_t>> expected_row_splits = {
    //  {0, 3, 4, 6}, {0, 1, 3, 4, 7, 9, 10}};
    // std::vector<std::vector<int32_t>> expected_row_ids = {
    //   {0, 0, 0, 1, 2, 2}, {0, 1, 1, 2, 3, 3, 3, 4, 4, 5}};

    Array1<int32_t> splits1(context, row_splits1);
    Array1<int32_t> ids1(context, row_ids1);
    Array1<int32_t> splits2(context, row_splits2);
    Array1<int32_t> ids2(context, row_ids2);
    RaggedShape shape =
        RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2, &ids2, ids2.Dim());

    std::vector<int32_t> new2old_vec = {1, 2, 0};
    Array1<int32_t> new2old(context, new2old_vec);
    RaggedShape result = Renumber(shape, new2old);

    CheckResultOfRenumber(context, shape, new2old, result);
  }

  {
    // test with random large size
    for (int32_t i = 0; i < 2; ++i) {
      int32_t num_axes = RandInt(2, 4);
      RaggedShape shape =
          RandomRaggedShape(true, num_axes, num_axes, 0, 1000).To(context);
      int32_t dim0 = shape.Dim0();
      std::vector<int32_t> new2old_vec(dim0);
      std::iota(new2old_vec.begin(), new2old_vec.end(), 0);
      std::random_device rd;
      std::mt19937 g(rd());
      std::shuffle(new2old_vec.begin(), new2old_vec.end(), g);
      Array1<int32_t> new2old(context, new2old_vec);
      RaggedShape result = Renumber(shape, new2old);
      CheckResultOfRenumber(context, shape, new2old, result);
    }
  }
}
TEST(RaggedShapeOpsTest, TestRenumber) {
  TestRenumber<kCpu>();
  TestRenumber<kCuda>();
}
TEST(GetTransposeReordering, NoDuplicates) {
  // 0 0 0 9 2
  // 5 8 0 0 1
  // 0 0 3 0 0
  // 0 6 0 0 0
  std::vector<int32_t> col_indexes{3, 4, 0, 1, 4, 2, 1};
  std::vector<int32_t> _row_splits{0, 2, 5, 6, 7};
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    Array1<int32_t> row_splits(context, _row_splits);
    RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
    Array1<int32_t> values(context, col_indexes);

    Ragged<int32_t> ragged(shape, values);
    Array1<int32_t> order = GetTransposeReordering(ragged, 5);
    //   index 0 1 2 3 4 5 6
    // it maps 9 2 5 8 1 3 6 to
    //         5 8 6 3 9 2 1
    // so it returns
    //         2 3 6 5 0 1 4
    CheckArrayData(order, {2, 3, 6, 5, 0, 1, 4});
    EXPECT_TRUE(context->IsCompatible(*order.Context()));
  }
}

TEST(GetTransposeReordering, WithDuplicates) {
  // 0 0 0 (9,9,9)
  // 5 8 0     0
  // 0 0 (3,3) 0
  // 0 6 0     0
  std::vector<int32_t> col_indexes{3, 3, 3, 0, 1, 2, 2, 1};
  std::vector<int32_t> _row_splits{0, 3, 5, 7, 8};
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    Array1<int32_t> row_splits(context, _row_splits);
    RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
    Array1<int32_t> values(context, col_indexes);

    Ragged<int32_t> ragged(shape, values);
    Array1<int32_t> order = GetTransposeReordering(ragged, 4);
    //   index 0 1 2 3 4 5 6 7
    // it maps 9 9 9 5 8 3 3 6 to
    //         5 8 6 3 3 9 9 9
    // so it returns
    //         3 4 7 5 6 0 1 2   Note that it is stable
    CheckArrayData(order, {3, 4, 7, 5, 6, 0, 1, 2});
    EXPECT_TRUE(context->IsCompatible(*order.Context()));
  }
}

template <DeviceType d>
void TestGetCountsPartitioned() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  // Testing with simple case is good enough as we have tested GetCounts() with
  // random large size and GetCountsPartitioned just calls GetCounts.
  std::vector<int32_t> src_row_splits_vec = {0, 3, 4, 6, 10};
  Array1<int32_t> src_row_splits(context, src_row_splits_vec);
  RaggedShape src_shape = RaggedShape2(&src_row_splits, nullptr, -1);
  std::vector<int32_t> src_values_vec = {0, 1, 0, 2, 5, 5, 7, 7, 9, 7};
  Array1<int32_t> src_values(context, src_values_vec);
  Ragged<int32_t> src(src_shape, src_values);

  std::vector<int32_t> ans_row_splits_vec = {0, 2, 4, 7, 10};
  Array1<int32_t> ans_row_splits(context, ans_row_splits_vec);
  RaggedShape ans_shape = RaggedShape2(&ans_row_splits, nullptr, -1);

  Ragged<int32_t> result = GetCountsPartitioned(src, ans_shape);

  ASSERT_EQ(result.NumAxes(), 2);
  // Check row_splits
  Array1<int32_t> row_splits = result.shape.RowSplits(1).To(cpu);
  std::vector<int32_t> result_row_splits(row_splits.Data(),
                                         row_splits.Data() + row_splits.Dim());
  EXPECT_EQ(result_row_splits, ans_row_splits_vec);
  // check values
  std::vector<int32_t> expected_data = {2, 1, 1, 0, 0, 2, 0, 3, 0, 1};
  Array1<int32_t> values = result.values.To(cpu);
  std::vector<int32_t> data(values.Data(), values.Data() + values.Dim());
  EXPECT_EQ(data, expected_data);
}

TEST(RaggedShapeOpsTest, TestGetCountsPartitioned) {
  TestGetCountsPartitioned<kCpu>();
  TestGetCountsPartitioned<kCuda>();
}

template <DeviceType d>
void TestStack() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  {
    // simple case
    std::vector<RaggedShape> shapes(2);
    std::vector<const RaggedShape *> shapes_ptr(2);
    std::vector<std::vector<Array1<int32_t>>> row_splits_vec(2);
    {
      const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
      const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
      Array1<int32_t> splits1(context, row_splits1);
      Array1<int32_t> splits2(context, row_splits2);
      row_splits_vec[0].push_back(splits1);
      row_splits_vec[1].push_back(splits2);
      shapes[0] = RaggedShape3(&splits1, nullptr, -1, &splits2, nullptr, -1);
      shapes_ptr[0] = &shapes[0];
    }
    {
      const std::vector<int32_t> row_splits1 = {0, 1, 3, 4};
      const std::vector<int32_t> row_splits2 = {0, 3, 4, 5, 7};
      Array1<int32_t> splits1(context, row_splits1);
      Array1<int32_t> splits2(context, row_splits2);
      row_splits_vec[0].push_back(splits1);
      row_splits_vec[1].push_back(splits2);
      shapes[1] = RaggedShape3(&splits1, nullptr, -1, &splits2, nullptr, -1);
      shapes_ptr[1] = &shapes[1];
    }
    std::vector<std::vector<int32_t>> expected_row_splits = {
        {0, 3, 6},
        {0, 2, 5, 6, 7, 9, 10},
        {0, 2, 3, 4, 6, 7, 10, 13, 14, 15, 17}};

    {
      // axis == 0
      int32_t axis = 0;
      RaggedShape result = Stack(axis, 2, shapes_ptr.data());
      for (int32_t i = 0; i != 3; ++i) {
        CheckArrayData(result.RowSplits(i + 1), expected_row_splits[i]);
      }
    }
    {
      // axis == 1
      int32_t axis = 1;
      RaggedShape result = Stack(axis, 2, shapes_ptr.data());
      RaggedShape transpose = Transpose(result);
      for (int32_t i = 0; i != 3; ++i) {
        CheckArrayData(transpose.RowSplits(i + 1), expected_row_splits[i]);
      }
    }
  }

  {
    // test with random large size
    for (int32_t m = 0; m < 2; ++m) {
      int32_t num_shape = RandInt(2, 100);
      int32_t num_axes = RandInt(2, 4);
      int32_t dim0 = RandInt(1, 100);
      std::vector<RaggedShape> shape_vec(num_shape);
      std::vector<const RaggedShape *> shapes(num_shape);
      for (int32_t j = 0; j != num_shape; ++j) {
        RaggedShape shape =
            RandomRaggedShape(false, num_axes, num_axes, 0, 1000).To(context);
        int32_t src_dim0 = shape.Dim0();
        std::vector<int32_t> row_splits_vec(dim0 + 1);
        row_splits_vec[0] = 0;
        for (int32_t n = 1; n < dim0; ++n) {
          row_splits_vec[n] = RandInt(0, src_dim0);
        }
        row_splits_vec[dim0] = src_dim0;
        std::sort(row_splits_vec.begin(), row_splits_vec.end());
        Array1<int32_t> row_splits(context, row_splits_vec);
        RaggedShape first = RaggedShape2(&row_splits, nullptr, -1);
        RaggedShape new_shape = ComposeRaggedShapes(first, shape);
        shape_vec[j] = new_shape;
        shapes[j] = &shape_vec[j];
      }
      std::vector<RaggedShape> cpu_shapes(num_shape);
      for (auto i = 0; i != num_shape; ++i) {
        cpu_shapes[i] = shape_vec[i].To(cpu);
      }

      {
        // axis == 0
        int32_t axis = 0;
        RaggedShape result = Stack(axis, num_shape, shapes.data());
        ASSERT_EQ(result.NumAxes(),
                  num_axes + 2);  // note we append one axis in each shape in
                                  // `shapes` before `Stack`
        ASSERT_EQ(result.Dim0(), num_shape);
        result = result.To(cpu);
        for (auto iter = result.Iterator(); !iter.Done(); iter.Next()) {
          std::vector<int32_t> index = iter.Value();
          int32_t t = result[index];  // don't need the value, just make sure
                                      // it's a valid index.
          int32_t i = index[0];
          index.erase(index.begin());
          // result[i,j,k,l] = (shape[i])[j,k,l]
          i = cpu_shapes[i][index];  // don't need the value, just need to make
                                     // sure it's an allowable index.
        }
      }
      {
        // axis == 1
        int32_t axis = 1;
        RaggedShape result = Stack(axis, num_shape, shapes.data());
        ASSERT_EQ(result.NumAxes(),
                  num_axes + 2);  // note we append one axis in each shape in
                                  // `shapes` before `Stack`
        ASSERT_EQ(result.Dim0(), dim0);
        result = result.To(cpu);
        for (auto iter = result.Iterator(); !iter.Done(); iter.Next()) {
          std::vector<int32_t> index = iter.Value();
          int32_t t = result[index];  // don't need the value, just make sure
                                      // it's a valid index.
          int32_t i = index[1];
          index.erase(index.begin() + 1);
          // result[i,j,k,l] = (shape[j])[i,k,l]
          i = cpu_shapes[i][index];  // don't need the value, just need to make
                                     // sure it's an allowable index.
        }
      }
    }
  }
}
TEST(RaggedShapeOpsTest, TestStack) {
  TestStack<kCpu>();
  TestStack<kCuda>();
}

}  // namespace k2
