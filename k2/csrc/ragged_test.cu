/**
 * @brief
 * ragged_test
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey, Haowen Qiu)
 *                      Mobvoi Inc.        (authors: Fangjun Kuang)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include <algorithm>
#include <numeric>
#include <utility>
#include <vector>

#include "k2/csrc/array.h"
#include "k2/csrc/array_ops.h"
#include "k2/csrc/context.h"
#include "k2/csrc/math.h"
#include "k2/csrc/ragged.h"
#include "k2/csrc/tensor.h"

namespace {
// TODO(haowen): may move below functions to some file like `test_utils.h`,
// in case other Tests may use it?
template <typename T>
static void CheckArrayData(const k2::Array1<T> &array,
                           const std::vector<T> &target) {
  ASSERT_EQ(array.Dim(), target.size());
  const T *array_data = array.Data();
  // copy data from CPU/GPU to CPU
  auto kind = k2::GetMemoryCopyKind(*array.Context(), *k2::GetCpuContext());
  std::vector<T> cpu_data(array.Dim());
  k2::MemoryCopy(static_cast<void *>(cpu_data.data()),
                 static_cast<const void *>(array_data),
                 array.Dim() * array.ElementSize(), kind,
                 nullptr);
  EXPECT_EQ(cpu_data, target);
}

static void CheckRowSplits(k2::RaggedShape &shape,
                           const std::vector<std::vector<int32_t>> &target) {
  for (int32_t i = 1; i < shape.NumAxes(); ++i) {
    k2::Array1<int32_t> curr_row_splits = shape.RowSplits(i);
    CheckArrayData<int32_t>(curr_row_splits, target[i - 1]);
  }
}

// check if `array` and `target` have the same values
template <typename T>
static void CheckArrayData(const k2::Array1<T> &array,
                           const k2::Array1<T> &target) {
  ASSERT_EQ(array.Dim(), target.Dim());
  int32_t dim = array.Dim();
  k2::ContextPtr cpu = k2::GetCpuContext();
  k2::Array1<T> cpu_array = array.To(cpu);
  k2::Array1<T> cpu_target = target.To(cpu);
  std::vector<T> array_data(cpu_array.Data(), cpu_array.Data() + dim);
  std::vector<T> target_data(cpu_target.Data(), cpu_target.Data() + dim);
  EXPECT_EQ(array_data, target_data);
}
}  // namespace

namespace k2 {
class RaggedShapeOpsSuiteTest : public ::testing::Test {
 protected:
  RaggedShapeOpsSuiteTest() {
    ContextPtr context = GetCpuContext();
    const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
    const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
    const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
    const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};
    const std::vector<int32_t> row_splits3 = {0,  2,  3,  5,  8, 9,
                                              12, 13, 15, 15, 16};
    const std::vector<int32_t> row_ids3 = {0, 0, 1, 2, 2, 3, 3, 3,
                                           4, 5, 5, 5, 6, 7, 7, 9};
    std::vector<RaggedShapeDim> axes;
    axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits1),
                                     Array1<int32_t>(context, row_ids1),
                                     static_cast<int32_t>(row_ids1.size())});
    axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits2),
                                     Array1<int32_t>(context, row_ids2),
                                     static_cast<int32_t>(row_ids2.size())});
    axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits3),
                                     Array1<int32_t>(context, row_ids3),
                                     static_cast<int32_t>(row_ids3.size())});

    simple_shape_ = RaggedShape(axes, true);

    // random_shape_ is on CPU
    random_shape_ = RandomRaggedShape(true,   // set_row_ids
                                      3,      // min_num_axes
                                      4,      // max_num_axes
                                      0,      // min_num_elements
                                      1000);  // max_num_elements
  }

  RaggedShape simple_shape_;
  RaggedShape random_shape_;
};

void TestUnsqueeze(ContextPtr context, const RaggedShape &input_shape) {
  RaggedShape src_shape = input_shape.To(context);
  src_shape.Populate();  // set row_ids
  {
    // axis = 0.
    RaggedShape shape = Unsqueeze(src_shape, 0);
    int32_t dim0 = src_shape.Dim0();
    const std::vector<RaggedShapeDim> &src_axes = src_shape.Axes();
    const std::vector<RaggedShapeDim> &dest_axes = shape.Axes();

    {
      const Array1<int32_t> &row_splits0 = dest_axes[0].row_splits;
      std::vector<int32_t> data = {0, dim0};
      CheckArrayData(row_splits0, data);
    }

    {
      const Array1<int32_t> &row_ids0 = dest_axes[0].row_ids;
      std::vector<int32_t> data(dim0, 0);
      CheckArrayData(row_ids0, data);
    }

    {
      for (auto i = 0; i != src_axes.size(); ++i) {
        CheckArrayData(src_axes[i].row_splits, dest_axes[i + 1].row_splits);
        CheckArrayData(src_axes[i].row_ids, dest_axes[i + 1].row_ids);
      }
    }
  }

  {
    // axis = 1
    int32_t axis = 1;
    RaggedShape shape = Unsqueeze(src_shape, axis);
    int32_t tot_size = shape.TotSize(axis);
    const std::vector<RaggedShapeDim> &src_axes = src_shape.Axes();
    const std::vector<RaggedShapeDim> &dest_axes = shape.Axes();

    {
      for (auto i = 0; i < axis; ++i) {
        CheckArrayData(src_axes[i].row_splits, dest_axes[i].row_splits);
        CheckArrayData(src_axes[i].row_ids, dest_axes[i].row_ids);
      }
    }

    {
      const Array1<int32_t> &row_splits = dest_axes[axis].row_splits;
      std::vector<int32_t> data(tot_size + 1);
      std::iota(data.begin(), data.end(), 0);
      CheckArrayData(row_splits, data);
    }

    {
      const Array1<int32_t> &row_ids = dest_axes[axis].row_ids;
      std::vector<int32_t> data(tot_size);
      std::iota(data.begin(), data.end(), 0);
      CheckArrayData(row_ids, data);
    }

    {
      for (auto i = axis; i < src_axes.size(); ++i) {
        CheckArrayData(src_axes[i].row_splits, dest_axes[i + 1].row_splits);
        CheckArrayData(src_axes[i].row_ids, dest_axes[i + 1].row_ids);
      }
    }
  }
}

TEST_F(RaggedShapeOpsSuiteTest, TestUnsqueezeCpu) {
  TestUnsqueeze(GetCpuContext(), simple_shape_);
  TestUnsqueeze(GetCpuContext(), random_shape_);
}
TEST_F(RaggedShapeOpsSuiteTest, TestUnsqueezeGpu) {
  TestUnsqueeze(GetCudaContext(), simple_shape_);
  TestUnsqueeze(GetCudaContext(), random_shape_);
}

void TestRemoveAxis(ContextPtr context, const RaggedShape &input_shape) {
  RaggedShape src_shape = input_shape.To(context);
  ASSERT_EQ(src_shape.NumAxes(), 4);
  {
    // axis = 0.
    int32_t axis = 0;
    RaggedShape shape = RemoveAxis(src_shape, axis);
    const std::vector<RaggedShapeDim> &src_axes = src_shape.Axes();
    const std::vector<RaggedShapeDim> &dest_axes = shape.Axes();
    ASSERT_EQ(src_axes.size(), 3);
    ASSERT_EQ(dest_axes.size(), 2);

    {
      for (auto i = 0; i != dest_axes.size(); ++i) {
        CheckArrayData(dest_axes[i].row_splits, src_axes[i + 1].row_splits);
        CheckArrayData(dest_axes[i].row_ids, src_axes[i + 1].row_ids);
      }
    }
  }

  {
    // axis = 1
    int32_t axis = 1;
    RaggedShape shape = RemoveAxis(src_shape, axis);
    const std::vector<RaggedShapeDim> &src_axes = src_shape.Axes();
    const std::vector<RaggedShapeDim> &dest_axes = shape.Axes();
    ASSERT_EQ(src_axes.size(), 3);
    ASSERT_EQ(dest_axes.size(), 2);

    {
      const Array1<int32_t> &row_splits0 = dest_axes[0].row_splits;
      std::vector<int32_t> data = {0, 3, 7, 10};
      CheckArrayData(row_splits0, data);
    }

    {
      const Array1<int32_t> &row_ids0 = dest_axes[0].row_ids;
      std::vector<int32_t> data = {0, 0, 0, 1, 1, 1, 1, 2, 2, 2};
      CheckArrayData(row_ids0, data);
    }

    {
      for (auto i = 1; i != dest_axes.size(); ++i) {
        CheckArrayData(dest_axes[i].row_splits, src_axes[i + 1].row_splits);
        CheckArrayData(dest_axes[i].row_ids, src_axes[i + 1].row_ids);
      }
    }
  }

  {
    // axis = 3
    int32_t axis = 3;  // the last axis
    RaggedShape shape = RemoveAxis(src_shape, axis);
    const std::vector<RaggedShapeDim> &src_axes = src_shape.Axes();
    const std::vector<RaggedShapeDim> &dest_axes = shape.Axes();
    ASSERT_EQ(src_axes.size(), 3);
    ASSERT_EQ(dest_axes.size(), 2);

    {
      for (auto i = 0; i != dest_axes.size(); ++i) {
        CheckArrayData(dest_axes[i].row_splits, src_axes[i].row_splits);
        CheckArrayData(dest_axes[i].row_ids, src_axes[i].row_ids);
      }
    }
  }
}

TEST_F(RaggedShapeOpsSuiteTest, TestRemoveAxisCpu) {
  TestRemoveAxis(GetCpuContext(), simple_shape_);
}
TEST_F(RaggedShapeOpsSuiteTest, TestRemoveAxisGpu) {
  TestRemoveAxis(GetCudaContext(), simple_shape_);
}

void TestGetOffsets(ContextPtr context) {
  for (int32_t i = 0; i != 2; ++i) {
    int32_t num_shape = RandInt(10, 100);
    int32_t num_axes = RandInt(2, 4);
    std::vector<RaggedShape> shape_vec(num_shape);
    std::vector<RaggedShape *> shapes(num_shape);
    for (int32_t j = 0; j != num_shape; ++j) {
      shape_vec[j] =
          RandomRaggedShape(false, num_axes, num_axes, 0, 1000).To(context);
      shapes[j] = &shape_vec[j];
    }
    RaggedShape **shapes_ptr = shapes.data();
    Array2<int32_t> offsets = GetOffsets(num_shape, shapes_ptr);
    ASSERT_EQ(offsets.Dim0(), num_axes + 1);
    ASSERT_EQ(offsets.Dim1(), num_shape + 1);
    auto acc = offsets.Accessor();
    for (int32_t axis = 0; axis <= num_axes; ++axis) {
      int32_t sum = 0;
      for (int32_t j = 0; j <= num_shape; ++j) {
        EXPECT_EQ(acc(axis, j), sum);
        if (j < num_shape) {
          sum += (axis == 0 ? 1 : shape_vec[j].TotSize(axis - 1));
        }
      }
    }
  }
}

TEST(RaggedShapeOpsTest, TestGetOffsets) {
  TestGetOffsets(GetCpuContext());
  TestGetOffsets(GetCudaContext());
}

// returns a random ragged shape where the dims on axis 1 are all the same
// (so: can be transposed).
RaggedShape RandomRaggedShapeToTranspose(ContextPtr c) {
  ContextPtr c_cpu = GetCpuContext();

  RaggedShape random = RandomRaggedShape(false, 2, 4, 0, 5000).To(c);

  int32_t input_dim0 = random.Dim0(), divisor = 1;
  for (int32_t i = 1; i * i <= input_dim0; i++) {
    if (input_dim0 % i == 0 && i > divisor) divisor = i;
  }

  int32_t output_dim0 = divisor, output_dim1 = input_dim0 / divisor;

  Array1<int32_t> row_splits =
      Range<int32_t>(c, output_dim0 + 1, 0, output_dim1);
  int32_t cached_tot_size = input_dim0;

  RaggedShape top_level_shape =
      RaggedShape2(&row_splits, nullptr, cached_tot_size);
  return ComposeRaggedShapes(top_level_shape, random);
}

template <DeviceType d>
void TestTranspose() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  {
    const std::vector<int32_t> row_splits1_vec = {0, 2, 4, 6};
    const std::vector<int32_t> row_splits2_vec = {0, 3, 4, 7, 8, 10, 12};
    Array1<int32_t> row_splits1(context, row_splits1_vec);
    Array1<int32_t> row_splits2(context, row_splits2_vec);
    RaggedShape src_shape =
        RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);
    ASSERT_EQ(src_shape.Dim0(), 3);
    ASSERT_EQ(src_shape.TotSize(1), 6);
    RaggedShape shape = Transpose(src_shape);
    EXPECT_EQ(shape.Dim0(), 2);
    ASSERT_EQ(shape.TotSize(1), 6);
    const std::vector<int32_t> expected_row_splits = {0, 3, 6};
    const std::vector<int32_t> expected_row_ids = {0, 0, 0, 1, 1, 1};
    CheckArrayData(shape.RowSplits(1), expected_row_splits);
    CheckArrayData(shape.RowIds(1), expected_row_ids);
    CheckArrayData(shape.RowSplits(2), src_shape.RowSplits(2));
    CheckArrayData(shape.RowIds(2), src_shape.RowIds(2));
  }

  {
    // random case
    RaggedShape to_transpose = RandomRaggedShapeToTranspose(context);
    RaggedShape transposed = Transpose(to_transpose);

    if (d != kCpu) {
      to_transpose = to_transpose.To(cpu);
      transposed = transposed.To(cpu);
    }

    for (auto iter = transposed.Iterator(); !iter.Done(); iter.Next()) {
      std::vector<int32_t> index = iter.Value();
      int32_t i = transposed[index];  // Just make sure this doesn't crash, dont
                                      // need the value.
      std::swap(index[0], index[1]);
      i = to_transpose[index];  // don't need the value, just need to make
                                // sure it's an allowable index.
    }
    for (auto iter = to_transpose.Iterator(); !iter.Done(); iter.Next()) {
      std::vector<int32_t> index = iter.Value();
      std::swap(index[0], index[1]);
      int32_t i = transposed[index];  // don't need the value, just need to make
                                      // sure it's an allowable index.
    }
  }
}
TEST(RaggedShapeOpsTest, TestTranspose) {
  // TODO(haowen): uncomment after testing Renumber
  // TestTranspose<kCpu>();
  // TestTranspose<kCuda>();
}

template <DeviceType d>
void TestRowSplitsPtr() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }
  RaggedShape shape = RandomRaggedShape().To(context);
  ASSERT_GE(shape.NumAxes(), 2);
  Array1<int32_t *> ptrs = GetRowSplitsPtr(shape);
  ASSERT_EQ(ptrs.Dim(), shape.NumAxes() - 1);
  // as num_axes is not so big, access (may copy memory) it in a loop is fine.
  for (int32_t i = 0; i != ptrs.Dim(); ++i) {
    EXPECT_EQ(ptrs[i], shape.RowSplits(i + 1).Data());
  }
}
TEST(RaggedShapeOpsTest, TestRowSplitsPtr) {
  TestRowSplitsPtr<kCpu>();
  TestRowSplitsPtr<kCuda>();
}

void TestRaggedShape2(ContextPtr context, const RaggedShape &shape) {
  RaggedShape src_shape = shape.To(context);
  src_shape.Populate();
  ASSERT_GE(src_shape.NumAxes(), 2);
  Array1<int32_t> row_splits = src_shape.RowSplits(1);
  Array1<int32_t> row_ids = src_shape.RowIds(1);
  int32_t cached_tot_size = src_shape.TotSize(1);

  {
    // both row_splits and row_ids are non-null
    RaggedShape result = RaggedShape2(&row_splits, &row_ids, cached_tot_size);
    CheckArrayData(result.RowSplits(1), row_splits);
    CheckArrayData(result.RowIds(1), row_ids);
    EXPECT_EQ(result.TotSize(1), cached_tot_size);
  }
  {
    // both row_splits and row_ids are non-null, cached_tot_size = -1
    RaggedShape result = RaggedShape2(&row_splits, &row_ids, -1);
    CheckArrayData(result.RowSplits(1), row_splits);
    CheckArrayData(result.RowIds(1), row_ids);
    EXPECT_EQ(result.TotSize(1), cached_tot_size);
  }
  {
    // row_ids is null
    RaggedShape result = RaggedShape2(&row_splits, nullptr, cached_tot_size);
    CheckArrayData(result.RowSplits(1), row_splits);
    CheckArrayData(result.RowIds(1), row_ids);
    EXPECT_EQ(result.TotSize(1), cached_tot_size);
  }
  {
    // row_ids is null, cached_tot_size = -1
    RaggedShape result = RaggedShape2(&row_splits, nullptr, -1);
    CheckArrayData(result.RowSplits(1), row_splits);
    CheckArrayData(result.RowIds(1), row_ids);
    EXPECT_EQ(result.TotSize(1), cached_tot_size);
  }

  // note if row_splits == null, then we suppose there's no empty rows after the
  // last row-id in row_ids
  if (row_splits.Dim() == (row_ids.Dim() == 0 ? 1 : row_ids.Back() + 2)) {
    {
      // row_splits is null
      RaggedShape result = RaggedShape2(nullptr, &row_ids, cached_tot_size);
      CheckArrayData(result.RowSplits(1), row_splits);
      CheckArrayData(result.RowIds(1), row_ids);
      EXPECT_EQ(result.TotSize(1), cached_tot_size);
    }
    {
      // row_splits is null, cached_tot_size = -1
      RaggedShape result = RaggedShape2(nullptr, &row_ids, -1);
      CheckArrayData(result.RowSplits(1), row_splits);
      CheckArrayData(result.RowIds(1), row_ids);
      EXPECT_EQ(result.TotSize(1), cached_tot_size);
    }
  }
}
TEST_F(RaggedShapeOpsSuiteTest, TestRaggedShape2Cpu) {
  TestRaggedShape2(GetCpuContext(), simple_shape_);
  TestRaggedShape2(GetCpuContext(), random_shape_);
}
TEST_F(RaggedShapeOpsSuiteTest, TestRaggedShape2Gpu) {
  TestRaggedShape2(GetCudaContext(), simple_shape_);
  TestRaggedShape2(GetCudaContext(), random_shape_);
}

void TestRaggedShape3(ContextPtr context, const RaggedShape &shape) {
  RaggedShape src_shape = shape.To(context);
  src_shape.Populate();
  ASSERT_GE(src_shape.NumAxes(), 3);
  Array1<int32_t> row_splits1 = src_shape.RowSplits(1);
  Array1<int32_t> row_ids1 = src_shape.RowIds(1);
  int32_t cached_tot_size1 = src_shape.TotSize(1);
  Array1<int32_t> row_splits2 = src_shape.RowSplits(2);
  Array1<int32_t> row_ids2 = src_shape.RowIds(2);
  int32_t cached_tot_size2 = src_shape.TotSize(2);

  {
    // both row_splits and row_ids are non-null
    RaggedShape result =
        RaggedShape3(&row_splits1, &row_ids1, cached_tot_size1, &row_splits2,
                     &row_ids2, cached_tot_size2);
    CheckArrayData(result.RowSplits(1), row_splits1);
    CheckArrayData(result.RowIds(1), row_ids1);
    EXPECT_EQ(result.TotSize(1), cached_tot_size1);
    CheckArrayData(result.RowSplits(2), row_splits2);
    CheckArrayData(result.RowIds(2), row_ids2);
    EXPECT_EQ(result.TotSize(2), cached_tot_size2);
  }
  {
    // row_ids is non-null, cached_tot_size = -1
    RaggedShape result =
        RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);
    CheckArrayData(result.RowSplits(1), row_splits1);
    CheckArrayData(result.RowIds(1), row_ids1);
    EXPECT_EQ(result.TotSize(1), cached_tot_size1);
    CheckArrayData(result.RowSplits(2), row_splits2);
    CheckArrayData(result.RowIds(2), row_ids2);
    EXPECT_EQ(result.TotSize(2), cached_tot_size2);
  }

  // note if row_splits == null, then we suppose there's no empty rows after the
  // last row-id in row_ids
  bool valid1 =
      (row_splits1.Dim() == (row_ids1.Dim() == 0 ? 1 : row_ids1.Back() + 2));
  bool valid2 =
      (row_splits2.Dim() == (row_ids2.Dim() == 0 ? 1 : row_ids2.Back() + 2));
  if (valid1 && valid2) {
    RaggedShape result =
        RaggedShape3(nullptr, &row_ids1, -1, nullptr, &row_ids2, -1);
    CheckArrayData(result.RowSplits(1), row_splits1);
    CheckArrayData(result.RowIds(1), row_ids1);
    EXPECT_EQ(result.TotSize(1), cached_tot_size1);
    CheckArrayData(result.RowSplits(2), row_splits2);
    CheckArrayData(result.RowIds(2), row_ids2);
    EXPECT_EQ(result.TotSize(2), cached_tot_size2);
  }
  // TODO(haowen): add more cases for other branches
}
TEST_F(RaggedShapeOpsSuiteTest, TestRaggedShape3Cpu) {
  TestRaggedShape3(GetCpuContext(), simple_shape_);
  TestRaggedShape3(GetCpuContext(), random_shape_);
}
TEST_F(RaggedShapeOpsSuiteTest, TestRaggedShape3Gpu) {
  TestRaggedShape3(GetCudaContext(), simple_shape_);
  TestRaggedShape3(GetCudaContext(), random_shape_);
}

void TestComposeShape(ContextPtr context, const RaggedShape &shape) {
  RaggedShape src_shape = shape.To(context);
  ASSERT_GE(src_shape.NumAxes(), 3);
  Array1<int32_t> row_splits1 = src_shape.RowSplits(1);
  Array1<int32_t> row_ids1 = src_shape.RowIds(1);
  Array1<int32_t> row_splits2 = src_shape.RowSplits(2);
  Array1<int32_t> row_ids2 = src_shape.RowIds(2);

  RaggedShape shape1 = RaggedShape2(&row_splits1, nullptr, -1);
  RaggedShape shape2 = RaggedShape2(&row_splits2, nullptr, -1);

  RaggedShape result = ComposeRaggedShapes(shape1, shape2);

  ASSERT_EQ(result.NumAxes(), 3);

  CheckArrayData(result.RowSplits(1), row_splits1);
  CheckArrayData(result.RowIds(1), row_ids1);
  CheckArrayData(result.RowSplits(2), row_splits2);
  CheckArrayData(result.RowIds(2), row_ids2);
}
TEST_F(RaggedShapeOpsSuiteTest, TestComposeShapeCpu) {
  TestComposeShape(GetCpuContext(), simple_shape_);
  TestComposeShape(GetCpuContext(), random_shape_);
}
TEST_F(RaggedShapeOpsSuiteTest, TestComposeShapeGpu) {
  TestComposeShape(GetCudaContext(), simple_shape_);
  TestComposeShape(GetCudaContext(), random_shape_);
}

void TestShapeFromTotSize(ContextPtr context, const RaggedShape &shape) {
  RaggedShape src_shape = shape.To(context);
  ASSERT_GE(src_shape.NumAxes(), 2);

  int32_t num_axes = src_shape.NumAxes();
  std::vector<int32_t> tot_sizes(num_axes);
  for (int32_t i = 0; i != num_axes; ++i) {
    tot_sizes[i] = src_shape.TotSize(i);
  }

  RaggedShape result =
      RaggedShapeFromTotSizes(context, num_axes, tot_sizes.data());

  ASSERT_EQ(result.NumAxes(), num_axes);
  for (int32_t i = 0; i < num_axes; ++i) {
    EXPECT_EQ(result.TotSize(i), src_shape.TotSize(i));
    if (i > 0) {
      EXPECT_EQ(result.RowSplits(i).Dim(), src_shape.RowSplits(i).Dim());
      EXPECT_EQ(result.RowIds(i).Dim(), src_shape.RowIds(i).Dim());
    }
  }
}
TEST_F(RaggedShapeOpsSuiteTest, TestShapeFromTotSizeCpu) {
  TestShapeFromTotSize(GetCpuContext(), simple_shape_);
  TestShapeFromTotSize(GetCpuContext(), random_shape_);
}
TEST_F(RaggedShapeOpsSuiteTest, TestShapeFromTotSizeGpu) {
  TestShapeFromTotSize(GetCudaContext(), simple_shape_);
  TestShapeFromTotSize(GetCudaContext(), random_shape_);
}

template <typename T, DeviceType d>
void TestRagged() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  {
    // constructed with row_splits and row_ids
    // RaggedTensor4 t = [
    //  [ [[ 1, 2], [4]],  [[3, 0]] ],
    //  [ [[7, 8, 9]], [[6], [3, 5, 7]], [[2]] ],
    //  [ [[3, 4], [], [8]] ]
    // ]
    const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
    const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
    const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
    const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};
    const std::vector<int32_t> row_splits3 = {0,  2,  3,  5,  8, 9,
                                              12, 13, 15, 15, 16};
    const std::vector<int32_t> row_ids3 = {0, 0, 1, 2, 2, 3, 3, 3,
                                           4, 5, 5, 5, 6, 7, 7, 9};
    const std::vector<T> values_vec = {1, 2, 4, 3, 0, 7, 8, 9,
                                       6, 3, 5, 7, 2, 3, 4, 8};
    std::vector<RaggedShapeDim> axes;
    axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits1),
                                     Array1<int32_t>(context, row_ids1),
                                     static_cast<int32_t>(row_ids1.size())});
    axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits2),
                                     Array1<int32_t>(context, row_ids2),
                                     static_cast<int32_t>(row_ids2.size())});
    axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits3),
                                     Array1<int32_t>(context, row_ids3),
                                     static_cast<int32_t>(row_ids3.size())});

    RaggedShape shape(axes, true);
    Array1<T> values(context, values_vec);
    Ragged<T> ragged(shape, values);

    // test Index(axis, i)
    {
      // values: [[[ 1, 2], [4]], [[3, 0]]]
      Ragged<T> sub_raggged = ragged.Index(0, 0);
      RaggedShape &sub_shape = sub_raggged.shape;
      EXPECT_EQ(sub_shape.NumAxes(), 3);
      const std::vector<std::vector<int32_t>> sub_row_splits_vec = {
          {0, 2, 3}, {0, 2, 3, 5}};
      CheckRowSplits(sub_shape, sub_row_splits_vec);
      const Array1<T> &sub_values = sub_raggged.values;
      const std::vector<T> sub_values_vec = {1, 2, 4, 3, 0};
      CheckArrayData<T>(sub_values, sub_values_vec);
    }
    {
      // values: [[[7, 8, 9]], [[6], [3, 5, 7]], [[2]]]
      Ragged<T> sub_raggged = ragged.Index(0, 1);
      RaggedShape &sub_shape = sub_raggged.shape;
      EXPECT_EQ(sub_shape.NumAxes(), 3);
      const std::vector<std::vector<int32_t>> sub_row_splits_vec = {
          {0, 1, 3, 4}, {0, 3, 4, 7, 8}};
      CheckRowSplits(sub_shape, sub_row_splits_vec);
      const Array1<T> &sub_values = sub_raggged.values;
      const std::vector<T> sub_values_vec = {7, 8, 9, 6, 3, 5, 7, 2};
      CheckArrayData<T>(sub_values, sub_values_vec);
    }
    {
      // values: [[[3, 4], [], [8]]]
      Ragged<T> sub_raggged = ragged.Index(0, 2);
      RaggedShape &sub_shape = sub_raggged.shape;
      EXPECT_EQ(sub_shape.NumAxes(), 3);
      const std::vector<std::vector<int32_t>> sub_row_splits_vec = {
          {0, 3}, {0, 2, 2, 3}};
      CheckRowSplits(sub_shape, sub_row_splits_vec);
      const Array1<T> &sub_values = sub_raggged.values;
      const std::vector<T> sub_values_vec = {3, 4, 8};
      CheckArrayData<T>(sub_values, sub_values_vec);
    }

    // test operator[](const std::vector<int32_t> &indexes)
    if (d == kCpu) {
      {
        std::vector<int32_t> indexes = {0, 0, 0, 0};
        EXPECT_EQ(ragged.shape[indexes], 0);
        EXPECT_EQ(ragged[indexes], 1);
      }
      {
        std::vector<int32_t> indexes = {0, 1, 0, 0};
        EXPECT_EQ(ragged.shape[indexes], 3);
        EXPECT_EQ(ragged[indexes], 3);
      }
      {
        std::vector<int32_t> indexes = {1, 0, 0, 1};
        EXPECT_EQ(ragged.shape[indexes], 6);
        EXPECT_EQ(ragged[indexes], 8);
      }
      {
        std::vector<int32_t> indexes = {1, 1, 1, 0};
        EXPECT_EQ(ragged.shape[indexes], 9);
        EXPECT_EQ(ragged[indexes], 3);
      }
      {
        std::vector<int32_t> indexes = {2, 0, 0, 1};
        EXPECT_EQ(ragged.shape[indexes], 14);
        EXPECT_EQ(ragged[indexes], 4);
      }
      {
        std::vector<int32_t> indexes = {2, 0, 2, 0};
        EXPECT_EQ(ragged.shape[indexes], 15);
        EXPECT_EQ(ragged[indexes], 8);
      }
    }

    const std::vector<std::vector<int32_t>> row_splits_vec = {
        row_splits1, row_splits2, row_splits3};
    // test To(ctx)
    {
      // to GPU
      Ragged<T> other = ragged.To(GetCudaContext());
      CheckRowSplits(other.shape, row_splits_vec);
      CheckArrayData<T>(other.values, values_vec);
    }
    {
      // to CPU
      Ragged<T> other = ragged.To(GetCpuContext());
      CheckRowSplits(other.shape, row_splits_vec);
      CheckArrayData<T>(other.values, values_vec);
    }
  }
}

template <typename T, typename OP = LessThan<T>>
static void CpuSortSublists(const Array1<int32_t> &row_splits, Array1<T> *src) {
  K2_CHECK(src->Context()->GetDeviceType() == kCpu);
  T *p = src->Data();
  OP comp = OP();
  for (int32_t i = 0; i < row_splits.Dim() - 1; ++i) {
    int32_t cur = row_splits[i];
    int32_t next = row_splits[i + 1];
    std::sort(p + cur, p + next, comp);
  }
}

template <typename T, typename OP = LessThan<T>>
static void TestSortSublists() {
  auto cpu_context = GetCpuContext();
  auto cuda_context = GetCudaContext();

  RaggedShape shape = RandomRaggedShape(false,  // set_row_ids
                                        2,      // min_num_axes
                                        4,      // max_num_axes
                                        1,      // min_num_elements
                                        2000);  // max_num_elements

  Array1<T> values =
      RandUniformArray1<T>(shape.Context(), shape.NumElements(), -2000, 2000);
  Ragged<T> ragged(shape, values);
  ragged = ragged.To(cuda_context);
  values = values.To(cpu_context);  // to be sorted by cpu

  // TODO(fangjun): add a `Clone` method to Array1<T>
  Array1<T> unsorted = values.To(cuda_context).To(cpu_context);

  Array1<int32_t> order(ragged.Context(), ragged.values.Dim());
  SortSublists<T, OP>(&ragged, &order);

  Array1<int32_t> &segment = ragged.shape.RowSplits(ragged.NumAxes() - 1);
  CpuSortSublists<T, OP>(segment, &values);

  int32_t n = order.Dim();
  for (int i = 0; i != n; ++i) {
    EXPECT_EQ(values[i], ragged.values[i]);
    EXPECT_EQ(ragged.values[i], unsorted[order[i]]);
  }
}

TEST(RaggedTest, Ragged) {
  TestRagged<int32_t, kCuda>();
  TestRagged<int32_t, kCpu>();
  TestRagged<double, kCuda>();
  TestRagged<double, kCpu>();

  TestSortSublists<int32_t>();
  TestSortSublists<double>();
}

template <DeviceType d>
void TestAppend() {
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  ContextPtr context = nullptr;
  if (d == kCpu) {
    context = GetCpuContext();
  } else {
    K2_CHECK_EQ(d, kCuda);
    context = GetCudaContext();
  }

  {
    // simple case
    std::vector<RaggedShape> shapes(2);
    std::vector<RaggedShape *> shapes_ptr(2);
    std::vector<std::vector<Array1<int32_t>>> row_splits_vec(2);
    {
      const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
      const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
      const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
      const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};
      Array1<int32_t> splits1(context, row_splits1);
      Array1<int32_t> ids1(context, row_ids1);
      Array1<int32_t> splits2(context, row_splits2);
      Array1<int32_t> ids2(context, row_ids2);
      row_splits_vec[0].push_back(splits1);
      row_splits_vec[1].push_back(splits2);
      shapes[0] = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2, &ids2,
                               ids2.Dim());
      shapes_ptr[0] = &shapes[0];
    }
    {
      const std::vector<int32_t> row_splits1 = {0, 1, 3};
      const std::vector<int32_t> row_ids1 = {0, 1, 1};
      const std::vector<int32_t> row_splits2 = {0, 3, 4, 7};
      const std::vector<int32_t> row_ids2 = {0, 0, 0, 1, 2, 2, 2};
      Array1<int32_t> splits1(context, row_splits1);
      Array1<int32_t> ids1(context, row_ids1);
      Array1<int32_t> splits2(context, row_splits2);
      Array1<int32_t> ids2(context, row_ids2);
      row_splits_vec[0].push_back(splits1);
      row_splits_vec[1].push_back(splits2);
      RaggedShape shape = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2,
                                       &ids2, ids2.Dim());
      shapes[1] = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2, &ids2,
                               ids2.Dim());
      shapes_ptr[1] = &shapes[1];
    }

    RaggedShape result = Append(0, 2, shapes_ptr.data());

    // get result splits with `SpliceRowSplits` and get result row-ids with
    // `RowSplitsToRowIds``
    std::vector<Array1<int32_t>> result_splits;
    std::vector<Array1<int32_t>> result_ids;
    for (auto i = 0; i < 2; ++i) {
      std::vector<const Array1<int32_t> *> splits_ptr = {&row_splits_vec[i][0],
                                                         &row_splits_vec[i][1]};
      Array1<int32_t> curr_row_splits = SpliceRowSplits(2, splits_ptr.data());
      result_splits.push_back(curr_row_splits);
      Array1<int32_t> curr_row_ids(context, curr_row_splits.Back());
      RowSplitsToRowIds(curr_row_splits, curr_row_ids);
      result_ids.push_back(curr_row_ids);
    }
    for (int32_t i = 0; i < 2; ++i) {
      CheckArrayData(result.RowSplits(i + 1), result_splits[i]);
      CheckArrayData(result.RowIds(i + 1), result_ids[i]);
    }
  }

  {
    // test with random large size
    for (int32_t i = 0; i < 2; ++i) {
      int32_t num_shape = RandInt(2, 100);
      int32_t num_axes = RandInt(2, 4);
      std::vector<RaggedShape> shape_vec(num_shape);
      std::vector<RaggedShape *> shapes(num_shape);
      for (int32_t j = 0; j != num_shape; ++j) {
        shape_vec[j] =
            RandomRaggedShape(true, num_axes, num_axes, 0, 1000).To(context);
        shapes[j] = &shape_vec[j];
      }
      RaggedShape result = Append(0, num_shape, shapes.data());
      ASSERT_EQ(result.NumAxes(), num_axes);

      // get result splits with `SpliceRowSplits` and get result row-ids with
      // `RowSplitsToRowIds``
      std::vector<Array1<int32_t>> result_splits;
      std::vector<Array1<int32_t>> result_ids;
      for (int32_t axis = 1; axis < num_axes; ++axis) {
        std::vector<Array1<int32_t>> splits_vec(num_shape);
        std::vector<const Array1<int32_t> *> splits_vec_ptr(num_shape);
        for (int32_t n = 0; n != num_shape; ++n) {
          splits_vec[n] = shape_vec[n].RowSplits(axis);
          splits_vec_ptr[n] = &splits_vec[n];
        }
        Array1<int32_t> curr_row_splits =
            SpliceRowSplits(num_shape, splits_vec_ptr.data());
        result_splits.push_back(curr_row_splits);
        Array1<int32_t> curr_row_ids(context, curr_row_splits.Back());
        RowSplitsToRowIds(curr_row_splits, curr_row_ids);
        result_ids.push_back(curr_row_ids);
      }

      // check data
      for (int32_t axis = 1; axis < num_axes; ++axis) {
        CheckArrayData(result.RowSplits(axis), result_splits[axis - 1]);
        CheckArrayData(result.RowIds(axis), result_ids[axis - 1]);
      }
    }
  }
}
TEST(RaggedShapeOpsTest, TestAppend) {
  TestAppend<kCpu>();
  TestAppend<kCuda>();
}

}  // namespace k2
