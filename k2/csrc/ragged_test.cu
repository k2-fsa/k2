/**
 * @brief
 * ragged_test
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey, Haowen Qiu)
 *                      Mobvoi Inc.        (authors: Fangjun Kuang)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include <algorithm>
#include <numeric>
#include <utility>
#include <vector>

#include "k2/csrc/array.h"
#include "k2/csrc/array_ops.h"
#include "k2/csrc/context.h"
#include "k2/csrc/math.h"
#include "k2/csrc/ragged.h"
#include "k2/csrc/ragged_ops.h"
#include "k2/csrc/tensor.h"
#include "k2/csrc/test_utils.h"

namespace k2 {

class RaggedShapeOpsSuiteTest : public ::testing::Test {
 protected:
  RaggedShapeOpsSuiteTest() {
    ContextPtr context = GetCpuContext();
    const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
    const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
    const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
    const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};
    const std::vector<int32_t> row_splits3 = {0,  2,  3,  5,  8, 9,
                                              12, 13, 15, 15, 16};
    const std::vector<int32_t> row_ids3 = {0, 0, 1, 2, 2, 3, 3, 3,
                                           4, 5, 5, 5, 6, 7, 7, 9};
    std::vector<RaggedShapeDim> axes;
    axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits1),
                                     Array1<int32_t>(context, row_ids1),
                                     static_cast<int32_t>(row_ids1.size())});
    axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits2),
                                     Array1<int32_t>(context, row_ids2),
                                     static_cast<int32_t>(row_ids2.size())});
    axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits3),
                                     Array1<int32_t>(context, row_ids3),
                                     static_cast<int32_t>(row_ids3.size())});

    simple_shape_ = RaggedShape(axes, true);

    // random_shape_ is on CPU
    random_shape_ = RandomRaggedShape(true,   // set_row_ids
                                      3,      // min_num_axes
                                      4,      // max_num_axes
                                      0,      // min_num_elements
                                      1000);  // max_num_elements
  }

  RaggedShape simple_shape_;
  RaggedShape random_shape_;
};

TEST(RaggedShapeTest, TestConstructFromString) {
  RaggedShape rs(" [ [ x x ] [x] ]");
  Array1<int32_t> row_splits1(GetCpuContext(), std::vector<int32_t>{0, 2, 3});
  K2_LOG(INFO) << rs.RowSplits(1);
  K2_CHECK(Equal(rs.RowSplits(1), row_splits1));

  RaggedShape rs2(" [ [ [ x x ] ] [[x]] ]");
  K2_LOG(INFO) << "rs2 = " << rs2;

  K2_CHECK_EQ(RaggedShape("[ ]").Dim0(), 0);

  ASSERT_DEATH(RaggedShape(" [ [ x x ] [x] "), "");
  ASSERT_DEATH(RaggedShape(" [ [ x x ] [[x]]] "), "");
  ASSERT_DEATH(RaggedShape(" [ [ x [] x ] "), "");
  ASSERT_DEATH(RaggedShape(" [ x ] "), "");
  ASSERT_DEATH(RaggedShape(" [ x ] [ x ] "), "");
  ASSERT_DEATH(RaggedShape(" [ x | x ] "), "");

  for (int i = 0; i < 5; i++) {
    RaggedShape rs = RandomRaggedShape(true,
                                       2,      // min_num_axes
                                       4,      // max_num_axes
                                       0,      // min_num_elements
                                       1000);  // max_num_elements
    std::ostringstream os;
    os << rs;
    RaggedShape rs2;
    std::istringstream is(os.str());
    K2_LOG(INFO) << "Shape is: " << os.str();
    is >> rs2;
    K2_CHECK(is.good());
    // the reason for the || below is that in "[ ]", the number of
    // axes is ambiguous; we assume 2.
    K2_CHECK(Equal(rs, rs2) || rs.NumElements() == 0);
  }
}

TEST(RaggedTest, TestRaggedFromString) {
  Ragged<int32_t> rs(" [ [ 1 2 ] [3] ]");
  Array1<int32_t> row_splits1(GetCpuContext(), std::vector<int32_t>{0, 2, 3});
  K2_LOG(INFO) << rs.RowSplits(1);
  K2_CHECK(Equal(rs.RowSplits(1), row_splits1));
  K2_CHECK_EQ(rs.values.Back(), 3);
  K2_CHECK_EQ(rs.values[0], 1);

  Ragged<int32_t> rs2(" [ [ [ 0 5 ] ] [[10]] ]");
  K2_LOG(INFO) << "rs2 = " << rs2;

  ASSERT_DEATH(RaggedShape(" [ [ 0 0 ] [0] "), "");
  ASSERT_DEATH(RaggedShape(" [ [ 0 0 ] [[0]]] "), "");
  ASSERT_DEATH(RaggedShape(" [ [ 0 [] 0 ] "), "");
  ASSERT_DEATH(RaggedShape(" [ 0 ] "), "");
  ASSERT_DEATH(RaggedShape(" [ 0 ] [ 0 ] "), "");
  ASSERT_DEATH(RaggedShape(" [ 0 | 0 ] "), "");

  for (int32_t i = 0; i < 5; i++) {
    Ragged<int32_t> r = RandomRagged<int32_t>();
    std::ostringstream os;
    os << r;
    Ragged<int32_t> r2(os.str());
    // the reason for the || below is that in "[ ]", the number of
    // axes is ambiguous; we assume 2.
    K2_CHECK(Equal(r, r2) || r.values.Dim() == 0);
  }
}

template <typename T>
void TestMaxPerSubListTest() {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // empty case
      const std::vector<int32_t> row_splits = {0};
      RaggedShapeDim shape_dim;
      shape_dim.row_splits = Array1<int32_t>(context, row_splits);
      shape_dim.cached_tot_size = 0;
      std::vector<RaggedShapeDim> axes = {shape_dim};
      RaggedShape shape(axes, true);
      Array1<T> values(context, 0);
      Ragged<T> ragged(shape, values);

      int32_t num_rows = ragged.shape.Dim0();
      ASSERT_EQ(num_rows, 0);
      Array1<T> max_values(context, num_rows);
      // just run to check if there's any error
      MaxPerSublist(ragged, 1, &max_values);
      EXPECT_EQ(max_values.Dim(), 0);
    }

    {
      const std::vector<int32_t> row_splits = {0, 2, 2, 5, 6};
      RaggedShapeDim shape_dim;
      shape_dim.row_splits = Array1<int32_t>(context, row_splits);
      shape_dim.cached_tot_size = row_splits.back();
      std::vector<RaggedShapeDim> axes = {shape_dim};
      RaggedShape shape(axes, true);
      const std::vector<T> values_vec = {1, 3, 2, 8, 0, -1};
      Array1<T> values(context, values_vec);
      Ragged<T> ragged(shape, values);

      int32_t num_rows = ragged.shape.Dim0();
      Array1<T> max_values(context, num_rows);
      T default_value = 2;
      MaxPerSublist(ragged, default_value, &max_values);
      // copy memory from GPU/CPU to CPU
      std::vector<T> cpu_data(max_values.Dim());
      auto kind = GetMemoryCopyKind(*max_values.Context(), *cpu);
      MemoryCopy(static_cast<void *>(cpu_data.data()),
                 static_cast<const void *>(max_values.Data()),
                 max_values.Dim() * max_values.ElementSize(), kind, nullptr);
      std::vector<T> expected_data = {3, default_value, 8, default_value};
      EXPECT_EQ(cpu_data, expected_data);
    }

    {
      // test with random large size
      const int32_t min_num_elements = 2000;
      // not random shape is on CPU
      RaggedShape shape =
          RandomRaggedShape(false, 2, 2, min_num_elements, 5000);
      ASSERT_EQ(shape.NumAxes(), 2);
      RaggedShape gpu_shape;
      if (context->GetDeviceType() == kCuda) {
        // copy shape to GPU
        const Array1<T> &row_splits = shape.RowSplits(1);
        RaggedShapeDim shape_dim;
        shape_dim.row_splits = row_splits.To(GetCudaContext());
        shape_dim.cached_tot_size = shape.NumElements();
        std::vector<RaggedShapeDim> axes = {shape_dim};
        gpu_shape = RaggedShape(axes, true);
      }

      int32_t num_elems = shape.NumElements();
      std::vector<T> data(num_elems);
      for (int32_t i = 0; i != 10; ++i) {
        std::iota(data.begin(), data.end(), 0);
        // randomly set data[pos] = num_elems which is
        // greater than any element in data
        int32_t pos = RandInt(0, num_elems - 1);
        data[pos] = num_elems;
        // find the corresponding row
        int32_t num_rows = shape.Dim0();
        const int32_t *row_splits_data = shape.RowSplits(1).Data();
        int32_t row = 0;
        for (int32_t i = 0; i < num_rows; ++i) {
          if (pos >= row_splits_data[i] && pos < row_splits_data[i + 1]) {
            row = i;
            break;
          }
        }

        Array1<T> values(context, data);
        Ragged<T> ragged(context->GetDeviceType() == kCuda ? gpu_shape : shape,
                         values);
        Array1<T> max_values(context, num_rows);
        T default_value = 0;
        MaxPerSublist(ragged, default_value, &max_values);
        EXPECT_EQ(max_values[row], num_elems);
      }
    }
  }
}

TEST(RaggedShapeOpsTest, MaxPerSubListTest) {
  TestMaxPerSubListTest<int32_t>();
}

template <typename T>
void TestMinPerSubListTest() {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // empty case
      std::vector<int32_t> row_splits_vec = {0};
      Array1<T> row_splits(context, row_splits_vec);
      RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
      Array1<T> values(context, 0);
      Ragged<T> ragged(shape, values);

      int32_t num_rows = ragged.shape.Dim0();
      ASSERT_EQ(num_rows, 0);
      Array1<T> min_values(context, num_rows);
      // just run to check if there's any error
      MinPerSublist(ragged, 1, &min_values);
      EXPECT_EQ(min_values.Dim(), 0);
    }

    {
      std::vector<int32_t> row_splits_vec = {0, 2, 2, 5, 6};
      Array1<T> row_splits(context, row_splits_vec);
      RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
      const std::vector<T> values_vec = {1, 3, 3, 8, 4, -1};
      Array1<T> values(context, values_vec);
      Ragged<T> ragged(shape, values);

      int32_t num_rows = ragged.shape.Dim0();
      Array1<T> min_values(context, num_rows);
      T default_value = 2;
      MinPerSublist(ragged, default_value, &min_values);
      // copy memory from GPU/CPU to CPU
      min_values = min_values.To(cpu);
      std::vector<T> cpu_data(min_values.Data(),
                              min_values.Data() + min_values.Dim());
      std::vector<T> expected_data = {1, default_value, default_value, -1};
      EXPECT_EQ(cpu_data, expected_data);
    }

    // May add tests for random large size? (but maybe it's fine to not add as
    // we have tested large cases in MaxPerSubList)
  }
}

TEST(RaggedShapeOpsTest, MinPerSubListTest) {
  TestMinPerSubListTest<int32_t>();
}

template <typename T>
void TestAndOrPerSubListTest() {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // And
      const std::vector<int32_t> row_splits = {0, 2, 2, 5, 6};
      RaggedShapeDim shape_dim;
      shape_dim.row_splits = Array1<int32_t>(context, row_splits);
      shape_dim.cached_tot_size = row_splits.back();
      std::vector<RaggedShapeDim> axes = {shape_dim};
      RaggedShape shape(axes, true);
      const std::vector<T> values_vec = {1, 3, 3, 6, 11, 0};
      Array1<T> values(context, values_vec);
      Ragged<T> ragged(shape, values);

      int32_t num_rows = ragged.shape.Dim0();
      Array1<T> dst(context, num_rows);
      T default_value = -1;
      AndPerSublist(ragged, default_value, &dst);
      // copy memory from GPU/CPU to CPU
      dst = dst.To(cpu);
      std::vector<T> cpu_data(dst.Data(), dst.Data() + dst.Dim());
      std::vector<T> expected_data = {1, -1, 2, 0};
      EXPECT_EQ(cpu_data, expected_data);
    }

    {
      // Or
      const std::vector<int32_t> row_splits = {0, 2, 2, 5, 6};
      RaggedShapeDim shape_dim;
      shape_dim.row_splits = Array1<int32_t>(context, row_splits);
      shape_dim.cached_tot_size = row_splits.back();
      std::vector<RaggedShapeDim> axes = {shape_dim};
      RaggedShape shape(axes, true);
      const std::vector<T> values_vec = {1, 3, 3, 4, 6, 0};
      Array1<T> values(context, values_vec);
      Ragged<T> ragged(shape, values);

      int32_t num_rows = ragged.shape.Dim0();
      Array1<T> dst(context, num_rows);
      T default_value = 0;
      OrPerSublist(ragged, default_value, &dst);
      // copy memory from GPU/CPU to CPU
      dst = dst.To(cpu);
      std::vector<T> cpu_data(dst.Data(), dst.Data() + dst.Dim());
      std::vector<T> expected_data = {3, 0, 7, 0};
      EXPECT_EQ(cpu_data, expected_data);
    }
  }
}

TEST(RaggedShapeOpsTest, AndOrPerSubListTest) {
  TestAndOrPerSubListTest<int32_t>();
}

void TestUnsqueeze(const RaggedShape &input_shape) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    RaggedShape src_shape = input_shape.To(context);
    src_shape.Populate();  // set row_ids
    {
      // axis = 0.
      RaggedShape shape = Unsqueeze(src_shape, 0);
      int32_t dim0 = src_shape.Dim0();
      const std::vector<RaggedShapeDim> &src_axes = src_shape.Axes();
      const std::vector<RaggedShapeDim> &dest_axes = shape.Axes();

      {
        const Array1<int32_t> &row_splits0 = dest_axes[0].row_splits;
        std::vector<int32_t> data = {0, dim0};
        CheckArrayData(row_splits0, data);
      }

      {
        const Array1<int32_t> &row_ids0 = dest_axes[0].row_ids;
        std::vector<int32_t> data(dim0, 0);
        CheckArrayData(row_ids0, data);
      }

      {
        for (auto i = 0; i != src_axes.size(); ++i) {
          CheckArrayData(src_axes[i].row_splits, dest_axes[i + 1].row_splits);
          CheckArrayData(src_axes[i].row_ids, dest_axes[i + 1].row_ids);
        }
      }
    }

    {
      // axis = 1
      int32_t axis = 1;
      RaggedShape shape = Unsqueeze(src_shape, axis);
      int32_t tot_size = shape.TotSize(axis);
      const std::vector<RaggedShapeDim> &src_axes = src_shape.Axes();
      const std::vector<RaggedShapeDim> &dest_axes = shape.Axes();

      {
        for (auto i = 0; i < axis; ++i) {
          CheckArrayData(src_axes[i].row_splits, dest_axes[i].row_splits);
          CheckArrayData(src_axes[i].row_ids, dest_axes[i].row_ids);
        }
      }

      {
        const Array1<int32_t> &row_splits = dest_axes[axis].row_splits;
        std::vector<int32_t> data(tot_size + 1);
        std::iota(data.begin(), data.end(), 0);
        CheckArrayData(row_splits, data);
      }

      {
        const Array1<int32_t> &row_ids = dest_axes[axis].row_ids;
        std::vector<int32_t> data(tot_size);
        std::iota(data.begin(), data.end(), 0);
        CheckArrayData(row_ids, data);
      }

      {
        for (auto i = axis; i < src_axes.size(); ++i) {
          CheckArrayData(src_axes[i].row_splits, dest_axes[i + 1].row_splits);
          CheckArrayData(src_axes[i].row_ids, dest_axes[i + 1].row_ids);
        }
      }
    }
  }
}

TEST_F(RaggedShapeOpsSuiteTest, TestUnsqueeze) {
  TestUnsqueeze(simple_shape_);
  TestUnsqueeze(random_shape_);
}

TEST(RaggedShapeOpsTest, TestUnsqueezeParallel) {
  for (int32_t i = 0; i < 10; i++) {
    ContextPtr c = (i % 2 == 0 ? GetCpuContext() : GetCudaContext());
    int32_t num_shapes = RandInt(0, 10);

    std::vector<RaggedShape *> orig_shapes;
    for (int32_t i = 0; i < num_shapes; i++)
      orig_shapes.push_back(
          new RaggedShape(RandomRaggedShape(false, 2, 5, 0, 1000).To(c)));
    int32_t axis = 0;  // only one supported for now.
    std::vector<RaggedShape> unsqueezed =
        UnsqueezeParallel(num_shapes, orig_shapes.data(), axis);
    for (int32_t i = 0; i < num_shapes; i++) {
      ASSERT_EQ(unsqueezed[i].Validate(), true);
      RaggedShape temp = RemoveAxis(unsqueezed[i], axis);
      ASSERT_EQ(Equal(temp, *(orig_shapes[i])), true);
      delete orig_shapes[i];
    }
  }
}

void TestRemoveAxis(const RaggedShape &input_shape) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    RaggedShape src_shape = input_shape.To(context);
    ASSERT_EQ(src_shape.NumAxes(), 4);
    {
      // axis = 0.
      int32_t axis = 0;
      RaggedShape shape = RemoveAxis(src_shape, axis);
      const std::vector<RaggedShapeDim> &src_axes = src_shape.Axes();
      const std::vector<RaggedShapeDim> &dest_axes = shape.Axes();
      ASSERT_EQ(src_axes.size(), 3);
      ASSERT_EQ(dest_axes.size(), 2);

      {
        for (auto i = 0; i != dest_axes.size(); ++i) {
          CheckArrayData(dest_axes[i].row_splits, src_axes[i + 1].row_splits);
          CheckArrayData(dest_axes[i].row_ids, src_axes[i + 1].row_ids);
        }
      }
    }

    {
      // axis = 1
      int32_t axis = 1;
      RaggedShape shape = RemoveAxis(src_shape, axis);
      const std::vector<RaggedShapeDim> &src_axes = src_shape.Axes();
      const std::vector<RaggedShapeDim> &dest_axes = shape.Axes();
      ASSERT_EQ(src_axes.size(), 3);
      ASSERT_EQ(dest_axes.size(), 2);

      {
        const Array1<int32_t> &row_splits0 = dest_axes[0].row_splits;
        std::vector<int32_t> data = {0, 3, 7, 10};
        CheckArrayData(row_splits0, data);
      }

      {
        const Array1<int32_t> &row_ids0 = dest_axes[0].row_ids;
        std::vector<int32_t> data = {0, 0, 0, 1, 1, 1, 1, 2, 2, 2};
        CheckArrayData(row_ids0, data);
      }

      {
        for (auto i = 1; i != dest_axes.size(); ++i) {
          CheckArrayData(dest_axes[i].row_splits, src_axes[i + 1].row_splits);
          CheckArrayData(dest_axes[i].row_ids, src_axes[i + 1].row_ids);
        }
      }
    }

    {
      // axis = 3
      int32_t axis = 3;  // the last axis
      RaggedShape shape = RemoveAxis(src_shape, axis);
      const std::vector<RaggedShapeDim> &src_axes = src_shape.Axes();
      const std::vector<RaggedShapeDim> &dest_axes = shape.Axes();
      ASSERT_EQ(src_axes.size(), 3);
      ASSERT_EQ(dest_axes.size(), 2);

      {
        for (auto i = 0; i != dest_axes.size(); ++i) {
          CheckArrayData(dest_axes[i].row_splits, src_axes[i].row_splits);
          CheckArrayData(dest_axes[i].row_ids, src_axes[i].row_ids);
        }
      }
    }
  }
}

TEST_F(RaggedShapeOpsSuiteTest, TestRemoveAxis) {
  TestRemoveAxis(simple_shape_);
}

TEST(RaggedShapeOpsTest, TestGetOffsets) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    for (int32_t i = 0; i != 2; ++i) {
      int32_t num_shape = RandInt(10, 100);
      int32_t num_axes = RandInt(2, 4);
      std::vector<RaggedShape> shape_vec(num_shape);
      std::vector<RaggedShape *> shapes(num_shape);
      for (int32_t j = 0; j != num_shape; ++j) {
        shape_vec[j] =
            RandomRaggedShape(false, num_axes, num_axes, 0, 1000).To(context);
        shapes[j] = &shape_vec[j];
      }
      RaggedShape **shapes_ptr = shapes.data();
      Array2<int32_t> offsets = GetOffsets(num_shape, shapes_ptr);
      ASSERT_EQ(offsets.Dim0(), num_axes + 1);
      ASSERT_EQ(offsets.Dim1(), num_shape + 1);
      auto acc = offsets.Accessor();
      for (int32_t axis = 0; axis <= num_axes; ++axis) {
        int32_t sum = 0;
        for (int32_t j = 0; j <= num_shape; ++j) {
          EXPECT_EQ(acc(axis, j), sum);
          if (j < num_shape) {
            sum += (axis == 0 ? 1 : shape_vec[j].TotSize(axis - 1));
          }
        }
      }
    }
  }
}

// returns a random ragged shape where the dims on axis 1 are all the same
// (so: can be transposed).
RaggedShape RandomRaggedShapeToTranspose(ContextPtr c) {
  ContextPtr c_cpu = GetCpuContext();

  RaggedShape random = RandomRaggedShape(false, 2, 4, 0, 5000).To(c);

  int32_t input_dim0 = random.Dim0(), divisor = 1;
  for (int32_t i = 1; i * i <= input_dim0; i++) {
    if (input_dim0 % i == 0 && i > divisor) divisor = i;
  }

  int32_t output_dim0 = divisor, output_dim1 = input_dim0 / divisor;

  Array1<int32_t> row_splits =
      Range<int32_t>(c, output_dim0 + 1, 0, output_dim1);
  int32_t cached_tot_size = input_dim0;

  RaggedShape top_level_shape =
      RaggedShape2(&row_splits, nullptr, cached_tot_size);
  return ComposeRaggedShapes(top_level_shape, random);
}

TEST(RaggedShapeOpsTest, TestTranspose) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      const std::vector<int32_t> row_splits1_vec = {0, 2, 4, 6};
      const std::vector<int32_t> row_splits2_vec = {0, 3, 4, 7, 8, 10, 12};
      Array1<int32_t> row_splits1(context, row_splits1_vec);
      Array1<int32_t> row_splits2(context, row_splits2_vec);
      RaggedShape src_shape =
          RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);
      ASSERT_EQ(src_shape.Dim0(), 3);
      ASSERT_EQ(src_shape.TotSize(1), 6);
      RaggedShape shape = Transpose(src_shape);
      EXPECT_EQ(shape.Dim0(), 2);
      ASSERT_EQ(shape.TotSize(1), 6);
      const std::vector<int32_t> expected_row_splits = {0, 3, 6};
      const std::vector<int32_t> expected_row_ids = {0, 0, 0, 1, 1, 1};
      CheckArrayData(shape.RowSplits(1), expected_row_splits);
      CheckArrayData(shape.RowIds(1), expected_row_ids);
      CheckArrayData(shape.RowSplits(2), {0, 3, 6, 8, 9, 10, 12});
      CheckArrayData(shape.RowIds(2), {0, 0, 0, 1, 1, 1, 2, 2, 3, 4, 5, 5});
    }

    {
      // random case
      for (int32_t j = 0; j != 2; ++j) {
        RaggedShape to_transpose = RandomRaggedShapeToTranspose(context);
        RaggedShape transposed = Transpose(to_transpose);

        if (context->GetDeviceType() != kCpu) {
          to_transpose = to_transpose.To(cpu);
          transposed = transposed.To(cpu);
        }

        for (auto iter = transposed.Iterator(); !iter.Done(); iter.Next()) {
          std::vector<int32_t> index = iter.Value();
          int32_t i = transposed[index];  // Just make sure this doesn't crash,
                                          // dont need the value.
          std::swap(index[0], index[1]);
          i = to_transpose[index];  // don't need the value, just need to make
                                    // sure it's an allowable index.
        }
        for (auto iter = to_transpose.Iterator(); !iter.Done(); iter.Next()) {
          std::vector<int32_t> index = iter.Value();
          std::swap(index[0], index[1]);
          int32_t i = transposed[index];  // don't need the value, just need to
                                          // make sure it's an allowable index.
        }
      }
    }
  }
}

template <typename T>
void TestTransposeRagged() {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      const std::vector<int32_t> row_splits1_vec = {0, 2, 4, 6};
      const std::vector<int32_t> row_splits2_vec = {0, 3, 4, 7, 8, 10, 12};
      Array1<int32_t> row_splits1(context, row_splits1_vec);
      Array1<int32_t> row_splits2(context, row_splits2_vec);
      RaggedShape src_shape =
          RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);
      ASSERT_EQ(src_shape.Dim0(), 3);
      ASSERT_EQ(src_shape.TotSize(1), 6);
      std::vector<T> values = {0, 1, 2, 3, 4, 5, 8, 7, 6, 9, 10, 15};
      ASSERT_EQ(values.size(), src_shape.NumElements());
      Array1<T> values_array(context, values);
      Ragged<T> ragged(src_shape, values_array);
      Ragged<T> ans = Transpose(ragged);
      RaggedShape shape = ans.shape;
      // Check shape
      ASSERT_EQ(shape.Dim0(), 2);
      ASSERT_EQ(shape.TotSize(1), 6);
      const std::vector<int32_t> expected_row_splits = {0, 3, 6};
      const std::vector<int32_t> expected_row_ids = {0, 0, 0, 1, 1, 1};
      CheckArrayData(shape.RowSplits(1), expected_row_splits);
      CheckArrayData(shape.RowIds(1), expected_row_ids);
      CheckArrayData(shape.RowSplits(2), {0, 3, 6, 8, 9, 10, 12});
      CheckArrayData(shape.RowIds(2), {0, 0, 0, 1, 1, 1, 2, 2, 3, 4, 5, 5});
      // Check values
      CheckArrayData(ans.values, {0, 1, 2, 4, 5, 8, 6, 9, 3, 7, 10, 15});
    }

    {
      // random case
      for (int32_t j = 0; j != 2; ++j) {
        RaggedShape to_transpose = RandomRaggedShapeToTranspose(context);
        int32_t num_elems = to_transpose.NumElements();
        Array1<T> src_values =
            RandUniformArray1<T>(context, num_elems, 0, 10000);
        Ragged<T> src(to_transpose, src_values);
        Ragged<T> ans = Transpose(src);
        if (context->GetDeviceType() == kCuda) {
          src = src.To(cpu);
          ans = ans.To(cpu);
          to_transpose = to_transpose.To(cpu);
        }
        RaggedShape transposed = ans.shape;

        for (auto iter = transposed.Iterator(); !iter.Done(); iter.Next()) {
          std::vector<int32_t> index = iter.Value();
          T value = ans[index];
          std::swap(index[0], index[1]);
          EXPECT_EQ(value, src[index]);
        }
        for (auto iter = to_transpose.Iterator(); !iter.Done(); iter.Next()) {
          std::vector<int32_t> index = iter.Value();
          T value = src[index];
          std::swap(index[0], index[1]);
          EXPECT_EQ(value, ans[index]);
        }
      }
    }
  }
}
TEST(RaggedTest, TestTransposeRagged) {
  TestTransposeRagged<int32_t>();
  TestTransposeRagged<double>();
}

TEST(RaggedShapeOpsTest, TestRowSplitsPtr) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    RaggedShape shape = RandomRaggedShape().To(context);
    ASSERT_GE(shape.NumAxes(), 2);
    Array1<int32_t *> ptrs = GetRowSplitsPtr(shape);
    ASSERT_EQ(ptrs.Dim(), shape.NumAxes() - 1);
    // as num_axes is not so big, access (may copy memory) it in a loop is fine.
    for (int32_t i = 0; i != ptrs.Dim(); ++i) {
      EXPECT_EQ(ptrs[i], shape.RowSplits(i + 1).Data());
    }
  }
}

void TestRaggedShape2(const RaggedShape &shape) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    RaggedShape src_shape = shape.To(context);
    src_shape.Populate();
    ASSERT_GE(src_shape.NumAxes(), 2);
    Array1<int32_t> row_splits = src_shape.RowSplits(1);
    Array1<int32_t> row_ids = src_shape.RowIds(1);
    int32_t cached_tot_size = src_shape.TotSize(1);

    {
      // both row_splits and row_ids are non-null
      RaggedShape result = RaggedShape2(&row_splits, &row_ids, cached_tot_size);
      CheckArrayData(result.RowSplits(1), row_splits);
      CheckArrayData(result.RowIds(1), row_ids);
      EXPECT_EQ(result.TotSize(1), cached_tot_size);
    }
    {
      // both row_splits and row_ids are non-null, cached_tot_size = -1
      RaggedShape result = RaggedShape2(&row_splits, &row_ids, -1);
      CheckArrayData(result.RowSplits(1), row_splits);
      CheckArrayData(result.RowIds(1), row_ids);
      EXPECT_EQ(result.TotSize(1), cached_tot_size);
    }
    {
      // row_ids is null
      RaggedShape result = RaggedShape2(&row_splits, nullptr, cached_tot_size);
      CheckArrayData(result.RowSplits(1), row_splits);
      CheckArrayData(result.RowIds(1), row_ids);
      EXPECT_EQ(result.TotSize(1), cached_tot_size);
    }
    {
      // row_ids is null, cached_tot_size = -1
      RaggedShape result = RaggedShape2(&row_splits, nullptr, -1);
      CheckArrayData(result.RowSplits(1), row_splits);
      CheckArrayData(result.RowIds(1), row_ids);
      EXPECT_EQ(result.TotSize(1), cached_tot_size);
    }

    // note if row_splits == null, then we suppose there's no empty rows after
    // the last row-id in row_ids
    if (row_splits.Dim() == (row_ids.Dim() == 0 ? 1 : row_ids.Back() + 2)) {
      {
        // row_splits is null
        RaggedShape result = RaggedShape2(nullptr, &row_ids, cached_tot_size);
        CheckArrayData(result.RowSplits(1), row_splits);
        CheckArrayData(result.RowIds(1), row_ids);
        EXPECT_EQ(result.TotSize(1), cached_tot_size);
      }
      {
        // row_splits is null, cached_tot_size = -1
        RaggedShape result = RaggedShape2(nullptr, &row_ids, -1);
        CheckArrayData(result.RowSplits(1), row_splits);
        CheckArrayData(result.RowIds(1), row_ids);
        EXPECT_EQ(result.TotSize(1), cached_tot_size);
      }
    }
  }
}
TEST_F(RaggedShapeOpsSuiteTest, TestRaggedShape2) {
  TestRaggedShape2(simple_shape_);
  TestRaggedShape2(random_shape_);
}

void TestRaggedShape3(const RaggedShape &shape) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    RaggedShape src_shape = shape.To(context);
    src_shape.Populate();
    ASSERT_GE(src_shape.NumAxes(), 3);
    Array1<int32_t> row_splits1 = src_shape.RowSplits(1);
    Array1<int32_t> row_ids1 = src_shape.RowIds(1);
    int32_t cached_tot_size1 = src_shape.TotSize(1);
    Array1<int32_t> row_splits2 = src_shape.RowSplits(2);
    Array1<int32_t> row_ids2 = src_shape.RowIds(2);
    int32_t cached_tot_size2 = src_shape.TotSize(2);

    {
      // both row_splits and row_ids are non-null
      RaggedShape result =
          RaggedShape3(&row_splits1, &row_ids1, cached_tot_size1, &row_splits2,
                       &row_ids2, cached_tot_size2);
      CheckArrayData(result.RowSplits(1), row_splits1);
      CheckArrayData(result.RowIds(1), row_ids1);
      EXPECT_EQ(result.TotSize(1), cached_tot_size1);
      CheckArrayData(result.RowSplits(2), row_splits2);
      CheckArrayData(result.RowIds(2), row_ids2);
      EXPECT_EQ(result.TotSize(2), cached_tot_size2);
    }
    {
      // row_ids is non-null, cached_tot_size = -1
      RaggedShape result =
          RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);
      CheckArrayData(result.RowSplits(1), row_splits1);
      CheckArrayData(result.RowIds(1), row_ids1);
      EXPECT_EQ(result.TotSize(1), cached_tot_size1);
      CheckArrayData(result.RowSplits(2), row_splits2);
      CheckArrayData(result.RowIds(2), row_ids2);
      EXPECT_EQ(result.TotSize(2), cached_tot_size2);
    }

    // note if row_splits == null, then we suppose there's no empty rows after
    // the last row-id in row_ids
    bool valid1 =
        (row_splits1.Dim() == (row_ids1.Dim() == 0 ? 1 : row_ids1.Back() + 2));
    bool valid2 =
        (row_splits2.Dim() == (row_ids2.Dim() == 0 ? 1 : row_ids2.Back() + 2));
    if (valid1 && valid2) {
      RaggedShape result =
          RaggedShape3(nullptr, &row_ids1, -1, nullptr, &row_ids2, -1);
      CheckArrayData(result.RowSplits(1), row_splits1);
      CheckArrayData(result.RowIds(1), row_ids1);
      EXPECT_EQ(result.TotSize(1), cached_tot_size1);
      CheckArrayData(result.RowSplits(2), row_splits2);
      CheckArrayData(result.RowIds(2), row_ids2);
      EXPECT_EQ(result.TotSize(2), cached_tot_size2);
    }
    // TODO(haowen): add more cases for other branches
  }
}
TEST_F(RaggedShapeOpsSuiteTest, TestRaggedShape3) {
  TestRaggedShape3(simple_shape_);
  TestRaggedShape3(random_shape_);
}

void TestComposeShape(const RaggedShape &shape) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    RaggedShape src_shape = shape.To(context);
    ASSERT_GE(src_shape.NumAxes(), 3);
    Array1<int32_t> row_splits1 = src_shape.RowSplits(1);
    Array1<int32_t> row_ids1 = src_shape.RowIds(1);
    Array1<int32_t> row_splits2 = src_shape.RowSplits(2);
    Array1<int32_t> row_ids2 = src_shape.RowIds(2);

    RaggedShape shape1 = RaggedShape2(&row_splits1, nullptr, -1);
    RaggedShape shape2 = RaggedShape2(&row_splits2, nullptr, -1);

    RaggedShape result = ComposeRaggedShapes(shape1, shape2);

    ASSERT_EQ(result.NumAxes(), 3);

    CheckArrayData(result.RowSplits(1), row_splits1);
    CheckArrayData(result.RowIds(1), row_ids1);
    CheckArrayData(result.RowSplits(2), row_splits2);
    CheckArrayData(result.RowIds(2), row_ids2);
  }
}
TEST_F(RaggedShapeOpsSuiteTest, TestComposeShape) {
  TestComposeShape(simple_shape_);
  TestComposeShape(random_shape_);
}

void TestShapeFromTotSize(const RaggedShape &shape) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    RaggedShape src_shape = shape.To(context);
    ASSERT_GE(src_shape.NumAxes(), 2);

    int32_t num_axes = src_shape.NumAxes();
    std::vector<int32_t> tot_sizes(num_axes);
    for (int32_t i = 0; i != num_axes; ++i) {
      tot_sizes[i] = src_shape.TotSize(i);
    }

    RaggedShape result =
        RaggedShapeFromTotSizes(context, num_axes, tot_sizes.data());

    ASSERT_EQ(result.NumAxes(), num_axes);
    for (int32_t i = 0; i < num_axes; ++i) {
      EXPECT_EQ(result.TotSize(i), src_shape.TotSize(i));
      if (i > 0) {
        EXPECT_EQ(result.RowSplits(i).Dim(), src_shape.RowSplits(i).Dim());
        EXPECT_EQ(result.RowIds(i).Dim(), src_shape.RowIds(i).Dim());
      }
    }
  }
}
TEST_F(RaggedShapeOpsSuiteTest, TestShapeFromTotSize) {
  TestShapeFromTotSize(simple_shape_);
  TestShapeFromTotSize(random_shape_);
}

template <typename T>
void TestRagged() {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // constructed with row_splits and row_ids
      // RaggedTensor4 t = [
      //  [ [[ 1, 2], [4]],  [[3, 0]] ],
      //  [ [[7, 8, 9]], [[6], [3, 5, 7]], [[2]] ],
      //  [ [[3, 4], [], [8]] ]
      // ]
      const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
      const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
      const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
      const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};
      const std::vector<int32_t> row_splits3 = {0,  2,  3,  5,  8, 9,
                                                12, 13, 15, 15, 16};
      const std::vector<int32_t> row_ids3 = {0, 0, 1, 2, 2, 3, 3, 3,
                                             4, 5, 5, 5, 6, 7, 7, 9};
      const std::vector<T> values_vec = {1, 2, 4, 3, 0, 7, 8, 9,
                                         6, 3, 5, 7, 2, 3, 4, 8};
      std::vector<RaggedShapeDim> axes;
      axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits1),
                                       Array1<int32_t>(context, row_ids1),
                                       static_cast<int32_t>(row_ids1.size())});
      axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits2),
                                       Array1<int32_t>(context, row_ids2),
                                       static_cast<int32_t>(row_ids2.size())});
      axes.emplace_back(RaggedShapeDim{Array1<int32_t>(context, row_splits3),
                                       Array1<int32_t>(context, row_ids3),
                                       static_cast<int32_t>(row_ids3.size())});

      RaggedShape shape(axes, true);
      Array1<T> values(context, values_vec);
      Ragged<T> ragged(shape, values);

      // test Index(axis, i)
      {
        // values: [[[ 1, 2], [4]], [[3, 0]]]
        Ragged<T> sub_raggged = ragged.Index(0, 0);
        RaggedShape &sub_shape = sub_raggged.shape;
        EXPECT_EQ(sub_shape.NumAxes(), 3);
        const std::vector<std::vector<int32_t>> sub_row_splits_vec = {
            {0, 2, 3}, {0, 2, 3, 5}};
        CheckRowSplits(sub_shape, sub_row_splits_vec);
        const Array1<T> &sub_values = sub_raggged.values;
        const std::vector<T> sub_values_vec = {1, 2, 4, 3, 0};
        CheckArrayData<T>(sub_values, sub_values_vec);
      }
      {
        // values: [[[7, 8, 9]], [[6], [3, 5, 7]], [[2]]]
        Ragged<T> sub_raggged = ragged.Index(0, 1);
        RaggedShape &sub_shape = sub_raggged.shape;
        EXPECT_EQ(sub_shape.NumAxes(), 3);
        const std::vector<std::vector<int32_t>> sub_row_splits_vec = {
            {0, 1, 3, 4}, {0, 3, 4, 7, 8}};
        CheckRowSplits(sub_shape, sub_row_splits_vec);
        const Array1<T> &sub_values = sub_raggged.values;
        const std::vector<T> sub_values_vec = {7, 8, 9, 6, 3, 5, 7, 2};
        CheckArrayData<T>(sub_values, sub_values_vec);
      }
      {
        // values: [[[3, 4], [], [8]]]
        Ragged<T> sub_raggged = ragged.Index(0, 2);
        RaggedShape &sub_shape = sub_raggged.shape;
        EXPECT_EQ(sub_shape.NumAxes(), 3);
        const std::vector<std::vector<int32_t>> sub_row_splits_vec = {
            {0, 3}, {0, 2, 2, 3}};
        CheckRowSplits(sub_shape, sub_row_splits_vec);
        const Array1<T> &sub_values = sub_raggged.values;
        const std::vector<T> sub_values_vec = {3, 4, 8};
        CheckArrayData<T>(sub_values, sub_values_vec);
      }

      // test operator[](const std::vector<int32_t> &indexes)
      if (context->GetDeviceType() == kCpu) {
        {
          std::vector<int32_t> indexes = {0, 0, 0, 0};
          EXPECT_EQ(ragged.shape[indexes], 0);
          EXPECT_EQ(ragged[indexes], 1);
        }
        {
          std::vector<int32_t> indexes = {0, 1, 0, 0};
          EXPECT_EQ(ragged.shape[indexes], 3);
          EXPECT_EQ(ragged[indexes], 3);
        }
        {
          std::vector<int32_t> indexes = {1, 0, 0, 1};
          EXPECT_EQ(ragged.shape[indexes], 6);
          EXPECT_EQ(ragged[indexes], 8);
        }
        {
          std::vector<int32_t> indexes = {1, 1, 1, 0};
          EXPECT_EQ(ragged.shape[indexes], 9);
          EXPECT_EQ(ragged[indexes], 3);
        }
        {
          std::vector<int32_t> indexes = {2, 0, 0, 1};
          EXPECT_EQ(ragged.shape[indexes], 14);
          EXPECT_EQ(ragged[indexes], 4);
        }
        {
          std::vector<int32_t> indexes = {2, 0, 2, 0};
          EXPECT_EQ(ragged.shape[indexes], 15);
          EXPECT_EQ(ragged[indexes], 8);
        }
      }

      const std::vector<std::vector<int32_t>> row_splits_vec = {
          row_splits1, row_splits2, row_splits3};
      // test To(ctx)
      {
        // to GPU
        Ragged<T> other = ragged.To(GetCudaContext());
        CheckRowSplits(other.shape, row_splits_vec);
        CheckArrayData<T>(other.values, values_vec);
      }
      {
        // to CPU
        Ragged<T> other = ragged.To(GetCpuContext());
        CheckRowSplits(other.shape, row_splits_vec);
        CheckArrayData<T>(other.values, values_vec);
      }
    }
  }
}

template <typename T, typename OP = LessThan<T>>
static void CpuSortSublists(const Array1<int32_t> &row_splits, Array1<T> *src) {
  K2_CHECK(src->Context()->GetDeviceType() == kCpu);
  T *p = src->Data();
  OP comp = OP();
  for (int32_t i = 0; i < row_splits.Dim() - 1; ++i) {
    int32_t cur = row_splits[i];
    int32_t next = row_splits[i + 1];
    std::sort(p + cur, p + next, comp);
  }
}

template <typename T, typename OP = LessThan<T>>
static void TestSortSublists() {
  auto cpu_context = GetCpuContext();
  auto cuda_context = GetCudaContext();

  RaggedShape shape = RandomRaggedShape(false,  // set_row_ids
                                        2,      // min_num_axes
                                        4,      // max_num_axes
                                        1,      // min_num_elements
                                        2000);  // max_num_elements

  Array1<T> values =
      RandUniformArray1<T>(shape.Context(), shape.NumElements(), -2000, 2000);
  Ragged<T> ragged(shape, values);
  ragged = ragged.To(cuda_context);
  values = values.To(cpu_context);  // to be sorted by cpu

  Array1<T> unsorted = values.Clone();

  Array1<int32_t> order(ragged.Context(), ragged.values.Dim());
  SortSublists<T, OP>(&ragged, &order);

  Array1<int32_t> &segment = ragged.shape.RowSplits(ragged.NumAxes() - 1);
  CpuSortSublists<T, OP>(segment, &values);

  int32_t n = order.Dim();
  for (int i = 0; i != n; ++i) {
    EXPECT_EQ(values[i], ragged.values[i]);
    EXPECT_EQ(ragged.values[i], unsorted[order[i]]);
  }
}

TEST(RaggedTest, Ragged) {
  TestRagged<int32_t>();
  TestRagged<double>();

  TestSortSublists<int32_t>();
  TestSortSublists<double>();
}

TEST(RaggedShapeOpsTest, TestAppend) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // simple case
      std::vector<RaggedShape> shapes(2);
      std::vector<RaggedShape *> shapes_ptr(2);
      std::vector<std::vector<Array1<int32_t>>> row_splits_vec(2);
      {
        const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
        const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
        const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
        const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};
        Array1<int32_t> splits1(context, row_splits1);
        Array1<int32_t> ids1(context, row_ids1);
        Array1<int32_t> splits2(context, row_splits2);
        Array1<int32_t> ids2(context, row_ids2);
        row_splits_vec[0].push_back(splits1);
        row_splits_vec[1].push_back(splits2);
        shapes[0] = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2, &ids2,
                                 ids2.Dim());
        shapes_ptr[0] = &shapes[0];
      }
      {
        const std::vector<int32_t> row_splits1 = {0, 1, 3, 4};
        const std::vector<int32_t> row_ids1 = {0, 1, 1, 2};
        const std::vector<int32_t> row_splits2 = {0, 3, 4, 5, 7};
        const std::vector<int32_t> row_ids2 = {0, 0, 0, 1, 2, 3, 3};
        Array1<int32_t> splits1(context, row_splits1);
        Array1<int32_t> ids1(context, row_ids1);
        Array1<int32_t> splits2(context, row_splits2);
        Array1<int32_t> ids2(context, row_ids2);
        row_splits_vec[0].push_back(splits1);
        row_splits_vec[1].push_back(splits2);
        RaggedShape shape = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2,
                                         &ids2, ids2.Dim());
        shapes[1] = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2, &ids2,
                                 ids2.Dim());
        shapes_ptr[1] = &shapes[1];
      }

      {
        // axis == 1
        RaggedShape result = Append(1, 2, shapes_ptr.data());
        std::vector<std::vector<int32_t>> expected_row_splits = {
            {0, 3, 8, 10}, {0, 2, 3, 6, 7, 9, 10, 11, 12, 15, 17}};
        std::vector<std::vector<int32_t>> expected_row_ids = {
            {0, 0, 0, 1, 1, 1, 1, 1, 2, 2},
            {0, 0, 1, 2, 2, 2, 3, 4, 4, 5, 6, 7, 8, 8, 8, 9, 9}};
        for (int32_t i = 0; i < 2; ++i) {
          CheckArrayData(result.RowSplits(i + 1), expected_row_splits[i]);
          CheckArrayData(result.RowIds(i + 1), expected_row_ids[i]);
        }
      }

      {
        // axis == 0
        RaggedShape result = Append(0, 2, shapes_ptr.data());

        // get result splits with `SpliceRowSplits` and get result row-ids with
        // `RowSplitsToRowIds``
        std::vector<Array1<int32_t>> result_splits;
        std::vector<Array1<int32_t>> result_ids;
        for (auto i = 0; i < 2; ++i) {
          std::vector<const Array1<int32_t> *> splits_ptr = {
              &row_splits_vec[i][0], &row_splits_vec[i][1]};
          Array1<int32_t> curr_row_splits =
              SpliceRowSplits(2, splits_ptr.data());
          result_splits.push_back(curr_row_splits);
          Array1<int32_t> curr_row_ids(context, curr_row_splits.Back());
          RowSplitsToRowIds(curr_row_splits, &curr_row_ids);
          result_ids.push_back(curr_row_ids);
        }
        for (int32_t i = 0; i < 2; ++i) {
          CheckArrayData(result.RowSplits(i + 1), result_splits[i]);
          CheckArrayData(result.RowIds(i + 1), result_ids[i]);
        }
      }
    }

    {
      // test with random large size
      for (int32_t i = 0; i < 2; ++i) {
        int32_t num_shape = RandInt(2, 100);
        int32_t num_axes = RandInt(2, 4);
        std::vector<RaggedShape> shape_vec(num_shape);
        std::vector<RaggedShape *> shapes(num_shape);
        for (int32_t j = 0; j != num_shape; ++j) {
          shape_vec[j] =
              RandomRaggedShape(true, num_axes, num_axes, 0, 1000).To(context);
          shapes[j] = &shape_vec[j];
        }
        // only test case axis == 0, test axis==1 with simple case is good
        // enough as it just calls Stack
        RaggedShape result = Append(0, num_shape, shapes.data());
        ASSERT_EQ(result.NumAxes(), num_axes);

        // get result splits with `SpliceRowSplits` and get result row-ids with
        // `RowSplitsToRowIds``
        std::vector<Array1<int32_t>> result_splits;
        std::vector<Array1<int32_t>> result_ids;
        for (int32_t axis = 1; axis < num_axes; ++axis) {
          std::vector<Array1<int32_t>> splits_vec(num_shape);
          std::vector<const Array1<int32_t> *> splits_vec_ptr(num_shape);
          for (int32_t n = 0; n != num_shape; ++n) {
            splits_vec[n] = shape_vec[n].RowSplits(axis);
            splits_vec_ptr[n] = &splits_vec[n];
          }
          Array1<int32_t> curr_row_splits =
              SpliceRowSplits(num_shape, splits_vec_ptr.data());
          result_splits.push_back(curr_row_splits);
          Array1<int32_t> curr_row_ids(context, curr_row_splits.Back());
          RowSplitsToRowIds(curr_row_splits, &curr_row_ids);
          result_ids.push_back(curr_row_ids);
        }

        // check data
        for (int32_t axis = 1; axis < num_axes; ++axis) {
          CheckArrayData(result.RowSplits(axis), result_splits[axis - 1]);
          CheckArrayData(result.RowIds(axis), result_ids[axis - 1]);
        }
      }
    }
  }
}

template <typename T>
void TestAppendRagged() {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    // TODO(haowen): remove duplicate code in TestAppend above.
    // test with simple case could be good enough, as we have tested
    // Append(RaggedShape&) already.
    std::vector<Ragged<T>> ragged_vec(2);
    std::vector<Ragged<T> *> ragged(2);
    std::vector<std::vector<Array1<int32_t>>> row_splits_vec(2);
    {
      const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
      const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
      const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
      const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};
      const std::vector<T> values_vec = {1, 2, 5, 7, 9, 10, 12, 14, 15, 18};
      Array1<int32_t> splits1(context, row_splits1);
      Array1<int32_t> ids1(context, row_ids1);
      Array1<int32_t> splits2(context, row_splits2);
      Array1<int32_t> ids2(context, row_ids2);
      RaggedShape shape = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2,
                                       &ids2, ids2.Dim());
      Array1<T> values(context, values_vec);
      ragged_vec[0] = Ragged<T>(shape, values);
      ragged[0] = &ragged_vec[0];
    }

    {
      const std::vector<int32_t> row_splits1 = {0, 1, 3, 4};
      const std::vector<int32_t> row_ids1 = {0, 1, 1, 2};
      const std::vector<int32_t> row_splits2 = {0, 3, 4, 5, 7};
      const std::vector<int32_t> row_ids2 = {0, 0, 0, 1, 2, 3, 3};
      const std::vector<T> values_vec = {20, 21, 23, 28, 30, 32, 35};
      Array1<int32_t> splits1(context, row_splits1);
      Array1<int32_t> ids1(context, row_ids1);
      Array1<int32_t> splits2(context, row_splits2);
      Array1<int32_t> ids2(context, row_ids2);
      RaggedShape shape = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2,
                                       &ids2, ids2.Dim());
      Array1<T> values(context, values_vec);
      ragged_vec[1] = Ragged<T>(shape, values);
      ragged[1] = &ragged_vec[1];
    }

    {
      // axis == 0
      Ragged<T> result = Append(0, 2, ragged.data());
      std::vector<std::vector<int32_t>> expected_row_splits = {
          {0, 2, 5, 6, 7, 9, 10}, {0, 2, 3, 4, 6, 7, 10, 13, 14, 15, 17}};
      std::vector<std::vector<int32_t>> expected_row_ids = {
          {0, 0, 1, 1, 1, 2, 3, 4, 4, 5},
          {0, 0, 1, 2, 3, 3, 4, 5, 5, 5, 6, 6, 6, 7, 8, 9, 9}};
      for (int32_t i = 0; i < 2; ++i) {
        CheckArrayData(result.RowSplits(i + 1), expected_row_splits[i]);
        CheckArrayData(result.RowIds(i + 1), expected_row_ids[i]);
      }
      std::vector<T> expected_data = {1,  2,  5,  7,  9,  10, 12, 14, 15,
                                      18, 20, 21, 23, 28, 30, 32, 35};
      CheckArrayData(result.values, expected_data);
    }

    {
      // axis == 1
      Ragged<T> result = Append(1, 2, ragged.data());
      std::vector<std::vector<int32_t>> expected_row_splits = {
          {0, 3, 8, 10}, {0, 2, 3, 6, 7, 9, 10, 11, 12, 15, 17}};
      std::vector<std::vector<int32_t>> expected_row_ids = {
          {0, 0, 0, 1, 1, 1, 1, 1, 2, 2},
          {0, 0, 1, 2, 2, 2, 3, 4, 4, 5, 6, 7, 8, 8, 8, 9, 9}};
      for (int32_t i = 0; i < 2; ++i) {
        CheckArrayData(result.RowSplits(i + 1), expected_row_splits[i]);
        CheckArrayData(result.RowIds(i + 1), expected_row_ids[i]);
      }
      std::vector<T> expected_data = {1,  2,  5,  20, 21, 23, 7,  9, 10,
                                      12, 28, 30, 14, 15, 18, 32, 35};
      CheckArrayData(result.values, expected_data);
    }
  }
}
TEST(RaggedTest, TestAppendRagged) {
  TestAppendRagged<int32_t>();
  TestAppendRagged<double>();
}

void CheckResultOfIndex(const ContextPtr &context, RaggedShape shape,
                        Array1<int32_t> new2old, RaggedShape result) {
  K2_CHECK(context->IsCompatible(*shape.Context()));
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  int32_t num_axes = shape.NumAxes();
  int32_t src_dim0 = shape.Dim0(), result_dim0 = result.Dim0();
  if (result_dim0 == 0) {
    std::vector<int32_t> empty_row_splits = {0};
    for (int32_t i = 0; i < num_axes - 1; ++i) {
      CheckArrayData(result.RowSplits(i + 1), empty_row_splits);
      EXPECT_EQ(result.RowIds(i + 1).Dim(), 0);
    }
    return;
  }
  Array2<int32_t> old_offsets(context, num_axes, src_dim0 + 1);
  auto old_offsets_acc = old_offsets.Accessor();
  Array1<int32_t *> row_splits_ptrs = GetRowSplitsPtr(shape);
  int32_t **row_splits_ptrs_data = row_splits_ptrs.Data();
  // Set old_offsets
  auto lambda_get_old_offsets = [=] __host__ __device__(int32_t i) {
    // 0 <= i <= dim0
    int32_t cur_offset = i;
    for (int32_t axis = 0; axis < num_axes; axis++) {
      old_offsets_acc(axis, i) = cur_offset;
      if (axis + 1 == num_axes) return;
      cur_offset = row_splits_ptrs_data[axis][cur_offset];
    }
  };
  Eval(context, src_dim0 + 1, lambda_get_old_offsets);
  old_offsets = old_offsets.To(cpu);
  auto cpu_offsets_acc = old_offsets.Accessor();
  shape = shape.To(cpu);
  new2old = new2old.To(cpu);
  // get result splits with `SpliceRowSplits` and get result row-ids with
  // `RowSplitsToRowIds``
  std::vector<Array1<int32_t>> result_splits;
  std::vector<Array1<int32_t>> result_ids;
  for (auto axis = 0; axis < num_axes - 1; ++axis) {
    Array1<int32_t> curr_row_splits = shape.RowSplits(axis + 1);
    std::vector<Array1<int32_t>> splits_vec(result_dim0);
    std::vector<const Array1<int32_t> *> splits_vec_ptr(result_dim0);
    for (int32_t m = 0; m != result_dim0; ++m) {
      int32_t old_idx = new2old[m];
      int32_t start = cpu_offsets_acc(axis, old_idx);
      int32_t end = cpu_offsets_acc(axis, old_idx + 1);
      Array1<int32_t> sub_list = curr_row_splits.Range(start, end - start + 1);
      Array1<int32_t> copy_sub_list(cpu, sub_list.Dim());
      copy_sub_list.CopyFrom(sub_list);
      int32_t *data = copy_sub_list.Data();
      int32_t init = data[0];
      for (int32_t n = 0; n != copy_sub_list.Dim(); ++n) {
        data[n] -= init;
      }
      splits_vec[m] = copy_sub_list;
      splits_vec_ptr[m] = &splits_vec[m];
    }
    Array1<int32_t> result_row_splits =
        SpliceRowSplits(result_dim0, splits_vec_ptr.data());
    result_splits.push_back(result_row_splits);
    Array1<int32_t> result_row_ids(cpu, result_row_splits.Back());
    RowSplitsToRowIds(result_row_splits, &result_row_ids);
    result_ids.push_back(result_row_ids);
  }
  for (int32_t i = 0; i < num_axes - 1; ++i) {
    CheckArrayData(result.RowSplits(i + 1), result_splits[i]);
    CheckArrayData(result.RowIds(i + 1), result_ids[i]);
  }
}

TEST(RaggedShapeOpsTest, TestIndex) {
  for (int i = 0; i < 5; i++) {
    ContextPtr cpu = GetCpuContext();  // will be used to copy data
    for (auto &context : {GetCpuContext(), GetCudaContext()}) {
      {
        // simple case
        const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
        const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
        const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
        const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};

        Array1<int32_t> splits1(context, row_splits1);
        Array1<int32_t> ids1(context, row_ids1);
        Array1<int32_t> splits2(context, row_splits2);
        Array1<int32_t> ids2(context, row_ids2);
        RaggedShape shape = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2,
                                         &ids2, ids2.Dim());

        std::vector<int32_t> new2old_vec = {2, 1};
        Array1<int32_t> new2old(context, new2old_vec);
        Array1<int32_t> value_indexes_out;
        RaggedShape result = Index(shape, new2old, &value_indexes_out);
        // fsa 2, state_idx01 {5}, arc_idx012 {7, 8, 9}
        // fsa 1, state_idx01 {2, 3, 4}, arc_idx012 {{3},{4, 5}, {6}}
        CheckArrayData(value_indexes_out,
                       std::vector<int32_t>{7, 8, 9, 3, 4, 5, 6});
        CheckResultOfIndex(context, shape, new2old, result);
      }

      {
        // test with random large size
        for (int32_t i = 0; i < 2; ++i) {
          int32_t num_axes = RandInt(2, 4);
          RaggedShape shape =
              RandomRaggedShape(true, num_axes, num_axes, 0, 1000).To(context);
          int32_t dim0 = shape.Dim0(), result_dim0 = RandInt(0, 10);
          if (dim0 == 0) result_dim0 = 0;
          std::vector<int32_t> new2old_vec(result_dim0);
          for (int i = 0; i < result_dim0; i++)
            new2old_vec[i] = RandInt(0, dim0 - 1);
          Array1<int32_t> new2old(context, new2old_vec);
          Array1<int32_t> value_indexes;
          RaggedShape result = Index(shape, new2old, &value_indexes);
          CheckResultOfIndex(context, shape, new2old, result);
          K2_LOG(INFO) << "Value_indexes = " << value_indexes;
        }
      }
    }
  }
}

TEST(GetTransposeReordering, NoDuplicates) {
  //       col0  col1  col2  col3  col4  col5
  // row0                           a0    b1
  // row1   c2    d3                      e4
  // row2                     f5
  // row3   g6          h7          i8
  // row4                                 j9
  // row5         k10               l11
  std::vector<int32_t> col_indexes{4, 5, 0, 1, 5, 3, 0, 2, 4, 5, 1, 4};
  std::vector<int32_t> _row_splits{0, 2, 5, 6, 9, 10, 12};
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    Array1<int32_t> row_splits(context, _row_splits);
    RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
    Array1<int32_t> values(context, col_indexes);

    Ragged<int32_t> ragged(shape, values);
    Array1<int32_t> order = GetTransposeReordering(ragged, 6);
    CheckArrayData(order, {2, 6, 3, 10, 7, 5, 0, 8, 11, 1, 4, 9});
    EXPECT_TRUE(context->IsCompatible(*order.Context()));
  }
}

TEST(GetTransposeReordering, ThreeAxesEmptyCase) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    Ragged<int32_t> ragged("[ [ [ ] ] ]");
    ragged = ragged.To(context);
    Array1<int32_t> order = GetTransposeReordering(ragged, 0);
  }
}

TEST(GetTransposeReordering, NoDuplicatesThreeAxes) {
  //       col0  col1  col2  col3  col4  col5
  // row0         a0          b1
  // row1   c2          d3
  // row2         e4
  // row3   f5    g6          h7
  // row4                                  i8
  // row5                            j9    k10
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    Array1<int32_t> col_indexes(
        context, std::vector<int32_t>{1, 3, 0, 2, 1, 0, 1, 3, 5, 4, 5});
    Array1<int32_t> row_splits1(context, std::vector<int32_t>{0, 4, 6});
    Array1<int32_t> row_splits2(context,
                                std::vector<int32_t>{0, 2, 4, 5, 8, 9, 11});
    RaggedShape shape =
        RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);
    Ragged<int32_t> ragged(shape, col_indexes);
    Array1<int32_t> order = GetTransposeReordering(ragged, 6);
    CheckArrayData(order, {2, 5, 0, 4, 6, 3, 1, 7, 9, 8, 10});
    EXPECT_TRUE(context->IsCompatible(*order.Context()));
  }
}

TEST(GetTransposeReordering, WithDuplicates) {
  //       col0   col1   col2    col3      col4      col5
  // row0         a0,a1         b2,b3,b4
  // row1  c5,c6          d7
  // row2         e8
  // row3   f9   g10,g11         h12
  // row4                                i13,i14,i15
  // row5                        j16                  k17
  std::vector<int32_t> col_indexes{1, 1, 3, 3, 3, 0, 0, 2, 1,
                                   0, 1, 1, 3, 4, 4, 4, 3, 5};
  std::vector<int32_t> _row_splits{0, 5, 8, 9, 13, 16, 18};
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    Array1<int32_t> row_splits(context, _row_splits);
    RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
    Array1<int32_t> values(context, col_indexes);
    Ragged<int32_t> ragged(shape, values);
    Array1<int32_t> order = GetTransposeReordering(ragged, 6);
    CheckArrayData(
        order, {5, 6, 9, 0, 1, 8, 10, 11, 7, 2, 3, 4, 12, 16, 13, 14, 15, 17});
    EXPECT_TRUE(context->IsCompatible(*order.Context()));
  }
}

TEST(GetTransposeReordering, WithDuplicatesThreeAxes) {
  //       col0   col1   col2    col3      col4      col5
  // row0         a0,a1         b2,b3,b4
  // row1  c5,c6          d7
  // row2         e8
  // row3   f9   g10,g11         h12
  // row4                                i13,i14,i15
  // row5                                 j16         k17
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    Array1<int32_t> col_indexes(
        context, std::vector<int32_t>{1, 1, 3, 3, 3, 0, 0, 2, 1, 0, 1, 1, 3, 4,
                                      4, 4, 4, 5});
    Array1<int32_t> row_splits1(context, std::vector<int32_t>{0, 4, 6});
    Array1<int32_t> row_splits2(context,
                                std::vector<int32_t>{0, 5, 8, 9, 13, 16, 18});
    RaggedShape shape =
        RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);
    Ragged<int32_t> ragged(shape, col_indexes);
    Array1<int32_t> order = GetTransposeReordering(ragged, 6);
    CheckArrayData(
        order, {5, 6, 9, 0, 1, 8, 10, 11, 7, 2, 3, 4, 12, 13, 14, 15, 16, 17});
    EXPECT_TRUE(context->IsCompatible(*order.Context()));
  }
}

TEST(ChangeSublistSize, TwoAxes) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    Array1<int32_t> row_splits1(context, std::vector<int32_t>{0, 2, 5});
    RaggedShape src = RaggedShape2(&row_splits1, nullptr, -1);

    int32_t size_delta = 2;
    RaggedShape dst = ChangeSublistSize(src, size_delta);
    CheckArrayData(dst.RowSplits(1), std::vector<int32_t>{0, 4, 9});

    size_delta = -2;
    dst = ChangeSublistSize(src, size_delta);
    CheckArrayData(dst.RowSplits(1), std::vector<int32_t>{0, 0, 1});

    size_delta = 0;
    dst = ChangeSublistSize(src, size_delta);
    CheckArrayData(dst.RowSplits(1), std::vector<int32_t>{0, 2, 5});
  }
}

TEST(ChangeSublistSize, ThreeAxes) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    /*
     [
       [ [x, x, x], [x, x] ]
       [ [x], [x, x], [x, x, x] ]
     ]
     */
    Array1<int32_t> row_splits1(context, std::vector<int32_t>{0, 2, 5});
    Array1<int32_t> row_splits2(context,
                                std::vector<int32_t>{0, 3, 5, 6, 8, 11});
    RaggedShape src =
        RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);

    int32_t size_delta = 2;
    RaggedShape dst = ChangeSublistSize(src, size_delta);
    CheckArrayData(dst.RowSplits(2), std::vector<int32_t>{0, 5, 9, 12, 16, 21});

    // it is an error to use -2 here
    // because the state (state_idx01 == 2) has only 1 entry
    size_delta = -1;

    dst = ChangeSublistSize(src, size_delta);
    CheckArrayData(dst.RowSplits(2), std::vector<int32_t>{0, 2, 3, 3, 4, 6});

    size_delta = 0;
    dst = ChangeSublistSize(src, size_delta);
    CheckArrayData(dst.RowSplits(2), std::vector<int32_t>{0, 3, 5, 6, 8, 11});
  }
}

TEST(RaggedShapeOpsTest, TestGetCountsPartitioned) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    // Testing with simple case is good enough as we have tested GetCounts()
    // with random large size and GetCountsPartitioned just calls GetCounts.
    std::vector<int32_t> src_row_splits_vec = {0, 3, 4, 6, 10};
    Array1<int32_t> src_row_splits(context, src_row_splits_vec);
    RaggedShape src_shape = RaggedShape2(&src_row_splits, nullptr, -1);
    std::vector<int32_t> src_values_vec = {0, 1, 0, 2, 5, 5, 7, 7, 9, 7};
    Array1<int32_t> src_values(context, src_values_vec);
    Ragged<int32_t> src(src_shape, src_values);

    std::vector<int32_t> ans_row_splits_vec = {0, 2, 4, 7, 10};
    Array1<int32_t> ans_row_splits(context, ans_row_splits_vec);
    RaggedShape ans_shape = RaggedShape2(&ans_row_splits, nullptr, -1);

    Ragged<int32_t> result = GetCountsPartitioned(src, ans_shape);

    ASSERT_EQ(result.NumAxes(), 2);
    // Check row_splits
    Array1<int32_t> row_splits = result.shape.RowSplits(1).To(cpu);
    std::vector<int32_t> result_row_splits(
        row_splits.Data(), row_splits.Data() + row_splits.Dim());
    EXPECT_EQ(result_row_splits, ans_row_splits_vec);
    // check values
    std::vector<int32_t> expected_data = {2, 1, 1, 0, 0, 2, 0, 3, 0, 1};
    Array1<int32_t> values = result.values.To(cpu);
    std::vector<int32_t> data(values.Data(), values.Data() + values.Dim());
    EXPECT_EQ(data, expected_data);
  }
}

TEST(RaggedShapeOpsTest, TestStack) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // simple case
      std::vector<RaggedShape> shapes(2);
      std::vector<RaggedShape *> shapes_ptr(2);
      std::vector<std::vector<Array1<int32_t>>> row_splits_vec(2);
      {
        const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
        const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
        Array1<int32_t> splits1(context, row_splits1);
        Array1<int32_t> splits2(context, row_splits2);
        row_splits_vec[0].push_back(splits1);
        row_splits_vec[1].push_back(splits2);
        shapes[0] = RaggedShape3(&splits1, nullptr, -1, &splits2, nullptr, -1);
        shapes_ptr[0] = &shapes[0];
      }
      {
        const std::vector<int32_t> row_splits1 = {0, 1, 3, 4};
        const std::vector<int32_t> row_splits2 = {0, 3, 4, 5, 7};
        Array1<int32_t> splits1(context, row_splits1);
        Array1<int32_t> splits2(context, row_splits2);
        row_splits_vec[0].push_back(splits1);
        row_splits_vec[1].push_back(splits2);
        shapes[1] = RaggedShape3(&splits1, nullptr, -1, &splits2, nullptr, -1);
        shapes_ptr[1] = &shapes[1];
      }
      std::vector<std::vector<int32_t>> expected_row_splits = {
          {0, 3, 6},
          {0, 2, 5, 6, 7, 9, 10},
          {0, 2, 3, 4, 6, 7, 10, 13, 14, 15, 17}};

      {
        // axis == 0
        int32_t axis = 0;
        RaggedShape result = Stack(axis, 2, shapes_ptr.data());
        for (int32_t i = 0; i != 3; ++i) {
          CheckArrayData(result.RowSplits(i + 1), expected_row_splits[i]);
        }
      }
      {
        // axis == 1
        int32_t axis = 1;
        RaggedShape result = Stack(axis, 2, shapes_ptr.data());
        RaggedShape transpose = Transpose(result);
        for (int32_t i = 0; i != 3; ++i) {
          CheckArrayData(transpose.RowSplits(i + 1), expected_row_splits[i]);
        }
      }
    }

    {
      // test with random large size
      for (int32_t m = 0; m < 2; ++m) {
        int32_t num_shape = RandInt(2, 100);
        int32_t num_axes = RandInt(2, 4);
        int32_t dim0 = RandInt(1, 100);
        std::vector<RaggedShape> shape_vec(num_shape);
        std::vector<RaggedShape *> shapes(num_shape);
        for (int32_t j = 0; j != num_shape; ++j) {
          RaggedShape shape =
              RandomRaggedShape(false, num_axes, num_axes, 0, 1000).To(context);
          int32_t src_dim0 = shape.Dim0();
          std::vector<int32_t> row_splits_vec(dim0 + 1);
          row_splits_vec[0] = 0;
          for (int32_t n = 1; n < dim0; ++n) {
            row_splits_vec[n] = RandInt(0, src_dim0);
          }
          row_splits_vec[dim0] = src_dim0;
          std::sort(row_splits_vec.begin(), row_splits_vec.end());
          Array1<int32_t> row_splits(context, row_splits_vec);
          RaggedShape first = RaggedShape2(&row_splits, nullptr, -1);
          RaggedShape new_shape = ComposeRaggedShapes(first, shape);
          shape_vec[j] = new_shape;
          shapes[j] = &shape_vec[j];
        }
        std::vector<RaggedShape> cpu_shapes(num_shape);
        for (auto i = 0; i != num_shape; ++i) {
          cpu_shapes[i] = shape_vec[i].To(cpu);
        }

        {
          // axis == 0
          int32_t axis = 0;
          RaggedShape result = Stack(axis, num_shape, shapes.data());
          ASSERT_EQ(result.NumAxes(),
                    num_axes + 2);  // note we append one axis in each shape in
                                    // `shapes` before `Stack`
          ASSERT_EQ(result.Dim0(), num_shape);
          result = result.To(cpu);
          for (auto iter = result.Iterator(); !iter.Done(); iter.Next()) {
            std::vector<int32_t> index = iter.Value();
            int32_t t = result[index];  // don't need the value, just make sure
                                        // it's a valid index.
            int32_t i = index[0];
            index.erase(index.begin());
            // result[i,j,k,l] = (shape[i])[j,k,l]
            i = cpu_shapes[i][index];  // don't need the value, just need to
                                       // make sure it's an allowable index.
          }
        }
        {
          // axis == 1
          int32_t axis = 1;
          RaggedShape result = Stack(axis, num_shape, shapes.data());
          ASSERT_EQ(result.NumAxes(),
                    num_axes + 2);  // note we append one axis in each shape in
                                    // `shapes` before `Stack`
          ASSERT_EQ(result.Dim0(), dim0);
          result = result.To(cpu);
          for (auto iter = result.Iterator(); !iter.Done(); iter.Next()) {
            std::vector<int32_t> index = iter.Value();
            int32_t t = result[index];  // don't need the value, just make sure
                                        // it's a valid index.
            int32_t i = index[1];
            index.erase(index.begin() + 1);
            // result[i,j,k,l] = (shape[j])[i,k,l]
            i = cpu_shapes[i][index];  // don't need the value, just need to
                                       // make sure it's an allowable index.
          }
        }
      }
    }
  }
}

template <typename T>
void TestStackRagged() {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    // test with random large size
    for (int32_t m = 0; m < 2; ++m) {
      int32_t num_shape = RandInt(2, 100);
      int32_t num_axes = RandInt(2, 4);
      int32_t dim0 = RandInt(1, 100);
      std::vector<Ragged<T>> ragged_vec(num_shape);
      std::vector<Ragged<T> *> ragged(num_shape);
      for (int32_t j = 0; j != num_shape; ++j) {
        RaggedShape shape =
            RandomRaggedShape(false, num_axes, num_axes, 0, 1000).To(context);
        int32_t src_dim0 = shape.Dim0();
        std::vector<int32_t> row_splits_vec(dim0 + 1);
        row_splits_vec[0] = 0;
        for (int32_t n = 1; n < dim0; ++n) {
          row_splits_vec[n] = RandInt(0, src_dim0);
        }
        row_splits_vec[dim0] = src_dim0;
        std::sort(row_splits_vec.begin(), row_splits_vec.end());
        Array1<int32_t> row_splits(context, row_splits_vec);
        RaggedShape first = RaggedShape2(&row_splits, nullptr, -1);
        RaggedShape new_shape = ComposeRaggedShapes(first, shape);
        int32_t num_elems = new_shape.NumElements();
        Array1<T> src_values =
            RandUniformArray1<T>(context, num_elems, 0, 10000);
        ragged_vec[j] = Ragged<T>(new_shape, src_values);
        ragged[j] = &ragged_vec[j];
      }
      std::vector<Ragged<T>> cpu_ragged_vec(num_shape);
      for (auto j = 0; j != num_shape; ++j) {
        cpu_ragged_vec[j] = ragged_vec[j].To(cpu);
      }

      {
        // axis == 0
        int32_t axis = 0;
        Ragged<T> result = Stack(axis, num_shape, ragged.data());
        ASSERT_EQ(result.NumAxes(),
                  num_axes + 2);  // note we append one axis in each shape in
                                  // `shapes` before `Stack`
        ASSERT_EQ(result.Dim0(), num_shape);
        result = result.To(cpu);
        RaggedShape &shape = result.shape;
        for (auto iter = shape.Iterator(); !iter.Done(); iter.Next()) {
          std::vector<int32_t> index = iter.Value();
          T value = result[index];
          int32_t i = index[0];
          index.erase(index.begin());
          // result[i,j,k,l] = (shape[i])[j,k,l]
          EXPECT_EQ(value, cpu_ragged_vec[i][index]);
        }
      }
      {
        // axis == 1
        int32_t axis = 1;
        Ragged<T> result = Stack(axis, num_shape, ragged.data());
        ASSERT_EQ(result.NumAxes(),
                  num_axes + 2);  // note we append one axis in each shape in
                                  // `shapes` before `Stack`
        ASSERT_EQ(result.Dim0(), dim0);
        result = result.To(cpu);
        RaggedShape &shape = result.shape;
        for (auto iter = shape.Iterator(); !iter.Done(); iter.Next()) {
          std::vector<int32_t> index = iter.Value();
          T value = result[index];
          int32_t j = index[1];
          index.erase(index.begin() + 1);
          // result[i,j,k,l] = (shape[j])[i,k,l]
          EXPECT_EQ(value, cpu_ragged_vec[j][index]);
        }
      }
    }
  }
}
TEST(RaggedTest, TestStackRagged) {
  TestStackRagged<int32_t>();
  TestStackRagged<double>();
}

TEST(RaggedTest, TestMaxSize) {
  for (int32_t i = 0; i <= 10; i++) {
    ContextPtr c = (i % 2 == 0 ? GetCpuContext() : GetCudaContext());
    int32_t num_axes = RandInt(2, 4);
    RaggedShape shape =
        RandomRaggedShape(true, num_axes, num_axes, 0, 1000).To(c);
    int32_t axis = RandInt(1, num_axes - 1);
    int32_t max_size = shape.MaxSize(axis);
    if (axis == 0) {
      K2_CHECK(max_size == shape.Dim0());
    } else {
      Array1<int32_t> row_splits = shape.RowSplits(axis).To(GetCpuContext());
      int32_t *row_splits_data = row_splits.Data();
      int32_t m = 0;
      for (int32_t i = 0; i + 1 < row_splits.Dim(); i++) {
        int32_t size = row_splits_data[i + 1] - row_splits_data[i];
        if (size > m) m = size;
      }
      ASSERT_EQ(m, max_size);
    }
  }
}

TEST(RaggedShapeOpsTest, TestMakeTransposable) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // simple case
      const std::vector<int32_t> row_splits1 = {0, 2, 5, 6, 8};
      // const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2, 3, 3};
      const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10, 12, 13};
      // const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5, 6,
      //                                        6, 7};
      Array1<int32_t> row_splits1_array(context, row_splits1);
      Array1<int32_t> row_splits2_array(context, row_splits2);
      RaggedShape shape = RaggedShape3(&row_splits1_array, nullptr, -1,
                                       &row_splits2_array, nullptr, -1);

      std::vector<std::vector<int32_t>> expected_row_splits = {
          {0, 3, 6, 9, 12}, {0, 2, 3, 3, 4, 6, 7, 10, 10, 10, 12, 13, 13}};
      std::vector<std::vector<int32_t>> expected_row_ids = {
          {0, 0, 0, 1, 1, 1, 2, 2, 2, 3, 3, 3},
          {0, 0, 1, 3, 4, 4, 5, 6, 6, 6, 9, 9, 10}};

      RaggedShape result = MakeTransposable(shape);
      for (int32_t i = 1; i != 3; ++i) {
        CheckArrayData(result.RowSplits(i), expected_row_splits[i - 1]);
        CheckArrayData(result.RowIds(i), expected_row_ids[i - 1]);
      }
    }

    {
      // test with random large size
      for (int32_t i = 0; i < 2; ++i) {
        int32_t num_axes = RandInt(2, 4);
        RaggedShape shape =
            RandomRaggedShape(true, num_axes, num_axes, 0, 1000).To(context);
        int32_t dim0 = shape.Dim0();
        int32_t max_size = shape.MaxSize(1);
        RaggedShape result = MakeTransposable(shape);
        shape = shape.To(cpu);
        result = result.To(cpu);
        EXPECT_EQ(result.Dim0(), dim0);
        EXPECT_EQ(result.TotSize(1), dim0 * max_size);
        // check if every sub list in axis 1 has the same size
        int32_t *row_splits1 = result.RowSplits(1).Data();
        for (int32_t j = 0; j != dim0 + 1; ++j) {
          EXPECT_EQ(row_splits1[j], j * max_size);
        }
        if (num_axes > 2) {
          for (auto iter = shape.Iterator(); !iter.Done(); iter.Next()) {
            std::vector<int32_t> index = iter.Value();
            EXPECT_EQ(shape[index], result[index]);
          }
        }
      }
    }
  }
}

TEST(RaggedShapeOpsTest, PrefixTest) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // simple case
      const std::vector<int32_t> row_splits1 = {0, 2, 5, 6, 8};
      const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10, 12, 13};
      Array1<int32_t> row_splits1_array(context, row_splits1);
      Array1<int32_t> row_splits2_array(context, row_splits2);
      RaggedShape shape = RaggedShape3(&row_splits1_array, nullptr, -1,
                                       &row_splits2_array, nullptr, -1);
      int32_t dim0 = shape.Dim0();
      int32_t num_axes = shape.NumAxes();
      EXPECT_EQ(dim0, 4);
      EXPECT_EQ(num_axes, 3);
      {
        // n == 0
        int32_t n = 0;
        std::vector<std::vector<int32_t>> expected_row_splits = {{0}, {0}};
        RaggedShape result = Prefix(shape, n);
        EXPECT_TRUE(IsCompatible(shape, result));
        EXPECT_EQ(result.Dim0(), n);
        EXPECT_EQ(result.NumAxes(), num_axes);
        for (int32_t i = 1; i != num_axes; ++i) {
          CheckArrayData(result.RowSplits(i), expected_row_splits[i - 1]);
        }
      }

      {
        // n > 0 && n < dim0
        int32_t n = 2;
        std::vector<std::vector<int32_t>> expected_row_splits = {
            {0, 2, 5}, {0, 2, 3, 4, 6, 7}};
        RaggedShape result = Prefix(shape, n);
        EXPECT_TRUE(IsCompatible(shape, result));
        EXPECT_EQ(result.Dim0(), n);
        EXPECT_EQ(result.NumAxes(), num_axes);
        for (int32_t i = 1; i != num_axes; ++i) {
          CheckArrayData(result.RowSplits(i), expected_row_splits[i - 1]);
        }
      }

      {
        // n == dim0
        int32_t n = 4;
        std::vector<std::vector<int32_t>> expected_row_splits = {
            {0, 2, 5}, {0, 2, 3, 4, 6, 7}};
        RaggedShape result = Prefix(shape, n);
        EXPECT_TRUE(IsCompatible(shape, result));
        EXPECT_EQ(result.Dim0(), n);
        EXPECT_EQ(result.NumAxes(), num_axes);
        CheckArrayData(result.RowSplits(1), row_splits1);
        CheckArrayData(result.RowSplits(2), row_splits2);
      }
    }

    {
      // test with random large size
      for (int32_t i = 0; i < 2; ++i) {
        RaggedShape shape = RandomRaggedShape(false, 2, 4, 0, 1000).To(context);
        int32_t dim0 = shape.Dim0();
        int32_t num_axes = shape.NumAxes();
        int32_t n = RandInt(0, dim0);
        RaggedShape result = Prefix(shape, n);
        EXPECT_TRUE(IsCompatible(shape, result));
        EXPECT_EQ(result.Dim0(), n);
        EXPECT_EQ(result.NumAxes(), num_axes);
        // just check row_splits1 here would be fine, as we have tested it with
        // simple case. We just confirm it can run successfully with kinds of
        // different random shapes.
        CheckArrayData(result.RowSplits(1), shape.RowSplits(1).Range(0, n + 1));
      }
    }
  }
}

TEST(RaggedShapeOpsTest, GetPrefixesTest) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // test with random large size
      for (int32_t i = 0; i < 2; ++i) {
        RaggedShape shape = RandomRaggedShape(false, 2, 4, 0, 1000).To(context);
        int32_t dim0 = shape.Dim0();
        int32_t num_axes = shape.NumAxes();
        int32_t ans_num = RandInt(0, 10);
        std::vector<int32_t> sizes;
        for (int32_t j = 0; j != ans_num; ++j)
          sizes.push_back(RandInt(0, dim0));
        ASSERT_EQ(sizes.size(), ans_num);
        std::vector<RaggedShape> ans = GetPrefixes(shape, sizes);
        ASSERT_EQ(ans.size(), ans_num);

        for (int32_t j = 0; j != ans_num; ++j) {
          int32_t n = sizes[j];

          RaggedShape ans_j = ans[j];
          EXPECT_TRUE(IsCompatible(shape, ans_j));
          EXPECT_EQ(ans_j.Dim0(), n);
          EXPECT_EQ(ans_j.NumAxes(), num_axes);

          RaggedShape result = Prefix(shape, n);
          EXPECT_TRUE(IsCompatible(shape, result));
          EXPECT_EQ(result.Dim0(), n);
          EXPECT_EQ(result.NumAxes(), num_axes);

          for (int32_t m = 1; m != num_axes; ++m) {
            EXPECT_TRUE(Equal(result.RowSplits(m), ans_j.RowSplits(m)));
          }
        }
      }
    }
  }
}

}  // namespace k2
