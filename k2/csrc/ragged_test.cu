#include "hip/hip_runtime.h"
/**
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey, Haowen Qiu)
 *                      Mobvoi Inc.        (authors: Fangjun Kuang)
 *                      Yiming Wang
 *
 * See LICENSE for clarification regarding multiple authors
 */

#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include <algorithm>
#include <limits>
#include <numeric>
#include <set>
#include <utility>
#include <vector>

#include "k2/csrc/array.h"
#include "k2/csrc/array_ops.h"
#include "k2/csrc/context.h"
#include "k2/csrc/fsa_utils.h"
#include "k2/csrc/math.h"
#include "k2/csrc/ragged.h"
#include "k2/csrc/ragged_ops.h"
#include "k2/csrc/tensor.h"
#include "k2/csrc/test_utils.h"

namespace k2 {


TEST(RaggedShapeOpsTest, CatMoreAxes) {
  for (auto &c : {GetCpuContext(), GetCudaContext()}) {
    RaggedShape shape1 =
                    RaggedShape("[ [ [ [ x x ] ] [ [x ] ] ] [[[x]]]]").To(c),
                shape2 =
                    RaggedShape("[ [ [ [x ] ] [ [x ] ] ] [[[x x]]]]").To(c),
                shape3 = RaggedShape("[ [ [ [ ] ] [ [ x ] ] ] [[[]]]]").To(c);

    RaggedShape cat_axis2_ref =
        RaggedShape("[ [ [[ x x ][ x ][]] [[x ][x][ x ]] ] [[[x ][ x x][]]]]")
            .To(c);
    RaggedShape cat_axis3_ref =
        RaggedShape("[ [ [[ x x x ]] [[x x x ]] ] [[[x x x]]]]").To(c);
    RaggedShape *srcs[] = {&shape1, &shape2, &shape3};
    Array1<uint32_t> merge_map2;
    Array1<uint32_t> merge_map3;
    RaggedShape cat_axis2 = Cat(2, 3, srcs, &merge_map2);
    RaggedShape cat_axis3 = Cat(3, 3, srcs, &merge_map3);
    K2_LOG(INFO) << "cat_axis2 = " << cat_axis2;
    K2_LOG(INFO) << "cat_axis3 = " << cat_axis3;

    K2_CHECK(Equal(cat_axis2, cat_axis2_ref));
    K2_CHECK(Equal(cat_axis2, cat_axis2_ref));

    std::vector<uint32_t> merge_values = {0, 3, 1, 6, 4, 2, 9, 7, 10};
    CheckArrayData(merge_map2, merge_values);
    CheckArrayData(merge_map3, merge_values);
  }
}

TEST(RaggedShapeOpsTest, StackMoreAxes) {
  for (auto &c : {GetCpuContext(), GetCudaContext()}) {
    RaggedShape shape1 =
                    RaggedShape("[ [ [ [ x x ] ] [ [x ] ] ] [[[x]]]]").To(c),
                shape2 =
                    RaggedShape("[ [ [ [x ] ] [ [x ] ] ] [[[x x]]]]").To(c),
                shape3 = RaggedShape("[ [ [ [ ] ] [ [ x ] ] ] [[[]]]]").To(c);

    RaggedShape stacked_ref =
        RaggedShape(
            "[ [ [[[ x x ]][[ x ]][[]]] [[[x ]][[x]][[ x ]]] ] "
            "[[[[x ]][[ x x]][[]]]]]")
            .To(c);
    RaggedShape *srcs[] = {&shape1, &shape2, &shape3};
    Array1<uint32_t> merge_map2;
    Array1<uint32_t> merge_map3;
    RaggedShape stacked_axis2 = Stack(2, 3, srcs, &merge_map2);
    RaggedShape stacked_axis3 = Stack(3, 3, srcs, &merge_map3);
    K2_LOG(INFO) << "stacked_axis2 = " << stacked_axis2;
    K2_LOG(INFO) << "stacked_axis3 = " << stacked_axis3;

    K2_CHECK(Equal(stacked_axis2, stacked_ref));
    K2_CHECK(Equal(stacked_axis2, stacked_ref));

    std::vector<uint32_t> merge_values = {0, 3, 1, 6, 4, 2, 9, 7, 10};
    CheckArrayData(merge_map2, merge_values);
    CheckArrayData(merge_map3, merge_values);
  }
}


class RaggedShapeOpsSuiteTest : public ::testing::Test {
 protected:
  RaggedShapeOpsSuiteTest() {
    ContextPtr context = GetCpuContext();
    const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
    const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
    const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
    const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};
    const std::vector<int32_t> row_splits3 = {0,  2,  3,  5,  8, 9,
                                              12, 13, 15, 15, 16};
    const std::vector<int32_t> row_ids3 = {0, 0, 1, 2, 2, 3, 3, 3,
                                           4, 5, 5, 5, 6, 7, 7, 9};
    std::vector<RaggedShapeLayer> axes;
    axes.emplace_back(RaggedShapeLayer{Array1<int32_t>(context, row_splits1),
                                       Array1<int32_t>(context, row_ids1),
                                       static_cast<int32_t>(row_ids1.size())});
    axes.emplace_back(RaggedShapeLayer{Array1<int32_t>(context, row_splits2),
                                       Array1<int32_t>(context, row_ids2),
                                       static_cast<int32_t>(row_ids2.size())});
    axes.emplace_back(RaggedShapeLayer{Array1<int32_t>(context, row_splits3),
                                       Array1<int32_t>(context, row_ids3),
                                       static_cast<int32_t>(row_ids3.size())});

    simple_shape_ = RaggedShape(axes, true);

    // random_shape_ is on CPU
    random_shape_ = RandomRaggedShape(true,   // set_row_ids
                                      3,      // min_num_axes
                                      4,      // max_num_axes
                                      0,      // min_num_elements
                                      1000);  // max_num_elements
  }

  RaggedShape simple_shape_;
  RaggedShape random_shape_;
};

TEST(RaggedShapeTest, TestConstructFromString) {
  RaggedShape rs(" [ [ x x ] [x] ]");
  Array1<int32_t> row_splits1(GetCpuContext(), std::vector<int32_t>{0, 2, 3});
  K2_LOG(INFO) << rs.RowSplits(1);
  K2_CHECK(Equal(rs.RowSplits(1), row_splits1));

  RaggedShape rs2(" [ [ [ x x ] ] [[x]] ]");
  K2_LOG(INFO) << "rs2 = " << rs2;

  K2_CHECK_EQ(RaggedShape("[ ]").Dim0(), 0);

  ASSERT_THROW(RaggedShape(" [ [ x x ] [x] "), std::runtime_error);
  ASSERT_THROW(RaggedShape(" [ [ x x ] [[x]]] "), std::runtime_error);
  ASSERT_THROW(RaggedShape(" [ [ x [] x ] "), std::runtime_error);
  ASSERT_THROW(RaggedShape(" [ x ] "), std::runtime_error);
  ASSERT_THROW(RaggedShape(" [ x ] [ x ] "), std::runtime_error);
  ASSERT_THROW(RaggedShape(" [ x | x ] "), std::runtime_error);

  for (int i = 0; i < 5; i++) {
    RaggedShape rs = RandomRaggedShape(true,
                                       2,      // min_num_axes
                                       4,      // max_num_axes
                                       0,      // min_num_elements
                                       1000);  // max_num_elements
    std::ostringstream os;
    os << rs;
    RaggedShape rs2;
    std::istringstream is(os.str());
    K2_LOG(INFO) << "Shape is: " << os.str();
    is >> rs2;
    K2_CHECK(is.good());
    // the reason for the || below is that in "[ ]", the number of
    // axes is ambiguous; we assume 2.
    K2_CHECK(Equal(rs, rs2) || rs.NumElements() == 0);
  }
}

TEST(RaggedTest, TestRaggedFromString) {
  Ragged<int32_t> rs(" [ [ 1 2 ] [3] ]");
  Array1<int32_t> row_splits1(GetCpuContext(), std::vector<int32_t>{0, 2, 3});
  K2_LOG(INFO) << rs.RowSplits(1);
  K2_CHECK(Equal(rs.RowSplits(1), row_splits1));
  K2_CHECK_EQ(rs.values.Back(), 3);
  K2_CHECK_EQ(rs.values[0], 1);

  Ragged<int32_t> rs2(" [ [ [ 0 5 ] ] [[10]] ]");
  K2_LOG(INFO) << "rs2 = " << rs2;

  ASSERT_THROW(RaggedShape(" [ [ 0 0 ] [0] "), std::runtime_error);
  ASSERT_THROW(RaggedShape(" [ [ 0 0 ] [[0]]] "), std::runtime_error);
  ASSERT_THROW(RaggedShape(" [ [ 0 [] 0 ] "), std::runtime_error);
  ASSERT_THROW(RaggedShape(" [ 0 ] "), std::runtime_error);
  ASSERT_THROW(RaggedShape(" [ 0 ] [ 0 ] "), std::runtime_error);
  ASSERT_THROW(RaggedShape(" [ 0 | 0 ] "), std::runtime_error);

  for (int32_t i = 0; i < 5; i++) {
    Ragged<int32_t> r = RandomRagged<int32_t>();
    std::ostringstream os;
    os << r;
    Ragged<int32_t> r2(os.str());
    // the reason for the || below is that in "[ ]", the number of
    // axes is ambiguous; we assume 2.
    K2_CHECK(Equal(r, r2) || r.values.Dim() == 0);
  }
}

template <typename T>
void TestMaxPerSubListTest() {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // empty case
      const std::vector<int32_t> row_splits = {0};
      RaggedShapeLayer shape_dim;
      shape_dim.row_splits = Array1<int32_t>(context, row_splits);
      shape_dim.cached_tot_size = 0;
      std::vector<RaggedShapeLayer> axes = {shape_dim};
      RaggedShape shape(axes, true);
      Array1<T> values(context, 0);
      Ragged<T> ragged(shape, values);

      int32_t num_rows = ragged.shape.Dim0();
      ASSERT_EQ(num_rows, 0);
      Array1<T> max_values(context, num_rows);
      // just run to check if there's any error
      MaxPerSublist(ragged, 1, &max_values);
      EXPECT_EQ(max_values.Dim(), 0);
    }

    {
      const std::vector<int32_t> row_splits = {0, 2, 2, 5, 6};
      RaggedShapeLayer shape_dim;
      shape_dim.row_splits = Array1<int32_t>(context, row_splits);
      shape_dim.cached_tot_size = row_splits.back();
      std::vector<RaggedShapeLayer> axes = {shape_dim};
      RaggedShape shape(axes, true);
      const std::vector<T> values_vec = {1, 3, 2, 8, 0, -1};
      Array1<T> values(context, values_vec);
      Ragged<T> ragged(shape, values);

      int32_t num_rows = ragged.shape.Dim0();
      Array1<T> max_values(context, num_rows);
      T default_value = 2;
      MaxPerSublist(ragged, default_value, &max_values);
      // copy memory from GPU/CPU to CPU
      std::vector<T> cpu_data(max_values.Dim());
      max_values.Context()->CopyDataTo(
          max_values.Dim() * max_values.ElementSize(), max_values.Data(), cpu,
          cpu_data.data());
      std::vector<T> expected_data = {3, default_value, 8, default_value};
      EXPECT_EQ(cpu_data, expected_data);
    }

    {
      // test with random large size
      const int32_t min_num_elements = 2000;
      // not random shape is on CPU
      RaggedShape shape =
          RandomRaggedShape(false, 2, 2, min_num_elements, 5000);
      ASSERT_EQ(shape.NumAxes(), 2);
      RaggedShape gpu_shape;
      if (context->GetDeviceType() == kCuda) {
        // copy shape to GPU
        const Array1<T> &row_splits = shape.RowSplits(1);
        RaggedShapeLayer shape_dim;
        shape_dim.row_splits = row_splits.To(GetCudaContext());
        shape_dim.cached_tot_size = shape.NumElements();
        std::vector<RaggedShapeLayer> axes = {shape_dim};
        gpu_shape = RaggedShape(axes, true);
      }

      int32_t num_elems = shape.NumElements();
      std::vector<T> data(num_elems);
      for (int32_t i = 0; i != 10; ++i) {
        std::iota(data.begin(), data.end(), 0);
        // randomly set data[pos] = num_elems which is
        // greater than any element in data
        int32_t pos = RandInt(0, num_elems - 1);
        data[pos] = num_elems;
        // find the corresponding row
        int32_t num_rows = shape.Dim0();
        const int32_t *row_splits_data = shape.RowSplits(1).Data();
        int32_t row = 0;
        for (int32_t i = 0; i < num_rows; ++i) {
          if (pos >= row_splits_data[i] && pos < row_splits_data[i + 1]) {
            row = i;
            break;
          }
        }

        Array1<T> values(context, data);
        Ragged<T> ragged(context->GetDeviceType() == kCuda ? gpu_shape : shape,
                         values);
        Array1<T> max_values(context, num_rows);
        T default_value = 0;
        MaxPerSublist(ragged, default_value, &max_values);
        EXPECT_EQ(max_values[row], num_elems);
      }
    }
  }
}

TEST(RaggedShapeOpsTest, MaxPerSubListTest) {
  TestMaxPerSubListTest<int32_t>();
}

template <typename T>
void TestArgMaxPerSubListTest() {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // empty case
      const std::vector<int32_t> row_splits_vec = {0};
      Array1<int32_t> row_splits(context, row_splits_vec);
      RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
      Array1<T> values(context, 0);
      Ragged<T> ragged(shape, values);

      int32_t num_rows = ragged.shape.Dim0();
      ASSERT_EQ(num_rows, 0);
      Array1<int32_t> argmax_values(context, num_rows);
      // just run to check if there's any error
      ArgMaxPerSublist(ragged, 1, &argmax_values);
      EXPECT_EQ(argmax_values.Dim(), 0);
    }

    {
      const std::vector<int32_t> row_splits_vec = {0, 3, 3, 6, 7};
      Array1<int32_t> row_splits(context, row_splits_vec);
      RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
      const std::vector<T> values_vec = {1, 3, 3, 2, 1, 0, -1};
      Array1<T> values(context, values_vec);
      Ragged<T> ragged(shape, values);

      int32_t num_rows = ragged.shape.Dim0();
      Array1<T> argmax_values(context, num_rows);
      T default_value = 2;
      ArgMaxPerSublist(ragged, default_value, &argmax_values);
      std::vector<T> expected_data = {2, -1, 3, -1};
      CheckArrayData(argmax_values, expected_data);
    }

    {
      // test with random large size
      ContextPtr cpu = GetCpuContext();
      for (int32_t i = 0; i != 10; ++i) {
        Ragged<int32_t> ragged =
            RandomRagged<int32_t>(0, 1000, 2, 4, 0, 5000).To(context);
        int32_t last_axis = ragged.NumAxes() - 1;
        Array1<int32_t> argmax_values(context,
                                      ragged.RowSplits(last_axis).Dim() - 1);
        int32_t default_value = 2;
        ArgMaxPerSublist(ragged, default_value, &argmax_values);

        ragged = ragged.To(cpu);
        argmax_values = argmax_values.To(cpu);
        Array1<int32_t> row_splits = ragged.RowSplits(last_axis);
        int32_t rows = row_splits.Dim() - 1;
        for (int32_t row = 0; row < rows; row++) {
          int32_t begin = row_splits[row], end = row_splits[row + 1];
          int32_t max_val = 2, best_pos = -1;
          for (int32_t pos = begin; pos < end; pos++) {
            if (ragged.values[pos] >= max_val) {
              max_val = ragged.values[pos];
              best_pos = pos;
            }
          }
          EXPECT_EQ(argmax_values[row], best_pos);
        }
      }
    }
  }
}

TEST(RaggedShapeOpsTest, ArgMaxPerSubListTest) {
  TestArgMaxPerSubListTest<int32_t>();
}

template <typename T>
void TestMinPerSubListTest() {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // empty case
      std::vector<int32_t> row_splits_vec = {0};
      Array1<T> row_splits(context, row_splits_vec);
      RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
      Array1<T> values(context, 0);
      Ragged<T> ragged(shape, values);

      int32_t num_rows = ragged.shape.Dim0();
      ASSERT_EQ(num_rows, 0);
      Array1<T> min_values(context, num_rows);
      // just run to check if there's any error
      MinPerSublist(ragged, 1, &min_values);
      EXPECT_EQ(min_values.Dim(), 0);
    }

    {
      std::vector<int32_t> row_splits_vec = {0, 2, 2, 5, 6};
      Array1<T> row_splits(context, row_splits_vec);
      RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
      const std::vector<T> values_vec = {1, 3, 3, 8, 4, -1};
      Array1<T> values(context, values_vec);
      Ragged<T> ragged(shape, values);

      int32_t num_rows = ragged.shape.Dim0();
      Array1<T> min_values(context, num_rows);
      T default_value = 2;
      MinPerSublist(ragged, default_value, &min_values);
      // copy memory from GPU/CPU to CPU
      min_values = min_values.To(cpu);
      std::vector<T> cpu_data(min_values.Data(),
                              min_values.Data() + min_values.Dim());
      std::vector<T> expected_data = {1, default_value, default_value, -1};
      EXPECT_EQ(cpu_data, expected_data);
    }

    // May add tests for random large size? (but maybe it's fine to not add as
    // we have tested large cases in MaxPerSubList)
  }
}

TEST(RaggedShapeOpsTest, MinPerSubListTest) {
  TestMinPerSubListTest<int32_t>();
}

template <typename T>
void TestAndOrPerSubListTest() {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // And
      const std::vector<int32_t> row_splits = {0, 2, 2, 5, 6};
      RaggedShapeLayer shape_dim;
      shape_dim.row_splits = Array1<int32_t>(context, row_splits);
      shape_dim.cached_tot_size = row_splits.back();
      std::vector<RaggedShapeLayer> axes = {shape_dim};
      RaggedShape shape(axes, true);
      const std::vector<T> values_vec = {1, 3, 3, 6, 11, 0};
      Array1<T> values(context, values_vec);
      Ragged<T> ragged(shape, values);

      int32_t num_rows = ragged.shape.Dim0();
      Array1<T> dst(context, num_rows);
      T default_value = -1;
      AndPerSublist(ragged, default_value, &dst);
      // copy memory from GPU/CPU to CPU
      dst = dst.To(cpu);
      std::vector<T> cpu_data(dst.Data(), dst.Data() + dst.Dim());
      std::vector<T> expected_data = {1, -1, 2, 0};
      EXPECT_EQ(cpu_data, expected_data);
    }

    {
      // Or
      const std::vector<int32_t> row_splits = {0, 2, 2, 5, 6};
      RaggedShapeLayer shape_dim;
      shape_dim.row_splits = Array1<int32_t>(context, row_splits);
      shape_dim.cached_tot_size = row_splits.back();
      std::vector<RaggedShapeLayer> axes = {shape_dim};
      RaggedShape shape(axes, true);
      const std::vector<T> values_vec = {1, 3, 3, 4, 6, 0};
      Array1<T> values(context, values_vec);
      Ragged<T> ragged(shape, values);

      int32_t num_rows = ragged.shape.Dim0();
      Array1<T> dst(context, num_rows);
      T default_value = 0;
      OrPerSublist(ragged, default_value, &dst);
      // copy memory from GPU/CPU to CPU
      dst = dst.To(cpu);
      std::vector<T> cpu_data(dst.Data(), dst.Data() + dst.Dim());
      std::vector<T> expected_data = {3, 0, 7, 0};
      EXPECT_EQ(cpu_data, expected_data);
    }
  }
}

TEST(RaggedShapeOpsTest, AndOrPerSubListTest) {
  TestAndOrPerSubListTest<int32_t>();
}

void TestUnsqueeze(const RaggedShape &input_shape) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    RaggedShape src_shape = input_shape.To(context);
    src_shape.Populate();  // set row_ids
    {
      // axis = 0.
      RaggedShape shape = Unsqueeze(src_shape, 0);
      int32_t dim0 = src_shape.Dim0();
      const std::vector<RaggedShapeLayer> &src_axes = src_shape.Layers();
      const std::vector<RaggedShapeLayer> &dest_axes = shape.Layers();

      {
        const Array1<int32_t> &row_splits0 = dest_axes[0].row_splits;
        std::vector<int32_t> data = {0, dim0};
        CheckArrayData(row_splits0, data);
      }

      {
        const Array1<int32_t> &row_ids0 = dest_axes[0].row_ids;
        std::vector<int32_t> data(dim0, 0);
        CheckArrayData(row_ids0, data);
      }

      {
        for (size_t i = 0; i != src_axes.size(); ++i) {
          CheckArrayData(src_axes[i].row_splits, dest_axes[i + 1].row_splits);
          CheckArrayData(src_axes[i].row_ids, dest_axes[i + 1].row_ids);
        }
      }
    }

    {
      // axis = 1
      int32_t axis = 1;
      RaggedShape shape = Unsqueeze(src_shape, axis);
      int32_t tot_size = shape.TotSize(axis);
      const std::vector<RaggedShapeLayer> &src_axes = src_shape.Layers();
      const std::vector<RaggedShapeLayer> &dest_axes = shape.Layers();

      {
        for (int32_t i = 0; i < axis; ++i) {
          CheckArrayData(src_axes[i].row_splits, dest_axes[i].row_splits);
          CheckArrayData(src_axes[i].row_ids, dest_axes[i].row_ids);
        }
      }

      {
        const Array1<int32_t> &row_splits = dest_axes[axis].row_splits;
        std::vector<int32_t> data(tot_size + 1);
        std::iota(data.begin(), data.end(), 0);
        CheckArrayData(row_splits, data);
      }

      {
        const Array1<int32_t> &row_ids = dest_axes[axis].row_ids;
        std::vector<int32_t> data(tot_size);
        std::iota(data.begin(), data.end(), 0);
        CheckArrayData(row_ids, data);
      }

      {
        for (std::size_t i = axis; i < src_axes.size(); ++i) {
          CheckArrayData(src_axes[i].row_splits, dest_axes[i + 1].row_splits);
          CheckArrayData(src_axes[i].row_ids, dest_axes[i + 1].row_ids);
        }
      }
    }
  }
}

TEST_F(RaggedShapeOpsSuiteTest, TestUnsqueeze) {
  TestUnsqueeze(simple_shape_);
  TestUnsqueeze(random_shape_);
}

TEST(RaggedShapeOpsTest, TestUnsqueezeParallel) {
  for (int32_t i = 0; i < 10; i++) {
    ContextPtr c = (i % 2 == 0 ? GetCpuContext() : GetCudaContext());
    int32_t num_shapes = RandInt(0, 10);

    std::vector<RaggedShape *> orig_shapes;
    for (int32_t i = 0; i < num_shapes; i++)
      orig_shapes.push_back(
          new RaggedShape(RandomRaggedShape(false, 2, 5, 0, 1000).To(c)));
    int32_t axis = 0;  // only one supported for now.
    std::vector<RaggedShape> unsqueezed =
        UnsqueezeParallel(num_shapes, orig_shapes.data(), axis);
    for (int32_t i = 0; i < num_shapes; i++) {
      unsqueezed[i].Check();
      RaggedShape temp = RemoveAxis(unsqueezed[i], axis);
      ASSERT_EQ(Equal(temp, *(orig_shapes[i])), true);
      delete orig_shapes[i];
    }
  }
}

void TestRemoveAxis(const RaggedShape &input_shape) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    RaggedShape src_shape = input_shape.To(context);
    ASSERT_EQ(src_shape.NumAxes(), 4);
    {
      // axis = 0.
      int32_t axis = 0;
      RaggedShape shape = RemoveAxis(src_shape, axis);
      const std::vector<RaggedShapeLayer> &src_axes = src_shape.Layers();
      const std::vector<RaggedShapeLayer> &dest_axes = shape.Layers();
      ASSERT_EQ(src_axes.size(), 3);
      ASSERT_EQ(dest_axes.size(), 2);

      {
        for (std::size_t i = 0; i != dest_axes.size(); ++i) {
          CheckArrayData(dest_axes[i].row_splits, src_axes[i + 1].row_splits);
          CheckArrayData(dest_axes[i].row_ids, src_axes[i + 1].row_ids);
        }
      }
    }

    {
      // axis = 1
      int32_t axis = 1;
      RaggedShape shape = RemoveAxis(src_shape, axis);
      const std::vector<RaggedShapeLayer> &src_axes = src_shape.Layers();
      const std::vector<RaggedShapeLayer> &dest_axes = shape.Layers();
      ASSERT_EQ(src_axes.size(), 3);
      ASSERT_EQ(dest_axes.size(), 2);

      {
        const Array1<int32_t> &row_splits0 = dest_axes[0].row_splits;
        std::vector<int32_t> data = {0, 3, 7, 10};
        CheckArrayData(row_splits0, data);
      }

      {
        const Array1<int32_t> &row_ids0 = dest_axes[0].row_ids;
        std::vector<int32_t> data = {0, 0, 0, 1, 1, 1, 1, 2, 2, 2};
        CheckArrayData(row_ids0, data);
      }

      {
        for (std::size_t i = 1; i != dest_axes.size(); ++i) {
          CheckArrayData(dest_axes[i].row_splits, src_axes[i + 1].row_splits);
          CheckArrayData(dest_axes[i].row_ids, src_axes[i + 1].row_ids);
        }
      }
    }

    {
      // axis = 3
      int32_t axis = 3;  // the last axis
      RaggedShape shape = RemoveAxis(src_shape, axis);
      const std::vector<RaggedShapeLayer> &src_axes = src_shape.Layers();
      const std::vector<RaggedShapeLayer> &dest_axes = shape.Layers();
      ASSERT_EQ(src_axes.size(), 3);
      ASSERT_EQ(dest_axes.size(), 2);

      {
        for (std::size_t i = 0; i != dest_axes.size(); ++i) {
          CheckArrayData(dest_axes[i].row_splits, src_axes[i].row_splits);
          CheckArrayData(dest_axes[i].row_ids, src_axes[i].row_ids);
        }
      }
    }
  }
}

TEST_F(RaggedShapeOpsSuiteTest, TestRemoveAxis) {
  TestRemoveAxis(simple_shape_);
}

TEST(RaggedShapeOpsTest, TestGetOffsets) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    for (int32_t i = 0; i != 2; ++i) {
      int32_t num_shape = RandInt(10, 100);
      int32_t num_axes = RandInt(2, 4);
      std::vector<RaggedShape> shape_vec(num_shape);
      std::vector<RaggedShape *> shapes(num_shape);
      for (int32_t j = 0; j != num_shape; ++j) {
        shape_vec[j] =
            RandomRaggedShape(false, num_axes, num_axes, 0, 1000).To(context);
        shapes[j] = &shape_vec[j];
      }
      RaggedShape **shapes_ptr = shapes.data();
      Array2<int32_t> offsets = GetOffsets(num_shape, shapes_ptr);
      ASSERT_EQ(offsets.Dim0(), num_axes + 1);
      ASSERT_EQ(offsets.Dim1(), num_shape + 1);
      auto acc = offsets.Accessor();
      for (int32_t axis = 0; axis <= num_axes; ++axis) {
        int32_t sum = 0;
        for (int32_t j = 0; j <= num_shape; ++j) {
          EXPECT_EQ(acc(axis, j), sum);
          if (j < num_shape) {
            sum += (axis == 0 ? 1 : shape_vec[j].TotSize(axis - 1));
          }
        }
      }
    }
  }
}

// returns a random ragged shape where the dims on axis 1 are all the same
// (so: can be transposed).
RaggedShape RandomRaggedShapeToTranspose(ContextPtr c) {
  ContextPtr c_cpu = GetCpuContext();

  RaggedShape random = RandomRaggedShape(false, 2, 4, 0, 5000).To(c);

  int32_t input_dim0 = random.Dim0(), divisor = 1;
  for (int32_t i = 1; i * i <= input_dim0; i++) {
    if (input_dim0 % i == 0 && i > divisor) divisor = i;
  }

  int32_t output_dim0 = divisor, output_dim1 = input_dim0 / divisor;

  Array1<int32_t> row_splits =
      Range<int32_t>(c, output_dim0 + 1, 0, output_dim1);
  int32_t cached_tot_size = input_dim0;

  RaggedShape top_level_shape =
      RaggedShape2(&row_splits, nullptr, cached_tot_size);
  return ComposeRaggedShapes(top_level_shape, random);
}

TEST(RaggedShapeOpsTest, TestTranspose) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      const std::vector<int32_t> row_splits1_vec = {0, 2, 4, 6};
      const std::vector<int32_t> row_splits2_vec = {0, 3, 4, 7, 8, 10, 12};
      Array1<int32_t> row_splits1(context, row_splits1_vec);
      Array1<int32_t> row_splits2(context, row_splits2_vec);
      RaggedShape src_shape =
          RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);
      ASSERT_EQ(src_shape.Dim0(), 3);
      ASSERT_EQ(src_shape.TotSize(1), 6);
      RaggedShape shape = Transpose(src_shape);
      EXPECT_EQ(shape.Dim0(), 2);
      ASSERT_EQ(shape.TotSize(1), 6);
      const std::vector<int32_t> expected_row_splits = {0, 3, 6};
      const std::vector<int32_t> expected_row_ids = {0, 0, 0, 1, 1, 1};
      CheckArrayData(shape.RowSplits(1), expected_row_splits);
      CheckArrayData(shape.RowIds(1), expected_row_ids);
      CheckArrayData(shape.RowSplits(2), {0, 3, 6, 8, 9, 10, 12});
      CheckArrayData(shape.RowIds(2), {0, 0, 0, 1, 1, 1, 2, 2, 3, 4, 5, 5});
    }

    {
      // random case
      for (int32_t j = 0; j != 2; ++j) {
        RaggedShape to_transpose = RandomRaggedShapeToTranspose(context);
        RaggedShape transposed = Transpose(to_transpose);

        if (context->GetDeviceType() != kCpu) {
          to_transpose = to_transpose.To(cpu);
          transposed = transposed.To(cpu);
        }

        for (auto iter = transposed.Iterator(); !iter.Done(); iter.Next()) {
          std::vector<int32_t> index = iter.Value();
          int32_t i = transposed[index];  // Just make sure this doesn't crash,
                                          // don't need the value.
          std::swap(index[0], index[1]);
          i = to_transpose[index];  // don't need the value, just need to make
                                    // sure it's an allowable index.
          ++i;  // this line just suppresses the warning `variable i set but not
                // used`
        }
        for (auto iter = to_transpose.Iterator(); !iter.Done(); iter.Next()) {
          std::vector<int32_t> index = iter.Value();
          std::swap(index[0], index[1]);
          int32_t i = transposed[index];  // don't need the value, just need to
                                          // make sure it's an allowable index.
        }
      }
    }
  }
}

template <typename T>
void TestTransposeRagged() {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      const std::vector<int32_t> row_splits1_vec = {0, 2, 4, 6};
      const std::vector<int32_t> row_splits2_vec = {0, 3, 4, 7, 8, 10, 12};
      Array1<int32_t> row_splits1(context, row_splits1_vec);
      Array1<int32_t> row_splits2(context, row_splits2_vec);
      RaggedShape src_shape =
          RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);
      ASSERT_EQ(src_shape.Dim0(), 3);
      ASSERT_EQ(src_shape.TotSize(1), 6);
      std::vector<T> values = {0, 1, 2, 3, 4, 5, 8, 7, 6, 9, 10, 15};
      ASSERT_EQ(values.size(), src_shape.NumElements());
      Array1<T> values_array(context, values);
      Ragged<T> ragged(src_shape, values_array);
      Ragged<T> ans = Transpose(ragged);
      RaggedShape shape = ans.shape;
      // Check shape
      ASSERT_EQ(shape.Dim0(), 2);
      ASSERT_EQ(shape.TotSize(1), 6);
      const std::vector<int32_t> expected_row_splits = {0, 3, 6};
      const std::vector<int32_t> expected_row_ids = {0, 0, 0, 1, 1, 1};
      CheckArrayData(shape.RowSplits(1), expected_row_splits);
      CheckArrayData(shape.RowIds(1), expected_row_ids);
      CheckArrayData(shape.RowSplits(2), {0, 3, 6, 8, 9, 10, 12});
      CheckArrayData(shape.RowIds(2), {0, 0, 0, 1, 1, 1, 2, 2, 3, 4, 5, 5});
      // Check values
      CheckArrayData(ans.values, {0, 1, 2, 4, 5, 8, 6, 9, 3, 7, 10, 15});
    }

    {
      // random case
      for (int32_t j = 0; j != 2; ++j) {
        RaggedShape to_transpose = RandomRaggedShapeToTranspose(context);
        int32_t num_elems = to_transpose.NumElements();
        Array1<T> src_values =
            RandUniformArray1<T>(context, num_elems, 0, 10000);
        Ragged<T> src(to_transpose, src_values);
        Ragged<T> ans = Transpose(src);
        if (context->GetDeviceType() == kCuda) {
          src = src.To(cpu);
          ans = ans.To(cpu);
          to_transpose = to_transpose.To(cpu);
        }
        RaggedShape transposed = ans.shape;

        for (auto iter = transposed.Iterator(); !iter.Done(); iter.Next()) {
          std::vector<int32_t> index = iter.Value();
          T value = ans[index];
          std::swap(index[0], index[1]);
          EXPECT_EQ(value, src[index]);
        }
        for (auto iter = to_transpose.Iterator(); !iter.Done(); iter.Next()) {
          std::vector<int32_t> index = iter.Value();
          T value = src[index];
          std::swap(index[0], index[1]);
          EXPECT_EQ(value, ans[index]);
        }
      }
    }
  }
}
TEST(RaggedTest, TestTransposeRagged) {
  TestTransposeRagged<int32_t>();
  TestTransposeRagged<double>();
}

void TestRaggedShape2(const RaggedShape &shape) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    RaggedShape src_shape = shape.To(context);
    src_shape.Populate();
    ASSERT_GE(src_shape.NumAxes(), 2);
    Array1<int32_t> row_splits = src_shape.RowSplits(1);
    Array1<int32_t> row_ids = src_shape.RowIds(1);
    int32_t cached_tot_size = src_shape.TotSize(1);

    {
      // both row_splits and row_ids are non-null
      RaggedShape result = RaggedShape2(&row_splits, &row_ids, cached_tot_size);
      CheckArrayData(result.RowSplits(1), row_splits);
      CheckArrayData(result.RowIds(1), row_ids);
      EXPECT_EQ(result.TotSize(1), cached_tot_size);
    }
    {
      // both row_splits and row_ids are non-null, cached_tot_size = -1
      RaggedShape result = RaggedShape2(&row_splits, &row_ids, -1);
      CheckArrayData(result.RowSplits(1), row_splits);
      CheckArrayData(result.RowIds(1), row_ids);
      EXPECT_EQ(result.TotSize(1), cached_tot_size);
    }
    {
      // row_ids is null
      RaggedShape result = RaggedShape2(&row_splits, nullptr, cached_tot_size);
      CheckArrayData(result.RowSplits(1), row_splits);
      CheckArrayData(result.RowIds(1), row_ids);
      EXPECT_EQ(result.TotSize(1), cached_tot_size);
    }
    {
      // row_ids is null, cached_tot_size = -1
      RaggedShape result = RaggedShape2(&row_splits, nullptr, -1);
      CheckArrayData(result.RowSplits(1), row_splits);
      CheckArrayData(result.RowIds(1), row_ids);
      EXPECT_EQ(result.TotSize(1), cached_tot_size);
    }

    // note if row_splits == null, then we suppose there's no empty rows after
    // the last row-id in row_ids
    if (row_splits.Dim() == (row_ids.Dim() == 0 ? 1 : row_ids.Back() + 2)) {
      {
        // row_splits is null
        RaggedShape result = RaggedShape2(nullptr, &row_ids, cached_tot_size);
        CheckArrayData(result.RowSplits(1), row_splits);
        CheckArrayData(result.RowIds(1), row_ids);
        EXPECT_EQ(result.TotSize(1), cached_tot_size);
      }
      {
        // row_splits is null, cached_tot_size = -1
        RaggedShape result = RaggedShape2(nullptr, &row_ids, -1);
        CheckArrayData(result.RowSplits(1), row_splits);
        CheckArrayData(result.RowIds(1), row_ids);
        EXPECT_EQ(result.TotSize(1), cached_tot_size);
      }
    }
  }
}

TEST_F(RaggedShapeOpsSuiteTest, TestRaggedShape2) {
  TestRaggedShape2(simple_shape_);
  TestRaggedShape2(random_shape_);
}

void TestRaggedShape3(const RaggedShape &shape) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    RaggedShape src_shape = shape.To(context);
    src_shape.Populate();
    ASSERT_GE(src_shape.NumAxes(), 3);
    Array1<int32_t> row_splits1 = src_shape.RowSplits(1);
    Array1<int32_t> row_ids1 = src_shape.RowIds(1);
    int32_t cached_tot_size1 = src_shape.TotSize(1);
    Array1<int32_t> row_splits2 = src_shape.RowSplits(2);
    Array1<int32_t> row_ids2 = src_shape.RowIds(2);
    int32_t cached_tot_size2 = src_shape.TotSize(2);

    {
      // both row_splits and row_ids are non-null
      RaggedShape result =
          RaggedShape3(&row_splits1, &row_ids1, cached_tot_size1, &row_splits2,
                       &row_ids2, cached_tot_size2);
      CheckArrayData(result.RowSplits(1), row_splits1);
      CheckArrayData(result.RowIds(1), row_ids1);
      EXPECT_EQ(result.TotSize(1), cached_tot_size1);
      CheckArrayData(result.RowSplits(2), row_splits2);
      CheckArrayData(result.RowIds(2), row_ids2);
      EXPECT_EQ(result.TotSize(2), cached_tot_size2);
    }
    {
      // row_ids is non-null, cached_tot_size = -1
      RaggedShape result =
          RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);
      CheckArrayData(result.RowSplits(1), row_splits1);
      CheckArrayData(result.RowIds(1), row_ids1);
      EXPECT_EQ(result.TotSize(1), cached_tot_size1);
      CheckArrayData(result.RowSplits(2), row_splits2);
      CheckArrayData(result.RowIds(2), row_ids2);
      EXPECT_EQ(result.TotSize(2), cached_tot_size2);
    }

    // note if row_splits == null, then we suppose there's no empty rows after
    // the last row-id in row_ids
    bool valid1 =
        (row_splits1.Dim() == (row_ids1.Dim() == 0 ? 1 : row_ids1.Back() + 2));
    bool valid2 =
        (row_splits2.Dim() == (row_ids2.Dim() == 0 ? 1 : row_ids2.Back() + 2));
    if (valid1 && valid2) {
      RaggedShape result =
          RaggedShape3(nullptr, &row_ids1, -1, nullptr, &row_ids2, -1);
      CheckArrayData(result.RowSplits(1), row_splits1);
      CheckArrayData(result.RowIds(1), row_ids1);
      EXPECT_EQ(result.TotSize(1), cached_tot_size1);
      CheckArrayData(result.RowSplits(2), row_splits2);
      CheckArrayData(result.RowIds(2), row_ids2);
      EXPECT_EQ(result.TotSize(2), cached_tot_size2);
    }
    // TODO(haowen): add more cases for other branches
  }
}
TEST_F(RaggedShapeOpsSuiteTest, TestRaggedShape3) {
  TestRaggedShape3(simple_shape_);
  TestRaggedShape3(random_shape_);
}

void TestComposeShape(const RaggedShape &shape) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    RaggedShape src_shape = shape.To(context);
    ASSERT_GE(src_shape.NumAxes(), 3);
    Array1<int32_t> row_splits1 = src_shape.RowSplits(1);
    Array1<int32_t> row_ids1 = src_shape.RowIds(1);
    Array1<int32_t> row_splits2 = src_shape.RowSplits(2);
    Array1<int32_t> row_ids2 = src_shape.RowIds(2);

    RaggedShape shape1 = RaggedShape2(&row_splits1, nullptr, -1);
    RaggedShape shape2 = RaggedShape2(&row_splits2, nullptr, -1);

    RaggedShape result = ComposeRaggedShapes(shape1, shape2);

    ASSERT_EQ(result.NumAxes(), 3);

    CheckArrayData(result.RowSplits(1), row_splits1);
    CheckArrayData(result.RowIds(1), row_ids1);
    CheckArrayData(result.RowSplits(2), row_splits2);
    CheckArrayData(result.RowIds(2), row_ids2);
  }
}
TEST_F(RaggedShapeOpsSuiteTest, TestComposeShape) {
  TestComposeShape(simple_shape_);
  TestComposeShape(random_shape_);
}

void TestShapeFromTotSize(const RaggedShape &shape) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    RaggedShape src_shape = shape.To(context);
    ASSERT_GE(src_shape.NumAxes(), 2);

    int32_t num_axes = src_shape.NumAxes();
    std::vector<int32_t> tot_sizes(num_axes);
    for (int32_t i = 0; i != num_axes; ++i) {
      tot_sizes[i] = src_shape.TotSize(i);
    }

    RaggedShape result =
        RaggedShapeFromTotSizes(context, num_axes, tot_sizes.data());

    ASSERT_EQ(result.NumAxes(), num_axes);
    for (int32_t i = 0; i < num_axes; ++i) {
      EXPECT_EQ(result.TotSize(i), src_shape.TotSize(i));
      if (i > 0) {
        EXPECT_EQ(result.RowSplits(i).Dim(), src_shape.RowSplits(i).Dim());
        EXPECT_EQ(result.RowIds(i).Dim(), src_shape.RowIds(i).Dim());
      }
    }
  }
}
TEST_F(RaggedShapeOpsSuiteTest, TestShapeFromTotSize) {
  TestShapeFromTotSize(simple_shape_);
  TestShapeFromTotSize(random_shape_);
}

template <typename T>
void TestRagged() {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // constructed with row_splits and row_ids
      // RaggedTensor4 t = [
      //  [ [[ 1, 2], [4]],  [[3, 0]] ],
      //  [ [[7, 8, 9]], [[6], [3, 5, 7]], [[2]] ],
      //  [ [[3, 4], [], [8]] ]
      // ]
      const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
      const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
      const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
      const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};
      const std::vector<int32_t> row_splits3 = {0,  2,  3,  5,  8, 9,
                                                12, 13, 15, 15, 16};
      const std::vector<int32_t> row_ids3 = {0, 0, 1, 2, 2, 3, 3, 3,
                                             4, 5, 5, 5, 6, 7, 7, 9};
      const std::vector<T> values_vec = {1, 2, 4, 3, 0, 7, 8, 9,
                                         6, 3, 5, 7, 2, 3, 4, 8};
      std::vector<RaggedShapeLayer> axes;
      axes.emplace_back(
          RaggedShapeLayer{Array1<int32_t>(context, row_splits1),
                           Array1<int32_t>(context, row_ids1),
                           static_cast<int32_t>(row_ids1.size())});
      axes.emplace_back(
          RaggedShapeLayer{Array1<int32_t>(context, row_splits2),
                           Array1<int32_t>(context, row_ids2),
                           static_cast<int32_t>(row_ids2.size())});
      axes.emplace_back(
          RaggedShapeLayer{Array1<int32_t>(context, row_splits3),
                           Array1<int32_t>(context, row_ids3),
                           static_cast<int32_t>(row_ids3.size())});

      RaggedShape shape(axes, true);
      Array1<T> values(context, values_vec);
      Ragged<T> ragged(shape, values);

      // test Index(axis, i)
      {
        // values: [[[ 1, 2], [4]], [[3, 0]]]
        Ragged<T> sub_raggged = ragged.Index(0, 0);
        RaggedShape &sub_shape = sub_raggged.shape;
        EXPECT_EQ(sub_shape.NumAxes(), 3);
        const std::vector<std::vector<int32_t>> sub_row_splits_vec = {
            {0, 2, 3}, {0, 2, 3, 5}};
        CheckRowSplits(sub_shape, sub_row_splits_vec);
        const Array1<T> &sub_values = sub_raggged.values;
        const std::vector<T> sub_values_vec = {1, 2, 4, 3, 0};
        CheckArrayData<T>(sub_values, sub_values_vec);
      }
      {
        // values: [[[7, 8, 9]], [[6], [3, 5, 7]], [[2]]]
        Ragged<T> sub_raggged = ragged.Index(0, 1);
        RaggedShape &sub_shape = sub_raggged.shape;
        EXPECT_EQ(sub_shape.NumAxes(), 3);
        const std::vector<std::vector<int32_t>> sub_row_splits_vec = {
            {0, 1, 3, 4}, {0, 3, 4, 7, 8}};
        CheckRowSplits(sub_shape, sub_row_splits_vec);
        const Array1<T> &sub_values = sub_raggged.values;
        const std::vector<T> sub_values_vec = {7, 8, 9, 6, 3, 5, 7, 2};
        CheckArrayData<T>(sub_values, sub_values_vec);
      }
      {
        // values: [[[3, 4], [], [8]]]
        Ragged<T> sub_raggged = ragged.Index(0, 2);
        RaggedShape &sub_shape = sub_raggged.shape;
        EXPECT_EQ(sub_shape.NumAxes(), 3);
        const std::vector<std::vector<int32_t>> sub_row_splits_vec = {
            {0, 3}, {0, 2, 2, 3}};
        CheckRowSplits(sub_shape, sub_row_splits_vec);
        const Array1<T> &sub_values = sub_raggged.values;
        const std::vector<T> sub_values_vec = {3, 4, 8};
        CheckArrayData<T>(sub_values, sub_values_vec);
      }

      // test operator[](const std::vector<int32_t> &indexes)
      if (context->GetDeviceType() == kCpu) {
        {
          std::vector<int32_t> indexes = {0, 0, 0, 0};
          EXPECT_EQ(ragged.shape[indexes], 0);
          EXPECT_EQ(ragged[indexes], 1);
        }
        {
          std::vector<int32_t> indexes = {0, 1, 0, 0};
          EXPECT_EQ(ragged.shape[indexes], 3);
          EXPECT_EQ(ragged[indexes], 3);
        }
        {
          std::vector<int32_t> indexes = {1, 0, 0, 1};
          EXPECT_EQ(ragged.shape[indexes], 6);
          EXPECT_EQ(ragged[indexes], 8);
        }
        {
          std::vector<int32_t> indexes = {1, 1, 1, 0};
          EXPECT_EQ(ragged.shape[indexes], 9);
          EXPECT_EQ(ragged[indexes], 3);
        }
        {
          std::vector<int32_t> indexes = {2, 0, 0, 1};
          EXPECT_EQ(ragged.shape[indexes], 14);
          EXPECT_EQ(ragged[indexes], 4);
        }
        {
          std::vector<int32_t> indexes = {2, 0, 2, 0};
          EXPECT_EQ(ragged.shape[indexes], 15);
          EXPECT_EQ(ragged[indexes], 8);
        }
      }

      const std::vector<std::vector<int32_t>> row_splits_vec = {
          row_splits1, row_splits2, row_splits3};
      // test To(ctx)
      {
        // to GPU
        Ragged<T> other = ragged.To(GetCudaContext());
        CheckRowSplits(other.shape, row_splits_vec);
        CheckArrayData<T>(other.values, values_vec);
      }
      {
        // to CPU
        Ragged<T> other = ragged.To(GetCpuContext());
        CheckRowSplits(other.shape, row_splits_vec);
        CheckArrayData<T>(other.values, values_vec);
      }
    }
  }
}

template <typename T, typename OP = LessThan<T>>
static void CpuSortSublists(const Array1<int32_t> &row_splits, Array1<T> *src) {
  K2_CHECK(src->Context()->GetDeviceType() == kCpu);
  T *p = src->Data();
  OP comp = OP();
  for (int32_t i = 0; i < row_splits.Dim() - 1; ++i) {
    int32_t cur = row_splits[i];
    int32_t next = row_splits[i + 1];
    std::sort(p + cur, p + next, comp);
  }
}

template <typename T, typename OP = LessThan<T>>
static void TestSortSublists() {
  auto cpu_context = GetCpuContext();
  auto cuda_context = GetCudaContext();

  RaggedShape shape = RandomRaggedShape(false,  // set_row_ids
                                        2,      // min_num_axes
                                        4,      // max_num_axes
                                        1,      // min_num_elements
                                        2000);  // max_num_elements

  Array1<T> values =
      RandUniformArray1<T>(shape.Context(), shape.NumElements(), -2000, 2000);
  Ragged<T> ragged(shape, values);
  ragged = ragged.To(cuda_context);
  values = values.To(cpu_context);  // to be sorted by cpu

  Array1<T> unsorted = values.Clone();

  Array1<int32_t> order(ragged.Context(), ragged.values.Dim());
  SortSublists<T, OP>(&ragged, &order);

  Array1<int32_t> &segment = ragged.shape.RowSplits(ragged.NumAxes() - 1);
  CpuSortSublists<T, OP>(segment, &values);

  int32_t n = order.Dim();
  for (int i = 0; i != n; ++i) {
    EXPECT_EQ(values[i], ragged.values[i]);
    EXPECT_EQ(ragged.values[i], unsorted[order[i]]);
  }
}

TEST(RaggedTest, Ragged) {
  TestRagged<int32_t>();
  TestRagged<double>();

  TestSortSublists<int32_t>();
  TestSortSublists<double>();
}

TEST(RaggedShapeOpsTest, TestCat) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // simple case
      std::vector<RaggedShape> shapes(2);
      std::vector<RaggedShape *> shapes_ptr(2);
      std::vector<std::vector<Array1<int32_t>>> row_splits_vec(2);
      {
        const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
        const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
        const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
        const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};
        Array1<int32_t> splits1(context, row_splits1);
        Array1<int32_t> ids1(context, row_ids1);
        Array1<int32_t> splits2(context, row_splits2);
        Array1<int32_t> ids2(context, row_ids2);
        row_splits_vec[0].push_back(splits1);
        row_splits_vec[1].push_back(splits2);
        shapes[0] = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2, &ids2,
                                 ids2.Dim());
        shapes_ptr[0] = &shapes[0];
      }
      {
        const std::vector<int32_t> row_splits1 = {0, 1, 3, 4};
        const std::vector<int32_t> row_ids1 = {0, 1, 1, 2};
        const std::vector<int32_t> row_splits2 = {0, 3, 4, 5, 7};
        const std::vector<int32_t> row_ids2 = {0, 0, 0, 1, 2, 3, 3};
        Array1<int32_t> splits1(context, row_splits1);
        Array1<int32_t> ids1(context, row_ids1);
        Array1<int32_t> splits2(context, row_splits2);
        Array1<int32_t> ids2(context, row_ids2);
        row_splits_vec[0].push_back(splits1);
        row_splits_vec[1].push_back(splits2);
        RaggedShape shape = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2,
                                         &ids2, ids2.Dim());
        shapes[1] = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2, &ids2,
                                 ids2.Dim());
        shapes_ptr[1] = &shapes[1];
      }

      {
        // axis == 1
        RaggedShape result = Cat(1, 2, shapes_ptr.data());
        std::vector<std::vector<int32_t>> expected_row_splits = {
            {0, 3, 8, 10}, {0, 2, 3, 6, 7, 9, 10, 11, 12, 15, 17}};
        std::vector<std::vector<int32_t>> expected_row_ids = {
            {0, 0, 0, 1, 1, 1, 1, 1, 2, 2},
            {0, 0, 1, 2, 2, 2, 3, 4, 4, 5, 6, 7, 8, 8, 8, 9, 9}};
        for (int32_t i = 0; i < 2; ++i) {
          CheckArrayData(result.RowSplits(i + 1), expected_row_splits[i]);
          CheckArrayData(result.RowIds(i + 1), expected_row_ids[i]);
        }
      }

      {
        // axis == 0
        RaggedShape result = Cat(0, 2, shapes_ptr.data());

        // get result splits with `SpliceRowSplits` and get result row-ids with
        // `RowSplitsToRowIds``
        std::vector<Array1<int32_t>> result_splits;
        std::vector<Array1<int32_t>> result_ids;
        for (auto i = 0; i < 2; ++i) {
          std::vector<const Array1<int32_t> *> splits_ptr = {
              &row_splits_vec[i][0], &row_splits_vec[i][1]};
          Array1<int32_t> curr_row_splits =
              SpliceRowSplits(2, splits_ptr.data());
          result_splits.push_back(curr_row_splits);
          Array1<int32_t> curr_row_ids(context, curr_row_splits.Back());
          RowSplitsToRowIds(curr_row_splits, &curr_row_ids);
          result_ids.push_back(curr_row_ids);
        }
        for (int32_t i = 0; i < 2; ++i) {
          CheckArrayData(result.RowSplits(i + 1), result_splits[i]);
          CheckArrayData(result.RowIds(i + 1), result_ids[i]);
        }
      }
    }

    {
      // test with random large size
      for (int32_t i = 0; i < 2; ++i) {
        int32_t num_shape = RandInt(2, 100);
        int32_t num_axes = RandInt(2, 4);
        std::vector<RaggedShape> shape_vec(num_shape);
        std::vector<RaggedShape *> shapes(num_shape);
        for (int32_t j = 0; j != num_shape; ++j) {
          shape_vec[j] =
              RandomRaggedShape(true, num_axes, num_axes, 0, 1000).To(context);
          shapes[j] = &shape_vec[j];
        }
        // only test case axis == 0, test axis==1 with simple case is good
        // enough as it just calls Stack
        RaggedShape result = Cat(0, num_shape, shapes.data());
        ASSERT_EQ(result.NumAxes(), num_axes);

        // get result splits with `SpliceRowSplits` and get result row-ids with
        // `RowSplitsToRowIds``
        std::vector<Array1<int32_t>> result_splits;
        std::vector<Array1<int32_t>> result_ids;
        for (int32_t axis = 1; axis < num_axes; ++axis) {
          std::vector<Array1<int32_t>> splits_vec(num_shape);
          std::vector<const Array1<int32_t> *> splits_vec_ptr(num_shape);
          for (int32_t n = 0; n != num_shape; ++n) {
            splits_vec[n] = shape_vec[n].RowSplits(axis);
            splits_vec_ptr[n] = &splits_vec[n];
          }
          Array1<int32_t> curr_row_splits =
              SpliceRowSplits(num_shape, splits_vec_ptr.data());
          result_splits.push_back(curr_row_splits);
          Array1<int32_t> curr_row_ids(context, curr_row_splits.Back());
          RowSplitsToRowIds(curr_row_splits, &curr_row_ids);
          result_ids.push_back(curr_row_ids);
        }

        // check data
        for (int32_t axis = 1; axis < num_axes; ++axis) {
          CheckArrayData(result.RowSplits(axis), result_splits[axis - 1]);
          CheckArrayData(result.RowIds(axis), result_ids[axis - 1]);
        }
      }
    }
  }
}

template <typename T>
void TestCatRagged() {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    // TODO(haowen): remove duplicate code in TestCat above.
    // test with simple case could be good enough, as we have tested
    // Cat(RaggedShape&) already.
    std::vector<Ragged<T>> ragged_vec(2);
    std::vector<Ragged<T> *> ragged(2);
    std::vector<std::vector<Array1<int32_t>>> row_splits_vec(2);
    {
      const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
      const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
      const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
      const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};
      const std::vector<T> values_vec = {1, 2, 5, 7, 9, 10, 12, 14, 15, 18};
      Array1<int32_t> splits1(context, row_splits1);
      Array1<int32_t> ids1(context, row_ids1);
      Array1<int32_t> splits2(context, row_splits2);
      Array1<int32_t> ids2(context, row_ids2);
      RaggedShape shape = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2,
                                       &ids2, ids2.Dim());
      Array1<T> values(context, values_vec);
      ragged_vec[0] = Ragged<T>(shape, values);
      ragged[0] = &ragged_vec[0];
    }

    {
      const std::vector<int32_t> row_splits1 = {0, 1, 3, 4};
      const std::vector<int32_t> row_ids1 = {0, 1, 1, 2};
      const std::vector<int32_t> row_splits2 = {0, 3, 4, 5, 7};
      const std::vector<int32_t> row_ids2 = {0, 0, 0, 1, 2, 3, 3};
      const std::vector<T> values_vec = {20, 21, 23, 28, 30, 32, 35};
      Array1<int32_t> splits1(context, row_splits1);
      Array1<int32_t> ids1(context, row_ids1);
      Array1<int32_t> splits2(context, row_splits2);
      Array1<int32_t> ids2(context, row_ids2);
      RaggedShape shape = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2,
                                       &ids2, ids2.Dim());
      Array1<T> values(context, values_vec);
      ragged_vec[1] = Ragged<T>(shape, values);
      ragged[1] = &ragged_vec[1];
    }

    {
      // axis == 0
      Ragged<T> result = Cat(0, 2, ragged.data());
      std::vector<std::vector<int32_t>> expected_row_splits = {
          {0, 2, 5, 6, 7, 9, 10}, {0, 2, 3, 4, 6, 7, 10, 13, 14, 15, 17}};
      std::vector<std::vector<int32_t>> expected_row_ids = {
          {0, 0, 1, 1, 1, 2, 3, 4, 4, 5},
          {0, 0, 1, 2, 3, 3, 4, 5, 5, 5, 6, 6, 6, 7, 8, 9, 9}};
      for (int32_t i = 0; i < 2; ++i) {
        CheckArrayData(result.RowSplits(i + 1), expected_row_splits[i]);
        CheckArrayData(result.RowIds(i + 1), expected_row_ids[i]);
      }
      std::vector<T> expected_data = {1,  2,  5,  7,  9,  10, 12, 14, 15,
                                      18, 20, 21, 23, 28, 30, 32, 35};
      CheckArrayData(result.values, expected_data);
    }

    {
      // axis == 1
      Ragged<T> result = Cat(1, 2, ragged.data());
      std::vector<std::vector<int32_t>> expected_row_splits = {
          {0, 3, 8, 10}, {0, 2, 3, 6, 7, 9, 10, 11, 12, 15, 17}};
      std::vector<std::vector<int32_t>> expected_row_ids = {
          {0, 0, 0, 1, 1, 1, 1, 1, 2, 2},
          {0, 0, 1, 2, 2, 2, 3, 4, 4, 5, 6, 7, 8, 8, 8, 9, 9}};
      for (int32_t i = 0; i < 2; ++i) {
        CheckArrayData(result.RowSplits(i + 1), expected_row_splits[i]);
        CheckArrayData(result.RowIds(i + 1), expected_row_ids[i]);
      }
      std::vector<T> expected_data = {1,  2,  5,  20, 21, 23, 7,  9, 10,
                                      12, 28, 30, 14, 15, 18, 32, 35};
      CheckArrayData(result.values, expected_data);
    }
  }
}
TEST(RaggedTest, TestCatRagged) {
  TestCatRagged<int32_t>();
  TestCatRagged<double>();
}

void CheckResultOfIndex(const ContextPtr &context, RaggedShape shape,
                        Array1<int32_t> new2old, RaggedShape result) {
  K2_CHECK(context->IsCompatible(*shape.Context()));
  ContextPtr cpu = GetCpuContext();  // will use to copy data
  int32_t num_axes = shape.NumAxes();
  int32_t src_dim0 = shape.Dim0(), result_dim0 = result.Dim0();
  EXPECT_EQ(result_dim0, new2old.Dim());

  result.Check();

  for (int32_t i = 0; i < result_dim0; i++) {
    RaggedShape result_part = Arange(result, 0, i, i + 1);
    if (new2old[i] == -1) {
      K2_CHECK_EQ(0, result_part.TotSize(1));
    } else {
      RaggedShape src_part = Arange(shape, 0, new2old[i], new2old[i] + 1);
      K2_CHECK_EQ(true, Equal(src_part, result_part));
    }
  }
}

TEST(RaggedShapeOpsTest, TestIndex) {
  for (int i = 0; i < 5; i++) {
    ContextPtr cpu = GetCpuContext();  // will be used to copy data
    for (auto &context : {GetCpuContext(), GetCudaContext()}) {
      {
        // simple case
        const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
        const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2};
        const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
        const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};

        Array1<int32_t> splits1(context, row_splits1);
        Array1<int32_t> ids1(context, row_ids1);
        Array1<int32_t> splits2(context, row_splits2);
        Array1<int32_t> ids2(context, row_ids2);
        RaggedShape shape = RaggedShape3(&splits1, &ids1, ids1.Dim(), &splits2,
                                         &ids2, ids2.Dim());

        std::vector<int32_t> new2old_vec = {2, 1};
        Array1<int32_t> new2old(context, new2old_vec);
        Array1<int32_t> value_indexes_out;
        RaggedShape result = Index(shape, 0, new2old, &value_indexes_out);
        // fsa 2, state_idx01 {5}, arc_idx012 {7, 8, 9}
        // fsa 1, state_idx01 {2, 3, 4}, arc_idx012 {{3},{4, 5}, {6}}
        CheckArrayData(value_indexes_out,
                       std::vector<int32_t>{7, 8, 9, 3, 4, 5, 6});
        CheckResultOfIndex(context, shape, new2old, result);
      }

      {
        // test with random large size
        for (int32_t i = 0; i < 2; ++i) {
          int32_t num_axes = RandInt(2, 4);
          RaggedShape shape =
              RandomRaggedShape(true, num_axes, num_axes, 0, 1000).To(context);
          int32_t dim0 = shape.Dim0(), result_dim0 = RandInt(0, 10);
          if (dim0 == 0) result_dim0 = 0;
          std::vector<int32_t> new2old_vec(result_dim0);
          for (int i = 0; i < result_dim0; i++)
            new2old_vec[i] = RandInt(-1, dim0 - 1);
          Array1<int32_t> new2old(context, new2old_vec);
          Array1<int32_t> value_indexes;
          RaggedShape result = Index(shape, 0, new2old, &value_indexes);
          CheckResultOfIndex(context, shape, new2old, result);
          K2_LOG(INFO) << "Value_indexes = " << value_indexes;
        }
      }
    }
  }
}


TEST(RaggedShapeOpsTest, TestIndexAxis1) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      Ragged<int32_t> input = Ragged<int32_t>(" [ [ 1 2 ] [ 3 4 5 ] [ 6 7 ] [ ] ]").To(context);  // NOLINT
      Array1<int32_t> indexes = Array1<int32_t>(" [ 1 0 4 2 6 5 ]").To(context);
      Ragged<int32_t> output = Ragged<int32_t>(" [ [ 2 1 ] [ 5 3 ] [ 7 6 ] [ ] ]").To(context);  // NOLINT

      Ragged<int32_t> indexed = Index(input, 1, indexes);
      EXPECT_EQ(Equal(output, indexed), true);
    }
  }
}



TEST(GetTransposeReordering, NoDuplicates) {
  //       col0  col1  col2  col3  col4  col5
  // row0                           a0    b1
  // row1   c2    d3                      e4
  // row2                     f5
  // row3   g6          h7          i8
  // row4                                 j9
  // row5         k10               l11
  std::vector<int32_t> col_indexes{4, 5, 0, 1, 5, 3, 0, 2, 4, 5, 1, 4};
  std::vector<int32_t> _row_splits{0, 2, 5, 6, 9, 10, 12};
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    Array1<int32_t> row_splits(context, _row_splits);
    RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
    Array1<int32_t> values(context, col_indexes);

    Ragged<int32_t> ragged(shape, values);
    Array1<int32_t> order = GetTransposeReordering(ragged, 6);
    CheckArrayData(order, {2, 6, 3, 10, 7, 5, 0, 8, 11, 1, 4, 9});
    EXPECT_TRUE(context->IsCompatible(*order.Context()));
  }
}

TEST(GetTransposeReordering, ThreeAxesEmptyCase) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    Ragged<int32_t> ragged("[ [ [ ] ] ]");
    ragged = ragged.To(context);
    Array1<int32_t> order = GetTransposeReordering(ragged, 0);
  }
}

TEST(GetTransposeReordering, NoDuplicatesThreeAxes) {
  //       col0  col1  col2  col3  col4  col5
  // row0         a0          b1
  // row1   c2          d3
  // row2         e4
  // row3   f5    g6          h7
  // row4                                  i8
  // row5                            j9    k10
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    Array1<int32_t> col_indexes(
        context, std::vector<int32_t>{1, 3, 0, 2, 1, 0, 1, 3, 5, 4, 5});
    Array1<int32_t> row_splits1(context, std::vector<int32_t>{0, 4, 6});
    Array1<int32_t> row_splits2(context,
                                std::vector<int32_t>{0, 2, 4, 5, 8, 9, 11});
    RaggedShape shape =
        RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);
    Ragged<int32_t> ragged(shape, col_indexes);
    Array1<int32_t> order = GetTransposeReordering(ragged, 6);
    CheckArrayData(order, {2, 5, 0, 4, 6, 3, 1, 7, 9, 8, 10});
    EXPECT_TRUE(context->IsCompatible(*order.Context()));
  }
}

TEST(GetTransposeReordering, WithDuplicates) {
  //       col0   col1   col2    col3      col4      col5
  // row0         a0,a1         b2,b3,b4
  // row1  c5,c6          d7
  // row2         e8
  // row3   f9   g10,g11         h12
  // row4                                i13,i14,i15
  // row5                        j16                  k17
  std::vector<int32_t> col_indexes{1, 1, 3, 3, 3, 0, 0, 2, 1,
                                   0, 1, 1, 3, 4, 4, 4, 3, 5};
  std::vector<int32_t> _row_splits{0, 5, 8, 9, 13, 16, 18};
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    Array1<int32_t> row_splits(context, _row_splits);
    RaggedShape shape = RaggedShape2(&row_splits, nullptr, -1);
    Array1<int32_t> values(context, col_indexes);
    Ragged<int32_t> ragged(shape, values);
    Array1<int32_t> order = GetTransposeReordering(ragged, 6);
    CheckArrayData(
        order, {5, 6, 9, 0, 1, 8, 10, 11, 7, 2, 3, 4, 12, 16, 13, 14, 15, 17});
    EXPECT_TRUE(context->IsCompatible(*order.Context()));
  }
}

TEST(GetTransposeReordering, WithDuplicatesThreeAxes) {
  //       col0   col1   col2    col3      col4      col5
  // row0         a0,a1         b2,b3,b4
  // row1  c5,c6          d7
  // row2         e8
  // row3   f9   g10,g11         h12
  // row4                                i13,i14,i15
  // row5                                 j16         k17
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    Array1<int32_t> col_indexes(
        context, std::vector<int32_t>{1, 1, 3, 3, 3, 0, 0, 2, 1, 0, 1, 1, 3, 4,
                                      4, 4, 4, 5});
    Array1<int32_t> row_splits1(context, std::vector<int32_t>{0, 4, 6});
    Array1<int32_t> row_splits2(context,
                                std::vector<int32_t>{0, 5, 8, 9, 13, 16, 18});
    RaggedShape shape =
        RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);
    Ragged<int32_t> ragged(shape, col_indexes);
    Array1<int32_t> order = GetTransposeReordering(ragged, 6);
    CheckArrayData(
        order, {5, 6, 9, 0, 1, 8, 10, 11, 7, 2, 3, 4, 12, 13, 14, 15, 16, 17});
    EXPECT_TRUE(context->IsCompatible(*order.Context()));
  }
}

TEST(GetTransposeReordering, RandomFsaVecTest) {
  for (int32_t iter = 0; iter != 8; ++iter) {
    for (auto &context : {GetCpuContext(), GetCudaContext()}) {
      int n = RandInt(100, 200);
      int32_t min_num_fsas = n;
      int32_t max_num_fsas = n * 2;
      bool acyclic = false;
      int32_t max_symbol = 100;
      int32_t min_num_arcs = min_num_fsas * 10;
      int32_t max_num_arcs = max_num_fsas * 20;

      FsaVec fsas = RandomFsaVec(min_num_fsas, max_num_fsas, acyclic,
                                 max_symbol, min_num_arcs, max_num_arcs);
      fsas = fsas.To(context);
      Array1<int32_t> dest_states = GetDestStates(fsas, true);
      Ragged<int32_t> dest_states_tensor(fsas.shape, dest_states);
      int32_t num_states = fsas.TotSize(1);
      int32_t num_arcs = fsas.TotSize(2);
      Array1<int32_t> order =
          GetTransposeReordering(dest_states_tensor, num_states);
      Sort(&order);
      ASSERT_EQ(order.Dim(), num_arcs);
      Array1<int32_t> expected = Range<int32_t>(context, num_arcs, 0);
      CheckArrayData(order, expected);
    }
  }
}

TEST(ChangeSublistSize, TwoAxes) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    Array1<int32_t> row_splits1(context, std::vector<int32_t>{0, 2, 5});
    RaggedShape src = RaggedShape2(&row_splits1, nullptr, -1);

    int32_t size_delta = 2;
    RaggedShape dst = ChangeSublistSize(src, size_delta);
    CheckArrayData(dst.RowSplits(1), std::vector<int32_t>{0, 4, 9});

    size_delta = -2;
    dst = ChangeSublistSize(src, size_delta);
    CheckArrayData(dst.RowSplits(1), std::vector<int32_t>{0, 0, 1});

    size_delta = 0;
    dst = ChangeSublistSize(src, size_delta);
    CheckArrayData(dst.RowSplits(1), std::vector<int32_t>{0, 2, 5});
  }
}

TEST(ChangeSublistSizePinned, TwoAxes) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      Array1<int32_t> row_splits1(context, std::vector<int32_t>{0, 2, 5, 5});
      RaggedShape src = RaggedShape2(&row_splits1, nullptr, -1);

      int32_t size_delta = 2;
      RaggedShape dst = ChangeSublistSizePinned(src, size_delta);
      CheckArrayData(dst.RowSplits(1), std::vector<int32_t>{0, 4, 9, 9});

      size_delta = -3;
      dst = ChangeSublistSizePinned(src, size_delta);
      CheckArrayData(dst.RowSplits(1), std::vector<int32_t>{0, 0, 0, 0});

      size_delta = 0;
      dst = ChangeSublistSizePinned(src, size_delta);
      CheckArrayData(dst.RowSplits(1), std::vector<int32_t>{0, 2, 5, 5});
    }
  }
}

TEST(ChangeSublistSize, ThreeAxes) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    /*
     [
       [ [x, x, x], [x, x] ]
       [ [x], [x, x], [x, x, x] ]
     ]
     */
    Array1<int32_t> row_splits1(context, std::vector<int32_t>{0, 2, 5});
    Array1<int32_t> row_splits2(context,
                                std::vector<int32_t>{0, 3, 5, 6, 8, 11});
    RaggedShape src =
        RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);

    int32_t size_delta = 2;
    RaggedShape dst = ChangeSublistSize(src, size_delta);
    CheckArrayData(dst.RowSplits(2), std::vector<int32_t>{0, 5, 9, 12, 16, 21});

    // it is an error to use -2 here
    // because the state (state_idx01 == 2) has only 1 entry
    size_delta = -1;

    dst = ChangeSublistSize(src, size_delta);
    CheckArrayData(dst.RowSplits(2), std::vector<int32_t>{0, 2, 3, 3, 4, 6});

    size_delta = 0;
    dst = ChangeSublistSize(src, size_delta);
    CheckArrayData(dst.RowSplits(2), std::vector<int32_t>{0, 3, 5, 6, 8, 11});
  }
}

TEST(ChangeSublistSizePinned, ThreeAxes) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    /*
     [
       [ [x, x, x], [x, x] ]
       [ [x], [x, x], [], [x, x, x] ]
     ]
     */
    Array1<int32_t> row_splits1(context, std::vector<int32_t>{0, 2, 6});
    Array1<int32_t> row_splits2(context,
                                std::vector<int32_t>{0, 3, 5, 6, 8, 8, 11});
    RaggedShape src =
        RaggedShape3(&row_splits1, nullptr, -1, &row_splits2, nullptr, -1);

    int32_t size_delta = 2;
    RaggedShape dst = ChangeSublistSizePinned(src, size_delta);
    CheckArrayData(dst.RowSplits(2),
                   std::vector<int32_t>{0, 5, 9, 12, 16, 16, 21});

    size_delta = -2;

    dst = ChangeSublistSizePinned(src, size_delta);
    CheckArrayData(dst.RowSplits(2), std::vector<int32_t>{0, 1, 1, 1, 1, 1, 2});

    size_delta = 0;
    dst = ChangeSublistSizePinned(src, size_delta);
    CheckArrayData(dst.RowSplits(2),
                   std::vector<int32_t>{0, 3, 5, 6, 8, 8, 11});
  }
}

TEST(RaggedShapeOpsTest, TestGetCountsPartitioned) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    // Testing with simple case is good enough as we have tested GetCounts()
    // with random large size and GetCountsPartitioned just calls GetCounts.
    std::vector<int32_t> src_row_splits_vec = {0, 3, 4, 6, 10};
    Array1<int32_t> src_row_splits(context, src_row_splits_vec);
    RaggedShape src_shape = RaggedShape2(&src_row_splits, nullptr, -1);
    std::vector<int32_t> src_values_vec = {0, 1, 0, 2, 5, 5, 7, 7, 9, 7};
    Array1<int32_t> src_values(context, src_values_vec);
    Ragged<int32_t> src(src_shape, src_values);

    std::vector<int32_t> ans_row_splits_vec = {0, 2, 4, 7, 10};
    Array1<int32_t> ans_row_splits(context, ans_row_splits_vec);
    RaggedShape ans_shape = RaggedShape2(&ans_row_splits, nullptr, -1);

    Ragged<int32_t> result = GetCountsPartitioned(src, ans_shape);

    ASSERT_EQ(result.NumAxes(), 2);
    // Check row_splits
    Array1<int32_t> row_splits = result.shape.RowSplits(1).To(cpu);
    std::vector<int32_t> result_row_splits(
        row_splits.Data(), row_splits.Data() + row_splits.Dim());
    EXPECT_EQ(result_row_splits, ans_row_splits_vec);
    // check values
    std::vector<int32_t> expected_data = {2, 1, 1, 0, 0, 2, 0, 3, 0, 1};
    Array1<int32_t> values = result.values.To(cpu);
    std::vector<int32_t> data(values.Data(), values.Data() + values.Dim());
    EXPECT_EQ(data, expected_data);
  }
}

TEST(RaggedShapeOpsTest, TestStack) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // simple case
      std::vector<RaggedShape> shapes(2);
      std::vector<RaggedShape *> shapes_ptr(2);
      std::vector<std::vector<Array1<int32_t>>> row_splits_vec(2);
      {
        const std::vector<int32_t> row_splits1 = {0, 2, 5, 6};
        const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10};
        Array1<int32_t> splits1(context, row_splits1);
        Array1<int32_t> splits2(context, row_splits2);
        row_splits_vec[0].push_back(splits1);
        row_splits_vec[1].push_back(splits2);
        shapes[0] = RaggedShape3(&splits1, nullptr, -1, &splits2, nullptr, -1);
        shapes_ptr[0] = &shapes[0];
      }
      {
        const std::vector<int32_t> row_splits1 = {0, 1, 3, 4};
        const std::vector<int32_t> row_splits2 = {0, 3, 4, 5, 7};
        Array1<int32_t> splits1(context, row_splits1);
        Array1<int32_t> splits2(context, row_splits2);
        row_splits_vec[0].push_back(splits1);
        row_splits_vec[1].push_back(splits2);
        shapes[1] = RaggedShape3(&splits1, nullptr, -1, &splits2, nullptr, -1);
        shapes_ptr[1] = &shapes[1];
      }
      std::vector<std::vector<int32_t>> expected_row_splits = {
          {0, 3, 6},
          {0, 2, 5, 6, 7, 9, 10},
          {0, 2, 3, 4, 6, 7, 10, 13, 14, 15, 17}};

      {
        // axis == 0
        int32_t axis = 0;
        RaggedShape result = Stack(axis, 2, shapes_ptr.data());
        for (int32_t i = 0; i != 3; ++i) {
          CheckArrayData(result.RowSplits(i + 1), expected_row_splits[i]);
        }
        RaggedShape result2 = Stack(axis, 1, shapes_ptr.data());
        RaggedShape orig = result2.Index(0, 0);
        EXPECT_TRUE(Equal(orig, shapes[0]));
      }
      {
        // axis == 1
        int32_t axis = 1;
        RaggedShape result = Stack(axis, 2, shapes_ptr.data());
        RaggedShape transpose = Transpose(result);
        for (int32_t i = 0; i != 3; ++i) {
          CheckArrayData(transpose.RowSplits(i + 1), expected_row_splits[i]);
        }
      }
    }

    {
      // test with random large size
      for (int32_t m = 0; m < 2; ++m) {
        int32_t num_shape = RandInt(2, 100);
        int32_t num_axes = RandInt(2, 4);
        int32_t dim0 = RandInt(1, 100);
        std::vector<RaggedShape> shape_vec(num_shape);
        std::vector<RaggedShape *> shapes(num_shape);
        for (int32_t j = 0; j != num_shape; ++j) {
          RaggedShape shape =
              RandomRaggedShape(false, num_axes, num_axes, 0, 1000).To(context);
          int32_t src_dim0 = shape.Dim0();
          std::vector<int32_t> row_splits_vec(dim0 + 1);
          row_splits_vec[0] = 0;
          for (int32_t n = 1; n < dim0; ++n) {
            row_splits_vec[n] = RandInt(0, src_dim0);
          }
          row_splits_vec[dim0] = src_dim0;
          std::sort(row_splits_vec.begin(), row_splits_vec.end());
          Array1<int32_t> row_splits(context, row_splits_vec);
          RaggedShape first = RaggedShape2(&row_splits, nullptr, -1);
          RaggedShape new_shape = ComposeRaggedShapes(first, shape);
          shape_vec[j] = new_shape;
          shapes[j] = &shape_vec[j];
        }
        std::vector<RaggedShape> cpu_shapes(num_shape);
        for (auto i = 0; i != num_shape; ++i) {
          cpu_shapes[i] = shape_vec[i].To(cpu);
        }

        {
          // axis == 0
          int32_t axis = 0;
          RaggedShape result = Stack(axis, num_shape, shapes.data());
          ASSERT_EQ(result.NumAxes(),
                    num_axes + 2);  // note we append one axis in each shape in
                                    // `shapes` before `Stack`
          ASSERT_EQ(result.Dim0(), num_shape);
          result = result.To(cpu);
          for (auto iter = result.Iterator(); !iter.Done(); iter.Next()) {
            std::vector<int32_t> index = iter.Value();
            int32_t t = result[index];  // don't need the value, just make sure
                                        // it's a valid index.
            int32_t i = index[0];
            index.erase(index.begin());
            // result[i,j,k,l] = (shape[i])[j,k,l]
            i = cpu_shapes[i][index];  // don't need the value, just need to
                                       // make sure it's an allowable index.
          }
        }
        {
          // axis == 1
          int32_t axis = 1;
          RaggedShape result = Stack(axis, num_shape, shapes.data());
          ASSERT_EQ(result.NumAxes(),
                    num_axes + 2);  // note we append one axis in each shape in
                                    // `shapes` before `Stack`
          ASSERT_EQ(result.Dim0(), dim0);
          result = result.To(cpu);
          for (auto iter = result.Iterator(); !iter.Done(); iter.Next()) {
            std::vector<int32_t> index = iter.Value();
            int32_t t = result[index];  // don't need the value, just make sure
                                        // it's a valid index.
            int32_t i = index[1];
            index.erase(index.begin() + 1);
            // result[i,j,k,l] = (shape[j])[i,k,l]
            i = cpu_shapes[i][index];  // don't need the value, just need to
                                       // make sure it's an allowable index.
          }
        }
      }
    }
  }
}

template <typename T>
void TestStackRagged() {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    // test with random large size
    for (int32_t m = 0; m < 2; ++m) {
      int32_t num_shape = RandInt(2, 100);
      int32_t num_axes = RandInt(2, 4);
      int32_t dim0 = RandInt(1, 100);
      std::vector<Ragged<T>> ragged_vec(num_shape);
      std::vector<Ragged<T> *> ragged(num_shape);
      for (int32_t j = 0; j != num_shape; ++j) {
        RaggedShape shape =
            RandomRaggedShape(false, num_axes, num_axes, 0, 1000).To(context);
        int32_t src_dim0 = shape.Dim0();
        std::vector<int32_t> row_splits_vec(dim0 + 1);
        row_splits_vec[0] = 0;
        for (int32_t n = 1; n < dim0; ++n) {
          row_splits_vec[n] = RandInt(0, src_dim0);
        }
        row_splits_vec[dim0] = src_dim0;
        std::sort(row_splits_vec.begin(), row_splits_vec.end());
        Array1<int32_t> row_splits(context, row_splits_vec);
        RaggedShape first = RaggedShape2(&row_splits, nullptr, -1);
        RaggedShape new_shape = ComposeRaggedShapes(first, shape);
        int32_t num_elems = new_shape.NumElements();
        Array1<T> src_values =
            RandUniformArray1<T>(context, num_elems, 0, 10000);
        ragged_vec[j] = Ragged<T>(new_shape, src_values);
        ragged[j] = &ragged_vec[j];
      }
      std::vector<Ragged<T>> cpu_ragged_vec(num_shape);
      for (auto j = 0; j != num_shape; ++j) {
        cpu_ragged_vec[j] = ragged_vec[j].To(cpu);
      }

      {
        // axis == 0
        int32_t axis = 0;
        Ragged<T> result = Stack(axis, num_shape, ragged.data());
        ASSERT_EQ(result.NumAxes(),
                  num_axes + 2);  // note we append one axis in each shape in
                                  // `shapes` before `Stack`
        ASSERT_EQ(result.Dim0(), num_shape);
        result = result.To(cpu);
        RaggedShape &shape = result.shape;
        for (auto iter = shape.Iterator(); !iter.Done(); iter.Next()) {
          std::vector<int32_t> index = iter.Value();
          T value = result[index];
          int32_t i = index[0];
          index.erase(index.begin());
          // result[i,j,k,l] = (shape[i])[j,k,l]
          EXPECT_EQ(value, cpu_ragged_vec[i][index]);
        }
      }
      {
        // axis == 1
        int32_t axis = 1;
        Ragged<T> result = Stack(axis, num_shape, ragged.data());
        ASSERT_EQ(result.NumAxes(),
                  num_axes + 2);  // note we append one axis in each shape in
                                  // `shapes` before `Stack`
        ASSERT_EQ(result.Dim0(), dim0);
        result = result.To(cpu);
        RaggedShape &shape = result.shape;
        for (auto iter = shape.Iterator(); !iter.Done(); iter.Next()) {
          std::vector<int32_t> index = iter.Value();
          T value = result[index];
          int32_t j = index[1];
          index.erase(index.begin() + 1);
          // result[i,j,k,l] = (shape[j])[i,k,l]
          EXPECT_EQ(value, cpu_ragged_vec[j][index]);
        }
      }
    }
  }
}

TEST(RaggedTest, TestStackRagged) {
  TestStackRagged<int32_t>();
  TestStackRagged<double>();
}

TEST(RaggedTest, TestMaxSize) {
  for (int32_t i = 0; i <= 10; i++) {
    ContextPtr c = (i % 2 == 0 ? GetCpuContext() : GetCudaContext());
    int32_t num_axes = RandInt(2, 4);
    RaggedShape shape =
        RandomRaggedShape(true, num_axes, num_axes, 0, 1000).To(c);
    int32_t axis = RandInt(1, num_axes - 1);
    int32_t max_size = shape.MaxSize(axis);
    if (axis == 0) {
      K2_CHECK(max_size == shape.Dim0());
    } else {
      Array1<int32_t> row_splits = shape.RowSplits(axis).To(GetCpuContext());
      int32_t *row_splits_data = row_splits.Data();
      int32_t m = 0;
      for (int32_t i = 0; i + 1 < row_splits.Dim(); i++) {
        int32_t size = row_splits_data[i + 1] - row_splits_data[i];
        if (size > m) m = size;
      }
      ASSERT_EQ(m, max_size);
    }
  }
}

TEST(RaggedShapeOpsTest, TestMakeTransposable) {
  ContextPtr cpu = GetCpuContext();  // will be used to copy data
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // simple case
      const std::vector<int32_t> row_splits1 = {0, 2, 5, 6, 8};
      // const std::vector<int32_t> row_ids1 = {0, 0, 1, 1, 1, 2, 3, 3};
      const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10, 12, 13};
      // const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5, 6,
      //                                        6, 7};
      Array1<int32_t> row_splits1_array(context, row_splits1);
      Array1<int32_t> row_splits2_array(context, row_splits2);
      RaggedShape shape = RaggedShape3(&row_splits1_array, nullptr, -1,
                                       &row_splits2_array, nullptr, -1);

      std::vector<std::vector<int32_t>> expected_row_splits = {
          {0, 3, 6, 9, 12}, {0, 2, 3, 3, 4, 6, 7, 10, 10, 10, 12, 13, 13}};
      std::vector<std::vector<int32_t>> expected_row_ids = {
          {0, 0, 0, 1, 1, 1, 2, 2, 2, 3, 3, 3},
          {0, 0, 1, 3, 4, 4, 5, 6, 6, 6, 9, 9, 10}};

      RaggedShape result = MakeTransposable(shape);
      for (int32_t i = 1; i != 3; ++i) {
        CheckArrayData(result.RowSplits(i), expected_row_splits[i - 1]);
        CheckArrayData(result.RowIds(i), expected_row_ids[i - 1]);
      }
    }

    {
      // test with random large size
      for (int32_t i = 0; i < 2; ++i) {
        int32_t num_axes = RandInt(2, 4);
        RaggedShape shape =
            RandomRaggedShape(true, num_axes, num_axes, 0, 1000).To(context);
        int32_t dim0 = shape.Dim0();
        int32_t max_size = shape.MaxSize(1);
        RaggedShape result = MakeTransposable(shape);
        shape = shape.To(cpu);
        result = result.To(cpu);
        EXPECT_EQ(result.Dim0(), dim0);
        EXPECT_EQ(result.TotSize(1), dim0 * max_size);
        // check if every sub list in axis 1 has the same size
        int32_t *row_splits1 = result.RowSplits(1).Data();
        for (int32_t j = 0; j != dim0 + 1; ++j) {
          EXPECT_EQ(row_splits1[j], j * max_size);
        }
        if (num_axes > 2) {
          for (auto iter = shape.Iterator(); !iter.Done(); iter.Next()) {
            const std::vector<int32_t> &index = iter.Value();
            EXPECT_EQ(shape[index], result[index]);
          }
        }
      }
    }
  }
}

TEST(RaggedShapeOpsTest, PrefixTest) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // simple case
      const std::vector<int32_t> row_splits1 = {0, 2, 5, 6, 8};
      const std::vector<int32_t> row_splits2 = {0, 2, 3, 4, 6, 7, 10, 12, 13};
      Array1<int32_t> row_splits1_array(context, row_splits1);
      Array1<int32_t> row_splits2_array(context, row_splits2);
      RaggedShape shape = RaggedShape3(&row_splits1_array, nullptr, -1,
                                       &row_splits2_array, nullptr, -1);
      int32_t dim0 = shape.Dim0();
      int32_t num_axes = shape.NumAxes();
      EXPECT_EQ(dim0, 4);
      EXPECT_EQ(num_axes, 3);
      {
        // n == 0
        int32_t n = 0;
        std::vector<std::vector<int32_t>> expected_row_splits = {{0}, {0}};
        RaggedShape result = Prefix(shape, n);
        EXPECT_TRUE(IsCompatible(shape, result));
        EXPECT_EQ(result.Dim0(), n);
        EXPECT_EQ(result.NumAxes(), num_axes);
        for (int32_t i = 1; i != num_axes; ++i) {
          CheckArrayData(result.RowSplits(i), expected_row_splits[i - 1]);
        }
      }

      {
        // n > 0 && n < dim0
        int32_t n = 2;
        std::vector<std::vector<int32_t>> expected_row_splits = {
            {0, 2, 5}, {0, 2, 3, 4, 6, 7}};
        RaggedShape result = Prefix(shape, n);
        EXPECT_TRUE(IsCompatible(shape, result));
        EXPECT_EQ(result.Dim0(), n);
        EXPECT_EQ(result.NumAxes(), num_axes);
        for (int32_t i = 1; i != num_axes; ++i) {
          CheckArrayData(result.RowSplits(i), expected_row_splits[i - 1]);
        }
      }

      {
        // n == dim0
        int32_t n = 4;
        std::vector<std::vector<int32_t>> expected_row_splits = {
            {0, 2, 5}, {0, 2, 3, 4, 6, 7}};
        RaggedShape result = Prefix(shape, n);
        EXPECT_TRUE(IsCompatible(shape, result));
        EXPECT_EQ(result.Dim0(), n);
        EXPECT_EQ(result.NumAxes(), num_axes);
        CheckArrayData(result.RowSplits(1), row_splits1);
        CheckArrayData(result.RowSplits(2), row_splits2);
      }
    }

    {
      // test with random large size
      for (int32_t i = 0; i < 2; ++i) {
        RaggedShape shape = RandomRaggedShape(false, 2, 4, 0, 1000).To(context);
        int32_t dim0 = shape.Dim0();
        int32_t num_axes = shape.NumAxes();
        int32_t n = RandInt(0, dim0);
        RaggedShape result = Prefix(shape, n);
        EXPECT_TRUE(IsCompatible(shape, result));
        EXPECT_EQ(result.Dim0(), n);
        EXPECT_EQ(result.NumAxes(), num_axes);
        // just check row_splits1 here would be fine, as we have tested it with
        // simple case. We just confirm it can run successfully with kinds of
        // different random shapes.
        CheckArrayData(result.RowSplits(1), shape.RowSplits(1).Range(0, n + 1));
      }
    }
  }
}

TEST(RaggedShapeOpsTest, GetPrefixesTest) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // test with random large size
      for (int32_t i = 0; i < 2; ++i) {
        RaggedShape shape = RandomRaggedShape(false, 2, 4, 0, 1000).To(context);
        int32_t dim0 = shape.Dim0();
        int32_t num_axes = shape.NumAxes();
        int32_t ans_num = RandInt(0, 10);
        std::vector<int32_t> sizes;
        for (int32_t j = 0; j != ans_num; ++j)
          sizes.push_back(RandInt(0, dim0));
        ASSERT_EQ(sizes.size(), ans_num);
        std::vector<RaggedShape> ans = GetPrefixes(shape, sizes);
        ASSERT_EQ(ans.size(), ans_num);

        for (int32_t j = 0; j != ans_num; ++j) {
          int32_t n = sizes[j];

          RaggedShape ans_j = ans[j];
          EXPECT_TRUE(IsCompatible(shape, ans_j));
          EXPECT_EQ(ans_j.Dim0(), n);
          EXPECT_EQ(ans_j.NumAxes(), num_axes);

          RaggedShape result = Prefix(shape, n);
          EXPECT_TRUE(IsCompatible(shape, result));
          EXPECT_EQ(result.Dim0(), n);
          EXPECT_EQ(result.NumAxes(), num_axes);

          for (int32_t m = 1; m != num_axes; ++m) {
            EXPECT_TRUE(Equal(result.RowSplits(m), ans_j.RowSplits(m)));
          }
        }
      }
    }
  }
}

TEST(RaggedShapeOpsTest, ArangeTest) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // simple case
      const std::vector<int32_t> row_splits1 = {0, 2, 3, 4, 6, 7, 10};
      // const std::vector<int32_t> row_ids1 = {0, 0, 1, 2, 3, 3, 4, 5, 5, 5};
      const std::vector<int32_t> row_splits2 = {0,  2,  3,  5,  8, 9,
                                                12, 13, 15, 15, 16};
      // const std::vector<int32_t> row_ids2 = {0, 0, 1, 2, 2, 3, 3, 3,
      // 4, 5, 5, 5, 6, 7, 7, 9};
      Array1<int32_t> row_splits1_array(context, row_splits1);
      Array1<int32_t> row_splits2_array(context, row_splits2);
      RaggedShape shape = RaggedShape3(&row_splits1_array, nullptr, -1,
                                       &row_splits2_array, nullptr, -1);
      std::vector<int32_t> values(shape.NumElements());
      std::iota(values.begin(), values.end(), 10);
      Array1<int32_t> values_array(context, values);
      Ragged<int32_t> ragged(shape, values_array);
      int32_t dim0 = shape.Dim0();
      int32_t num_axes = shape.NumAxes();
      EXPECT_EQ(dim0, 6);
      EXPECT_EQ(num_axes, 3);
      {
        // axis == 0, begin == end
        int32_t axis = 0;
        int32_t begin = 1, end = 1;
        std::vector<std::vector<int32_t>> expected_row_splits = {{0}, {0}};
        std::pair<int32_t, int32_t> value_range;
        RaggedShape result = Arange(shape, axis, begin, end, &value_range);
        EXPECT_TRUE(IsCompatible(shape, result));
        EXPECT_EQ(result.Dim0(), 0);
        EXPECT_EQ(result.NumAxes(), num_axes);
        for (int32_t i = 1; i != num_axes; ++i) {
          CheckArrayData(result.RowSplits(i), expected_row_splits[i - 1]);
        }
        std::pair<int32_t, int32_t> expected_value_range = {1, 1};
        EXPECT_EQ(value_range, expected_value_range);
        EXPECT_EQ(result.NumElements(), value_range.second - value_range.first);

        // test `Arange` for ragged array
        Ragged<int32_t> ragged_result = Arange(ragged, axis, begin, end);
        EXPECT_EQ(ragged_result.values.Dim(), 0);
      }

      {
        // axis == 0, begin  < end == Dim0() + 1
        int32_t axis = 0;
        int32_t begin = 3, end = 6;
        std::vector<std::vector<int32_t>> expected_row_splits = {
            {0, 2, 3, 6}, {0, 1, 4, 5, 7, 7, 8}};
        std::pair<int32_t, int32_t> value_range;
        RaggedShape result = Arange(shape, axis, begin, end, &value_range);
        EXPECT_TRUE(IsCompatible(shape, result));
        EXPECT_EQ(result.NumAxes(), num_axes);
        for (int32_t i = 1; i != num_axes; ++i) {
          CheckArrayData(result.RowSplits(i), expected_row_splits[i - 1]);
        }
        std::pair<int32_t, int32_t> expected_value_range = {8, 16};
        EXPECT_EQ(value_range, expected_value_range);
        EXPECT_EQ(result.NumElements(), value_range.second - value_range.first);

        // test `Arange` for ragged array
        Ragged<int32_t> ragged_result = Arange(ragged, axis, begin, end);
        std::vector<int32_t> expected_values = {18, 19, 20, 21, 22, 23, 24, 25};
        CheckArrayData(ragged_result.values, expected_values);
      }

      {
        // axis == 1
        int32_t axis = 1;
        int32_t begin = 6, end = 8;
        std::vector<int32_t> expected_row_splits = {0, 1, 3};
        std::pair<int32_t, int32_t> value_range;
        RaggedShape result = Arange(shape, axis, begin, end, &value_range);
        EXPECT_TRUE(IsCompatible(shape, result));
        EXPECT_EQ(result.NumAxes(), 2);
        CheckArrayData(result.RowSplits(1), expected_row_splits);
        std::pair<int32_t, int32_t> expected_value_range = {12, 15};
        EXPECT_EQ(value_range, expected_value_range);
        EXPECT_EQ(result.NumElements(), value_range.second - value_range.first);

        // test `Arange` for ragged array
        Ragged<int32_t> ragged_result = Arange(ragged, axis, begin, end);
        std::vector<int32_t> expected_values = {22, 23, 24};
        CheckArrayData(ragged_result.values, expected_values);
      }
    }

    {
      // test with random large size
      for (int32_t i = 0; i < 2; ++i) {
        RaggedShape shape = RandomRaggedShape(false, 2, 4, 0, 1000).To(context);
        int32_t num_axes = shape.NumAxes();
        int32_t axis = RandInt(0, num_axes - 2);
        int32_t tot_size = shape.TotSize(axis);
        int32_t begin = RandInt(0, tot_size);
        int32_t end = RandInt(begin, tot_size);
        std::pair<int32_t, int32_t> value_range;
        RaggedShape result = Arange(shape, axis, begin, end, &value_range);
        EXPECT_TRUE(IsCompatible(shape, result));
        EXPECT_EQ(result.Dim0(), std::max(0, end - begin));
        EXPECT_EQ(result.NumAxes(), num_axes - axis);
        // just check row_splits1 here would be fine, as we have tested it with
        // simple case. We just confirm it can run successfully with kinds of
        // different random shapes.
        if (begin == end) {
          CheckArrayData(result.RowSplits(1), std::vector<int32_t>{0});
        } else {
          Array1<int32_t> row_splits1 =
              shape.RowSplits(axis + 1).Arange(begin, end + 1);
          row_splits1 = Minus(row_splits1, row_splits1[0]);
          CheckArrayData(result.RowSplits(1), row_splits1);
        }
        EXPECT_EQ(result.NumElements(), value_range.second - value_range.first);
      }
    }
  }
}

TEST(RaggedShapeOpsTest, Merge) {
  for (auto &c : {GetCpuContext(), GetCudaContext()}) {
    RaggedShape shape1 = RaggedShape("[ [ x x ] [ x ] [] ]")
                             .To(c),  // m: 0 3 6, m_out:  0 3, 6,
        shape2 = RaggedShape("[ [ x] [ x x x ] ]")
                     .To(c),  // m: 1 4, m_out: 1, 4 7 10
        shape3 =
            RaggedShape("[ [ ] [ x x ] [] ]").To(c);  // m: 2 5 8, m_out: ,2 5,

    RaggedShape ans_ref =
        RaggedShape("[ [] [x] [x x x] [] [] [x x] [x x] [x] ]").To(c);

    // This is a mixed-up kind of merge map that doesn't appear naturally (they
    // are always in-order from each source, right now) but it should still
    // work.
    std::vector<uint32_t> merge_map_data = {6, 1, 4, 8, 2, 5, 0, 3};
    Array1<uint32_t> merge_map_in(c, merge_map_data);
    RaggedShape *srcs[] = {&shape1, &shape2, &shape3};

    Array1<uint32_t> merge_map_out;
    RaggedShape merged = Merge(3, srcs, merge_map_in, &merge_map_out);

    ASSERT_EQ(true, Equal(ans_ref, merged));

    std::vector<uint32_t> merge_map_out_data = {1, 4, 7, 10, 2, 5, 0, 3, 6};
    CheckArrayData(merge_map_out, merge_map_out_data);
  }
}

TEST(RaggedTest, AddSuffixToRaggedTest) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // test with random large size
      for (int32_t i = 0; i < 10; ++i) {
        Ragged<int32_t> src = RandomRagged<int32_t>().To(context);
        int32_t num_axes = src.NumAxes();
        Array1<int32_t> suffix = RandUniformArray1<int32_t>(
            context, src.TotSize(num_axes - 2), 0, 100);
        Ragged<int32_t> dst = AddSuffixToRagged(src, suffix);
        EXPECT_EQ(dst.NumAxes(), num_axes);
        EXPECT_EQ(dst.NumElements(), src.NumElements() + suffix.Dim());
        Ragged<int32_t> src_cpu = src.To(GetCpuContext());
        Ragged<int32_t> dst_cpu = dst.To(GetCpuContext());
        for (RaggedShapeIndexIterator src_iter = src_cpu.shape.Iterator();
             !src_iter.Done(); src_iter.Next()) {
          const std::vector<int32_t> &src_indexes = src_iter.Value();
          EXPECT_EQ(dst_cpu[src_indexes], src_cpu[src_indexes]);
        }
        Array1<int32_t> src_row_splits = src_cpu.RowSplits(num_axes - 1);
        Array1<int32_t> suffix_cpu = suffix.To(GetCpuContext());
        for (int32_t i = 0; i < suffix.Dim(); ++i) {
          EXPECT_EQ(dst_cpu.values[src_row_splits[i + 1] + i], suffix_cpu[i]);
        }
      }
    }
  }
}

TEST(RaggedTest, AddPrefixToRaggedTest) {
  for (auto &context : {GetCpuContext(), GetCudaContext()}) {
    {
      // test with random large size
      for (int32_t i = 0; i < 10; ++i) {
        Ragged<int32_t> src = RandomRagged<int32_t>().To(context);
        int32_t num_axes = src.NumAxes();
        Array1<int32_t> prefix = RandUniformArray1<int32_t>(
            context, src.TotSize(num_axes - 2), 0, 100);
        Ragged<int32_t> dst = AddPrefixToRagged(src, prefix);
        EXPECT_EQ(dst.NumAxes(), num_axes);
        EXPECT_EQ(dst.NumElements(), src.NumElements() + prefix.Dim());
        Ragged<int32_t> src_cpu = src.To(GetCpuContext());
        Ragged<int32_t> dst_cpu = dst.To(GetCpuContext());
        for (RaggedShapeIndexIterator src_iter = src_cpu.shape.Iterator();
             !src_iter.Done(); src_iter.Next()) {
          const std::vector<int32_t> &src_indexes = src_iter.Value();
          std::vector<int32_t> dst_indexes(src_indexes);
          dst_indexes.back() += 1;  // increase the last index by 1
          EXPECT_EQ(dst_cpu[dst_indexes], src_cpu[src_indexes]);
        }
        Array1<int32_t> src_row_splits = src_cpu.RowSplits(num_axes - 1);
        Array1<int32_t> prefix_cpu = prefix.To(GetCpuContext());
        for (int32_t i = 0; i < prefix.Dim(); ++i) {
          EXPECT_EQ(dst_cpu.values[src_row_splits[i] + i], prefix_cpu[i]);
        }
      }
    }
  }
}

TEST(RaggedTest, RemoveValuesLeq) {
  for (auto &c : {GetCpuContext(), GetCudaContext()}) {
    Ragged<int32_t> r = Ragged<int32_t>(" [ [ 3 4 ] [ 5 7 8 ] ]").To(c),
                    s3 = Ragged<int32_t>(" [ [4] [5 7 8]]").To(c),
                    s5 = Ragged<int32_t>(" [ [] [ 7 8]]").To(c);
    Ragged<int32_t> ans1 = RemoveValuesLeq(r, 3), ans2 = RemoveValuesLeq(r, 5);
    K2_LOG(INFO) << "ans2 = " << ans2;
    EXPECT_EQ(true, Equal(ans1, s3));
    EXPECT_EQ(true, Equal(ans2, s5));
  }
}

TEST(RaggedTest, IndexArrayRagged) {
  for (auto &c : {GetCpuContext(), GetCudaContext()}) {
    Ragged<int32_t> r = Ragged<int32_t>(" [ [ 2 0 ] [ 1 2 3 ] ]").To(c);
    Array1<float> f(c, std::vector<float>({0.0, 1.0, 2.0, 3.0, 4.0}));

    Ragged<float> fr = Ragged<float>(" [ [ 2.0 0.0 ] [ 1.0 2.0 3.0 ] ]").To(c),
                  ans = Index(f, r);
    EXPECT_EQ(true, Equal(ans, fr));
  }
}

TEST(RaggedTest, IndexRaggedRagged) {
  for (auto &c : {GetCpuContext(), GetCudaContext()}) {
    Ragged<int32_t> r = Ragged<int32_t>(" [ [ 2 0 ] [ 1 2 3 ] ]").To(c);

    Ragged<int32_t> s =
        Ragged<int32_t>(" [ [ 10 10 ] [ 11 ] [ 12 12 ] [ 13 ] [ 14 14] ]")
            .To(c);  // NOLINT

    Ragged<int32_t> sr1 =
        Ragged<int32_t>(" [ [ [12 12] [10 10] ] [ [11] [12 12] [13] ] ]")
            .To(c);  // NOLINT

    Ragged<int32_t> sr2 =
        Ragged<int32_t>(" [ [ 12 12 10 10 ] [ 11 12 12 13 ] ]")
            .To(c);  // NOLINT

    EXPECT_EQ(true, Equal(Index(s, r, false), sr1));
    EXPECT_EQ(true, Equal(Index(s, r, true), sr2));
  }
}

TEST(RaggedShapeOpsTest, CoveringShape) {
  for (auto &c : {GetCpuContext(), GetCudaContext()}) {
    {
      // simple case
      RaggedShape shape1 = RaggedShape("[ [ x x ] [] [ x ] ]").To(c),
                  shape2 = RaggedShape("[ [ x] [] [ x x x ] ]").To(c),
                  shape3 = RaggedShape("[ [] [] [ x x ] ]").To(c);

      RaggedShape expected = RaggedShape("[ [x x] [] [x x x] ]").To(c);
      RaggedShape *srcs[] = {&shape1, &shape2, &shape3};
      RaggedShape ans = CoveringShape(3, srcs);
      EXPECT_TRUE(Equal(expected, ans));

      // test CoveringShapeForwardMap
      {
        Array1<int32_t> elem_map = CoveringShapeForwardMap(shape1, ans);
        std::vector<int32_t> expected_map = {0, 1, 2, -1, -1};
        CheckArrayData(elem_map, expected_map);
      }
      {
        Array1<int32_t> elem_map = CoveringShapeForwardMap(shape2, ans);
        std::vector<int32_t> expected_map = {0, -1, 1, 2, 3};
        CheckArrayData(elem_map, expected_map);
      }
      {
        Array1<int32_t> elem_map = CoveringShapeForwardMap(shape3, ans);
        std::vector<int32_t> expected_map = {-1, -1, 0, 1, -1};
        CheckArrayData(elem_map, expected_map);
      }
    }
    {
      // another simple case: only one src
      RaggedShape shape1 = RaggedShape("[ [ x x ] [ x ] [] ]").To(c);
      RaggedShape *srcs[] = {&shape1};
      RaggedShape ans = CoveringShape(1, srcs);
      EXPECT_TRUE(Equal(shape1, ans));

      // test CoveringShapeForwardMap
      Array1<int32_t> elem_map = CoveringShapeForwardMap(shape1, ans);
      std::vector<int32_t> expected_map = {0, 1, 2};
      CheckArrayData(elem_map, expected_map);
    }
    {
      // random case
      for (int32_t i = 0; i != 1; ++i) {
        int32_t num_shape = RandInt(1, 100);
        int32_t dim0 = RandInt(1, 1000);
        std::vector<RaggedShape> shape_vec(num_shape);
        std::vector<RaggedShape *> shapes(num_shape);
        for (int32_t j = 0; j != num_shape; ++j) {
          Array1<int32_t> row_sizes =
              RandUniformArray1<int32_t>(c, dim0 + 1, 0, 100);
          ExclusiveSum(row_sizes, &row_sizes);
          shape_vec[j] = RaggedShape2(&row_sizes, nullptr, -1);
          ASSERT_TRUE(shape_vec[j].Context()->IsCompatible(*c));
          ASSERT_EQ(shape_vec[j].Dim0(), dim0);
          shapes[j] = &shape_vec[j];
        }
        RaggedShape ans = CoveringShape(num_shape, shapes.data());
        std::vector<Array1<int32_t>> elem_map(num_shape);
        for (int32_t j = 0; j != num_shape; ++j) {
          elem_map[j] = CoveringShapeForwardMap(shape_vec[j], ans);
        }
        // check ans
        ASSERT_EQ(ans.NumAxes(), 2);
        ASSERT_EQ(ans.Dim0(), dim0);
        ASSERT_TRUE(ans.Context()->IsCompatible(*c));
        ContextPtr cpu = GetCpuContext();
        ans = ans.To(cpu);
        for (int32_t j = 0; j != num_shape; ++j)
          shape_vec[j] = shape_vec[j].To(cpu);
        for (int32_t d = 0; d != dim0; ++d) {
          int32_t max_row_size = 0;
          for (int32_t j = 0; j != num_shape; ++j)
            max_row_size = std::max(
                shape_vec[j].RowSplits(1)[d + 1] - shape_vec[j].RowSplits(1)[d],
                max_row_size);
          EXPECT_EQ(max_row_size,
                    ans.RowSplits(1)[d + 1] - ans.RowSplits(1)[d]);
        }

        // test CoveringShapeForwardMap
        for (int32_t j = 0; j != num_shape; ++j) {
          Array1<int32_t> cur_elem_map = elem_map[j].To(cpu);
          ASSERT_EQ(cur_elem_map.Dim(), ans.NumElements());
          int32_t n = 0;
          for (RaggedShapeIndexIterator ans_iter = ans.Iterator();
               !ans_iter.Done(); ans_iter.Next()) {
            const std::vector<int32_t> &ans_indexes = ans_iter.Value();
            int32_t src_shape_linear_index = cur_elem_map[n];
            if (src_shape_linear_index != -1) {
              EXPECT_EQ(src_shape_linear_index, shape_vec[j][ans_indexes]);
            }
            ++n;
          }
        }
      }
    }
  }
}

TEST(RaggedShapeOpsTest, RaggedShapeAxis0Splitter) {
  for (int32_t i = 0; i < 20; i++) {
    for (auto &context : {GetCpuContext(), GetCudaContext()}) {
      RaggedShape random = RandomRaggedShape(false, 3, 6, 0, 2000);
      int32_t dim0 = random.Dim0();
      RaggedShapeAxis0Splitter splitter(random);
      for (int32_t i = 0; i < dim0; i++) {
        int32_t offset, offset2, offset3;
        RaggedShape sub_shape1 = random.Index(0, i, &offset),
                    sub_shape2 = splitter.GetElement(i, &offset2);
        offset3 = splitter.GetOffset(i, random.NumAxes() - 1);
        EXPECT_EQ(offset, offset2);
        EXPECT_EQ(offset, offset3);
        EXPECT_EQ(Equal(sub_shape1, sub_shape2), true);
      }
    }
  }
}
template <typename T>
static void TestSegmentedExclusiveSum() {
  for (auto &c : {GetCpuContext(), GetCudaContext()}) {
    {
      // simple case
      Ragged<T> src("[ [1 2 3 -1] [3 4 -1] [] [5 6 7 -1] ]");
      src = src.To(c);
      Array1<T> dst(c, src.NumElements());
      SegmentedExclusiveSum(src, &dst);

      std::vector<T> expected = {0, 1, 3, 6,
                                 //
                                 0, 3, 7,
                                 //
                                 0, 5, 11, 18};
      CheckArrayData(dst, expected);

      // &src.values == dst
      SegmentedExclusiveSum(src, &src.values);
      CheckArrayData(src.values, expected);
    }
    {
      // random case, we assume the implementation for cpu is correct and only
      // test for Cuda version
      if (c->GetDeviceType() == kCuda) {
        for (int32_t i = 0; i != 2; ++i) {
          Ragged<T> cpu_ragged = RandomRagged<T>(-1000, 1000, 2, 4, 0, 5000);
          int32_t dim = cpu_ragged.NumElements();
          Array1<T> cpu_dst(GetCpuContext(), dim);
          SegmentedExclusiveSum(cpu_ragged, &cpu_dst);

          Ragged<T> ragged = cpu_ragged.To(c);
          Array1<T> dst(c, dim);
          SegmentedExclusiveSum(ragged, &dst);
          CheckArrayData(dst, cpu_dst, (T)0.1);
        }
      }
    }
  }
}

TEST(RaggedOpsTest, SegmentedExclusiveSum) {
  TestSegmentedExclusiveSum<int32_t>();
  TestSegmentedExclusiveSum<float>();
  TestSegmentedExclusiveSum<double>();
}

TEST(RaggedOpsTest, TestComputeHash) {
  for (int32_t i = 0; i < 20; i++) {
    Ragged<int32_t> src = RandomRagged<int32_t>(
                        std::numeric_limits<int32_t>::min(),
                        std::numeric_limits<int32_t>::max(), 2, 4, 0, 20000),
                    src_gpu = src.To(GetCudaContext());
    {
      Array1<int64_t> hash1 = ComputeHash<int64_t>(src),
                      hash2 = ComputeHash<int64_t>(src_gpu).To(GetCpuContext());
      EXPECT_EQ(Equal(hash1, hash2), true);
    }

    {
      Array1<int32_t> hash1 = ComputeHash<int32_t>(src),
                      hash2 = ComputeHash<int32_t>(src_gpu).To(GetCpuContext());
      EXPECT_EQ(Equal(hash1, hash2), true);
    }
  }
}



TEST(RaggedOpsTest, TestUniqueSequences) {
  for (int32_t i = 0; i < 20; i++) {
    for (auto &c : {GetCpuContext(), GetCudaContext()}) {
      Ragged<int32_t> src = RandomRagged<int32_t>(0, 3, 2, 4, 0, 20000).To(c);

      Ragged<int32_t> unique = UniqueSequences(src);

      if (src.NumAxes() == 2) {
        src = Unsqueeze(src, 0);
        unique = Unsqueeze(unique, 0);
      }

      ContextPtr cpu = GetCpuContext();
      Array1<int32_t> hash_src = ComputeHash<int32_t>(src).To(cpu),
          hash_unique = ComputeHash<int32_t>(unique).To(cpu);

      RaggedShape src_hash_shape =
          RemoveAxis(src.shape, src.NumAxes() - 1).To(cpu);
      src_hash_shape = GetLayer(src_hash_shape, src_hash_shape.NumLayers() - 1);

      RaggedShape unique_hash_shape =
          RemoveAxis(unique.shape, unique.NumAxes() - 1).To(cpu);
      unique_hash_shape =
          GetLayer(unique_hash_shape, unique_hash_shape.NumLayers() - 1);

      K2_CHECK_EQ(src_hash_shape.Dim0(), unique_hash_shape.Dim0());

      const int32_t *src_hash_row_splits = src_hash_shape.RowSplits(1).Data(),
          *unique_hash_row_splits = unique_hash_shape.RowSplits(1).Data();
      const int32_t *src_hash_data = hash_src.Data(),
          *unique_hash_data = hash_unique.Data();

      for (int32_t r = 0; r < src_hash_shape.Dim0(); r++) {
        int32_t src_begin = src_hash_row_splits[r],
                src_end = src_hash_row_splits[r + 1],
                unique_begin = unique_hash_row_splits[r],
                unique_end = unique_hash_row_splits[r + 1];
        std::set<int32_t> src_set(src_hash_data + src_begin,
                                  src_hash_data + src_end),
            unique_set(unique_hash_data + unique_begin,
                       unique_hash_data + unique_end);
        EXPECT_EQ((src_set == unique_set), true);
      }
    }
  }
}

TEST(RaggedIntTest, TestCreateRagged2Int) {
  std::vector<std::vector<int32_t>> vecs{{7, 9}, {10, 12, 13}, {}};
  std::vector<int32_t> expected_values{7, 9, 10, 12, 13};
  std::vector<int32_t> expected_row_splits = {0, 2, 5, 5};
  Ragged<int32_t> r = CreateRagged2(vecs);
  EXPECT_EQ(r.Context()->GetDeviceType(), kCpu);
  CheckArrayData(r.RowSplits(1), expected_row_splits);
  EXPECT_EQ(r.NumAxes(), 2);
  CheckArrayData(r.values, expected_values);

  Ragged<int32_t> r2("[ [7 9] [10 12 13] [] ]");
  K2_CHECK(Equal(r, r2));
}


TEST(RaggedFloatTest, TestCreateRagged2Float) {
  std::vector<std::vector<float>> vecs{{1.2, 2.3}, {}, {3.4, 5.6}};
  std::vector<float> expected_values{1.2, 2.3, 3.4, 5.6};
  std::vector<int32_t> expected_row_splits = {0, 2, 2, 4};
  Ragged<float> r = CreateRagged2(vecs);

  Ragged<float> &r2 = r.Generic().Specialize<float>();

  EXPECT_EQ(r.Context()->GetDeviceType(), kCpu);
  EXPECT_EQ(r2.Context()->GetDeviceType(), kCpu);
  CheckArrayData(r.RowSplits(1), expected_row_splits);
  EXPECT_EQ(r.NumAxes(), 2);
  EXPECT_EQ(r2.NumAxes(), 2);
  CheckArrayData(r.values, expected_values);
}


}  // namespace k2
