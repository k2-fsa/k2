/**
 * @brief
 * ragged
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey
 *                                                   Haowen Qiu)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <hipcub/hipcub.hpp>
#include <vector>

#include "k2/csrc/array_ops.h"
#include "k2/csrc/math.h"
#include "k2/csrc/ragged.h"
namespace {

// will be used in RaggedShape::MaxSize(int32_t axis) to call
// hipcub::DeviceReduce::Max
struct RowSplitsDiff {
  const int32_t *row_splits_data;
  explicit RowSplitsDiff(const int32_t *row_splits)
      : row_splits_data(row_splits) {}
  // operator[] and operator+ are required by hipcub::DeviceReduce::Max
  __device__ int32_t operator[](int32_t i) const {
    return row_splits_data[i + 1] - row_splits_data[i];
  }
  __device__ RowSplitsDiff operator+(int32_t n) const {
    RowSplitsDiff tmp(*this);
    tmp.row_splits_data += n;
    return tmp;
  }
};
}  // namespace

namespace std {
// vaule_type is required by hipcub::DeviceReduce::Max
template <>
struct iterator_traits<::RowSplitsDiff> {
  typedef int32_t value_type;
};
}  // namespace std

namespace k2 {

RaggedShape RandomRaggedShape(int32_t min_num_axes, int32_t max_num_axes,
                              int32_t min_num_elements,
                              int32_t max_num_elements) {
  ContextPtr c = GetCpuContext();
  K2_CHECK(min_num_axes >= 2 && max_num_axes >= min_num_axes &&
           min_num_elements >= 0 && max_num_elements >= min_num_elements);
  int32_t num_axes = RandInt(min_num_axes, max_num_axes);

  // int32_t done_repeats = 0;

  std::vector<RaggedShapeDim> axes(num_axes - 1);
  int32_t num_elements = RandIntGeometric(min_num_elements, max_num_elements);
  for (int32_t axis = num_axes - 2; axis >= 0; axis--) {
    // this axis will have row_ids of length num_elements and row_splits of
    // length
    // to be determined.

    int32_t cur_row_split = 0;
    std::vector<int32_t> row_splits_vec;
    row_splits_vec.push_back(cur_row_split);
    // The reason for "|| RandInt(0, 2) == 0)" is so that even if there
    // are no elements we can still potentially generate empty row-splits.
    while (cur_row_split < num_elements || RandInt(0, 2) == 0) {
      int32_t split_size = RandIntGeometric(0, num_elements - cur_row_split);
      cur_row_split += split_size;
      // sometimes we have a bunch of empty rows in a row (this will test out
      // more of the code).
      // TODO(haowen): what do below code do?
      /*
      int32_t num_repeats = 1;
      if (split_size == 0 && RandInt(0, 30) == 0 && done_repeats == 0) {
        num_repeats = RandIntGeometric(1, 128);
        done_repeats = 0;
      }
      */
      row_splits_vec.push_back(cur_row_split);
    }
    axes[axis].row_splits = Array1<int32_t>(c, row_splits_vec);
    axes[axis].cached_tot_size = num_elements;
    num_elements = cur_row_split;
  }
  return RaggedShape(axes);
}

// Recursive function that prints (part of) a ragged shape.
// 0 <=  begin_pos <= end_pos < shape.TotSize(axis).

void PrintRaggedShapePart(std::ostream &stream, RaggedShape &shape,
                          int32_t axis, int32_t begin_pos, int32_t end_pos) {
  K2_CHECK(axis >= 0 && axis < shape.NumAxes() && begin_pos >= 0 &&
           begin_pos <= end_pos && end_pos <= shape.TotSize(axis));
  for (int32_t d = begin_pos; d < end_pos; d++) {
    if (axis == shape.NumAxes() - 1) {
      stream << d << " ";
    } else {
      stream << "[ ";
      const int32_t *row_splits = shape.RowSplits(axis + 1).Data();
      K2_DCHECK(d < shape.RowSplits(axis + 1).Dim());
      int32_t row_start = row_splits[d], row_end = row_splits[d + 1];
      PrintRaggedShapePart(stream, shape, axis + 1, row_start, row_end);
      stream << "] ";
    }
  }
}

// prints a RaggedShape as e.g. [ [ 0 1 ] [ 2 ] [] ].  Note, the 'values'
// are just the positions in the array, this is for readability.
std::ostream &operator<<(std::ostream &stream, RaggedShape &shape) {
  if (shape.Context()->GetDeviceType() != kCpu) {
    return stream << shape.To(GetCpuContext());
  } else {
    stream << "[ ";
    PrintRaggedShapePart(stream, shape, 0, 0, shape.Dim0());
    stream << "]";
    return stream;
  }
}

RaggedShape RaggedShapeFromTotSizes(ContextPtr &c, int32_t num_axes,
                                    int32_t *tot_sizes) {
  std::vector<RaggedShapeDim> axes(num_axes - 1);
  // In future we might choose to allocate everything in one big array, to avoid
  // multiple allocations, but for now just do it the simple way.
  for (int32_t axis = 1; axis < num_axes; axis++) {
    axes[axis - 1].row_splits = Array1<int32_t>(c, tot_sizes[axis - 1] + 1);
    axes[axis - 1].row_ids = Array1<int32_t>(c, tot_sizes[axis] + 1);
    axes[axis - 1].cached_tot_size = tot_sizes[axis];
  }
  return RaggedShape(axes);
}

RaggedShape RaggedShapeFromTotSizes(int32_t num_axes, int32_t *tot_sizes) {
  // TODO
  std::vector<RaggedShapeDim> axes;
  return RaggedShape(axes);
}

Array1<int32_t> &RaggedShape::RowIds(int32_t axis) {
  K2_CHECK_GT(axis, 0);
  K2_CHECK_LT(axis, NumAxes());
  RaggedShapeDim &rsd = axes_[axis - 1];
  auto &row_splits = rsd.row_splits;
  auto &row_ids = rsd.row_ids;
  // there must be row_splits.Dim() >=1 according to the definition of
  // RaggedShapeDim.
  K2_CHECK_GE(row_splits.Dim(), 1);
  if (row_splits.Dim() != 1 && row_ids.Dim() == 0) {
    // create row_ids as it does not exist
    row_ids = Array1<int32_t>(Context(), row_splits[row_splits.Dim() - 1]);
    const int32_t *row_splits_data = row_splits.Data();
    int32_t *row_ids_data = row_ids.Data();
    RowSplitsToRowIds(Context(), row_splits.Dim() - 1, row_splits_data,
                      row_ids.Dim(), row_ids_data);
    // set cached_tot_size
    rsd.cached_tot_size = row_ids.Dim();
  }
  return row_ids;
}

int32_t RaggedShape::MaxSize(int32_t axis) {
  K2_CHECK_GT(axis, 0);
  K2_CHECK_LT(axis, NumAxes());
  const auto &row_splits = axes_[axis - 1].row_splits;
  const int32_t num_rows = row_splits.Dim() - 1;
  if (num_rows == 0) return 0;
  const int32_t *row_splits_data = row_splits.Data();
  ContextPtr c = Context();
  if (c->GetDeviceType() == kCpu) {
    int32_t max_value = 0;
    for (int32_t i = 0; i < num_rows; ++i) {
      int32_t value = row_splits_data[i + 1] - row_splits_data[i];
      if (value > max_value) max_value = value;
    }
    return max_value;
  } else {
    K2_CHECK_EQ(c->GetDeviceType(), kCuda);
    ::RowSplitsDiff row_splits_diff(row_splits_data);
    Array1<int32_t> max_array(Context(), 1, 0);
    int32_t *max_value = max_array.Data();

    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    // the first time is to determine temporary device storage requirements
    K2_CHECK_CUDA_ERROR(hipcub::DeviceReduce::Max(
        d_temp_storage, temp_storage_bytes, row_splits_diff, max_value,
        num_rows, c->GetCudaStream()));
    void *deleter_context;
    d_temp_storage = c->Allocate(temp_storage_bytes, &deleter_context);
    K2_CHECK_CUDA_ERROR(hipcub::DeviceReduce::Max(
        d_temp_storage, temp_storage_bytes, row_splits_diff, max_value,
        num_rows, c->GetCudaStream()));
    c->Deallocate(d_temp_storage, deleter_context);
    // this will convert to memory on CPU
    return max_array[0];
  }
}

RaggedShape RaggedShape::Index(int32_t axis, int32_t i) {
  // only support `axis == 0` for now
  K2_CHECK_EQ(axis, 0);
  K2_CHECK_GE(i, 0);
  int32_t num_axes = NumAxes();
  K2_CHECK_GE(num_axes, 2);
  const auto &src_axes = Axes();
  K2_CHECK_LT(i + 1, src_axes[0].row_splits.Dim());

  int32_t idx = src_axes[0].row_splits[i];
  int32_t idx_next = src_axes[0].row_splits[i + 1];
  std::vector<RaggedShapeDim> axes(src_axes.size() - 1);
  ContextPtr c = Context();
  for (int32_t i = 2; i < num_axes; ++i) {
    const Array1<int32_t> &src_row_splits = src_axes[i - 1].row_splits;
    int num_rows = idx_next - idx;
    int32_t offset = idx;
    idx = src_row_splits[idx];
    idx_next = src_row_splits[idx_next];
    // allocate new memory here as we need to change the values,
    // i.e. subtracts the offset.
    axes[i - 2].row_splits = Array1<int32_t>(c, num_rows + 1);
    int32_t *data = axes[i - 2].row_splits.Data();
    const int32_t *src_data = src_row_splits.Data();
    auto lambda_set_values = [=] __host__ __device__(int32_t i) -> void {
      data[i] = src_data[i + offset] - idx;
    };
    Eval(c, num_rows + 1, lambda_set_values);
    // leave row_ids and cached_tot_size unset
    axes[i - 2].cached_tot_size = -1;
  }
  RaggedShape shape(axes, true);
  return shape;
}

void RaggedShape::Populate() {
  int32_t num_axes = NumAxes();
  for (int32_t i = 1; i < num_axes; ++i) {
    // row_splits is always non-empty
    RaggedShapeDim &rsd = axes_[i - 1];
    const auto &row_splits = rsd.row_splits;
    auto &row_ids = rsd.row_ids;
    K2_CHECK_GE(row_splits.Dim(), 1);
    if (row_splits.Dim() != 1 && row_ids.Dim() == 0) {
      // create row_ids as it does not exist
      row_ids = Array1<int32_t>(Context(), row_splits[row_splits.Dim() - 1]);
      const int32_t *row_splits_data = row_splits.Data();
      int32_t *row_ids_data = row_ids.Data();
      RowSplitsToRowIds(Context(), row_splits.Dim() - 1, row_splits_data,
                        row_ids.Dim(), row_ids_data);
      rsd.cached_tot_size = row_ids.Dim();
    }
  }
}

RaggedShape RaggedShape::To(ContextPtr ctx) const {
  if (ctx->IsCompatible(*Context())) return *this;
  std::vector<RaggedShapeDim> axes(axes_.size());
  int32_t num_axes = NumAxes();
  for (int32_t i = 1; i < num_axes; ++i) {
    axes[i - 1].row_splits = axes_[i - 1].row_splits.To(ctx);
    // leave row_ids and cached_tot_size unset
    axes[i - 1].cached_tot_size = -1;
  }
  return RaggedShape(axes);
}

RaggedShapeIndexIterator RaggedShape::Iterator() {
  return RaggedShapeIndexIterator(*this);
}

int32_t RaggedShape::operator[](const std::vector<int32_t> &indexes) {
  K2_CHECK(indexes.size() == NumAxes());
  K2_CHECK(Context()->GetDeviceType() == kCpu);
  int32_t cur_idx = indexes[0];
  for (int32_t i = 1; i < NumAxes(); i++) {
    Array1<int32_t> &row_splits = axes_[i - 1].row_splits;
    K2_CHECK(cur_idx >= 0 && cur_idx + 1 < row_splits.Dim());
    cur_idx = row_splits[cur_idx];
    cur_idx += indexes[i];
  }
  return cur_idx;
}

void RaggedShape::Check() {
  ContextPtr c = Context();
  int32_t num_axes = axes_.size();
  for (int32_t axis = 0; axis < axes_.size(); axis++) {
    RaggedShapeDim &rsd = axes_[axis];
    CHECK_GE(rsd.row_splits.Dim(), 0);
    if (rsd.cached_tot_size >= 0) {
      K2_CHECK(rsd.row_splits.Dim() == 0 ||
               rsd.cached_tot_size == rsd.row_splits[rsd.row_splits.Dim() - 1]);
      K2_CHECK(rsd.row_ids.Dim() == 0 ||
               rsd.cached_tot_size == rsd.row_ids.Dim());
    } else {
      K2_CHECK_EQ(rsd.cached_tot_size, -1);
      K2_CHECK_EQ(rsd.row_ids.Dim(), 0);
    }

    int32_t num_elems;
    // Check row_splits.
    {
      // meta[0] is a bool, ok == 1, not-ok == 0.
      // meta[1] will contain the number of row_splits.
      Array1<int32_t> meta(c, 2, 1);
      int32_t *ok_data = meta.Data(), *num_elems_data = ok_data + 1;
      const int32_t *row_splits_data = rsd.row_splits.Data();
      int32_t num_rows = rsd.row_splits.Dim() - 1;

      auto lambda_check_row_splits =
          [=] __host__ __device__(int32_t i) -> void {
        int32_t this_idx = row_splits_data[i];
        if (i == 0 && this_idx != 0) *ok_data = 0;
        if (i < num_rows) {
          int32_t next_idx = row_splits_data[i + 1];
          if (next_idx < this_idx) *ok_data = 0;
        } else {
          K2_CHECK(i == num_rows);
          *num_elems_data = this_idx;
        }
      };
      Eval(c, num_rows + 1, lambda_check_row_splits);
      meta = meta.To(GetCpuContext());
      num_elems = meta[1];
      int32_t ok = meta[0];
      if (!ok) {
        K2_LOG(FATAL) << "Problem validating row-splits: for axes_[" << axis
                      << "], row_splits = " << rsd.row_splits;
      }
      if (rsd.cached_tot_size > 0 && rsd.cached_tot_size != num_elems) {
        K2_LOG(FATAL) << "Problem validating row-splits: for axes_[" << axis
                      << "], row_splits[-1] = " << num_elems
                      << " but cached_tot_size == " << rsd.cached_tot_size;
      }
    }
    if (axis + 1 < axes_.size()) {
      int32_t next_num_rows = axes_[axis + 1].row_splits.Dim() - 1;
      if (num_elems != next_num_rows) {
        K2_LOG(FATAL) << "Ragged shape has num_elems for axes_[" << axis
                      << "] == " << num_elems << " and num-rows for axes_["
                      << (axis + 1) << "] == " << next_num_rows;
      }
    }

    if (rsd.row_ids.Dim() != 0) {  // check row_ids.
      K2_CHECK(IsCompatible(rsd.row_ids, rsd.row_splits));
      // 1st elem is `ok` (1 or 0); 2nd elem is location of bad index
      // into row_splits
      Array1<int32_t> meta(c, 2, 1);
      int32_t *ok_data = meta.Data(), *bad_index_data = ok_data + 1;

      const int32_t *row_splits_data = rsd.row_splits.Data(),
                    *row_ids_data = rsd.row_ids.Data();
      int32_t num_elems_from_row_ids = rsd.row_ids.Dim(),
              num_rows = rsd.row_splits.Dim() - 1;

      K2_CHECK_EQ(num_elems, num_elems_from_row_ids);
      auto lambda_check_row_ids = [=] __host__ __device__(int32_t i) -> void {
        int32_t this_row = row_ids_data[i];
        if (this_row < 0 || this_row >= num_rows ||
            i < row_splits_data[this_row] ||
            i >= row_splits_data[this_row + 1]) {
          *ok_data = 0;
          *bad_index_data = i;
        }
      };
      // TODO: could do this and the other one in separate streams.
      Eval(c, num_elems, lambda_check_row_ids);
      meta = meta.To(GetCpuContext());  // since we have 2 accesses, this should
                                        // be faster.
      int32_t ok = meta[0];
      if (!ok) {
        K2_LOG(FATAL) << "Problem validating row-ids: for axes_[" << axis
                      << "], row_splits = " << rsd.row_splits
                      << ", row_ids = " << rsd.row_ids << ", see index "
                      << meta[1] << " of row_ids, whose dim is "
                      << rsd.row_ids.Dim();
      }
    }
    if (axis + 1 < axes_.size()) {
      K2_CHECK(IsCompatible(rsd.row_splits, axes_[axis + 1].row_splits));
    }
  }
}

RaggedShape RaggedShape2(Array1<int32_t> *row_splits, Array1<int32_t> *row_ids,
                         int32_t cached_tot_size) {
  if (!row_splits && !row_ids) {
    LOG(FATAL) << "At least one of row_splits and row_ids must be defined";
  }
  if (cached_tot_size != -1) {
    if (row_ids != nullptr) CHECK(cached_tot_size == row_ids->Dim() - 1);
    if (row_splits != nullptr) {  // caution: next check may be slow...
      const auto &row_splits_ref = *row_splits;
      K2_CHECK_EQ(cached_tot_size, row_splits_ref[row_splits->Dim() - 1]);
    }
  }
  std::vector<RaggedShapeDim> axes(1);
  if (row_splits) axes[0].row_splits = *row_splits;
  if (row_ids) axes[0].row_ids = *row_ids;
  axes[0].cached_tot_size = cached_tot_size;
  return RaggedShape(axes);
}

RaggedShape ComposeRaggedShapes(RaggedShape &a, RaggedShape &b) {
  if (a.NumElements() != b.Dim0()) {
    LOG(FATAL) << "ComposeRaggedShapes: shape mismatch: " << a.NumElements()
               << " vs. " << b.Dim0();
  }
  const auto &a_axes = a.Axes();
  const auto &b_axes = b.Axes();
  std::vector<RaggedShapeDim> axes(a_axes.size() + b_axes.size());
  size_t a_size = a_axes.size(), b_size = b_axes.size();
  for (size_t i = 0; i < a_size; i++) axes[i] = a_axes[i];
  for (size_t i = 0; i < b_size; i++) axes[i + a_size] = b_axes[i];
  return RaggedShape(axes);
}

RaggedShape RaggedShape3(Array1<int32_t> *row_splits1,
                         Array1<int32_t> *row_ids1, int32_t cached_tot_size1,
                         Array1<int32_t> *row_splits2,
                         Array1<int32_t> *row_ids2, int32_t cached_tot_size2) {
  // This is a slightly lazy implementation, could save a couple copies of
  // metadata by
  // implementing it directly.
  auto shape1 = RaggedShape2(row_splits1, row_ids1, cached_tot_size1);
  auto shape2 = RaggedShape2(row_splits2, row_ids2, cached_tot_size2);
  return ComposeRaggedShapes(shape1, shape2);
}

Array1<int32_t *> GetRowSplitsPtr(RaggedShape &src) {
  Array1<int32_t *> array;
  // TODO(haowen): implement
  return array;
}

// See declaration in ragged.h for documentation of its purpose and interface.
RaggedShape Unsqueeze(RaggedShape &src, int32_t axis) {
  // If axis == 0, initial row_splits and row_ids will look like the following,
  // if for example src.Dim0() was 5: [ 0 5 ],  [ 0 0 0 0 0 ].  The other axes
  // would be pushed forward.
  //
  // If 0 < axis <= src.NumAxes(), the inserted row_splits and row_ids would
  // look like the following, if for instance the src.TotSize(axis-1) = 8:
  //   [ 0 1 2 3 4 5 6 7 8 ], [ 0 1 2 3 4 5 6 7 ].
  //
  // The reason why the code is different for axis == 0, is that in that case we
  // are really making visible an "implicit" axis of the input `src`; we could
  // call it axis 0 of the original RaggedShape.  Imagine that "implicit" axis's
  // row_splits and row_ids map respectively from an idx_minus1 -> idx0 and from
  // an idx_0 to idx_minus1, where idx_minus1 is always 0 and 0 <= idx0 <
  // Dim0().

  ContextPtr c = src.Context();
  K2_CHECK(axis >= 0 && axis <= src.NumAxes());

  const std::vector<RaggedShapeDim> &axes_in = src.Axes();

  int32_t num_axes_in = src.NumAxes();

  // Note: in RaggedShape, the vector of RaggedShapeDim is of length num_axes -
  // 1,
  // so the output will have one more axis than the input.
  std::vector<RaggedShapeDim> axes_out(num_axes_in);

  int32_t row_splits_dim, row_ids_dim;
  Array1<int32_t> mem;

  if (axis == 0) {
    row_splits_dim = 2;        // e.g. [ 0 5 ]
    row_ids_dim = src.Dim0();  // e.g. [ 0 0 0 0 0 ]
    mem = Array1<int32_t>(c, row_splits_dim + row_ids_dim);
    int32_t *mem_data = mem.Data();
    auto lambda_set_mem = [=] __host__ __device__(int32_t i) -> void {
      if (i == 1)
        mem_data[i] = row_ids_dim;
      else
        mem_data[i] = 0;
    };
    Eval(c, mem.Dim(), lambda_set_mem);
    axes_out[0].row_splits = mem.Range(0, 2);
  } else {
    int32_t tot_size = src.TotSize(axis - 1);
    row_splits_dim = tot_size + 1;
    row_ids_dim = tot_size;
    mem = Array1<int32_t>(c, row_splits_dim + row_ids_dim);
    int32_t *mem_data = mem.Data();
    auto lambda_set_mem2 = [=] __host__ __device__(int32_t i) -> void {
      mem_data[i] = i % (tot_size + 1);
    };
    Eval(c, mem.Dim(), lambda_set_mem2);
  }
  axes_out[axis].row_splits = mem.Range(0, row_splits_dim);
  axes_out[axis].row_ids = mem.Range(row_splits_dim, row_ids_dim);
  for (int32_t i = 0; i < axis; i++) axes_out[axis] = axes_in[axis];
  // Note: the returned array has `num_axes_in + 1` axes, so its
  // array of RaggedShapeDim is of length `num_axes_in`.
  for (int32_t i = axis + 1; i < num_axes_in; i++)
    axes_out[axis] = axes_in[axis - 1];
  return RaggedShape(axes_out);
}

RaggedShape Renumber(RaggedShape &src, const Array1<int32_t> &new2old) {
  ContextPtr c = src.Context();
  K2_CHECK(IsCompatible(src, new2old));
  int32_t num_axes = src.NumAxes(), dim0 = src.Dim0();
  K2_CHECK_EQ(new2old.Dim(), dim0);
  std::vector<int32_t> tot_sizes_out(num_axes);
  for (int32_t axis = 0; axis < num_axes; axis++)
    tot_sizes_out[axis] = src.TotSize(axis);
  // the arrays in `ans` will be the same sizes as those in `src`.
  RaggedShape ans = RaggedShapeFromTotSizes(num_axes, tot_sizes_out.data());

  src.Populate();
  Array2<int32_t> old_offsets(c, num_axes, dim0 + 1),
      new_offsets(c, num_axes, dim0 + 1);
  auto old_offsets_acc = old_offsets.Accessor(),
       new_offsets_acc = new_offsets.Accessor();

  Array1<int32_t *> row_splits_ptrs = GetRowSplitsPtr(src);
  int32_t **row_splits_ptrs_data = row_splits_ptrs.Data();

  // Set old_offsets
  auto lambda_get_old_offsets = [=] __host__ __device__(int32_t i) {
    // 0 <= i <= dim0
    int32_t cur_offset = i;
    for (int32_t axis = 0; axis < num_axes; axis++) {
      old_offsets_acc(0, i) = cur_offset;
      if (axis + 1 == num_axes) return;
      cur_offset = row_splits_ptrs_data[axis][cur_offset];
    }
  };
  Eval(c, dim0 + 1, lambda_get_old_offsets);
  const int32_t *new2old_data = new2old.Data();
  auto lambda_get_new_offsets = [=] __host__ __device__(int32_t axis,
                                                        int32_t new_i) {
    // 0 <= axis < num_axes;  0 <= new_i < dim0
    int32_t old_i = new2old_data[new_i],
            this_old_offset = old_offsets_acc(axis, old_i),
            next_old_offset = old_offsets_acc(axis, old_i + 1),
            size = next_old_offset - this_old_offset;
    new_offsets_acc(axis, new_i) = size;
  };
  Eval2(c, num_axes, dim0, lambda_get_new_offsets);
  ExclusiveSum(new_offsets, &new_offsets);
  // Now new_offsets contains the offsets, not the sizes.

  ParallelRunner pr(c);
  std::vector<hipStream_t> streams(num_axes);
  int32_t num_jobs = dim0 * 2;  // note: this formula is not a heuristic; it's
                                // how TaskRedirect works..
  Array2<TaskRedirect> task_redirects(c, num_axes, num_jobs);
  auto task_redirects_acc = task_redirects.Accessor();
  for (int32_t axis = 0; axis < num_axes; axis++) {
    streams[axis] = pr.NewStream();
    With w(streams[axis]);
    const int32_t *new_offsets_ptr = new_offsets_acc.Row(axis);
    TaskRedirect *task_redirect_ptr = task_redirects_acc.Row(axis);
    GetTaskRedirect(c, dim0, new_offsets_ptr, task_redirect_ptr);
  }

  for (int32_t axis = 0; axis < num_axes - 1; axis++) {
    {
      int32_t *this_new_row_splits = ans.RowSplits(axis).Data();
      const int32_t *this_old_row_splits = src.RowSplits(axis).Data();

      auto lambda_set_row_splits = [=] __host__ __device__(
                                       int32_t new_idx, int32_t num_threads,
                                       int32_t thread_idx) -> void {
        //  0 <= new_idx < dim0; and 0 <= thread_idx < num_threads,
        //  num_threads may have any value > 0 as far as this code is concerned.
        //
        // Reminder of how row_splits work dimensionally: they are a map
        // from, e.g. an idx0 to an idx01.   An offsets_acc(0,n) is
        // dimensionally an idx0; an offsets_acc(1,n) an idx01, and so on.
        // The locations in the row_splits array are as given by
        // the `axis`'th row of `offsets`; the values in the array
        // are related to those in the `axis+1`'th row.
        int32_t old_idx = new2old_data[new_idx],
                this_old_offset = old_offsets_acc(axis, old_idx),
                next_old_offset = old_offsets_acc(axis, old_idx + 1),
                this_new_offset = new_offsets_acc(axis, old_idx),
                num_rows = next_old_offset - this_old_offset,
                value_offset = new_offsets_acc(axis + 1, new_idx) -
                               old_offsets_acc(axis + 1, old_idx);

        // Using <= instead of < below causes threads for different src_idx to
        // write a single overlapping value, but also ensures that the
        // terminating value is written.  This only works because row_splits
        // vectors always start with 0, which is not necessarily the case
        // for row-ids.
        for (; thread_idx <= num_rows; thread_idx += num_threads) {
          this_new_row_splits[this_new_offset + thread_idx] =
              value_offset + this_old_row_splits[thread_idx];
        }
      };
      int32_t min_threads_per_job = 2, tot_work = tot_sizes_out[axis],
              target_num_loops = (tot_work > 1000000 ? 4 : 2);
      // bool include_final_task = false;
      EvalWithRedirect(streams[axis], num_jobs, task_redirects_acc.Row(axis),
                       min_threads_per_job, tot_work, target_num_loops,
                       lambda_set_row_splits);
    }

    {
      int32_t *this_new_row_ids = ans.RowIds(axis).Data();
      const int32_t *this_old_row_ids = src.RowIds(axis).Data();

      auto lambda_set_row_ids = [=] __host__ __device__(
                                    int32_t new_idx, int32_t num_threads,
                                    int32_t thread_idx) -> void {
        //  0 <= new_idx < dim0; and 0 <= thread_idx < num_threads,
        //  num_threads may have any value > 0 as far as this code is concerned.
        //
        // Reminder of how row_ids work dimensionally: they are a map
        // from, e.g. an idx01 to an idx0.   An offsets_acc(0,n) is
        // dimensionally an idx0; an offsets_acc(1,n) an idx01, and so on.
        // The locations in the row_ids array are as given by
        // the `axis+1`'th row of `offsets`; the values in the array
        // are related to those in the `axis`'th row.
        int32_t old_idx = new2old_data[new_idx],
                this_old_offset = old_offsets_acc(axis + 1, old_idx),
                next_old_offset = old_offsets_acc(axis + 1, old_idx + 1),
                this_new_offset = new_offsets_acc(axis + 1, old_idx),
                num_rows = next_old_offset - this_old_offset,
                value_offset = new_offsets_acc(axis, new_idx) -
                               old_offsets_acc(axis, old_idx);

        // Using <= instead of < below causes threads for different src_idx to
        // write a single overlapping value, but also ensures that the
        // terminating value is written.  This only works because row_splits
        // vectors always start with 0, which is not necessarily the case
        // for row-ids.
        for (; thread_idx < num_rows; thread_idx += num_threads) {
          this_new_row_ids[this_new_offset + thread_idx] =
              value_offset + this_old_row_ids[thread_idx];
        }
        // TODO: maybe remove this if I decide last value is not needed.
        if (new_idx == dim0 - 1 && thread_idx == num_rows) {
          int32_t next_value_offset = new_offsets_acc(axis, new_idx + 1) -
                                      old_offsets_acc(axis, old_idx + 1);
          this_new_row_ids[this_new_offset + thread_idx] = next_value_offset;
        }
      };
      int32_t min_threads_per_job = 2, tot_work = tot_sizes_out[axis],
              target_num_loops = (tot_work > 1000000 ? 4 : 2);
      EvalWithRedirect(streams[axis], num_jobs, task_redirects_acc.Row(axis),
                       min_threads_per_job, tot_work, target_num_loops,
                       lambda_set_row_ids);
    }
  }
#ifndef NDEBUG
  ans.Check();
#endif
  return ans;
}

/*
  Returns a CPU array of shape (src[0]->NumAxes()+1) by (num_srcs + 1), where
  each row is the exclusive-sum of the TotSize() of the respective sources,
  on the previous axis (or 1 for axis 0).  Specifically: it's the same
  as setting ans(i,j) to (i == 0 ? 1 : src[j]->TotSize(i-1)), and then
  doing an exclusive-sum on each row of i.

     @param [in] num_srcs  The number of `RaggedShape`s in `src`
     @param [in] src    The shapes whose sizes we want.  Must all have the
                      same NumAxes().
     @return   Returns a freshly allocated CPU Array2<int32_t> of dimension
               src[0]->NumAxes() by (num_srcs + 1), where each
               row is the exclusive-sum of the TotSize() of the respective
               sources, on that axis.  Its last column contains the totals.

 */
inline Array2<int32_t> GetOffsets(int32_t num_srcs, RaggedShape **src) {
  //  src_offsets[i,j]  == src_offsets.Data()[i*num_axes_in + j] contains:
  //          sum(0 <= k < i) src[k]->TotSize(j).
  int32_t num_axes_in = src[0]->NumAxes();
  Array2<int32_t> src_offsets(GetCpuContext(), num_srcs + 1, num_axes_in);
  int32_t *src_offsets_data = src_offsets.Data();
  int32_t src_offsets_stride0 = num_srcs + 1;
  DCHECK_EQ(src_offsets.ElemStride0(), src_offsets_stride0);

  for (int32_t axis = 0; axis < num_axes_in; axis++) {
    int32_t sum = 0;
    for (int32_t i = 0; i <= num_srcs; i++) {
      src_offsets_data[i * src_offsets_stride0 + axis] = sum;
      if (i < num_srcs) {
        sum += (axis == 0 ? 1 : src[i]->TotSize(axis - 1));
      }
    }
  }
  return src_offsets;
}

/*
  TODO: fix this documentation...

  Extract meta-info from the shape (this will include populating any row_ids and
  row_splits that were not already populated).  This is used inside algorithms
  when we need to transfer meta-info to GPU.

     @param [in]   src   Ragged shape that we're extracting meta-info from
     @param [out] row_splits  This will be set to an array of size
  src.NumAxes()-1,
                         containing pointers to the row_splits' Data() vectors.
                         The array will be on the same device as `src`.
     @param [out] row_splits  This will be set to an array of size
  src.NumAxes()-1,
                         containing pointers to the row_splits' Data() vectors.
                         The array will be on the same device as `src`.


     Host (i.e. CPU memory) array of RowInfo that we're
                         writing to (but the pointers inside them are to the
                         memory of the device used in shape.Context());
                         must be the start of an array of length
                         shape.NumAxes().
                         Note: element 0 of the output `ptrs` contains a
                         row_splits with [ 0, shape.Dim0() ]
                         and the row_ids are [ 0 0 0 (repeats shape.Dim0()
  times) 1 ]
*/
// outputs have dims src.NumAxes() - 1.
void GetRowInfo(RaggedShape &src, Array1<int32_t *> *row_splits,
                Array1<int32_t *> *row_ids) {
  // TODO
}
void GetRowInfoMulti(int32_t num_src, RaggedShape **src,
                     Array2<int32_t *> *row_splits,
                     Array2<int32_t *> *row_ids) {
  // TODO
}

RaggedShape Append(int32_t num_srcs, RaggedShape **src, int32_t axis) {
  K2_CHECK_EQ(axis, 0) << "Append() with axis > 0 not yet supported";
  K2_CHECK_GT(num_srcs, 0);

  int32_t num_axes = src[0]->NumAxes();
  ContextPtr c = src[0]->Context();
  for (int32_t i = 1; i < num_srcs; i++) {
    // Check they have the same num-axes.
    K2_CHECK_EQ(num_axes, src[i]->NumAxes());
    K2_CHECK(IsCompatible(*src[0], *src[i]));
  }

  // `offsets` will be on CPU for now.
  Array2<int32_t> offsets = GetOffsets(num_srcs, src);
  auto offsets_acc = offsets.Accessor();

  std::vector<int32_t> tot_sizes_out(num_axes);
  for (int32_t axis = 0; axis < num_axes; axis++)
    tot_sizes_out[axis] = offsets_acc(axis, num_srcs);

  RaggedShape ans = RaggedShapeFromTotSizes(num_axes, tot_sizes_out.data());
  Array1<int32_t *> dest_row_splits, dest_row_ids;
  GetRowInfo(ans, &dest_row_splits, &dest_row_ids);

  Array2<int32_t *> src_row_splits, src_row_ids;
  GetRowInfoMulti(num_srcs, src, &src_row_splits, &src_row_ids);

  if (c->GetDeviceType() != kCpu) offsets = offsets.To(c);

  int32_t **dest_row_splits_data = dest_row_splits.Data(),
          **dest_row_ids_data = dest_row_ids.Data();
  auto src_row_splits_acc = src_row_splits.Accessor(),
       src_row_ids_acc = src_row_ids.Accessor();
  offsets_acc = offsets.Accessor();  // on GPU now (if we're using one)

  ParallelRunner pr(c);
  std::vector<hipStream_t> streams(num_axes + 1);
  int32_t num_jobs = num_srcs * 2;
  // task_redirects is a device array (if using GPU).
  // We have `num_axes - 1` different sets of row_splits/row_ids to
  // populate but they have different sizes; the total number of distinct
  // sizes is `num_axes`.
  Array2<TaskRedirect> task_redirects(c, num_axes, num_jobs);
  auto task_redirects_acc = task_redirects.Accessor();
  // populate task_redirects (these allocate blocks of threads roughly
  // proportionally to the amount of data to process from this source.
  for (int32_t axis = 0; axis < num_axes; axis++) {
    streams[axis] = pr.NewStream();
    const int32_t *offsets = &(offsets_acc(axis, 0));
    GetTaskRedirect(c, num_srcs, offsets, task_redirects_acc.Row(axis));
  }

  for (int32_t axis = 0; axis < num_axes - 1; axis++) {
    // first set the row-splits.
    TaskRedirect *tr = &(task_redirects_acc(axis, 0));

    int32_t **this_src_row_splits = &(src_row_splits_acc(axis, 0)),
            **this_src_row_ids = &(src_row_ids_acc(axis, 0));
    int32_t *this_dest_row_splits = ans.RowSplits(axis + 1).Data(),
            *this_dest_row_ids = ans.RowIds(axis + 1).Data();
    const int32_t *offsets_this_axis = &(offsets_acc(axis, 0)),
                  *offsets_next_axis = &(offsets_acc(axis + 1, 0));
    auto lambda_set_row_splits = [=] __host__ __device__(
                                     int32_t src_idx, int32_t num_threads,
                                     int32_t thread_idx) -> void {
      // Reminder of how row_splits work dimensionally: they are a map
      // from, e.g. an idx0 to an idx01.   An offsets_acc(0,n) is
      // dimensionally an idx0; an offsets_acc(1,n) an idx01, and so on.
      int32_t this_offset = offsets_this_axis[src_idx],
              next_offset = offsets_this_axis[src_idx + 1],
              this_value_offset = offsets_next_axis[src_idx],
              num_rows = next_offset - this_offset;
      int32_t *src_row_splits_ptr = this_src_row_splits[src_idx];
      // Using <= instead of < below causes threads for different src_idx to
      // write a single overlapping value, but also ensures that the
      // terminating value is written.  This only works because row_splits
      // vectors always start with 0, which is not necessarily the case
      // for row-ids.
      for (; thread_idx <= num_rows; thread_idx += num_threads) {
        this_dest_row_splits[this_offset + thread_idx] =
            this_value_offset + src_row_splits_ptr[thread_idx];
      }
    };

    int32_t min_threads_per_job = 2, tot_work = tot_sizes_out[axis],
            target_num_loops = (tot_work > 1000000 ? 4 : 2);
    EvalWithRedirect(streams[axis], num_jobs, task_redirects_acc.Row(axis),
                     min_threads_per_job, tot_work, target_num_loops,
                     lambda_set_row_splits);

    {  // set the row-ids
      auto lambda_set_row_ids = [=] __host__ __device__(
                                    int32_t src_idx, int32_t num_threads,
                                    int32_t thread_idx) -> void {
        // Reminder of how row_ids work dimensionally: they are a map
        // from, e.g. an idx01 to an idx0.   An offsets_acc(0,n) is
        // dimensionally an idx0; an offsets_acc(1,n) an idx01, and so on.
        int32_t this_offset = offsets_next_axis[src_idx],
                next_offset = offsets_next_axis[src_idx + 1],
                this_value_offset = offsets_this_axis[src_idx],
                num_elems = next_offset - this_offset;
        int32_t *src_row_ids_ptr = this_src_row_ids[src_idx];
        // We need to write the very last value at the end of all the
        // arrays; the last job (for src_idx == num_srcs - 1) does this
        // by adding 1 to num_srcs.  We can't let them all write an
        // extra value, because unlike row_splits, row_ids vectors may not
        // start with 0 in general; so having 2 threads write that
        // value (the 1st of each; one past the last of each) would cause
        // indeterminacy.
        if (src_idx == num_srcs - 1) num_elems++;
        for (; thread_idx <= num_elems; thread_idx += num_threads) {
          this_dest_row_ids[this_offset + thread_idx] =
              this_value_offset + src_row_ids_ptr[thread_idx];
        }
      };
      int32_t min_threads_per_job = 2, tot_work = tot_sizes_out[axis + 1],
              target_num_loops = (tot_work > 1000000 ? 4 : 2);
      // bool include_final_task = false;
      EvalWithRedirect(streams[axis + 1], num_jobs,
                       task_redirects_acc.Row(axis + 1), min_threads_per_job,
                       tot_work, target_num_loops, lambda_set_row_ids);
    }
  }
  return ans;
}

/*
  Get some meta-info for an array of RaggedShape, and transfer them
  to the
  device that `src` is located on

     @param [in] num_src  Number of source arrays to process.
     @param [in] src      Source arrays.  Let num_axes be src[0]->NumAxes().
     @param [in] row_splits  Output array of row_splits pointers,
                          will be of dimension num_axes-1 by num_src
     @param [in] row_splits  Output array of row_splits pointers,
                          will be of dimension num_axes-1 by num_src
     @param [out] offsets   Output array of `offsets` pointers,
                          will be of dimension num_axes by num_src+1;
                          these are the exclusive-sum of the TotSize(axis)
                          of the respective sources.
     @param [out] tot_sizes  The last column of `offsets`, as a std::vector
*/
void GetInfoMulti(int32_t num_src, RaggedShape **src,
                  Array2<int32_t *> *row_splits, Array2<int32_t *> *row_ids,
                  Array2<int32_t *> *offsets, std::vector<int32_t> *tot_sizes);

struct RowInfoWithOffsets {
  int32_t *row_splits;
  int32_t *row_ids;
  int32_t num_rows;
  int32_t num_elems;
  int32_t row_splits_offset;
  int32_t row_ids_offset;
};

RaggedShape RemoveAxis(RaggedShape &src, int32_t axis) {
  CHECK_GT(src.NumAxes(), 2);
  CHECK(axis >= 0 && axis < src.NumAxes());

  // note, `axes` is of dim src.NumAxes() - 1.
  // Also note: axes_in[i] pertains to the relationship between
  // axes i and i+1 in the source.
  src.Populate();
  const std::vector<RaggedShapeDim> &axes_in = src.Axes();

  std::vector<RaggedShapeDim> axes_out(axes_in.size() - 1);

  for (int32_t i = 0; i < axis - 1; i++) axes_out[i] = axes_in[i];

  if (axis > 0 && axis + 1 < src.NumAxes()) {
    axes_out[axis - 1].row_ids =
        axes_in[axis - 1].row_ids[axes_in[axis].row_ids];
    axes_out[axis - 1].row_splits =
        axes_in[axis].row_splits[axes_in[axis - 1].row_splits];
  }
  for (int32_t i = axis; i < axes_out.size(); i++) axes_out[i] = axes_in[i + 1];
  return RaggedShape(axes_out);
}

// transpose axes 0 and 1.
RaggedShape Transpose(RaggedShape &src) {
  K2_CHECK(src.NumAxes() > 2);
  int32_t src_dim0 = src.Dim0(), src_tot_size1 = src.TotSize(1);
  int32_t src_dim1 = src_tot_size1 % src_dim0;
  if (src_tot_size1 % src_dim0 != 0) {
    K2_LOG(FATAL) << "Transpose(): all dims on axis 0 must be the same.";
  }
  RaggedShape src_no_axis0 = RemoveAxis(src, 0);
  assert(src_no_axis0.Dim0() == src_tot_size1);
  ContextPtr c = src.Context();
  // `renumbering` is a `new2old` map, that maps from the first index in
  // src_no_axis0_renumbered
  // to the first index into src_no_axis0.
  Array1<int32_t> renumbering(c, src_tot_size1);
  int32_t *renumbering_data = renumbering.Data();
  auto lambda_set_renumbering = [=] __host__ __device__(int32_t i) {
    int32_t j = i % src_dim1, k = i / src_dim1, i_old = j * src_dim0 + k;
    renumbering_data[i] = i_old;
  };
  Eval(c, src_tot_size1, lambda_set_renumbering);

  RaggedShape src_no_axis0_renumbered = Renumber(src_no_axis0, renumbering);

  int32_t num_rows = src_dim1, row_splits_dim = num_rows + 1,
          row_ids_dim = src_tot_size1;
  std::vector<RaggedShapeDim> ans_axis0(1);
  Array1<int32_t> mem(c, row_splits_dim + row_ids_dim);
  ans_axis0[0].row_splits = mem.Range(0, row_splits_dim);
  ans_axis0[0].row_ids = mem.Range(0, row_ids_dim);
  ans_axis0[0].cached_tot_size = row_ids_dim;

  int32_t *mem_data = mem.Data();
  auto lambda_set_row_info = [=] __host__ __device__(int32_t i) {
    int32_t val;
    if (i >= row_splits_dim) {
      int32_t elem_idx = i - row_splits_dim;
      val = elem_idx / src_dim0;
    } else {
      int32_t row_idx = i;
      val = row_idx * src_dim0;
    }
    mem_data[i] = val;
  };
  Eval(c, row_splits_dim + row_ids_dim, lambda_set_row_info);
  RaggedShape temp(ans_axis0);
  return ComposeRaggedShapes(temp, src_no_axis0_renumbered);
}

RaggedShape Stack(int32_t num_srcs, RaggedShape **src, int32_t axis) {
  CHECK_GT(num_srcs, 0);
  CHECK(axis >= 0 && axis <= 1);

  ContextPtr c = src[0]->Context();

  std::vector<RaggedShape> unsqueezed(num_srcs);
  std::vector<RaggedShape *> unsqueezed_ptrs(num_srcs);
  {
    ParallelRunner pr(c);
    for (int32_t i = 0; i < num_srcs; i++) {
      With w(pr.NewStream());
      unsqueezed[i] = Unsqueeze(*src[i], 0);
      unsqueezed_ptrs[i] = &unsqueezed[i];
    }
    // destructor will wait for work in those launched streams to finish.
    // (well it won't actually wait, but it will force the current stream to
    // wait.
  }

  RaggedShape ans = Append(num_srcs, &(unsqueezed_ptrs[0]), 0);
  if (axis == 1) ans = Transpose(ans);
  return ans;
}

}  // namespace k2
