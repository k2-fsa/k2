/**
 * @brief
 * ragged
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey, Haowen Qiu)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <hipcub/hipcub.hpp>
#include <vector>

#include "k2/csrc/array_ops.h"
#include "k2/csrc/math.h"
#include "k2/csrc/ragged.h"
namespace {

// will be used in RaggedShape::MaxSize(int32_t axis) to call
// hipcub::DeviceReduce::Max
struct RowSplitsDiff {
  const int32_t *row_splits_data;
  explicit RowSplitsDiff(const int32_t *row_splits)
      : row_splits_data(row_splits) {}
  // operator[] and operator+ are required by hipcub::DeviceReduce::Max
  __device__ int32_t operator[](int32_t i) const {
    return row_splits_data[i + 1] - row_splits_data[i];
  }
  __device__ RowSplitsDiff operator+(int32_t n) const {
    RowSplitsDiff tmp(*this);
    tmp.row_splits_data += n;
    return tmp;
  }
};

/*
A helper function used in RaggedShape3;
  if both first and second are non-NULL, it will check if the context of them
     is compatible or not and return that context if compatible;
  if one of them is NULL, returns the other one's context.
 */
static k2::ContextPtr GetContext(const k2::Array1<int32_t> *first,
                                 const k2::Array1<int32_t> *second) {
  K2_CHECK(first != nullptr || second != nullptr)
      << "At least one of first and second must be non-NULL";
  if (first == nullptr)
    return second->Context();
  else if (second == nullptr)
    return first->Context();
  else
    return k2::GetContext(*first, *second);
}

}  // namespace

namespace std {
// vaule_type is required by hipcub::DeviceReduce::Max
template <>
struct iterator_traits<::RowSplitsDiff> {
  typedef int32_t value_type;
};
}  // namespace std

namespace k2 {

RaggedShape RandomRaggedShape(bool set_row_ids, int32_t min_num_axes,
                              int32_t max_num_axes, int32_t min_num_elements,
                              int32_t max_num_elements) {
  ContextPtr c = GetCpuContext();
  K2_CHECK(min_num_axes >= 2 && max_num_axes >= min_num_axes &&
           min_num_elements >= 0 && max_num_elements >= min_num_elements);
  int32_t num_axes = RandInt(min_num_axes, max_num_axes);
  int32_t num_elements = RandIntGeometric(min_num_elements, max_num_elements);

  bool done_repeats = false;
  std::vector<RaggedShapeDim> axes(num_axes - 1);
  for (int32_t axis = num_axes - 2; axis >= 0; axis--) {
    // this axis will have row_ids of length num_elements and
    // row_splits of length to be determined.
    int32_t cur_row_split = 0;
    std::vector<int32_t> row_splits_vec;
    std::vector<int32_t> row_ids_vec;
    row_splits_vec.push_back(cur_row_split);
    // The reason for "|| RandInt(0, 2) == 0)" is so that even if there
    // are no elements we can still potentially generate empty row-splits.
    while (cur_row_split < num_elements || RandInt(0, 2) == 0) {
      int32_t split_size = RandIntGeometric(0, num_elements - cur_row_split);
      cur_row_split += split_size;
      // sometimes we have a bunch of empty rows in a row (this will test out
      // more of the code), so here we generate a bunch of empty rows, but we
      // just do this only once (that's why we declare `done_repeats` here).
      if (split_size == 0 && RandInt(0, 30) == 0 && !done_repeats) {
        int32_t num_repeats = RandIntGeometric(1, 128);
        row_splits_vec.insert(row_splits_vec.end(), num_repeats, cur_row_split);
        // don't need to set `row_ids_vec` as there's no element.
        done_repeats = true;
      }
      row_splits_vec.push_back(cur_row_split);
      if (set_row_ids) {
        int32_t cur_row = static_cast<int32_t>(row_splits_vec.size()) - 2;
        row_ids_vec.insert(row_ids_vec.end(), split_size, cur_row);
      }
    }
    axes[axis].row_splits = Array1<int32_t>(c, row_splits_vec);
    if (set_row_ids) axes[axis].row_ids = Array1<int32_t>(c, row_ids_vec);
    axes[axis].cached_tot_size = num_elements;
    num_elements = axes[axis].row_splits.Dim() - 1;
  }
  // RaggedShape(axes, true) will check the returned RaggedShape for
  // consistency.
  return RaggedShape(axes, true);
}

// Recursive function that prints (part of) a ragged shape.
// 0 <=  begin_pos <= end_pos < shape.TotSize(axis).

void PrintRaggedShapePart(std::ostream &stream, RaggedShape &shape,
                          int32_t axis, int32_t begin_pos, int32_t end_pos) {
  K2_CHECK(axis >= 0 && axis < shape.NumAxes() && begin_pos >= 0 &&
           begin_pos <= end_pos && end_pos <= shape.TotSize(axis));
  for (int32_t d = begin_pos; d < end_pos; ++d) {
    if (axis == shape.NumAxes() - 1) {
      stream << d << " ";
    } else {
      stream << "[ ";
      const int32_t *row_splits = shape.RowSplits(axis + 1).Data();
      K2_DCHECK(d < shape.RowSplits(axis + 1).Dim());
      int32_t row_start = row_splits[d], row_end = row_splits[d + 1];
      PrintRaggedShapePart(stream, shape, axis + 1, row_start, row_end);
      stream << "] ";
    }
  }
}

// prints a RaggedShape as e.g. [ [ 0 1 ] [ 2 ] [] ].  Note, the 'values'
// are just the positions in the array, this is for readability.
std::ostream &operator<<(std::ostream &stream, RaggedShape &shape) {
  if (shape.Context()->GetDeviceType() != kCpu) {
    return stream << shape.To(GetCpuContext());
  } else {
    stream << "[ ";
    PrintRaggedShapePart(stream, shape, 0, 0, shape.Dim0());
    stream << "]";
    return stream;
  }
}

Array1<int32_t> &RaggedShape::RowIds(int32_t axis) {
  K2_CHECK_GT(axis, 0);
  K2_CHECK_LT(axis, NumAxes());
  RaggedShapeDim &rsd = axes_[axis - 1];
  auto &row_splits = rsd.row_splits;
  auto &row_ids = rsd.row_ids;
  // there must be row_splits.Dim() >=1 according to the definition of
  // RaggedShapeDim.
  K2_CHECK_GE(row_splits.Dim(), 1);
  if (!row_ids.IsValid()) {
    if (rsd.cached_tot_size < 0)
      rsd.cached_tot_size = row_splits[row_splits.Dim() - 1];
    // create row_ids as it does not exist
    row_ids = Array1<int32_t>(Context(), rsd.cached_tot_size);
    const int32_t *row_splits_data = row_splits.Data();
    int32_t *row_ids_data = row_ids.Data();
    RowSplitsToRowIds(Context(), row_splits.Dim() - 1, row_splits_data,
                      row_ids.Dim(), row_ids_data);
  }
  return row_ids;
}

int32_t RaggedShape::MaxSize(int32_t axis) {
  K2_CHECK_GT(axis, 0);
  K2_CHECK_LT(axis, NumAxes());
  const auto &row_splits = axes_[axis - 1].row_splits;
  const int32_t num_rows = row_splits.Dim() - 1;
  if (num_rows == 0) return 0;
  const int32_t *row_splits_data = row_splits.Data();
  ContextPtr c = Context();
  if (c->GetDeviceType() == kCpu) {
    int32_t max_value = 0;
    for (int32_t i = 0; i < num_rows; ++i) {
      int32_t value = row_splits_data[i + 1] - row_splits_data[i];
      if (value > max_value) max_value = value;
    }
    return max_value;
  } else {
    K2_CHECK_EQ(c->GetDeviceType(), kCuda);
    ::RowSplitsDiff row_splits_diff(row_splits_data);
    Array1<int32_t> max_array(Context(), 1, 0);
    int32_t *max_value = max_array.Data();

    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    // the first time is to determine temporary device storage requirements
    K2_CUDA_SAFE_CALL(hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes,
                                             row_splits_diff, max_value,
                                             num_rows, c->GetCudaStream()));
    void *deleter_context;
    d_temp_storage = c->Allocate(temp_storage_bytes, &deleter_context);
    K2_CUDA_SAFE_CALL(hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes,
                                             row_splits_diff, max_value,
                                             num_rows, c->GetCudaStream()));
    c->Deallocate(d_temp_storage, deleter_context);
    // this will convert to memory on CPU
    return max_array[0];
  }
}

RaggedShape RaggedShape::Index(int32_t axis, int32_t i) {
  // only support `axis == 0` for now
  K2_CHECK_EQ(axis, 0);
  K2_CHECK_GE(i, 0);
  int32_t num_axes = NumAxes();
  K2_CHECK_GE(num_axes, 2);
  const auto &src_axes = Axes();
  K2_CHECK_LT(i + 1, src_axes[0].row_splits.Dim());

  int32_t idx = src_axes[0].row_splits[i];
  int32_t idx_next = src_axes[0].row_splits[i + 1];
  std::vector<RaggedShapeDim> axes(src_axes.size() - 1);
  ContextPtr c = Context();
  for (int32_t i = 2; i < num_axes; ++i) {
    const Array1<int32_t> &src_row_splits = src_axes[i - 1].row_splits;
    int32_t num_rows = idx_next - idx;
    int32_t offset = idx;
    idx = src_row_splits[idx];
    idx_next = src_row_splits[idx_next];
    // allocate new memory here as we need to change the values,
    // i.e. subtracts the offset.
    axes[i - 2].row_splits = Array1<int32_t>(c, num_rows + 1);
    int32_t *data = axes[i - 2].row_splits.Data();
    const int32_t *src_data = src_row_splits.Data();
    auto lambda_set_values = [=] __host__ __device__(int32_t i) -> void {
      data[i] = src_data[i + offset] - idx;
    };
    Eval(c, num_rows + 1, lambda_set_values);
    // leave row_ids and cached_tot_size unset
    axes[i - 2].cached_tot_size = -1;
  }
  RaggedShape shape(axes, true);
  return shape;
}

void RaggedShape::Populate() {
  int32_t num_axes = NumAxes();
  for (int32_t i = 1; i < num_axes; ++i) {
    // ignore return values of the following calls.
    this->TotSize(i);
    this->RowIds(i);
  }
}

RaggedShape RaggedShape::To(ContextPtr ctx) const {
  if (ctx->IsCompatible(*Context())) return *this;
  std::vector<RaggedShapeDim> axes(axes_.size());
  int32_t num_axes = NumAxes();
  for (int32_t i = 1; i < num_axes; ++i) {
    axes[i - 1].row_splits = axes_[i - 1].row_splits.To(ctx);
    // leave row_ids and cached_tot_size unset
    axes[i - 1].cached_tot_size = -1;
  }
  return RaggedShape(axes);
}

RaggedShapeIndexIterator RaggedShape::Iterator() {
  return RaggedShapeIndexIterator(*this);
}

int32_t RaggedShape::operator[](const std::vector<int32_t> &indexes) {
  K2_CHECK(indexes.size() == NumAxes());
  K2_CHECK(Context()->GetDeviceType() == kCpu);
  int32_t cur_idx = indexes[0];
  for (int32_t i = 1; i < NumAxes(); i++) {
    Array1<int32_t> &row_splits = axes_[i - 1].row_splits;
    K2_CHECK(cur_idx >= 0 && cur_idx + 1 < row_splits.Dim());
    cur_idx = row_splits[cur_idx];
    cur_idx += indexes[i];
  }
  return cur_idx;
}

int32_t RaggedShape::TotSize(int32_t axis) const {
  K2_CHECK_GE(axis, 0);
  K2_CHECK_LT(axis, NumAxes());
  if (axis == 0)
    return Dim0();
  else {
    const RaggedShapeDim &rsd = axes_[axis - 1];
    if (rsd.cached_tot_size >= 0) {
      return rsd.cached_tot_size;
    } else {
      // if we had row_ids set up, we should have set cached_tot_size.
      K2_CHECK_EQ(rsd.row_ids.Dim(), 0);
      K2_CHECK_GT(rsd.row_splits.Dim(), 0);
      const_cast<RaggedShapeDim &>(rsd).cached_tot_size = rsd.row_splits.Back();
      return rsd.cached_tot_size;
    }
  }
}

// TODO(dan): change this so that on error it prints a warning if
// print_warnings==true, and then returns false.
bool RaggedShape::Validate(bool print_warnings) {
  ContextPtr c = Context();
  int32_t num_axes = axes_.size();
  for (int32_t axis = 0; axis < num_axes; ++axis) {
    RaggedShapeDim &rsd = axes_[axis];
    K2_CHECK_GE(rsd.row_splits.Dim(), 0);
    if (rsd.cached_tot_size >= 0) {
      K2_CHECK(rsd.row_splits.Dim() == 0 ||
               rsd.cached_tot_size == rsd.row_splits.Back());
      K2_CHECK(rsd.row_ids.Dim() == 0 ||
               rsd.cached_tot_size == rsd.row_ids.Dim());
    } else {
      K2_CHECK_EQ(rsd.cached_tot_size, -1);
      K2_CHECK_EQ(rsd.row_ids.Dim(), 0);
    }

    int32_t num_elems;
    // Check row_splits.
    {
      // meta[0] is a bool, ok == 1, not-ok == 0.
      // meta[1] will contain the number of row_splits.
      Array1<int32_t> meta(c, 2, 1);
      int32_t *ok_data = meta.Data(), *num_elems_data = ok_data + 1;
      const int32_t *row_splits_data = rsd.row_splits.Data();
      int32_t num_rows = rsd.row_splits.Dim() - 1;

      auto lambda_check_row_splits =
          [=] __host__ __device__(int32_t i) -> void {
        int32_t this_idx = row_splits_data[i];
        if (i == 0 && this_idx != 0) *ok_data = 0;
        if (i < num_rows) {
          int32_t next_idx = row_splits_data[i + 1];
          if (next_idx < this_idx) *ok_data = 0;
        } else {
          K2_CHECK(i == num_rows);
          *num_elems_data = this_idx;
        }
      };
      Eval(c, num_rows + 1, lambda_check_row_splits);
      meta = meta.To(GetCpuContext());
      num_elems = meta[1];
      int32_t ok = meta[0];
      if (!ok) {
        K2_LOG(FATAL) << "Problem validating row-splits: for axes_[" << axis
                      << "], row_splits = " << rsd.row_splits;
      }
      if (rsd.cached_tot_size > 0 && rsd.cached_tot_size != num_elems) {
        K2_LOG(FATAL) << "Problem validating row-splits: for axes_[" << axis
                      << "], row_splits[-1] = " << num_elems
                      << " but cached_tot_size == " << rsd.cached_tot_size;
      }
    }
    if (axis + 1 < num_axes) {
      int32_t next_num_rows = axes_[axis + 1].row_splits.Dim() - 1;
      if (num_elems != next_num_rows) {
        K2_LOG(FATAL) << "Ragged shape has num_elems for axes_[" << axis
                      << "] == " << num_elems << " and num-rows for axes_["
                      << (axis + 1) << "] == " << next_num_rows;
      }
    }

    if (rsd.row_ids.Dim() != 0) {  // check row_ids.
      K2_CHECK(IsCompatible(rsd.row_ids, rsd.row_splits));
      // 1st elem is `ok` (1 or 0); 2nd elem is location of bad index
      // into row_splits
      Array1<int32_t> meta(c, 2, 1);
      int32_t *ok_data = meta.Data(), *bad_index_data = ok_data + 1;

      const int32_t *row_splits_data = rsd.row_splits.Data(),
                    *row_ids_data = rsd.row_ids.Data();
      int32_t num_elems_from_row_ids = rsd.row_ids.Dim(),
              num_rows = rsd.row_splits.Dim() - 1;

      K2_CHECK_EQ(num_elems, num_elems_from_row_ids);
      auto lambda_check_row_ids = [=] __host__ __device__(int32_t i) -> void {
        int32_t this_row = row_ids_data[i];
        if (this_row < 0 || this_row >= num_rows ||
            i < row_splits_data[this_row] ||
            i >= row_splits_data[this_row + 1]) {
          *ok_data = 0;
          *bad_index_data = i;
        }
      };
      // TODO: could do this and the other one in separate streams.
      Eval(c, num_elems, lambda_check_row_ids);
      meta = meta.To(GetCpuContext());  // since we have 2 accesses, this should
                                        // be faster.
      int32_t ok = meta[0];
      if (!ok) {
        K2_LOG(FATAL) << "Problem validating row-ids: for axes_[" << axis
                      << "], row_splits = " << rsd.row_splits
                      << ", row_ids = " << rsd.row_ids << ", see index "
                      << meta[1] << " of row_ids, whose dim is "
                      << rsd.row_ids.Dim();
      }
    }
    if (axis + 1 < axes_.size()) {
      K2_CHECK(IsCompatible(rsd.row_splits, axes_[axis + 1].row_splits));
    }
  }
  return true;
}

RaggedShape RaggedShape2(Array1<int32_t> *row_splits, Array1<int32_t> *row_ids,
                         int32_t cached_tot_size) {
  K2_CHECK(row_splits != nullptr || row_ids != nullptr)
      << "At least one of row_splits and row_ids must be defined";
  ContextPtr ctx = ::GetContext(row_splits, row_ids);
  if (cached_tot_size != -1) {
    if (row_ids != nullptr) K2_CHECK_EQ(cached_tot_size, row_ids->Dim());
    if (row_splits != nullptr) {
      // may be slow as it may copy memory from device to host
      K2_DCHECK_EQ(cached_tot_size, row_splits->Back());
    }
  }
  std::vector<RaggedShapeDim> axes(1);
  if (row_splits != nullptr) {
    axes[0].row_splits = *row_splits;
  } else {
    // we need to work out row_splits as we always require row_splits is not
    // empty for RaggedShape. Note here we suppose the last element in row_ids
    // is num_rows - 1, i.e. there's no empty rows after row `row_ids[-1]`.
    int32_t num_rows = row_ids->Dim() == 0 ? 0 : row_ids->Back() + 1;
    Array1<int32_t> row_splits_array(ctx, num_rows + 1);
    RowIdsToRowSplits(*row_ids, row_splits_array);
    axes[0].row_splits = row_splits_array;
  }
  if (row_ids != nullptr) axes[0].row_ids = *row_ids;
  if (cached_tot_size == -1) {
    cached_tot_size =
        row_ids != nullptr ? row_ids->Dim() : axes[0].row_splits.Back();
  }
  axes[0].cached_tot_size = cached_tot_size;
  // note below line will check if row_splits and row_ids are valid and agree
  // with each other.
  return RaggedShape(axes);
}

RaggedShape ComposeRaggedShapes(const RaggedShape &a, const RaggedShape &b) {
  if (a.NumElements() != b.Dim0()) {
    K2_LOG(FATAL) << "ComposeRaggedShapes: shape mismatch: " << a.NumElements()
                  << " vs. " << b.Dim0();
  }
  const auto &a_axes = a.Axes();
  const auto &b_axes = b.Axes();
  std::vector<RaggedShapeDim> axes(a_axes.size() + b_axes.size());
  std::size_t a_size = a_axes.size(), b_size = b_axes.size();
  for (std::size_t i = 0; i < a_size; ++i) axes[i] = a_axes[i];
  for (std::size_t i = 0; i < b_size; ++i) axes[i + a_size] = b_axes[i];
  return RaggedShape(axes);
}

RaggedShape RaggedShape3(Array1<int32_t> *row_splits1,
                         Array1<int32_t> *row_ids1, int32_t cached_tot_size1,
                         Array1<int32_t> *row_splits2,
                         Array1<int32_t> *row_ids2, int32_t cached_tot_size2) {
  K2_CHECK(row_splits1 != nullptr || row_ids1 != nullptr)
      << "At least one of row_splits1 and row_ids1 must be defined";
  K2_CHECK(row_splits2 != nullptr || row_ids2 != nullptr)
      << "At least one of row_splits2 and row_ids2 must be defined";

  // check context
  ContextPtr ctx1 = ::GetContext(row_splits1, row_ids1);
  ContextPtr ctx2 = ::GetContext(row_splits2, row_ids2);
  K2_CHECK(ctx1->IsCompatible(*ctx2));

  // check row_splits and row_ids of axis-1
  if (cached_tot_size1 != -1) {
    if (row_ids1 != nullptr) K2_CHECK_EQ(cached_tot_size1, row_ids1->Dim());
    if (row_splits1 != nullptr) {
      // may be slow as it may copy memory from device to host
      K2_DCHECK_EQ(cached_tot_size1, row_splits1->Back());
    }
  }

  // check row_splits and row_ids of axis-2
  if (cached_tot_size2 != -1) {
    if (row_ids2 != nullptr) K2_CHECK_EQ(cached_tot_size2, row_ids2->Dim());
    if (row_splits2 != nullptr) {
      // may be slow as it may copy memory from device to host
      K2_DCHECK_EQ(cached_tot_size2, row_splits2->Back());
    }
  }

  std::vector<RaggedShapeDim> axes(2);
  // set row_splits and row_ids for axis 1
  if (row_splits1 != nullptr) {
    axes[0].row_splits = *row_splits1;
  } else {
    // work out row_splits1, see code in RaggedShape2 above for the reason
    int32_t num_rows = row_ids1->Dim() == 0 ? 0 : row_ids1->Back() + 1;
    Array1<int32_t> row_splits_array(ctx1, num_rows + 1);
    RowIdsToRowSplits(*row_ids1, row_splits_array);
    axes[0].row_splits = row_splits_array;
  }
  if (row_ids1 != nullptr) axes[0].row_ids = *row_ids1;
  if (cached_tot_size1 == -1) {
    cached_tot_size1 =
        row_ids1 != nullptr ? row_ids1->Dim() : axes[0].row_splits.Back();
  }
  axes[0].cached_tot_size = cached_tot_size1;

  // set row_splits and row_ids for axis 2
  if (row_splits2 != nullptr) {
    axes[1].row_splits = *row_splits2;
  } else {
    // work out row_splits1, see code in RaggedShape2 above for the reason
    int32_t num_rows = row_ids2->Dim() == 0 ? 0 : row_ids2->Back() + 1;
    Array1<int32_t> row_splits_array(ctx1, num_rows + 1);
    RowIdsToRowSplits(*row_ids2, row_splits_array);
    axes[1].row_splits = row_splits_array;
  }
  if (row_ids2 != nullptr) axes[1].row_ids = *row_ids2;
  if (cached_tot_size2 == -1) {
    cached_tot_size2 =
        row_ids2 != nullptr ? row_ids2->Dim() : axes[1].row_splits.Back();
  }
  axes[1].cached_tot_size = cached_tot_size2;

  // we don't check here if
  // row_splits1[row_splits1.Dim() - 1] == row_ids1.Dim()
  //   == (row_splits2.Dim() - 1)
  //   >= (row_ids2[row_ids2.Dim() - 1] + 1)
  // but RaggedShape(axes) below will check this.
  return RaggedShape(axes);
}

RaggedShape RaggedShapeFromTotSizes(ContextPtr &c, int32_t num_axes,
                                    int32_t *tot_sizes) {
  K2_CHECK_GE(num_axes, 2);
  std::vector<RaggedShapeDim> axes(num_axes - 1);
  // In future we might choose to allocate everything in one big array, to avoid
  // multiple allocations, but for now just do it the simple way.
  for (int32_t axis = 1; axis < num_axes; ++axis) {
    axes[axis - 1].row_splits = Array1<int32_t>(c, tot_sizes[axis - 1] + 1);
    axes[axis - 1].row_ids = Array1<int32_t>(c, tot_sizes[axis]);
    axes[axis - 1].cached_tot_size = tot_sizes[axis];
  }
  // Not check here as we did not set the values of row_splits and row_ids
  return RaggedShape(axes, false);
}

Array1<int32_t *> GetRowSplitsPtr(RaggedShape &src) {
  int32_t axes = src.NumAxes();
  K2_CHECK_GE(axes, 2);
  std::vector<int32_t *> row_splits_start(axes - 1);
  for (int32_t i = 1; i != axes; ++i) {
    Array1<int32_t> &cur_splits = src.RowSplits(i);
    row_splits_start[i - 1] = cur_splits.Data();
  }
  return Array1<int32_t *>(src.Context(), row_splits_start);
}

// See declaration in ragged.h for documentation of its purpose and interface.
RaggedShape Unsqueeze(const RaggedShape &src, int32_t axis) {
  // If axis == 0, initial row_splits and row_ids will look like the following,
  // if for example src.Dim0() was 5: [ 0 5 ],  [ 0 0 0 0 0 ].  The other axes
  // would be pushed forward.
  //
  // If 0 < axis <= src.NumAxes(), the inserted row_splits and row_ids would
  // look like the following, if for instance the src.TotSize(axis) = 8:
  //   [ 0 1 2 3 4 5 6 7 8 ], [ 0 1 2 3 4 5 6 7 ].
  //
  // The reason why the code is different for axis == 0, is that in that case we
  // are really making visible an "implicit" axis of the input `src`; we could
  // call it axis 0 of the original RaggedShape.  Imagine that "implicit" axis's
  // row_splits and row_ids map respectively from an idx_minus1 -> idx0 and from
  // an idx_0 to idx_minus1, where idx_minus1 is always 0 and 0 <= idx0 <
  // Dim0().

  ContextPtr c = src.Context();
  K2_CHECK(axis >= 0 && axis <= src.NumAxes());

  const std::vector<RaggedShapeDim> &axes_in = src.Axes();
  int32_t num_axes_in = src.NumAxes();

  // Note: in RaggedShape, the vector of RaggedShapeDim is of length
  // num_axes - 1, so the output will have one more axis than the input.
  std::vector<RaggedShapeDim> axes_out(num_axes_in);

  int32_t row_splits_dim, row_ids_dim;
  Array1<int32_t> mem;

  if (axis == 0) {
    row_splits_dim = 2;        // e.g. [ 0 5 ]
    row_ids_dim = src.Dim0();  // e.g. [ 0 0 0 0 0 ]
    mem = Array1<int32_t>(c, row_splits_dim + row_ids_dim);
    int32_t *mem_data = mem.Data();
    auto lambda_set_mem = [=] __host__ __device__(int32_t i) -> void {
      if (i == 1)
        mem_data[i] = row_ids_dim;
      else
        mem_data[i] = 0;
    };
    Eval(c, mem.Dim(), lambda_set_mem);
  } else {
    int32_t tot_size = src.TotSize(axis);
    row_splits_dim = tot_size + 1;
    row_ids_dim = tot_size;
    mem = Array1<int32_t>(c, row_splits_dim + row_ids_dim);
    int32_t *mem_data = mem.Data();
    auto lambda_set_mem2 = [=] __host__ __device__(int32_t i) -> void {
      mem_data[i] = i % (tot_size + 1);
    };
    Eval(c, mem.Dim(), lambda_set_mem2);
  }
  axes_out[axis].row_splits = mem.Range(0, row_splits_dim);
  axes_out[axis].row_ids = mem.Range(row_splits_dim, row_ids_dim);
  axes_out[axis].cached_tot_size = row_ids_dim;
  for (int32_t i = 0; i < axis; ++i) axes_out[i] = axes_in[i];
  // Note: the returned array has `num_axes_in + 1` axes, so its
  // array of RaggedShapeDim is of length `num_axes_in`.
  for (int32_t i = axis + 1; i < num_axes_in; ++i) axes_out[i] = axes_in[i - 1];
  return RaggedShape(axes_out);
}

RaggedShape Renumber(RaggedShape &src, const Array1<int32_t> &new2old) {
  ContextPtr c = src.Context();
  K2_CHECK(IsCompatible(src, new2old));
  int32_t num_axes = src.NumAxes(), dim0 = src.Dim0();
  K2_CHECK_EQ(new2old.Dim(), dim0);
  std::vector<int32_t> tot_sizes_out(num_axes);
  for (int32_t axis = 0; axis < num_axes; axis++)
    tot_sizes_out[axis] = src.TotSize(axis);
  // the arrays in `ans` will be the same sizes as those in `src`.
  RaggedShape ans = RaggedShapeFromTotSizes(c, num_axes, tot_sizes_out.data());

  src.Populate();
  Array2<int32_t> old_offsets(c, num_axes, dim0 + 1),
      new_offsets(c, num_axes, dim0 + 1);
  auto old_offsets_acc = old_offsets.Accessor(),
       new_offsets_acc = new_offsets.Accessor();

  Array1<int32_t *> row_splits_ptrs = GetRowSplitsPtr(src);
  int32_t **row_splits_ptrs_data = row_splits_ptrs.Data();

  // Set old_offsets
  auto lambda_get_old_offsets = [=] __host__ __device__(int32_t i) {
    // 0 <= i <= dim0
    int32_t cur_offset = i;
    for (int32_t axis = 0; axis < num_axes; axis++) {
      old_offsets_acc(0, i) = cur_offset;
      if (axis + 1 == num_axes) return;
      cur_offset = row_splits_ptrs_data[axis][cur_offset];
    }
  };
  Eval(c, dim0 + 1, lambda_get_old_offsets);
  const int32_t *new2old_data = new2old.Data();
  auto lambda_get_new_offsets = [=] __host__ __device__(int32_t axis,
                                                        int32_t new_i) {
    // 0 <= axis < num_axes;  0 <= new_i < dim0
    int32_t old_i = new2old_data[new_i],
            this_old_offset = old_offsets_acc(axis, old_i),
            next_old_offset = old_offsets_acc(axis, old_i + 1),
            size = next_old_offset - this_old_offset;
    new_offsets_acc(axis, new_i) = size;
  };
  Eval2(c, num_axes, dim0, lambda_get_new_offsets);
  ExclusiveSum(new_offsets, &new_offsets);
  // Now new_offsets contains the offsets, not the sizes.

  ParallelRunner pr(c);
  std::vector<hipStream_t> streams(num_axes);
  int32_t num_jobs = dim0 * 2;  // note: this formula is not a heuristic; it's
                                // how TaskRedirect works..
  Array2<TaskRedirect> task_redirects(c, num_axes, num_jobs);
  auto task_redirects_acc = task_redirects.Accessor();
  for (int32_t axis = 0; axis < num_axes; axis++) {
    streams[axis] = pr.NewStream();
    With w(streams[axis]);
    const int32_t *new_offsets_ptr = new_offsets_acc.Row(axis);
    TaskRedirect *task_redirect_ptr = task_redirects_acc.Row(axis);
    GetTaskRedirect(c, dim0, new_offsets_ptr, task_redirect_ptr);
  }

  for (int32_t axis = 0; axis < num_axes - 1; axis++) {
    {
      int32_t *this_new_row_splits = ans.RowSplits(axis).Data();
      const int32_t *this_old_row_splits = src.RowSplits(axis).Data();

      auto lambda_set_row_splits = [=] __host__ __device__(
                                       int32_t new_idx, int32_t num_threads,
                                       int32_t thread_idx) -> void {
        //  0 <= new_idx < dim0; and 0 <= thread_idx < num_threads,
        //  num_threads may have any value > 0 as far as this code is concerned.
        //
        // Reminder of how row_splits work dimensionally: they are a map
        // from, e.g. an idx0 to an idx01.   An offsets_acc(0,n) is
        // dimensionally an idx0; an offsets_acc(1,n) an idx01, and so on.
        // The locations in the row_splits array are as given by
        // the `axis`'th row of `offsets`; the values in the array
        // are related to those in the `axis+1`'th row.
        int32_t old_idx = new2old_data[new_idx],
                this_old_offset = old_offsets_acc(axis, old_idx),
                next_old_offset = old_offsets_acc(axis, old_idx + 1),
                this_new_offset = new_offsets_acc(axis, old_idx),
                num_rows = next_old_offset - this_old_offset,
                value_offset = new_offsets_acc(axis + 1, new_idx) -
                               old_offsets_acc(axis + 1, old_idx);

        // Using <= instead of < below causes threads for different src_idx to
        // write a single overlapping value, but also ensures that the
        // terminating value is written.  This only works because row_splits
        // vectors always start with 0, which is not necessarily the case
        // for row-ids.
        for (; thread_idx <= num_rows; thread_idx += num_threads) {
          this_new_row_splits[this_new_offset + thread_idx] =
              value_offset + this_old_row_splits[thread_idx];
        }
      };
      int32_t min_threads_per_job = 2, tot_work = tot_sizes_out[axis],
              target_num_loops = (tot_work > 1000000 ? 4 : 2);
      // bool include_final_task = false;
      EvalWithRedirect(streams[axis], num_jobs, task_redirects_acc.Row(axis),
                       min_threads_per_job, tot_work, target_num_loops,
                       lambda_set_row_splits);
    }

    {
      int32_t *this_new_row_ids = ans.RowIds(axis).Data();
      const int32_t *this_old_row_ids = src.RowIds(axis).Data();

      auto lambda_set_row_ids = [=] __host__ __device__(
                                    int32_t new_idx, int32_t num_threads,
                                    int32_t thread_idx) -> void {
        //  0 <= new_idx < dim0; and 0 <= thread_idx < num_threads,
        //  num_threads may have any value > 0 as far as this code is concerned.
        //
        // Reminder of how row_ids work dimensionally: they are a map
        // from, e.g. an idx01 to an idx0.   An offsets_acc(0,n) is
        // dimensionally an idx0; an offsets_acc(1,n) an idx01, and so on.
        // The locations in the row_ids array are as given by
        // the `axis+1`'th row of `offsets`; the values in the array
        // are related to those in the `axis`'th row.
        int32_t old_idx = new2old_data[new_idx],
                this_old_offset = old_offsets_acc(axis + 1, old_idx),
                next_old_offset = old_offsets_acc(axis + 1, old_idx + 1),
                this_new_offset = new_offsets_acc(axis + 1, old_idx),
                num_rows = next_old_offset - this_old_offset,
                value_offset = new_offsets_acc(axis, new_idx) -
                               old_offsets_acc(axis, old_idx);

        // Using <= instead of < below causes threads for different src_idx to
        // write a single overlapping value, but also ensures that the
        // terminating value is written.  This only works because row_splits
        // vectors always start with 0, which is not necessarily the case
        // for row-ids.
        for (; thread_idx < num_rows; thread_idx += num_threads) {
          this_new_row_ids[this_new_offset + thread_idx] =
              value_offset + this_old_row_ids[thread_idx];
        }
        // TODO: maybe remove this if I decide last value is not needed.
        if (new_idx == dim0 - 1 && thread_idx == num_rows) {
          int32_t next_value_offset = new_offsets_acc(axis, new_idx + 1) -
                                      old_offsets_acc(axis, old_idx + 1);
          this_new_row_ids[this_new_offset + thread_idx] = next_value_offset;
        }
      };
      int32_t min_threads_per_job = 2, tot_work = tot_sizes_out[axis],
              target_num_loops = (tot_work > 1000000 ? 4 : 2);
      EvalWithRedirect(streams[axis], num_jobs, task_redirects_acc.Row(axis),
                       min_threads_per_job, tot_work, target_num_loops,
                       lambda_set_row_ids);
    }
  }
#ifndef NDEBUG
  ans.Check();
#endif
  return ans;
}

Array2<int32_t> GetOffsets(int32_t num_srcs, RaggedShape **src) {
  K2_CHECK_GT(num_srcs, 0);
  int32_t num_axes_in = src[0]->NumAxes();
  ContextPtr ctx = src[0]->Context();
  Array2<int32_t> src_offsets(GetCpuContext(), num_axes_in + 1, num_srcs + 1);
  int32_t *src_offsets_data = src_offsets.Data();
  int32_t src_offsets_stride0 = src_offsets.ElemStride0();

  // Check if they have same num-axes and compatible context
  for (int32_t i = 1; i < num_srcs; ++i) {
    K2_CHECK_EQ(src[i]->NumAxes(), num_axes_in);
    K2_CHECK(ctx->IsCompatible(*src[i]->Context()));
  }

  for (int32_t axis = 0; axis <= num_axes_in; ++axis) {
    int32_t sum = 0;
    for (int32_t i = 0; i <= num_srcs; ++i) {  // i is the column
      src_offsets_data[axis * src_offsets_stride0 + i] = sum;
      if (i < num_srcs) {
        sum += (axis == 0 ? 1 : src[i]->TotSize(axis - 1));
      }
    }
  }
  return src_offsets;
}

/*
  Extract meta-info from the shape (this will include populating any row_ids and
  row_splits that were not already populated).  This is used inside algorithms
  when we need to transfer meta-info to GPU.

     @param [in]   src   Ragged shape that we're extracting meta-info from
     @param [out] row_splits  This will be set to an array of size
                              src.NumAxes()-1, containing pointers to the
                              row_splits' Data() vectors. The array will be
                              allocated on the same device as `src`.
     @param [out] row_ids     This will be set to an array of size
                              src.NumAxes()-1, containing pointers to the
                              row_ids' Data() vectors. The array will be
                              allocated on the same device as `src`.
*/
void GetRowInfo(RaggedShape &src, Array1<int32_t *> *row_splits,
                Array1<int32_t *> *row_ids) {
  int32_t axes = src.NumAxes();
  K2_CHECK_GE(axes, 2);
  src.Populate();
  std::vector<int32_t *> row_splits_ptrs(axes - 1);
  std::vector<int32_t *> row_ids_ptrs(axes - 1);
  for (int32_t i = 1; i != axes; ++i) {
    row_splits_ptrs[i - 1] = src.RowSplits(i).Data();
    row_ids_ptrs[i - 1] = src.RowIds(i).Data();
  }
  ContextPtr ctx = src.Context();
  *row_splits = Array1<int32_t *>(ctx, row_splits_ptrs);
  *row_ids = Array1<int32_t *>(ctx, row_ids_ptrs);
}

/*
  Get some meta-info for an array of RaggedShape, and transfer them
  to the device that `src` is located on. Just same with `GetRowInfo`
  above, but for multiple RaggedShapes.

     @param [in] num_srcs  Number of source arrays to process.
     @param [in] src      Source arrays.  All of them must have same num_axes
                          and on the same device, but we just check this in
                          debug mode.
     @param [in] row_splits  Output array of row_splits pointers,
                          will be of dimension num_axes-1 by num_src
     @param [in] row_splits  Output array of row_splits pointers,
                          will be of dimension num_axes-1 by num_src
*/
void GetRowInfoMulti(int32_t num_srcs, RaggedShape **src,
                     Array2<int32_t *> *row_splits,
                     Array2<int32_t *> *row_ids) {
  K2_CHECK_GT(num_srcs, 0);
  int32_t num_axes_in = src[0]->NumAxes();
  K2_CHECK_GE(num_axes_in, 2);
  ContextPtr ctx = src[0]->Context();

  // check if they have same num-axes and compatible context
  for (int32_t i = 1; i < num_srcs; ++i) {
    K2_CHECK_EQ(src[i]->NumAxes(), num_axes_in);
    K2_CHECK(ctx->IsCompatible(*src[i]->Context()));
  }

  Array2<int32_t *> row_splits_ptrs(GetCpuContext(), num_axes_in - 1, num_srcs);
  Array2<int32_t *> row_ids_ptrs(GetCpuContext(), num_axes_in - 1, num_srcs);
  int32_t **splits_ptr_data = row_splits_ptrs.Data();
  int32_t **ids_ptr_data = row_ids_ptrs.Data();

  int32_t stride0 = row_splits_ptrs.ElemStride0();
  K2_CHECK_EQ(stride0, row_ids_ptrs.ElemStride0());

  for (int32_t axis = 0; axis != num_axes_in - 1; ++axis) {
    for (int32_t i = 0; i != num_srcs; ++i) {
      splits_ptr_data[axis * stride0 + i] = src[i]->RowSplits(axis + 1).Data();
      ids_ptr_data[axis * stride0 + i] = src[i]->RowIds(axis + 1).Data();
    }
  }
  *row_splits = row_splits_ptrs.To(ctx);
  *row_ids = row_ids_ptrs.To(ctx);
}

RaggedShape Append(int32_t axis, int32_t num_srcs, RaggedShape **src) {
  K2_CHECK_EQ(axis, 0) << "Append() with axis > 0 not yet supported";
  K2_CHECK_GT(num_srcs, 1);
  int32_t num_axes = src[0]->NumAxes();
  ContextPtr c = src[0]->Context();

  // Check if they have same num-axes and compatible context
  for (int32_t i = 1; i < num_srcs; ++i) {
    K2_CHECK_EQ(num_axes, src[i]->NumAxes());
    K2_CHECK(IsCompatible(*src[0], *src[i]));
  }

  // `offsets` will be on CPU for now.
  Array2<int32_t> offsets = GetOffsets(num_srcs, src);
  auto offsets_acc = offsets.Accessor();

  std::vector<int32_t> tot_sizes_out(num_axes);
  for (int32_t axis = 0; axis < num_axes; ++axis)
    tot_sizes_out[axis] = offsets_acc(axis + 1, num_srcs);

  RaggedShape ans = RaggedShapeFromTotSizes(c, num_axes, tot_sizes_out.data());

  Array2<int32_t *> src_row_splits, src_row_ids;
  GetRowInfoMulti(num_srcs, src, &src_row_splits, &src_row_ids);
  auto src_row_splits_acc = src_row_splits.Accessor(),
       src_row_ids_acc = src_row_ids.Accessor();
  offsets = offsets.To(c);
  offsets_acc = offsets.Accessor();  // on GPU now (if we're using one)

  ParallelRunner pr(c);
  std::vector<hipStream_t> streams(num_axes);
  int32_t num_jobs = num_srcs * 2;

  // task_redirects is a device array (if using GPU).
  // We have `num_axes - 1` different sets of row_splits/row_ids to
  // populate but they have different sizes; the total number of distinct
  // sizes is `num_axes`.
  Array2<TaskRedirect> task_redirects(c, num_axes, num_jobs);
  auto task_redirects_acc = task_redirects.Accessor();
  // populate task_redirects (these allocate blocks of threads roughly
  // proportionally to the amount of data to process from this source.
  for (int32_t axis = 0; axis < num_axes; ++axis) {
    streams[axis] = pr.NewStream();
    With w(streams[axis]);
    const int32_t *offsets = offsets_acc.Row(axis + 1);
    // c->GetCudaStream() == stream[axis] as it has been overridden by With
    GetTaskRedirect(c, num_srcs, offsets, task_redirects_acc.Row(axis));
  }

  for (int32_t axis = 0; axis < num_axes - 1; axis++) {
    // first set the row-splits.
    int32_t **this_src_row_splits = src_row_splits_acc.Row(axis),
            **this_src_row_ids = src_row_ids_acc.Row(axis);
    int32_t *this_dest_row_splits = ans.RowSplits(axis + 1).Data(),
            *this_dest_row_ids = ans.RowIds(axis + 1).Data();
    const int32_t *offsets_this_axis = offsets_acc.Row(axis + 1),
                  *offsets_next_axis = offsets_acc.Row(axis + 2);
    auto lambda_set_row_splits = [=] __host__ __device__(
                                     int32_t src_idx, int32_t num_threads,
                                     int32_t thread_idx) -> void {
      // Reminder of how row_splits work dimensionally: they are a map
      // from, e.g. an idx0 to an idx0x.   An offsets_acc(0,n) is
      // dimensionally an idx0; an offsets_acc(1,n) an idx01, and so on.
      int32_t this_offset = offsets_this_axis[src_idx],
              next_offset = offsets_this_axis[src_idx + 1],
              this_value_offset = offsets_next_axis[src_idx],
              num_rows = next_offset - this_offset;
      int32_t *src_row_splits_ptr = this_src_row_splits[src_idx];
      // Using <= instead of < below causes threads for different src_idx to
      // write a single overlapping value, but also ensures that the
      // terminating value is written.  This only works because row_splits
      // vectors always start with 0, which is not necessarily the case
      // for row-ids.
      for (; thread_idx <= num_rows; thread_idx += num_threads) {
        this_dest_row_splits[this_offset + thread_idx] =
            this_value_offset + src_row_splits_ptr[thread_idx];
      }
    };

    int32_t min_threads_per_job = 2, tot_work = tot_sizes_out[axis],
            target_num_loops = (tot_work > 1000000 ? 4 : 2);
    EvalWithRedirect(streams[axis], num_jobs, task_redirects_acc.Row(axis),
                     min_threads_per_job, tot_work, target_num_loops,
                     lambda_set_row_splits);

    {  // set the row-ids
      auto lambda_set_row_ids = [=] __host__ __device__(
                                    int32_t src_idx, int32_t num_threads,
                                    int32_t thread_idx) -> void {
        // Reminder of how row_ids work dimensionally: they are a map
        // from, e.g. an idx01 to an idx0.   An offsets_acc(0,n) is
        // dimensionally an idx0; an offsets_acc(1,n) an idx01, and so on.
        int32_t this_offset = offsets_next_axis[src_idx],
                next_offset = offsets_next_axis[src_idx + 1],
                this_value_offset = offsets_this_axis[src_idx],
                num_elems = next_offset - this_offset;
        int32_t *src_row_ids_ptr = this_src_row_ids[src_idx];
        for (; thread_idx < num_elems; thread_idx += num_threads) {
          this_dest_row_ids[this_offset + thread_idx] =
              this_value_offset + src_row_ids_ptr[thread_idx];
        }
      };
      int32_t min_threads_per_job = 2, tot_work = tot_sizes_out[axis + 1],
              target_num_loops = (tot_work > 1000000 ? 4 : 2);
      // TODO(haowen): maybe we should launch kernels for row_splits and row_ids
      // in different streams
      EvalWithRedirect(streams[axis + 1], num_jobs,
                       task_redirects_acc.Row(axis + 1), min_threads_per_job,
                       tot_work, target_num_loops, lambda_set_row_ids);
    }
  }
  return ans;
}

RaggedShape RemoveAxis(RaggedShape &src, int32_t axis) {
  K2_CHECK_GT(src.NumAxes(), 2);
  K2_CHECK(axis >= 0 && axis < src.NumAxes());

  // note, `axes_in` is of dim src.NumAxes() - 1.
  // Also note: axes_in[i] pertains to the relationship between
  // axes i and i+1 in the source.
  src.Populate();

  const std::vector<RaggedShapeDim> &axes_in = src.Axes();

  std::vector<RaggedShapeDim> axes_out(axes_in.size() - 1);
  int32_t axes_out_size = static_cast<int32_t>(axes_out.size());

  for (int32_t i = 0; i < axis - 1; ++i) axes_out[i] = axes_in[i];

  if (axis > 0 && axis + 1 < src.NumAxes()) {
    axes_out[axis - 1].row_ids =
        axes_in[axis - 1].row_ids[axes_in[axis].row_ids];
    axes_out[axis - 1].row_splits =
        axes_in[axis].row_splits[axes_in[axis - 1].row_splits];
    axes_out[axis - 1].cached_tot_size = axes_out[axis - 1].row_ids.Dim();
  }
  for (int32_t i = axis; i < axes_out_size; ++i) axes_out[i] = axes_in[i + 1];
  return RaggedShape(axes_out);
}

// transpose axes 0 and 1.
RaggedShape Transpose(RaggedShape &src) {
  K2_CHECK_GT(src.NumAxes(), 2);
  int32_t src_dim0 = src.Dim0(), src_tot_size1 = src.TotSize(1);
  K2_CHECK_EQ(src_tot_size1 % src_dim0, 0)
      << "Transpose(): all dims on axis 0 must be the same.";
  int32_t src_dim1 = src_tot_size1 / src_dim0;
  RaggedShape src_no_axis0 = RemoveAxis(src, 0);
  K2_CHECK_EQ(src_no_axis0.Dim0(), src_tot_size1);
  ContextPtr c = src.Context();
  // `renumbering` is a `new2old` map, that maps from the first index in
  // src_no_axis0_renumbered
  // to the first index into src_no_axis0.
  Array1<int32_t> renumbering(c, src_tot_size1);
  int32_t *renumbering_data = renumbering.Data();
  auto lambda_set_renumbering = [=] __host__ __device__(int32_t i) {
    int32_t j = i % src_dim1, k = i / src_dim1, i_old = j * src_dim0 + k;
    renumbering_data[i] = i_old;
  };
  Eval(c, src_tot_size1, lambda_set_renumbering);

  RaggedShape src_no_axis0_renumbered = Renumber(src_no_axis0, renumbering);

  int32_t num_rows = src_dim1, row_splits_dim = num_rows + 1,
          row_ids_dim = src_tot_size1;
  std::vector<RaggedShapeDim> ans_axis0(1);
  Array1<int32_t> mem(c, row_splits_dim + row_ids_dim);
  int32_t *mem_data = mem.Data();
  auto lambda_set_row_info = [=] __host__ __device__(int32_t i) {
    int32_t val;
    if (i >= row_splits_dim) {
      // row_ids
      int32_t elem_idx = i - row_splits_dim;
      val = elem_idx / src_dim0;
    } else {
      // row_splits
      int32_t row_idx = i;
      val = row_idx * src_dim0;
    }
    mem_data[i] = val;
  };
  Eval(c, row_splits_dim + row_ids_dim, lambda_set_row_info);
  ans_axis0[0].row_splits = mem.Range(0, row_splits_dim);
  ans_axis0[0].row_ids = mem.Range(row_splits_dim, row_ids_dim);
  ans_axis0[0].cached_tot_size = row_ids_dim;

  RaggedShape temp(ans_axis0);
  return ComposeRaggedShapes(temp, src_no_axis0_renumbered);
}

RaggedShape Stack(int32_t axis, int32_t num_srcs, const RaggedShape **src) {
  K2_CHECK_GT(num_srcs, 0);
  K2_CHECK(axis >= 0 && axis <= 1);

  ContextPtr c = src[0]->Context();
  int32_t num_axes = src[0]->NumAxes();

  // Check if they have the same num-axes and compatible context
  for (int32_t i = 1; i < num_srcs; ++i) {
    K2_CHECK_EQ(num_axes, src[i]->NumAxes());
    K2_CHECK(c->IsCompatible(*src[i]->Context()));
  }

  std::vector<RaggedShape> unsqueezed(num_srcs);
  std::vector<RaggedShape *> unsqueezed_ptrs(num_srcs);
  {
    ParallelRunner pr(c);
    for (int32_t i = 0; i < num_srcs; i++) {
      With w(pr.NewStream());
      unsqueezed[i] = Unsqueeze(*src[i], 0);
      unsqueezed_ptrs[i] = &unsqueezed[i];
    }
    // destructor will wait for work in those launched streams to finish.
    // (well it won't actually wait, but it will force the current stream to
    // wait.)
  }

  RaggedShape ans = Append(0, num_srcs, unsqueezed_ptrs.data());
  // Transpose will check if all src->Dim0() has the same value.
  if (axis == 1) ans = Transpose(ans);
  return ans;
}

RaggedShape TrivialShape(ContextPtr &c, int32_t num_elems) {
  // row_splits= [
  Array1<int32_t> row_splits = Range<int32_t>(c, 2, 0, num_elems);
  int32_t *row_splits_data = row_splits.Data();

  Array1<int32_t> row_ids(c, num_elems, 0);
  return RaggedShape2(&row_splits, &row_ids, num_elems);
}

}  // namespace k2
