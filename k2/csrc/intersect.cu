#include "hip/hip_runtime.h"
/**
 * Copyright (c)  2020  Xiaomi Corporation (authors: Daniel Povey)
 *
 * See LICENSE for clarification regarding multiple authors
 */

#include <limits>
#include <vector>

#include "k2/csrc/array_ops.h"
#include "k2/csrc/fsa_algo.h"
#include "k2/csrc/fsa_utils.h"
#include "k2/csrc/hash.h"
#include "k2/csrc/macros.h"
#include "k2/csrc/ragged_ops.h"

namespace k2 {

namespace intersect_internal {

struct StateInfo {
  // the state_idx01 in a_fsas_.
  int32_t a_fsas_state_idx01;
  // the state_idx01 in b_fsas_.
  int32_t b_fsas_state_idx01;
};

struct ArcInfo {
  int32_t src_ostate;    // source state-index which is index into states_.
  int32_t dest_ostate;   // dest state-index which is index into states_.
  int32_t a_arc_idx012;  // The idx012 of the source arc in a_fsas_.
  int32_t b_arc_idx012;  // The idx012 of the source arc in b_fsas_.
  // Note: other fields, e.g. the label and score, can be worked
  // out from the arc-indexes.
};

/*
static std::ostream &operator<<(std::ostream &os, const StateInfo &s) {
  os << "StateInfo{" << s.a_fsas_state_idx01 << ","
     << s.b_fsas_state_idx01 << "}";
  return os;
}

static std::ostream &operator<<(std::ostream &os, const ArcInfo &a) {
  os << "ArcInfo{" << a.src_ostate << "," << a.dest_ostate << ","
     << a.a_arc_idx012 << "," << a.b_arc_idx012 << "}";
  return os;
}
*/


}  // namespace intersect_internal

using namespace intersect_internal;  // NOLINT

/*
   Intersection (a.k.a. composition) that corresponds to decoding for
   speech recognition-type tasks.

   Can use either different decoding graphs (one per acoustic sequence) or a
   shared graph.

   How to use this object:
       Construct it
       Call Intersect()
       Call FormatOutput()
*/
class DeviceIntersector {
 public:
  /**
     This object does intersection on device (the general case, but without treating
     epsilons specially)

       @param [in] a_fsas  An FsaVec (3 axes), must be valid.  Caution: in future,
                           we may require that it be arc-sorted.
       @param [in] b_fsas  An FsaVec (3 axes), must be valid.
       @param [in] b_to_a_map  Map from fsa-index in b_fsas to the index of the FSA
                           in a_fsas that we want to intersect it with.

     Does not fully check its args (see wrapping code).  After constructing this object,
     call Intersect() and then FormatOutput().
   */
  DeviceIntersector(FsaVec &a_fsas, FsaVec &b_fsas,
                    const Array1<int32_t> &b_to_a_map):
      c_(a_fsas.Context()),
      a_fsas_(a_fsas),
      b_fsas_(b_fsas),
      b_to_a_map_(b_to_a_map),
      b_state_bits_(2 + HighestBitSet(b_fsas_.TotSize(1))),
      key_bits_(b_state_bits_ + 2 + HighestBitSet(a_fsas_.shape.MaxSize(1))) {

    if (key_bits_ < 32)  // TEMP!!
      key_bits_ = 32;

    // We may want to tune this hash size eventually.
    // Note: the hash size
    int32_t hash_size = 4 * RoundUpToNearestPowerOfTwo(b_fsas.NumElements()),
        min_hash_size = 1 << 16;
    if (hash_size < min_hash_size)
      hash_size = min_hash_size;
    // caution: also use hash_size in FirstIter() as default size of various arrays.
    state_pair_to_state_ = Hash(c_, hash_size);

    K2_CHECK(c_->IsCompatible(*b_fsas.Context()));
    K2_CHECK(c_->IsCompatible(*b_to_a_map.Context()));
  }


  void FirstIter() {
    int32_t initial_size = state_pair_to_state_.NumBuckets();
    arcs_row_ids_ = Array1<int32_t>(c_, initial_size);
    arcs_row_ids_.Resize(0, true);
    arcs_ = Array1<ArcInfo>(c_, initial_size);
    arcs_.Resize(0, true);

    int32_t num_fsas = b_fsas_.Dim0();

    states_ = Array1<StateInfo>(c_, initial_size);

    Renumbering renumber_initial_states(c_, num_fsas);

    char *keep_initial_states = renumber_initial_states.Keep().Data();

    const int32_t *b_fsas_row_splits1_data = b_fsas_.RowSplits(1).Data(),
                          *b_to_a_map_data = b_to_a_map_.Data(),
                 *a_fsas_row_splits1_data = a_fsas_.RowSplits(1).Data();

    K2_EVAL(c_, num_fsas, lambda_set_keep, (int32_t i) -> void {
        int nonempty_b = b_fsas_row_splits1_data[i+1] > b_fsas_row_splits1_data[i],
                   i_a = b_to_a_map_data[i],
            nonempty_a = a_fsas_row_splits1_data[i_a+1] > a_fsas_row_splits1_data[i_a];
        keep_initial_states[i] = (char)(nonempty_a & nonempty_b);
      });
    int32_t num_initial_states = renumber_initial_states.New2Old().Dim();

    states_.Resize(num_initial_states, true);
    final_states_ = Array1<StateInfo>(c_, num_initial_states);

    StateInfo *states_data = states_.Data(),
        *final_states_data = final_states_.Data();
    const int32_t *new2old_data = renumber_initial_states.New2Old().Data();
    K2_EVAL(c_, num_initial_states, lambda_set_state_info, (int32_t new_i) -> void {
        int32_t b_idx0 = new2old_data[new_i],
               b_idx01 = b_fsas_row_splits1_data[b_idx0],
                a_idx0 = b_to_a_map_data[b_idx0],
               a_idx01 = a_fsas_row_splits1_data[a_idx0];
        StateInfo info;
        info.a_fsas_state_idx01 = a_idx01;
        info.b_fsas_state_idx01 = b_idx01;
        states_data[new_i] = info;

        // now set final-state info.
        info.a_fsas_state_idx01 = a_fsas_row_splits1_data[a_idx0 + 1] - 1;
        info.b_fsas_state_idx01 = b_fsas_row_splits1_data[b_idx0 + 1] - 1;
        final_states_data[new_i] = info;
      });

    iter_to_state_row_splits_cpu_.reserve(128);
    iter_to_state_row_splits_cpu_.push_back(0);
    iter_to_state_row_splits_cpu_.push_back(num_initial_states);
  }


  /*
    Adds the StateInfo for the final-states to the states_ array.
  */
  void LastIter() {
    int32_t num_final_states = final_states_.Dim();
    int32_t cur_num_states = states_.Dim(),
            tot_num_states = cur_num_states + num_final_states;
    states_.Resize(tot_num_states);
    Array1<StateInfo> dest = states_.Arange(cur_num_states,
                                            tot_num_states);
    Assign(final_states_, &dest);
    K2_CHECK_EQ(cur_num_states, iter_to_state_row_splits_cpu_.back());  // Remove this line.
    iter_to_state_row_splits_cpu_.push_back(tot_num_states);
  }

  /* Does the main work of intersection/composition, but doesn't produce any
     output; the output is provided when you call FormatOutput(). */
  void Intersect() {
    NVTX_RANGE(K2_FUNC);

    FirstIter();
    Forward();
    LastIter();
  }


  /*
    Creates and returns a ragged array indexed [fsa][state][arc],
    containing the result of intersection.  (Note: we don't guarantee that
    all states are coaccessible (i.e. can reach the end); if that might be
    an issue in your case, you can call Connect() afterward.

         @param [out] arc_map_a_out  If non-NULL, the map from (arc-index of
                                  returned FsaVec) to (arc-index in a_fsas_)
                                  will be written to here.
         @param [out] arc_map_b_out  If non-NULL, the map from (arc-index of
                                  returned FsaVec) to (arc-index in b_fsas_)
                                  will be written to here.
         @return  Returns a FsaVec that is the composed result.  It may
                  contain states and/or arcs that are not co-accessible.
   */
  FsaVec FormatOutput(Array1<int32_t> *arc_map_a_out,
                      Array1<int32_t> *arc_map_b_out) {
    NVTX_RANGE(K2_FUNC);

    int32_t num_states = iter_to_state_row_splits_cpu_.back(),
             num_iters = iter_to_state_row_splits_cpu_.size() - 1,
              num_fsas = b_fsas_.Dim0();
    Array1<int32_t> row_splits1(c_, iter_to_state_row_splits_cpu_),
        row_ids1(c_, num_states);
    RowSplitsToRowIds(row_splits1, &row_ids1);

    const int32_t *b_fsas_row_ids1_data = b_fsas_.RowIds(1).Data();
    int32_t *row_ids1_data = row_ids1.Data();
    StateInfo *states_data = states_.Data();
    K2_CHECK_EQ(num_states, states_.Dim());

    /*
      currently, row_ids1 maps from state-index (in states_) to iteration
      index 0 <= t < num_iters.  We next modify it so it maps from state to
      a number that encodes (iter, FSA-index), i.e. we modify it from
           iter  ->  iter * num_fsas + fsa_idx0.
      Later we'll reorder the rows so that each FSA has all its states
      together.
    */
    K2_EVAL(c_, num_states, lambda_modify_row_ids, (int32_t i) -> void {
        int32_t iter = row_ids1_data[i];
        StateInfo info = states_data[i];
        // note: the FSA-index of the output is the same as that in b_fsas_,
        // but not necessarily in a_fsas_, thanks to b_to_a_map_.
        int32_t fsa_idx0 = b_fsas_row_ids1_data[info.b_fsas_state_idx01],
              new_row_id = iter * num_fsas + fsa_idx0;
        K2_DCHECK_LT(static_cast<uint32_t>(fsa_idx0),
                     static_cast<uint32_t>(num_fsas));
        row_ids1_data[i] = new_row_id;
    });

    Array1<int32_t> row_ids2(row_ids1),  // we'll later interpret this as the 2nd
                                         // level's row-ids.
        row_splits2(c_, num_iters * num_fsas + 1);
    RowIdsToRowSplits(row_ids2, &row_splits2);

    // We'll use 'fsaiter_new2old' to effectively transpose two axes, the
    // iteration and FSA axes.  We want the FSA to be the more-slowly-varying
    // index, so we can have all states for FSA 0 first.
    Array1<int32_t> fsaiter_new2old(c_, num_iters * num_fsas);
    int32_t *fsaiter_new2old_data = fsaiter_new2old.Data();
    K2_EVAL(c_, num_iters * num_fsas, lambda_set_reordering, (int32_t i) -> void {
        int32_t fsa_idx = i / num_iters,
               iter_idx = i % num_iters;
        int32_t old_i = iter_idx * num_fsas + fsa_idx;
        fsaiter_new2old_data[i] = old_i;
      });

    Array1<int32_t> &row_ids3(arcs_row_ids_);
    Array1<int32_t> row_splits3(c_, num_states + 1);
    RowIdsToRowSplits(row_ids3, &row_splits3);

    RaggedShape layer2 = RaggedShape2(&row_splits2, &row_ids2, -1),
                layer3 = RaggedShape2(&row_splits3, &row_ids3, -1);

    Array1<int32_t> states_new2old, arcs_new2old;
    RaggedShape layer2_new = Index(layer2, 0, fsaiter_new2old,
                                   &states_new2old),
                layer3_new = Index(layer3, 0, states_new2old,
                                   &arcs_new2old);

    RaggedShape layer1_new = RegularRaggedShape(c_, num_fsas, num_iters);

    // We remove axis 1, which represents 'iteration-index' (this is not
    // something the user needs to know or care about).
    RaggedShape temp = ComposeRaggedShapes3(layer1_new, layer2_new, layer3_new);
    RaggedShape ans_shape = RemoveAxis(temp, 1);

    int32_t num_arcs = arcs_.Dim();
    K2_CHECK_EQ(ans_shape.NumElements(), num_arcs);

    Array1<Arc> ans_values(c_, num_arcs);

    int32_t *arc_map_a_data = nullptr,
            *arc_map_b_data = nullptr;
    if (arc_map_a_out) {
      *arc_map_a_out = Array1<int32_t>(c_, num_arcs);
      arc_map_a_data = arc_map_a_out->Data();
    }
    if (arc_map_b_out) {
      *arc_map_b_out = Array1<int32_t>(c_, num_arcs);
      arc_map_b_data = arc_map_b_out->Data();
    }

    Array1<int32_t> states_old2new = InvertPermutation(states_new2old);

    ArcInfo *arc_info_data = arcs_.Data();
    const Arc *a_arcs_data = a_fsas_.values.Data(),
              *b_arcs_data = b_fsas_.values.Data();
    Arc *arcs_out_data = ans_values.Data();
    const int32_t *arcs_new2old_data = arcs_new2old.Data(),
                *states_new2old_data = states_new2old.Data(),
                *states_old2new_data = states_old2new.Data();

    const int32_t *ans_shape_row_ids2 = ans_shape.RowIds(2).Data(),
                  *ans_shape_row_ids1 = ans_shape.RowIds(1).Data(),
               *ans_shape_row_splits1 = ans_shape.RowSplits(1).Data();

    // arc_idx012 here is w.r.t. ans_shape that currently has axes, indexed
    // [fsa][state][arc].
    K2_EVAL(c_, num_arcs, lambda_set_output_data, (int32_t new_arc_idx012) -> void {
        int32_t new_src_state_idx01 = ans_shape_row_ids2[new_arc_idx012],
                     old_arc_idx012 = arcs_new2old_data[new_arc_idx012],
                old_src_state_idx01 = states_new2old_data[new_src_state_idx01];

        ArcInfo info = arc_info_data[old_arc_idx012];
        K2_CHECK_EQ(old_src_state_idx01, info.src_ostate);
        int32_t fsa_idx0 = ans_shape_row_ids1[new_src_state_idx01];
        int32_t dest_state_idx01;
        if (info.dest_ostate >= 0) {
          dest_state_idx01 = states_old2new_data[info.dest_ostate];
        } else {
          dest_state_idx01 = ans_shape_row_splits1[fsa_idx0 + 1] - 1;
        }
        int32_t fsa_idx0x = ans_shape_row_splits1[fsa_idx0],
          dest_state_idx1 = dest_state_idx01 - fsa_idx0x,
           src_state_idx1 = new_src_state_idx01 - fsa_idx0x;

        Arc a_arc = a_arcs_data[info.a_arc_idx012],
            b_arc = b_arcs_data[info.b_arc_idx012];
        if (arc_map_a_data) arc_map_a_data[new_arc_idx012] = info.a_arc_idx012;
        if (arc_map_b_data) arc_map_b_data[new_arc_idx012] = info.b_arc_idx012;

        Arc out_arc;
        out_arc.src_state = src_state_idx1;
        out_arc.dest_state = dest_state_idx1;
        K2_CHECK_EQ(a_arc.label, b_arc.label);
        out_arc.label = a_arc.label;
        out_arc.score = a_arc.score + b_arc.score;
        arcs_out_data[new_arc_idx012] = out_arc;
      });

    return Ragged<Arc>(ans_shape, ans_values);
  }

  void Forward() {
    NVTX_RANGE(K2_FUNC);
    for (int32_t t = 0; ; t++) {
      NVTX_RANGE("LoopT");

      if (states_.Dim() * 4 > state_pair_to_state_.NumBuckets()) {
        // enlarge hash..
        state_pair_to_state_.Resize(state_pair_to_state_.NumBuckets() * 2,
                                    key_bits_);
      }

      K2_CHECK_EQ(t + 2, int32_t(iter_to_state_row_splits_cpu_.size()));

      int32_t state_begin = iter_to_state_row_splits_cpu_[t],
          state_end = iter_to_state_row_splits_cpu_[t + 1],
          num_states = state_end - state_begin;

      if (num_states == 0) {
        // It saves a little processing later to remove the last, empty,
        // iteration-index.
        iter_to_state_row_splits_cpu_.pop_back();
        break;  // Nothing left to process.
      }

      // We need to process output-states numbered state_begin..state_end-1.

      // Row 0 of num_arcs will contain the num_arcs leaving each state
      // in b in this batch; row 1 will contain (num_arcs in a * num_arcs in b).
      // If the total of row 1 is small enough and we're using the device,
      // we'll process all pairs of arcs; otherwise we'll do a logarithmic
      // search.
      Array2<int32_t> num_arcs(c_, 2, num_states + 1);

      auto num_arcs_acc = num_arcs.Accessor();
      StateInfo *states_data = states_.Data();
      const int32_t *a_fsas_row_splits2_data = a_fsas_.RowSplits(2).Data(),
          *b_fsas_row_splits2_data = b_fsas_.RowSplits(2).Data();

      K2_EVAL(c_, num_states, lambda_find_num_arcs, (int32_t i) -> void {
        int32_t state_idx = state_begin + i;
        StateInfo info = states_data[state_idx];
        int32_t b_fsas_state_idx01 = info.b_fsas_state_idx01,
            b_start_arc = b_fsas_row_splits2_data[b_fsas_state_idx01],
            b_end_arc =  b_fsas_row_splits2_data[b_fsas_state_idx01 + 1],
            b_num_arcs = b_end_arc - b_start_arc;
        num_arcs_acc(0, i) = b_num_arcs;
        int32_t a_fsas_state_idx01 = info.a_fsas_state_idx01,
            a_start_arc = a_fsas_row_splits2_data[a_fsas_state_idx01],
            a_end_arc =  a_fsas_row_splits2_data[a_fsas_state_idx01 + 1],
            a_num_arcs = a_end_arc - a_start_arc;
        num_arcs_acc(1, i) = b_num_arcs * a_num_arcs;
        });

      Array1<int32_t> row_splits_ab = num_arcs.Row(1),
                         num_arcs_b = num_arcs.Row(0);
      ExclusiveSum(row_splits_ab, &row_splits_ab);

      // tot_ab is total of (num-arcs from state a * num-arcs from state b).
      int32_t tot_ab = row_splits_ab[num_states],
              cutoff = 1 << 30;  // Eventually I'll make cutoff smaller, like 16384,
                           // and implement the other branch.

      const Arc *a_arcs_data = a_fsas_.values.Data(),
          *b_arcs_data = b_fsas_.values.Data();

      int32_t key_bits = key_bits_, b_state_bits = b_state_bits_,
          value_bits = 64 - key_bits;

      // `value_max` is the limit for how large values in the hash can be.
      uint64_t value_max = ((uint64_t)1) << value_bits;
      auto state_pair_to_state_acc =
          state_pair_to_state_.GetGenericAccessor(key_bits);

      K2_CHECK_GT(value_max, (uint64_t)tot_ab) << "Problem size too large "
          "for hash table... redesign or reduce problem size.";

      if (tot_ab < cutoff) {
        Array1<int32_t> row_ids_ab(c_, tot_ab);
        RowSplitsToRowIds(row_splits_ab, &row_ids_ab);

        const int32_t *row_ids_ab_data = row_ids_ab.Data(),
                   *row_splits_ab_data = row_splits_ab.Data(),
                      *num_arcs_b_data = num_arcs_b.Data();

        const int32_t *b_fsas_row_ids1_data = b_fsas_.RowIds(1).Data();

        // arcs_newstates_renumbering serves two purposes:
        //  - we'll keep some subset of the `tot_ab` arcs.
        //  - some subset of the dest-states of those arcs will be "new" dest-states
        //    that need to be assigned a state-id.
        // To avoid sequential kernels for computing Old2New() and computing New2Old(),
        // we combine those two renumberings into one.
        Renumbering arcs_newstates_renumbering(c_, tot_ab * 2);
        char *keep_arc_data = arcs_newstates_renumbering.Keep().Data(),
            *new_dest_state_data = keep_arc_data + tot_ab;
        const int32_t *a_fsas_row_splits2 = a_fsas_.RowSplits(2).Data(),
                      *b_fsas_row_splits2 = b_fsas_.RowSplits(2).Data();

        K2_EVAL(c_, tot_ab, lambda_set_keep_arc_newstate, (int32_t i) -> void {
          // state_i is the index into the block of ostates that we're
          // processing, the actual state index is state_i + state_begin.
          int32_t state_i = row_ids_ab_data[i],
              // arc_pair_idx encodes a_arc_idx2 and b_arc_idx2
              arc_pair_idx = i - row_splits_ab_data[state_i],
              state_idx = state_i + state_begin;
          StateInfo sinfo = states_data[state_idx];
          int32_t num_arcs_b = num_arcs_b_data[state_i],
              a_arc_idx2 = arc_pair_idx / num_arcs_b,
              b_arc_idx2 = arc_pair_idx % num_arcs_b;
          // the idx2's above are w.r.t. a_fsas_ and b_fsas_.
          int32_t a_arc_idx01x = a_fsas_row_splits2[sinfo.a_fsas_state_idx01],
              b_arc_idx01x = b_fsas_row_splits2[sinfo.b_fsas_state_idx01],
              a_arc_idx012 = a_arc_idx01x + a_arc_idx2,
              b_arc_idx012 = b_arc_idx01x + b_arc_idx2;
          // Not treating epsilons specially here, see documentation for
          // IntersectDevice() in [currently] fsa_algo.h.
          int keep_arc = (a_arcs_data[a_arc_idx012].label ==
                          b_arcs_data[b_arc_idx012].label);
          keep_arc_data[i] = (char)keep_arc;
          int new_dest_state = 0;
          if (keep_arc && a_arcs_data[a_arc_idx012].label != -1) {
            // investigate whether the dest-state is new (not currently allocated
            // a state-id).  We don't allocate ids for the final-state, so skip this
            // if label is -1.

            int32_t b_dest_state_idx1 = b_arcs_data[b_arc_idx012].dest_state,
                b_dest_state_idx01 = b_dest_state_idx1 + sinfo.b_fsas_state_idx01 -
                                     b_arcs_data[b_arc_idx012].src_state,
                a_dest_state_idx1 = a_arcs_data[a_arc_idx012].dest_state;
            uint64_t hash_key = (((uint64_t)a_dest_state_idx1) << b_state_bits) |
                   b_dest_state_idx01, hash_value = i;
            // If it was successfully inserted, then this arc is assigned
            // responsibility for creating the state-id for its destination
            // state.
            if (state_pair_to_state_acc.Insert(hash_key, hash_value)) {
              new_dest_state = 1;
            }
          }
          new_dest_state_data[i] = (char)new_dest_state;
        });

        // When reading the code below, remember this code is a little unusual
        // because we have combined the renumberings for arcs and new-states
        // into one.
        int32_t num_kept_arcs = arcs_newstates_renumbering.Old2New(true)[tot_ab],
                 num_kept_tot = arcs_newstates_renumbering.New2Old().Dim(),
              num_kept_states = num_kept_tot - num_kept_arcs;

        int32_t next_state_end = state_end + num_kept_states;
        iter_to_state_row_splits_cpu_.push_back(next_state_end);
        states_.Resize(next_state_end);  // Note: this Resize() won't actually reallocate each time.
        states_data = states_.Data();  // In case it changed (unlikely)

        Array1<int32_t> states_new2old =
            arcs_newstates_renumbering.New2Old().Arange(num_kept_arcs, num_kept_tot);
        const int32_t *states_new2old_data = states_new2old.Data(),
            *b_to_a_map_data = b_to_a_map_.Data(),
            *a_fsas_row_splits1_data = a_fsas_.RowSplits(1).Data();

        // set new elements of `states_data`, setting up the StateInfo on the next
        // frame and setting the state indexes in the hash (to be looked up when
        // creating the arcs.
        K2_EVAL(c_, num_kept_states, lambda_set_states_data, (int32_t i) -> void {
          // the reason for the "- tot_ab" is that this was in the second half of
          // the array of 'kept' of size tot_ab * 2.
          int32_t arc_i = states_new2old_data[i] - tot_ab;

          // The code below repeats what we did when processing arcs in the
          // previous lambda (now just for a small subset of arcs).

          // src_state_i is the index into the block of ostates that we're
          // processing, the actual state index is state_i + state_begin.
          int32_t src_state_i = row_ids_ab_data[arc_i],
              // arc_pair_idx encodes a_arc_idx2 and b_arc_idx2
              arc_pair_idx = arc_i - row_splits_ab_data[src_state_i],
             src_state_idx = src_state_i + state_begin;
          StateInfo src_sinfo = states_data[src_state_idx];
          int32_t num_arcs_b = num_arcs_b_data[src_state_i],
              a_arc_idx2 = arc_pair_idx / num_arcs_b,
              b_arc_idx2 = arc_pair_idx % num_arcs_b;
          // the idx2's above are w.r.t. a_fsas_ and b_fsas_.
          int32_t a_arc_idx01x = a_fsas_row_splits2[src_sinfo.a_fsas_state_idx01],
              b_arc_idx01x = b_fsas_row_splits2[src_sinfo.b_fsas_state_idx01],
              a_arc_idx012 = a_arc_idx01x + a_arc_idx2,
              b_arc_idx012 = b_arc_idx01x + b_arc_idx2;
          Arc b_arc = b_arcs_data[b_arc_idx012],
              a_arc = a_arcs_data[a_arc_idx012];
          K2_DCHECK_EQ(a_arc.label, b_arc.label);

          int32_t b_dest_state_idx1 = b_arcs_data[b_arc_idx012].dest_state,
                b_dest_state_idx01 = b_dest_state_idx1 + src_sinfo.b_fsas_state_idx01 -
                                     b_arcs_data[b_arc_idx012].src_state,
                b_fsa_idx0 = b_fsas_row_ids1_data[b_dest_state_idx01],
              a_dest_state_idx1 = a_arcs_data[a_arc_idx012].dest_state,
              a_dest_state_idx01 = a_fsas_row_splits1_data[b_to_a_map_data[b_fsa_idx0]] +
                    a_dest_state_idx1;
          uint64_t hash_key = (((uint64_t)a_dest_state_idx1) << b_state_bits) |
              b_dest_state_idx01;
          uint64_t value, *key_value_location = nullptr;
          bool ans = state_pair_to_state_acc.Find(hash_key, &value,
                                                  &key_value_location);
          K2_DCHECK(ans);
          K2_DCHECK_EQ(value, (uint64_t)arc_i);
          int32_t dest_state_idx = state_end + i;
          state_pair_to_state_acc.SetValue(key_value_location, hash_key,
                                             (uint64_t)dest_state_idx);

          StateInfo dest_sinfo;
          dest_sinfo.a_fsas_state_idx01 = a_dest_state_idx01;
          dest_sinfo.b_fsas_state_idx01 = b_dest_state_idx01;
          states_data[dest_state_idx] = dest_sinfo;
        });

        int32_t old_num_arcs = arcs_.Dim(),
            new_num_arcs = old_num_arcs + num_kept_arcs;
        if (static_cast<uint64_t>(tot_ab) >= value_max ||
            static_cast<uint64_t>(next_state_end) >= value_max) {
          K2_LOG(FATAL) << "Problem size is too large for this code: b_state_bits="
                        << b_state_bits_ << ", key_bits=" << key_bits_
                        << ", value_bits=" << value_bits
                        << ", value_max=" << value_max
                        << ", tot_ab=" << tot_ab
                        << ", next_state_end=" << next_state_end;
        }

        arcs_.Resize(new_num_arcs);
        arcs_row_ids_.Resize(new_num_arcs);
        ArcInfo *arcs_data = arcs_.Data();
        int32_t *arcs_row_ids_data = arcs_row_ids_.Data();

        const int32_t *arcs_new2old_data =
            arcs_newstates_renumbering.New2Old().Data();

        K2_EVAL(c_, num_kept_arcs, lambda_set_arc_info, (int32_t new_arc_i) -> void {
          // 0 <= old_arc_i < tot_ab.
          int32_t old_arc_i = arcs_new2old_data[new_arc_i];

          // The code below repeats what we did when processing arcs in the
          // previous lambdas (we do this for all arcs that were kept).

          // src_state_i is the index into the block of ostates that we're
          // processing, the actual state index is src_state_i + state_begin.
          int32_t src_state_i = row_ids_ab_data[old_arc_i];
          // arc_pair_idx encodes a_arc_idx2 and b_arc_idx2
          int32_t arc_pair_idx = old_arc_i - row_splits_ab_data[src_state_i],
              src_state_idx = src_state_i + state_begin;
          StateInfo src_sinfo = states_data[src_state_idx];
          int32_t num_arcs_b = num_arcs_b_data[src_state_i],
              a_arc_idx2 = arc_pair_idx / num_arcs_b,
              b_arc_idx2 = arc_pair_idx % num_arcs_b;
          // the idx2's above are w.r.t. a_fsas_ and b_fsas_.
          int32_t a_arc_idx01x = a_fsas_row_splits2[src_sinfo.a_fsas_state_idx01],
              b_arc_idx01x = b_fsas_row_splits2[src_sinfo.b_fsas_state_idx01],
              a_arc_idx012 = a_arc_idx01x + a_arc_idx2,
              b_arc_idx012 = b_arc_idx01x + b_arc_idx2;
          Arc b_arc = b_arcs_data[b_arc_idx012],
              a_arc = a_arcs_data[a_arc_idx012];
          K2_DCHECK_EQ(a_arc.label, b_arc.label);

          int32_t dest_state_idx = -1;
          if (a_arc.label != -1) {
            int32_t b_dest_state_idx1 = b_arcs_data[b_arc_idx012].dest_state,
                b_dest_state_idx01 = b_dest_state_idx1 + src_sinfo.b_fsas_state_idx01 -
                                     b_arcs_data[b_arc_idx012].src_state,
              a_dest_state_idx1 = a_arcs_data[a_arc_idx012].dest_state;
            uint64_t hash_key = (((uint64_t)a_dest_state_idx1) << b_state_bits) +
                b_dest_state_idx01;

            uint64_t value = 0;
            bool ans = state_pair_to_state_acc.Find(hash_key, &value);
            dest_state_idx = static_cast<uint32_t>(value);
          }  // else leave it at -1, it's a final-state and we allocate their
             // state-ids at the end.

          ArcInfo info;
          info.src_ostate = src_state_idx;
          info.dest_ostate = dest_state_idx;
          info.a_arc_idx012 = a_arc_idx012;
          info.b_arc_idx012 = b_arc_idx012;
          arcs_data[old_num_arcs + new_arc_i] = info;
          arcs_row_ids_data[old_num_arcs + new_arc_i] = src_state_idx;
        });
      } else {
        ExclusiveSum(num_arcs, &num_arcs, 1);  // sum
        // Plan to implement binary search here at some point, to get arc ranges...
        K2_LOG(FATAL) << "Not implemented yet, see code..";
      }
    }
  }

  ~DeviceIntersector() {
    // Prevent crash in destructor of hash (at exit, it still contains values, by design).
    state_pair_to_state_.Destroy();
  }


  ContextPtr c_;
  FsaVec a_fsas_;  // a_fsas_: decoding graphs
                   // Note: a_fsas_ has 3 axes.

  FsaVec b_fsas_;

  // map from fsa-index in b_fsas_ to the fsa-index in a_fsas_ that we want to
  // intersect it with.
  Array1<int32_t> b_to_a_map_;

  // iter_to_state_row_splits_cpu_, which is resized on each iteration of the
  // algorithm, is a row-splits array that maps from iteration index to
  // state_idx (index into states_).
  std::vector<int32_t> iter_to_state_row_splits_cpu_;

  // states_ is a resizable array of StateInfo that conceptually is the elements
  // of a ragged array indexed [iter][state], with row_splits1 ==
  // iter_to_state_row_splits_cpu_.
  Array1<StateInfo> states_;

  // final_states_ is an array of StateInfo, of dimension <= b_fsas_.Dim0(),
  // that contains the final state-pairs of each composed FSA that has initial
  // state-pairs.  These will be added to the end of states_ after composition
  // has finished.
  Array1<StateInfo> final_states_;

  // arcs_ is a resizable array of ArcInfo that conceptually is the elements
  // of a ragged array indexed [iter][state][arc], with row_splits1 == iter_to_state_row_splits_cpu_
  // and row_ids2 == arcs_row_ids_.
  Array1<ArcInfo> arcs_;

  // arcs_row_ids_, which always maintained as having the same size as `arcs_`,
  // maps from the output arc to the corresponding ostate index that the arc
  // leaves from (index into states_).  Actually this may be redu
  Array1<int32_t> arcs_row_ids_;



  // The hash maps from state-pair, as:
  //   state_pair = (a_fsas_state_idx1 << b_state_bits_) + b_fsas_state_idx01
  //
  // The number of bits in the key (max bits set in `state_pair`) is
  // key_bits_ == b_state_bits_ + HighestBitSet(a_fsas_.MaxSize(1)) + 2.
  // The number of bits in the value is 64 minus this; we'll crash if
  // the number of states ends up being too large to store in this
  // value.
  int32_t b_state_bits_;  // == HighestBitSet(b_fsas_.TotSize(1)) + 2.
  int32_t key_bits_;  // b_state_bits_ + HighestBitSet(a_fsas_.MaxSize(1)) + 2.


  Hash state_pair_to_state_;
};



FsaVec IntersectDevice(FsaVec &a_fsas, int32_t properties_a,
                     FsaVec &b_fsas, int32_t properties_b,
                     const Array1<int32_t> &b_to_a_map,
                     Array1<int32_t> *arc_map_a,
                     Array1<int32_t> *arc_map_b) {
  NVTX_RANGE("IntersectDevice");
  K2_CHECK_NE(properties_a & kFsaPropertiesValid, 0);
  K2_CHECK_NE(properties_b & kFsaPropertiesValid, 0);
  K2_CHECK_EQ(a_fsas.NumAxes(), 3);
  K2_CHECK_EQ(b_fsas.NumAxes(), 3);
  K2_CHECK_EQ(b_to_a_map.Dim(), b_fsas.Dim0());
  K2_CHECK_LT(static_cast<uint32_t>(MaxValue(b_to_a_map)),
              static_cast<uint32_t>(a_fsas.Dim0()));

  DeviceIntersector intersector(a_fsas, b_fsas, b_to_a_map);
  intersector.Intersect();
  return intersector.FormatOutput(arc_map_a, arc_map_b);
}
}  // namespace k2
