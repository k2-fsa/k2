#include "hip/hip_runtime.h"
/**
 * Copyright      2021  Xiaomi Corporation (authors: Fangjun Kuang)
 *
 * See LICENSE for clarification regarding multiple authors
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gtest/gtest.h"
#include "k2/csrc/log.h"
#include "k2/torch/csrc/test_wave_data.h"
#include "k2/torch/csrc/wave_reader.h"

namespace k2 {

TEST(WaveReader, Mono) {
  std::stringstream ss;
  ss.write(reinterpret_cast<const char *>(kTestWav), sizeof(kTestWav));
  WaveReader reader(ss);
  torch::Tensor expected = torch::arange(16, torch::kShort);
  expected.data_ptr<int16_t>()[0] = 32767;
  expected = (expected / 32768.).to(torch::kFloat32);
  EXPECT_TRUE(reader.Data().allclose(expected, 1e-6));
  EXPECT_EQ(reader.SampleRate(), 16000);
}

}  // namespace k2
